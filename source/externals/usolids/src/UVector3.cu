#include "hip/hip_runtime.h"
//
// ********************************************************************
// * This Software is part of the AIDA Unified Solids Library package *
// * See: https://aidasoft.web.cern.ch/USolids                        *
// ********************************************************************
//
// $Id:$
//
// --------------------------------------------------------------------
//
// UVector3
//
// 19.09.12 Marek Gayer
//          Created from original implementation in CLHEP
// --------------------------------------------------------------------

#include "UVector3.hh"
#include "UUtils.hh"

//______________________________________________________________________________
UVector3::UVector3(double theta, double phi)
{
  // Creates a unit vector based on theta and phi angles
  x_ = std::sin(theta) * std::cos(phi);
  y_ = std::sin(theta) * std::sin(phi);
  z_ = std::cos(theta);
}

//______________________________________________________________________________
double UVector3::Angle(const UVector3& q) const
{
  // return the angle w.r.t. another 3-vector
  double ptot2 = Mag2() * q.Mag2();
  if (ptot2 <= 0)
  {
    return 0.0;
  }
  else
  {
    double arg = Dot(q) / std::sqrt(ptot2);
    if (arg >  1.0) arg =  1.0;
    if (arg < -1.0) arg = -1.0;
    return UUtils::ACos(arg);
  }
}

//______________________________________________________________________________
double UVector3::Mag() const
{
  // return the magnitude (rho in spherical coordinate system)

  return std::sqrt(Mag2());
}

//______________________________________________________________________________
double UVector3::Perp() const
{
  //return the transverse component  (R in cylindrical coordinate system)

  return std::sqrt(Perp2());
}

//______________________________________________________________________________
double UVector3::Phi() const
{
  //return the  azimuth angle. returns phi from -pi to pi
  return x_ == 0.0 && y_ == 0.0 ? 0.0 : UUtils::ATan2(y_, x_);
}

//______________________________________________________________________________
double UVector3::Theta() const
{
  //return the polar angle from 0 to pi
  double mag2 = Mag2();
  if (mag2 == 0.0) return 0.0;
  return UUtils::ACos(z_ / std::sqrt(mag2));
}

//______________________________________________________________________________
UVector3 UVector3::Unit() const
{
  // return unit vector parallel to this.
  double  tot = Mag2();
  UVector3 p(x_, y_, z_);
  return tot > 0.0 ? p *= (1.0 / std::sqrt(tot)) : p;
}

//______________________________________________________________________________
double UVector3::Normalize()
{
  // Normalize to unit. Return normalization factor.
  double  mag = Mag2();
  if (mag == 0.0) return mag;;
  mag = std::sqrt(mag);
  x_ /= mag;
  y_ /= mag;
  z_ /= mag;
  return mag;
}

//______________________________________________________________________________
void UVector3::RotateX(double angle)
{
  //rotate vector around X
  double s = std::sin(angle);
  double c = std::cos(angle);
  double yy = y_;
  y_ = c * yy - s * z_;
  z_ = s * yy + c * z_;
}

//______________________________________________________________________________
void UVector3::RotateY(double angle)
{
  //rotate vector around Y
  double s = std::sin(angle);
  double c = std::cos(angle);
  double zz = z_;
  z_ = c * zz - s * x_;
  x_ = s * zz + c * x_;
}

//______________________________________________________________________________
void UVector3::RotateZ(double angle)
{
  //rotate vector around Z
  double s = std::sin(angle);
  double c = std::cos(angle);
  double xx = x_;
  x_ = c * xx - s * y_;
  y_ = s * xx + c * y_;
}
