#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4Ellipsoid.cc 83572 2014-09-01 15:23:27Z gcosmo $
//
// class G4Ellipsoid
//
// Implementation for G4Ellipsoid class
//
// History:
//
// 10.11.99 G.Horton-Smith  -- first writing, based on G4Sphere class
// 25.02.05 G.Guerrieri -- Modified for future Geant4 release
//
// --------------------------------------------------------------------

#include "globals.hh"

#include "G4Ellipsoid.hh"

#include "G4VoxelLimits.hh"
#include "G4AffineTransform.hh"
#include "G4GeometryTolerance.hh"

#include "meshdefs.hh"
#include "Randomize.hh"

#include "G4VPVParameterisation.hh"

#include "G4VGraphicsScene.hh"
#include "G4VisExtent.hh"

#include "G4AutoLock.hh"

namespace
{
  G4Mutex polyhedronMutex = G4MUTEX_INITIALIZER;
}

using namespace CLHEP;

///////////////////////////////////////////////////////////////////////////////
//
// constructor - check parameters, convert angles so 0<sphi+dpshi<=2_PI
//             - note if pDPhi>2PI then reset to 2PI

G4Ellipsoid::G4Ellipsoid(const G4String& pName,
                               G4double pxSemiAxis,
                               G4double pySemiAxis,
                               G4double pzSemiAxis,
                               G4double pzBottomCut,
                               G4double pzTopCut)
  : G4VSolid(pName), fRebuildPolyhedron(false), fpPolyhedron(0),
    fCubicVolume(0.), fSurfaceArea(0.), zBottomCut(0.), zTopCut(0.)
{
  // note: for users that want to use the full ellipsoid it is useful
  // to include a default for the cuts 

  kRadTolerance = G4GeometryTolerance::GetInstance()->GetRadialTolerance();

  halfCarTolerance = kCarTolerance*0.5;
  halfRadTolerance = kRadTolerance*0.5;

  // Check Semi-Axis
  if ( (pxSemiAxis<=0.) || (pySemiAxis<=0.) || (pzSemiAxis<=0.) )
  {
     std::ostringstream message;
     message << "Invalid semi-axis - " << GetName();
     G4Exception("G4Ellipsoid::G4Ellipsoid()", "GeomSolids0002",
                 FatalErrorInArgument, message);
  }
  SetSemiAxis(pxSemiAxis, pySemiAxis, pzSemiAxis);

  if ( pzBottomCut == 0 && pzTopCut == 0 )
  {
     SetZCuts(-pzSemiAxis, pzSemiAxis);
  }
  else if ( (pzBottomCut < pzSemiAxis) && (pzTopCut > -pzSemiAxis)
         && (pzBottomCut < pzTopCut) )
  {
     SetZCuts(pzBottomCut, pzTopCut);
  }
  else
  {
     std::ostringstream message;
     message << "Invalid z-coordinate for cutting plane - " << GetName();
     G4Exception("G4Ellipsoid::G4Ellipsoid()", "GeomSolids0002",
                 FatalErrorInArgument, message);
  }
}

///////////////////////////////////////////////////////////////////////////////
//
// Fake default constructor - sets only member data and allocates memory
//                            for usage restricted to object persistency.
//
G4Ellipsoid::G4Ellipsoid( __void__& a )
  : G4VSolid(a), fRebuildPolyhedron(false), fpPolyhedron(0), kRadTolerance(0.),
    halfCarTolerance(0.), halfRadTolerance(0.), fCubicVolume(0.),
    fSurfaceArea(0.), xSemiAxis(0.), ySemiAxis(0.), zSemiAxis(0.),
    semiAxisMax(0.), zBottomCut(0.), zTopCut(0.)
{
}

///////////////////////////////////////////////////////////////////////////////
//
// Destructor

G4Ellipsoid::~G4Ellipsoid()
{
  delete fpPolyhedron; fpPolyhedron = 0;
}

///////////////////////////////////////////////////////////////////////////////
//
// Copy constructor

G4Ellipsoid::G4Ellipsoid(const G4Ellipsoid& rhs)
  : G4VSolid(rhs),
    fRebuildPolyhedron(false), fpPolyhedron(0),
    kRadTolerance(rhs.kRadTolerance),
    halfCarTolerance(rhs.halfCarTolerance),
    halfRadTolerance(rhs.halfRadTolerance),
    fCubicVolume(rhs.fCubicVolume), fSurfaceArea(rhs.fSurfaceArea),
    xSemiAxis(rhs.xSemiAxis), ySemiAxis(rhs.ySemiAxis),
    zSemiAxis(rhs.zSemiAxis), semiAxisMax(rhs.semiAxisMax),
    zBottomCut(rhs.zBottomCut), zTopCut(rhs.zTopCut)
{
}

///////////////////////////////////////////////////////////////////////////////
//
// Assignment operator

G4Ellipsoid& G4Ellipsoid::operator = (const G4Ellipsoid& rhs) 
{
   // Check assignment to self
   //
   if (this == &rhs)  { return *this; }

   // Copy base class data
   //
   G4VSolid::operator=(rhs);

   // Copy data
   //
   kRadTolerance = rhs.kRadTolerance;
   halfCarTolerance = rhs.halfCarTolerance;
   halfRadTolerance = rhs.halfRadTolerance;
   fCubicVolume = rhs.fCubicVolume; fSurfaceArea = rhs.fSurfaceArea;
   xSemiAxis = rhs.xSemiAxis; ySemiAxis = rhs.ySemiAxis;
   zSemiAxis = rhs.zSemiAxis; semiAxisMax = rhs.semiAxisMax;
   zBottomCut = rhs.zBottomCut; zTopCut = rhs.zTopCut;
   fRebuildPolyhedron = false;
   delete fpPolyhedron; fpPolyhedron = 0;

   return *this;
}

////////////////////////////////////////////////////////////////////////
//
// Dispatch to parameterisation for replication mechanism dimension
// computation & modification.

void G4Ellipsoid::ComputeDimensions(G4VPVParameterisation* p,
                                    const G4int n,
                                    const G4VPhysicalVolume* pRep)
{
  p->ComputeDimensions(*this,n,pRep);
}

///////////////////////////////////////////////////////////////////////////////
//
// Calculate extent under transform and specified limit

G4bool
G4Ellipsoid::CalculateExtent(const EAxis pAxis,
                             const G4VoxelLimits& pVoxelLimit,
                             const G4AffineTransform& pTransform,
                                   G4double& pMin, G4double& pMax) const
{
  if (!pTransform.IsRotated())
  {
    // Special case handling for unrotated solid ellipsoid
    // Compute x/y/z mins and maxs for bounding box respecting limits,
    // with early returns if outside limits. Then switch() on pAxis,
    // and compute exact x and y limit for x/y case

    G4double xoffset,xMin,xMax;
    G4double yoffset,yMin,yMax;
    G4double zoffset,zMin,zMax;

    G4double maxDiff,newMin,newMax;
    G4double xoff,yoff;

    xoffset=pTransform.NetTranslation().x();
    xMin=xoffset - xSemiAxis;
    xMax=xoffset + xSemiAxis;
    if (pVoxelLimit.IsXLimited())
    {
      if ( (xMin>pVoxelLimit.GetMaxXExtent()+kCarTolerance)
        || (xMax<pVoxelLimit.GetMinXExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (xMin<pVoxelLimit.GetMinXExtent())
        {
          xMin=pVoxelLimit.GetMinXExtent();
        }
        if (xMax>pVoxelLimit.GetMaxXExtent())
        {
          xMax=pVoxelLimit.GetMaxXExtent();
        }
      }
    }

    yoffset=pTransform.NetTranslation().y();
    yMin=yoffset - ySemiAxis;
    yMax=yoffset + ySemiAxis;
    if (pVoxelLimit.IsYLimited())
    {
      if ( (yMin>pVoxelLimit.GetMaxYExtent()+kCarTolerance)
        || (yMax<pVoxelLimit.GetMinYExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (yMin<pVoxelLimit.GetMinYExtent())
        {
          yMin=pVoxelLimit.GetMinYExtent();
        }
        if (yMax>pVoxelLimit.GetMaxYExtent())
        {
          yMax=pVoxelLimit.GetMaxYExtent();
        }
      }
    }

    zoffset=pTransform.NetTranslation().z();
    zMin=zoffset + (-zSemiAxis > zBottomCut ? -zSemiAxis : zBottomCut);
    zMax=zoffset + ( zSemiAxis < zTopCut ? zSemiAxis : zTopCut);
    if (pVoxelLimit.IsZLimited())
    {
      if ( (zMin>pVoxelLimit.GetMaxZExtent()+kCarTolerance)
        || (zMax<pVoxelLimit.GetMinZExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (zMin<pVoxelLimit.GetMinZExtent())
        {
          zMin=pVoxelLimit.GetMinZExtent();
        }
        if (zMax>pVoxelLimit.GetMaxZExtent())
        {
          zMax=pVoxelLimit.GetMaxZExtent();
        }
      }
    }

    // if here, then known to cut bounding box around ellipsoid
    //
    xoff = (xoffset < xMin) ? (xMin-xoffset)
         : (xoffset > xMax) ? (xoffset-xMax) : 0.0;
    yoff = (yoffset < yMin) ? (yMin-yoffset)
         : (yoffset > yMax) ? (yoffset-yMax) : 0.0;

    // detailed calculations
    // NOTE: does not use X or Y offsets to adjust Z range,
    // and does not use Z offset to adjust X or Y range,
    // which is consistent with G4Sphere::CalculateExtent behavior
    //
    switch (pAxis)
    {
      case kXAxis:
        if (yoff==0.)
        {
          // YZ limits cross max/min x => no change
          //
          pMin=xMin;
          pMax=xMax;
        }
        else
        {
          // YZ limits don't cross max/min x => compute max delta x,
          // hence new mins/maxs
          //
          maxDiff= 1.0-sqr(yoff/ySemiAxis);
          if (maxDiff < 0.0) { return false; }
          maxDiff= xSemiAxis * std::sqrt(maxDiff);
          newMin=xoffset-maxDiff;
          newMax=xoffset+maxDiff;
          pMin=(newMin<xMin) ? xMin : newMin;
          pMax=(newMax>xMax) ? xMax : newMax;
        }
        break;
      case kYAxis:
        if (xoff==0.)
        {
          // XZ limits cross max/min y => no change
          //
          pMin=yMin;
          pMax=yMax;
        }
        else
        {
          // XZ limits don't cross max/min y => compute max delta y,
          // hence new mins/maxs
          //
          maxDiff= 1.0-sqr(xoff/xSemiAxis);
          if (maxDiff < 0.0) { return false; }
          maxDiff= ySemiAxis * std::sqrt(maxDiff);
          newMin=yoffset-maxDiff;
          newMax=yoffset+maxDiff;
          pMin=(newMin<yMin) ? yMin : newMin;
          pMax=(newMax>yMax) ? yMax : newMax;
        }
        break;
      case kZAxis:
        pMin=zMin;
        pMax=zMax;
        break;
      default:
        break;
    }
  
    pMin-=kCarTolerance;
    pMax+=kCarTolerance;
    return true;
  }
  else  // not rotated
  {
    G4int i,j,noEntries,noBetweenSections;
    G4bool existsAfterClip=false;

    // Calculate rotated vertex coordinates

    G4int noPolygonVertices=0;
    G4ThreeVectorList* vertices =
      CreateRotatedVertices(pTransform,noPolygonVertices);

    pMin=+kInfinity;
    pMax=-kInfinity;

    noEntries=vertices->size(); // noPolygonVertices*noPhiCrossSections
    noBetweenSections=noEntries-noPolygonVertices;
    
    G4ThreeVectorList ThetaPolygon;
    for (i=0;i<noEntries;i+=noPolygonVertices)
    {
      for(j=0;j<(noPolygonVertices/2)-1;j++)
      {
        ThetaPolygon.push_back((*vertices)[i+j]);  
        ThetaPolygon.push_back((*vertices)[i+j+1]);  
        ThetaPolygon.push_back((*vertices)[i+noPolygonVertices-2-j]);
        ThetaPolygon.push_back((*vertices)[i+noPolygonVertices-1-j]);
        CalculateClippedPolygonExtent(ThetaPolygon,pVoxelLimit,pAxis,pMin,pMax);
        ThetaPolygon.clear();
      }
    }
    for (i=0;i<noBetweenSections;i+=noPolygonVertices)
    {
      for(j=0;j<noPolygonVertices-1;j++)
      {
        ThetaPolygon.push_back((*vertices)[i+j]);  
        ThetaPolygon.push_back((*vertices)[i+j+1]);  
        ThetaPolygon.push_back((*vertices)[i+noPolygonVertices+j+1]);
        ThetaPolygon.push_back((*vertices)[i+noPolygonVertices+j]);
        CalculateClippedPolygonExtent(ThetaPolygon,pVoxelLimit,pAxis,pMin,pMax);
        ThetaPolygon.clear();
      }
      ThetaPolygon.push_back((*vertices)[i+noPolygonVertices-1]);
      ThetaPolygon.push_back((*vertices)[i]);
      ThetaPolygon.push_back((*vertices)[i+noPolygonVertices]);
      ThetaPolygon.push_back((*vertices)[i+2*noPolygonVertices-1]);
      CalculateClippedPolygonExtent(ThetaPolygon,pVoxelLimit,pAxis,pMin,pMax);
      ThetaPolygon.clear();
    }
    if ( (pMin!=kInfinity) || (pMax!=-kInfinity) )
    {
      existsAfterClip=true;
    
      // Add 2*tolerance to avoid precision troubles
      //
      pMin-=kCarTolerance;
      pMax+=kCarTolerance;

    }
    else
    {
      // Check for case where completely enveloping clipping volume
      // If point inside then we are confident that the solid completely
      // envelopes the clipping volume. Hence set min/max extents according
      // to clipping volume extents along the specified axis.
      //
      G4ThreeVector
      clipCentre((pVoxelLimit.GetMinXExtent()+pVoxelLimit.GetMaxXExtent())*0.5,
                 (pVoxelLimit.GetMinYExtent()+pVoxelLimit.GetMaxYExtent())*0.5,
                 (pVoxelLimit.GetMinZExtent()+pVoxelLimit.GetMaxZExtent())*0.5);

      if (Inside(pTransform.Inverse().TransformPoint(clipCentre))!=kOutside)
      {
        existsAfterClip=true;
        pMin=pVoxelLimit.GetMinExtent(pAxis);
        pMax=pVoxelLimit.GetMaxExtent(pAxis);
      }
    }
    delete vertices;
    return existsAfterClip;
  }
}

///////////////////////////////////////////////////////////////////////////////
//
// Return whether point inside/outside/on surface
// Split into radius, phi, theta checks
// Each check modifies `in', or returns as approprate

EInside G4Ellipsoid::Inside(const G4ThreeVector& p) const
{
  G4double rad2oo,  // outside surface outer tolerance
           rad2oi;  // outside surface inner tolerance
  EInside in;

  // check this side of z cut first, because that's fast
  //
  if (p.z() < zBottomCut-halfRadTolerance) { return in=kOutside; }
  if (p.z() > zTopCut+halfRadTolerance)    { return in=kOutside; }

  rad2oo= sqr(p.x()/(xSemiAxis+halfRadTolerance))
        + sqr(p.y()/(ySemiAxis+halfRadTolerance))
        + sqr(p.z()/(zSemiAxis+halfRadTolerance));

  if (rad2oo > 1.0)  { return in=kOutside; }
    
  rad2oi= sqr(p.x()*(1.0+halfRadTolerance/xSemiAxis)/xSemiAxis)
      + sqr(p.y()*(1.0+halfRadTolerance/ySemiAxis)/ySemiAxis)
      + sqr(p.z()*(1.0+halfRadTolerance/zSemiAxis)/zSemiAxis);

  // Check radial surfaces
  //  sets `in' (already checked for rad2oo > 1.0)
  //
  if (rad2oi < 1.0)
  {
    in = ( (p.z() < zBottomCut+halfRadTolerance)
        || (p.z() > zTopCut-halfRadTolerance) ) ? kSurface : kInside;
    if ( rad2oi > 1.0-halfRadTolerance )  { in=kSurface; }
  }
  else 
  {
    in = kSurface;
  }
  return in;

}

///////////////////////////////////////////////////////////////////////////////
//
// Return unit normal of surface closest to p not protected against p=0

G4ThreeVector G4Ellipsoid::SurfaceNormal( const G4ThreeVector& p) const
{
  G4double distR, distZBottom, distZTop;

  // normal vector with special magnitude:  parallel to normal, units 1/length
  // norm*p == 1.0 if on surface, >1.0 if outside, <1.0 if inside
  //
  G4ThreeVector norm(p.x()/(xSemiAxis*xSemiAxis),
                     p.y()/(ySemiAxis*ySemiAxis),
                     p.z()/(zSemiAxis*zSemiAxis));
  G4double radius = 1.0/norm.mag();

  // approximate distance to curved surface
  //
  distR = std::fabs( (p*norm - 1.0) * radius ) / 2.0;

  // Distance to z-cut plane
  //
  distZBottom = std::fabs( p.z() - zBottomCut );
  distZTop = std::fabs( p.z() - zTopCut );

  if ( (distZBottom < distR) || (distZTop < distR) )
  {
    return G4ThreeVector(0.,0.,(distZBottom < distZTop) ? -1.0 : 1.0);
  }
  return ( norm *= radius );
}

///////////////////////////////////////////////////////////////////////////////
//
// Calculate distance to shape from outside, along normalised vector
// - return kInfinity if no intersection, or intersection distance <= tolerance
//

G4double G4Ellipsoid::DistanceToIn( const G4ThreeVector& p,
                                    const G4ThreeVector& v  ) const
{
  G4double distMin = std::min(xSemiAxis,ySemiAxis);
  const G4double dRmax = 100.*std::min(distMin,zSemiAxis);
  distMin= kInfinity;

  // check to see if Z plane is relevant
  if (p.z() <= zBottomCut+halfCarTolerance)
  {
    if (v.z() <= 0.0) { return distMin; }
    G4double distZ = (zBottomCut - p.z()) / v.z();

    if ( (distZ > -halfRadTolerance) && (Inside(p+distZ*v) != kOutside) )
    {
      // early exit since can't intercept curved surface if we reach here
      if ( std::fabs(distZ) < halfRadTolerance ) { distZ=0.; }
      return distMin= distZ;
    }
  }
  if (p.z() >= zTopCut-halfCarTolerance)
  {
    if (v.z() >= 0.0) { return distMin;}
    G4double distZ = (zTopCut - p.z()) / v.z();
    if ( (distZ > -halfRadTolerance) && (Inside(p+distZ*v) != kOutside) )
    {
      // early exit since can't intercept curved surface if we reach here
      if ( std::fabs(distZ) < halfRadTolerance ) { distZ=0.; }
      return distMin= distZ;
    }
  }
  // if fZCut1 <= p.z() <= fZCut2, then must hit curved surface

  // now check curved surface intercept
  G4double A,B,C;

  A= sqr(v.x()/xSemiAxis) + sqr(v.y()/ySemiAxis) + sqr(v.z()/zSemiAxis);
  C= sqr(p.x()/xSemiAxis) + sqr(p.y()/ySemiAxis) + sqr(p.z()/zSemiAxis) - 1.0;
  B= 2.0 * ( p.x()*v.x()/(xSemiAxis*xSemiAxis)
           + p.y()*v.y()/(ySemiAxis*ySemiAxis)
           + p.z()*v.z()/(zSemiAxis*zSemiAxis) );

  C= B*B - 4.0*A*C;
  if (C > 0.0)
  {    
    G4double distR= (-B - std::sqrt(C)) / (2.0*A);
    G4double intZ = p.z()+distR*v.z();
    if ( (distR > halfRadTolerance)
      && (intZ >= zBottomCut-halfRadTolerance)
      && (intZ <= zTopCut+halfRadTolerance) )
    { 
      distMin = distR;
    }
    else if( (distR >- halfRadTolerance)
	    && (intZ >= zBottomCut-halfRadTolerance)
	    && (intZ <= zTopCut+halfRadTolerance) )
    {
      // p is on the curved surface, DistanceToIn returns 0 or kInfinity:
      // DistanceToIn returns 0, if second root is positive (means going inside)
      // If second root is negative, DistanceToIn returns kInfinity (outside)
      //
      distR = (-B + std::sqrt(C) ) / (2.0*A);
      if(distR>0.) { distMin=0.; }
    }
    else
    {
      distR= (-B + std::sqrt(C)) / (2.0*A);
      intZ = p.z()+distR*v.z();
      if ( (distR > halfRadTolerance)
        && (intZ >= zBottomCut-halfRadTolerance)
        && (intZ <= zTopCut+halfRadTolerance) )
      {
        G4ThreeVector norm=SurfaceNormal(p);
        if (norm.dot(v)<0.) { distMin = distR; }
      }
    }
    if ( (distMin!=kInfinity) && (distMin>dRmax) ) 
    {                    // Avoid rounding errors due to precision issues on
                         // 64 bits systems. Split long distances and recompute
      G4double fTerm = distMin-std::fmod(distMin,dRmax);
      distMin = fTerm + DistanceToIn(p+fTerm*v,v);
    }
  }
  
  if (std::fabs(distMin)<halfRadTolerance) { distMin=0.; }
  return distMin;
} 

///////////////////////////////////////////////////////////////////////////////
//
// Calculate distance (<= actual) to closest surface of shape from outside
// - Return 0 if point inside

G4double G4Ellipsoid::DistanceToIn(const G4ThreeVector& p) const
{
  G4double distR, distZ;

  // normal vector:  parallel to normal, magnitude 1/(characteristic radius)
  //
  G4ThreeVector norm(p.x()/(xSemiAxis*xSemiAxis),
                     p.y()/(ySemiAxis*ySemiAxis),
                     p.z()/(zSemiAxis*zSemiAxis));
  G4double radius= 1.0/norm.mag();

  // approximate distance to curved surface ( <= actual distance )
  //
  distR= (p*norm - 1.0) * radius / 2.0;

  // Distance to z-cut plane
  //
  distZ= zBottomCut - p.z();
  if (distZ < 0.0)
  {
    distZ = p.z() - zTopCut;
  }

  // Distance to closest surface from outside
  //
  if (distZ < 0.0)
  {
    return (distR < 0.0) ? 0.0 : distR;
  }
  else if (distR < 0.0)
  {
    return distZ;
  }
  else
  {
    return (distZ < distR) ? distZ : distR;
  }
}

///////////////////////////////////////////////////////////////////////////////
//
// Calculate distance to surface of shape from `inside', allowing for tolerance

G4double G4Ellipsoid::DistanceToOut(const G4ThreeVector& p,
                                    const G4ThreeVector& v,
                                    const G4bool calcNorm,
                                          G4bool *validNorm,
                                          G4ThreeVector *n  ) const
{
  G4double distMin;
  enum surface_e {kPlaneSurf, kCurvedSurf, kNoSurf} surface;
  
  distMin= kInfinity;
  surface= kNoSurf;

  // check to see if Z plane is relevant
  //
  if (v.z() < 0.0)
  {
    G4double distZ = (zBottomCut - p.z()) / v.z();
    if (distZ < 0.0)
    {
      distZ= 0.0;
      if (!calcNorm) {return 0.0;}
    }
    distMin= distZ;
    surface= kPlaneSurf;
  }
  if (v.z() > 0.0)
  {
    G4double distZ = (zTopCut - p.z()) / v.z();
    if (distZ < 0.0)
    {
      distZ= 0.0;
      if (!calcNorm) {return 0.0;}
    }
    distMin= distZ;
    surface= kPlaneSurf;
  }

  // normal vector:  parallel to normal, magnitude 1/(characteristic radius)
  //
  G4ThreeVector nearnorm(p.x()/(xSemiAxis*xSemiAxis),
                         p.y()/(ySemiAxis*ySemiAxis),
                         p.z()/(zSemiAxis*zSemiAxis));
  
  // now check curved surface intercept
  //
  G4double A,B,C;
  
  A= sqr(v.x()/xSemiAxis) + sqr(v.y()/ySemiAxis) + sqr(v.z()/zSemiAxis);
  C= (p * nearnorm) - 1.0;
  B= 2.0 * (v * nearnorm);

  C= B*B - 4.0*A*C;
  if (C > 0.0)
  {
    G4double distR= (-B + std::sqrt(C) ) / (2.0*A);
    if (distR < 0.0)
    {
      distR= 0.0;
      if (!calcNorm) {return 0.0;}
    }
    if (distR < distMin)
    {
      distMin= distR;
      surface= kCurvedSurf;
    }
  }

  // set normal if requested
  //
  if (calcNorm)
  {
    if (surface == kNoSurf)
    {
      *validNorm = false;
    }
    else
    {
      *validNorm = true;
      switch (surface)
      {
        case kPlaneSurf:
          *n= G4ThreeVector(0.,0.,(v.z() > 0.0 ? 1. : -1.));
          break;
        case kCurvedSurf:
        {
          G4ThreeVector pexit= p + distMin*v;
          G4ThreeVector truenorm(pexit.x()/(xSemiAxis*xSemiAxis),
                                 pexit.y()/(ySemiAxis*ySemiAxis),
                                 pexit.z()/(zSemiAxis*zSemiAxis));
          truenorm *= 1.0/truenorm.mag();
          *n= truenorm;
        } break;
        default:           // Should never reach this case ...
          DumpInfo();
          std::ostringstream message;
          G4int oldprc = message.precision(16);
          message << "Undefined side for valid surface normal to solid."
                  << G4endl
                  << "Position:"  << G4endl
                  << "   p.x() = "   << p.x()/mm << " mm" << G4endl
                  << "   p.y() = "   << p.y()/mm << " mm" << G4endl
                  << "   p.z() = "   << p.z()/mm << " mm" << G4endl
                  << "Direction:" << G4endl << G4endl
                  << "   v.x() = "   << v.x() << G4endl
                  << "   v.y() = "   << v.y() << G4endl
                  << "   v.z() = "   << v.z() << G4endl
                  << "Proposed distance :" << G4endl
                  << "   distMin = "    << distMin/mm << " mm";
          message.precision(oldprc);
          G4Exception("G4Ellipsoid::DistanceToOut(p,v,..)",
                      "GeomSolids1002", JustWarning, message);
          break;
      }
    }
  }
   
  return distMin;
}

///////////////////////////////////////////////////////////////////////////////
//
// Calculate distance (<=actual) to closest surface of shape from inside

G4double G4Ellipsoid::DistanceToOut(const G4ThreeVector& p) const
{
  G4double distR, distZ;

#ifdef G4SPECSDEBUG
  if( Inside(p) == kOutside )
  {
     DumpInfo();
     std::ostringstream message;
     G4int oldprc = message.precision(16);
     message << "Point p is outside !?" << G4endl
             << "Position:"  << G4endl
             << "   p.x() = "   << p.x()/mm << " mm" << G4endl
             << "   p.y() = "   << p.y()/mm << " mm" << G4endl
             << "   p.z() = "   << p.z()/mm << " mm";
     message.precision(oldprc) ;
     G4Exception("G4Ellipsoid::DistanceToOut(p)", "GeomSolids1002",
                 JustWarning, message);
  }
#endif

  // Normal vector:  parallel to normal, magnitude 1/(characteristic radius)
  //
  G4ThreeVector norm(p.x()/(xSemiAxis*xSemiAxis),
                     p.y()/(ySemiAxis*ySemiAxis),
                     p.z()/(zSemiAxis*zSemiAxis));

  // the following is a safe inlined "radius= min(1.0/norm.mag(),p.mag())
  //
  G4double radius= p.mag();
  G4double tmp= norm.mag();
  if ( (tmp > 0.0) && (1.0 < radius*tmp) ) {radius = 1.0/tmp;}

  // Approximate distance to curved surface ( <= actual distance )
  //
  distR = (1.0 - p*norm) * radius / 2.0;
    
  // Distance to z-cut plane
  //
  distZ = p.z() - zBottomCut;
  if (distZ < 0.0) {distZ= zTopCut - p.z();}

  // Distance to closest surface from inside
  //
  if ( (distZ < 0.0) || (distR < 0.0) )
  {
    return 0.0;
  }
  else
  {
    return (distZ < distR) ? distZ : distR;
  }
}

///////////////////////////////////////////////////////////////////////////////
//
// Create a List containing the transformed vertices
// Ordering [0-3] -fDz cross section
//          [4-7] +fDz cross section such that [0] is below [4],
//                                             [1] below [5] etc.
// Note:
//  Caller has deletion resposibility
//  Potential improvement: For last slice, use actual ending angle
//                         to avoid rounding error problems.

G4ThreeVectorList*
G4Ellipsoid::CreateRotatedVertices(const G4AffineTransform& pTransform,
                                         G4int& noPolygonVertices) const
{
  G4ThreeVectorList *vertices;
  G4ThreeVector vertex;
  G4double meshAnglePhi, meshRMaxFactor,
           crossAnglePhi, coscrossAnglePhi, sincrossAnglePhi, sAnglePhi;
  G4double meshTheta, crossTheta, startTheta;
  G4double rMaxX, rMaxY, rMaxZ, rMaxMax, rx, ry, rz;
  G4int crossSectionPhi, noPhiCrossSections, crossSectionTheta, noThetaSections;

  // Phi cross sections
  //
  noPhiCrossSections=G4int (twopi/kMeshAngleDefault)+1;  // = 9!
    
/*
  if (noPhiCrossSections<kMinMeshSections)        // <3
  {
    noPhiCrossSections=kMinMeshSections;
  }
  else if (noPhiCrossSections>kMaxMeshSections)   // >37
  {
    noPhiCrossSections=kMaxMeshSections;
  }
*/
  meshAnglePhi=twopi/(noPhiCrossSections-1);
    
  // Set start angle such that mesh will be at fRMax
  // on the x axis. Will give better extent calculations when not rotated.
    
  sAnglePhi = -meshAnglePhi*0.5;

  // Theta cross sections
    
  noThetaSections = G4int(pi/kMeshAngleDefault)+3;  //  = 7!

/*
  if (noThetaSections<kMinMeshSections)       // <3
  {
    noThetaSections=kMinMeshSections;
  }
  else if (noThetaSections>kMaxMeshSections)  // >37
  {
    noThetaSections=kMaxMeshSections;
  }
*/
  meshTheta= pi/(noThetaSections-2);
    
  // Set start angle such that mesh will be at fRMax
  // on the z axis. Will give better extent calculations when not rotated.
    
  startTheta = -meshTheta*0.5;

  meshRMaxFactor =  1.0/std::cos(0.5*
                    std::sqrt(meshAnglePhi*meshAnglePhi+meshTheta*meshTheta));
  rMaxMax= (xSemiAxis > ySemiAxis ? xSemiAxis : ySemiAxis);
  if (zSemiAxis > rMaxMax) rMaxMax= zSemiAxis;
  rMaxX= xSemiAxis + rMaxMax*(meshRMaxFactor-1.0);
  rMaxY= ySemiAxis + rMaxMax*(meshRMaxFactor-1.0);
  rMaxZ= zSemiAxis + rMaxMax*(meshRMaxFactor-1.0);
  G4double* cosCrossTheta = new G4double[noThetaSections];
  G4double* sinCrossTheta = new G4double[noThetaSections];    
  vertices=new G4ThreeVectorList(noPhiCrossSections*noThetaSections);
  if (vertices && cosCrossTheta && sinCrossTheta)
  {
    for (crossSectionTheta=0; crossSectionTheta<noThetaSections;
         crossSectionTheta++)
    {
      // Compute sine and cosine table (for historical reasons)
      //
      crossTheta=startTheta+crossSectionTheta*meshTheta;
      cosCrossTheta[crossSectionTheta]=std::cos(crossTheta);
      sinCrossTheta[crossSectionTheta]=std::sin(crossTheta);
    }
    for (crossSectionPhi=0; crossSectionPhi<noPhiCrossSections;
         crossSectionPhi++)
    {
      crossAnglePhi=sAnglePhi+crossSectionPhi*meshAnglePhi;
      coscrossAnglePhi=std::cos(crossAnglePhi);
      sincrossAnglePhi=std::sin(crossAnglePhi);
      for (crossSectionTheta=0; crossSectionTheta<noThetaSections;
           crossSectionTheta++)
      {
        // Compute coordinates of cross section at section crossSectionPhi
        //
        rx= sinCrossTheta[crossSectionTheta]*coscrossAnglePhi*rMaxX;
        ry= sinCrossTheta[crossSectionTheta]*sincrossAnglePhi*rMaxY;
        rz= cosCrossTheta[crossSectionTheta]*rMaxZ;
        if (rz < zBottomCut)
          { rz= zBottomCut; }
        if (rz > zTopCut)
          { rz= zTopCut; }
        vertex= G4ThreeVector(rx,ry,rz);
        vertices->push_back(pTransform.TransformPoint(vertex));
      }    // Theta forward     
    }    // Phi
    noPolygonVertices = noThetaSections ;
  }
  else
  {
    DumpInfo();
    G4Exception("G4Ellipsoid::CreateRotatedVertices()",
                "GeomSolids0003", FatalException,
                "Error in allocation of vertices. Out of memory !");
  }

  delete[] cosCrossTheta;
  delete[] sinCrossTheta;

  return vertices;
}

//////////////////////////////////////////////////////////////////////////
//
// G4EntityType

G4GeometryType G4Ellipsoid::GetEntityType() const
{
  return G4String("G4Ellipsoid");
}

//////////////////////////////////////////////////////////////////////////
//
// Make a clone of the object

G4VSolid* G4Ellipsoid::Clone() const
{
  return new G4Ellipsoid(*this);
}

//////////////////////////////////////////////////////////////////////////
//
// Stream object contents to an output stream

std::ostream& G4Ellipsoid::StreamInfo( std::ostream& os ) const
{
  G4int oldprc = os.precision(16);
  os << "-----------------------------------------------------------\n"
     << "    *** Dump for solid - " << GetName() << " ***\n"
     << "    ===================================================\n"
     << " Solid type: G4Ellipsoid\n"
     << " Parameters: \n"

     << "    semi-axis x: " << xSemiAxis/mm << " mm \n"
     << "    semi-axis y: " << ySemiAxis/mm << " mm \n"
     << "    semi-axis z: " << zSemiAxis/mm << " mm \n"
     << "    max semi-axis: " << semiAxisMax/mm << " mm \n"
     << "    lower cut plane level z: " << zBottomCut/mm << " mm \n"
     << "    upper cut plane level z: " << zTopCut/mm << " mm \n"
     << "-----------------------------------------------------------\n";
  os.precision(oldprc);

  return os;
}

////////////////////////////////////////////////////////////////////
//
// GetPointOnSurface

G4ThreeVector G4Ellipsoid::GetPointOnSurface() const
{
  G4double aTop, aBottom, aCurved, chose, xRand, yRand, zRand, phi;
  G4double cosphi, sinphi, costheta, sintheta, alpha, beta, max1, max2, max3;

  max1  = xSemiAxis > ySemiAxis ? xSemiAxis : ySemiAxis;
  max1  = max1 > zSemiAxis ? max1 : zSemiAxis;
  if (max1 == xSemiAxis)      { max2 = ySemiAxis; max3 = zSemiAxis; }
  else if (max1 == ySemiAxis) { max2 = xSemiAxis; max3 = zSemiAxis; }
  else                        { max2 = xSemiAxis; max3 = ySemiAxis; }

  phi   = RandFlat::shoot(0.,twopi);
  
  cosphi = std::cos(phi);   sinphi = std::sin(phi);
  costheta = RandFlat::shoot(zBottomCut,zTopCut)/zSemiAxis;
  sintheta = std::sqrt(1.-sqr(costheta));
  
  alpha = 1.-sqr(max2/max1); beta  = 1.-sqr(max3/max1);
  
  aTop    = pi*xSemiAxis*ySemiAxis*(1 - sqr(zTopCut/zSemiAxis));
  aBottom = pi*xSemiAxis*ySemiAxis*(1 - sqr(zBottomCut/zSemiAxis));
  
  // approximation
  // from:" http://www.citr.auckland.ac.nz/techreports/2004/CITR-TR-139.pdf"
  aCurved = 4.*pi*max1*max2*(1.-1./6.*(alpha+beta)-
                            1./120.*(3.*sqr(alpha)+2.*alpha*beta+3.*sqr(beta)));

  aCurved *= 0.5*(1.2*zTopCut/zSemiAxis - 1.2*zBottomCut/zSemiAxis);
  
  if( ( zTopCut >= zSemiAxis && zBottomCut <= -1.*zSemiAxis )
   || ( zTopCut == 0 && zBottomCut ==0 ) )
  {
    aTop = 0; aBottom = 0;
  }
  
  chose = RandFlat::shoot(0.,aTop + aBottom + aCurved); 
  
  if(chose < aCurved)
  { 
    xRand = xSemiAxis*sintheta*cosphi;
    yRand = ySemiAxis*sintheta*sinphi;
    zRand = zSemiAxis*costheta;
    return G4ThreeVector (xRand,yRand,zRand); 
  }
  else if(chose >= aCurved && chose < aCurved + aTop)
  {
    xRand = RandFlat::shoot(-1.,1.)*xSemiAxis
          * std::sqrt(1-sqr(zTopCut/zSemiAxis));
    yRand = RandFlat::shoot(-1.,1.)*ySemiAxis
          * std::sqrt(1.-sqr(zTopCut/zSemiAxis)-sqr(xRand/xSemiAxis));
    zRand = zTopCut;
    return G4ThreeVector (xRand,yRand,zRand);
  }
  else
  {
    xRand = RandFlat::shoot(-1.,1.)*xSemiAxis
          * std::sqrt(1-sqr(zBottomCut/zSemiAxis));
    yRand = RandFlat::shoot(-1.,1.)*ySemiAxis
          * std::sqrt(1.-sqr(zBottomCut/zSemiAxis)-sqr(xRand/xSemiAxis)); 
    zRand = zBottomCut;
    return G4ThreeVector (xRand,yRand,zRand);
  }
}

/////////////////////////////////////////////////////////////////////////////
//
// Methods for visualisation

void G4Ellipsoid::DescribeYourselfTo (G4VGraphicsScene& scene) const
{
  scene.AddSolid(*this);
}

G4VisExtent G4Ellipsoid::GetExtent() const
{
  // Define the sides of the box into which the G4Ellipsoid instance would fit.
  //
  return G4VisExtent (-semiAxisMax, semiAxisMax,
                      -semiAxisMax, semiAxisMax,
                      -semiAxisMax, semiAxisMax);
}

G4Polyhedron* G4Ellipsoid::CreatePolyhedron () const
{
  return new G4PolyhedronEllipsoid(xSemiAxis, ySemiAxis, zSemiAxis,
                                   zBottomCut, zTopCut);
}

G4Polyhedron* G4Ellipsoid::GetPolyhedron () const
{
  if (!fpPolyhedron ||
      fRebuildPolyhedron ||
      fpPolyhedron->GetNumberOfRotationStepsAtTimeOfCreation() !=
      fpPolyhedron->GetNumberOfRotationSteps())
    {
      G4AutoLock l(&polyhedronMutex);
      delete fpPolyhedron;
      fpPolyhedron = CreatePolyhedron();
      fRebuildPolyhedron = false;
      l.unlock();
    }
  return fpPolyhedron;
}
