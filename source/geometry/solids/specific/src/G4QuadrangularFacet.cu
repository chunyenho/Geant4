#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration and of QinetiQ Ltd,   *
// * subject to DEFCON 705 IPR conditions.                            *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4QuadrangularFacet.cc 66819 2013-01-12 16:20:10Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// CHANGE HISTORY
// --------------
//
// 31 October 2004, P R Truscott, QinetiQ Ltd, UK - Created.
// 12 October 2012, M Gayer, CERN
//                  New implementation reducing memory requirements by 50%,
//                  and considerable CPU speedup together with the new
//                  implementation of G4TessellatedSolid.
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

#include "G4QuadrangularFacet.hh"
#include "geomdefs.hh"
#include "Randomize.hh"
 
using namespace std;

///////////////////////////////////////////////////////////////////////////////
//
// !!!THIS IS A FUDGE!!!  IT'S TWO ADJACENT G4TRIANGULARFACETS
// --- NOT EFFICIENT BUT PRACTICAL.
//
G4QuadrangularFacet::G4QuadrangularFacet (const G4ThreeVector &vt0,
                                          const G4ThreeVector &vt1,
                                          const G4ThreeVector &vt2,
                                          const G4ThreeVector &vt3,
                                                G4FacetVertexType vertexType)
{
  G4ThreeVector e1, e2, e3;

  SetVertex(0, vt0);
  if (vertexType == ABSOLUTE)
  {
    SetVertex(1, vt1);
    SetVertex(2, vt2);
    SetVertex(3, vt3);

    e1 = vt1 - vt0;
    e2 = vt2 - vt0;
    e3 = vt3 - vt0;
  }
  else
  {
    SetVertex(1, vt0 + vt1);
    SetVertex(2, vt0 + vt2);
    SetVertex(3, vt0 + vt3);

    e1 = vt1;
    e2 = vt2;
    e3 = vt3;
  }
  G4double length1 = e1.mag();
  G4double length2 = (GetVertex(2)-GetVertex(1)).mag();
  G4double length3 = (GetVertex(3)-GetVertex(2)).mag();
  G4double length4 = e3.mag();

  G4ThreeVector normal1 = e1.cross(e2).unit();
  G4ThreeVector normal2 = e2.cross(e3).unit(); 

  bool isDefined = (length1 > kCarTolerance && length2 > kCarTolerance &&
    length3 > kCarTolerance && length4 > kCarTolerance &&
    normal1.dot(normal2) >= 0.9999999999);

  if (isDefined)
  {
    fFacet1 = G4TriangularFacet (GetVertex(0),GetVertex(1),
                                 GetVertex(2),ABSOLUTE);
    fFacet2 = G4TriangularFacet (GetVertex(0),GetVertex(2),
                                 GetVertex(3),ABSOLUTE);

    G4TriangularFacet facet3 (GetVertex(0),GetVertex(1),GetVertex(3),ABSOLUTE);
    G4TriangularFacet facet4 (GetVertex(1),GetVertex(2),GetVertex(3),ABSOLUTE);

    G4ThreeVector normal12 = fFacet1.GetSurfaceNormal()
                           + fFacet2.GetSurfaceNormal();
    G4ThreeVector normal34 = facet3.GetSurfaceNormal()
                           + facet4.GetSurfaceNormal();
    G4ThreeVector normal = 0.25 * (normal12 + normal34);

    fFacet1.SetSurfaceNormal (normal);
    fFacet2.SetSurfaceNormal (normal);

    G4ThreeVector vtmp = 0.5 * (e1 + e2);
    fCircumcentre = GetVertex(0) + vtmp;
    G4double radiusSqr = vtmp.mag2();
    fRadius = std::sqrt(radiusSqr);
  }
  else
  {
    G4Exception("G4QuadrangularFacet::G4QuadrangularFacet()",
                "GeomSolids0002", JustWarning,
                "Length of sides of facet are too small or sides not planar.");
    G4cout << G4endl;
    G4cout << "P0 = " << GetVertex(0) << G4endl;
    G4cout << "P1 = " << GetVertex(1) << G4endl;
    G4cout << "P2 = " << GetVertex(2) << G4endl;
    G4cout << "P3 = " << GetVertex(3) << G4endl;
    G4cout << "Side lengths = P0->P1" << length1 << G4endl;    
    G4cout << "Side lengths = P1->P2" << length2 << G4endl;    
    G4cout << "Side lengths = P2->P3" << length3 << G4endl;    
    G4cout << "Side lengths = P3->P0" << length4 << G4endl;    
    G4cout << G4endl;
    fRadius = 0.0;
  }
}

///////////////////////////////////////////////////////////////////////////////
//
G4QuadrangularFacet::~G4QuadrangularFacet ()
{
}

///////////////////////////////////////////////////////////////////////////////
//
G4QuadrangularFacet::G4QuadrangularFacet (const G4QuadrangularFacet &rhs)
  : G4VFacet(rhs)
{
  fFacet1 = rhs.fFacet1;
  fFacet2 = rhs.fFacet2;
  fRadius = 0.0;
}

///////////////////////////////////////////////////////////////////////////////
//
G4QuadrangularFacet &
G4QuadrangularFacet::operator=(const G4QuadrangularFacet &rhs)
{
  if (this == &rhs)
    return *this;

  fFacet1 = rhs.fFacet1;
  fFacet2 = rhs.fFacet2;
  fRadius = 0.0;

  return *this;
}

///////////////////////////////////////////////////////////////////////////////
//
G4VFacet *G4QuadrangularFacet::GetClone ()
{
  G4QuadrangularFacet *c = new G4QuadrangularFacet (GetVertex(0), GetVertex(1),
                                                    GetVertex(2), GetVertex(3),
                                                    ABSOLUTE);
  return c;
}

///////////////////////////////////////////////////////////////////////////////
//
G4ThreeVector G4QuadrangularFacet::Distance (const G4ThreeVector &p)
{
  G4ThreeVector v1 = fFacet1.Distance(p);
  G4ThreeVector v2 = fFacet2.Distance(p);

  if (v1.mag2() < v2.mag2()) return v1;
  else return v2;
}

///////////////////////////////////////////////////////////////////////////////
//
G4double G4QuadrangularFacet::Distance (const G4ThreeVector &p,
                                              G4double)
{  
  G4double dist = Distance(p).mag();
  return dist;
}

///////////////////////////////////////////////////////////////////////////////
//
G4double G4QuadrangularFacet::Distance (const G4ThreeVector &p, G4double,
                                        const G4bool outgoing)
{
  G4double dist;

  G4ThreeVector v = Distance(p);
  G4double dir = v.dot(GetSurfaceNormal());
  if ( ((dir > dirTolerance) && (!outgoing))
    || ((dir < -dirTolerance) && outgoing))
    dist = kInfinity;
  else 
    dist = v.mag();
  return dist;
}

///////////////////////////////////////////////////////////////////////////////
//
G4double G4QuadrangularFacet::Extent (const G4ThreeVector axis)
{
  G4double ss  = 0;

  for (G4int i = 0; i <= 3; ++i)
  {
    G4double sp = GetVertex(i).dot(axis);
    if (sp > ss) ss = sp;
  }
  return ss;
}

///////////////////////////////////////////////////////////////////////////////
//
G4bool G4QuadrangularFacet::Intersect (const G4ThreeVector &p,
                                       const G4ThreeVector &v,
                                             G4bool outgoing,
                                             G4double &distance,
                                             G4double &distFromSurface,
                                             G4ThreeVector &normal)
{
  G4bool intersect =
    fFacet1.Intersect(p,v,outgoing,distance,distFromSurface,normal);
  if (!intersect) intersect =
    fFacet2.Intersect(p,v,outgoing,distance,distFromSurface,normal);
  if (!intersect)
  {
    distance = distFromSurface = kInfinity;
    normal.set(0,0,0);
  }
  return intersect;
}

///////////////////////////////////////////////////////////////////////////////
//
// Auxiliary method to get a random point on surface
//
G4ThreeVector G4QuadrangularFacet::GetPointOnFace() const
{
  G4ThreeVector pr = (G4RandFlat::shoot(0.,1.) < 0.5)
                   ? fFacet1.GetPointOnFace() : fFacet2.GetPointOnFace();
  return pr;
}

///////////////////////////////////////////////////////////////////////////////
//
// Auxiliary method for returning the surface area
//
G4double G4QuadrangularFacet::GetArea()
{
  G4double area = fFacet1.GetArea() + fFacet2.GetArea();
  return area;
}

///////////////////////////////////////////////////////////////////////////////
//
G4String G4QuadrangularFacet::GetEntityType () const
{
  return "G4QuadrangularFacet";
}

///////////////////////////////////////////////////////////////////////////////
//
G4ThreeVector G4QuadrangularFacet::GetSurfaceNormal () const
{
  return fFacet1.GetSurfaceNormal();
}
