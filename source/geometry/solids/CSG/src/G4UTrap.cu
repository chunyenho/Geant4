#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id:$
//
// 
// Implementation for G4UTrap wrapper class
// --------------------------------------------------------------------

#include "G4Trap.hh"
#include "G4UTrap.hh"
#include "G4VPVParameterisation.hh"
#include "G4Polyhedron.hh"

/////////////////////////////////////////////////////////////////////////
//
// Constructors
//
G4UTrap::G4UTrap( const G4String& pName,
                        G4double pdz,
                        G4double pTheta, G4double pPhi,
                        G4double pdy1, G4double pdx1, G4double pdx2,
                        G4double pAlp1,
                        G4double pdy2, G4double pdx3, G4double pdx4,
                        G4double pAlp2 )
  : G4USolid(pName, new UTrap(pName, pdz, pTheta, pPhi,
                              pdy1, pdx1, pdx2, pAlp1, pdy2, pdx3, pdx4, pAlp2))
{
}

G4UTrap::G4UTrap( const G4String& pName,
                  const G4ThreeVector pt[8] )
  : G4USolid(pName, new UTrap(pName))
{
  SetPlanes(pt);
}

G4UTrap::G4UTrap( const G4String& pName,
                        G4double pZ,
                        G4double pY,
                        G4double pX, G4double pLTX )
  : G4USolid(pName, new UTrap(pName, pZ, pY, pX, pLTX))
{
}

G4UTrap::G4UTrap( const G4String& pName,
                        G4double pdx1,  G4double pdx2,
                        G4double pdy1,  G4double pdy2,
                        G4double pdz )
  : G4USolid(pName, new UTrap(pName, pdx1, pdx2, pdy1, pdy2, pdz))
{
}

G4UTrap::G4UTrap(const G4String& pName,
                       G4double pdx, G4double pdy, G4double pdz,
                       G4double pAlpha, G4double pTheta, G4double pPhi )
  : G4USolid(pName, new UTrap(pName, pdx, pdy, pdz, pAlpha, pTheta, pPhi))
{
}

G4UTrap::G4UTrap( const G4String& pName )
  : G4USolid(pName, new UTrap(pName))
{
}

///////////////////////////////////////////////////////////////////////
//
// Fake default constructor - sets only member data and allocates memory
//                            for usage restricted to object persistency.
//
G4UTrap::G4UTrap( __void__& a )
  : G4USolid(a)
{
}

//////////////////////////////////////////////////////////////////////////
//
// Destructor
//
G4UTrap::~G4UTrap()
{
}

//////////////////////////////////////////////////////////////////////////
//
// Copy constructor
//
G4UTrap::G4UTrap(const G4UTrap& rhs)
  : G4USolid(rhs)
{
}

//////////////////////////////////////////////////////////////////////////
//
// Assignment operator
//
G4UTrap& G4UTrap::operator = (const G4UTrap& rhs) 
{
   // Check assignment to self
   //
   if (this == &rhs)  { return *this; }

   // Copy base class data
   //
   G4USolid::operator=(rhs);

   return *this;
}

/////////////////////////////////////////////////////////////////////////
//
// Dispatch to parameterisation for replication mechanism dimension
// computation & modification.
//
void G4UTrap::ComputeDimensions(      G4VPVParameterisation* p,
                                const G4int n,
                                const G4VPhysicalVolume* pRep)
{
  p->ComputeDimensions(*(G4Trap*)this,n,pRep);
}

//////////////////////////////////////////////////////////////////////////
//
// Make a clone of the object
//
G4VSolid* G4UTrap::Clone() const
{
  return new G4UTrap(*this);
}

//////////////////////////////////////////////////////////////////////////
//
// CreatePolyhedron()
//
G4Polyhedron* G4UTrap::CreatePolyhedron() const
{
  G4double fTthetaSphi = GetShape()->GetThetaSphi();
  G4double fTthetaCphi = GetShape()->GetThetaCphi();
  G4double phi = std::atan2(fTthetaSphi, fTthetaCphi);
  G4double alpha1 = std::atan(GetTanAlpha1());
  G4double alpha2 = std::atan(GetTanAlpha2());
  G4double theta = std::atan(std::sqrt(fTthetaCphi*fTthetaCphi+fTthetaSphi*fTthetaSphi));

  return new G4PolyhedronTrap(GetZHalfLength(), theta, phi,
                              GetYHalfLength1(),
                              GetXHalfLength1(), GetXHalfLength2(), alpha1,
                              GetYHalfLength2(),
                              GetXHalfLength3(), GetXHalfLength4(), alpha2);
}
