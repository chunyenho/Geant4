#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4Orb.cc 83572 2014-09-01 15:23:27Z gcosmo $
//
// class G4Orb
//
// Implementation for G4Orb class
//
// History:
//
// 05.04.12 M.Kelsey   - GetPointOnSurface() throw flat in cos(theta)
// 30.06.04 V.Grichine - bug fixed in DistanceToIn(p,v) on Rmax surface
// 20.08.03 V.Grichine - created
//
//////////////////////////////////////////////////////////////

#include "G4Orb.hh"

#if !defined(G4GEOM_USE_UORB)

#include "G4VoxelLimits.hh"
#include "G4AffineTransform.hh"
#include "G4GeometryTolerance.hh"

#include "G4VPVParameterisation.hh"

#include "Randomize.hh"

#include "meshdefs.hh"

#include "G4VGraphicsScene.hh"

using namespace CLHEP;

// Private enum: Not for external use - used by distanceToOut

enum ESide {kNull,kRMax};

// used by normal

enum ENorm {kNRMax};


////////////////////////////////////////////////////////////////////////
//
// constructor - check positive radius
//             

G4Orb::G4Orb( const G4String& pName, G4double pRmax )
: G4CSGSolid(pName), fRmax(pRmax)
{

  const G4double fEpsilon = 2.e-11;  // relative tolerance of fRmax

  G4double kRadTolerance
    = G4GeometryTolerance::GetInstance()->GetRadialTolerance();

  // Check radius
  //
  if ( pRmax < 10*kCarTolerance )
  {
    G4Exception("G4Orb::G4Orb()", "GeomSolids0002", FatalException,
                "Invalid radius > 10*kCarTolerance.");
  }
  fRmaxTolerance =  std::max( kRadTolerance, fEpsilon*fRmax);

}

///////////////////////////////////////////////////////////////////////
//
// Fake default constructor - sets only member data and allocates memory
//                            for usage restricted to object persistency.
//
G4Orb::G4Orb( __void__& a )
  : G4CSGSolid(a), fRmax(0.), fRmaxTolerance(0.)
{
}

/////////////////////////////////////////////////////////////////////
//
// Destructor

G4Orb::~G4Orb()
{
}

//////////////////////////////////////////////////////////////////////////
//
// Copy constructor

G4Orb::G4Orb(const G4Orb& rhs)
  : G4CSGSolid(rhs), fRmax(rhs.fRmax), fRmaxTolerance(rhs.fRmaxTolerance)
{
}

//////////////////////////////////////////////////////////////////////////
//
// Assignment operator

G4Orb& G4Orb::operator = (const G4Orb& rhs) 
{
   // Check assignment to self
   //
   if (this == &rhs)  { return *this; }

   // Copy base class data
   //
   G4CSGSolid::operator=(rhs);

   // Copy data
   //
   fRmax = rhs.fRmax;
   fRmaxTolerance = rhs.fRmaxTolerance;

   return *this;
}

//////////////////////////////////////////////////////////////////////////
//
// Dispatch to parameterisation for replication mechanism dimension
// computation & modification.

void G4Orb::ComputeDimensions(       G4VPVParameterisation* p,
                               const G4int n,
                               const G4VPhysicalVolume* pRep )
{
  p->ComputeDimensions(*this,n,pRep);
}

////////////////////////////////////////////////////////////////////////////
//
// Calculate extent under transform and specified limit

G4bool G4Orb::CalculateExtent( const EAxis pAxis,
                               const G4VoxelLimits& pVoxelLimit,
                               const G4AffineTransform& pTransform,
                                        G4double& pMin, G4double& pMax ) const
{
    // Compute x/y/z mins and maxs for bounding box respecting limits,
    // with early returns if outside limits. Then switch() on pAxis,
    // and compute exact x and y limit for x/y case
      
    G4double xoffset,xMin,xMax;
    G4double yoffset,yMin,yMax;
    G4double zoffset,zMin,zMax;

    G4double diff1,diff2,delta,maxDiff,newMin,newMax;
    G4double xoff1,xoff2,yoff1,yoff2;

    xoffset=pTransform.NetTranslation().x();
    xMin=xoffset-fRmax;
    xMax=xoffset+fRmax;

    if (pVoxelLimit.IsXLimited())
    {
      if ( (xMin>pVoxelLimit.GetMaxXExtent()+kCarTolerance)
        || (xMax<pVoxelLimit.GetMinXExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (xMin<pVoxelLimit.GetMinXExtent())
        {
          xMin=pVoxelLimit.GetMinXExtent();
        }
        if (xMax>pVoxelLimit.GetMaxXExtent())
        {
          xMax=pVoxelLimit.GetMaxXExtent();
        }
      }
    }
    yoffset=pTransform.NetTranslation().y();
    yMin=yoffset-fRmax;
    yMax=yoffset+fRmax;

    if (pVoxelLimit.IsYLimited())
    {
      if ( (yMin>pVoxelLimit.GetMaxYExtent()+kCarTolerance)
        || (yMax<pVoxelLimit.GetMinYExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (yMin<pVoxelLimit.GetMinYExtent())
        {
          yMin=pVoxelLimit.GetMinYExtent();
        }
        if (yMax>pVoxelLimit.GetMaxYExtent())
        {
          yMax=pVoxelLimit.GetMaxYExtent();
        }
      }
    }
    zoffset=pTransform.NetTranslation().z();
    zMin=zoffset-fRmax;
    zMax=zoffset+fRmax;

    if (pVoxelLimit.IsZLimited())
    {
      if ( (zMin>pVoxelLimit.GetMaxZExtent()+kCarTolerance)
        || (zMax<pVoxelLimit.GetMinZExtent()-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (zMin<pVoxelLimit.GetMinZExtent())
        {
          zMin=pVoxelLimit.GetMinZExtent();
        }
        if (zMax>pVoxelLimit.GetMaxZExtent())
        {
          zMax=pVoxelLimit.GetMaxZExtent();
        }
      }
    }

    // Known to cut sphere

    switch (pAxis)
    {
      case kXAxis:
        yoff1=yoffset-yMin;
        yoff2=yMax-yoffset;

        if ( yoff1 >= 0 && yoff2 >= 0 )
        {
          // Y limits cross max/min x => no change
          //
          pMin=xMin;
          pMax=xMax;
        }
        else
        {
          // Y limits don't cross max/min x => compute max delta x,
          // hence new mins/maxs
          //
          delta=fRmax*fRmax-yoff1*yoff1;
          diff1=(delta>0.) ? std::sqrt(delta) : 0.;
          delta=fRmax*fRmax-yoff2*yoff2;
          diff2=(delta>0.) ? std::sqrt(delta) : 0.;
          maxDiff=(diff1>diff2) ? diff1:diff2;
          newMin=xoffset-maxDiff;
          newMax=xoffset+maxDiff;
          pMin=(newMin<xMin) ? xMin : newMin;
          pMax=(newMax>xMax) ? xMax : newMax;
        }
        break;
      case kYAxis:
        xoff1=xoffset-xMin;
        xoff2=xMax-xoffset;
        if (xoff1>=0&&xoff2>=0)
        {
          // X limits cross max/min y => no change
          //
          pMin=yMin;
          pMax=yMax;
        }
        else
        {
          // X limits don't cross max/min y => compute max delta y,
          // hence new mins/maxs
          //
          delta=fRmax*fRmax-xoff1*xoff1;
          diff1=(delta>0.) ? std::sqrt(delta) : 0.;
          delta=fRmax*fRmax-xoff2*xoff2;
          diff2=(delta>0.) ? std::sqrt(delta) : 0.;
          maxDiff=(diff1>diff2) ? diff1:diff2;
          newMin=yoffset-maxDiff;
          newMax=yoffset+maxDiff;
          pMin=(newMin<yMin) ? yMin : newMin;
          pMax=(newMax>yMax) ? yMax : newMax;
        }
        break;
      case kZAxis:
        pMin=zMin;
        pMax=zMax;
        break;
      default:
        break;
    }
    pMin -= fRmaxTolerance;
    pMax += fRmaxTolerance;

    return true;  
  
}

///////////////////////////////////////////////////////////////////////////
//
// Return whether point inside/outside/on surface
// Split into radius checks
// 

EInside G4Orb::Inside( const G4ThreeVector& p ) const
{
  G4double rad2,tolRMax;
  EInside in;


  rad2 = p.x()*p.x()+p.y()*p.y()+p.z()*p.z();

  G4double radius = std::sqrt(rad2);

  // G4double radius = std::sqrt(rad2);
  // Check radial surface
  // sets `in'
  
  tolRMax = fRmax - fRmaxTolerance*0.5;
    
  if ( radius <= tolRMax )  { in = kInside; }
  else
  {
    tolRMax = fRmax + fRmaxTolerance*0.5;       
    if ( radius <= tolRMax )  { in = kSurface; }
    else                   { in = kOutside; }
  }
  return in;
}

/////////////////////////////////////////////////////////////////////
//
// Return unit normal of surface closest to p
// - note if point on z axis, ignore phi divided sides
// - unsafe if point close to z axis a rmin=0 - no explicit checks

G4ThreeVector G4Orb::SurfaceNormal( const G4ThreeVector& p ) const
{
  ENorm side = kNRMax;
  G4ThreeVector norm;
  G4double radius = std::sqrt(p.x()*p.x()+p.y()*p.y()+p.z()*p.z());

  switch (side)
  {
    case kNRMax: 
      norm = G4ThreeVector(p.x()/radius,p.y()/radius,p.z()/radius);
      break;
   default:        // Should never reach this case ...
      DumpInfo();
      G4Exception("G4Orb::SurfaceNormal()", "GeomSolids1002", JustWarning,
                  "Undefined side for valid surface normal to solid.");
      break;    
  } 

  return norm;
}

///////////////////////////////////////////////////////////////////////////////
//
// Calculate distance to shape from outside, along normalised vector
// - return kInfinity if no intersection, or intersection distance <= tolerance
//
// -> If point is outside outer radius, compute intersection with rmax
//        - if no intersection return
//        - if  valid phi,theta return intersection Dist

G4double G4Orb::DistanceToIn( const G4ThreeVector& p,
                              const G4ThreeVector& v  ) const
{
  G4double snxt = kInfinity;      // snxt = default return value

  G4double radius, pDotV3d; // , tolORMax2, tolIRMax2;
  G4double c, d2, sd = kInfinity;

  const G4double dRmax = 100.*fRmax;

  // General Precalcs

  radius  = std::sqrt(p.x()*p.x() + p.y()*p.y() + p.z()*p.z());
  pDotV3d = p.x()*v.x() + p.y()*v.y() + p.z()*v.z();

  // Radial Precalcs

  // tolORMax2 = (fRmax+fRmaxTolerance*0.5)*(fRmax+fRmaxTolerance*0.5);
  // tolIRMax2 = (fRmax-fRmaxTolerance*0.5)*(fRmax-fRmaxTolerance*0.5);

  // Outer spherical shell intersection
  // - Only if outside tolerant fRmax
  // - Check for if inside and outer G4Orb heading through solid (-> 0)
  // - No intersect -> no intersection with G4Orb
  //
  // Shell eqn: x^2+y^2+z^2 = RSPH^2
  //
  // => (px+svx)^2+(py+svy)^2+(pz+svz)^2=R^2
  //
  // => (px^2+py^2+pz^2) +2sd(pxvx+pyvy+pzvz)+sd^2(vx^2+vy^2+vz^2)=R^2
  // =>      rad2        +2sd(pDotV3d)      +sd^2                =R^2
  //
  // => sd=-pDotV3d+-std::sqrt(pDotV3d^2-(rad2-R^2))

  c = (radius - fRmax)*(radius + fRmax);

  if( radius > fRmax-fRmaxTolerance*0.5 ) // not inside in terms of Inside(p)
  {
    if ( c > fRmaxTolerance*fRmax )
    {
      // If outside tolerant boundary of outer G4Orb in terms of c
      // [ should be std::sqrt(rad2) - fRmax > fRmaxTolerance*0.5 ]

      d2 = pDotV3d*pDotV3d - c;

      if ( d2 >= 0 )
      {
        sd = -pDotV3d - std::sqrt(d2);
        if ( sd >= 0 )
        {
          if ( sd > dRmax ) // Avoid rounding errors due to precision issues seen on
          {                 // 64 bits systems. Split long distances and recompute
            G4double fTerm = sd - std::fmod(sd,dRmax);
            sd = fTerm + DistanceToIn(p+fTerm*v,v);
          } 
          return snxt = sd;
        }
      }
      else    // No intersection with G4Orb
      {
        return snxt = kInfinity;
      }
    }
    else // not outside in terms of c
    {
      if ( c > -fRmaxTolerance*fRmax )  // on surface  
      {
        d2 = pDotV3d*pDotV3d - c;             
        if ( (d2 < fRmaxTolerance*fRmax) || (pDotV3d >= 0) )
        {
          return snxt = kInfinity;
        }
        else
        {
          return snxt = 0.;
        }
      }
    }
  }
#ifdef G4CSGDEBUG
  else // inside ???
  {
      G4Exception("G4Orb::DistanceToIn(p,v)", "GeomSolids1002",
                  JustWarning, "Point p is inside !?");
  }
#endif

  return snxt;
}

//////////////////////////////////////////////////////////////////////
//
// Calculate distance (<= actual) to closest surface of shape from outside
// - Calculate distance to radial plane
// - Return 0 if point inside

G4double G4Orb::DistanceToIn( const G4ThreeVector& p ) const
{
  G4double safe = 0.0,
           radius  = std::sqrt(p.x()*p.x()+p.y()*p.y()+p.z()*p.z());
  safe = radius - fRmax;
  if( safe < 0 ) { safe = 0.; }
  return safe;
}

/////////////////////////////////////////////////////////////////////
//
// Calculate distance to surface of shape from `inside', allowing for tolerance
// 

G4double G4Orb::DistanceToOut( const G4ThreeVector& p,
                               const G4ThreeVector& v,
                               const G4bool calcNorm,
                                     G4bool *validNorm,
                                     G4ThreeVector *n   ) const
{
  G4double snxt = kInfinity;     // ??? snxt is default return value
  ESide    side = kNull;
  
  G4double rad2,pDotV3d; 
  G4double xi,yi,zi;      // Intersection point
  G4double c,d2;
                 
  rad2    = p.x()*p.x() + p.y()*p.y() + p.z()*p.z();
  pDotV3d = p.x()*v.x() + p.y()*v.y() + p.z()*v.z();
    
  // Radial Intersection from G4Orb::DistanceToIn
  //
  // Outer spherical shell intersection
  // - Only if outside tolerant fRmax
  // - Check for if inside and outer G4Orb heading through solid (-> 0)
  // - No intersect -> no intersection with G4Orb
  //
  // Shell eqn: x^2+y^2+z^2=RSPH^2
  //
  // => (px+svx)^2+(py+svy)^2+(pz+svz)^2=R^2
  //
  // => (px^2+py^2+pz^2) +2s(pxvx+pyvy+pzvz)+s^2(vx^2+vy^2+vz^2)=R^2
  // =>      rad2        +2s(pDotV3d)       +s^2                =R^2
  //
  // => s=-pDotV3d+-std::sqrt(pDotV3d^2-(rad2-R^2))
  
  const G4double  Rmax_plus = fRmax + fRmaxTolerance*0.5;
  G4double radius = std::sqrt(rad2);

  if ( radius <= Rmax_plus )
  {
    c = (radius - fRmax)*(radius + fRmax);

    if ( c < fRmaxTolerance*fRmax ) 
    {
      // Within tolerant Outer radius 
      // 
      // The test is
      //     radius  - fRmax < 0.5*fRmaxTolerance
      // =>  radius  < fRmax + 0.5*kRadTol
      // =>  rad2 < (fRmax + 0.5*kRadTol)^2
      // =>  rad2 < fRmax^2 + 2.*0.5*fRmax*kRadTol + 0.25*kRadTol*kRadTol
      // =>  rad2 - fRmax^2    <~    fRmax*kRadTol 

      d2 = pDotV3d*pDotV3d - c;

      if( ( c > -fRmaxTolerance*fRmax) &&         // on tolerant surface
          ( ( pDotV3d >= 0 )   || ( d2 < 0 )) )   // leaving outside from Rmax 
                                                  // not re-entering
      {
        if(calcNorm)
        {
          *validNorm = true;
          *n         = G4ThreeVector(p.x()/fRmax,p.y()/fRmax,p.z()/fRmax);
        }
        return snxt = 0;
      }
      else 
      {
        snxt = -pDotV3d + std::sqrt(d2);    // second root since inside Rmax
        side = kRMax; 
      }
    }
  }
  else // p is outside ???
  {
    G4cout << G4endl;
    DumpInfo();
    std::ostringstream message;
    G4int oldprc = message.precision(16);
    message << "Logic error: snxt = kInfinity ???" << G4endl
            << "Position:"  << G4endl << G4endl
            << "p.x() = "   << p.x()/mm << " mm" << G4endl
            << "p.y() = "   << p.y()/mm << " mm" << G4endl
            << "p.z() = "   << p.z()/mm << " mm" << G4endl << G4endl
            << "Rp = "<< std::sqrt( p.x()*p.x()+p.y()*p.y()+p.z()*p.z() )/mm
            << " mm" << G4endl << G4endl
            << "Direction:" << G4endl << G4endl
            << "v.x() = "   << v.x() << G4endl
            << "v.y() = "   << v.y() << G4endl
            << "v.z() = "   << v.z() << G4endl << G4endl
            << "Proposed distance :" << G4endl << G4endl
            << "snxt = "    << snxt/mm << " mm" << G4endl;
    message.precision(oldprc);
    G4Exception("G4Orb::DistanceToOut(p,v,..)", "GeomSolids1002",
                JustWarning, message);
  }
  if (calcNorm)    // Output switch operator
  {
    switch( side )
    {
      case kRMax:
        xi=p.x()+snxt*v.x();
        yi=p.y()+snxt*v.y();
        zi=p.z()+snxt*v.z();
        *n=G4ThreeVector(xi/fRmax,yi/fRmax,zi/fRmax);
        *validNorm=true;
        break;
      default:
        G4cout << G4endl;
        DumpInfo();
        std::ostringstream message;
        G4int oldprc = message.precision(16);
        message << "Undefined side for valid surface normal to solid."
                << G4endl
                << "Position:"  << G4endl << G4endl
                << "p.x() = "   << p.x()/mm << " mm" << G4endl
                << "p.y() = "   << p.y()/mm << " mm" << G4endl
                << "p.z() = "   << p.z()/mm << " mm" << G4endl << G4endl
                << "Direction:" << G4endl << G4endl
                << "v.x() = "   << v.x() << G4endl
                << "v.y() = "   << v.y() << G4endl
                << "v.z() = "   << v.z() << G4endl << G4endl
                << "Proposed distance :" << G4endl << G4endl
                << "snxt = "    << snxt/mm << " mm" << G4endl;
        message.precision(oldprc);
        G4Exception("G4Orb::DistanceToOut(p,v,..)","GeomSolids1002",
                    JustWarning, message);
        break;
    }
  }
  return snxt;
}

/////////////////////////////////////////////////////////////////////////
//
// Calculate distance (<=actual) to closest surface of shape from inside

G4double G4Orb::DistanceToOut( const G4ThreeVector& p ) const
{
  G4double safe=0.0,radius = std::sqrt(p.x()*p.x()+p.y()*p.y()+p.z()*p.z());

#ifdef G4CSGDEBUG
  if( Inside(p) == kOutside )
  {
     G4int oldprc = G4cout.precision(16);
     G4cout << G4endl;
     DumpInfo();
     G4cout << "Position:"  << G4endl << G4endl;
     G4cout << "p.x() = "   << p.x()/mm << " mm" << G4endl;
     G4cout << "p.y() = "   << p.y()/mm << " mm" << G4endl;
     G4cout << "p.z() = "   << p.z()/mm << " mm" << G4endl << G4endl;
     G4cout.precision(oldprc);
     G4Exception("G4Orb::DistanceToOut(p)", "GeomSolids1002",
                 JustWarning, "Point p is outside !?" );
  }
#endif

  safe = fRmax - radius;
  if ( safe < 0. ) safe = 0.;
  return safe;
}

//////////////////////////////////////////////////////////////////////////
//
// G4EntityType

G4GeometryType G4Orb::GetEntityType() const
{
  return G4String("G4Orb");
}

//////////////////////////////////////////////////////////////////////////
//
// Make a clone of the object
//
G4VSolid* G4Orb::Clone() const
{
  return new G4Orb(*this);
}

//////////////////////////////////////////////////////////////////////////
//
// Stream object contents to an output stream

std::ostream& G4Orb::StreamInfo( std::ostream& os ) const
{
  G4int oldprc = os.precision(16);
  os << "-----------------------------------------------------------\n"
     << "    *** Dump for solid - " << GetName() << " ***\n"
     << "    ===================================================\n"
     << " Solid type: G4Orb\n"
     << " Parameters: \n"

     << "    outer radius: " << fRmax/mm << " mm \n"
     << "-----------------------------------------------------------\n";
  os.precision(oldprc);

  return os;
}

/////////////////////////////////////////////////////////////////////////
//
// GetPointOnSurface

G4ThreeVector G4Orb::GetPointOnSurface() const
{
  //  generate a random number from zero to 2pi...
  //
  G4double phi      = RandFlat::shoot(0.,2.*pi);
  G4double cosphi   = std::cos(phi);
  G4double sinphi   = std::sin(phi);

  // generate a random point uniform in area
  G4double costheta = RandFlat::shoot(-1.,1.);
  G4double sintheta = std::sqrt(1.-sqr(costheta));
  
  return G4ThreeVector (fRmax*sintheta*cosphi,
                        fRmax*sintheta*sinphi, fRmax*costheta); 
}

////////////////////////////////////////////////////////////////////////
//
// Methods for visualisation

void G4Orb::DescribeYourselfTo ( G4VGraphicsScene& scene ) const
{
  scene.AddSolid (*this);
}

G4Polyhedron* G4Orb::CreatePolyhedron () const
{
  return new G4PolyhedronSphere (0., fRmax, 0., 2*pi, 0., pi);
}

#endif
