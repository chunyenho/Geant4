
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4OpenGLStoredQtViewer.cc 87164 2014-11-26 08:48:31Z gcosmo $
//
//
// Class G4OpenGLStoredQtViewer : a class derived from G4OpenGLQtViewer and
//                                G4OpenGLStoredViewer.

#ifdef G4VIS_BUILD_OPENGLQT_DRIVER

#include "G4OpenGLStoredQtViewer.hh"

#include "G4OpenGLStoredSceneHandler.hh"
#include "G4ios.hh"
#ifdef G4MULTITHREADED
#include "G4Threading.hh"
#endif

#include <qapplication.h>

G4OpenGLStoredQtViewer::G4OpenGLStoredQtViewer
(G4OpenGLStoredSceneHandler& sceneHandler,
 const G4String&  name):
  G4VViewer (sceneHandler, sceneHandler.IncrementViewCount (), name),
  G4OpenGLViewer (sceneHandler),
  G4OpenGLQtViewer (sceneHandler),
  G4OpenGLStoredViewer (sceneHandler),             // FIXME : gerer le pb du parent !
  QGLWidget()
{

    // Indicates that the widget has no background, i.e. when the widget receives paint events, the background is not automatically repainted. Note: Unlike WA_OpaquePaintEvent, newly exposed areas are never filled with the background (e.g., after showing a window for the first time the user can see "through" it until the application processes the paint events). This flag is set or cleared by the widget's author.
  QGLWidget::setAttribute (Qt::WA_NoSystemBackground);

  setFocusPolicy(Qt::StrongFocus); // enable keybord events
  fHasToRepaint = false;
  fPaintEventLock = false;
  fUpdateGLLock = false;

  resize(fVP.GetWindowSizeHintX(),fVP.GetWindowSizeHintY());

  if (fViewId < 0) return;  // In case error in base class instantiation.
}

G4OpenGLStoredQtViewer::~G4OpenGLStoredQtViewer() {
  makeCurrent();
  // this is connect to the Dialog for deleting it properly
  // when close event.
  //   ((QDialog*)window())->reject();
}

void G4OpenGLStoredQtViewer::Initialise() {
  makeCurrent();
#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::Initialise 1\n");
#endif
  hide();
  fQGLWidgetInitialiseCompleted = false;
  CreateMainWindow (this,QString(GetName()));

  glDrawBuffer (GL_BACK);
  fQGLWidgetInitialiseCompleted = true;

}

void G4OpenGLStoredQtViewer::initializeGL () {

  InitializeGLView ();

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::InitialiseGL () 1 %d\n", this);
#endif

  if (fSceneHandler.GetScene() == 0) {
    fHasToRepaint =false;
  } else {
    fHasToRepaint =true;
  }

   // Set the component visible
   setVisible(true) ;

   // and update it immediatly before wait for SessionStart() (batch mode)
  QCoreApplication::sendPostedEvents () ;

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::InitialiseGL  END\n");
#endif
}

G4bool G4OpenGLStoredQtViewer::CompareForKernelVisit(G4ViewParameters& lastVP)
{
  // Identical to G4OpenGLStoredViewer::CompareForKernelVisit except
  // for checking of VisAttributesModifiers, because
  // G4OpenGLStoredQtViewer keeps track of its own touchable
  // modifiers (fTreeItemModels, etc.).
  if (
      (lastVP.GetDrawingStyle ()    != fVP.GetDrawingStyle ())    ||
      (lastVP.IsAuxEdgeVisible ()   != fVP.IsAuxEdgeVisible ())   ||
      (lastVP.IsCulling ()          != fVP.IsCulling ())          ||
      (lastVP.IsCullingInvisible () != fVP.IsCullingInvisible ()) ||
      (lastVP.IsDensityCulling ()   != fVP.IsDensityCulling ())   ||
      (lastVP.IsCullingCovered ()   != fVP.IsCullingCovered ())   ||
      (lastVP.IsSection ()          != fVP.IsSection ())          ||
      // Section (DCUT) implemented locally.  But still need to visit
      // kernel if status changes so that back plane culling can be
      // switched.
      (lastVP.IsCutaway ()          != fVP.IsCutaway ())          ||
      // Cutaways implemented locally.  But still need to visit kernel
      // if status changes so that back plane culling can be switched.
      (lastVP.IsExplode ()          != fVP.IsExplode ())          ||
      (lastVP.GetNoOfSides ()       != fVP.GetNoOfSides ())       ||
      (lastVP.GetDefaultVisAttributes()->GetColour() !=
       fVP.GetDefaultVisAttributes()->GetColour())                ||
      (lastVP.GetDefaultTextVisAttributes()->GetColour() !=
       fVP.GetDefaultTextVisAttributes()->GetColour())            ||
      (lastVP.GetBackgroundColour ()!= fVP.GetBackgroundColour ())||
      (lastVP.IsPicking ()          != fVP.IsPicking ())
//      ||
//      (lastVP.GetVisAttributesModifiers().size() !=
//       fVP.GetVisAttributesModifiers().size())
      )
    return true;

  if (lastVP.IsDensityCulling () &&
      (lastVP.GetVisibleDensity () != fVP.GetVisibleDensity ()))
    return true;

  /**************************************************************
   Section (DCUT) implemented locally.  No need to visit kernel if
   section plane itself changes.
   if (lastVP.IsSection () &&
   (lastVP.GetSectionPlane () != fVP.GetSectionPlane ()))
   return true;
   ***************************************************************/

  /**************************************************************
   Cutaways implemented locally.  No need to visit kernel if cutaway
   planes themselves change.
   if (lastVP.IsCutaway ()) {
   if (lastVP.GetCutawayPlanes ().size () !=
   fVP.GetCutawayPlanes ().size ()) return true;
   for (size_t i = 0; i < lastVP.GetCutawayPlanes().size(); ++i)
   if (lastVP.GetCutawayPlanes()[i] != fVP.GetCutawayPlanes()[i])
   return true;
   }
   ***************************************************************/

  if (lastVP.IsExplode () &&
      (lastVP.GetExplodeFactor () != fVP.GetExplodeFactor ()))
    return true;

  return false;
}

G4bool G4OpenGLStoredQtViewer::POSelected(size_t POListIndex)
{
  return isTouchableVisible(POListIndex);
}

G4bool G4OpenGLStoredQtViewer::TOSelected(size_t)
{
  return true;
}

void G4OpenGLStoredQtViewer::DrawView () {
#ifdef G4MULTITHREADED
  if (G4Threading::G4GetThreadId() == G4Threading::MASTER_ID) {
    updateQWidget();
  }
#else
  updateQWidget();
#endif
}

void G4OpenGLStoredQtViewer::ComputeView () {

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::ComputeView %d %d   VVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVV\n",getWinWidth(), getWinHeight());
#endif
  makeCurrent();
  G4ViewParameters::DrawingStyle dstyle = GetViewParameters().GetDrawingStyle();

  //Make sure current viewer is attached and clean...

  //See if things have changed from last time and remake if necessary...
  // The fNeedKernelVisit flag might have been set by the user in
  // /vis/viewer/rebuild, but if not, make decision and set flag only
  // if necessary...
  if (!fNeedKernelVisit) {
    KernelVisitDecision ();
  }
  G4bool kernelVisitWasNeeded = fNeedKernelVisit; // Keep (ProcessView resets).
  ProcessView ();
   
  if (fNeedKernelVisit) {
    displaySceneTreeComponent();
  }
  if(dstyle!=G4ViewParameters::hlr &&
     haloing_enabled) {
#ifdef G4DEBUG_VIS_OGL
    printf("G4OpenGLStoredQtViewer::ComputeView DANS LE IF\n");
#endif

    HaloingFirstPass ();
    DrawDisplayLists ();
    glFlush ();

    HaloingSecondPass ();

    DrawDisplayLists ();
    FinishView ();

  } else {
     
    // If kernel visit was needed, drawing and FinishView will already
    // have been done, so...
    if (!kernelVisitWasNeeded) {
#ifdef G4DEBUG_VIS_OGL
      printf("**************************  G4OpenGLStoredQtViewer::ComputeView Don't need kernel Visit \n");
#endif
      DrawDisplayLists ();
      FinishView ();
    } else {
#ifdef G4DEBUG_VIS_OGL
      printf("**************************  G4OpenGLStoredQtViewer::ComputeView need kernel Visit \n");
#endif
      // However, union cutaways are implemented in DrawDisplayLists, so make
      // an extra pass...
      if (fVP.IsCutaway() &&
          fVP.GetCutawayMode() == G4ViewParameters::cutawayUnion) {
        ClearView();
        DrawDisplayLists ();
        FinishView ();
#ifdef G4DEBUG_VIS_OGL
        printf("***************************  CASE 4 ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^ \n");
#endif
      } else { // ADD TO AVOID KernelVisit=1 and nothing to display
        DrawDisplayLists ();
        FinishView ();
      }
    }
  }

  if (isRecording()) {
    savePPMToTemp();
  }

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::ComputeView %d %d ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^ \n",getWinWidth(), getWinHeight());
#endif
  fHasToRepaint = true;
}


/**
   - Lors du resize de la fenetre, on doit non pas redessiner le detecteur, mais aussi les evenements
*/
void G4OpenGLStoredQtViewer::resizeGL(
                                      int aWidth
                                      ,int aHeight)
{  
  // Set new size, it will be update when next Repaint()->SetView() called
  if ((aWidth > 0) && (aHeight > 0)) {
    ResizeWindow(aWidth,aHeight);
    fHasToRepaint = sizeHasChanged();
  }
}


// We have to get several case :
// - Only activate the windows (mouse click for example) -> Do not redraw
// - resize window -> redraw
// - try to avoid recompute everything if we do not rescale picture (side is the same)
 
void G4OpenGLStoredQtViewer::paintGL()
{
  updateToolbarAndMouseContextMenu();

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::paintGL \n");
#endif
  if (fPaintEventLock) {
//    return ;
  }
  fPaintEventLock = true;
  if ((getWinWidth() == 0) && (getWinHeight() == 0)) {
    return;
  }

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::paintGL ready:%d fHasTo:%d??\n",fReadyToPaint,fHasToRepaint);
#endif
  if (!fQGLWidgetInitialiseCompleted) {
    fPaintEventLock = false;
    return;
  }
  // DO NOT RESIZE IF SIZE HAS NOT CHANGE :
  //    WHEN CLICK ON THE FRAME FOR EXAMPLE
  //    EXECEPT WHEN MOUSE MOVE EVENT
  if ( !fHasToRepaint) {
    // L. Garnier : Trap to get the size with mac OSX 10.6 and Qt 4.6(devel)
    // Tested on Qt4.5 on mac, 4.4 on windows, 4.5 on unbuntu
    int sw = 0;
    int sh = 0;
    if (!isMaximized() && !isFullScreen()) {
      sw = normalGeometry().width();
      sh = normalGeometry().height();
    } else {
      sw = frameGeometry().width();
      sh = frameGeometry().height();
    }
    if ((getWinWidth() == (unsigned int)sw) &&(getWinHeight() == (unsigned int)sh)) {
      return;
    }
  }
#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::paintGL VVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVVV ready %d\n",fReadyToPaint);
#endif
  // Ensure that we really draw the BACK buffer
  glDrawBuffer (GL_BACK);

  SetView();
  
  ClearView (); //ok, put the background correct
  ComputeView();

  fHasToRepaint = false;

#ifdef G4DEBUG_VIS_OGL
  printf("G4OpenGLStoredQtViewer::paintGL ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^ ready %d\n",fReadyToPaint);
#endif
  fPaintEventLock = false;
}

void G4OpenGLStoredQtViewer::paintEvent(QPaintEvent *) {
  if (! fQGLWidgetInitialiseCompleted) {
    return;
  }
  if ( fHasToRepaint) {
    // Will really update the widget by calling CGLFlushDrawable
    // The widget's rendering context will become the current context and initializeGL()
    // will be called if it hasn't already been called.
    // Copies the back buffer of a double-buffered context to the front buffer.
    updateGL();
  }
}

void G4OpenGLStoredQtViewer::mousePressEvent(QMouseEvent *event)
{
  G4MousePressEvent(event);
}

void G4OpenGLStoredQtViewer::keyPressEvent (QKeyEvent * event) 
{
  G4keyPressEvent(event);
}

void G4OpenGLStoredQtViewer::keyReleaseEvent (QKeyEvent * event)
{
  G4keyReleaseEvent(event);
}

void G4OpenGLStoredQtViewer::wheelEvent (QWheelEvent * event)
{
  G4wheelEvent(event);
}

void G4OpenGLStoredQtViewer::showEvent (QShowEvent *) 
{
  fHasToRepaint = true;
}

/**
 * This function was build in order to make a zoom on double clic event.
 * It was think to build a rubberband on the zoom area, but never work fine
 */
void G4OpenGLStoredQtViewer::mouseDoubleClickEvent(QMouseEvent *)
{
  G4MouseDoubleClickEvent();
}

void G4OpenGLStoredQtViewer::mouseReleaseEvent(QMouseEvent *event)
{
  G4MouseReleaseEvent(event);
}

void G4OpenGLStoredQtViewer::mouseMoveEvent(QMouseEvent *event)
{
  G4MouseMoveEvent(event);
}


void G4OpenGLStoredQtViewer::contextMenuEvent(QContextMenuEvent *e)
{
  G4manageContextMenuEvent(e);
}

void G4OpenGLStoredQtViewer::updateQWidget() {
  if (fUpdateGLLock) {
    return;
  }
  fUpdateGLLock = true;
  fHasToRepaint= true;
  // Will really update the widget by calling CGLFlushDrawable
  // The widget's rendering context will become the current context and initializeGL()
  // will be called if it hasn't already been called.
  // Copies the back buffer of a double-buffered context to the front buffer.
  updateGL();
  updateSceneTreeComponentTreeWidgetInfos();
  fHasToRepaint= false;
  fUpdateGLLock = false;
}

void G4OpenGLStoredQtViewer::ShowView (
) 
//////////////////////////////////////////////////////////////////////////////
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!//
{
  // Some X servers fail to draw all trajectories, particularly Mac
  // XQuartz.  Revisit this at a future date.  Meanwhile, issue an
  // extra...
  ClearView();
  DrawView();
  activateWindow();
  //  glFlush(); // NO NEED and as drawView will already cause a flush
  // that could do a double flush
  
}


void G4OpenGLStoredQtViewer::DisplayTimePOColourModification (
G4Colour& c,
size_t poIndex) {
  c = getColorForPoIndex(poIndex);
}

#endif
