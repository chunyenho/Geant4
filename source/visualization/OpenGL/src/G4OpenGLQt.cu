
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id:$
//
// John Allison  27th October 2012
// Base class for OpenGLImmediate/StoredQt graphics system factories.

#ifdef G4VIS_BUILD_OPENGLQT_DRIVER

#include "G4OpenGLQt.hh"

#include "G4UIQt.hh"
#include "G4UImanager.hh"
#include "G4UIbatch.hh"

G4OpenGLQt::G4OpenGLQt (const G4String& name,
                        const G4String& nickname,
                        const G4String& description,
                        Functionality f):
G4VGraphicsSystem (name,
                   nickname,
                   description,
                   f)
{}

G4bool G4OpenGLQt::IsUISessionCompatible () const
{
  G4bool isCompatible = false;
  G4UImanager* ui = G4UImanager::GetUIpointer();
  G4UIsession* session = ui->GetSession();
  // In case it's a G4UIbatch, find original session by recursive search until
  // the session is no longer a G4UIbatch, in which case it will be the
  // original session, if any.
  while (G4UIbatch* batch = dynamic_cast<G4UIbatch*>(session)) {
    session = batch->GetPreviousSession();
  }
  if (!session) {
    // The user has not instantiated a session - must be batch.
    // It's OK to have a Qt window in batch - you can open a viewer, create a
    // scene, set view parameters and /vis/ogl/printEPS.
    isCompatible = true;
  } else {
    // The user has instantiated a session...
    if (dynamic_cast<G4UIQt*>(session)) {
      // ...and it's a G4UIQt session, which is OK.
      isCompatible = true;
    } else {
      // Not OK - go and find the fallback graphics system.
      isCompatible = false;
    }
  }
  return isCompatible;
}

#endif
