//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4PhysicsOrderedFreeVector.cc 74256 2013-10-02 14:24:02Z gcosmo $
//
////////////////////////////////////////////////////////////////////////
// PhysicsOrderedFreeVector Class Implementation
////////////////////////////////////////////////////////////////////////
//
// File:        G4PhysicsOrderedFreeVector.cc
// Version:     2.0
// Created:     1996-08-13
// Author:      Juliet Armstrong
// Updated:     1997-03-25 by Peter Gumplinger
//              > cosmetics (only)
//              1998-11-11 by Peter Gumplinger
//              > initialize all data members of the base class in 
//                derived class constructors
//              2000-11-11 by H.Kurashige
//              > use STL vector for dataVector and binVector
//              2009-06-19 by V.Ivanchenko 
//              > removed hidden bin 
//              2013-10-02 by V.Ivanchenko removed FindBinLocation   
//
// mail:        gum@triumf.ca
//
////////////////////////////////////////////////////////////////////////

#include "G4PhysicsOrderedFreeVector.hh"

/////////////////////////
// Class Implementation
/////////////////////////

        /////////////////
        // Constructors
        /////////////////

G4PhysicsOrderedFreeVector::G4PhysicsOrderedFreeVector(G4double *Energies,
                                                       G4double *Values,
                                                       size_t VectorLength)
  : G4PhysicsVector()
{
  type = T_G4PhysicsOrderedFreeVector;

  for (size_t i = 0 ; i < VectorLength ; i++)
    {
      InsertValues(Energies[i], Values[i]);
    }
}

G4PhysicsOrderedFreeVector::G4PhysicsOrderedFreeVector()
  : G4PhysicsVector()
{
  type = T_G4PhysicsOrderedFreeVector;
}

        ////////////////
        // Destructors
        ////////////////

G4PhysicsOrderedFreeVector::~G4PhysicsOrderedFreeVector() {}

        ////////////
        // Methods
        ////////////
  
void G4PhysicsOrderedFreeVector::InsertValues(G4double energy, G4double value)
{
        std::vector<G4double>::iterator binLoc =
                 std::lower_bound(binVector.begin(), binVector.end(), energy);

        size_t binIdx = binLoc - binVector.begin();	// Iterator difference!

        std::vector<G4double>::iterator dataLoc = dataVector.begin() + binIdx;

        binVector.insert(binLoc, energy);
        dataVector.insert(dataLoc, value);

        numberOfNodes++;
        edgeMin = binVector.front();
        edgeMax = binVector.back();
}

G4double  G4PhysicsOrderedFreeVector::GetEnergy(G4double aValue)
{

        if (aValue <= GetMinValue()) {
                return GetMinLowEdgeEnergy();
        } else if (aValue >= GetMaxValue()) {
                return GetMaxLowEdgeEnergy();
        } else { 
        size_t closestBin = FindValueBinLocation(aValue);
        G4double theEnergy = LinearInterpolationOfEnergy(aValue, closestBin);

        return theEnergy;
        }
}

size_t G4PhysicsOrderedFreeVector::FindValueBinLocation(G4double aValue)
{
   G4int n1 = 0;
   G4int n2 = numberOfNodes/2;
   G4int n3 = numberOfNodes - 1;
   while (n1 != n3 - 1) {
      if (aValue > dataVector[n2])
         { n1 = n2; }
      else
         { n3 = n2; }
      n2 = n1 + (n3 - n1 + 1)/2;
   }
   return (size_t)n1;
}

G4double G4PhysicsOrderedFreeVector::LinearInterpolationOfEnergy(G4double aValue,
								 size_t theLocBin)
{
  G4double intplFactor = (aValue-dataVector[theLocBin])
     / (dataVector[theLocBin+1]-dataVector[theLocBin]); // Interpolation factor

  return binVector[theLocBin] +
         ( binVector[theLocBin+1]-binVector[theLocBin] ) * intplFactor;
}

