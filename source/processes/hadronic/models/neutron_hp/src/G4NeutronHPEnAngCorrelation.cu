#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// neutron_hp -- source file
// J.P. Wellisch, Nov-1996
// A prototype of the low energy neutron transport model.
//
// 100413 Fix bug in incidence energy by T. Koi  
//
#include "G4NeutronHPEnAngCorrelation.hh"
#include "G4LorentzRotation.hh"
#include "G4LorentzVector.hh"
#include "G4RotationMatrix.hh"

G4ReactionProduct * G4NeutronHPEnAngCorrelation::SampleOne(G4double anEnergy)
{  
  G4ReactionProduct * result = new G4ReactionProduct;
  
  // do we have an appropriate distribution
  if(nProducts!=1) throw G4HadronicException(__FILE__, __LINE__, "More than one product in SampleOne");
  
  // get the result
  G4ReactionProductVector * temp=0;
  G4int i=0;
  while(temp == 0) temp = theProducts[i++].Sample(anEnergy);
  
  // is the multiplicity correct
  if(temp->size()!=1) throw G4HadronicException(__FILE__, __LINE__, "SampleOne: Yield not correct");
  
  // fill result
  result = temp->operator[](0);
  
  // some garbage collection
  delete temp;
  
  // return result
  return result;
}

G4ReactionProductVector * G4NeutronHPEnAngCorrelation::Sample(G4double anEnergy)
{
  G4ReactionProductVector * result = new G4ReactionProductVector;
  G4int i;
  G4ReactionProductVector * it;
  G4ReactionProduct theCMS;
  G4LorentzRotation toZ;
  //TK120515 migrate frameFlag (MF6 LCT) = 3 
  //if(frameFlag==2)
  if(frameFlag==2||frameFlag==3)
  {
    // simplify and double check @
    G4ThreeVector the3Neutron = fCache.Get().theNeutron->GetMomentum(); //theNeutron has value in LAB
    G4double nEnergy = fCache.Get().theNeutron->GetTotalEnergy();
    G4ThreeVector the3Target = fCache.Get().theTarget->GetMomentum();  //theTarget has value in LAB
    G4double tEnergy = fCache.Get().theTarget->GetTotalEnergy();
    G4double totE = nEnergy+tEnergy;
    G4ThreeVector the3CMS = the3Target+the3Neutron;
    theCMS.SetMomentum(the3CMS);
    G4double cmsMom = std::sqrt(the3CMS*the3CMS);
    G4double sqrts = std::sqrt((totE-cmsMom)*(totE+cmsMom));
    theCMS.SetMass(sqrts);
    theCMS.SetTotalEnergy(totE);
    G4ReactionProduct aNeutron;
    aNeutron.Lorentz(*fCache.Get().theNeutron, theCMS);
    //TKDB 100413 
    //ENDF-6 Formats Manual ENDF-102
    //CHAPTER 6. FILE 6: PRODUCT ENERGY-ANGLE DISTRIBUTIONS
    //LCT Reference system for secondary energy and angle (incident energy is always given in the LAB system)
    //anEnergy = aNeutron.GetKineticEnergy();
    anEnergy = fCache.Get().theNeutron->GetKineticEnergy(); //should be same argumment of "anEnergy"

    G4LorentzVector Ptmp (aNeutron.GetMomentum(), aNeutron.GetTotalEnergy());

    toZ.rotateZ(-1*Ptmp.phi());
    toZ.rotateY(-1*Ptmp.theta());
  }
  fCache.Get().theTotalMeanEnergy=0;
  G4LorentzRotation toLab(toZ.inverse()); //toLab only change axis NOT to LAB system
  for(i=0; i<nProducts; i++)
  {
    it = theProducts[i].Sample(anEnergy);
    G4double aMeanEnergy = theProducts[i].MeanEnergyOfThisInteraction();
    if(aMeanEnergy>0)
    {
      fCache.Get().theTotalMeanEnergy += aMeanEnergy;
    }
    else
    {
      fCache.Get().theTotalMeanEnergy = anEnergy/nProducts+theProducts[i].GetQValue();
    }
    if(it!=0)
    {
      for(unsigned int ii=0; ii<it->size(); ii++)
      {
        G4LorentzVector pTmp1 (it->operator[](ii)->GetMomentum(),
	                       it->operator[](ii)->GetTotalEnergy());
	pTmp1 = toLab*pTmp1;
	it->operator[](ii)->SetMomentum(pTmp1.vect());
	it->operator[](ii)->SetTotalEnergy(pTmp1.e());
	if(frameFlag==1) // target rest //TK 100413 should be LAB?
	{
          it->operator[](ii)->Lorentz(*(it->operator[](ii)), -1.*(*fCache.Get().theTarget)); //TK 100413 Is this really need?
	}
	else if(frameFlag==2) // CMS
	{
#ifdef G4_NHP_DEBUG
          cout <<"G4NeutronHPEnAngCorrelation: "<<
        	 it->at(ii)->GetTotalEnergy()<<" "<<
        	 it->at(ii)->GetMomentum()<<G4endl;
#endif
          it->operator[](ii)->Lorentz(*(it->operator[](ii)), -1.*theCMS);
	}
        //TK120515 migrate frameFlag (MF6 LCT) = 3 
	else if(frameFlag==3) // CMS A<=4 other LAB
        {
           if ( theProducts[i].GetMassCode() > 4 ) //Alpha AWP 3.96713
           {
              //LAB
              it->operator[](ii)->Lorentz(*(it->operator[](ii)), -1.*(*fCache.Get().theTarget)); //TK 100413 Is this really need?
           }
           else
           {
              //CMS
              it->operator[](ii)->Lorentz(*(it->operator[](ii)), -1.*theCMS);
           }
        }
	else
	{
          throw G4HadronicException(__FILE__, __LINE__, "G4NeutronHPEnAngCorrelation::Sample: The frame of the finalstate is not specified");
	}
	result->push_back(it->operator[](ii));
      }
    delete it;
    }
  }   
  return result;
}

