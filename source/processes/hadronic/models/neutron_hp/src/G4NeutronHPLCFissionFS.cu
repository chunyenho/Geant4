//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// neutron_hp -- source file
// J.P. Wellisch, Nov-1996
// A prototype of the low energy neutron transport model.
//
#include "G4NeutronHPLCFissionFS.hh"

  void G4NeutronHPLCFissionFS::Init (G4double A, G4double Z, G4int M, G4String & dirName, G4String & )
  {
    G4String aString = "/LC/";
    G4NeutronHPFissionBaseFS::Init(A, Z, M, dirName, aString);
  }
  
  G4DynamicParticleVector * G4NeutronHPLCFissionFS::ApplyYourself(G4int NNeutrons)
  {  
    G4DynamicParticleVector * aResult;
//    G4cout <<"G4NeutronHPLCFissionFS::ApplyYourself +"<<G4endl;
    aResult = G4NeutronHPFissionBaseFS::ApplyYourself(NNeutrons);    
    return aResult;
  }
