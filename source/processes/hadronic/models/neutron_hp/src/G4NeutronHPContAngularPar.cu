#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// neutron_hp -- source file
// J.P. Wellisch, Nov-1996
// A prototype of the low energy neutron transport model.
//
// 09-May-06 fix in Sample by T. Koi
// 080318 Fix Compilation warnings - gcc-4.3.0 by T. Koi
//        (This fix has a real effect to the code.) 
// 080409 Fix div0 error with G4FPE by T. Koi
// 080612 Fix contribution from Benoit Pirard and Laurent Desorgher (Univ. Bern) #1
// 080714 Limiting the sum of energy of secondary particles by T. Koi
// 080801 Fix div0 error wiht G4FPE and memory leak by T. Koi
// 081024 G4NucleiPropertiesTable:: to G4NucleiProperties::
//

#include "G4NeutronHPContAngularPar.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4NeutronHPLegendreStore.hh"
#include "G4Gamma.hh"
#include "G4Electron.hh"
#include "G4Positron.hh"
#include "G4Neutron.hh"
#include "G4Proton.hh"
#include "G4Deuteron.hh"
#include "G4Triton.hh"
#include "G4He3.hh"
#include "G4Alpha.hh"
#include "G4NeutronHPVector.hh"
#include "G4NucleiProperties.hh"
#include "G4NeutronHPKallbachMannSyst.hh"
#include "G4IonTable.hh"
 
  void G4NeutronHPContAngularPar::Init(std::istream & aDataFile)
  {
    aDataFile >> theEnergy >> nEnergies >> nDiscreteEnergies >> nAngularParameters;
    theEnergy *= eV;
    theAngular = new G4NeutronHPList [nEnergies];
    for(G4int i=0; i<nEnergies; i++)
    {
      G4double sEnergy;
      aDataFile >> sEnergy;
      sEnergy*=eV;
      theAngular[i].SetLabel(sEnergy);
      theAngular[i].Init(aDataFile, nAngularParameters, 1.);
    }
  }

  G4ReactionProduct * 
  G4NeutronHPContAngularPar::Sample(G4double anEnergy, G4double massCode, G4double /*targetMass*/, 
                                    G4int angularRep, G4int /*interpolE*/ )
  {

      if ( fCache.Get() == NULL ) cacheInit();

    G4ReactionProduct * result = new G4ReactionProduct;
    G4int Z = static_cast<G4int>(massCode/1000);
    G4int A = static_cast<G4int>(massCode-1000*Z);
    if(massCode==0)
    {
      result->SetDefinition(G4Gamma::Gamma());
    }
    else if(A==0)
    {
      result->SetDefinition(G4Electron::Electron());     
      if(Z==1) result->SetDefinition(G4Positron::Positron());
    }
    else if(A==1)
    {
      result->SetDefinition(G4Neutron::Neutron());
      if(Z==1) result->SetDefinition(G4Proton::Proton());
    }
    else if(A==2)
    {
      result->SetDefinition(G4Deuteron::Deuteron());      
    }
    else if(A==3)
    {
      result->SetDefinition(G4Triton::Triton());  
      if(Z==2) result->SetDefinition(G4He3::He3());
    }
    else if(A==4)
    {
      result->SetDefinition(G4Alpha::Alpha());
      if(Z!=2) throw G4HadronicException(__FILE__, __LINE__, "G4NeutronHPContAngularPar: Unknown ion case 1");    
    }
    else
    {
      //result->SetDefinition(G4ParticleTable::GetParticleTable()->FindIon(Z,A,0,Z));
      result->SetDefinition(G4IonTable::GetIonTable()->GetIon(Z,A,0));
    }
    G4int i(0);
    G4int it(0);
    G4double fsEnergy(0);
    G4double cosTh(0);

   if( angularRep == 1 )
   {
// 080612 Fix contribution from Benoit Pirard and Laurent Desorgher (Univ. Bern) #1
       //if (interpolE == 2)
//110609 above was wrong interupition, pointed out by E.Mendoza and D.Cano (CIMAT)
//Following are reviesd version written by T.Koi (SLAC)
      if ( nDiscreteEnergies != 0 )
      {

//1st check remaining_energy 
//	if this is the first set it. (How?)
         if ( fCache.Get()->fresh == true ) 
         { 
            //Discrete Lines, larger energies come first 
            //Continues Emssions, low to high                                      LAST  
            fCache.Get()->remaining_energy = std::max ( theAngular[0].GetLabel() , theAngular[nEnergies-1].GetLabel() );
            fCache.Get()->fresh = false; 
         }

         //Cheating for small remaining_energy 
         //TEMPORAL SOLUTION
         if ( nDiscreteEnergies == nEnergies )
         {
            fCache.Get()->remaining_energy = std::max ( fCache.Get()->remaining_energy , theAngular[nDiscreteEnergies-1].GetLabel() ); //Minimum Line
         }
         else
         {
            //G4double cont_min = theAngular[nDiscreteEnergies].GetLabel();   
            //if ( theAngular[nDiscreteEnergies].GetLabel() == 0.0 ) cont_min = theAngular[nDiscreteEnergies+1].GetLabel();   
            G4double cont_min=0.0; 
            for ( G4int j = nDiscreteEnergies ; j < nEnergies ; j++ )
            {
               cont_min = theAngular[j].GetLabel();   
               if ( theAngular[j].GetValue(0) != 0.0 ) break;  
            }
            fCache.Get()->remaining_energy = std::max ( fCache.Get()->remaining_energy , std::min ( theAngular[nDiscreteEnergies-1].GetLabel() , cont_min ) );   //Minimum Line or grid 
         }
//
	 G4double random = G4UniformRand();

	 G4double * running = new G4double[nEnergies+1];
	 running[0] = 0.0;

         for ( G4int j = 0 ; j < nDiscreteEnergies ; j++ ) 
         {
            G4double delta = 0.0;
            if ( theAngular[j].GetLabel() <= fCache.Get()->remaining_energy ) delta = theAngular[i].GetValue(0);
            running[j+1] = running[j] + delta;
         }
         G4double tot_prob_DIS = running[ nDiscreteEnergies ];
 
         for ( G4int j = nDiscreteEnergies ; j < nEnergies ; j++ ) 
         {
            G4double delta = 0.0;
            G4double e_low = 0.0;
            G4double e_high = 0.0;
            if ( theAngular[j].GetLabel() <= fCache.Get()->remaining_energy ) delta = theAngular[j].GetValue(0);

            //To calculate Prob. e_low and e_high should be in eV 
            //There are two case
            //1:theAngular[nDiscreteEnergies].GetLabel() != 0.0
            //   delta should be used between j-1 and j 
            //   At j = nDiscreteEnergies (the first) e_low should be set explicitly  
            if ( theAngular[j].GetLabel() != 0 )
            {
               if ( j == nDiscreteEnergies ) {
                  e_low = 0.0/eV;
               } else {
                  e_low = theAngular[j-1].GetLabel()/eV;
               }
               e_high = theAngular[j].GetLabel()/eV;
            }
            //2:theAngular[nDiscreteEnergies].GetLabel() == 0.0
            //   delta should be used between j and j+1 
            if ( theAngular[j].GetLabel() == 0.0 ) {
               e_low = theAngular[j].GetLabel()/eV;
               if ( j != nEnergies-1 ) {
                  e_high = theAngular[j+1].GetLabel()/eV;
               } else {
                  e_high = theAngular[j].GetLabel()/eV;
                  if ( theAngular[j].GetValue(0) != 0.0 ) {
                     throw G4HadronicException(__FILE__, __LINE__, "G4NeutronHPContAngularPar: Unexpected non zero value of theAngular[nEnergies-1].GetValue(0)");    
                  }
               }
            }

            running[j+1] = running[j] + ( ( e_high - e_low ) * delta );
         }
         G4double tot_prob_CON = running[ nEnergies ] - running[ nDiscreteEnergies ];

/*
         For FPE debugging 
         if (tot_prob_DIS + tot_prob_CON == 0 ) { 
            G4cout << "TKDB tot_prob_DIS + tot_prob_CON " << tot_prob_DIS + tot_prob_CON << G4endl;
            G4cout << "massCode " << massCode << G4endl;
            G4cout << "nDiscreteEnergies " << nDiscreteEnergies << " nEnergies " << nEnergies << G4endl;
            for ( int j = nDiscreteEnergies ; j < nEnergies ; j++ ) {
               G4cout << j << " " << theAngular[j].GetLabel() << " " << theAngular[j].GetValue(0) << G4endl;
            }
          }
*/
         // Normalize random 
         random *= (tot_prob_DIS + tot_prob_CON);
//2nd Judge Discrete or not             This shoudl be relatively close to 1  For safty 
         if ( random <= ( tot_prob_DIS / ( tot_prob_DIS + tot_prob_CON ) ) || nDiscreteEnergies == nEnergies )      
         {
//          Discrete Emission 
            for ( G4int j = 0 ; j < nDiscreteEnergies ; j++ )
	    {
               //Here we should use i+1
	       if ( random < running[ j+1 ] ) 
               {
                  it = j; 
                  break;
               }
            }
            fsEnergy = theAngular[ it ].GetLabel();

 	    G4NeutronHPLegendreStore theStore(1);
	    theStore.Init(0,fsEnergy,nAngularParameters);
	    for (G4int j=0;j<nAngularParameters;j++)
	    {
	       theStore.SetCoeff(0,j,theAngular[it].GetValue(j));
	    }
	    // use it to sample.
	    cosTh = theStore.SampleMax(fsEnergy);
         //Done 
         }
         else
         {
//          Continuous Emission
            for ( G4int j = nDiscreteEnergies ; j < nEnergies ; j++ )
	    {
               //Here we should use i
	       if ( random < running[ j ] ) 
               {
                  it = j; 
                  break;
               }
            }

            G4double x1 = running[it-1];
            G4double x2 = running[it];

            G4double y1 = 0.0;
            if ( it != nDiscreteEnergies ) 
                y1 = theAngular[it-1].GetLabel();
            G4double y2 = theAngular[it].GetLabel();

            fsEnergy = theInt.Interpolate(theManager.GetInverseScheme(it),
                                         random,x1,x2,y1,y2);

            G4NeutronHPLegendreStore theStore(2);
            theStore.Init(0,y1,nAngularParameters);
            theStore.Init(1,y2,nAngularParameters);
            theStore.SetManager(theManager);
            for (G4int j=0;j<nAngularParameters;j++)
            {
               G4int itt = it;
               if ( it == nDiscreteEnergies ) itt = it+1; //"This case "it-1" has data for Discrete, so we will use an extrpolate values it and it+1
               if ( it == 0 ) 
               {
                  //Safty for unexpected it = 0;
                  //G4cout << "110611 G4NeutronHPContAngularPar::Sample it = 0; invetigation required " << G4endl;
                  itt = it+1; 
               }
               theStore.SetCoeff(0,j,theAngular[itt-1].GetValue(j));
               theStore.SetCoeff(1,j,theAngular[itt].GetValue(j));
            }
            // use it to sample.
            cosTh = theStore.SampleMax(fsEnergy);

        //Done 
        }

         //TK080711
         fCache.Get()->remaining_energy -= fsEnergy;
         //TK080711

         //080801b
	 delete[] running;
         //080801b
      } 
      else 
      {
         // Only continue, TK will clean up 

         //080714 
         if ( fCache.Get()->fresh == true )
         {
            fCache.Get()->remaining_energy = theAngular[ nEnergies-1 ].GetLabel();
            fCache.Get()->fresh = false;
         }
         //080714 
         G4double random = G4UniformRand();
         G4double * running = new G4double[nEnergies];
         running[0]=0;
         G4double weighted = 0;
         for(i=1; i<nEnergies; i++)
         {
/*
           if(i!=0) 
           {
             running[i]=running[i-1];
           }
           running[i] += theInt.GetBinIntegral(theManager.GetScheme(i-1),
                                theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                                theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
           weighted += theInt.GetWeightedBinIntegral(theManager.GetScheme(i-1),
                                theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                                theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
*/

             running[i]=running[i-1];
             if ( fCache.Get()->remaining_energy >= theAngular[i].GetLabel() )
             {
                running[i] += theInt.GetBinIntegral(theManager.GetScheme(i-1),
                                 theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                                 theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
                weighted += theInt.GetWeightedBinIntegral(theManager.GetScheme(i-1),
                                 theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                                 theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
             }
         }
         // cash the mean energy in this distribution
         //080409 TKDB
         if ( nEnergies == 1 || running[nEnergies-1] == 0 )  
            fCache.Get()->currentMeanEnergy = 0.0;
         else
         { 
            fCache.Get()->currentMeanEnergy = weighted/running[nEnergies-1];
         }
         
         //080409 TKDB
         if ( nEnergies == 1 ) it = 0; 

         //080729
         if ( running[nEnergies-1] != 0 )  
         {
            for ( i = 1 ; i < nEnergies ; i++ )
            {
               it = i;
               if ( random < running [ i ] / running [ nEnergies-1 ] ) break;
            } 
         }

         //080714
         if ( running [ nEnergies-1 ] == 0 ) it = 0;
         //080714

         if (it<nDiscreteEnergies||it==0) 
         {
           if(it == 0)
           {
             fsEnergy = theAngular[it].GetLabel();
             G4NeutronHPLegendreStore theStore(1);
             theStore.Init(0,fsEnergy,nAngularParameters);
             for(i=0;i<nAngularParameters;i++)
             {
               theStore.SetCoeff(0,i,theAngular[it].GetValue(i));
             }
             // use it to sample.
             cosTh = theStore.SampleMax(fsEnergy);
           }
           else
           {
             G4double e1, e2;
             e1 = theAngular[it-1].GetLabel();
             e2 = theAngular[it].GetLabel();
             fsEnergy = theInt.Interpolate(theManager.GetInverseScheme(it),
                                           random,
                                           running[it-1]/running[nEnergies-1], 
                                           running[it]/running[nEnergies-1],
                                           e1, e2);
             // fill a Legendrestore
             G4NeutronHPLegendreStore theStore(2);
             theStore.Init(0,e1,nAngularParameters);
             theStore.Init(1,e2,nAngularParameters);
             for(i=0;i<nAngularParameters;i++)
             {
               theStore.SetCoeff(0,i,theAngular[it-1].GetValue(i));
               theStore.SetCoeff(1,i,theAngular[it].GetValue(i));
             }
             // use it to sample.
             theStore.SetManager(theManager);
             cosTh = theStore.SampleMax(fsEnergy);
           }
         }
         else // continuum contribution
         {
           G4double x1 = running[it-1]/running[nEnergies-1];
           G4double x2 = running[it]/running[nEnergies-1];
           G4double y1 = theAngular[it-1].GetLabel();
           G4double y2 = theAngular[it].GetLabel();
           fsEnergy = theInt.Interpolate(theManager.GetInverseScheme(it),
                                         random,x1,x2,y1,y2);
           G4NeutronHPLegendreStore theStore(2);
           theStore.Init(0,y1,nAngularParameters);
           theStore.Init(1,y2,nAngularParameters);
           theStore.SetManager(theManager);
           for(i=0;i<nAngularParameters;i++)
           {
             theStore.SetCoeff(0,i,theAngular[it-1].GetValue(i));
             theStore.SetCoeff(1,i,theAngular[it].GetValue(i));
           }
           // use it to sample.
           cosTh = theStore.SampleMax(fsEnergy);
         }
         delete [] running;

         //080714
         fCache.Get()->remaining_energy -= fsEnergy;
         //080714
      }
   }
    else if(angularRep==2)
    {
      // first get the energy (already the right for this incoming energy)
      G4int j;
      G4double * running = new G4double[nEnergies];
      running[0]=0;
      G4double weighted = 0;
      for(j=1; j<nEnergies; j++)
      {
        if(j!=0) running[j]=running[j-1];
        running[j] += theInt.GetBinIntegral(theManager.GetScheme(j-1),
                             theAngular[j-1].GetLabel(), theAngular[j].GetLabel(),
                             theAngular[j-1].GetValue(0), theAngular[j].GetValue(0));
        weighted += theInt.GetWeightedBinIntegral(theManager.GetScheme(j-1),
                             theAngular[j-1].GetLabel(), theAngular[j].GetLabel(),
                             theAngular[j-1].GetValue(0), theAngular[j].GetValue(0));
      }
      // cash the mean energy in this distribution
      //080409 TKDB
      //currentMeanEnergy = weighted/running[nEnergies-1];
      if ( nEnergies == 1 )
         fCache.Get()->currentMeanEnergy = 0.0;
      else
         fCache.Get()->currentMeanEnergy = weighted/running[nEnergies-1];
      
      G4int itt(0);
      G4double randkal = G4UniformRand();
      //080409 TKDB
      //for(i=0; i<nEnergies; i++)
      for(j=1; j<nEnergies; j++)
      {
        itt = j;
        if(randkal<running[j]/running[nEnergies-1]) break;
      }
      
      // interpolate the secondary energy.
      G4double x, x1,x2,y1,y2;
      if(itt==0) itt=1;
      x = randkal*running[nEnergies-1];
      x1 = running[itt-1];
      x2 = running[itt];
      G4double compoundFraction;
      // interpolate energy
      y1 = theAngular[itt-1].GetLabel();
      y2 = theAngular[itt].GetLabel();
      fsEnergy = theInt.Interpolate(theManager.GetInverseScheme(itt-1), 
                                    x, x1,x2,y1,y2);
      // for theta interpolate the compoundFractions
      G4double cLow = theAngular[itt-1].GetValue(1);
      G4double cHigh = theAngular[itt].GetValue(1);
      compoundFraction = theInt.Interpolate(theManager.GetScheme(itt),
                                            fsEnergy, y1, y2, cLow,cHigh);
      delete [] running;
      
      // get cosTh
      G4double incidentEnergy = anEnergy;
      G4double incidentMass = G4Neutron::Neutron()->GetPDGMass();
      G4double productEnergy = fsEnergy;
      G4double productMass = result->GetMass();
      G4int targetZ = G4int(theTargetCode/1000);
      G4int targetA = G4int(theTargetCode-1000*targetZ);
      // To correspond to natural composition (-nat-) data files. 
      if ( targetA == 0 ) 
         targetA = G4int ( theTarget->GetMass()/amu_c2 + 0.5 );
      G4double targetMass = theTarget->GetMass();
      G4int residualA = targetA+1-A;
      G4int residualZ = targetZ-Z;
      G4double residualMass =  residualZ*G4Proton::Proton()->GetPDGMass();
               residualMass +=(residualA-residualZ)*G4Neutron::Neutron()->GetPDGMass();
               residualMass -= G4NucleiProperties::GetBindingEnergy( residualA , residualZ );
      G4NeutronHPKallbachMannSyst theKallbach(compoundFraction,
                                              incidentEnergy, incidentMass,
                                              productEnergy, productMass,
                                              residualMass, residualA, residualZ,
                                              targetMass, targetA, targetZ);
      cosTh = theKallbach.Sample(anEnergy);
    }
    else if(angularRep>10&&angularRep<16)
    {
      G4double random = G4UniformRand();
      G4double * running = new G4double[nEnergies];
      running[0]=0;      
      G4double weighted = 0;
      for(i=1; i<nEnergies; i++)
      {
        if(i!=0) running[i]=running[i-1];
        running[i] += theInt.GetBinIntegral(theManager.GetScheme(i-1),
                             theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                             theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
        weighted += theInt.GetWeightedBinIntegral(theManager.GetScheme(i-1),
                             theAngular[i-1].GetLabel(), theAngular[i].GetLabel(),
                             theAngular[i-1].GetValue(0), theAngular[i].GetValue(0));
      }
       // cash the mean energy in this distribution
      //currentMeanEnergy = weighted/running[nEnergies-1];
      if ( nEnergies == 1 )  
         fCache.Get()->currentMeanEnergy = 0.0;
      else
         fCache.Get()->currentMeanEnergy = weighted/running[nEnergies-1];
      
      //080409 TKDB
      if ( nEnergies == 1 ) it = 0; 
      //for(i=0; i<nEnergies; i++)
      for(i=1; i<nEnergies; i++)
      {
        it = i;
        if(random<running[i]/running[nEnergies-1]) break;
      }
      if(it<nDiscreteEnergies||it==0) 
      {
        if(it==0)
        {
          fsEnergy = theAngular[0].GetLabel();          
          G4NeutronHPVector theStore; 
	  G4int aCounter = 0;
          for(G4int j=1; j<nAngularParameters; j+=2) 
          {
            theStore.SetX(aCounter, theAngular[0].GetValue(j));
            theStore.SetY(aCounter, theAngular[0].GetValue(j+1));
	    aCounter++;	    
          }
          G4InterpolationManager aMan;
          aMan.Init(angularRep-10, nAngularParameters-1);
          theStore.SetInterpolationManager(aMan);
          cosTh = theStore.Sample();
        }
        else 
        {
          fsEnergy = theAngular[it].GetLabel();
          G4NeutronHPVector theStore; 
          G4InterpolationManager aMan;
          aMan.Init(angularRep-10, nAngularParameters-1);
          theStore.SetInterpolationManager(aMan); // Store interpolates f(costh)
          G4InterpolationScheme currentScheme = theManager.GetInverseScheme(it);
	  G4int aCounter = 0;
          for(G4int j=1; j<nAngularParameters; j+=2) 
          {
            theStore.SetX(aCounter, theAngular[it].GetValue(j));
            theStore.SetY(aCounter, theInt.Interpolate(currentScheme, 
                                       random,
                                       running[it-1]/running[nEnergies-1],
                                       running[it]/running[nEnergies-1],
                                       theAngular[it-1].GetValue(j+1),
                                       theAngular[it].GetValue(j+1)));
	    aCounter++;	    
          }
          cosTh = theStore.Sample();
        }
      }
      else
      {
        G4double x1 = running[it-1]/running[nEnergies-1];
        G4double x2 = running[it]/running[nEnergies-1];
        G4double y1 = theAngular[it-1].GetLabel();
        G4double y2 = theAngular[it].GetLabel();
        fsEnergy = theInt.Interpolate(theManager.GetInverseScheme(it),
                                      random,x1,x2,y1,y2);
        G4NeutronHPVector theBuff1;
        G4NeutronHPVector theBuff2;
        G4InterpolationManager aMan;
        aMan.Init(angularRep-10, nAngularParameters-1);
//        theBuff1.SetInterpolationManager(aMan); // Store interpolates f(costh)
//        theBuff2.SetInterpolationManager(aMan); // Store interpolates f(costh)
//      Bug Report #1366 from L. Russell 
        //for(i=0; i<nAngularParameters; i++) // i=1 ist wichtig!
        //{
        //  theBuff1.SetX(i, theAngular[it-1].GetValue(i));
        //  theBuff1.SetY(i, theAngular[it-1].GetValue(i+1));
        //  theBuff2.SetX(i, theAngular[it].GetValue(i));
        //  theBuff2.SetY(i, theAngular[it].GetValue(i+1));
        //  i++;
        //}
        {
        G4int j;
        for(i=0,j=1; i<nAngularParameters; i++,j+=2) 
        {
          theBuff1.SetX(i, theAngular[it-1].GetValue(j));
          theBuff1.SetY(i, theAngular[it-1].GetValue(j+1));
          theBuff2.SetX(i, theAngular[it].GetValue(j));
          theBuff2.SetY(i, theAngular[it].GetValue(j+1));
        }
        }
        G4NeutronHPVector theStore;
        theStore.SetInterpolationManager(aMan); // Store interpolates f(costh)        
        x1 = y1;
        x2 = y2;
        G4double x, y;
        //for(i=0;i<theBuff1.GetVectorLength(); i++);
        for(i=0;i<theBuff1.GetVectorLength(); i++)
        {
          x = theBuff1.GetX(i); // costh binning identical
          y1 = theBuff1.GetY(i);
          y2 = theBuff2.GetY(i);
          y = theInt.Interpolate(theManager.GetScheme(it),
                                 fsEnergy, theAngular[it-1].GetLabel(), 
                                 theAngular[it].GetLabel(), y1, y2);
          theStore.SetX(i, x);
          theStore.SetY(i, y);
        }
        cosTh = theStore.Sample();
      }
      delete [] running;
    }
    else
    {
      throw G4HadronicException(__FILE__, __LINE__, "G4NeutronHPContAngularPar::Sample: Unknown angular representation");
    }
    result->SetKineticEnergy(fsEnergy);
    G4double phi = twopi*G4UniformRand();
    G4double theta = std::acos(cosTh);
    G4double sinth = std::sin(theta);
    G4double mtot = result->GetTotalMomentum();
    G4ThreeVector tempVector(mtot*sinth*std::cos(phi), mtot*sinth*std::sin(phi), mtot*std::cos(theta) );
    result->SetMomentum(tempVector);
//  return the result.    
    return result;
  }
