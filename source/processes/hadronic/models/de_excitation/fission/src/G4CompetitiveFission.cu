#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4CompetitiveFission.cc 85841 2014-11-05 15:35:06Z gcosmo $
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Oct 1998)
//
// J. M. Quesada (March 2009). Bugs fixed:
//          - Full relativistic calculation (Lorentz boosts)
//          - Fission pairing energy is included in fragment excitation energies
// Now Energy and momentum are conserved in fission 

#include "G4CompetitiveFission.hh"
#include "G4PairingCorrection.hh"
#include "G4ParticleMomentum.hh"
#include "G4Pow.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

G4CompetitiveFission::G4CompetitiveFission() : G4VEvaporationChannel("fission")
{
  theFissionBarrierPtr = new G4FissionBarrier;
  MyOwnFissionBarrier = true;

  theFissionProbabilityPtr = new G4FissionProbability;
  MyOwnFissionProbability = true;
  
  theLevelDensityPtr = new G4FissionLevelDensityParameter;
  MyOwnLevelDensity = true;

  MaximalKineticEnergy = -1000.0*MeV;
  FissionBarrier = 0.0;
  FissionProbability = 0.0;
  LevelDensityParameter = 0.0;
  pairingCorrection = G4PairingCorrection::GetInstance();
}

G4CompetitiveFission::~G4CompetitiveFission()
{
  if (MyOwnFissionBarrier) delete theFissionBarrierPtr;
  if (MyOwnFissionProbability) delete theFissionProbabilityPtr;
  if (MyOwnLevelDensity) delete theLevelDensityPtr;
}

G4double G4CompetitiveFission::GetEmissionProbability(G4Fragment* fragment)
{
  G4int anA = fragment->GetA_asInt();
  G4int aZ  = fragment->GetZ_asInt();
  G4double ExEnergy = fragment->GetExcitationEnergy() - 
    pairingCorrection->GetFissionPairingCorrection(anA,aZ);
  

  // Saddle point excitation energy ---> A = 65
  // Fission is excluded for A < 65
  if (anA >= 65 && ExEnergy > 0.0) {
    FissionBarrier = theFissionBarrierPtr->FissionBarrier(anA,aZ,ExEnergy);
    MaximalKineticEnergy = ExEnergy - FissionBarrier;
    LevelDensityParameter = 
      theLevelDensityPtr->LevelDensityParameter(anA,aZ,ExEnergy);
    FissionProbability = 
      theFissionProbabilityPtr->EmissionProbability(*fragment,MaximalKineticEnergy);
    }
  else {
    MaximalKineticEnergy = -1000.0*MeV;
    LevelDensityParameter = 0.0;
    FissionProbability = 0.0;
  }
  return FissionProbability;
}

G4FragmentVector * G4CompetitiveFission::BreakUp(const G4Fragment & theNucleus)
{
  G4FragmentVector * theResult = new G4FragmentVector();
  G4Fragment* frag0 = new G4Fragment(theNucleus);
  G4Fragment* frag1 = EmittedFragment(frag0);
  if(frag1) { theResult->push_back(frag1); }
  theResult->push_back(frag0);
  return theResult;
}

G4Fragment* G4CompetitiveFission::EmittedFragment(G4Fragment* theNucleus)
{
  G4Fragment * Fragment1 = 0; 
  // Nucleus data
  // Atomic number of nucleus
  G4int A = theNucleus->GetA_asInt();
  // Charge of nucleus
  G4int Z = theNucleus->GetZ_asInt();
  //   Excitation energy (in MeV)
  G4double U = theNucleus->GetExcitationEnergy();
  G4double pcorr = pairingCorrection->GetFissionPairingCorrection(A,Z);
  if (U <= pcorr) { return Fragment1; }

  // Atomic Mass of Nucleus (in MeV)
  G4double M = theNucleus->GetGroundStateMass();

  // Nucleus Momentum
  G4LorentzVector theNucleusMomentum = theNucleus->GetMomentum();

  // Calculate fission parameters
  G4FissionParameters theParameters(A,Z,U-pcorr,FissionBarrier);
  
  // First fragment
  G4int A1 = 0;
  G4int Z1 = 0;
  G4double M1 = 0.0;

  // Second fragment
  G4int A2 = 0;
  G4int Z2 = 0;
  G4double M2 = 0.0;

  G4double FragmentsExcitationEnergy = 0.0;
  G4double FragmentsKineticEnergy = 0.0;

  //JMQ 04/03/09 It will be used latter to fix the bug in energy conservation
  G4double FissionPairingEnergy=
    pairingCorrection->GetFissionPairingCorrection(A,Z);

  G4int Trials = 0;
  do {

    // First fragment 
    A1 = FissionAtomicNumber(A,theParameters);
    Z1 = FissionCharge(A,Z,A1);
    M1 = G4NucleiProperties::GetNuclearMass(A1, Z1);

    // Second Fragment
    A2 = A - A1;
    Z2 = Z - Z1;
    if (A2 < 1 || Z2 < 0 || Z2 > A2) {
      FragmentsExcitationEnergy = -1.0;
      continue;
    }
    M2 = G4NucleiProperties::GetNuclearMass(A2, Z2);
    // Maximal Kinetic Energy (available energy for fragments)
    G4double Tmax = M + U - M1 - M2;

    // Check that fragment masses are less or equal than total energy
    if (Tmax < 0.0) {
      FragmentsExcitationEnergy = -1.0;
      continue;
    }

    FragmentsKineticEnergy = FissionKineticEnergy( A , Z,
						   A1, Z1,
						   A2, Z2,
						   U , Tmax,
						   theParameters);
    
    // Excitation Energy
    //	FragmentsExcitationEnergy = Tmax - FragmentsKineticEnergy;
    // JMQ 04/03/09 BUG FIXED: in order to fulfill energy conservation the
    // fragments carry the fission pairing energy in form of 
    // excitation energy

    FragmentsExcitationEnergy = 
      Tmax - FragmentsKineticEnergy+FissionPairingEnergy;

  } while (FragmentsExcitationEnergy < 0.0 && Trials++ < 100);
    
  if (FragmentsExcitationEnergy <= 0.0) { 
    throw G4HadronicException(__FILE__, __LINE__, 
      "G4CompetitiveFission::BreakItUp: Excitation energy for fragments < 0.0!");
  }

  // Fragment 1
  M1 += FragmentsExcitationEnergy * A1/static_cast<G4double>(A);
  // Fragment 2
  M2 += FragmentsExcitationEnergy * A2/static_cast<G4double>(A);
  // primary
  M += U;

  G4double etot1 = (M*M - M2*M2 + M1*M1)/(2*M);
  G4ParticleMomentum Momentum1(IsotropicVector(std::sqrt((etot1 - M1)*(etot1+M1))));
  G4LorentzVector FourMomentum1(Momentum1, etot1);
  FourMomentum1.boost(theNucleusMomentum.boostVector());
    
  // Create Fragments
  Fragment1 = new G4Fragment( A1, Z1, FourMomentum1);
  theNucleusMomentum -= FourMomentum1;
  theNucleus->SetZandA_asInt(Z2, A2);
  theNucleus->SetMomentum(theNucleusMomentum);
  return Fragment1;
}

G4int 
G4CompetitiveFission::FissionAtomicNumber(G4int A, 
					  const G4FissionParameters & theParam)
  // Calculates the atomic number of a fission product
{

  // For Simplicity reading code
  G4double A1 = theParam.GetA1();
  G4double A2 = theParam.GetA2();
  G4double As = theParam.GetAs();
  G4double Sigma2 = theParam.GetSigma2();
  G4double SigmaS = theParam.GetSigmaS();
  G4double w = theParam.GetW();
  
  //    G4double FasymAsym = 2.0*std::exp(-((A2-As)*(A2-As))/(2.0*Sigma2*Sigma2)) + 
  //	std::exp(-((A1-As)*(A1-As))/(2.0*Sigma1*Sigma1));

  //    G4double FsymA1A2 = std::exp(-((As-(A1+A2))*(As-(A1+A2)))/(2.0*SigmaS*SigmaS));

  G4double C2A = A2 + 3.72*Sigma2;
  G4double C2S = As + 3.72*SigmaS;
  
  G4double C2 = 0.0;
  if (w > 1000.0 ) C2 = C2S;
  else if (w < 0.001) C2 = C2A;
  else C2 =  std::max(C2A,C2S);

  G4double C1 = A-C2;
  if (C1 < 30.0) {
    C2 = A-30.0;
    C1 = 30.0;
  }

  G4double Am1 = (As + A1)/2.0;
  G4double Am2 = (A1 + A2)/2.0;

  // Get Mass distributions as sum of symmetric and asymmetric Gasussians
  G4double Mass1 = MassDistribution(As,A,theParam); 
  G4double Mass2 = MassDistribution(Am1,A,theParam); 
  G4double Mass3 = MassDistribution(A1,A,theParam); 
  G4double Mass4 = MassDistribution(Am2,A,theParam); 
  G4double Mass5 = MassDistribution(A2,A,theParam); 
  // get maximal value among Mass1,...,Mass5
  G4double MassMax = Mass1;
  if (Mass2 > MassMax) MassMax = Mass2;
  if (Mass3 > MassMax) MassMax = Mass3;
  if (Mass4 > MassMax) MassMax = Mass4;
  if (Mass5 > MassMax) MassMax = Mass5;

  // Sample a fragment mass number, which lies between C1 and C2
  G4double xm;
  G4double Pm;
  do {
    xm = C1+G4UniformRand()*(C2-C1);
    Pm = MassDistribution(xm,A,theParam); 
  } while (MassMax*G4UniformRand() > Pm);
  G4int ires = G4lrint(xm);

  return ires;
}

G4double 
G4CompetitiveFission::MassDistribution(G4double x, G4double A, 
				       const G4FissionParameters & theParam)
  // This method gives mass distribution F(x) = F_{asym}(x)+w*F_{sym}(x)
  // which consist of symmetric and asymmetric sum of gaussians components.
{
  G4double Xsym = std::exp(-0.5*(x-theParam.GetAs())*(x-theParam.GetAs())/
			   (theParam.GetSigmaS()*theParam.GetSigmaS()));

  G4double Xasym = std::exp(-0.5*(x-theParam.GetA2())*(x-theParam.GetA2())/
			    (theParam.GetSigma2()*theParam.GetSigma2())) + 
    std::exp(-0.5*(x-(A-theParam.GetA2()))*(x-(A-theParam.GetA2()))/
	     (theParam.GetSigma2()*theParam.GetSigma2())) +
    0.5*std::exp(-0.5*(x-theParam.GetA1())*(x-theParam.GetA1())/
		 (theParam.GetSigma1()*theParam.GetSigma1())) +
    0.5*std::exp(-0.5*(x-(A-theParam.GetA1()))*(x-(A-theParam.GetA1()))/
		 (theParam.GetSigma1()*theParam.GetSigma1()));

  if (theParam.GetW() > 1000) return Xsym;
  else if (theParam.GetW() < 0.001) return Xasym;
  else return theParam.GetW()*Xsym+Xasym;
}

G4int G4CompetitiveFission::FissionCharge(G4double A, G4double Z,
					  G4double Af)
  // Calculates the charge of a fission product for a given atomic number Af
{
  static const G4double sigma = 0.6;
  G4double DeltaZ = 0.0;
  if (Af >= 134.0) DeltaZ = -0.45;                    //                      134 <= Af
  else if (Af <= (A-134.0)) DeltaZ = 0.45;             // Af <= (A-134) 
  else DeltaZ = -0.45*(Af-(A/2.0))/(134.0-(A/2.0));   //       (A-134) < Af < 134

  G4double Zmean = (Af/A)*Z + DeltaZ;
 
  G4double theZ;
  do {
    theZ = G4RandGauss::shoot(Zmean,sigma);
  } while (theZ  < 1.0 || theZ > (Z-1.0) || theZ > Af);
  //  return static_cast<G4int>(theZ+0.5);
  return static_cast<G4int>(theZ+0.5);
}

G4double 
G4CompetitiveFission::FissionKineticEnergy(G4int A, G4int Z,
					   G4double Af1, G4double /*Zf1*/,
					   G4double Af2, G4double /*Zf2*/,
					   G4double /*U*/, G4double Tmax,
					   const G4FissionParameters & theParam)
  // Gives the kinetic energy of fission products
{
  // Find maximal value of A for fragments
  G4double AfMax = std::max(Af1,Af2);
  if (AfMax < (A/2.0)) AfMax = A - AfMax;

  // Weights for symmetric and asymmetric components
  G4double Pas;
  if (theParam.GetW() > 1000) Pas = 0.0;
  else {
    G4double P1 = 0.5*std::exp(-0.5*(AfMax-theParam.GetA1())*(AfMax-theParam.GetA1())/
			       (theParam.GetSigma1()*theParam.GetSigma1()));

    G4double P2 = std::exp(-0.5*(AfMax-theParam.GetA2())*(AfMax-theParam.GetA2())/
			   (theParam.GetSigma2()*theParam.GetSigma2()));

    Pas = P1+P2;
  }

  G4double Ps;
  if (theParam.GetW() < 0.001) Ps = 0.0;
  else {
    Ps = theParam.GetW()*std::exp(-0.5*(AfMax-theParam.GetAs())*(AfMax-theParam.GetAs())/
				  (theParam.GetSigmaS()*theParam.GetSigmaS()));
  }
  G4double Psy = Ps/(Pas+Ps);

  // Fission fractions Xsy and Xas formed in symmetric and asymmetric modes
  G4double PPas = theParam.GetSigma1() + 2.0 * theParam.GetSigma2();
  G4double PPsy = theParam.GetW() * theParam.GetSigmaS();
  G4double Xas = PPas / (PPas+PPsy);
  G4double Xsy = PPsy / (PPas+PPsy);

  // Average kinetic energy for symmetric and asymmetric components
  G4double Eaverage = 0.1071*MeV*(Z*Z)/G4Pow::GetInstance()->Z13(A) + 22.2*MeV;

  // Compute maximal average kinetic energy of fragments and Energy Dispersion (sqrt)
  G4double TaverageAfMax;
  G4double ESigma = 10*MeV;
  // Select randomly fission mode (symmetric or asymmetric)
  if (G4UniformRand() > Psy) { // Asymmetric Mode
    G4double A11 = theParam.GetA1()-0.7979*theParam.GetSigma1();
    G4double A12 = theParam.GetA1()+0.7979*theParam.GetSigma1();
    G4double A21 = theParam.GetA2()-0.7979*theParam.GetSigma2();
    G4double A22 = theParam.GetA2()+0.7979*theParam.GetSigma2();
    // scale factor
    G4double ScaleFactor = 0.5*theParam.GetSigma1()*
      (AsymmetricRatio(A,A11)+AsymmetricRatio(A,A12))+
      theParam.GetSigma2()*(AsymmetricRatio(A,A21)+AsymmetricRatio(A,A22));
    // Compute average kinetic energy for fragment with AfMax
    TaverageAfMax = (Eaverage + 12.5 * Xsy) * (PPas/ScaleFactor) * 
      AsymmetricRatio(A,AfMax);

  } else { // Symmetric Mode
    G4double As0 = theParam.GetAs() + 0.7979*theParam.GetSigmaS();
    // scale factor
    G4double ScaleFactor = theParam.GetW()*theParam.GetSigmaS()*SymmetricRatio(A,As0);
    // Compute average kinetic energy for fragment with AfMax
    TaverageAfMax = (Eaverage - 12.5*MeV*Xas) * (PPsy/ScaleFactor) * 
      SymmetricRatio(A,AfMax);
    ESigma = 8.0*MeV;
  }


  // Select randomly, in accordance with Gaussian distribution, fragment kinetic energy
  G4double KineticEnergy;
  G4int i = 0;
  do {
    KineticEnergy = G4RandGauss::shoot(TaverageAfMax,ESigma);
    if (i++ > 100) return Eaverage;
  } while (KineticEnergy < Eaverage-3.72*ESigma || 
	   KineticEnergy > Eaverage+3.72*ESigma ||
	   KineticEnergy > Tmax);
  
  return KineticEnergy;
}

G4double G4CompetitiveFission::AsymmetricRatio(G4int A, G4double A11)
{
  static const G4double B1 = 23.5;
  static const G4double A00 = 134.0;
  return Ratio(G4double(A),A11,B1,A00);
}

G4double G4CompetitiveFission::SymmetricRatio(G4int A, G4double A11)
{
  static const G4double B1 = 5.32;
  const G4double A00 = A/2.0;
  return Ratio(G4double(A),A11,B1,A00);
}

G4double G4CompetitiveFission::Ratio(G4double A, G4double A11,
				     G4double B1, G4double A00) 
{
  if (A == 0.0) {
    throw G4HadronicException(__FILE__, __LINE__, 
			      "G4CompetitiveFission::Ratio: A == 0!");
  }
  if (A11 >= A/2.0 && A11 <= (A00+10.0)) {
    return 1.0-B1*((A11-A00)/A)*((A11-A00)/A);
  } else {
    return 1.0-B1*(10.0/A)*(10.0/A)-2.0*(10.0/A)*B1*((A11-A00-10.0)/A);
  }
}

G4ThreeVector G4CompetitiveFission::IsotropicVector(const G4double Magnitude)
  // Samples a isotropic random vectorwith a magnitud given by Magnitude.
  // By default Magnitude = 1.0
{
  G4double CosTheta = 1.0 - 2.0*G4UniformRand();
  G4double SinTheta = std::sqrt(1.0 - CosTheta*CosTheta);
  G4double Phi = twopi*G4UniformRand();
  G4ThreeVector Vector(Magnitude*std::cos(Phi)*SinTheta,
		       Magnitude*std::sin(Phi)*SinTheta,
		       Magnitude*CosTheta);
  return Vector;
}

#ifdef debug
void G4CompetitiveFission::CheckConservation(const G4Fragment & theInitialState,
					     G4FragmentVector * Result) const
{
    G4double ProductsEnergy =0;
    G4ThreeVector ProductsMomentum;
    G4int ProductsA = 0;
    G4int ProductsZ = 0;
    G4FragmentVector::iterator h;
    for (h = Result->begin(); h != Result->end(); h++) {
	G4LorentzVector tmp = (*h)->GetMomentum();
	ProductsEnergy += tmp.e();
	ProductsMomentum += tmp.vect();
	ProductsA += (*h)->GetA_asInt();
	ProductsZ += (*h)->GetZ_asInt();
    }

    if (ProductsA != theInitialState.GetA_asInt()) {
	G4cout << "!!!!!!!!!! Baryonic Number Conservation Violation !!!!!!!!!!" 
	       << G4endl;
	G4cout << "G4CompetitiveFission: Baryon Number Conservation test for fission fragments" 
	       << G4endl; 
	G4cout << "Initial A = " << theInitialState.GetA_asInt() 
	       << "   Fragments A = " << ProductsA << "   Diference --> " 
	       << theInitialState.GetA_asInt() - ProductsA << G4endl;
    }
    if (ProductsZ != theInitialState.GetZ_asInt()) {
	G4cout << "!!!!!!!!!! Charge Conservation Violation !!!!!!!!!!" << G4endl;
	G4cout << "G4CompetitiveFission.cc: Charge Conservation test for fission fragments" 
	       << G4endl; 
	G4cout << "Initial Z = " << theInitialState.GetZ_asInt() 
	       << "   Fragments Z = " << ProductsZ << "   Diference --> " 
	       << theInitialState.GetZ() - ProductsZ << G4endl;
    }
    if (std::fabs(ProductsEnergy-theInitialState.GetMomentum().e()) > 1.0*keV) {
	G4cout << "!!!!!!!!!! Energy Conservation Violation !!!!!!!!!!" << G4endl;
	G4cout << "G4CompetitiveFission.cc: Energy Conservation test for fission fragments" 
	       << G4endl; 
	G4cout << "Initial E = " << theInitialState.GetMomentum().e()/MeV << " MeV"
	       << "   Fragments E = " << ProductsEnergy/MeV  << " MeV   Diference --> " 
	       << (theInitialState.GetMomentum().e() - ProductsEnergy)/MeV << " MeV" << G4endl;
    } 
    if (std::fabs(ProductsMomentum.x()-theInitialState.GetMomentum().x()) > 1.0*keV || 
	std::fabs(ProductsMomentum.y()-theInitialState.GetMomentum().y()) > 1.0*keV ||
	std::fabs(ProductsMomentum.z()-theInitialState.GetMomentum().z()) > 1.0*keV) {
	G4cout << "!!!!!!!!!! Momentum Conservation Violation !!!!!!!!!!" << G4endl;
	G4cout << "G4CompetitiveFission.cc: Momentum Conservation test for fission fragments" 
	       << G4endl; 
	G4cout << "Initial P = " << theInitialState.GetMomentum().vect() << " MeV"
	       << "   Fragments P = " << ProductsMomentum  << " MeV   Diference --> " 
	       << theInitialState.GetMomentum().vect() - ProductsMomentum << " MeV" << G4endl;
    }
    return;
}
#endif




