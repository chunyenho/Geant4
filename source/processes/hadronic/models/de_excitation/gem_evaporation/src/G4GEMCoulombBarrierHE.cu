#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4GEMCoulombBarrierHE.cc 67983 2013-03-13 10:42:03Z gcosmo $
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Dec 1999)

#include "G4GEMCoulombBarrierHE.hh"
#include "G4HadronicException.hh"
#include "G4Pow.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

G4GEMCoulombBarrierHE::G4GEMCoulombBarrierHE(G4int anA, G4int aZ) :
  G4VCoulombBarrier(anA,aZ) 
{}

G4GEMCoulombBarrierHE::~G4GEMCoulombBarrierHE() 
{}

G4double G4GEMCoulombBarrierHE::GetCoulombBarrier(G4int ARes, G4int ZRes, G4double U) const 
  // Calculation of Coulomb potential energy (barrier) for outgoing fragment
{
  G4double Barrier = 0.0;
  if (ZRes > ARes || ARes < 1) {
    G4cout << "G4GEMCoulombBarrierHE::GetCoulombBarrier: "
	   << "Wrong values for "
	   << "residual nucleus A = " << ARes << " "
	   << "and residual nucleus Z = " << ZRes << G4endl;
    throw G4HadronicException(__FILE__, __LINE__,"FATAL error");
  }
  if (GetZ() == 0) {
    Barrier = 0.0;   // If there is no charge there is neither barrier
  } else {
    G4double CompoundRadius = CalcCompoundRadius(ARes);
    Barrier = ( elm_coupling * GetZ() * ZRes)/(CompoundRadius+3.75*fermi);
    
    // Barrier penetration coeficient
    //    G4double K = BarrierPenetrationFactor(ZRes);
    //    Barrier *= K;
    
    Barrier /= (1.0 + std::sqrt(U/static_cast<G4double>(2*ARes)));
  }
  return Barrier;
}


G4double G4GEMCoulombBarrierHE::CalcCompoundRadius(G4int ARes) const
{
  G4Pow* g4pow = G4Pow::GetInstance();
  G4double AresOneThird = g4pow->Z13(ARes);
  G4double AejectOneThird = g4pow->Z13(GetA());

  G4double Result = 1.12*(AresOneThird + AejectOneThird) - 
    0.86*(AresOneThird+AejectOneThird)/(AresOneThird*AejectOneThird);

  return Result*fermi;
}


