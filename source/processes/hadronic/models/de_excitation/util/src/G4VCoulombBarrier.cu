//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4VCoulombBarrier.cc 67983 2013-03-13 10:42:03Z gcosmo $
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Dec 1999)

#include "G4VCoulombBarrier.hh"
#include "G4HadronicException.hh"

G4VCoulombBarrier::G4VCoulombBarrier(G4int anA, G4int aZ)
{
  if (anA >= aZ && anA > 0) {
    theA = anA;
    theZ = aZ;
  } else {
    std::ostringstream errOs;
    G4cout << "G4VCoulombBarrier::G4VCoulombBarrier: "
	   << "Wrong values for "
	   << "A= " << anA << " "
	   << "and Z= " << aZ << G4endl;
    throw G4HadronicException(__FILE__, __LINE__,"FATAL Exception");
  }
}

G4VCoulombBarrier::~G4VCoulombBarrier()
{}

