#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4StatMFMicroPartition.cc 67983 2013-03-13 10:42:03Z gcosmo $
//
// by V. Lara
// --------------------------------------------------------------------

#include "G4StatMFMicroPartition.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4HadronicException.hh"

// Copy constructor
G4StatMFMicroPartition::G4StatMFMicroPartition(const G4StatMFMicroPartition & )
{
  throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMicroPartition::copy_constructor meant to not be accessable");
}

// Operators

G4StatMFMicroPartition & G4StatMFMicroPartition::
operator=(const G4StatMFMicroPartition & )
{
  throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMicroPartition::operator= meant to not be accessable");
  return *this;
}


G4bool G4StatMFMicroPartition::operator==(const G4StatMFMicroPartition & ) const
{
  //throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMicroPartition::operator== meant to not be accessable");
  return false;
}
 

G4bool G4StatMFMicroPartition::operator!=(const G4StatMFMicroPartition & ) const
{
  //throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMicroPartition::operator!= meant to not be accessable");
  return true;
}

void G4StatMFMicroPartition::CoulombFreeEnergy(G4int anA)
{
  // This Z independent factor in the Coulomb free energy 
  G4double  CoulombConstFactor = 1.0/std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1.0/3.0);
	
  CoulombConstFactor = elm_coupling * (3./5.) *
    (1. - CoulombConstFactor)/G4StatMFParameters::Getr0();

  // We use the aproximation Z_f ~ Z/A * A_f

  G4double ZA = G4double(theZ)/G4double(theA);
										
  if (anA == 0 || anA == 1) 
    {
      _theCoulombFreeEnergy.push_back(CoulombConstFactor*ZA*ZA);
    } 
  else if (anA == 2 || anA == 3 || anA == 4) 
    {
      // Z/A ~ 1/2
      _theCoulombFreeEnergy.push_back(CoulombConstFactor*0.5*std::pow(anA,5./3.));
    } 
  else  // anA > 4
    {
      _theCoulombFreeEnergy.push_back(CoulombConstFactor*ZA*ZA*std::pow(anA,5./3.));	
    }
}

G4double G4StatMFMicroPartition::GetCoulombEnergy(void)
{
  G4double  CoulombFactor = 1.0/
    std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1.0/3.0);	
			
  G4double CoulombEnergy = elm_coupling*(3./5.)*theZ*theZ*CoulombFactor/
    (G4StatMFParameters::Getr0()*std::pow(static_cast<G4double>(theA),1./3.));
	
  G4double ZA = G4double(theZ)/G4double(theA);
  for (unsigned int i = 0; i < _thePartition.size(); i++) 
    CoulombEnergy += _theCoulombFreeEnergy[i] - elm_coupling*(3./5.)*
      ZA*ZA*std::pow(static_cast<G4double>(_thePartition[i]),5./3.)/
      G4StatMFParameters::Getr0();
		
  return CoulombEnergy;
}

G4double G4StatMFMicroPartition::GetPartitionEnergy(G4double T)
{
  G4double  CoulombFactor = 1.0/
    std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1.0/3.0);	
  
  G4double PartitionEnergy = 0.0;
  
  // We use the aprox that Z_f ~ Z/A * A_f
  for (unsigned int i = 0; i < _thePartition.size(); i++) 
    {
      if (_thePartition[i] == 0 || _thePartition[i] == 1) 
        {	
          PartitionEnergy += _theCoulombFreeEnergy[i];
        }
      else if (_thePartition[i] == 2) 
        {		
          PartitionEnergy +=	
            -2.796 // Binding Energy of deuteron ??????
            + _theCoulombFreeEnergy[i];		
	}
      else if (_thePartition[i] == 3) 
        {	
          PartitionEnergy +=	
            -9.224 // Binding Energy of trtion/He3 ??????
            + _theCoulombFreeEnergy[i];		
	} 
      else if (_thePartition[i] == 4) 
        {	
          PartitionEnergy +=
            -30.11 // Binding Energy of ALPHA ??????
            + _theCoulombFreeEnergy[i] 
            + 4.*T*T/InvLevelDensity(4.);
	} 
      else 
        {											
          PartitionEnergy +=
            //Volume term						
            (- G4StatMFParameters::GetE0() + 
             T*T/InvLevelDensity(_thePartition[i]))
            *_thePartition[i] + 
            
            // Symmetry term
            G4StatMFParameters::GetGamma0()*
            (1.0-2.0*theZ/theA)*(1.0-2.0*theZ/theA)*_thePartition[i] +  
            
            // Surface term
            (G4StatMFParameters::Beta(T) - T*G4StatMFParameters::DBetaDT(T))*
            std::pow(static_cast<G4double>(_thePartition[i]),2./3.) +
            
            // Coulomb term 
            _theCoulombFreeEnergy[i];
	}
    }
	
  PartitionEnergy += elm_coupling*(3./5.)*theZ*theZ*CoulombFactor/
    (G4StatMFParameters::Getr0()*std::pow(static_cast<G4double>(theA),1./3.))
    + (3./2.)*T*(_thePartition.size()-1);
  
  return PartitionEnergy;
}

G4double G4StatMFMicroPartition::CalcPartitionTemperature(G4double U,
							  G4double FreeInternalE0)
{
  G4double PartitionEnergy = GetPartitionEnergy(0.0);
  
  // If this happens, T = 0 MeV, which means that probability for this
  // partition will be 0
  if (std::fabs(U + FreeInternalE0 - PartitionEnergy) < 0.003) return -1.0;
    
  // Calculate temperature by midpoint method
	
  // Bracketing the solution
  G4double Ta = 0.001;
  G4double Tb = std::max(std::sqrt(8.0*U/theA),0.0012*MeV);
  G4double Tmid = 0.0;
  
  G4double Da = (U + FreeInternalE0 - GetPartitionEnergy(Ta))/U;
  G4double Db = (U + FreeInternalE0 - GetPartitionEnergy(Tb))/U;
  
  G4int maxit = 0;
  while (Da*Db > 0.0 && maxit < 1000) 
    {
      ++maxit;
      Tb += 0.5*Tb; 	
      Db = (U + FreeInternalE0 - GetPartitionEnergy(Tb))/U;
    }
  
  G4double eps = 1.0e-14*std::abs(Ta-Tb);
  
  for (G4int i = 0; i < 1000; i++) 
    {
      Tmid = (Ta+Tb)/2.0;
      if (std::fabs(Ta-Tb) <= eps) return Tmid;
      G4double Dmid = (U + FreeInternalE0 - GetPartitionEnergy(Tmid))/U;
      if (std::fabs(Dmid) < 0.003) return Tmid;
      if (Da*Dmid < 0.0) 
        {
          Tb = Tmid;
          Db = Dmid;
        } 
      else 
        {
          Ta = Tmid;
          Da = Dmid;
        } 
    }
  // if we arrive here the temperature could not be calculated
  G4cerr << "G4StatMFMicroPartition::CalcPartitionTemperature: I can't calculate the temperature"  
         << G4endl;
  // and set probability to 0 returning T < 0
  return -1.0;
  
}

G4double G4StatMFMicroPartition::CalcPartitionProbability(G4double U,
							  G4double FreeInternalE0,
							  G4double SCompound)
{	
  G4double T = CalcPartitionTemperature(U,FreeInternalE0);
  if ( T <= 0.0) return _Probability = 0.0;
  _Temperature = T;
  
  
  // Factorial of fragment multiplicity
  G4double Fact = 1.0;
  unsigned int i;
  for (i = 0; i < _thePartition.size() - 1; i++) 
    {
      G4double f = 1.0;
      for (unsigned int ii = i+1; i< _thePartition.size(); i++) 
        {
          if (_thePartition[i] == _thePartition[ii]) f++;
        }
      Fact *= f;
  }
	
  G4double ProbDegeneracy = 1.0;
  G4double ProbA32 = 1.0;	
	
  for (i = 0; i < _thePartition.size(); i++) 
    {
      ProbDegeneracy *= GetDegeneracyFactor(static_cast<G4int>(_thePartition[i]));
      ProbA32 *= static_cast<G4double>(_thePartition[i])*
        std::sqrt(static_cast<G4double>(_thePartition[i]));
    }
	
  // Compute entropy
  G4double PartitionEntropy = 0.0;
  for (i = 0; i < _thePartition.size(); i++) 
    {
      // interaction entropy for alpha
      if (_thePartition[i] == 4) 
        {
          PartitionEntropy += 
            2.0*T*_thePartition[i]/InvLevelDensity(_thePartition[i]);
        }
      // interaction entropy for Af > 4
      else if (_thePartition[i] > 4) 
        {
          PartitionEntropy += 
            2.0*T*_thePartition[i]/InvLevelDensity(_thePartition[i])
            - G4StatMFParameters::DBetaDT(T)
            * std::pow(static_cast<G4double>(_thePartition[i]),2.0/3.0);
        } 
    }
	
  // Thermal Wave Lenght = std::sqrt(2 pi hbar^2 / nucleon_mass T)
  G4double ThermalWaveLenght3 = 16.15*fermi/std::sqrt(T);
  ThermalWaveLenght3 = ThermalWaveLenght3*ThermalWaveLenght3*ThermalWaveLenght3;
  
  // Translational Entropy
  G4double kappa = (1. + elm_coupling*(std::pow(static_cast<G4double>(_thePartition.size()),1./3.)-1.0)
                    /(G4StatMFParameters::Getr0()*std::pow(static_cast<G4double>(theA),1./3.)));
  kappa = kappa*kappa*kappa;
  kappa -= 1.;
  G4double V0 = (4./3.)*pi*theA*G4StatMFParameters::Getr0()*G4StatMFParameters::Getr0()*
    G4StatMFParameters::Getr0();
  G4double FreeVolume = kappa*V0;
  G4double TranslationalS = std::max(0.0, std::log(ProbA32/Fact) +
                                     (_thePartition.size()-1.0)*std::log(FreeVolume/ThermalWaveLenght3) +
                                     1.5*(_thePartition.size()-1.0) - (3./2.)*std::log(G4double(theA)));
  
  PartitionEntropy += std::log(ProbDegeneracy) + TranslationalS;
  _Entropy = PartitionEntropy;
	
  // And finally compute probability of fragment configuration
  G4double exponent = PartitionEntropy-SCompound;
  if (exponent > 700.0) exponent = 700.0;
  return _Probability = std::exp(exponent);
}

G4double G4StatMFMicroPartition::GetDegeneracyFactor(G4int A)
{
  // Degeneracy factors are statistical factors
  // DegeneracyFactor for nucleon is (2S_n + 1)(2I_n + 1) = 4
  G4double DegFactor = 0;
  if (A > 4) DegFactor = 1.0;
  else if (A == 1) DegFactor = 4.0;     // nucleon
  else if (A == 2) DegFactor = 3.0;     // Deuteron
  else if (A == 3) DegFactor = 4.0;     // Triton + He3
  else if (A == 4) DegFactor = 1.0;     // alpha
  return DegFactor;
}

G4StatMFChannel * G4StatMFMicroPartition::ChooseZ(G4int A0, G4int Z0, G4double MeanT)
// Gives fragments charges
{
  std::vector<G4int> FragmentsZ;
  
  G4int ZBalance = 0;
  do 
    {
      G4double CC = G4StatMFParameters::GetGamma0()*8.0;
      G4int SumZ = 0;
      for (unsigned int i = 0; i < _thePartition.size(); i++) 
        {
          G4double ZMean;
          G4double Af = _thePartition[i];
          if (Af > 1.5 && Af < 4.5) ZMean = 0.5*Af;
          else ZMean = Af*Z0/A0;
          G4double ZDispersion = std::sqrt(Af * MeanT/CC);
          G4int Zf;
          do 
            {
              Zf = static_cast<G4int>(G4RandGauss::shoot(ZMean,ZDispersion));
	    } 
          while (Zf < 0 || Zf > Af);
          FragmentsZ.push_back(Zf);
          SumZ += Zf;
	}
      ZBalance = Z0 - SumZ;
    } 
  while (std::abs(ZBalance) > 1);
  FragmentsZ[0] += ZBalance;
	
  G4StatMFChannel * theChannel = new G4StatMFChannel;
  for (unsigned int i = 0; i < _thePartition.size(); i++)
    {
      theChannel->CreateFragment(_thePartition[i],FragmentsZ[i]);
    }

  return theChannel;
}
