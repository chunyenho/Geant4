#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4StatMFMacroBiNucleon.cc 67983 2013-03-13 10:42:03Z gcosmo $
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara

#include "G4StatMFMacroBiNucleon.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

// Operators

G4StatMFMacroBiNucleon & G4StatMFMacroBiNucleon::
operator=(const G4StatMFMacroBiNucleon & )
{
    throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMacroBiNucleon::operator= meant to not be accessable");
    return *this;
}


G4bool G4StatMFMacroBiNucleon::operator==(const G4StatMFMacroBiNucleon & ) const
{
    throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMacroBiNucleon::operator== meant to not be accessable");
    return false;
}
 

G4bool G4StatMFMacroBiNucleon::operator!=(const G4StatMFMacroBiNucleon & ) const
{
    throw G4HadronicException(__FILE__, __LINE__, "G4StatMFMacroBiNucleon::operator!= meant to not be accessable");
    return true;
}


G4double G4StatMFMacroBiNucleon::CalcMeanMultiplicity(const G4double FreeVol, const G4double mu, 
						      const G4double nu, const G4double T)
{
    const G4double ThermalWaveLenght = 16.15*fermi/std::sqrt(T);
	
    const G4double lambda3 = ThermalWaveLenght*ThermalWaveLenght*ThermalWaveLenght;
    
    const G4double degeneracy = 3.0;
    
    const G4double Coulomb = (3./5.)*(elm_coupling/G4StatMFParameters::Getr0())*
	(1.0 - 1.0/std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1./3.));
    
    const G4double BindingE = G4NucleiProperties::GetBindingEnergy(theA,1); //old value was 2.796*MeV
    G4double exponent = (BindingE + theA*(mu+nu*theZARatio) - 
			 Coulomb*theZARatio*theZARatio*std::pow(G4double(theA),5./3.))/T;

    // To avoid numerical problems
    if (exponent < -700.0) exponent = -700.0;
    else if (exponent > 700.0) exponent = 700.0;

    _MeanMultiplicity = (degeneracy*FreeVol*static_cast<G4double>(theA)*std::sqrt(static_cast<G4double>(theA))/lambda3)*
	std::exp(exponent);
			 
    return _MeanMultiplicity;
}


G4double G4StatMFMacroBiNucleon::CalcEnergy(const G4double T)
{
    const G4double Coulomb = (3./5.)*(elm_coupling/G4StatMFParameters::Getr0())*
	(1.0 - 1.0/std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1./3.));
									
    _Energy  = -G4NucleiProperties::GetBindingEnergy(theA,1) + 
	Coulomb * theZARatio * theZARatio * std::pow(G4double(theA),5./3.) +
	(3./2.) * T;
							
    return 	_Energy;				
}



G4double G4StatMFMacroBiNucleon::CalcEntropy(const G4double T, const G4double FreeVol)
{
    const G4double ThermalWaveLenght = 16.15*fermi/std::sqrt(T);
    const G4double lambda3 = ThermalWaveLenght*ThermalWaveLenght*ThermalWaveLenght;

    G4double Entropy = 0.0;
    if (_MeanMultiplicity > 0.0)
	// Is this formula correct?
	Entropy = _MeanMultiplicity*(5./2.+
				     std::log(3.0*static_cast<G4double>(theA)*
					 std::sqrt(static_cast<G4double>(theA))*FreeVol/
					 (lambda3*_MeanMultiplicity)));
								
								
    return Entropy;
}
