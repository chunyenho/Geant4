#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4StatMFMacroTriNucleon.cc 68724 2013-04-05 09:26:32Z gcosmo $
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara

#include "G4StatMFMacroTriNucleon.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

G4StatMFMacroTriNucleon::G4StatMFMacroTriNucleon() 
  : G4VStatMFMacroCluster(3) 
{}

G4StatMFMacroTriNucleon::~G4StatMFMacroTriNucleon() 
{}

G4double 
G4StatMFMacroTriNucleon::CalcMeanMultiplicity(const G4double FreeVol, 
					      const G4double mu, 
					      const G4double nu, 
					      const G4double T)
{
  G4double ThermalWaveLenght = 16.15*fermi/std::sqrt(T);
  G4double lambda3 = ThermalWaveLenght*ThermalWaveLenght*ThermalWaveLenght;
  static const G4double degeneracy = 2.0+2.0;  // H3 + He3
  G4double Coulomb = (3./5.)*(elm_coupling/G4StatMFParameters::Getr0())*
    (1.0 - 1.0/std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1./3.));

  // old value was 9.224*MeV
  G4double BindingE = G4NucleiProperties::GetBindingEnergy(theA,1); 
  //	  + G4NucleiProperties::GetBindingEnergy(theA,2);

  G4double exponent = (BindingE+ theA*(mu+nu*theZARatio) - 
		       Coulomb*theZARatio*theZARatio
		       *std::pow(static_cast<G4double>(theA),5./3.))/T;
  if (exponent > 700.0) exponent = 700.0;

  _MeanMultiplicity = (degeneracy*FreeVol*theA*
		       std::sqrt(static_cast<G4double>(theA))/lambda3)*
    std::exp(exponent);
			 
  return _MeanMultiplicity;
}

G4double G4StatMFMacroTriNucleon::CalcEnergy(const G4double T)
{
  G4double Coulomb = (3./5.)*(elm_coupling/G4StatMFParameters::Getr0())*
    (1.0 - 1.0/std::pow(1.0+G4StatMFParameters::GetKappaCoulomb(),1./3.));
									
  return _Energy  = -G4NucleiProperties::GetBindingEnergy(theA,1) + 
    Coulomb * theZARatio * theZARatio 
    * std::pow(static_cast<G4double>(theA),5./3.) + (3./2.) * T;
}

G4double 
G4StatMFMacroTriNucleon::CalcEntropy(const G4double T, const G4double FreeVol)
{
  G4double ThermalWaveLenght = 16.15*fermi/std::sqrt(T);
  G4double lambda3 = ThermalWaveLenght*ThermalWaveLenght*ThermalWaveLenght;

  G4double Entropy = 0.0;
  if (_MeanMultiplicity > 0.0)
    Entropy = _MeanMultiplicity*(2.5 + std::log((4*theA)*
        std::sqrt(static_cast<G4double>(theA))*FreeVol
        /(lambda3*_MeanMultiplicity)));
								
  return Entropy;
}
