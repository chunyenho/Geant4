#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//J.M. Quesada (August2008). Based on:
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Oct 1998)
//
// Modif (03 September 2008) by J. M. Quesada for external choice of inverse 
// cross section option
// JMQ (06 September 2008) Also external choices have been added for 
// superimposed Coulomb barrier (if useSICB is set true, by default is false) 
//
// JMQ (14 february 2009) bug fixed in emission width: hbarc instead of hbar_Planck in the denominator
//
#include <iostream>

#include "G4EvaporationProbability.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4PairingCorrection.hh"
#include "G4ParticleTable.hh"
#include "G4IonTable.hh"

using namespace std;

G4EvaporationProbability::G4EvaporationProbability(G4int anA, G4int aZ, 
						   G4double aGamma,
						   G4VCoulombBarrier * aCoulombBarrier) 
  : theA(anA),
    theZ(aZ),
    Gamma(aGamma),
    theCoulombBarrierptr(aCoulombBarrier) 
{}

G4EvaporationProbability::G4EvaporationProbability()
  : theA(0),
    theZ(0),
    Gamma(0.0),
    theCoulombBarrierptr(0) 
{}

G4EvaporationProbability::~G4EvaporationProbability() 
{}
  
G4double 
G4EvaporationProbability::EmissionProbability(const G4Fragment & fragment, 
					      G4double anEnergy)
{
  G4double probability = 0.0;

  if (anEnergy > 0.0 && fragment.GetExcitationEnergy() > 0.0) {
    probability = CalculateProbability(fragment, anEnergy);

  }
  return probability;
}

////////////////////////////////////

// Computes the integrated probability of evaporation channel
G4double 
G4EvaporationProbability::CalculateProbability(const G4Fragment & fragment, 
					       G4double MaximalKineticEnergy)
{
  G4int ResidualA = fragment.GetA_asInt() - theA;
  G4int ResidualZ = fragment.GetZ_asInt() - theZ;
  G4double U = fragment.GetExcitationEnergy();
   
  if (OPTxs==0) {

    G4double NuclearMass = fragment.ComputeGroundStateMass(theZ,theA);

    G4double delta0 = fPairCorr->GetPairingCorrection(fragment.GetA_asInt(),
						      fragment.GetZ_asInt());

    G4double SystemEntropy = 2.0*std::sqrt(
      theEvapLDPptr->LevelDensityParameter(fragment.GetA_asInt(),fragment.GetZ_asInt(),U)*
      (U-delta0));
								  
    static const G4double RN = 1.5*fermi;

    G4double Alpha = CalcAlphaParam(fragment);
    G4double Beta = CalcBetaParam(fragment);
	
    G4double Rmax = MaximalKineticEnergy;
    G4double a = theEvapLDPptr->LevelDensityParameter(ResidualA,ResidualZ,Rmax);
    G4double GlobalFactor = Gamma * Alpha/(a*a) *
	(NuclearMass*RN*RN*fG4pow->Z23(ResidualA))/
	(twopi* hbar_Planck*hbar_Planck);
    G4double Term1 = (2.0*Beta*a-3.0)/2.0 + Rmax*a;
    G4double Term2 = (2.0*Beta*a-3.0)*std::sqrt(Rmax*a) + 2.0*a*Rmax;
	
    G4double ExpTerm1 = 0.0;
    if (SystemEntropy <= 600.0) { ExpTerm1 = std::exp(-SystemEntropy); }
	
    G4double ExpTerm2 = 2.*std::sqrt(a*Rmax) - SystemEntropy;
    if (ExpTerm2 > 700.0) { ExpTerm2 = 700.0; }
    ExpTerm2 = std::exp(ExpTerm2);
	
    G4double Width = GlobalFactor*(Term1*ExpTerm1 + Term2*ExpTerm2);
	
    return Width;
             
 } else if (OPTxs==1 || OPTxs==2 ||OPTxs==3 || OPTxs==4) {

   G4double EvaporatedMass = fragment.ComputeGroundStateMass(theZ,theA);
   G4double ResidulalMass = fragment.ComputeGroundStateMass(ResidualZ,ResidualA);
   G4double limit = std::max(0.0,fragment.GetGroundStateMass()-EvaporatedMass-ResidulalMass);
   if (useSICB) {
     limit = std::max(limit,theCoulombBarrierptr->GetCoulombBarrier(ResidualA,ResidualZ,U));
   }

   if (MaximalKineticEnergy <= limit) { return 0.0; }

   // if Coulomb barrier cutoff is superimposed for all cross sections 
   // then the limit is the Coulomb Barrier
   G4double LowerLimit= limit;

   //MaximalKineticEnergy: asimptotic value (already accounted for in G4EvaporationChannel)     

   G4double UpperLimit = MaximalKineticEnergy;

   G4double Width = IntegrateEmissionProbability(fragment,LowerLimit,UpperLimit);

   return Width;
 } else {
   std::ostringstream errOs;
   errOs << "Bad option for cross sections at evaporation"  <<G4endl;
   throw G4HadronicException(__FILE__, __LINE__, errOs.str());
 }
  
}

/////////////////////////////////////////////////////////////////////

G4double G4EvaporationProbability::
IntegrateEmissionProbability(const G4Fragment & fragment, 
			     const G4double & Low, const G4double & Up )
{
  static const G4int N = 10;
  // 10-Points Gauss-Legendre abcisas and weights
  static const G4double w[N] = {
    0.0666713443086881,
    0.149451349150581,
    0.219086362515982,
    0.269266719309996,
    0.295524224714753,
    0.295524224714753,
    0.269266719309996,
    0.219086362515982,
    0.149451349150581,
    0.0666713443086881
  };
  static const G4double x[N] = {
    -0.973906528517172,
    -0.865063366688985,
    -0.679409568299024,
    -0.433395394129247,
    -0.148874338981631,
    0.148874338981631,
    0.433395394129247,
    0.679409568299024,
    0.865063366688985,
    0.973906528517172
  };

  G4double Total = 0.0;


  for (G4int i = 0; i < N; i++) 
    {

      G4double KineticE = ((Up-Low)*x[i]+(Up+Low))/2.0;

      Total += w[i]*ProbabilityDistributionFunction(fragment, KineticE);

    }
  Total *= (Up-Low)/2.0;
  return Total;
}


/////////////////////////////////////////////////////////
//New method (OPT=1,2,3,4)

G4double 
G4EvaporationProbability::ProbabilityDistributionFunction( const G4Fragment & fragment, 
							   G4double K)
{ 
  G4int ResidualA = fragment.GetA_asInt() - theA;
  G4int ResidualZ = fragment.GetZ_asInt() - theZ;  
  G4double U = fragment.GetExcitationEnergy();
  //G4cout << "### G4EvaporationProbability::ProbabilityDistributionFunction" << G4endl;
  //G4cout << "FragZ= " << fragment.GetZ_asInt() << " FragA= " << fragment.GetA_asInt()
  //	 << " Z= " << theZ << "  A= " << theA << G4endl;
  //G4cout << "PC " << fPairCorr << "   DP " << theEvapLDPptr << G4endl;

  // if(K <= theCoulombBarrierptr->GetCoulombBarrier(ResidualA,ResidualZ,U)) return 0.0;

  G4double delta1 = fPairCorr->GetPairingCorrection(ResidualA,ResidualZ);
 
  G4double delta0 = fPairCorr->GetPairingCorrection(fragment.GetA_asInt(),
						    fragment.GetZ_asInt());

  
  G4double ParticleMass = fragment.ComputeGroundStateMass(theZ,theA);
  G4double ResidualMass = fragment.ComputeGroundStateMass(ResidualZ,ResidualA);

  G4double theSeparationEnergy = ParticleMass + ResidualMass 
    - fragment.GetGroundStateMass();

  G4double a0 = theEvapLDPptr->LevelDensityParameter(fragment.GetA_asInt(),
						     fragment.GetZ_asInt(),
						     U - delta0);

  G4double a1 = theEvapLDPptr->LevelDensityParameter(ResidualA, ResidualZ,
						     U - theSeparationEnergy - delta1);
  
  
  G4double E0 = U - delta0;

  G4double E1 = U - theSeparationEnergy - delta1 - K;

  if (E1<0.) { return 0.; }

  //JMQ 14/02/09 BUG fixed: hbarc should be in the denominator instead of hbar_Planck 
  //Without 1/hbar_Panck remains as a width

  //G4double Prob=Gamma*ParticleMass/((pi*hbarc)*(pi*hbarc)*std::exp(2*std::sqrt(a0*E0)))
  //  *K*CrossSection(fragment,K)*std::exp(2*std::sqrt(a1*E1))*millibarn;

  static const G4double pcoeff = millibarn/((pi*hbarc)*(pi*hbarc)); 

  // Fixed numerical problem
  G4double Prob = pcoeff*Gamma*ParticleMass*std::exp(2*(std::sqrt(a1*E1) - std::sqrt(a0*E0)))
    *K*CrossSection(fragment,K);

  return Prob;
}


