#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4TritonEvaporationProbability.cc 67983 2013-03-13 10:42:03Z gcosmo $
//
// J.M. Quesada (August2008). Based on:
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Oct 1998)
//
// Modified:
// 03-09-2008 J.M. Quesada for external choice of inverse cross section option
// 17-11-2010 V.Ivanchenko integer Z and A

#include "G4TritonEvaporationProbability.hh"
#include "G4SystemOfUnits.hh"

G4TritonEvaporationProbability::G4TritonEvaporationProbability() :
    G4EvaporationProbability(3,1,2,&theCoulombBarrier) // A,Z,Gamma,&theCoulombBarrier
{
  ResidualA = ResidualZ = theA = theZ = FragmentA = 0;
  ResidualAthrd = FragmentAthrd = 0.0;
}

G4TritonEvaporationProbability::~G4TritonEvaporationProbability()
{}

G4double G4TritonEvaporationProbability::CalcAlphaParam(const G4Fragment & fragment) 
{ 
  return 1.0 + CCoeficient(fragment.GetZ_asInt()-GetZ());
}
	
G4double G4TritonEvaporationProbability::CalcBetaParam(const G4Fragment & ) 
{ 
  return 0.0; 
}

G4double G4TritonEvaporationProbability::CCoeficient(G4int aZ) 
{
  // Data comes from 
  // Dostrovsky, Fraenkel and Friedlander
  // Physical Review, vol 116, num. 3 1959
  // 
  // const G4int size = 5;
  // G4double Zlist[5] = { 10.0, 20.0, 30.0, 50.0, 70.0};
  // G4double Cp[5] = { 0.50, 0.28, 0.20, 0.15, 0.10};
  // C for triton is equal to C for protons divided by 3
  G4double C = 0.0;
	
  if (aZ >= 70) {
    C = 0.10;
  } else {
    C = ((((0.15417e-06*aZ) - 0.29875e-04)*aZ + 0.21071e-02)*aZ - 0.66612e-01)*aZ + 0.98375;
  }
	
  return C/3.0;
}

///////////////////////////////////////////////////////////////////////////////////
//J. M. Quesada (Dec 2007-June 2008): New inverse reaction cross sections 
//OPT=0 Dostrovski's parameterization
//OPT=1,2 Chatterjee's paramaterization 
//OPT=3,4 Kalbach's parameterization 
// 
G4double 
G4TritonEvaporationProbability::CrossSection(const  G4Fragment & fragment, G4double K)
{
  theA=GetA();
  theZ=GetZ();
  ResidualA=fragment.GetA_asInt()-theA;
  ResidualZ=fragment.GetZ_asInt()-theZ; 
  
  ResidualAthrd=fG4pow->Z13(ResidualA);
  FragmentA=fragment.GetA_asInt();
  FragmentAthrd=fG4pow->Z13(FragmentA);

  if (OPTxs==0) {std::ostringstream errOs;
    errOs << "We should'n be here (OPT =0) at evaporation cross section calculation (tritons)!!"  
	  <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
    return 0.;}
  if( OPTxs==1 || OPTxs==2) return G4TritonEvaporationProbability::GetOpt12( K);
  else if (OPTxs==3 || OPTxs==4)  return G4TritonEvaporationProbability::GetOpt34( K);
  else{
    std::ostringstream errOs;
    errOs << "BAD Triton CROSS SECTION OPTION AT EVAPORATION!!"  <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
    return 0.;
  }
}

//
//********************* OPT=1,2 : Chatterjee's cross section *****************
//(fitting to cross section from Bechetti & Greenles OM potential)

G4double G4TritonEvaporationProbability::GetOpt12(G4double K)
{
  G4double Kc=K;

  // JMQ xsec is set constat above limit of validity
  if (K > 50*MeV) { Kc=50*MeV; }

  G4double landa ,mu ,nu ,p , Ec,q,r,ji,xs;
 
  G4double    p0 = -11.04;
  G4double    p1 = 619.1;
  G4double    p2 = -2147.;
  G4double    landa0 = -0.0426;
  G4double    landa1 = -10.33;
  G4double    mum0 = 601.9;
  G4double    mu1 = 0.37;
  G4double    nu0 = 583.0;
  G4double    nu1 = -546.2;
  G4double    nu2 = 1.718;  
  G4double    delta=1.2;            

  Ec = 1.44*theZ*ResidualZ/(1.5*ResidualAthrd+delta);
  p = p0 + p1/Ec + p2/(Ec*Ec);
  landa = landa0*ResidualA + landa1;

  G4double resmu1 = fG4pow->powZ(ResidualA,mu1); 
  mu = mum0*resmu1;
  nu = resmu1*(nu0 + nu1*Ec + nu2*(Ec*Ec));
  q = landa - nu/(Ec*Ec) - 2*p*Ec;
  r = mu + 2*nu/Ec + p*(Ec*Ec);
  
  ji=std::max(Kc,Ec);
  if(Kc < Ec) { xs = p*Kc*Kc + q*Kc + r;}
  else {xs = p*(Kc - ji)*(Kc - ji) + landa*Kc + mu + nu*(2 - Kc/ji)/ji ;}
                 
  if (xs <0.0) {xs=0.0;}
              
  return xs;
}

// *********** OPT=3,4 : Kalbach's cross sections (from PRECO code)*************
G4double G4TritonEvaporationProbability::GetOpt34(G4double K)
//     ** t from o.m. of hafele, flynn et al
{
  G4double landa, mu, nu, p , signor(1.),sig;
  G4double ec,ecsq,xnulam,etest(0.),a; 
  G4double b,ecut,cut,ecut2,geom,elab;

  G4double     flow = 1.e-18;
  G4double     spill= 1.e+18;

  G4double     p0 = -21.45;
  G4double     p1 = 484.7;
  G4double     p2 = -1608.;
  G4double     landa0 = 0.0186;
  G4double     landa1 = -8.90;
  G4double     mum0 = 686.3;
  G4double     mu1 = 0.325;
  G4double     nu0 = 368.9;
  G4double     nu1 = -522.2;
  G4double     nu2 = -4.998;  
  
  G4double      ra=0.80;
        
  //JMQ 13/02/09 increase of reduced radius to lower the barrier
  // ec = 1.44 * theZ * ResidualZ / (1.5*ResidualAthrd+ra);
  ec = 1.44 * theZ * ResidualZ / (1.7*ResidualAthrd+ra);
  ecsq = ec * ec;
  p = p0 + p1/ec + p2/ecsq;
  landa = landa0*ResidualA + landa1;
  a = fG4pow->powZ(ResidualA,mu1);
  mu = mum0 * a;
  nu = a* (nu0+nu1*ec+nu2*ecsq);  
  xnulam = nu / landa;
  if (xnulam > spill) { xnulam=0.; }
  if (xnulam >= flow) { etest = 1.2 *std::sqrt(xnulam); }
 
  a = -2.*p*ec + landa - nu/ecsq;
  b = p*ecsq + mu + 2.*nu/ec;
  ecut = 0.;
  cut = a*a - 4.*p*b;
  if (cut > 0.) { ecut = std::sqrt(cut); }
  ecut = (ecut-a) / (p+p);
  ecut2 = ecut;
  //JMQ 290310 for avoiding unphysical increase below minimum (at ecut)
  // ecut<0 means that there is no cut with energy axis, i.e. xs is set 
  // to 0 bellow minimum
  //  if (cut < 0.) ecut2 = ecut - 2.;
  if (cut < 0.) { ecut2 = ecut; }
  elab = K * FragmentA / G4double(ResidualA);
  sig = 0.;
 
  if (elab <= ec) { //start for E<Ec
    if (elab > ecut2) { sig = (p*elab*elab+a*elab+b) * signor; }
  }           //end for E<Ec
  else {           //start for E>Ec
    sig = (landa*elab+mu+nu/elab) * signor;
    geom = 0.;
    if (xnulam < flow || elab < etest) { return sig; }
    geom = std::sqrt(theA*K);
    geom = 1.23*ResidualAthrd + ra + 4.573/geom;
    geom = 31.416 * geom * geom;
    sig = std::max(geom,sig);
  }           //end for E>Ec
  return sig;
}

