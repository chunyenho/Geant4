#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4ProtonEvaporationProbability.cc 74869 2013-10-23 09:26:17Z gcosmo $
//
// J.M. Quesada (August2008). Based on:
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Oct 1998)
//
// Modified:
// 03-09-2008 J.M. Quesada for external choice of inverse cross section option
// 17-11-2010 V.Ivanchenko integer Z and A

#include "G4ProtonEvaporationProbability.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4Log.hh"
#include "G4Exp.hh"

G4ProtonEvaporationProbability::G4ProtonEvaporationProbability() :
    G4EvaporationProbability(1,1,2,&theCoulombBarrier) // A,Z,Gamma,&theCoulombBarrier
{
  ResidualA = ResidualZ = theA = theZ = FragmentA = 0;
  ResidualAthrd = FragmentAthrd = U = 0.0;
}

G4ProtonEvaporationProbability::~G4ProtonEvaporationProbability() 
{}

G4double G4ProtonEvaporationProbability::CalcAlphaParam(const G4Fragment & fragment) 
  { return 1.0 + CCoeficient(fragment.GetZ_asInt()-GetZ());}
	
G4double G4ProtonEvaporationProbability::CalcBetaParam(const G4Fragment & )  
  { return 0.0; }

G4double G4ProtonEvaporationProbability::CCoeficient(G4int aZ) 
{
  // Data comes from 
  // Dostrovsky, Fraenkel and Friedlander
  // Physical Review, vol 116, num. 3 1959
  // 
  // const G4int size = 5;
  // G4double Zlist[5] = { 10.0, 20.0, 30.0, 50.0, 70.0};
  // G4double Cp[5] = { 0.50, 0.28, 0.20, 0.15, 0.10};
  G4double C = 0.0;
	
  if (aZ >= 70) {
    C = 0.10;
  } else {
    C = ((((0.15417e-06*aZ) - 0.29875e-04)*aZ + 0.21071e-02)*aZ - 0.66612e-01)*aZ + 0.98375;
  }
	
  return C;
	
}

///////////////////////////////////////////////////////////////////////////////////
//J. M. Quesada (Dec 2007-June 2008): New inverse reaction cross sections for protons
//OPT=0 Dostrovski's parameterization
//OPT=1 Chatterjee's parameterization 
//OPT=2,4 Wellisch's parameterization 
//OPT=3 Kalbach's parameterization
// 
G4double 
G4ProtonEvaporationProbability::CrossSection(const  G4Fragment & fragment, G4double K)
{
  //  G4cout<<" In G4ProtonEVaporationProbability OPTxs="<<OPTxs<<G4endl;
  //  G4cout<<" In G4ProtonEVaporationProbability useSICB="<<useSICB<<G4endl;

  theA=GetA();
  theZ=GetZ();
  ResidualA=fragment.GetA_asInt()-theA;
  ResidualZ=fragment.GetZ_asInt()-theZ; 
  
  ResidualAthrd=fG4pow->Z13(ResidualA);
  FragmentA=fragment.GetA_asInt();
  FragmentAthrd=fG4pow->Z13(FragmentA);

  U=fragment.GetExcitationEnergy();

  if (OPTxs==0) {std::ostringstream errOs;
    errOs << "We should'n be here (OPT =0) at evaporation cross section calculation (protons)!!"  <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
    return 0.;}
  else if( OPTxs==1 ) return GetOpt1( K);
  else if( OPTxs==2 ||OPTxs==4) return GetOpt2( K);
  else if (OPTxs==3 )  return GetOpt3( K);
  else{
    std::ostringstream errOs;
    errOs << "BAD PROTON CROSS SECTION OPTION AT EVAPORATION!!"  <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
    return 0.;
  }
}

//********************* OPT=1 : Chatterjee's cross section *********************
//(fitting to cross section from Bechetti & Greenles OM potential)

G4double G4ProtonEvaporationProbability::GetOpt1(G4double K)
{
  G4double Kc=K; 

  // JMQ  xsec is set constat above limit of validity
  if (K > 50*MeV) { Kc = 50*MeV; }

  G4double landa, landa0, landa1, mu, mum0, mu1,nu, nu0, nu1, nu2,xs;
  G4double p, p0, p1, p2,Ec,delta,q,r,ji;
  
  p0 = 15.72;
  p1 = 9.65;
  p2 = -449.0;
  landa0 = 0.00437;
  landa1 = -16.58;
  mum0 = 244.7;
  mu1 = 0.503;
  nu0 = 273.1;
  nu1 = -182.4;
  nu2 = -1.872;  
  delta=0.;  

  Ec = 1.44*theZ*ResidualZ/(1.5*ResidualAthrd+delta);
  p = p0 + p1/Ec + p2/(Ec*Ec);
  landa = landa0*ResidualA + landa1;

  G4double resmu1 = fG4pow->powZ(ResidualA,mu1); 
  mu = mum0*resmu1;
  nu = resmu1*(nu0 + nu1*Ec + nu2*(Ec*Ec));
  q = landa - nu/(Ec*Ec) - 2*p*Ec;
  r = mu + 2*nu/Ec + p*(Ec*Ec);

  ji=std::max(Kc,Ec);
  if(Kc < Ec) { xs = p*Kc*Kc + q*Kc + r;}
  else {xs = p*(Kc - ji)*(Kc - ji) + landa*Kc + mu + nu*(2 - Kc/ji)/ji ;}
  if (xs <0.0) {xs=0.0;}

  return xs; 
}

//************* OPT=2 : Welisch's proton reaction cross section ***************

G4double G4ProtonEvaporationProbability::GetOpt2(G4double K)
{

  G4double eekin,ekin,ff1,ff2,ff3,r0,fac,fac1,fac2,b0,xine_th(0);
 
  // This is redundant when the Coulomb  barrier is overimposed to all 
  // cross sections 
  // It should be kept when Coulomb barrier only imposed at OPTxs=2

  if(!useSICB && K<=theCoulombBarrier.GetCoulombBarrier(ResidualA,ResidualZ,U)) 
    { return 0.0; }

  eekin=K;
  G4int rnneu=ResidualA-ResidualZ;
  ekin=eekin/1000;
  r0=1.36*1.e-15;
  fac=pi*r0*r0;
  b0=2.247-0.915*(1.-1./ResidualAthrd);
  fac1=b0*(1.-1./ResidualAthrd);
  fac2=1.;
  if(rnneu > 1.5) { fac2 = fG4pow->logZ(rnneu); }
  xine_th= 1.e+31*fac*fac2*(1.+ResidualAthrd-fac1);
  xine_th=(1.-0.15*G4Exp(-ekin))*xine_th/(1.00-0.0007*ResidualA);	
  ff1=0.70-0.0020*ResidualA;
  ff2=1.00+1/G4double(ResidualA);
  ff3=0.8+18/G4double(ResidualA)-0.002*ResidualA;
  G4double log10E = G4Log(ekin)/fG4pow->logZ(10);
  fac=1.-(1./(1.+G4Exp(-8.*ff1*(log10E + 1.37*ff2))));
  xine_th=xine_th*(1.+ff3*fac);
  ff1=1.-1/G4double(ResidualA)-0.001*ResidualA;
  ff2=1.17-2.7/G4double(ResidualA)-0.0014*ResidualA;
  fac=-8.*ff1*(log10E + 2.0*ff2);
  fac=1./(1.+G4Exp(fac));
  xine_th=xine_th*fac;            
  if (xine_th < 0.0){
    std::ostringstream errOs;
    G4cout<<"WARNING:  negative Wellisch cross section "<<G4endl; 
    errOs << "RESIDUAL: A=" << ResidualA << " Z=" << ResidualZ <<G4endl;
    errOs <<"  xsec("<<ekin<<" MeV) ="<<xine_th <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
  }
  return xine_th;
}

// *********** OPT=3 : Kalbach's cross sections (from PRECO code)*************
G4double G4ProtonEvaporationProbability::GetOpt3(const  G4double K)
{
  //     ** p from  becchetti and greenlees (but modified with sub-barrier
  //     ** correction function and xp2 changed from -449)

  G4double landa, landa0, landa1, mu, mum0, mu1,nu, nu0, nu1, nu2;
  G4double p, p0, p1, p2;
  p0 = 15.72;
  p1 = 9.65;
  p2 = -300.;
  landa0 = 0.00437;
  landa1 = -16.58;
  mum0 = 244.7;
  mu1 = 0.503;
  nu0 = 273.1;
  nu1 = -182.4;
  nu2 = -1.872;
  
  // parameters for  proton cross section refinement 
  /*
  G4double afit,bfit,a2,b2;
  afit=-0.0785656;
  bfit=5.10789;
  a2= -0.00089076;
  b2= 0.0231597;  
  */
  G4double ec,ecsq,xnulam,etest(0.),ra(0.),a,w,c,signor(1.),signor2,sig; 
  G4double b,ecut,cut,ecut2,geom,elab;
    
  G4double	flow = 1.e-18;
  G4double       spill= 1.e+18; 
   
  if (ResidualA <= 60)      { signor = 0.92; }
  else if (ResidualA < 100) { signor = 0.8 + ResidualA*0.002; }
  
  ec = 1.44 * theZ * ResidualZ / (1.5*ResidualAthrd+ra);
  ecsq = ec * ec;
  p = p0 + p1/ec + p2/ecsq;
  landa = landa0*ResidualA + landa1;
  a = fG4pow->powZ(ResidualA,mu1);
  mu = mum0 * a;
  nu = a* (nu0+nu1*ec+nu2*ecsq);
  
  c =std::min(3.15,ec*0.5);
  w = 0.7 * c / 3.15; 
  
  xnulam = nu / landa;
  if (xnulam > spill) { xnulam=0.; }
  if (xnulam >= flow) { etest =std::sqrt(xnulam) + 7.; }
  
  a = -2.*p*ec + landa - nu/ecsq;
  b = p*ecsq + mu + 2.*nu/ec;
  ecut = 0.;
  cut = a*a - 4.*p*b;
  if (cut > 0.) { ecut = std::sqrt(cut); }
  ecut = (ecut-a) / (p+p);
  ecut2 = ecut;
  //JMQ 290310 for avoiding unphysical increase below minimum (at ecut)
  // ecut<0 means that there is no cut with energy axis, i.e. xs is set 
  // to 0 bellow minimum
  //  if (cut < 0.) ecut2 = ecut - 2.;
  if (cut < 0.) { ecut2 = ecut; }
  elab = K * FragmentA /G4double(ResidualA);
  sig = 0.;
  if (elab <= ec) { //start for E<Ec 
    if (elab > ecut2) { sig = (p*elab*elab+a*elab+b) * signor; }
    
    signor2 = (ec-elab-c) / w;
    signor2 = 1. + G4Exp(signor2);
    sig = sig / signor2;
  }              //end for E<=Ec
  else{           //start for  E>Ec
    sig = (landa*elab+mu+nu/elab) * signor;
    geom = 0.;
    
    if (xnulam < flow || elab < etest) 
      {
        if (sig <0.0) {sig=0.0;}
        return sig;
      }
    geom = std::sqrt(theA*K);
    geom = 1.23*ResidualAthrd + ra + 4.573/geom;
    geom = 31.416 * geom * geom;
    sig = std::max(geom,sig);
    
  }   //end for E>Ec
  return sig;
}

