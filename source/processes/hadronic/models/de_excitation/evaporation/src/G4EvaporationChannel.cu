#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4EvaporationChannel.cc 85841 2014-11-05 15:35:06Z gcosmo $
//
//J.M. Quesada (August2008). Based on:
//
// Hadronic Process: Nuclear De-excitations
// by V. Lara (Oct 1998)
//
// Modified:
// 03-09-2008 J.M. Quesada for external choice of inverse cross section option
// 06-09-2008 J.M. Quesada Also external choices have been added for superimposed 
//                 Coulomb barrier (if useSICB is set true, by default is false) 
// 17-11-2010 V.Ivanchenko in constructor replace G4VEmissionProbability by 
//            G4EvaporationProbability and do not new and delete probability
//            object at each call; use G4Pow

#include "G4EvaporationChannel.hh"
#include "G4PairingCorrection.hh"
#include "G4NucleiProperties.hh"
#include "G4Pow.hh"
#include "G4Log.hh"
#include "G4Exp.hh"
#include "G4EvaporationLevelDensityParameter.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"
#include "G4Alpha.hh"

G4EvaporationChannel::G4EvaporationChannel(G4int anA, G4int aZ, 
					   const G4String & aName,
					   G4EvaporationProbability* aEmissionStrategy,
                                           G4VCoulombBarrier* aCoulombBarrier):
    G4VEvaporationChannel(aName),
    theA(anA),
    theZ(aZ),
    theEvaporationProbabilityPtr(aEmissionStrategy),
    theCoulombBarrierPtr(aCoulombBarrier),
    EmissionProbability(0.0),
    MaximalKineticEnergy(-1000.0)
{ 
  ResidualA = 0;
  ResidualZ = 0;
  ResidualMass = CoulombBarrier = 0.0;
  EvaporatedMass = G4NucleiProperties::GetNuclearMass(theA, theZ);
  theLevelDensityPtr = new G4EvaporationLevelDensityParameter;
  pairingCorrection = G4PairingCorrection::GetInstance();
}

G4EvaporationChannel::~G4EvaporationChannel()
{
  delete theLevelDensityPtr;
}

void G4EvaporationChannel::Initialise()
{
  //for inverse cross section choice
  theEvaporationProbabilityPtr->SetOPTxs(OPTxs);
  // for superimposed Coulomb Barrier for inverse cross sections
  theEvaporationProbabilityPtr->UseSICB(useSICB);

  G4VEvaporationChannel::Initialise();  
}

G4double G4EvaporationChannel::GetEmissionProbability(G4Fragment* fragment)
{
  G4int FragmentA = fragment->GetA_asInt();
  G4int FragmentZ = fragment->GetZ_asInt();
  ResidualA = FragmentA - theA;
  ResidualZ = FragmentZ - theZ;
  //G4cout << "G4EvaporationChannel::Initialize Z= " << theZ << " A= " << theA 
  //	 << " FragZ= " << FragmentZ << " FragA= " << FragmentA << G4endl;
  EmissionProbability = 0.0;

  // Only channels which are physically allowed are taken into account 
  if (ResidualA >= ResidualZ && ResidualZ > 0 && ResidualA >= theA) {
  
    //Effective excitation energy
    G4double ExEnergy = fragment->GetExcitationEnergy() - 
      pairingCorrection->GetPairingCorrection(FragmentA,FragmentZ);
    ResidualMass = G4NucleiProperties::GetNuclearMass(ResidualA, ResidualZ);
    G4double FragmentMass = fragment->GetGroundStateMass();
    G4double Etot = FragmentMass + ExEnergy;
  
    if(ExEnergy > 0.0 && Etot > ResidualMass + EvaporatedMass) {
  
      // Maximal Kinetic Energy
      MaximalKineticEnergy = ((Etot-ResidualMass)*(Etot+ResidualMass) 
	    + EvaporatedMass*EvaporatedMass)/(2.0*Etot) - EvaporatedMass;

      // Emission probability
      // Protection for the case Tmax<V. If not set in this way we could end up in an 
      // infinite loop in  the method GetKineticEnergy if OPTxs!=0 && useSICB=true. 
      // Of course for OPTxs=0 we have the Coulomb barrier 

      CoulombBarrier = 0.0;
      if (OPTxs==0 || (OPTxs!=0 && useSICB)) {
	CoulombBarrier = 
	  theCoulombBarrierPtr->GetCoulombBarrier(ResidualA,ResidualZ,ExEnergy);
      }
      // The threshold for charged particle emission must be  set to 0 if Coulomb 
      //cutoff  is included in the cross sections
      if (MaximalKineticEnergy > CoulombBarrier) {
	EmissionProbability = theEvaporationProbabilityPtr->
	  EmissionProbability(*fragment, MaximalKineticEnergy);
      }
    }
  }
  //G4cout << "G4EvaporationChannel:: probability= " << EmissionProbability << G4endl;   
  return EmissionProbability;
}

G4Fragment* G4EvaporationChannel::EmittedFragment(G4Fragment* theNucleus)
{
  G4Fragment* evFragment = 0;
  G4double evEnergy = SampleKineticEnergy(*theNucleus) + EvaporatedMass;

  G4ThreeVector momentum(IsotropicVector
    (std::sqrt((evEnergy - EvaporatedMass)*(evEnergy + EvaporatedMass))));
  
  G4LorentzVector EvaporatedMomentum(momentum, evEnergy);
  G4LorentzVector ResidualMomentum = theNucleus->GetMomentum();
  EvaporatedMomentum.boost(ResidualMomentum.boostVector());
  
  evFragment = new G4Fragment(theA,theZ,EvaporatedMomentum);
  ResidualMomentum -= EvaporatedMomentum;
  theNucleus->SetZandA_asInt(ResidualZ, ResidualA);
  theNucleus->SetMomentum(ResidualMomentum);

  return evFragment; 
} 

G4FragmentVector * G4EvaporationChannel::BreakUp(const G4Fragment & theNucleus)
{
  G4FragmentVector * theResult = new G4FragmentVector();
  G4Fragment* frag0 = new G4Fragment(theNucleus);
  G4Fragment* frag1 = EmittedFragment(frag0);
  if(frag1) { theResult->push_back(frag1); }
  theResult->push_back(frag0);
  return theResult;
} 

///////////////////////////////////////////
//JMQ: New method for MC sampling of kinetic energy. 
G4double G4EvaporationChannel::SampleKineticEnergy(const G4Fragment & aFragment)
{
  G4double T = 0.0;
  if (OPTxs==0) {
    // It uses Dostrovsky's approximation for the inverse reaction cross
    // in the probability for fragment emission
    // MaximalKineticEnergy energy in the original version (V.Lara) was calculated at 
    //the Coulomb barrier.
    
    G4double Rb = 4.0*theLevelDensityPtr->
      LevelDensityParameter(ResidualA+theA,ResidualZ+theZ,MaximalKineticEnergy)*
      MaximalKineticEnergy;
    G4double RbSqrt = std::sqrt(Rb);
    G4double PEX1 = 0.0;
    if (RbSqrt < 160.0) PEX1 = G4Exp(-RbSqrt);
    G4double Rk = 0.0;
    G4double FRk = 0.0;
    do {
      G4double RandNumber = G4UniformRand();
      Rk = 1.0 + (1./RbSqrt)*G4Log(RandNumber + (1.0-RandNumber)*PEX1);
      G4double Q1 = 1.0;
      G4double Q2 = 1.0;
      if (theZ == 0) { // for emitted neutron
        G4double Beta = (2.12/G4Pow::GetInstance()->Z23(ResidualA) - 0.05)*MeV/
          (0.76 + 2.2/G4Pow::GetInstance()->Z13(ResidualA));
        Q1 = 1.0 + Beta/(MaximalKineticEnergy);
        Q2 = Q1*std::sqrt(Q1);
      } 
      
      FRk = (3.0*std::sqrt(3.0)/2.0)/Q2 * Rk * (Q1 - Rk*Rk);
      
    } while (FRk < G4UniformRand());
    
    T =  MaximalKineticEnergy * (1.0-Rk*Rk) + CoulombBarrier;

  } else {    
    // Coulomb barrier is just included  in the cross sections
    G4double prob;
    do {  
      T=CoulombBarrier+G4UniformRand()*(MaximalKineticEnergy-CoulombBarrier);
      prob = theEvaporationProbabilityPtr->ProbabilityDistributionFunction(aFragment,T);
    } while (EmissionProbability*G4UniformRand() >= prob);
  }
  return T;
}

G4ThreeVector G4EvaporationChannel::IsotropicVector(G4double Magnitude)
    // Samples a isotropic random vectorwith a magnitud given by Magnitude.
    // By default Magnitude = 1.0
{
  G4double CosTheta = 1.0 - 2.0*G4UniformRand();
  G4double SinTheta = std::sqrt(1.0 - CosTheta*CosTheta);
  G4double Phi = twopi*G4UniformRand();
  G4ThreeVector Vector(Magnitude*std::cos(Phi)*SinTheta,
		       Magnitude*std::sin(Phi)*SinTheta,
		       Magnitude*CosTheta);
  return Vector;
}
