#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4E1Probability.cc 86783 2014-11-18 08:43:58Z gcosmo $
//
//---------------------------------------------------------------------
//
// Geant4 class G4E1Probability
//
// by V. Lara (May 2003)
//
// Modifications:
// 18.05.2010 V.Ivanchenko trying to speedup the most slow method
//            by usage of G4Pow, integer A and introduction of const members
// 17.11.2010 V.Ivanchenko perform general cleanup and simplification
//            of integration method; low-limit of integration is defined
//            by gamma energy or is zero (was always zero before)
//

#include "G4E1Probability.hh"
#include "Randomize.hh"
#include "G4Pow.hh"
#include "G4Exp.hh"
#include "G4SystemOfUnits.hh"

// Constructors and operators
//

G4E1Probability::G4E1Probability():G4VEmissionProbability()
{
  G4double x = CLHEP::pi*CLHEP::hbarc;
  normC = 1.0 / (x*x);
  theLevelDensityParameter = 0.125/MeV;
  fG4pow = G4Pow::GetInstance(); 
}

G4E1Probability::~G4E1Probability()
{}


G4double G4E1Probability::EmissionProbDensity(const G4Fragment& frag, 
					      G4double gammaE)
{
  // Calculate the probability density here

  // From nuclear fragment properties and the excitation energy, calculate
  // the probability density for photon evaporation from U to U - gammaE
  // (U = nucleus excitation energy, gammaE = total evaporated photon
  // energy). Fragment = nuclear fragment BEFORE de-excitation

  G4double theProb = 0.0;

  G4int Afrag = frag.GetA_asInt();
  G4double Uexcite = frag.GetExcitationEnergy();
  G4double U = Uexcite - gammaE;

  if(U < 0.0) { return theProb; }

  // Need a level density parameter.
  // For now, just use the constant approximation (not reliable near magic
  // nuclei) - is equivalent to G4ConstantLevelDensityParameter class

  G4double aLevelDensityParam = Afrag*theLevelDensityParameter;

  // VI reduce number of calls to exp 
  G4double levelDens = 
    G4Exp(2*(std::sqrt(aLevelDensityParam*U)-std::sqrt(aLevelDensityParam*Uexcite)));
  // Now form the probability density

  // Define constants for the photoabsorption cross-section (the reverse
  // process of our de-excitation)

  G4double sigma0 = 2.5 * Afrag * millibarn;  // millibarns

  G4double Egdp   = (40.3 / fG4pow->powZ(Afrag,0.2) )*MeV;
  G4double GammaR = 0.30 * Egdp;
 
  // CD
  //cout<<"  PROB TESTS "<<G4endl;
  //cout<<" hbarc = "<<hbarc<<G4endl;
  //cout<<" pi = "<<pi<<G4endl;
  //cout<<" Uexcite, gammaE = "<<Uexcite<<"  "<<gammaE<<G4endl;
  //cout<<" Uexcite, gammaE = "<<Uexcite*MeV<<"  "<<gammaE*MeV<<G4endl;
  //cout<<" lev density param = "<<aLevelDensityParam<<G4endl;
  //cout<<" level densities = "<<levelDensBef<<"  "<<levelDensAft<<G4endl;
  //cout<<" sigma0 = "<<sigma0<<G4endl;
  //cout<<" Egdp, GammaR = "<<Egdp<<"  "<<GammaR<<G4endl;
  //cout<<" normC = "<<normC<<G4endl;

  // VI implementation 18.05.2010
  G4double gammaE2 = gammaE*gammaE;
  G4double gammaR2 = gammaE2*GammaR*GammaR;
  G4double egdp2   = gammaE2 - Egdp*Egdp;
  G4double sigmaAbs = sigma0*gammaR2/(egdp2*egdp2 + gammaR2); 
  theProb = normC * sigmaAbs * gammaE2 * levelDens;

  // CD
  //cout<<" sigmaAbs = "<<sigmaAbs<<G4endl;
  //cout<<" Probability = "<<theProb<<G4endl;

  return theProb;

}

G4double G4E1Probability::EmissionProbability(const G4Fragment& frag, 
                                              G4double gammaE)
{
  // From nuclear fragment properties and the excitation energy, calculate
  // the probability for photon evaporation down to last ground level.
  // fragment = nuclear fragment BEFORE de-excitation

  G4double upperLim = gammaE;
  G4double lowerLim = 0.0; 

  //G4cout << "G4E1Probability::EmissionProbability:  Emin= " << lowerLim
  //	 << " Emax= " << upperLim << G4endl;
  if( upperLim - lowerLim <= CLHEP::keV ) { return 0.0; } 

  // Need to integrate EmissionProbDensity from lowerLim to upperLim 
  // and multiply by factor 3 (?!)

  G4double integ = EmissionIntegration(frag,lowerLim,upperLim);

  return integ;

}

G4double G4E1Probability::EmissionIntegration(const G4Fragment& frag, 
					      G4double lowLim, G4double upLim)

{
  // Simple integration
  // VI replace by direct integration over 100 point

  static const G4int numIters = 100;
  G4double Step = (upLim-lowLim)/G4double(numIters);

  G4double res = 0.0;
  G4double x = lowLim - 0.5*Step;

  for(G4int i = 0; i < numIters; ++i) {
    x += Step;
    res += EmissionProbDensity(frag, x);
  }

  if(res > 0.0) { res *= Step; }
  else { res = 0.0; }

  return res;

}


