#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4NuclearLevelManager.cc 87163 2014-11-26 08:46:54Z gcosmo $
//
// -------------------------------------------------------------------
//      GEANT 4 class file 
//
//      CERN, Geneva, Switzerland
//
//      File name:     G4NuclearLevelManager
//
//      Author:        Maria Grazia Pia (pia@genova.infn.it)
// 
//      Creation date: 24 October 1998
//
//      Modifications: 
//      
//        15 April 1999, Alessandro Brunengo (Alessandro.Brunengo@ge.infn.it)
//              Added half-life, angular momentum, parity, emissioni type
//              reading from experimental data. 
//        02 May 2003,   Vladimir Ivanchenko remove rublic copy constructor
//        06 Oct 2010, M. Kelsey -- Use object storage, not pointers, drop
//		public access to list, simplify list construction
// -------------------------------------------------------------------
#include <stdlib.h>
#include <fstream>
#include <sstream>
#include <algorithm>

#include "G4NuclearLevelManager.hh"

#include "globals.hh"
#include "G4SystemOfUnits.hh"
#include "G4NuclearLevel.hh"
#include "G4ios.hh"
#include "G4HadronicException.hh"
#include "G4HadTmpUtil.hh"
#include "G4NucleiProperties.hh"
#include "G4PhysicalConstants.hh"

G4double G4NuclearLevelManager::_levelEnergy=0.;
G4double G4NuclearLevelManager::_gammaEnergy=0.;
G4double G4NuclearLevelManager::_probability=0.;
G4double G4NuclearLevelManager::_polarity=0.;
G4double G4NuclearLevelManager::_halfLife=0.;
G4double G4NuclearLevelManager::_angularMomentum=0.;
G4double G4NuclearLevelManager::_kCC=0.;
G4double G4NuclearLevelManager::_l1CC=0.;
G4double G4NuclearLevelManager::_l2CC=0.;
G4double G4NuclearLevelManager::_l3CC=0.;
G4double G4NuclearLevelManager::_m1CC=0.;
G4double G4NuclearLevelManager::_m2CC=0.;
G4double G4NuclearLevelManager::_m3CC=0.;
G4double G4NuclearLevelManager::_m4CC=0.;
G4double G4NuclearLevelManager::_m5CC=0.;
G4double G4NuclearLevelManager::_nPlusCC=0.;
G4double G4NuclearLevelManager::_totalCC=0.;

G4NuclearLevelManager::G4NuclearLevelManager(G4int Z, G4int A, 
					     const G4String& filename) :
    _nucleusA(A), _nucleusZ(Z), _validity(false), 
    _levels(0)
{ 
  if (A <= 0 || Z <= 0 || Z > A ) {
    throw G4HadronicException(__FILE__, __LINE__, 
			      "==== G4NuclearLevelManager ==== (Z,A) <0, or Z>A");
  }
  MakeLevels(filename);
}

G4NuclearLevelManager::~G4NuclearLevelManager()
{
  ClearLevels();
}

void G4NuclearLevelManager::SetNucleus(G4int Z, G4int A, const G4String& filename)
{
  if (A <= 0 || Z <= 0 || Z > A ) {
    throw G4HadronicException(__FILE__, __LINE__, 
			      "==== G4NuclearLevelManager ==== (Z,A) <0, or Z>A");
  }
  if (_nucleusZ != Z || _nucleusA != A)
    {
      _nucleusA = A;
      _nucleusZ = Z;
      MakeLevels(filename);
    }
}

const G4NuclearLevel* G4NuclearLevelManager::GetLevel(G4int i) const {
  return (i>=0 && i<NumberOfLevels()) ? (*_levels)[i] : 0;
}

const G4NuclearLevel* 
G4NuclearLevelManager::NearestLevel(G4double energy, G4double) const 
{
  if (NumberOfLevels() <= 0) return 0;

  G4int iNear = -1;

  //G4cout << "G4NuclearLevelManager::NearestLevel E(MeV)= " 
  //	 << energy/MeV << " dEmax(MeV)= " << eDiffMax/MeV << G4endl;
  
  G4double diff = 1.e+10;
  for (unsigned int i=0; i<_levels->size(); ++i)
    {
      G4double e = GetLevel(i)->Energy();
      G4double eDiff = std::fabs(e - energy);
      //G4cout << i << ".   eDiff(MeV)= " << eDiff/MeV << G4endl;
      if (eDiff <= diff)
	{ 
	  diff = eDiff; 
	  iNear = i;
	}
    }
  
  return GetLevel(iNear);	// Includes range checking on iNear
}

G4double G4NuclearLevelManager::MinLevelEnergy() const
{
  return (NumberOfLevels() > 0) ? _levels->front()->Energy() : 9999.*GeV;
}

G4double G4NuclearLevelManager::MaxLevelEnergy() const
{
  return (NumberOfLevels() > 0) ? _levels->back()->Energy() : 0.*GeV;
}

const G4NuclearLevel* G4NuclearLevelManager::HighestLevel() const
{
  return (NumberOfLevels() > 0) ? _levels->front() : 0;
}

const G4NuclearLevel* G4NuclearLevelManager::LowestLevel() const
{
  return (NumberOfLevels() > 0) ? _levels->back() : 0;
}

G4bool G4NuclearLevelManager::Read(std::ifstream& dataFile) 
{
  G4bool goodRead = ReadDataLine(dataFile);
  
  if (goodRead) ProcessDataLine();
  return goodRead;
}

// NOTE:  Standard stream I/O generates a 45 byte std::string per item!

G4bool G4NuclearLevelManager::ReadDataLine(std::ifstream& dataFile) {
  /***** DO NOT USE REGULAR STREAM I/O
  G4bool result = true;
  if (dataFile >> _levelEnergy)
    {
      dataFile >> _gammaEnergy >> _probability >> _polarity >> _halfLife
	       >> _angularMomentum  >> _totalCC >> _kCC >> _l1CC >> _l2CC 
	       >> _l3CC >> _m1CC >> _m2CC >> _m3CC >> _m4CC >> _m5CC
	       >> _nPlusCC;
    }
  else result = false;
  *****/

  // Each item will return iostream status
  return (ReadDataItem(dataFile, _levelEnergy) &&
	  ReadDataItem(dataFile, _gammaEnergy) &&
	  ReadDataItem(dataFile, _probability) &&
	  ReadDataItem(dataFile, _polarity) &&
	  ReadDataItem(dataFile, _halfLife) &&
	  ReadDataItem(dataFile, _angularMomentum) &&
	  ReadDataItem(dataFile, _totalCC) &&
	  ReadDataItem(dataFile, _kCC) &&
	  ReadDataItem(dataFile, _l1CC) &&
	  ReadDataItem(dataFile, _l2CC) &&
	  ReadDataItem(dataFile, _l3CC) &&
	  ReadDataItem(dataFile, _m1CC) &&
	  ReadDataItem(dataFile, _m2CC) &&
	  ReadDataItem(dataFile, _m3CC) &&
	  ReadDataItem(dataFile, _m4CC) &&
	  ReadDataItem(dataFile, _m5CC) &&
	  ReadDataItem(dataFile, _nPlusCC) );
}

G4bool 
G4NuclearLevelManager::ReadDataItem(std::istream& dataFile, G4double& x) 
{
  // G4bool okay = (dataFile >> buffer) != 0;		// Get next token
  // if (okay) x = strtod(buffer, NULL);
  char buffer[30];
  for(G4int i=0; i<30; ++i) { buffer[i] = 0; }
  G4bool okay = true;
  dataFile >> buffer;
  if(dataFile.fail()) { okay = false; }
  else { x = strtod(buffer, NULL); }

  return okay;
}

void G4NuclearLevelManager::ProcessDataLine() 
{
  const G4double minProbability = 1e-8;
  
  // Assign units for dimensional quantities
  _levelEnergy *= keV;
  _gammaEnergy *= keV;
  _halfLife *= second;
  
  // The following adjustment is needed to take care of anomalies in 
  // data files, where some transitions show up with relative probability
  // zero
  if (_probability < minProbability) _probability = minProbability;
  // the folowwing is to convert icc probability to accumulative ones
  _l1CC += _kCC;
  _l2CC += _l1CC;
  _l3CC += _l2CC;
  _m1CC += _l3CC;
  _m2CC += _m1CC;
  _m3CC += _m2CC;
  _m4CC += _m3CC;
  _m5CC += _m4CC;
  _nPlusCC += _m5CC;

  if (_nPlusCC!=0) {	// Normalize to probabilities
    _kCC /= _nPlusCC;
    _l1CC /= _nPlusCC;
    _l2CC /= _nPlusCC;
    _l3CC /= _nPlusCC;
    _m1CC /= _nPlusCC;
    _m2CC /= _nPlusCC;
    _m3CC /= _nPlusCC;
    _m4CC /= _nPlusCC;
    _m5CC /= _nPlusCC;
    _nPlusCC /= _nPlusCC;  
  } else {		// Total was zero, reset to unity
    _kCC = 1;
    _l1CC = 1;
    _l2CC = 1;
    _l3CC = 1;
    _m1CC = 1;
    _m2CC = 1;
    _m3CC = 1;
    _m4CC = 1;
    _m5CC = 1;
    _nPlusCC = 1;
  }
	
  // G4cout << "Read " << _levelEnergy << " " << _gammaEnergy << " " << _probability << G4endl;
}

void G4NuclearLevelManager::ClearLevels()
{
  _validity = false;

  if (NumberOfLevels() > 0) {
    std::for_each(_levels->begin(), _levels->end(), DeleteLevel());
    _levels->clear();
  }

  delete _levels;
  _levels = 0;
}

void G4NuclearLevelManager::MakeLevels(const G4String& filename)
{
  _validity = false;
  if (NumberOfLevels() > 0) ClearLevels();	// Discard existing data

  std::ifstream inFile(filename, std::ios::in);
  if (! inFile) 
    {
#ifdef GAMMAFILEWARNING
      if (_nucleusZ > 10) G4cout << " G4NuclearLevelManager: nuclide (" 
                                 << _nucleusZ << "," << _nucleusA 
                                 << ") does not have a gamma levels file" << G4endl;
#endif
      return;
    }

  _levels = new G4PtrLevelVector;

  // Read individual gamma data and fill levels for this nucleus
 
  G4NuclearLevel* thisLevel = 0;
  G4int nData = 0;

  while (Read(inFile)) {
    thisLevel = UseLevelOrMakeNew(thisLevel);	// May create new pointer
    AddDataToLevel(thisLevel);
    nData++;					// For debugging purposes
  }

  FinishLevel(thisLevel);		// Final  must be completed by hand
  
  // ---- MGP ---- Don't forget to close the file 
  inFile.close();
	
  //  G4cout << " ==== MakeLevels ===== " << nData << " data read " << G4endl;

  G4PtrSort<G4NuclearLevel>(_levels);
  
  _validity = true;
  
  return;
}

G4NuclearLevel* 
G4NuclearLevelManager::UseLevelOrMakeNew(G4NuclearLevel* level) 
{
  if (level && _levelEnergy == level->Energy()) return level;	// No change

  if (level) FinishLevel(level);	// Save what we have up to now

  //  G4cout << "Making a new level... " << _levelEnergy << G4endl;
  return new G4NuclearLevel(_levelEnergy, _halfLife, _angularMomentum);
}

void G4NuclearLevelManager::AddDataToLevel(G4NuclearLevel* level) 
{
  if (!level) return;		// Sanity check

  level->_energies.push_back(_gammaEnergy);
  level->_weights.push_back(_probability);
  level->_polarities.push_back(_polarity);
  level->_kCC.push_back(_kCC);
  level->_l1CC.push_back(_l1CC);
  level->_l2CC.push_back(_l2CC);
  level->_l3CC.push_back(_l3CC);
  level->_m1CC.push_back(_m1CC);
  level->_m2CC.push_back(_m2CC);
  level->_m3CC.push_back(_m3CC);
  level->_m4CC.push_back(_m4CC);
  level->_m5CC.push_back(_m5CC);
  level->_nPlusCC.push_back(_nPlusCC);
  level->_totalCC.push_back(_totalCC);
}

void G4NuclearLevelManager::FinishLevel(G4NuclearLevel* level) 
{
  if (!level || !_levels) return;		// Sanity check

  level->Finalize();
  _levels->push_back(level);
}


void G4NuclearLevelManager::PrintAll()
{
  G4int nLevels = NumberOfLevels();
    
  G4cout << " ==== G4NuclearLevelManager ==== (" << _nucleusZ << ", " << _nucleusA
	 << ") has " << nLevels << " levels" << G4endl
	 << "Highest level is at energy " << MaxLevelEnergy() << " MeV "
	 << G4endl << "Lowest level is at energy " << MinLevelEnergy()
	 << " MeV " << G4endl;
    
  for (G4int i=0; i<nLevels; ++i) {
    GetLevel(i)->PrintAll();
  }
}

void G4NuclearLevelManager::PrintLevels()
{
  G4int nLevels = NumberOfLevels();
  G4double efermi = G4NucleiProperties::GetNuclearMass(_nucleusA-1, _nucleusZ)
    + neutron_mass_c2 -
    G4NucleiProperties::GetNuclearMass(_nucleusA, _nucleusZ);

  G4cout << "Z= " << _nucleusZ << " A= " << _nucleusA
	 << "  " << nLevels << " levels" 
	 << "  Efermi(MeV)= " << efermi << G4endl;
    
  for (G4int i=0; i<nLevels; ++i) {
    GetLevel(i)->PrintLevels();
  }
}

G4NuclearLevelManager::G4NuclearLevelManager(const G4NuclearLevelManager &right)
{
  _nucleusA = right._nucleusA;
  _nucleusZ = right._nucleusZ;
  _validity = right._validity;

  if (right._levels != 0)   
    {
      _levels = new G4PtrLevelVector;
      G4int n = right._levels->size();
      G4int i;
      for (i=0; i<n; ++i)
	{
	  _levels->push_back(new G4NuclearLevel(*(right.GetLevel(i))));
	}
      G4PtrSort<G4NuclearLevel>(_levels);
    }
  else 
    {
      _levels = 0;
    }
}

