#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4VGammaDeexcitation.cc 88987 2015-03-17 10:39:50Z gcosmo $
//
// -------------------------------------------------------------------
//      GEANT 4 class file 
//
//      CERN, Geneva, Switzerland
//
//      File name:     G4VGammaDeexcitation
//
//      Author:        Maria Grazia Pia (pia@genova.infn.it)
// 
//      Creation date: 23 October 1998
//
//      Modifications: 
//
//        21 Nov 2001, Fan Lei (flei@space.qinetiq.com)
//           Modified GenerateGamma() and UpdateUncleus() for implementation
//           of Internal Conversion processs 
//      
//        15 April 1999, Alessandro Brunengo (Alessandro.Brunengo@ge.infn.it)
//              Added creation time evaluation for products of evaporation
//
//        19 April 2010, J. M. Quesada calculations in CM system
//              pending final boost to lab system  (not critical)
//
//        23 April 2010, V.Ivanchenko rewite kinematic part using PDG formula
//                                    for 2-body decay
//
//        07 May   2011, V.Ivanchenko implement check ICM flag - produce or not e-
//
// -------------------------------------------------------------------

#include "G4VGammaDeexcitation.hh"

#include "globals.hh"
#include "G4PhysicalConstants.hh"
#include "Randomize.hh"
#include "G4Gamma.hh"
#include "G4Electron.hh"
#include "G4LorentzVector.hh"
#include "G4VGammaTransition.hh"
#include "G4Fragment.hh"
#include "G4FragmentVector.hh"

#include "G4ParticleTable.hh"
#include "G4IonTable.hh"

#include "G4DiscreteGammaTransition.hh"


G4VGammaDeexcitation::G4VGammaDeexcitation(): _transition(0), _verbose(0),
					      _electronO (0), _vSN(-1)
{ 
  _tolerance = 2*CLHEP::keV;
  _timeLimit = DBL_MAX;
}

G4VGammaDeexcitation::~G4VGammaDeexcitation()
{ 
  delete _transition; 
}

void G4VGammaDeexcitation::DoChain(G4FragmentVector* products, 
				   G4Fragment* nucleus)
{
  if (_verbose > 1) { G4cout << "G4VGammaDeexcitation::DoChain" << G4endl; }
  
  if(CanDoTransition(nucleus)) { 
    for(size_t i=0; i<100; ++i) {      
      _transition->SetEnergyFrom(nucleus->GetExcitationEnergy());
      G4Fragment* gamma = GenerateGamma(nucleus);
      if (gamma) { products->push_back(gamma); }
      else { break; } 
      //G4cout << i << ".  Egamma(MeV)= " << gamma->GetMomentum().e() 
      //	     << "; new Eex(MeV)= " << nucleus->GetExcitationEnergy() 
      //       << G4endl;
      if(nucleus->GetExcitationEnergy() <= _tolerance) { break; }
    } 
  }
  if (_verbose > 1) {
    G4cout << "G4VGammaDeexcitation::DoChain - end" << G4endl;
  }
}

G4Fragment* G4VGammaDeexcitation::GenerateGamma(G4Fragment* aNucleus)
{
  G4Fragment * thePhoton = 0;
  _vSN = -1;

  _transition->SelectGamma();  // it can be conversion electron too

  G4double etrans = _transition->GetGammaEnergy();

  //L.Desorgher 05/01/2015 need to add the bond energy for correct 
  //                       computation of a transition in case of ICM
  G4DiscreteGammaTransition* dtransition =
     dynamic_cast <G4DiscreteGammaTransition*> (_transition);
  G4double bond_energy=0.;

  if (dtransition && !dtransition->IsAGamma()) { 
    bond_energy = dtransition->GetBondEnergy(); 
  }
  etrans += bond_energy;
  //G4cout << "G4VGammaDeexcitation::GenerateGamma - Etrans(MeV)= " 
  //	 << etrans << G4endl; 
  if(etrans <= 0.0) { return thePhoton; }

  // final excitation
  G4double excitation = aNucleus->GetExcitationEnergy() - etrans;
  if(excitation <= _tolerance) { excitation = 0.0; } 

  G4double gammaTime = _transition->GetGammaCreationTime();
  if (_verbose > 1) {
    G4cout << "G4VGammaDeexcitation::GenerateGamma - Edeexc(MeV)= " 
           << etrans << "; Time(ns)= " << gammaTime/CLHEP::ns 
	   << "; left Eexc(MeV)= " << excitation << G4endl;
  }
  
  // Do complete Lorentz computation 
  G4LorentzVector lv = aNucleus->GetMomentum();
  G4double Mass = aNucleus->GetGroundStateMass() + excitation;

  // select secondary
  G4ParticleDefinition* gamma = G4Gamma::Gamma();

  if (dtransition && !dtransition->IsAGamma() ) {
    gamma = G4Electron::Electron(); 
    _vSN = dtransition->GetOrbitNumber();   
    _electronO.RemoveElectron(_vSN);
    //L. Desorgher 05/01/2015 need to remove atomic bond energy 
    //                        of the IC electron
    lv += G4LorentzVector(0.0,0.0,0.0,
                          CLHEP::electron_mass_c2 - bond_energy);
  }

  G4double cosTheta = 1. - 2. * G4UniformRand(); 
  G4double sinTheta = std::sqrt(1. - cosTheta * cosTheta);
  G4double phi = twopi * G4UniformRand();

  G4double eMass = gamma->GetPDGMass();
  G4LorentzVector Gamma4P;
  /*
  G4cout << " Mass= " << eMass << " t= " << gammaTime
  	 << " tlim= " << _timeLimit << G4endl;
  */
  // 2-body decay in rest frame
  G4double Ecm       = lv.mag();
  G4ThreeVector bst  = lv.boostVector();

  G4double GammaEnergy = 0.5*((Ecm - Mass)*(Ecm + Mass) + eMass*eMass)/Ecm;
  if(GammaEnergy < eMass) { GammaEnergy = eMass; }

  G4double mom = std::sqrt((GammaEnergy - eMass)*(GammaEnergy + eMass));
  Gamma4P.set(mom * sinTheta * std::cos(phi),
	      mom * sinTheta * std::sin(phi),
	      mom * cosTheta, GammaEnergy);

  // Lab system in normal case (_timeLimit = DBL_MAX)
  if(gammaTime <= _timeLimit) { 
    Gamma4P.boost(bst); 
    lv -= Gamma4P;
  } else {  
    // In exceptional case sample decay at rest at not correct position 
    // of stopping ion, 4-momentum balance is breaked but gamma energy
    // is correct
    lv -= Gamma4P;
    G4double E = lv.e();
    G4double P2= (E - Mass)*(E + Mass);
    G4ThreeVector v = lv.vect().unit();
    G4double p = 0.0;
    if(P2 > 0.0) { p = sqrt(P2); } 
    else { E = Mass; }
    lv.set(v.x()*p, v.y()*p, v.z()*p, E);  
  }

  // modified primary fragment 
  gammaTime += aNucleus->GetCreationTime();
  aNucleus->SetMomentum(lv);
  aNucleus->SetCreationTime(gammaTime);

  // gamma or e- are produced
  thePhoton = new G4Fragment(Gamma4P,gamma);
  thePhoton->SetCreationTime(gammaTime);

  //G4cout << "G4VGammaDeexcitation::GenerateGamma : " << thePhoton << G4endl;
  //G4cout << "       Left nucleus: " << aNucleus << G4endl;
  return thePhoton;
}

