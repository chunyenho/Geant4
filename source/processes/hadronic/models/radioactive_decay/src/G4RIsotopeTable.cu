#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4RIsotopeTable.cc
//
// Version:             0.b.4
// Date:                14/04/00
// Author:              F Lei & P R Truscott
// Organisation:        DERA UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            12115/96/JG/NL Work Order No. 3
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// CHANGE HISTORY
// --------------
//
// 29 February 2000, P R Truscott, DERA UK
// 0.b.3 release.
//
// 14 April 2000, F Lei, DERA UK
// 0.b.4 release. Minor changes to 
//            1) levelTolerance = 2.0 keV
//            2) changes to verbose control
//
// 18,July 2001 F.Lei
//  tidy up the print out at run level
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#include "G4DecayTable.hh"
#include "G4SystemOfUnits.hh"
#include "G4ParticleTable.hh"
#include "G4IsotopeProperty.hh"
#include "G4RIsotopeTable.hh"

#include "G4HadronicException.hh"
#include "G4NuclearLevelStore.hh"

/*
#include "G4RadioactiveDecayMode.hh"
#include "G4ITDecayChannel.hh"
#include "G4BetaMinusDecayChannel.hh"
#include "G4BetaPlusDecayChannel.hh"
#include "G4KshellECDecayChannel.hh"
#include "G4LshellECDecayChannel.hh"
#include "G4AlphaDecayChannel.hh"
#include "G4ProtonDecayChannel.hh"
*/
#include "G4ios.hh"
#include "globals.hh"
#include <iomanip>
#include <fstream>
#include <sstream>

const G4double G4RIsotopeTable::levelTolerance = 2.0*keV;


G4RIsotopeTable::G4RIsotopeTable()
{
  // Reset the list of user defined data files
  theUserRadioactiveDataFiles.clear();
}


G4RIsotopeTable::~G4RIsotopeTable()
{
  for (G4int i = 0; i < G4int(fIsotopeList.size()); i++) delete fIsotopeList[i];
  fIsotopeList.clear();
  fIsotopeNameList.clear(); 
}


G4int G4RIsotopeTable::GetVerboseLevel() const
{
  return G4ParticleTable::GetParticleTable()->GetVerboseLevel();
}


G4bool G4RIsotopeTable::FindIsotope(G4IsotopeProperty* )
{
  // do nothing, it is here just for the compiler
  // it is required by the base class
  return true;
}


G4IsotopeProperty* G4RIsotopeTable::GetIsotope(G4int Z, G4int A, G4double E)
{
  G4String fname = GetIsotopeName(Z, A, E);  
  G4int j = -1;
  for (G4int i = 0 ; i< Entries(); i++) {
    if(fIsotopeNameList[i] == fname) j = i;}
  if (j >=0) {
    if (GetVerboseLevel() > 1) {
      G4cout <<"G4RIsotopeTable::GetIsotope No. : ";
      G4cout <<j<<G4endl;   
    }
    return  GetIsotope(j);
    // isotope property data has been loaded already - just return the pointer

  } else {
    G4double meanlife = GetMeanLifeTime(Z, A, E);
    // E is passed as a refence hence on entry E is supplied by the user and it 
    // could be slightly different from the returned value which is the one 
    // defined in the database.
    // this call is to ensure the code uses a consistent E value throughout.
    
    G4IsotopeProperty* fProperty = new G4IsotopeProperty();   
    // Set Isotope Property
    fProperty->SetLifeTime(meanlife);
    fProperty->SetAtomicNumber(Z);
    fProperty->SetAtomicMass(A);
    // Notic that the value of E may have been changed
    fProperty->SetEnergy(E);
    // The spin is not being used in the current implementation
    fProperty->SetiSpin(0);
    // the decaytable will be loaded later in G4RadioactiveDecay when it is needed
    fProperty->SetDecayTable(0);
    
    fIsotopeList.push_back(fProperty);
    fname = GetIsotopeName(Z, A, E);
    fIsotopeNameList.push_back(fname);
    if (GetVerboseLevel()>1) {
      G4cout <<"G4RIsotopeTable::GetIsotope create: ";
      G4cout <<fname <<G4endl;  
    }
    return fProperty;

  }
}
///////////////////////////////////////////////////////////////////////////////
//
G4String G4RIsotopeTable::GetIsotopeName(G4int Z, G4int A, G4double E)  
{
  std::ostringstream os;
  os.setf(std::ios::fixed);
  os <<"A"<< A << "Z" << Z <<'[' << std::setprecision(1) << E/keV << ']';
  G4String name = os.str();
  if (GetVerboseLevel()>1) {
    G4cout <<"G4RIsotopeTable::GetIsotope Name: ";
    G4cout <<name <<G4endl;   
  }
  return name;
}


G4double G4RIsotopeTable::GetMeanLifeTime(G4int Z, G4int A, G4double& aE)
{

  G4double lifetime = -1.0;

  //Check if data have been provided by the user
  std::map<int,G4String>::iterator it = theUserRadioactiveDataFiles.find(1000*A+Z);
  G4String file="";
  if ( it != theUserRadioactiveDataFiles.end() ){
      file=it->second;
  }
  if (file ==""){
	if (!getenv("G4RADIOACTIVEDATA")) {
		G4cout << "Please setenv G4RADIOACTIVEDATA to point to the radioactive decay data files." << G4endl;
		throw G4HadronicException(__FILE__, __LINE__,
			      "Please setenv G4RADIOACTIVEDATA to point to the radioactive decay data files.");
	}
	G4String dirName = getenv("G4RADIOACTIVEDATA");

	std::ostringstream os;
	os <<dirName <<"/z" <<Z <<".a" <<A ;
	file = os.str();
  }
  std::ifstream DecaySchemeFile(file);

  G4bool found_in_raddecay_data(false);
  if (!DecaySchemeFile) {
    if (GetVerboseLevel()>1) {
      G4cout <<"G4RIsotopeTable::GetMeanLife() : "
	     <<"cannot find ion radioactive decay file: " 
	     <<file <<G4endl;
    }
  } else {
    char inputChars[100]={' '};
    G4String inputLine;
    G4String recordType("");
    G4double a(0.0);
    G4double b(0.0);

    while (!found_in_raddecay_data && !DecaySchemeFile.getline(inputChars, 100).eof()) {
      inputLine = inputChars;
      inputLine = inputLine.strip(1);

      if (inputChars[0] != '#' && inputLine.length() != 0) {
        std::istringstream tmpstream(inputLine);
        tmpstream >> recordType >> a >> b;
        if (recordType == "P") {
          if (std::abs(a*keV-aE) < levelTolerance) {
            found_in_raddecay_data    = true;
            lifetime = b/0.693147*s ;
          }
        }
      }
    }
    DecaySchemeFile.close();
  }

    if (!found_in_raddecay_data && aE) {
      G4double half_life=-1.;
      lifetime = 1.0E-20*s;


      //added by L.Desorgher If the life time is not found in  raddecay database
      // then it is deduced from photo-evaporation level
      const G4NuclearLevel* aLevel =
    		G4NuclearLevelStore::GetInstance()->GetManager(Z, A)
    									->NearestLevel(aE,levelTolerance);
      if (aLevel) {
    	  half_life = aLevel->HalfLife();
    	  lifetime = half_life/0.693147;
      }

      if (GetVerboseLevel()>1 && half_life<0) {
        G4cout << "G4RIsotopeTable::GetMeanLife() : ";
        G4cout << "cannot find ion of required excitation E = " << aE << G4endl;
        G4cout << "state in radioactive or photoevaporation data file " << G4endl;
        G4cout <<"The nucleus is assumed to be IT decayed with life = 1E-20 s" << G4endl;
        G4cout <<" -----------* THIS MAY CAUSE PROBLEM IN ITS DECAY-----------" << G4endl;
      }
    }

    if (!found_in_raddecay_data && !aE) {
      if (GetVerboseLevel()>1) {
        G4cout <<"G4RIsotopeTable::GetMeanLife() : ";
        G4cout <<"cannot find ion of required excitation E = " << aE << G4endl;
        G4cout <<"state in radioactive or photoevaporation data file" <<G4endl;
        G4cout <<"The nucleus is assumed to be stable" <<G4endl;
        lifetime = -1.0;
      }
    }

    if (GetVerboseLevel()>1) {
       G4cout <<"G4RIsotopeTable::GetMeanLifeTime: ";
       G4cout <<lifetime << " for " << GetIsotopeName(Z, A, aE) <<G4endl;
    }
  return lifetime;
}
////////////////////////////////////////////////////////////////////
//
void G4RIsotopeTable::AddUserDecayDataFile(G4int Z, G4int A,G4String filename)
{ if (Z<1 || A<2) {
	G4cout<<"Z and A not valid!"<<G4endl;
  }

  std::ifstream DecaySchemeFile(filename);
  if (DecaySchemeFile){
	G4int ID_ion=A*1000+Z;
	theUserRadioactiveDataFiles[ID_ion]=filename;
  }
  else {
	G4cout<<"The file "<<filename<<" does not exist!"<<G4endl;
  }
}

