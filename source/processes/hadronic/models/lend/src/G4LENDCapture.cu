//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//

#include "G4LENDCapture.hh"
#include "G4SystemOfUnits.hh"
#include "G4Nucleus.hh"
#include "G4ParticleTable.hh"
#include "G4IonTable.hh"
  
G4HadFinalState * G4LENDCapture::ApplyYourself(const G4HadProjectile& aTrack, G4Nucleus& aTarg )
{

   G4double temp = aTrack.GetMaterial()->GetTemperature();

   //G4int iZ = int ( aTarg.GetZ() );
   //G4int iA = int ( aTarg.GetN() );
   //migrate to integer A and Z (GetN_asInt returns number of neutrons in the nucleus since this) 
   G4int iZ = aTarg.GetZ_asInt();
   G4int iA = aTarg.GetA_asInt();
   G4int iM = 0;
   if ( aTarg.GetIsotope() != NULL ) {
      iM = aTarg.GetIsotope()->Getm();
   }

   G4double ke = aTrack.GetKineticEnergy();

   G4HadFinalState* theResult = &theParticleChange;
   theResult->Clear();

   G4GIDI_target* aTarget = usedTarget_map.find( lend_manager->GetNucleusEncoding( iZ , iA , iM ) )->second->GetTarget();
   std::vector<G4GIDI_Product>* products = aTarget->getCaptureFinalState( ke*MeV, temp, NULL, NULL );


   if ( products != NULL ) 
   {

      G4ThreeVector p(0,0,0);
      G4int totN = 0;

      for ( G4int j = 0; j < int( products->size() ); j++ ) 
      {
         G4int jZ = (*products)[j].Z; 
         G4int jA = (*products)[j].A; 

         //G4cout << "ZA = " << 1000 * (*products)[j].Z + (*products)[j].A << "  EK = "
         //     << (*products)[j].kineticEnergy
         //     << " px  " <<  (*products)[j].px
         //     << " py  " <<  (*products)[j].py
         //     << " pz  " <<  (*products)[j].pz
         //     << G4endl;

         G4ThreeVector dp((*products)[j].px,(*products)[j].py,(*products)[j].pz);
         p += dp;
          
         G4DynamicParticle* theSec = new G4DynamicParticle;

         if ( jA == 1 && jZ == 1 )
         {
            theSec->SetDefinition( G4Proton::Proton() );
            totN += 1;
         }
         else if ( jA == 1 && jZ == 0 )
         {
            theSec->SetDefinition( G4Neutron::Neutron() );
            totN += 1;
         } 
         else if ( jZ > 0 )
         {
            if ( jA != 0 )
            {
               theSec->SetDefinition( G4IonTable::GetIonTable()->GetIon( jZ , jA , iM ) );
               totN += jA;
            }
            else 
            {
               theSec->SetDefinition( G4IonTable::GetIonTable()->GetIon( jZ , iA+1-totN , iM ) );
            }
         } 
         else
         {
            theSec->SetDefinition( G4Gamma::Gamma() );
         } 

         theSec->SetMomentum( G4ThreeVector( (*products)[j].px*MeV , (*products)[j].py*MeV , (*products)[j].pz*MeV ) );

         if ( dp.mag() == 0 ) 
         {
            theSec->SetMomentum( -p*MeV ); 
         }

         theResult->AddSecondary( theSec );
      } 
   }
   delete products;

   theResult->SetStatusChange( stopAndKill );

   return theResult; 

}
