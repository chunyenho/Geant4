#include "hip/hip_runtime.h"
/*
# <<BEGIN-copyright>>
# Copyright (c) 2010, Lawrence Livermore National Security, LLC. 
# Produced at the Lawrence Livermore National Laboratory 
# Written by Bret R. Beck, beck6@llnl.gov. 
# CODE-461393
# All rights reserved. 
#  
# This file is part of GIDI. For details, see nuclear.llnl.gov. 
# Please also read the "Additional BSD Notice" at nuclear.llnl.gov. 
# 
# Redistribution and use in source and binary forms, with or without modification, 
# are permitted provided that the following conditions are met: 
#
#      1) Redistributions of source code must retain the above copyright notice, 
#         this list of conditions and the disclaimer below.
#      2) Redistributions in binary form must reproduce the above copyright notice, 
#         this list of conditions and the disclaimer (as noted below) in the 
#          documentation and/or other materials provided with the distribution.
#      3) Neither the name of the LLNS/LLNL nor the names of its contributors may be 
#         used to endorse or promote products derived from this software without 
#         specific prior written permission. 
#
# THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
# EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
# OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT 
# SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC, THE U.S. DEPARTMENT OF ENERGY OR 
# CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
# CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS 
# OR SERVICES;  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED 
# AND ON  ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
# (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
# EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE. 
# <<END-copyright>>
*/
#include <stdlib.h>
#include <limits.h>
#include <ctype.h>
#include "xData.h"

#if defined __cplusplus
namespace GIDI {
using namespace GIDI;
#endif

//char const * const xData_twod_xindex_y_ID = "2d.xindex_y";

static int toData( statusMessageReporting *smr, xDataType *xDT, xData_attributionList *attributes, const char *text );
static char *toString( statusMessageReporting *smr, xDataType *xDT );
static int release( statusMessageReporting *smr, xDataType *xDT );
static double *xData_2d_xindex_y_toFilled( statusMessageReporting *smr, xData_element *element, double *Xs, int size );
/*
************************************************************
*/
int xData_init_2d_xindex_y( statusMessageReporting *smr, xData_element *element ) {

    xDataType *xDT = &(element->xDataTypeInfo);

    xDT->status = xData_xDataType_Ok;
    xDT->typeString = xData_twod_xindex_y_ID;
    xDT->element = element;
    xDT->toData = toData;
    xDT->toString = toString;
    xDT->release = release;
    xDT->data = NULL;
    return( xData_xDataTypeConvertAttributes( smr, element ) );
}
/*
************************************************************
*/
int xData_is_2d_xindex_y( statusMessageReporting *smr, xDataType *xDT, int setMsg ) {

    return( xData_is_xDataType( smr, xDT, xData_twod_xindex_y_ID, setMsg ) );
}
/*
************************************************************
*/
int xData_isElement_2d_xindex_y( statusMessageReporting *smr, xData_element *element, int setMsg ) {

    return( xData_is_2d_xindex_y( smr, &(element->xDataTypeInfo), setMsg ) );
}
/*
************************************************************
*/
//static int toData( statusMessageReporting *smr, xDataType *xDT, xData_attributionList *attributes, const char *text ) {
static int toData( statusMessageReporting *smr, xDataType *xDT, xData_attributionList *, const char *text ) {

    xData_Int i, status = 0;
    char *e;
    const char *s;
    xData_2d_xindex_y *p;
    void *smrUser = xData_get_smrUserInterfaceFromElement( xDT->element );

    if( xDT->status != xData_xDataType_Ok ) return( xData_setMessageError_ReturnInt( 1, smr, smrUser, __FILE__, __LINE__, 1, "bad xDataType instance" ) );
    release( smr, xDT );
    if( ( xDT->data = xData_malloc2( smr, 2 * xDT->length * sizeof( xData_2d_xindex_y ), 0, "data" ) ) == NULL ) return( 1 );
    for( i = 0, s = text, p = (xData_2d_xindex_y *) xDT->data; i < xDT->length; i++, p++, s = e ) {
        if( xData_stringTo_xData_Int( smr, smrUser, s, &(p->index), " \n", &e ) ) { status = 1; break; }
        s = e;
        if( xData_stringTo_double( smr, smrUser, s, &(p->value), " \n", &e ) ) { status = 1; break; }
    }
    if( status == 0 ) {
        while( isspace( *e ) ) e++;
        if( *e != 0 ) {
            smr_setMessageError( smr, smrUser, __FILE__, __LINE__, 1, "2d_xindex_y contains extra data = %s", e );
            status = 1;
        }
    }
    if( status != 0 ) release( smr, xDT );
    return( status );
}
/*
************************************************************
*/
//static char *toString( statusMessageReporting *smr, xDataType *xDT ) {
static char *toString( statusMessageReporting *, xDataType *xDT ) {

    xData_Int i, n = xDT->length, recordSize = 5 + 16 + 1, indexMax = 9;
    int iFmt = 4;
    char *str, *p, fmt[32] = " %99d %15.7e\n";
    xData_2d_xindex_y *data = (xData_2d_xindex_y *) xDT->data;

    if( n < 0 ) n = 0;
    for( i = 0; i < n; i++, data++ ) {
        while( ( data->index > indexMax ) && ( indexMax > 0 ) ) {
            indexMax = 10 * indexMax + 9;
            recordSize++;
            iFmt++;
        }
    }
    sprintf( fmt, " %%%dld %%15.7e\n", iFmt );
    if( ( str = (char *) malloc( recordSize * ( n + 1 ) ) ) == NULL ) return( NULL );
    for( i = 0, p = str, data = (xData_2d_xindex_y *) xDT->data; i < n; i++, p += recordSize, data++ ) {
        sprintf( p, fmt, data->index, data->value );
    }
    *p = 0;
    return( str );
}
/*
************************************************************
*/
static int release( statusMessageReporting *smr, xDataType *xDT ) {

    if( xDT->data != NULL ) xDT->data = xData_free( smr, xDT->data );
    return( xDT->status = xData_xDataType_Ok );
}
/*
************************************************************
*/
xData_Int *xData_2d_xindex_y_rawIndices( statusMessageReporting *smr, xData_element *element ) {
/*
*   Returns NULL if length is 0 or memory could not be allocated.
*/
    xDataType *xDT = &(element->xDataTypeInfo);
    xData_Int i, index = xDT->start, *values;
    xData_2d_xindex_y *data = (xData_2d_xindex_y *) xDT->data;

    if( xDT->length == 0 ) return( NULL );
    //if( ( values = xData_malloc2( smr, xDT->length * sizeof( xData_Int ), 0, "values" ) ) == NULL ) return( NULL );
    if( ( values = (xData_Int*) xData_malloc2( smr, xDT->length * sizeof( xData_Int ), 0, "values" ) ) == NULL ) return( NULL );
    for( i = 0; i < xDT->length; i++ ) {
        values[i] = index;
        index += data[i].index;
    }
    return( values );
}
/*
************************************************************
*/
int xData_2d_xindex_y_free_rawIndices( statusMessageReporting *smr, void *data ) {

    xData_free( smr, data );
    return( 0 );
}
/*
************************************************************
*/
double *xData_2d_xindex_y_toXYs( statusMessageReporting *smr, xData_element *element, double *Xs ) {
/*
*   Returns NULL if length is 0 or memory could not be allocated.
*/
    xDataType *xDT = &(element->xDataTypeInfo);
    xData_Int i, index = xDT->start;
    double *values = NULL, *p;
    xData_2d_xindex_y *data = (xData_2d_xindex_y *) xDT->data;

    if( xDT->length == 0 ) return( NULL );
    //if( ( values = xData_malloc2( smr, 2 * xDT->length * sizeof( double ), 0, "values" ) ) == NULL ) return( NULL );
    if( ( values = (double*) xData_malloc2( smr, 2 * xDT->length * sizeof( double ), 0, "values" ) ) == NULL ) return( NULL );
    p = values;
    for( i = 0; i < xDT->length; i++, p++ ) {
        index += data[i].index;
        *p = Xs[index];
        p++;
        *p = data[i].value;
    }
    return( values );
}
/*
************************************************************
*/
double *xData_2d_xindex_y_toFilledYs( statusMessageReporting *smr, xData_element *element, double *Xs ) {

    return( xData_2d_xindex_y_toFilled( smr, element, Xs, 1 ) );
}
/*
************************************************************
*/
int xData_2d_xindex_y_free_toFilledYs( statusMessageReporting *smr, void *data ) {

    xData_free( smr, data );
    return( 0 );
}
/*
************************************************************
*/
double *xData_2d_xindex_y_toFilledXYs( statusMessageReporting *smr, xData_element *element, double *Xs ) {

    return( xData_2d_xindex_y_toFilled( smr, element, Xs, 2 ) );
}
/*
************************************************************
*/
static double *xData_2d_xindex_y_toFilled( statusMessageReporting *smr, xData_element *element, double *Xs, int size ) {
/*
*   Returns NULL if length is 0 or memory could not be allocated.
*/
    xDataType *xDT = &(element->xDataTypeInfo);
    xData_Int i, j, index = xDT->start, length = xDT->end - xDT->start;
    double x1, x2, *x, y1, y2, *values = NULL, *p;
    xData_2d_xindex_y *data = (xData_2d_xindex_y *) xDT->data;

    if( xDT->length == 0 ) return( NULL );
    //if( ( values = xData_malloc2( smr, size * length * sizeof( double ), 0, "values" ) ) == NULL ) return( NULL );
    if( ( values = (double*) xData_malloc2( smr, size * length * sizeof( double ), 0, "values" ) ) == NULL ) return( NULL );
    p = values;
    x = &(Xs[xDT->start]);
    x2 = 0.;                                        /* Dummy initializations, as x1 and y1 (set by x2 and y2) are not used first time thru loop. */
    y2 = 0.;
    for( i = 0; i < xDT->length; i++, x++, p++ ) {
        index += data[i].index;                     /* Note, data[0].index is 0; otherwise, following logic would not work. */
        x1 = x2;
        x2 = Xs[index];
        y1 = y2;
        y2 = data[i].value;
        for( j = data[i].index; j > 1; j--, x++, p++ ) {
            if( size == 2 ) {
                *p = *x;
                p++;
            }
            *p = ( y1 * ( x2 - *x ) + y2 * ( *x - x1 ) ) / ( x2 - x1 );
        }
        if( size == 2 ) *(p++) = *x;
        *p = y2;
    }
    return( values );
}

#if defined __cplusplus
}
#endif
