#include "hip/hip_runtime.h"
/*
# <<BEGIN-copyright>>
# Copyright (c) 2010, Lawrence Livermore National Security, LLC. 
# Produced at the Lawrence Livermore National Laboratory 
# Written by Bret R. Beck, beck6@llnl.gov. 
# CODE-461393
# All rights reserved. 
#  
# This file is part of GIDI. For details, see nuclear.llnl.gov. 
# Please also read the "Additional BSD Notice" at nuclear.llnl.gov. 
# 
# Redistribution and use in source and binary forms, with or without modification, 
# are permitted provided that the following conditions are met: 
#
#      1) Redistributions of source code must retain the above copyright notice, 
#         this list of conditions and the disclaimer below.
#      2) Redistributions in binary form must reproduce the above copyright notice, 
#         this list of conditions and the disclaimer (as noted below) in the 
#          documentation and/or other materials provided with the distribution.
#      3) Neither the name of the LLNS/LLNL nor the names of its contributors may be 
#         used to endorse or promote products derived from this software without 
#         specific prior written permission. 
#
# THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
# EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
# OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.  IN NO EVENT 
# SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC, THE U.S. DEPARTMENT OF ENERGY OR 
# CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
# CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS 
# OR SERVICES;  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED 
# AND ON  ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
# (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
# EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE. 
# <<END-copyright>>
*/
#include <iostream>
#include <float.h>
#include <string.h>
#include <cmath>
#include <tpia_target.h>

#if defined __cplusplus
namespace GIDI {
using namespace GIDI;
#endif 

/*
************************************************************
*/
int tpia_kinetics_2BodyReaction( statusMessageReporting *smr, tpia_decayChannel *decayChannel, double K, double mu, double phi, 
        tpia_productOutgoingData *outgoingData ) {

    tpia_product *pp3 = tpia_decayChannel_getFirstProduct( decayChannel ), *pp4;
    double m1 = decayChannel->m1_fullMass_MeV, m2 = decayChannel->m2_fullMass_MeV, m3, m4, mi, mf, Kp, x, beta;

    pp4 = tpia_decayChannel_getNextProduct( pp3 );
    m3 = pp3->productID->fullMass_MeV;
    m4 = pp4->productID->fullMass_MeV;
    mi = m1 + m2;
    mf = m3 + m4;
    beta = std::sqrt( K * ( K + 2. * m1 ) ) / ( K + mi );
    x = K * m2 / ( mi * mi );
    if( x < 2e-5 ) {                                        /* Kp is the total kinetic energy for m3 and m4 in the COM frame. */
        Kp = mi - mf + K * m2 / mi * ( 1 - 0.5 * x * ( 1 - x ) ); }
    else {
        Kp = std::sqrt( mi * mi + 2 * K * m2 ) - mf;
    }
    if( Kp < 0 ) Kp = 0.;           /* ???? There needs to be a better test here. */
    outgoingData[0].decayChannel = &(pp3->decayChannel);
    outgoingData[1].genre = outgoingData[0].genre;
    outgoingData[1].productID = pp4->productID;
    outgoingData[1].decayChannel = &(pp4->decayChannel);
    return( tpia_kinetics_COMKineticEnergy2LabEnergyAndMomentum( smr, beta, Kp, mu, phi, m3, m4, outgoingData ) );
}
/*
************************************************************
*/
//int tpia_kinetics_COMKineticEnergy2LabEnergyAndMomentum( statusMessageReporting *smr, double beta, double e_kinetic_com, double mu, double phi, 
int tpia_kinetics_COMKineticEnergy2LabEnergyAndMomentum( statusMessageReporting *, double beta, double e_kinetic_com, double mu, double phi, 
        double m3cc, double m4cc, tpia_productOutgoingData *outgoingData ) {
/*
*   beta            the velocity/speedOflight of the com frame relative to the lab frame.
*   e_kinetic_com   Total kinetic energy (K1 + K2) in the COM frame.
*   mu              std::cos( theta ) in the COM frame.
*/
    double x, v_p, p, pp3, pp4, px3, py3, pz3, pz4, pz, p_perp2, E3, E4, gamma, m3cc2 = m3cc * m3cc, m4cc2 = m4cc * m4cc;

    p = std::sqrt( e_kinetic_com * ( e_kinetic_com + 2. * m3cc ) * ( e_kinetic_com + 2. * m4cc )  * ( e_kinetic_com + 2. * ( m3cc + m4cc ) ) ) /
            ( 2. * ( e_kinetic_com + m3cc + m4cc ) );
    py3 = p * std::sqrt( 1 - mu * mu );
    px3 = py3 * std::cos( phi );
    py3 *= std::sin( phi );
    pz = p * mu;
    if( tpia_frame_getColumn( NULL, &(outgoingData[0].frame), 0 ) == tpia_referenceFrame_lab ) {
        E3 = std::sqrt( p * p + m3cc2 );
        E4 = std::sqrt( p * p + m4cc2 );
        gamma = std::sqrt( 1. / ( 1. - beta * beta ) );
        pz3 = gamma * (  pz + beta * E3 );
        pz4 = gamma * ( -pz + beta * E4 ); }
    else {
        pz3 = pz;
        pz4 = -pz;
    }
    outgoingData[1].isVelocity = outgoingData[0].isVelocity;
    outgoingData[1].frame = outgoingData[0].frame;

    p_perp2 = px3 * px3 + py3 * py3;

    outgoingData[0].px_vx = px3;
    outgoingData[0].py_vy = py3;
    outgoingData[0].pz_vz = pz3;
    pp3 = p_perp2 + pz3 * pz3;
//TK140602 Modified for protecting divided by 0 BEGIN
    if ( m3cc2 != 0 ) 
    x = pp3 / ( 2 * m3cc2 );
    else
    x = FLT_MIN;
//TK140602 Modified for protecting divided by 0 END
    if( x < 1e-5 ) {
        outgoingData[0].kineticEnergy = m3cc * x  * ( 1 - 0.5 * x * ( 1 - x ) ); }
    else {
        outgoingData[0].kineticEnergy = std::sqrt( m3cc2 + pp3 ) - m3cc;
    }
    outgoingData[1].px_vx = -px3;
    outgoingData[1].py_vy = -py3;
    outgoingData[1].pz_vz = pz4;
    pp4 = p_perp2 + pz4 * pz4;
    x = pp4 / ( 2 * m4cc2 );
    if( x < 1e-5 ) {
        outgoingData[1].kineticEnergy = m4cc * x  * ( 1 - 0.5 * x * ( 1 - x ) ); }
    else {
        outgoingData[1].kineticEnergy = std::sqrt( m4cc2 + pp4 ) - m4cc;
    }

    if( outgoingData[0].isVelocity ) {
        v_p = tpia_speedOfLight_cm_sec / std::sqrt( pp3 + m3cc2 );
        outgoingData[0].px_vx *= v_p;
        outgoingData[0].py_vy *= v_p;
        outgoingData[0].pz_vz *= v_p;

        v_p = tpia_speedOfLight_cm_sec / std::sqrt( pp4 + m4cc2 );
        outgoingData[1].px_vx *= v_p;
        outgoingData[1].py_vy *= v_p;
        outgoingData[1].pz_vz *= v_p;
    }

    return( 0 );
}

#if defined __cplusplus
}
#endif
