#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//

#include "globals.hh"
#include "G4ios.hh"
#include "G4HadronicException.hh"
#include "G4Clebsch.hh"
#include "Randomize.hh"
#include "G4Proton.hh"
#include "G4HadTmpUtil.hh"

G4Clebsch::G4Clebsch()
{
  G4int nLogs = 101;
  logs.push_back(0.);
  G4int i;
  for (i=1; i<nLogs; i++)
    {
      G4double previousLog = logs.back();
      G4double value = previousLog + std::log((G4double)i);
      logs.push_back(value);
    }
}


G4Clebsch::~G4Clebsch() 
{  }


G4bool G4Clebsch::operator==(const G4Clebsch &right) const
{
  return (this == (G4Clebsch *) &right);
}


G4bool G4Clebsch::operator!=(const G4Clebsch &right) const
{
  return (this != (G4Clebsch *) &right);
}


G4double G4Clebsch::Weight(G4int isoIn1,  G4int iso3In1, 
			   G4int isoIn2,  G4int iso3In2, 
			   G4int isoOut1, G4int isoOut2) const
{
  G4double value = 0.;
  
  G4int an_m = iso3In1 + iso3In2;

  G4int jMinIn = std::max(std::abs(isoIn1 - isoIn2), std::abs(an_m));
  G4int jMaxIn = isoIn1 + isoIn2;

  G4int jMinOut = std::max(std::abs(isoOut1 - isoOut2), std::abs(an_m));
  G4int jMaxOut = isoOut1 + isoOut2;

  G4int jMin = std::max(jMinIn,jMinOut);
  G4int jMax = std::min(jMaxIn,jMaxOut);

  G4int j;
  for (j=jMin; j<=jMax; j+=2)
  {
    value += ClebschGordan(isoIn1,iso3In1, isoIn2,iso3In2, j);
  }

  return value;
}


G4double G4Clebsch::ClebschGordan(G4int isoIn1, G4int iso3In1, 
				  G4int isoIn2, G4int iso3In2, 
				  G4int jOut) const
{
  // Calculates Clebsch-Gordan coefficient

  G4double j1 = isoIn1 / 2.0;
  G4double j2 = isoIn2 / 2.0;
  G4double j3 = jOut / 2.0;

  G4double m_1 = iso3In1 / 2.0;
  G4double m_2 = iso3In2 / 2.0;
  G4double m_3 = - (m_1 + m_2);

  G4int n = G4lrint(m_3+j1+j2+.1);
  G4double argument = 2. * j3 + 1.;
  if (argument < 0.) 
    throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::ClebschGordan - sqrt of negative argument");
  G4double coeff = std::sqrt(argument) / (std::pow(-1.,n));
  G4double clebsch = coeff * Wigner3J(j1,j2,j3, m_1,m_2,m_3);
  G4double value = clebsch * clebsch;

//   G4cout << "ClebschGordan(" 
// 	 << isoIn1 << "," << iso3In1 << ","
// 	 << isoIn2 << "," << iso3In2 << "," << jOut
// 	 << ") = " << value << G4endl;

  return value;
}


G4double G4Clebsch::Wigner3J(G4double j1, G4double j2, G4double j3, 
			     G4double m_1, G4double m_2, G4double m_3) const
{
  // Calculates Wigner 3-j symbols

  G4double value = 0.;

  G4double sigma = j1 + j2 + j3;
  std::vector<G4double> n;
  n.push_back(-j1 + j2 + j3);      // n0
  n.push_back(j1 - m_1);            // n1
  n.push_back(j1 + m_1);            // n2
  n.push_back(j1 - j2 + j3);       // n3
  n.push_back(j2 - m_2);            // n4
  n.push_back(j2 + m_2);            // n5
  n.push_back(j1 + j2 - j3);       // n6
  n.push_back(j3 - m_3);            // n7
  n.push_back(j3 + m_3);            // n8

  // Some preliminary checks

  G4bool ok = true;
  size_t i;
  for(i=1; i<=3; i++)
  {
    G4double sum1 = n[i-1] + n[i+2] + n[i+5];
    G4double sum2 = n[3*i-1] + n[3*i-2] + n[3*i-3];
    if (sum1 != sigma || sum2 != sigma) ok = false;
    G4int j;
    for(j=1; j<=3; j++) 
    {
      if (n[i+3*j-4] < 0.) ok = false; 
    }
  }

  if (ok)
  {
    G4int iMin = 1;
    G4int jMin = 1;
    G4double smallest = n[0];

    // Find the smallest n
    for (i=1; i<=3; i++)
    {
      G4int j;
      for (j=1; j<=3; j++)
      {
	if (n[i+3*j-4] < smallest)
	{
	  smallest = n[i+3*j-4];
	  iMin = i;
	  jMin = j;
	}
      }
    }

    G4int sign = 1;

    if(iMin > 1)
    {
      for(G4int j=1; j<=3; ++j)
      {
	G4double tmp = n[j*3-3];
	n[j*3-3] = n[iMin+j*3-4];
	n[iMin+j*3-4] = tmp;
      }
      sign = (G4int) std::pow(-1.,sigma);
    }

    if (jMin > 1)
    {
      for(i=1; i<=3; i++)
      {
	G4double tmp = n[i-1];
	n[i-1] = n[i+jMin*3-4];
	n[i+jMin*3-4] = tmp;
      }
      sign *= (G4int) std::pow(-1.,sigma);
    }

    const std::vector<G4double>& logVector = logs;//GetLogs();
    size_t n1 = G4lrint(n[0]);

    // Some boundary checks
    G4int logEntries = logVector.size() - 1;
    for (i=0; i<n.size(); i++)
    {
      if (n[i] < 0. || n[i] > logEntries)
	 throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::Wigner3J - Outside logVector boundaries, n");
    }

    G4double r1 = n[0];
    G4double r2 = n[3];
    G4double r3 = n[6];
    G4double r4 = n[1];
    G4double r5 = n[4];
    G4double r6 = n[7];
    G4double r7 = n[2];
    G4double r8 = n[5];
    G4double r9 = n[8];

    G4double l1 = logVector[(G4int)r1];
    G4double l2 = logVector[(G4int)r2];
    G4double l3 = logVector[(G4int)r3];
    G4double l4 = logVector[(G4int)r4];
    G4double l5 = logVector[(G4int)r5];
    G4double l6 = logVector[(G4int)r6];
    G4double l7 = logVector[(G4int)r7];
    G4double l8 = logVector[(G4int)r8];
    G4double l9 = logVector[(G4int)r9];

    G4double sigma1 = sigma + 1.;
    if (sigma1 < 0. || sigma1 > logEntries)
      throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::Wigner3J - Outside logVector boundaries, sigma");

    G4double ls = logVector[static_cast<G4int>(sigma1+.00001)];
    G4double hlp1 = (l2 + l3 + l4 +l7 -ls -l1 -l5 -l9 -l6 -l8) / 2.;
    G4int expon = static_cast<G4int>(r6 + r8+.00001);
    G4double sgn = std::pow(-1., expon);
    G4double coeff = std::exp(hlp1) * sgn;

    G4int n61 = static_cast<G4int>(r6 - r1+.00001);
    if (n61 < 0. || n61 > logEntries)
      throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::Wigner3J - Outside logVector boundaries, n61");
    G4int n81 = static_cast<G4int>(r8 - r1+.00001);
    if (n81 < 0. || n81 > logEntries)
      throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::Wigner3J - Outside logVector boundaries, n81");

    G4double hlp2 = l6 - logVector[n61] + l8 - logVector[n81];
    G4double sum = std::exp(hlp2);
    std::vector<G4double> S;
    S.push_back(sum);
    n1 = (size_t)r1;
    for (i=1; i<=n1; i++)
    {
      G4double last = S.back();
      G4double den = i * (r6 - r1 + i) * (r8 - r1 + i);
      if (den == 0) 
	throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::Wigner3J - divide by zero");
      G4double data = -last * (r1 + 1.0 - i) * (r5 + 1.0 - i) * (r9 + 1. - i) / den;
      S.push_back(data);
      sum += data;
    }
    value = coeff * sum * sign;
  } // endif ok
  else
  {
  }


//  G4cout << "Wigner3j(" 
//	 << j1 << "," << j2 << "," << j3 << "," 
//	 << m1 << "," << m2 << "," << m3 << ") = " 
//	 << value
//	 << G4endl;

  return value;
}



std::vector<G4double> G4Clebsch::GenerateIso3(G4int isoIn1, G4int iso3In1, 
						G4int isoIn2, G4int iso3In2, 
						G4int isoA,   G4int isoB) const
{
  std::vector<G4double> temp;

  // ---- Special cases first ----

  // Special case, both Jin are zero
  if (isoIn1 == 0 && isoIn2 == 0)
  {
    G4cout << "WARNING: G4Clebsch::GenerateIso3 - both isoIn are zero" << G4endl;
    temp.push_back(0.);
    temp.push_back(0.);
    return temp;
  }

  G4int iso3 = iso3In1 + iso3In2;

  // Special case, either Jout is zero
  if (isoA == 0)
  {  
    temp.push_back(0.);
    temp.push_back(iso3);
    return temp;
  }
  if (isoB == 0)
  {
    temp.push_back(iso3);
    temp.push_back(0.);
    return temp;
  }
  
  // Number of possible states, in 
  G4int jMinIn = std::max(std::abs(isoIn1 - isoIn2), std::abs(iso3));
  G4int jMaxIn = isoIn1 + isoIn2;

  // Number of possible states, out
    
  G4int jMinOut = 9999;
  G4int jTmp, i, j;
 
   for(i=-1; i<=1; i+=2)
   {
     for(j=-1; j<=1; j+=2)
     {
       jTmp= std::abs(i*isoA + j*isoB);
       if(jTmp < jMinOut) jMinOut = jTmp;
     }
   }
   jMinOut = std::max(jMinOut, std::abs(iso3));
   G4int jMaxOut = isoA + isoB;

   // Possible in and out common states 
   G4int jMin  =  std::max(jMinIn, jMinOut);
   G4int jMax  =  std::min(jMaxIn, jMaxOut);
   if (jMin > jMax)
   {
     throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - jMin > JMax");
   }
   
   // Number of possible isospins
   G4int nJ = (jMax - jMin) / 2 + 1;

   // A few consistency checks
   
   if ( (isoIn1 == 0 || isoIn2 == 0) && jMin != jMax )
     throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - J1 or J2 = 0, but jMin != JMax");

   // MGP ---- Shall it be a warning or an exception?
   if (nJ == 0)
     throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - nJ is zero, no overlap between in and out");

   // Loop over all possible combinations of isoIn1, isoIn2, iso3In11, iso3In2, jTot
   // to get the probability of each of the in-channel couplings

   std::vector<G4double> clebsch;

   for(j=jMin; j<=jMax; j+=2)
     {
       G4double cg = ClebschGordan(isoIn1, iso3In1, isoIn2, iso3In2, j);
       clebsch.push_back(cg);
     }     

   // Consistency check
   if (static_cast<G4int>(clebsch.size()) != nJ)
     throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - nJ inconsistency");

   G4double sum = clebsch[0];
   
   for (j=1; j<nJ; j++)
   {
     sum += clebsch[j];
   }
   // Consistency check
   if (sum <= 0.)
     throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - Sum of Clebsch-Gordan probabilities <=0");

   // Generate a normalized pdf 

   std::vector<G4double> clebschPdf;
   G4double previous = clebsch[0];
   clebschPdf.push_back(previous/sum);
   for (j=1; j<nJ; j++)
   {
     previous += clebsch[j];
     G4double prob = previous / sum;
     clebschPdf.push_back(prob);
   }

   // Generate a random jTot according to the Clebsch-Gordan pdf
   G4double rand = G4UniformRand();
   G4int jTot = 0;
   for (j=0; j<nJ; j++)
   {
     G4bool found = false;
     if (rand < clebschPdf[j])
     {
       found = true;
       jTot = jMin + 2*j;
     }
     if (found) break;
   }

   // Generate iso3Out

   std::vector<G4double> mMin;
   mMin.push_back(-isoA);
   mMin.push_back(-isoB);

   std::vector<G4double> mMax;
   mMax.push_back(isoA);
   mMax.push_back(isoB);

   // Calculate the possible |J_i M_i> combinations and their probability

   std::vector<G4double> m1Out;
   std::vector<G4double> m2Out;

   const G4int size = 20;
   G4double prbout[size][size];

   G4int m1pos(0), m2pos(0);
   G4int j12;
   G4int m1pr(0), m2pr(0);

   sum = 0.;
   for(j12 = std::abs(isoA-isoB); j12<=(isoA+isoB); j12+=2)
   {
     m1pos = -1;
     for (m1pr = static_cast<G4int>(mMin[0]+.00001); m1pr <= mMax[0]; m1pr+=2)
     {
       m1pos++;
       if (m1pos >= size)
	 throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - m1pos > size");
       m1Out.push_back(m1pr);
       m2pos = -1;
       for (m2pr = static_cast<G4int>(mMin[1]+.00001); m2pr <= mMax[1]; m2pr+=2)
       {
	 m2pos++;
	 if (m2pos >= size)
	 {
	   throw G4HadronicException(__FILE__, __LINE__,  "G4Clebsch::GenerateIso3 - m2pos > size");
	 }
	 m2Out.push_back(m2pr);

	 if(m1pr + m2pr == iso3) 
	 {
	   G4int m12 = m1pr + m2pr;
	   G4double c12 = ClebschGordan(isoA, m1pr, isoB,m2pr, j12);
	   G4double c34 = ClebschGordan(0,0,0,0,0);
           G4double ctot = ClebschGordan(j12, m12, 0, 0, jTot);
	   G4double cleb = c12*c34*ctot;
	   prbout[m1pos][m2pos] = cleb;
	   sum += cleb;
	 }
	 else
	 {
	   prbout[m1pos][m2pos] = 0.;
	 }
       }
     }
   }
   
   if (sum <= 0.)
     throw G4HadronicException(__FILE__, __LINE__, "G4Clebsch::GenerateIso3 - sum (out) <=0");

   for (i=0; i<size; i++)
   {
     for (j=0; j<size; j++)
     {
       prbout[i][j] /= sum;
     }
   }

   rand = G4UniformRand();

   G4int m1p, m2p;

   for (m1p=0; m1p<m1pos; m1p++)
   {
     for (m2p=0; m2p<m2pos; m2p++)
     {
       if (rand < prbout[m1p][m2p])
       {
	 temp.push_back(m1Out[m1p]);
	 temp.push_back(m2Out[m2p]);
	 return temp;
       }   
       else
       {
	 rand -= prbout[m1p][m2p];
       }
     }     
   }   

  throw G4HadronicException(__FILE__, __LINE__,  "Should never get here ");
  return temp;
}


G4double G4Clebsch::NormalizedClebschGordan(G4int J, G4int M, 
					    G4int J1, G4int J2,
					    G4int m_1, G4int m_2) const
{
  // Calculate the normalized Clebsch-Gordan coefficient, that is the prob 
  // of isospin decomposition of (J,m) into J1, J2, m1, m2

  G4double cleb = 0.;

  if(J1 == 0 || J2 == 0) return cleb; 
  
  G4double sum = 0.0;

  // Loop over all J1,J2,Jtot,m1,m2 combinations

  for(G4int m1Current=-J1; m1Current<=J1;  m1Current+=2) 
    {
      G4int m2Current = M - m1Current;
      
      G4double prob = ClebschGordan(J1, m1Current, J2, m2Current, J);
      sum += prob;
      if (m2Current == m_2 && m1Current == m_1) cleb += prob;
    }

  // Normalize probs to 1 
  if (sum > 0.) cleb /= sum; 

  return cleb;
}
