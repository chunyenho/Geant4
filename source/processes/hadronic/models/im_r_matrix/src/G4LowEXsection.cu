#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//

#include <cmath>

#include "G4LowEXsection.hh"
#include "G4SystemOfUnits.hh"

G4double G4LowEXsection::
 CrossSection(G4double aX) const
 {
   G4double result = 0;
   if(aX<front().first) return 0;
   G4LowEXsection::const_iterator i;
   G4LowEXsection::const_iterator it=end();
   for(i=begin(); i!=end(); i++)
   {
     if((*i).first/MeV>aX) break;
     it = i;
   }
   G4double x1 = std::log((*it).first);
   G4double x2 = std::log((*(it+1)).first);
   G4double y1 = std::log((*it).second);
   G4double y2 = std::log((*(it+1)).second);
   G4double x = std::log(aX);
   G4double y = y1+(x-x1)*(y2-y1)/(x2-x1);
   result = std::exp(y);
   return result*millibarn;
 }
