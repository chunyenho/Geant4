#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4PreCompoundNeutron.cc 90591 2015-06-04 13:45:29Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4PreCompoundNeutron
//
// Author:         V.Lara
//
// Modified:  
// 21.08.2008 J. M. Quesada add choice of options  
// 10.02.2009 J. M. Quesada set default opt3
// 20.08.2010 V.Ivanchenko added G4Pow and G4PreCompoundParameters pointers
//                         use int Z and A and cleanup
// 

#include "G4PreCompoundNeutron.hh"
#include "G4SystemOfUnits.hh"
#include "G4Neutron.hh"

G4PreCompoundNeutron::G4PreCompoundNeutron()
  : G4PreCompoundNucleon(G4Neutron::Neutron(), &theNeutronCoulombBarrier)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentAthrd = ResidualAthrd;
  FragmentA = theA + ResidualA;
}

G4PreCompoundNeutron::~G4PreCompoundNeutron()
{}

G4double G4PreCompoundNeutron::GetRj(G4int nParticles, G4int nCharged)
{
  G4double rj = 0.0;
  if(nParticles > 0) { 
    rj = static_cast<G4double>(nParticles - nCharged)/
      static_cast<G4double>(nParticles);
  }
  return rj;
}

////////////////////////////////////////////////////////////////////////////////////
//J. M. Quesada (Dec 2007-June 2008): New inverse reaction cross sections 
//OPT=0 Dostrovski's parameterization
//OPT=1,2 Chatterjee's paramaterization 
//OPT=3,4 Kalbach's parameterization 
// 
G4double G4PreCompoundNeutron::CrossSection(const  G4double K)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentA = theA + ResidualA;
  FragmentAthrd = g4pow->Z13(FragmentA);

  if (OPTxs==0)        { return GetOpt0( K); }
  else if( OPTxs <= 2) { return GetOpt12( K); }
  else                 { return GetOpt34( K); }
}

G4double G4PreCompoundNeutron::GetAlpha()
{
  return 0.76+2.2/ResidualAthrd;
}

G4double G4PreCompoundNeutron::GetBeta() 
{
  //   return (2.12/std::pow(GetRestA(),2.0/3.0)-0.05)*MeV/GetAlpha();
  return (2.12/(ResidualAthrd*ResidualAthrd)-0.05)*MeV/GetAlpha();
}

//********************* OPT=1,2 : Chatterjee's cross section ***************
//(fitting to cross section from Bechetti & Greenles OM potential)

G4double G4PreCompoundNeutron::GetOpt12(G4double K)
{
  G4double Kc=K;

  // Pramana (Bechetti & Greenles) for neutrons is chosen 

  // JMQ  xsec is set constat above limit of validity
  if (K > 50*MeV) { Kc = 50*MeV; }

  const G4double landa0 = 18.57;
  const G4double landa1 = -22.93;
  const G4double mm0 = 381.7;
  const G4double mu1 = 24.31;
  const G4double nu0 = 0.172;
  const G4double nu1 = -15.39;
  const G4double nu2 = 804.8;

  G4double landa = landa0/ResidualAthrd + landa1;
  G4double mu = mm0*ResidualAthrd + mu1*ResidualAthrd*ResidualAthrd;
  G4double nu = nu0*ResidualAthrd*ResidualA + nu1*ResidualAthrd*ResidualAthrd + nu2 ;
  G4double xs = landa*Kc + mu + nu/Kc;

  xs = std::max(xs, 0.0);
  return xs;
}

// *********** OPT=3,4 : Kalbach's cross sections (from PRECO code)*************
G4double G4PreCompoundNeutron::GetOpt34(G4double K)
{

  const G4double flow = 1.e-18;

  // PRECO xs for neutrons is choosen
  const G4double p0 = -312.;
  const G4double landa0 = 12.10;
  const G4double landa1=  -11.27;
  const G4double mm0 = 234.1;
  const G4double mu1 = 38.26;
  const G4double nu0 = 1.55;
  const G4double nu1 = -106.1;
  const G4double nu2 = 1280.8; 
  const G4double ra  = 0.0;

  G4double signor = 1.0;
  if(ResidualA < 40)       { signor =0.7 + ResidualA*0.0075; }
  else if(ResidualA > 210) { signor = 1. + (ResidualA-210)/250.; }

  G4double landa = landa0/ResidualAthrd + landa1;
  G4double mu = mm0*ResidualAthrd + mu1*ResidualAthrd*ResidualAthrd;
  G4double nu = nu0*ResidualAthrd*ResidualA + nu1*ResidualAthrd*ResidualAthrd + nu2;

  // JMQ very low energy behaviour corrected (problem  for A (apprx.)>60)
  if (nu < 0.) { nu = -nu; }

  G4double ec = 0.5;
  G4double ecsq = 0.25;
  G4double p = p0;
  G4double xnulam = 1.;
  G4double etest = 32.;
  //          ** etest is the energy above which the rxn cross section is
  //          ** compared with the geometrical limit and the max taken.
  //          ** xnulam here is a dummy value to be used later.

  G4double a = -2.*p*ec + landa - nu/ecsq;
  G4double b = p*ecsq + mu + 2.*nu/ec;
  G4double ecut = 0.;
  G4double cut = a*a - 4.*p*b;
  if (cut > 0.) { ecut = std::sqrt(cut); }
  ecut = (ecut-a) / (2*p);
  if (cut < 0.) { ecut -= 2.; }

  G4double elab = K * FragmentA / G4double(ResidualA);
  G4double sig = 0.;

  if (elab <= ec) { 
    if (elab > ecut) { sig = std::max(0.0,(p*elab*elab+a*elab+b) * signor); }

  } else {           
    sig = (landa*elab+mu+nu/elab) * signor;
    G4double geom = 0.;
    if (xnulam >= flow && elab >= etest) { 
      geom = std::sqrt(theA*K);
      geom = 1.23*ResidualAthrd + ra + 4.573/geom;
      geom = 31.416 * geom * geom;
    }
    sig = std::max(geom,sig);
  } 
  return sig; 
}
