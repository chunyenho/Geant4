//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4VPreCompoundFragment.cc 68028 2013-03-13 13:48:15Z gcosmo $
//
// J. M. Quesada (August 2008).  Based  on previous work by V. Lara
//
// Modified:
// 20.08.2010 V.Ivanchenko added G4Pow and G4PreCompoundParameters pointers
//                         use int Z and A and cleanup

#include "G4VPreCompoundFragment.hh"
#include "G4SystemOfUnits.hh"
#include "G4PreCompoundParameters.hh"
#include "G4NucleiProperties.hh"

G4VPreCompoundFragment::G4VPreCompoundFragment(
  const G4ParticleDefinition* part, G4VCoulombBarrier* aCoulombBarrier)
  : particle(part), theCoulombBarrierPtr(aCoulombBarrier),
    theRestNucleusA(0),theRestNucleusZ(0),theBindingEnergy(0.0), 
    theMaximalKineticEnergy(-MeV),theRestNucleusMass(0.0),
    theReducedMass(0.0),theMomentum(0.,0.,0.,0.),
    theEmissionProbability(0.0),theCoulombBarrier(0.0),
    OPTxs(3),useSICB(false)
{
  theA = particle->GetBaryonNumber();
  theZ = G4int(particle->GetPDGCharge()/eplus + 0.1);
  theMass = particle->GetPDGMass();
  theParameters = new G4PreCompoundParameters();
  g4pow = G4Pow::GetInstance();
  theRestNucleusA13 = 0;
}

G4VPreCompoundFragment::~G4VPreCompoundFragment()
{
  delete theParameters;
}

std::ostream& 
operator << (std::ostream &out, const G4VPreCompoundFragment &theFragment)
{
  out << &theFragment;
  return out; 
}

std::ostream& 
operator << (std::ostream &out, const G4VPreCompoundFragment *theFragment)
{
  out 
    << "PreCompoundModel Emitted Fragment: Z= " << theFragment->GetZ() 
    << " A= " << theFragment->GetA()
    << " Mass(GeV)= " << theFragment->GetNuclearMass()/CLHEP::GeV;
  return out;
}

void 
G4VPreCompoundFragment::Initialize(const G4Fragment & aFragment)
{
  theRestNucleusA = aFragment.GetA_asInt() - theA;
  theRestNucleusZ = aFragment.GetZ_asInt() - theZ;

  if ((theRestNucleusA < theRestNucleusZ) ||
      (theRestNucleusA < theA) ||
      (theRestNucleusZ < theZ)) 
    {
      // In order to be sure that emission probability will be 0.
      theMaximalKineticEnergy = 0.0;
      return;
    }

  theRestNucleusA13 = g4pow->Z13(theRestNucleusA);
    
  // Calculate Coulomb barrier
  theCoulombBarrier = theCoulombBarrierPtr->
    GetCoulombBarrier(theRestNucleusA,theRestNucleusZ,
		      aFragment.GetExcitationEnergy());

  // Calculate masses
  theRestNucleusMass = 
    G4NucleiProperties::GetNuclearMass(theRestNucleusA, theRestNucleusZ);
  theReducedMass = theRestNucleusMass*theMass/(theRestNucleusMass + theMass);

  // Compute Binding Energies for fragments 
  // needed to separate a fragment from the nucleus
  theBindingEnergy = 
    theRestNucleusMass + theMass - aFragment.GetGroundStateMass();
    
  // Compute Maximal Kinetic Energy which can be carried by fragments 
  // after separation - the true assimptotic value
  G4double Ecm  = aFragment.GetMomentum().m();
  theMaximalKineticEnergy = 
    ((Ecm-theRestNucleusMass)*(Ecm+theRestNucleusMass) + theMass*theMass)/(2.0*Ecm)-theMass;
}
