#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4PreCompoundIon.cc 82765 2014-07-08 14:25:39Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4PreCompoundIon
//
// Author:         V.Lara
//
// Modified:  
// 10.02.2009 J. M. Quesada fixed bug in level density of light fragments  
// 20.08.2010 V.Ivanchenko added G4Pow and G4PreCompoundParameters pointers
//                         use int Z and A and cleanup
//

#include "G4PreCompoundIon.hh"
#include "G4PhysicalConstants.hh"

G4PreCompoundIon::
G4PreCompoundIon(const G4ParticleDefinition* part,
		 G4VCoulombBarrier* aCoulombBarrier)
  : G4PreCompoundFragment(part,aCoulombBarrier)
{
  G4double r0 = theParameters->Getr0();
  fact = 0.75*CLHEP::millibarn/(CLHEP::pi*r0*r0*r0);
}

G4PreCompoundIon::~G4PreCompoundIon()
{}

G4double G4PreCompoundIon::
ProbabilityDistributionFunction(G4double eKin, 
				const G4Fragment& aFragment)
{
  if ( !IsItPossible(aFragment) ) { return 0.0; }
  G4double efinal = eKin + GetBindingEnergy();
  if(efinal <= 0.0 ) { return 0.0; } 

  G4double U = aFragment.GetExcitationEnergy();
  G4int P = aFragment.GetNumberOfParticles();
  G4int H = aFragment.GetNumberOfHoles();
  G4int A = GetA();
  G4int N = P + H;

  G4double g0 = (6.0/pi2)*aFragment.GetA_asInt()*theParameters->GetLevelDensity();
  G4double g1 = (6.0/pi2)*GetRestA()*theParameters->GetLevelDensity();

  // G4double gj = (6.0/pi2)*GetA() *
  //   G4PreCompoundParameters::GetAddress()->GetLevelDensity();

  G4double gj = g1;

  G4double A0 = G4double(P*P+H*H+P-3*H)/(4.0*g0);
  G4double A1 = std::max(0.0,(A0*g0 + A*(A-2*P-1)*0.25)/g1); 

  G4double E0 = U - A0;
  if (E0 <= 0.0) { return 0.0; }

  G4double E1 = (std::max(0.0,GetMaximalKineticEnergy() - eKin - A1)); 

  G4double Aj = A*(A+1)/(4.0*gj); 
  G4double Ej = std::max(0.0,efinal - Aj); 

  G4double rj = GetRj(P, aFragment.GetNumberOfCharged());
  G4double xs = CrossSection(eKin);
  
  G4double pA = fact*eKin*xs*rj 
    * CoalescenceFactor(aFragment.GetA_asInt()) * FactorialFactor(N,P)
    * std::sqrt(2.0/(GetReducedMass()*efinal)) 
    * g4pow->powN(g1*E1/(g0*E0), N-A-1)
    * g4pow->powN(gj*Ej/(g0*E0), A-1)*gj*g1/(g0*g0*E0*GetRestA()); 
   
  return pA;
}
