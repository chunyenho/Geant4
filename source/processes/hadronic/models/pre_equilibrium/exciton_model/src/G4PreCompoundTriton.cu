#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4PreCompoundTriton.cc 90591 2015-06-04 13:45:29Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4PreCompoundTriton
//
// Author:         V.Lara
//
// Modified:  
// 21.08.2008 J. M. Quesada add choice of options  
// 20.08.2010 V.Ivanchenko added G4Pow and G4PreCompoundParameters pointers
//                         use int Z and A and cleanup
// 05.07.2013 J.M. Quesada FactorialFactor fixed
//
 
#include "G4PreCompoundTriton.hh"
#include "G4SystemOfUnits.hh"
#include "G4Triton.hh"

G4PreCompoundTriton::G4PreCompoundTriton()
  : G4PreCompoundIon(G4Triton::Triton(), &theTritonCoulombBarrier)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentAthrd = ResidualAthrd;
  FragmentA = theA + ResidualA;
}

G4PreCompoundTriton::~G4PreCompoundTriton()
{}

G4double G4PreCompoundTriton::FactorialFactor(G4int N, const G4int P)
{
  return G4double((N-3)*(P-2)*(N-2)*(P-1)*(N-1)*P)/12.0; 
}
  
G4double G4PreCompoundTriton::CoalescenceFactor(G4int A)
{
  return 243.0/G4double(A*A);
}    

G4double G4PreCompoundTriton::GetRj(G4int nParticles, G4int nCharged)
{
  G4double rj = 0.0;
  if(nCharged >= 1 && (nParticles-nCharged) >= 2) {
    G4double denominator = 
      G4double(nParticles*(nParticles-1)*(nParticles-2));
    rj = G4double(3*nCharged*(nParticles-nCharged)*(nParticles-nCharged-1))
      /denominator; 
  }
  return rj;
}

//////////////////////////////////////////////////////////////////////////////////
//J. M. Quesada (Dec 2007-June 2008): New inverse reaction cross sections 
//OPT=0 Dostrovski's parameterization
//OPT=1,2 Chatterjee's paramaterization 
//OPT=3,4 Kalbach's parameterization 
// 
G4double G4PreCompoundTriton::CrossSection(G4double K)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentA = theA + ResidualA;
  FragmentAthrd = g4pow->Z13(FragmentA);

  if (OPTxs==0) { return GetOpt0( K); }
  else if( OPTxs==1 || OPTxs==2) { return GetOpt12( K); }
  else if (OPTxs==3 || OPTxs==4) { return GetOpt34( K); }
  else{
    std::ostringstream errOs;
    errOs << "BAD TRITON CROSS SECTION OPTION !!"  <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, errOs.str());
    return 0.;
  }
}

G4double G4PreCompoundTriton::GetAlpha()
{
  G4double C = 0.0;
  G4int aZ = theZ + ResidualZ;
  if (aZ >= 70) 
    {
      C = 0.10;
    } 
  else 
    {
      C = ((((0.15417e-06*aZ) - 0.29875e-04)*aZ + 0.21071e-02)*aZ - 0.66612e-01)*aZ + 0.98375; 
    }
 
  return 1.0 + C/3.0;
}

//
//********************* OPT=1,2 : Chatterjee's cross section *****************
//(fitting to cross section from Bechetti & Greenles OM potential)

G4double G4PreCompoundTriton::GetOpt12(G4double K)
{
  G4double Kc=K;

  // JMQ xsec is set constat above limit of validity
  if (K > 50*MeV) { Kc=50*MeV; }

  G4double landa ,mu ,nu ,p , Ec,q,r,ji,xs;
 
  const G4double p0 = -11.04;
  const G4double p1 = 619.1;
  const G4double p2 = -2147.;
  const G4double landa0 = -0.0426;
  const G4double landa1 = -10.33;
  const G4double mm0 = 601.9;
  const G4double mu1 = 0.37;
  const G4double nu0 = 583.0;
  const G4double nu1 = -546.2;
  const G4double nu2 = 1.718;  
  const G4double delta=1.2;            

  Ec = 1.44*theZ*ResidualZ/(1.5*ResidualAthrd+delta);
  p = p0 + p1/Ec + p2/(Ec*Ec);
  landa = landa0*ResidualA + landa1;

  G4double resmu1 = g4pow->powZ(ResidualA,mu1); 
  mu = mm0*resmu1;
  nu = resmu1*(nu0 + nu1*Ec + nu2*(Ec*Ec));
  q = landa - nu/(Ec*Ec) - 2*p*Ec;
  r = mu + 2*nu/Ec + p*(Ec*Ec);
  
  ji=std::max(Kc,Ec);
  if(Kc < Ec) { xs = p*Kc*Kc + q*Kc + r;}
  else {xs = p*(Kc - ji)*(Kc - ji) + landa*Kc + mu + nu*(2 - Kc/ji)/ji ;}
                               
  xs = std::max(xs, 0.0);
  return xs;
}

// *********** OPT=3,4 : Kalbach's cross sections (from PRECO code)*************
G4double G4PreCompoundTriton::GetOpt34(G4double K)
//     ** t from o.m. of hafele, flynn et al
{
  const G4double  flow = 1.e-18;
  const G4double  spill= 1.e+18;

  const G4double  p0 = -21.45;
  const G4double  p1 = 484.7;
  const G4double  p2 = -1608.;
  const G4double  landa0 = 0.0186;
  const G4double  landa1 = -8.90;
  const G4double  mm0 = 686.3;
  const G4double  mu1 = 0.325;
  const G4double  nu0 = 368.9;
  const G4double  nu1 = -522.2;
  const G4double  nu2 = -4.998;  
  
  const G4double  ra = 0.80;
  const G4double  signor = 1.0;
        
  //JMQ 13/02/09 increase of reduced radius to lower the barrier
  // ec = 1.44 * theZ * ResidualZ / (1.5*ResidualAthrd+ra);
  G4double ec = 1.44 * theZ * ResidualZ / (1.7*ResidualAthrd+ra);
  G4double ecsq = ec * ec;
  G4double p = p0 + p1/ec + p2/ecsq;
  G4double landa = landa0*ResidualA + landa1;
  G4double a = g4pow->powZ(ResidualA,mu1);
  G4double mu = mm0 * a;
  G4double nu = a* (nu0+nu1*ec+nu2*ecsq);  
  G4double xnulam = nu / landa;
  G4double etest = 0.0;
  if (xnulam > spill)      { xnulam=0.; }
  else if (xnulam >= flow) { etest = 1.2 *std::sqrt(xnulam); }
 
  a = -2.*p*ec + landa - nu/ecsq;
  G4double b = p*ecsq + mu + 2.*nu/ec;
  G4double ecut = 0.;
  G4double cut = a*a - 4.*p*b;
  if (cut > 0.) { ecut = std::sqrt(cut); }
  ecut = (ecut-a) / (2*p);
 
  //JMQ 290310 for avoiding unphysical increase below minimum (at ecut)
  // ecut<0 means that there is no cut with energy axis, i.e. xs is set 
  // to 0 bellow minimum

  G4double elab = K * FragmentA / G4double(ResidualA);
  G4double sig = 0.;

  if (elab <= ec) { 
    if (elab > ecut) { sig = std::max(0.0,(p*elab*elab+a*elab+b) * signor); }

  } else {           
    sig = (landa*elab+mu+nu/elab) * signor;
    G4double geom = 0.;
    if (xnulam >= flow && elab >= etest) { 
      geom = std::sqrt(theA*K);
      geom = 1.23*ResidualAthrd + ra + 4.573/geom;
      geom = 31.416 * geom * geom;
    }
    sig = std::max(geom,sig);
  }          
  return sig;
}
