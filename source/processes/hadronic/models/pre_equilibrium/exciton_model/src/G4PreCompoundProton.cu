#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4PreCompoundProton.cc 90591 2015-06-04 13:45:29Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4PreCompoundProton
//
// Author:         V.Lara
//
// Modified:  
// 21.08.2008 J. M. Quesada added external choice of inverse cross section option
// 21.08.2008 J. M. Quesada added external choice for superimposed Coulomb 
//                          barrier (if useSICB=true) 
// 20.08.2010 V.Ivanchenko added G4Pow and G4PreCompoundParameters pointers
//                         use int Z and A and cleanup
//

#include "G4PreCompoundProton.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4Proton.hh"
#include "G4Log.hh"
#include "G4Exp.hh"

G4PreCompoundProton::G4PreCompoundProton()
  : G4PreCompoundNucleon(G4Proton::Proton(), &theProtonCoulombBarrier)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentAthrd = ResidualAthrd;
  FragmentA = theA + ResidualA;
}

G4PreCompoundProton::~G4PreCompoundProton()
{}

G4double G4PreCompoundProton::GetRj(G4int nParticles, G4int nCharged)
{
  G4double rj = 0.0;
  if(nParticles > 0) { 
    rj = static_cast<G4double>(nCharged)/static_cast<G4double>(nParticles);
  }
  return rj;
}

////////////////////////////////////////////////////////////////////////////////////
//J. M. Quesada (Dec 2007-June 2008): New inverse reaction cross sections 
//OPT=0 Dostrovski's parameterization
//OPT=1 Chatterjee's paramaterization 
//OPT=2,4 Wellisch's parametarization
//OPT=3 Kalbach's parameterization 
// 
G4double G4PreCompoundProton::CrossSection(G4double K)
{
  ResidualA = GetRestA();
  ResidualZ = GetRestZ(); 
  theA = GetA();
  theZ = GetZ();
  ResidualAthrd = ResidualA13();
  FragmentA = theA + ResidualA;
  FragmentAthrd = g4pow->Z13(FragmentA);

  if (OPTxs==0)        { return GetOpt0( K); }
  else if( OPTxs == 1) { return GetOpt1( K); }
  else if( OPTxs == 2) { return GetOpt2( K); }
  else                 { return GetOpt3( K); }
}

G4double G4PreCompoundProton::GetAlpha()
{
  G4int aZ = ResidualZ;
  G4double C = 0.0;
  if (aZ >= 70) 
    {
      C = 0.10;
    } 
  else 
    {
      C = ((((0.15417e-06*aZ) - 0.29875e-04)*aZ + 0.21071e-02)*aZ 
	   - 0.66612e-01)*aZ + 0.98375;
    }
  return 1.0 + C;
}

G4double G4PreCompoundProton::GetBeta() 
{
  return -GetCoulombBarrier();
}
  
//********************* OPT=1 : Chatterjee's cross section *********************
//(fitting to cross section from Bechetti & Greenles OM potential)

G4double G4PreCompoundProton::GetOpt1(G4double K)
{
  G4double Kc=K; 

  // JMQ  xsec is set constat above limit of validity
  if (K > 50*MeV) { Kc = 50*MeV; }

  const G4double p0 = 15.72;
  const G4double p1 = 9.65;
  const G4double p2 = -449.0;
  const G4double landa0 = 0.00437;
  const G4double landa1 = -16.58;
  const G4double mm0 = 244.7;
  const G4double mu1 = 0.503;
  const G4double nu0 = 273.1;
  const G4double nu1 = -182.4;
  const G4double nu2 = -1.872;  
  const G4double delta = 0.;  

  G4double Ec = 1.44*theZ*ResidualZ/(1.5*ResidualAthrd+delta);
  G4double p = p0 + p1/Ec + p2/(Ec*Ec);
  G4double landa = landa0*ResidualA + landa1;

  G4double resmu1 = g4pow->powZ(ResidualA,mu1); 
  G4double mu = mm0*resmu1;
  G4double nu = resmu1*(nu0 + nu1*Ec + nu2*(Ec*Ec));
  G4double q = landa - nu/(Ec*Ec) - 2*p*Ec;
  G4double r = mu + 2*nu/Ec + p*(Ec*Ec);

  G4double ji = std::max(Kc,Ec);
  G4double xs = 0.0;

  if(Kc < Ec) { xs = p*Kc*Kc + q*Kc + r;}
  else {xs = p*(Kc - ji)*(Kc - ji) + landa*Kc + mu + nu*(2 - Kc/ji)/ji ;}

  xs = std::max(xs, 0.0);
  return xs; 
}

//************* OPT=2 : Welisch's proton reaction cross section ***************

G4double G4PreCompoundProton::GetOpt2(G4double K)
{
  // This is redundant when the Coulomb  barrier is overimposed to all 
  // cross sections 
  // It should be kept when Coulomb barrier only imposed at OPTxs=2

  if(!useSICB && K<=theCoulombBarrier) { return 0.0; }

  G4double eekin=K;
  G4int rnneu=ResidualA-ResidualZ;
  G4double ekin=eekin/1000;
  G4double r0=1.36*1.e-15;
  G4double fac=pi*r0*r0;
  G4double b0=2.247-0.915*(1.-1./ResidualAthrd);
  G4double fac1=b0*(1.-1./ResidualAthrd);
  G4double fac2=1.;
  if(rnneu > 1.5) { fac2 = g4pow->logZ(rnneu); }
  G4double xine_th= 1.e+31*fac*fac2*(1.+ResidualAthrd-fac1);
  xine_th=(1.-0.15*G4Exp(-ekin))*xine_th/(1.00-0.0007*ResidualA);	
  G4double ff1=0.70-0.0020*ResidualA;
  G4double ff2=1.00+1/G4double(ResidualA);
  G4double ff3=0.8+18/G4double(ResidualA)-0.002*ResidualA;
  G4double log10E = G4Log(ekin)/g4pow->logZ(10);
  fac=1.-(1./(1.+G4Exp(-8.*ff1*(log10E + 1.37*ff2))));
  xine_th=xine_th*(1.+ff3*fac);
  ff1=1.-1/G4double(ResidualA)-0.001*ResidualA;
  ff2=1.17-2.7/G4double(ResidualA)-0.0014*ResidualA;
  fac=-8.*ff1*(log10E + 2.0*ff2);
  xine_th /= (1.+G4Exp(fac));    
        
  xine_th = std::max(xine_th, 0.0);
  return xine_th;
}

// *********** OPT=3 : Kalbach's cross sections (from PRECO code)*************
G4double G4PreCompoundProton::GetOpt3(const  G4double K)
{
  //     ** p from  becchetti and greenlees (but modified with sub-barrier
  //     ** correction function and xp2 changed from -449)

  const G4double p0 = 15.72;
  const G4double p1 = 9.65;
  const G4double p2 = -300.;
  const G4double landa0 = 0.00437;
  const G4double landa1 = -16.58;
  const G4double mm0 = 244.7;
  const G4double mu1 = 0.503;
  const G4double nu0 = 273.1;
  const G4double nu1 = -182.4;
  const G4double nu2 = -1.872;
  
  const G4double flow  = 1.e-18;
  const G4double spill = 1.e+18; 
  const G4double ra = 0.0; 
   
  G4double signor = 1.0;
  if (ResidualA <= 60)      { signor = 0.92; }
  else if (ResidualA < 100) { signor = 0.8 + ResidualA*0.002; }
  
  G4double ec = 1.44 * theZ * ResidualZ / (1.5*ResidualAthrd+ra);
  G4double ecsq = ec * ec;
  G4double p = p0 + p1/ec + p2/ecsq;
  G4double landa = landa0*ResidualA + landa1;
  G4double a = g4pow->powZ(ResidualA,mu1);
  G4double mu = mm0 * a;
  G4double nu = a* (nu0+nu1*ec+nu2*ecsq);
  
  G4double c =std::min(3.15,ec*0.5);
  G4double w = 0.7 * c / 3.15; 
  
  G4double etest = 0.0;
  G4double xnulam = nu / landa;
  if(xnulam > spill)      { xnulam=0.; }
  else if(xnulam >= flow) { etest = std::sqrt(xnulam) + 7.; }
  
  a = -2.*p*ec + landa - nu/ecsq;
  G4double b = p*ecsq + mu + 2.*nu/ec;
  G4double ecut = 0.;
  G4double cut = a*a - 4.*p*b;
  if (cut > 0.) { ecut = std::sqrt(cut); }
  ecut = (ecut-a) / (2*p);
 
  //JMQ 290310 for avoiding unphysical increase below minimum (at ecut)
  // ecut<0 means that there is no cut with energy axis, i.e. xs is set 
  // to 0 bellow minimum

  G4double elab = K * FragmentA /G4double(ResidualA);
  G4double sig = 0.;
  if (elab <= ec) { 
    if (elab > ecut) { sig = (p*elab*elab+a*elab+b) * signor; }
    
    G4double signor2 = (ec-elab-c) / w;
    sig /= (1. + G4Exp(signor2));

  } else { 
    sig = (landa*elab+mu+nu/elab) * signor;
    G4double geom = 0.;    
    if (xnulam >= flow && elab >= etest) { 
      geom = std::sqrt(theA*K);
      geom = 1.23*ResidualAthrd + ra + 4.573/geom;
      geom = 31.416 * geom * geom;
      sig = std::max(geom, sig);
    }
  }  
  sig = std::max(sig, 0.0);
  return sig;
}
