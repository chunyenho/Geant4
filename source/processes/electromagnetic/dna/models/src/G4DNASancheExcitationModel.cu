//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4DNASancheExcitationModel.cc 87137 2014-11-25 09:12:48Z gcosmo $
//

// Created by Z. Francis

#include "G4DNASancheExcitationModel.hh"
#include "G4SystemOfUnits.hh"
#include "G4DNAMolecularMaterial.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4DNASancheExcitationModel::G4DNASancheExcitationModel(const G4ParticleDefinition*,
                                                       const G4String& nam) :
    G4VEmModel(nam), isInitialised(false)
{
  //  nistwater = G4NistManager::Instance()->FindOrBuildMaterial("G4_WATER");
  fpWaterDensity = 0;

  lowEnergyLimit = 2 * eV;
  highEnergyLimit = 100 * eV;
  SetLowEnergyLimit(lowEnergyLimit);
  SetHighEnergyLimit(highEnergyLimit);
  nLevels = 9;

  verboseLevel = 0;
  // Verbosity scale:
  // 0 = nothing
  // 1 = warning for energy non-conservation
  // 2 = details of energy budget
  // 3 = calculation of cross sections, file openings, sampling of atoms
  // 4 = entering in methods

  if (verboseLevel > 0)
  {
    G4cout << "Sanche Excitation model is constructed " << G4endl<< "Energy range: "
    << lowEnergyLimit / eV << " eV - "
    << highEnergyLimit / eV << " eV"
    << G4endl;
  }
  fParticleChangeForGamma = 0;
  fpWaterDensity = 0;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4DNASancheExcitationModel::~G4DNASancheExcitationModel()
{
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4DNASancheExcitationModel::Initialise(const G4ParticleDefinition* /*particle*/,
                                            const G4DataVector& /*cuts*/)
{

  if (verboseLevel > 3)
  G4cout << "Calling G4DNASancheExcitationModel::Initialise()" << G4endl;

  // Energy limits

  if (LowEnergyLimit() < lowEnergyLimit)
  {
    G4cout << "G4DNASancheExcitationModel: low energy limit increased from " <<
    LowEnergyLimit()/eV << " eV to " << lowEnergyLimit/eV << " eV" << G4endl;
    SetLowEnergyLimit(lowEnergyLimit);
  }

  if (HighEnergyLimit() > highEnergyLimit)
  {
    G4cout << "G4DNASancheExcitationModel: high energy limit decreased from " <<
    HighEnergyLimit()/eV << " eV to " << highEnergyLimit/eV << " eV" << G4endl;
    SetHighEnergyLimit(highEnergyLimit);
  }

  //

  if (verboseLevel > 0)
  {
    G4cout << "Sanche Excitation model is initialized " << G4endl
    << "Energy range: "
    << LowEnergyLimit() / eV << " eV - "
    << HighEnergyLimit() / eV << " eV"
    << G4endl;
  }

  // Initialize water density pointer
  fpWaterDensity = G4DNAMolecularMaterial::Instance()->
      GetNumMolPerVolTableFor(G4Material::GetMaterial("G4_WATER"));

  if (isInitialised) {return;} // RETURNS HERE

  fParticleChangeForGamma = GetParticleChangeForGamma();
  isInitialised = true;

  char *path = getenv("G4LEDATA");
  std::ostringstream eFullFileName;
  eFullFileName << path << "/dna/sigma_excitationvib_e_sanche.dat";
  std::ifstream input(eFullFileName.str().c_str());

  if (!input)
  {
    G4Exception("G4DNASancheExcitationModel::Initialise","em0003",
        FatalException,"Missing data file:/dna/sigma_excitationvib_e_sanche.dat");
  }

  // March 25th, 2014 - Vaclav Stepan, Sebastien Incerti
  // Added clear for MT
  tdummyVec.clear();
  //

  while(!input.eof())
  {
    double t;
    input>>t;
    tdummyVec.push_back(t);
    input>>map1[t][0]>>map1[t][1]>>map1[t][2]>>map1[t][3]>>map1[t][4]>>map1[t][5]>>map1[t][6]>>map1[t][7]>>map1[t][8];
    //G4cout<<t<<"  "<<map1[t][0]<<map1[t][1]<<map1[t][2]<<map1[t][3]<<map1[t][4]<<map1[t][5]<<map1[t][6]<<map1[t][7]<<map1[t][8]<<G4endl;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4DNASancheExcitationModel::CrossSectionPerVolume(const G4Material* material,
                                                           const G4ParticleDefinition* particleDefinition,
                                                           G4double ekin,
                                                           G4double,
                                                           G4double)
{
  if (verboseLevel > 3)
  G4cout << "Calling CrossSectionPerVolume() of G4DNASancheExcitationModel"
  << G4endl;

  // Calculate total cross section for model

  G4double sigma=0;

  G4double waterDensity = (*fpWaterDensity)[material->GetIndex()];

  if(waterDensity!= 0.0)
  //  if (material == nistwater || material->GetBaseMaterial() == nistwater)
  {

    if (particleDefinition == G4Electron::ElectronDefinition())
    {
      if (ekin >= lowEnergyLimit && ekin < highEnergyLimit)
      {
        sigma = Sum(ekin);
      }
    }

    if (verboseLevel > 2)
    {
      G4cout << "__________________________________" << G4endl;
      G4cout << "°°° G4DNASancheExcitationModel - XS INFO START" << G4endl;
      G4cout << "°°° Kinetic energy(eV)=" << ekin/eV << " particle : " << particleDefinition->GetParticleName() << G4endl;
      G4cout << "°°° Cross section per water molecule (cm^2)=" << sigma/cm/cm << G4endl;
      G4cout << "°°° Cross section per water molecule (cm^-1)=" << sigma*waterDensity/(1./cm) << G4endl;
      //      G4cout << " - Cross section per water molecule (cm^-1)=" << sigma*material->GetAtomicNumDensityVector()[1]/(1./cm) << G4endl;
      G4cout << "°°° G4DNASancheExcitationModel - XS INFO END" << G4endl;
    }

  } // if water

  //  return sigma*2*material->GetAtomicNumDensityVector()[1];
  return sigma*2*waterDensity;
  // see papers for factor 2 description

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4DNASancheExcitationModel::SampleSecondaries(std::vector<
                                                       G4DynamicParticle*>*,
                                                   const G4MaterialCutsCouple*,
                                                   const G4DynamicParticle* aDynamicElectron,
                                                   G4double,
                                                   G4double)
{

  if (verboseLevel > 3)
  G4cout << "Calling SampleSecondaries() of G4DNASancheExcitationModel"
  << G4endl;

  G4double electronEnergy0 = aDynamicElectron->GetKineticEnergy();
  G4int level = RandomSelect(electronEnergy0);
  G4double excitationEnergy = VibrationEnergy(level); // levels go from 0 to 8
  G4double newEnergy = electronEnergy0 - excitationEnergy;

  /*
   if (electronEnergy0 < highEnergyLimit)
   {
     if (newEnergy >= lowEnergyLimit)
     {
       fParticleChangeForGamma->ProposeMomentumDirection(aDynamicElectron->GetMomentumDirection());
       fParticleChangeForGamma->SetProposedKineticEnergy(newEnergy);
       fParticleChangeForGamma->ProposeLocalEnergyDeposit(excitationEnergy);
     }

     else
     {
       fParticleChangeForGamma->ProposeTrackStatus(fStopAndKill);
       fParticleChangeForGamma->ProposeLocalEnergyDeposit(electronEnergy0);
     }
   }
   */

  if (electronEnergy0 < highEnergyLimit && newEnergy>0.)
  {
    fParticleChangeForGamma->ProposeMomentumDirection(aDynamicElectron->GetMomentumDirection());
    fParticleChangeForGamma->SetProposedKineticEnergy(newEnergy);
    fParticleChangeForGamma->ProposeLocalEnergyDeposit(excitationEnergy);
  }

  //
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNASancheExcitationModel::PartialCrossSection(G4double t,
                                                         G4int level)
{
  std::vector<double>::iterator t2 = std::upper_bound(tdummyVec.begin(),
                                                      tdummyVec.end(), t / eV);
  std::vector<double>::iterator t1 = t2 - 1;

  double sigma = LinInterpolate((*t1), (*t2), t / eV, map1[*t1][level],
                                map1[*t2][level]);
  sigma *= 1e-16 * cm * cm;
  if (sigma == 0.) sigma = 1e-30;
  return (sigma);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNASancheExcitationModel::VibrationEnergy(G4int level)
{
  G4double energies[9] = { 0.01, 0.024, 0.061, 0.092, 0.204, 0.417, 0.460,
                           0.500, 0.835 };
  return (energies[level] * eV);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4int G4DNASancheExcitationModel::RandomSelect(G4double k)
{

  // Level Selection Counting can be done here !

  G4int i = nLevels;
  G4double value = 0.;
  std::deque<double> values;

  while (i > 0)
  {
    i--;
    G4double partial = PartialCrossSection(k, i);
    values.push_front(partial);
    value += partial;
  }

  value *= G4UniformRand();

  i = nLevels;

  while (i > 0)
  {
    i--;
    if (values[i] > value)
    {
      //outcount<<i<<"  "<<VibrationEnergy(i)<<G4endl;
      return i;
    }
    value -= values[i];
  }

  //outcount<<0<<"  "<<VibrationEnergy(0)<<G4endl;

  return 0;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNASancheExcitationModel::Sum(G4double k)
{
  G4double totalCrossSection = 0.;

  for (G4int i = 0; i < nLevels; i++)
  {
    totalCrossSection += PartialCrossSection(k, i);
  }
  return totalCrossSection;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNASancheExcitationModel::LinInterpolate(G4double e1,
                                                    G4double e2,
                                                    G4double e,
                                                    G4double xs1,
                                                    G4double xs2)
{
  G4double a = (xs2 - xs1) / (e2 - e1);
  G4double b = xs2 - a * e2;
  G4double value = a * e + b;
  // G4cout<<"interP >>  "<<e1<<"  "<<e2<<"  "<<e<<"  "<<xs1<<"  "<<xs2<<"  "<<a<<"  "<<b<<"  "<<value<<G4endl;

  return value;
}

