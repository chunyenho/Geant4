#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4DNAScreenedRutherfordElasticModel.cc 90769 2015-06-09 10:33:41Z gcosmo $
//

#include "G4DNAScreenedRutherfordElasticModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4DNAMolecularMaterial.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4DNAScreenedRutherfordElasticModel::G4DNAScreenedRutherfordElasticModel(const G4ParticleDefinition*,
                                                                         const G4String& nam) :
    G4VEmModel(nam), isInitialised(false)
{
  //  nistwater = G4NistManager::Instance()->FindOrBuildMaterial("G4_WATER");
  fpWaterDensity = 0;

  killBelowEnergy = 9 * eV;
  lowEnergyLimit = 0 * eV;
  intermediateEnergyLimit = 200 * eV; // Switch between two final state models
  highEnergyLimit = 1. * MeV;
  SetLowEnergyLimit(lowEnergyLimit);
  SetHighEnergyLimit(highEnergyLimit);

  verboseLevel = 0;
  // Verbosity scale:
  // 0 = nothing
  // 1 = warning for energy non-conservation
  // 2 = details of energy budget
  // 3 = calculation of cross sections, file openings, sampling of atoms
  // 4 = entering in methods

  if (verboseLevel > 0)
  {
    G4cout << "Screened Rutherford Elastic model is constructed " << G4endl<< "Energy range: "
    << lowEnergyLimit / eV << " eV - "
    << highEnergyLimit / MeV << " MeV"
    << G4endl;
  }
  fParticleChangeForGamma = 0;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4DNAScreenedRutherfordElasticModel::~G4DNAScreenedRutherfordElasticModel()
{
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4DNAScreenedRutherfordElasticModel::Initialise(const G4ParticleDefinition* /*particle*/,
                                                     const G4DataVector& /*cuts*/)
{

  if (verboseLevel > 3)
  G4cout << "Calling G4DNAScreenedRutherfordElasticModel::Initialise()"
  << G4endl;

  // Energy limits

  if (LowEnergyLimit() < lowEnergyLimit)
  {
    G4cout << "G4DNAScreenedRutherfordElasticModel: low energy limit increased from " <<
    LowEnergyLimit()/eV << " eV to " << lowEnergyLimit/eV << " eV" << G4endl;
    SetLowEnergyLimit(lowEnergyLimit);
  }

  if (HighEnergyLimit() > highEnergyLimit)
  {
    G4cout << "G4DNAScreenedRutherfordElasticModel: high energy limit decreased from " <<
    HighEnergyLimit()/MeV << " MeV to " << highEnergyLimit/MeV << " MeV" << G4endl;
    SetHighEnergyLimit(highEnergyLimit);
  }

  // Constants for final state by Brenner & Zaider
  // March 25th, 2014 - Vaclav Stepan, Sebastien Incerti
  // Added clear for MT

  betaCoeff.clear();
  betaCoeff.push_back(7.51525);
  betaCoeff.push_back(-0.41912);
  betaCoeff.push_back(7.2017E-3);
  betaCoeff.push_back(-4.646E-5);
  betaCoeff.push_back(1.02897E-7);

  deltaCoeff.clear();
  deltaCoeff.push_back(2.9612);
  deltaCoeff.push_back(-0.26376);
  deltaCoeff.push_back(4.307E-3);
  deltaCoeff.push_back(-2.6895E-5);
  deltaCoeff.push_back(5.83505E-8);

  gamma035_10Coeff.clear();
  gamma035_10Coeff.push_back(-1.7013);
  gamma035_10Coeff.push_back(-1.48284);
  gamma035_10Coeff.push_back(0.6331);
  gamma035_10Coeff.push_back(-0.10911);
  gamma035_10Coeff.push_back(8.358E-3);
  gamma035_10Coeff.push_back(-2.388E-4);

  gamma10_100Coeff.clear();
  gamma10_100Coeff.push_back(-3.32517);
  gamma10_100Coeff.push_back(0.10996);
  gamma10_100Coeff.push_back(-4.5255E-3);
  gamma10_100Coeff.push_back(5.8372E-5);
  gamma10_100Coeff.push_back(-2.4659E-7);

  gamma100_200Coeff.clear();
  gamma100_200Coeff.push_back(2.4775E-2);
  gamma100_200Coeff.push_back(-2.96264E-5);
  gamma100_200Coeff.push_back(-1.20655E-7);

  //

  if( verboseLevel>0 )
  {
    G4cout << "Screened Rutherford elastic model is initialized " << G4endl
    << "Energy range: "
    << LowEnergyLimit() / eV << " eV - "
    << HighEnergyLimit() / MeV << " MeV"
    << G4endl;
  }

  // Initialize water density pointer
  fpWaterDensity = G4DNAMolecularMaterial::Instance()->GetNumMolPerVolTableFor(G4Material::GetMaterial("G4_WATER"));

  if (isInitialised)
  { return;}
  fParticleChangeForGamma = GetParticleChangeForGamma();
  isInitialised = true;

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4DNAScreenedRutherfordElasticModel::CrossSectionPerVolume(const G4Material* material,
                                                                    const G4ParticleDefinition* particleDefinition,
                                                                    G4double ekin,
                                                                    G4double,
                                                                    G4double)
{
  if (verboseLevel > 3)
  G4cout << "Calling CrossSectionPerVolume() of G4DNAScreenedRutherfordElasticModel"
  << G4endl;

  // Calculate total cross section for model

  G4double sigma=0;

  G4double waterDensity = (*fpWaterDensity)[material->GetIndex()];

  if(waterDensity!= 0.0)
  //  if (material == nistwater || material->GetBaseMaterial() == nistwater)
  {

    if (ekin < highEnergyLimit)
    {

      if (ekin < killBelowEnergy) return DBL_MAX;

      G4double z = 10.;
      G4double n = ScreeningFactor(ekin,z);
      G4double crossSection = RutherfordCrossSection(ekin, z);
      sigma = pi * crossSection / (n * (n + 1.));
    }

    if (verboseLevel > 2)
    {
      G4cout << "__________________________________" << G4endl;
      G4cout << "=== G4DNAScreenedRutherfordElasticModel - XS INFO START" << G4endl;
      G4cout << "=== Kinetic energy(eV)=" << ekin/eV << " particle : " << particleDefinition->GetParticleName() << G4endl;
      G4cout << "=== Cross section per water molecule (cm^2)=" << sigma/cm/cm << G4endl;
      G4cout << "=== Cross section per water molecule (cm^-1)=" << sigma*waterDensity/(1./cm) << G4endl;
      // G4cout << " - Cross section per water molecule (cm^-1)=" << sigma*material->GetAtomicNumDensityVector()[1]/(1./cm) << G4endl;
      G4cout << "=== G4DNAScreenedRutherfordElasticModel - XS INFO END" << G4endl;
    }

  }

  return sigma*waterDensity;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNAScreenedRutherfordElasticModel::RutherfordCrossSection(G4double k,
                                                                     G4double z)
{
  //
  //                               e^4         /      K + m_e c^2      \^2
  // sigma_Ruth(K) = Z (Z+1) -------------------- | --------------------- |
  //                          (4 pi epsilon_0)^2  \  K * (K + 2 m_e c^2)  /
  //
  // Where K is the electron non-relativistic kinetic energy
  //
  // NIM 155, pp. 145-156, 1978

  G4double length = (e_squared * (k + electron_mass_c2))
      / (4 * pi * epsilon0 * k * (k + 2 * electron_mass_c2));
  G4double cross = z * (z + 1) * length * length;

  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNAScreenedRutherfordElasticModel::ScreeningFactor(G4double k,
                                                              G4double z)
{
  //
  //         alpha_1 + beta_1 ln(K/eV)   constK Z^(2/3)
  // n(T) = -------------------------- -----------------
  //              K/(m_e c^2)            2 + K/(m_e c^2)
  //
  // Where K is the electron non-relativistic kinetic energy
  //
  // n(T) > 0 for T < ~ 400 MeV
  //
  // NIM 155, pp. 145-156, 1978
  // Formulae (2) and (5)

  const G4double alpha_1(1.64);
  const G4double beta_1(-0.0825);
  const G4double constK(1.7E-5);

  G4double numerator = (alpha_1 + beta_1 * std::log(k / eV)) * constK
                       * std::pow(z, 2. / 3.);

  k /= electron_mass_c2;

  G4double denominator = k * (2 + k);

  G4double value = 0.;
  if (denominator > 0.) value = numerator / denominator;

  return value;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4DNAScreenedRutherfordElasticModel::SampleSecondaries(std::vector<
                                                                G4DynamicParticle*>* /*fvect*/,
                                                            const G4MaterialCutsCouple* /*couple*/,
                                                            const G4DynamicParticle* aDynamicElectron,
                                                            G4double,
                                                            G4double)
{

  if (verboseLevel > 3)
  {
    G4cout << "Calling SampleSecondaries() of G4DNAScreenedRutherfordElasticModel"
           << G4endl;
  }

  G4double electronEnergy0 = aDynamicElectron->GetKineticEnergy();

  if (electronEnergy0 < killBelowEnergy)
  {
    fParticleChangeForGamma->SetProposedKineticEnergy(0.);
    fParticleChangeForGamma->ProposeTrackStatus(fStopAndKill);
    fParticleChangeForGamma->ProposeLocalEnergyDeposit(electronEnergy0);
    return;
  }

  G4double cosTheta = 0.;

  if (electronEnergy0>= killBelowEnergy && electronEnergy0 < highEnergyLimit)
  {
    if (electronEnergy0<intermediateEnergyLimit)
    {
      if (verboseLevel > 3) G4cout << "---> Using Brenner & Zaider model" << G4endl;
      cosTheta = BrennerZaiderRandomizeCosTheta(electronEnergy0);
    }

    if (electronEnergy0>=intermediateEnergyLimit)
    {
      if (verboseLevel > 3) G4cout << "---> Using Screened Rutherford model" << G4endl;
      G4double z = 10.;
      cosTheta = ScreenedRutherfordRandomizeCosTheta(electronEnergy0,z);
    }

    G4double phi = 2. * pi * G4UniformRand();

    G4ThreeVector zVers = aDynamicElectron->GetMomentumDirection();
    G4ThreeVector xVers = zVers.orthogonal();
    G4ThreeVector yVers = zVers.cross(xVers);

    G4double xDir = std::sqrt(1. - cosTheta*cosTheta);
    G4double yDir = xDir;
    xDir *= std::cos(phi);
    yDir *= std::sin(phi);

    G4ThreeVector zPrimeVers((xDir*xVers + yDir*yVers + cosTheta*zVers));

    fParticleChangeForGamma->ProposeMomentumDirection(zPrimeVers.unit());

    fParticleChangeForGamma->SetProposedKineticEnergy(electronEnergy0);
  }

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4DNAScreenedRutherfordElasticModel::BrennerZaiderRandomizeCosTheta(G4double k)
{
  //  d sigma_el                         1                                 beta(K)
  // ------------ (K) ~ --------------------------------- + ---------------------------------
  //   d Omega           (1 + 2 gamma(K) - cos(theta))^2     (1 + 2 delta(K) + cos(theta))^2
  //
  // Maximum is < 1/(4 gamma(K)^2) + beta(K)/((2+2delta(K))^2)
  //
  // Phys. Med. Biol. 29 N.4 (1983) 443-447

  // gamma(K), beta(K) and delta(K) are polynomials with coefficients for energy measured in eV

  k /= eV;

  G4double beta = std::exp(CalculatePolynomial(k, betaCoeff));
  G4double delta = std::exp(CalculatePolynomial(k, deltaCoeff));
  G4double gamma;

  if (k > 100.)
  {
    gamma = CalculatePolynomial(k, gamma100_200Coeff);
    // Only in this case it is not the exponent of the polynomial
  }
  else
  {
    if (k > 10)
    {
      gamma = std::exp(CalculatePolynomial(k, gamma10_100Coeff));
    }
    else
    {
      gamma = std::exp(CalculatePolynomial(k, gamma035_10Coeff));
    }
  }

  // ***** Original method

  G4double oneOverMax = 1.
      / (1. / (4. * gamma * gamma) + beta
          / ((2. + 2. * delta) * (2. + 2. * delta)));

  G4double cosTheta = 0.;
  G4double leftDenominator = 0.;
  G4double rightDenominator = 0.;
  G4double fCosTheta = 0.;

  do
  {
    cosTheta = 2. * G4UniformRand()- 1.;

    leftDenominator = (1. + 2.*gamma - cosTheta);
    rightDenominator = (1. + 2.*delta + cosTheta);
    if ( (leftDenominator * rightDenominator) != 0. )
    {
      fCosTheta = oneOverMax * (1./(leftDenominator*leftDenominator) + beta/(rightDenominator*rightDenominator));
    }
  }
  while (fCosTheta < G4UniformRand());

  return cosTheta;

  // ***** Alternative method using cumulative probability
  /*
  G4double cosTheta = -1;
  G4double cumul = 0;
  G4double value = 0;
  G4double leftDenominator = 0.;
  G4double rightDenominator = 0.;

  // Number of integration steps in the -1,1 range
  G4int iMax=200;

  G4double random = G4UniformRand();

  // Cumulate differential cross section
  for (G4int i=0; i<iMax; i++)
  {
  cosTheta = -1 + i*2./(iMax-1);
  leftDenominator = (1. + 2.*gamma - cosTheta);
  rightDenominator = (1. + 2.*delta + cosTheta);
    if ( (leftDenominator * rightDenominator) != 0. )
    {
      cumul = cumul + (1./(leftDenominator*leftDenominator) + beta/(rightDenominator*rightDenominator));
    }
  }

  // Select cosTheta
  for (G4int i=0; i<iMax; i++)
  {
    cosTheta = -1 + i*2./(iMax-1);
    leftDenominator = (1. + 2.*gamma - cosTheta);
    rightDenominator = (1. + 2.*delta + cosTheta);
    if (cumul !=0 && (leftDenominator * rightDenominator) != 0.)
      value = value + (1./(leftDenominator*leftDenominator) + beta/(rightDenominator*rightDenominator)) / cumul;
    if (random < value) break;
  }

  return cosTheta;
   */

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNAScreenedRutherfordElasticModel::CalculatePolynomial(G4double k,
                                                                  std::vector<
                                                                      G4double>& vec)
{
  // Sum_{i=0}^{size-1} vector_i k^i
  //
  // Phys. Med. Biol. 29 N.4 (1983) 443-447

  G4double result = 0.;
  size_t size = vec.size();

  while (size > 0)
  {
    size--;

    result *= k;
    result += vec[size];
  }

  return result;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4DNAScreenedRutherfordElasticModel::ScreenedRutherfordRandomizeCosTheta(G4double k,
                                                                                  G4double z)
{

  //  d sigma_el                sigma_Ruth(K)
  // ------------ (K) ~ -----------------------------
  //   d Omega           (1 + 2 n(K) - cos(theta))^2
  //
  // We extract cos(theta) distributed as (1 + 2 n(K) - cos(theta))^-2
  //
  // Maximum is for theta=0: 1/(4 n(K)^2) (When n(K) is positive, that is always satisfied within the validity of the process)
  //
  // Phys. Med. Biol. 45 (2000) 3171-3194

  // ***** Original method

  G4double n = ScreeningFactor(k, z);

  G4double oneOverMax = (4. * n * n);

  G4double cosTheta = 0.;
  G4double fCosTheta;

  do
  {
    cosTheta = 2. * G4UniformRand()- 1.;
    fCosTheta = (1 + 2.*n - cosTheta);
    if (fCosTheta !=0.) fCosTheta = oneOverMax / (fCosTheta*fCosTheta);
  }
  while (fCosTheta < G4UniformRand());

  return cosTheta;

  // ***** Alternative method using cumulative probability
  /*
   G4double cosTheta = -1;
   G4double cumul = 0;
   G4double value = 0;
   G4double n = ScreeningFactor(k, z);
   G4double fCosTheta;

   // Number of integration steps in the -1,1 range
   G4int iMax=200;

   G4double random = G4UniformRand();

   // Cumulate differential cross section
   for (G4int i=0; i<iMax; i++)
   {
     cosTheta = -1 + i*2./(iMax-1);
     fCosTheta = (1 + 2.*n - cosTheta);
     if (fCosTheta !=0.) cumul = cumul + 1./(fCosTheta*fCosTheta);
   }

   // Select cosTheta
   for (G4int i=0; i<iMax; i++)
   {
     cosTheta = -1 + i*2./(iMax-1);
     fCosTheta = (1 + 2.*n - cosTheta);
     if (cumul !=0.) value = value + (1./(fCosTheta*fCosTheta)) / cumul;
     if (random < value) break;
   }
   return cosTheta;
   */
}

