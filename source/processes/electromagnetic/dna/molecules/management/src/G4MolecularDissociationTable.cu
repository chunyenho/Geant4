#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4MolecularDissociationTable.cc 84858 2014-10-21 16:08:22Z gcosmo $
//
// WARNING : This class is released as a prototype.
// It might strongly evolve or even disapear in the next releases.
//
// ----------------------------------------------------------------------
//      GEANT 4 class implementation file
//
//      History: first implementation by Alfonso Mantero 4 Mar 2009
//
// ----------------------------------------------------------------

#include "G4MolecularDissociationTable.hh"
#include "G4MolecularDissociationChannel.hh"

using namespace std;

G4MolecularDissociationTable::G4MolecularDissociationTable()
{
  ;
}

G4MolecularDissociationTable::~G4MolecularDissociationTable()
{
  channelsMap::iterator it_map = fDecayChannelsMap.begin();

  for (; it_map != fDecayChannelsMap.end(); it_map++)
  {
    vector<const G4MolecularDissociationChannel*>& decayChannels = it_map
        ->second;
    if (!decayChannels.empty())
    {
      for (int i = 0; i < (int) decayChannels.size(); i++)
      {
        if (decayChannels[i])
        {
          delete decayChannels[i];
          decayChannels[i] = 0;
        }
      }
      decayChannels.clear();
    }
  }
  fDecayChannelsMap.clear();
}

G4MolecularDissociationTable::G4MolecularDissociationTable(const G4MolecularDissociationTable& right)
{
  *this = right;
}

G4MolecularDissociationTable& G4MolecularDissociationTable::operator=(const G4MolecularDissociationTable& aMolecularDecayTable)
{
  fExcitedStatesMap = aMolecularDecayTable.fExcitedStatesMap;
  fDecayChannelsMap = channelsMap(aMolecularDecayTable.GetDecayChannelsMap());
  return *this;
}

const vector<const G4MolecularDissociationChannel*>* G4MolecularDissociationTable::GetDecayChannels(const G4ElectronOccupancy* conf) const
{
  statesMap::const_iterator it_exstates = fExcitedStatesMap.find(*conf);
  if (it_exstates == fExcitedStatesMap.end()) return 0;
  channelsMap::const_iterator it_decchannel = fDecayChannelsMap.find(
      it_exstates->second);
  if (it_decchannel == fDecayChannelsMap.end()) return 0;
  return &(it_decchannel->second);
}

const vector<const G4MolecularDissociationChannel*>* G4MolecularDissociationTable::GetDecayChannels(const G4String& exState) const
{
  channelsMap::const_iterator it_decchannel = fDecayChannelsMap.find(exState);
  if (it_decchannel == fDecayChannelsMap.end()) return 0;
  return &(it_decchannel->second);
}

const G4String& G4MolecularDissociationTable::GetExcitedState(const G4ElectronOccupancy* conf) const
{
  statesMap::const_iterator it_exstates = fExcitedStatesMap.find(*conf);

  if (it_exstates == fExcitedStatesMap.end())
  {
    G4String errMsg = "Excited state not found";
    G4Exception(
        "G4MolecularDecayTable::GetExcitedState(const G4ElectronOccupancy*)",
        "G4MolecularDecayTable001", FatalErrorInArgument, errMsg);
//        return *(new G4String("IM FAKE"));  // fake return statement
  }

  return it_exstates->second;
}

const G4ElectronOccupancy& G4MolecularDissociationTable::GetElectronOccupancy(const G4String& exState) const
{
  statesMap::const_iterator statesIter;
  const G4ElectronOccupancy* conf(0);
  for (statesIter = fExcitedStatesMap.begin();
      statesIter != fExcitedStatesMap.end(); statesIter++)
  {
    if (exState == statesIter->second) conf = &(statesIter->first);
  }

  if (statesIter == fExcitedStatesMap.end())
  {
    G4String errMsg = "Excited state" + exState + " not found";
    G4Exception("G4MolecularDecayTable::GetElectronOccupancy(const G4String&)",
                "G4MolecularDecayTable002", FatalErrorInArgument, errMsg);
  }
  return *conf;
}

void G4MolecularDissociationTable::AddExcitedState(const G4String& label)
{
  channelsMap::iterator channelsIter = fDecayChannelsMap.find(label);
  if (channelsIter != fDecayChannelsMap.end())
  {
    G4String errMsg = "Excited state" + label
                      + " already registered in the decay table.";
    G4Exception("G4MolecularDecayTable::AddExcitedState",
                "G4MolecularDecayTable003", FatalErrorInArgument, errMsg);
    return;
  }
  fDecayChannelsMap[label];
}

void G4MolecularDissociationTable::AddeConfToExcitedState(const G4String& label,
                                                          const G4ElectronOccupancy& conf)
{
  statesMap::iterator statesIter = fExcitedStatesMap.find(conf);

  if (statesIter == fExcitedStatesMap.end())
  {
    fExcitedStatesMap[conf] = label;
  }
  else
  {
    G4Exception(
        "G4MolecularDecayTable::AddExcitedState", "G4MolecularDecayTable004",
        FatalErrorInArgument,
        "Electronic configuration already registered in the decay table");
  }
}

void G4MolecularDissociationTable::AddDecayChannel(const G4String& label,
                                                   const G4MolecularDissociationChannel* channel)
{
  fDecayChannelsMap[label].push_back(channel);
}

void G4MolecularDissociationTable::CheckDataConsistency()
{
  channelsMap::iterator channelsIter;

  //Let's check probabilities

  for (channelsIter = fDecayChannelsMap.begin();
      channelsIter != fDecayChannelsMap.end(); channelsIter++)
  {

    vector<const G4MolecularDissociationChannel*>& decayVect = channelsIter
        ->second;
    G4double sum = 0;

    G4double max = decayVect.size();

    for (size_t i = 0; i < max; i++)
    {
      const G4MolecularDissociationChannel* decay = decayVect[i];
      const G4double prob = decay->GetProbability();
      sum += prob;
    }

    if (sum != 1)
    {
      G4String errMsg = "Deexcitation Channels probabilities in "
          + channelsIter->first + "excited state don't sum up to 1";
      G4Exception("G4MolecularDecayTable::CheckDataConsistency",
                  "G4MolecularDecayTable005", FatalErrorInArgument, errMsg);
    }
  }

}

