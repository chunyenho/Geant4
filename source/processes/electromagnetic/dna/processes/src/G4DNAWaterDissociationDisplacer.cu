#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4DNAWaterDissociationDisplacer.cc 84858 2014-10-21 16:08:22Z gcosmo $
//
// Author: Mathieu Karamitros (kara (AT) cenbg . in2p3 . fr) 
//
// WARNING : This class is released as a prototype.
// It might strongly evolve or even disapear in the next releases.
//
// History:
// -----------
// 10 Oct 2011 M.Karamitros created
//
// -------------------------------------------------------------------

#include "G4DNAWaterDissociationDisplacer.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4H2O.hh"
#include "G4H2.hh"
#include "G4Hydrogen.hh"
#include "G4OH.hh"
#include "G4H3O.hh"
#include "G4Electron_aq.hh"
#include "G4H2O2.hh"
#include "Randomize.hh"
#include "G4Molecule.hh"

using namespace std;

const DisplacementType G4DNAWaterDissociationDisplacer::Ionisation_DissociationDecay =
    G4VMolecularDecayDisplacer::AddDisplacement();
const DisplacementType G4DNAWaterDissociationDisplacer::A1B1_DissociationDecay =
    G4VMolecularDecayDisplacer::AddDisplacement();
const DisplacementType G4DNAWaterDissociationDisplacer::B1A1_DissociationDecay =
    G4VMolecularDecayDisplacer::AddDisplacement();
const DisplacementType G4DNAWaterDissociationDisplacer::AutoIonisation =
    G4VMolecularDecayDisplacer::AddDisplacement();
const DisplacementType G4DNAWaterDissociationDisplacer::DissociativeAttachment =
    G4VMolecularDecayDisplacer::AddDisplacement();

G4DNAWaterDissociationDisplacer::G4DNAWaterDissociationDisplacer() :
    G4VMolecularDecayDisplacer()
{
  ;
}

G4DNAWaterDissociationDisplacer::~G4DNAWaterDissociationDisplacer()
{
  ;
}

G4ThreeVector G4DNAWaterDissociationDisplacer::GetMotherMoleculeDisplacement(const G4MolecularDissociationChannel* theDecayChannel) const
{
  G4int decayType = theDecayChannel->GetDisplacementType();

  G4double RMSMotherMoleculeDisplacement = 0;

  if (decayType == Ionisation_DissociationDecay)
  {
    RMSMotherMoleculeDisplacement = 2.0 * nanometer;
  }
  else if (decayType == A1B1_DissociationDecay)
  {
    RMSMotherMoleculeDisplacement = 0. * nanometer;
  }
  else if (decayType == B1A1_DissociationDecay)
  {
    RMSMotherMoleculeDisplacement = 0. * nanometer;
  }
  else if (decayType == AutoIonisation)
  {
    RMSMotherMoleculeDisplacement = 2.0 * nanometer;
  }
  else if (decayType == DissociativeAttachment)
  {
    RMSMotherMoleculeDisplacement = 0. * nanometer;
  }

  if (RMSMotherMoleculeDisplacement == 0)
  {
    return G4ThreeVector(0, 0, 0);
  }
  G4ThreeVector RandDirection = radialDistributionOfProducts(
      RMSMotherMoleculeDisplacement);

  return RandDirection;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

vector<G4ThreeVector> G4DNAWaterDissociationDisplacer::GetProductsDisplacement(const G4MolecularDissociationChannel* theDecayChannel) const
{
  G4int nbProducts = theDecayChannel->GetNbProducts();
  vector<G4ThreeVector> theProductDisplacementVector(nbProducts);

  typedef map<const G4MoleculeDefinition*, G4double> RMSmap;
  RMSmap theRMSmap;

  G4int decayType = theDecayChannel->GetDisplacementType();

  if (decayType == Ionisation_DissociationDecay)
  {
    if (fVerbose) G4cout << "Ionisation_DissociationDecay" << G4endl;
    G4double RdmValue = G4UniformRand();

    if(RdmValue< 0.5)
    {
      // H3O
      theRMSmap[G4H3O::Definition()] = 0.* nanometer;
      // OH
      theRMSmap[G4OH::Definition()] = 0.8* nanometer;
    }
    else
    {
      // H3O
      theRMSmap[G4H3O::Definition()] = 0.8* nanometer;
      // OH
      theRMSmap[G4OH::Definition()] = 0.* nanometer;
    }

    for(int i = 0; i < nbProducts; i++)
    {
      G4double theRMSDisplacement;
      const G4Molecule* product = theDecayChannel->GetProduct(i);
      theRMSDisplacement = theRMSmap[product->GetDefinition()];

      if(theRMSDisplacement==0)
      {
        theProductDisplacementVector[i] = G4ThreeVector();
      }
      else
      {
        G4ThreeVector RandDirection = radialDistributionOfProducts(theRMSDisplacement);
        theProductDisplacementVector[i] = RandDirection;
      }
    }
  }
  else if(decayType == A1B1_DissociationDecay)
  {
    if(fVerbose)
    G4cout<<"A1B1_DissociationDecay"<<G4endl;
    G4double theRMSDisplacement = 2.4 * nanometer;
    G4ThreeVector RandDirection = radialDistributionOfProducts(theRMSDisplacement);

    for(G4int i =0; i < nbProducts; i++)
    {
      const G4Molecule* product = theDecayChannel->GetProduct(i);
      if(product->GetDefinition()== G4OH::Definition())
      {
        theProductDisplacementVector[i] = -1./18.*RandDirection;
      }
      else if(product->GetDefinition() == G4Hydrogen::Definition())
      {
        theProductDisplacementVector[i] = +17./18.*RandDirection;
      }
    }
  }
  else if(decayType == B1A1_DissociationDecay)
  {
    if(fVerbose)
    G4cout<<"B1A1_DissociationDecay"<<G4endl;
    G4double theRMSDisplacement = 0.8 * nanometer;
    G4ThreeVector RandDirection = radialDistributionOfProducts(theRMSDisplacement);

    G4int NbOfOH = 0;
    for(G4int i =0; i < nbProducts; i++)
    {
      const G4Molecule* product = theDecayChannel->GetProduct(i);
      if(product->GetDefinition() == G4H2::Definition())
      {
        theProductDisplacementVector[i] = -2./18.*RandDirection;
      }
      else if(product->GetDefinition() == G4OH::Definition())
      {
        G4ThreeVector OxygenDisplacement = +16./18.*RandDirection;
        G4double OHRMSDisplacement = 1.1 * nanometer;

        G4ThreeVector OHDisplacement = radialDistributionOfProducts(OHRMSDisplacement);

        if(NbOfOH==0)
        {
          OHDisplacement = 1./2.*OHDisplacement;
        }
        else
        {
          OHDisplacement = -1./2.*OHDisplacement;
        }

        theProductDisplacementVector[i] = OHDisplacement + OxygenDisplacement;

        NbOfOH ++;
      }
    }
  }
  else if(decayType == AutoIonisation)
  {
    if(fVerbose)
    G4cout<<"AutoIonisation"<<G4endl;
    G4double RdmValue = G4UniformRand();

    if(RdmValue< 0.5)
    {
      // H3O
      theRMSmap[G4H3O::Definition()] = 0.* nanometer;
      // OH
      theRMSmap[G4OH::Definition()] = 0.8* nanometer;
    }
    else
    {
      // H3O
      theRMSmap[G4H3O::Definition()] = 0.8* nanometer;
      // OH
      theRMSmap[G4OH::Definition()] = 0.* nanometer;
    }

    for(G4int i =0; i < nbProducts; i++)
    {
      G4double theRMSDisplacement;
      const G4Molecule* product = theDecayChannel->GetProduct(i);
      theRMSDisplacement = theRMSmap[product->GetDefinition()];

      if(theRMSDisplacement==0)
      {
        theProductDisplacementVector[i] = G4ThreeVector();
      }
      else
      {
        G4ThreeVector RandDirection = radialDistributionOfProducts(theRMSDisplacement);
        theProductDisplacementVector[i] = RandDirection;
      }
      if(product->GetDefinition() == G4Electron_aq::Definition())
      {
        theProductDisplacementVector[i]=radialDistributionOfElectron();
      }
    }
  }
  else if(decayType == DissociativeAttachment)
  {
    if(fVerbose)
    G4cout<<"DissociativeAttachment"<<G4endl;
    G4double theRMSDisplacement = 0.8 * nanometer;
    G4ThreeVector RandDirection = radialDistributionOfProducts(theRMSDisplacement);

    G4int NbOfOH = 0;
    for(G4int i =0; i < nbProducts; i++)
    {
      const G4Molecule* product = theDecayChannel->GetProduct(i);
      if(product->GetDefinition() == G4H2::Definition())
      {
        theProductDisplacementVector[i] = -2./18.*RandDirection;
      }
      else if(product->GetDefinition() == G4OH::Definition())
      {
        G4ThreeVector OxygenDisplacement = +16./18.*RandDirection;
        G4double OHRMSDisplacement = 1.1 * nanometer;

        G4ThreeVector OHDisplacement = radialDistributionOfProducts(OHRMSDisplacement);

        if(NbOfOH==0)
        {
          OHDisplacement = 1./2.*OHDisplacement;
        }
        else
        {
          OHDisplacement = -1./2.*OHDisplacement;
        }

        theProductDisplacementVector[i] = OHDisplacement + OxygenDisplacement;

        NbOfOH ++;
      }
    }
  }

  return theProductDisplacementVector;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4DNAWaterDissociationDisplacer::radialDistributionOfProducts(G4double Rrms) const
{
  G4double sigma = Rrms / sqrt(3.);
  G4double expectationValue = 2. * sqrt(2. / 3.14) * sigma;

  G4double XValueForfMax = sqrt(2. * sigma * sigma);
  G4double fMaxValue = sqrt(2. / 3.14) * 1. / (sigma * sigma * sigma)
      * (XValueForfMax * XValueForfMax)
      * exp(-1. / 2. * (XValueForfMax * XValueForfMax) / (sigma * sigma));

  G4double R(-1.);

  do
  {
    G4double aRandomfValue = fMaxValue * G4UniformRand();

    G4double sign;
    if(G4UniformRand() > 0.5)
    {
      sign = +1.;
    }
    else
    {
      sign = -1;
    }

    R = expectationValue + sign*3.*sigma* G4UniformRand();
    G4double f = sqrt(2./3.14) * 1/pow(sigma, 3) * R*R * exp(-1./2. * R*R/(sigma*sigma));

    if(aRandomfValue < f)
    {
      break;
    }
  }
  while(1);

  G4double costheta = (2. * G4UniformRand()-1.);
  G4double theta = acos(costheta);
  G4double phi = 2. * pi * G4UniformRand();

  G4double xDirection = R * cos(phi) * sin(theta);
  G4double yDirection = R * sin(theta) * sin(phi);
  G4double zDirection = R * costheta;
  G4ThreeVector RandDirection(xDirection, yDirection, zDirection);

  return RandDirection;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4DNAWaterDissociationDisplacer::radialDistributionOfElectron() const
{

  G4double sigma = 1. / 2.;
  G4double expectationValue = 1.;

  G4double XValueForfMax = 1. / 2.;
  G4double fMaxValue = 4. * XValueForfMax * exp(-2. * XValueForfMax);

  G4double R(-1.);

  do
  {
    G4double aRandomfValue = fMaxValue * G4UniformRand();

    G4double sign;
    if(G4UniformRand() > 0.5)
    {
      sign = +1;
    }
    else
    {
      sign = -1;
    }

    R = (expectationValue * G4UniformRand() )+ sign*3*sigma* G4UniformRand();
    G4double f = 4* R * exp(- 2. * R);

    if(aRandomfValue < f)
    {
      break;
    }
  }
  while(1);

  G4double Rnano = R * 10 * nanometer;

  G4double costheta = (2 * G4UniformRand()-1);
  G4double theta = acos(costheta);
  G4double phi = 2 * pi * G4UniformRand();

  G4double xDirection = Rnano * cos(phi) * sin(theta);
  G4double yDirection = Rnano * sin(theta) * sin(phi);
  G4double zDirection = Rnano * costheta;
  G4ThreeVector RandDirection(xDirection, yDirection, zDirection);

  return RandDirection;
}
