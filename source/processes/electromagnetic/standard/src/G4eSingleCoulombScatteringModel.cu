#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//	G4eSingleCoulombScatteringModel.cc
// -------------------------------------------------------------------
//
// GEANT4 Class header file
//
// File name:    G4eSingleCoulombScatteringModel
//
// Author:      Cristina Consolandi
//
// Creation date: 20.10.2012  
//                           
//	Class Description:
//	Single Scattering model for electron-nuclei interaction.
//	Suitable for high energy electrons and low scattering angles.
//
//
// Reference:
//      M.J. Boschini et al. "Non Ionizing Energy Loss induced by Electrons 
//      in the Space Environment" Proc. of the 13th International Conference 
//      on Particle Physics and Advanced Technology 
//
//	(13th ICPPAT, Como 3-7/10/2011), World Scientific (Singapore).
//	Available at: http://arxiv.org/abs/1111.4042v4
//
//
// -------------------------------------------------------------------
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....


#include "G4eSingleCoulombScatteringModel.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"
#include "G4ParticleChangeForGamma.hh"
#include "G4Proton.hh"
#include "G4ProductionCutsTable.hh"
#include "G4NucleiProperties.hh"
#include "G4NistManager.hh"
#include "G4ParticleTable.hh"
#include "G4IonTable.hh"

#include "G4UnitsTable.hh"


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

G4eSingleCoulombScatteringModel::G4eSingleCoulombScatteringModel(const G4String& nam)
  : G4VEmModel(nam),
    cosThetaMin(1.0),
    isInitialised(false)
{
  fNistManager = G4NistManager::Instance();
  theIonTable = G4ParticleTable::GetParticleTable()->GetIonTable();
  fParticleChange = 0;

  pCuts=0;
  currentMaterial = 0;
  currentElement  = 0;
  currentCouple = 0;

  lowEnergyLimit  = 0*eV;
  recoilThreshold = 0.*eV;
  particle = 0;
  mass=0;
  currentMaterialIndex = -1;

  Mottcross = new G4ScreeningMottCrossSection(); 
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4eSingleCoulombScatteringModel::~G4eSingleCoulombScatteringModel()
{ 
  delete  Mottcross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eSingleCoulombScatteringModel::Initialise(const G4ParticleDefinition* p,
						 const G4DataVector&  cuts)
{
  SetupParticle(p);
  currentCouple = 0;
  currentMaterialIndex = -1;
  //cosThetaMin = cos(PolarAngleLimit());
  Mottcross->Initialise(p,cosThetaMin);
 
  pCuts = &cuts; 
  //G4ProductionCutsTable::GetProductionCutsTable()->GetEnergyCutsVector(3);


  if(!isInitialised) {
    isInitialised = true;
    fParticleChange = GetParticleChangeForGamma();
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4eSingleCoulombScatteringModel::ComputeCrossSectionPerAtom(
                                const G4ParticleDefinition* p,
				G4double kinEnergy, 
				G4double Z, 
				G4double , 
				G4double, 
				G4double )
{
  SetupParticle(p);
 
  G4double cross =0.0;
  if(kinEnergy < lowEnergyLimit) return cross;

  DefineMaterial(CurrentCouple());

  //Total Cross section
  Mottcross->SetupKinematic(kinEnergy, Z);
  cross = Mottcross->NuclearCrossSection();

  //cout<< "....cross "<<G4BestUnit(cross,"Surface") << " cm2 "<< cross/cm2 <<endl;
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eSingleCoulombScatteringModel::SampleSecondaries(
			       std::vector<G4DynamicParticle*>* fvect,
			       const G4MaterialCutsCouple* couple,
			       const G4DynamicParticle* dp,
			       G4double cutEnergy, 
			       G4double)
{
  G4double kinEnergy = dp->GetKineticEnergy();
  //cout<<"--- kinEnergy "<<kinEnergy<<endl;

  if(kinEnergy < lowEnergyLimit) return;
	
  DefineMaterial(couple);
  SetupParticle(dp->GetDefinition());

  // Choose nucleus
  //last two :cutEnergy= min e kinEnergy=max
  currentElement = SelectRandomAtom(couple,particle,
				    kinEnergy,cutEnergy,kinEnergy);

  G4double Z  = currentElement->GetZ();
  G4int iz    = G4int(Z);
  G4int ia = SelectIsotopeNumber(currentElement);

  //cout<<"Element "<<currentElement->GetName()<<endl;;	

  G4double cross= Mottcross->GetTotalCross();

  if(cross == 0.0) { return; }
    		
  G4ThreeVector dir = dp->GetMomentumDirection(); //old direction
  G4ThreeVector newDirection=Mottcross->GetNewDirection();//new direction
  newDirection.rotateUz(dir);   
  
  fParticleChange->ProposeMomentumDirection(newDirection);   
  
  //Recoil energy
  G4double trec= Mottcross->GetTrec();

  //Energy after scattering	
  if(trec > kinEnergy) { trec = kinEnergy; }
  G4double finalT = kinEnergy - trec;
  G4double edep = 0.0;

  G4double tcut = recoilThreshold;
  if(pCuts) { 
    tcut= std::min(tcut,(*pCuts)[currentMaterialIndex]); 
  }

  if(trec > tcut) {

    //cout<<"Trec "<<trec/eV<<endl;
    G4ParticleDefinition* ion = theIonTable->GetIon(iz, ia, 0);

    //incident before scattering
    G4double ptot=sqrt(Mottcross->GetMom2Lab());
    //incident after scattering
    G4double plab = sqrt(finalT*(finalT + 2.0*mass));
    G4ThreeVector p2 = (ptot*dir - plab*newDirection).unit();
    //secondary particle
    G4DynamicParticle* newdp  = new G4DynamicParticle(ion, p2, trec);
    fvect->push_back(newdp);
  } else if(trec > 0.0) {
    edep = trec;
    fParticleChange->ProposeNonIonizingEnergyDeposit(trec);
  }

  // finelize primary energy and energy balance
  if(finalT <= lowEnergyLimit) { 
    edep += finalT;  
    finalT = 0.0;
  } 
  fParticleChange->SetProposedKineticEnergy(finalT);
  fParticleChange->ProposeLocalEnergyDeposit(edep);

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
		
