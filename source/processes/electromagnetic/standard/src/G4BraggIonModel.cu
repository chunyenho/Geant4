#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4BraggIonModel.cc 83008 2014-07-24 14:49:52Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:   G4BraggIonModel
//
// Author:        Vladimir Ivanchenko
//
// Creation date: 13.10.2004
//
// Modifications:
// 11-05-05 Major optimisation of internal interfaces (V.Ivantchenko)
// 29-11-05 Do not use G4Alpha class (V.Ivantchenko)
// 15-02-06 ComputeCrossSectionPerElectron, ComputeCrossSectionPerAtom (mma)
// 25-04-06 Add stopping data from ASTAR (V.Ivanchenko)
// 23-10-06 Reduce lowestKinEnergy to 0.25 keV (V.Ivanchenko)
// 12-08-08 Added methods GetParticleCharge, GetChargeSquareRatio, 
//          CorrectionsAlongStep needed for ions(V.Ivanchenko)
//

// Class Description:
//
// Implementation of energy loss and delta-electron production by
// slow charged heavy particles

// -------------------------------------------------------------------
//

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "G4BraggIonModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"
#include "G4Electron.hh"
#include "G4ParticleChangeForLoss.hh"
#include "G4LossTableManager.hh"
#include "G4EmCorrections.hh"
#include "G4DeltaAngle.hh"
#include "G4Log.hh"
#include "G4Exp.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

using namespace std;

G4ASTARStopping* G4BraggIonModel::fASTAR = 0;

G4BraggIonModel::G4BraggIonModel(const G4ParticleDefinition* p,
                                 const G4String& nam)
  : G4VEmModel(nam),
    corr(0),
    particle(0),
    fParticleChange(0),
    currentMaterial(0),
    iMolecula(-1),
    iASTAR(-1),
    isIon(false),
    isInitialised(false)
{
  SetHighEnergyLimit(2.0*MeV);

  HeMass           = 3.727417*GeV;
  rateMassHe2p     = HeMass/proton_mass_c2;
  lowestKinEnergy  = 1.0*keV/rateMassHe2p;
  massFactor       = 1000.*amu_c2/HeMass;
  theZieglerFactor = eV*cm2*1.0e-15;
  theElectron      = G4Electron::Electron();
  corrFactor       = 1.0;
  if(p) { SetParticle(p); }
  else  { SetParticle(theElectron); }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4BraggIonModel::~G4BraggIonModel()
{
  if(IsMaster()) { delete fASTAR; fASTAR = 0; }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4BraggIonModel::Initialise(const G4ParticleDefinition* p,
                                 const G4DataVector&)
{
  if(p != particle) { SetParticle(p); }

  corrFactor = chargeSquare;

  // always false before the run
  SetDeexcitationFlag(false);

  if(!isInitialised) {
    isInitialised = true;

    if(UseAngularGeneratorFlag() && !GetAngularDistribution()) {
      SetAngularDistribution(new G4DeltaAngle());
    }
    G4String pname = particle->GetParticleName();
    if(particle->GetParticleType() == "nucleus" &&
       pname != "deuteron" && pname != "triton" &&
       pname != "alpha+"   && pname != "helium" &&
       pname != "hydrogen") { isIon = true; }

    corr = G4LossTableManager::Instance()->EmCorrections();

    fParticleChange = GetParticleChangeForLoss();
    if(!fASTAR) { fASTAR = new G4ASTARStopping(); }
  }
  if(IsMaster() && particle->GetPDGMass() < GeV) { fASTAR->Initialise(); }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::MinEnergyCut(const G4ParticleDefinition*,
				       const G4MaterialCutsCouple* couple)
{
  return couple->GetMaterial()->GetIonisation()->GetMeanExcitationEnergy();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::GetChargeSquareRatio(const G4ParticleDefinition* p,
					       const G4Material* mat,
					       G4double kineticEnergy)
{
  //G4cout<<"G4BraggIonModel::GetChargeSquareRatio e= "<<kineticEnergy<<G4endl;
  // this method is called only for ions
  G4double q2 = corr->EffectiveChargeSquareRatio(p,mat,kineticEnergy);
  corrFactor  = q2*corr->EffectiveChargeCorrection(p,mat,kineticEnergy); 
  return corrFactor;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::GetParticleCharge(const G4ParticleDefinition* p,
					    const G4Material* mat,
					    G4double kineticEnergy)
{
  //G4cout<<"G4BraggIonModel::GetParticleCharge e= "<<kineticEnergy <<
  //  " q= " <<  corr->GetParticleCharge(p,mat,kineticEnergy) <<G4endl;
  // this method is called only for ions
  return corr->GetParticleCharge(p,mat,kineticEnergy);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::ComputeCrossSectionPerElectron(
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
                                                 G4double cutEnergy,
                                                 G4double maxKinEnergy)
{
  G4double cross     = 0.0;
  G4double tmax      = MaxSecondaryEnergy(p, kineticEnergy);
  G4double maxEnergy = std::min(tmax,maxKinEnergy);
  if(cutEnergy < tmax) {

    G4double energy  = kineticEnergy + mass;
    G4double energy2 = energy*energy;
    G4double beta2   = kineticEnergy*(kineticEnergy + 2.0*mass)/energy2;
    cross = (maxEnergy - cutEnergy)/(cutEnergy*maxEnergy) 
      - beta2*G4Log(maxEnergy/cutEnergy)/tmax;

    if( 0.5 == spin ) { cross += 0.5*(maxEnergy - cutEnergy)/energy2; }

    cross *= twopi_mc2_rcl2*chargeSquare/beta2;
  }
 //   G4cout << "BR: e= " << kineticEnergy << " tmin= " << cutEnergy 
 //          << " tmax= " << tmax << " cross= " << cross << G4endl;
 
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::ComputeCrossSectionPerAtom(
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
						 G4double Z, G4double,
                                                 G4double cutEnergy,
                                                 G4double maxEnergy)
{
  G4double cross = Z*ComputeCrossSectionPerElectron
                                         (p,kineticEnergy,cutEnergy,maxEnergy);
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::CrossSectionPerVolume(
					   const G4Material* material,
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
                                                 G4double cutEnergy,
                                                 G4double maxEnergy)
{
  G4double eDensity = material->GetElectronDensity();
  G4double cross = eDensity*ComputeCrossSectionPerElectron
                                         (p,kineticEnergy,cutEnergy,maxEnergy);
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::ComputeDEDXPerVolume(const G4Material* material,
					       const G4ParticleDefinition* p,
					       G4double kineticEnergy,
					       G4double cutEnergy)
{
  G4double tmax  = MaxSecondaryEnergy(p, kineticEnergy);
  G4double tmin  = min(cutEnergy, tmax);
  G4double tkin  = kineticEnergy/massRate;
  G4double dedx  = 0.0;

  if(tkin < lowestKinEnergy) {
    dedx = DEDX(material, lowestKinEnergy)*sqrt(tkin/lowestKinEnergy);
  } else {
    dedx = DEDX(material, tkin); 
  }

  if (cutEnergy < tmax) {

    G4double tau   = kineticEnergy/mass;
    G4double gam   = tau + 1.0;
    G4double bg2   = tau * (tau+2.0);
    G4double beta2 = bg2/(gam*gam);
    G4double x     = tmin/tmax;

    dedx += (G4Log(x) + (1.0 - x)*beta2) * twopi_mc2_rcl2
          * (material->GetElectronDensity())/beta2;
  }

  // now compute the total ionization loss

  if (dedx < 0.0) dedx = 0.0 ;

  dedx *= chargeSquare;

  //G4cout << " tkin(MeV) = " << tkin/MeV << " dedx(MeVxcm^2/g) = " 
  //       << dedx*gram/(MeV*cm2*material->GetDensity()) 
  //       << " q2 = " << chargeSquare <<  G4endl;

  return dedx;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4BraggIonModel::CorrectionsAlongStep(const G4MaterialCutsCouple* couple,
					   const G4DynamicParticle* dp,
					   G4double& eloss,
					   G4double&,
					   G4double /*length*/)
{
  // this method is called only for ions
  const G4ParticleDefinition* p = dp->GetDefinition();
  const G4Material* mat = couple->GetMaterial();
  G4double preKinEnergy = dp->GetKineticEnergy();
  G4double e = preKinEnergy - eloss*0.5;
  if(e < 0.0) { e = preKinEnergy*0.5; }

  G4double q2 = corr->EffectiveChargeSquareRatio(p,mat,e);
  GetModelOfFluctuations()->SetParticleAndCharge(p, q2);
  G4double qfactor = q2*corr->EffectiveChargeCorrection(p,mat,e)/corrFactor; 
  eloss *= qfactor; 

  //G4cout << "G4BraggIonModel::CorrectionsAlongStep e= " <<  e 
  //	 << " qfactor= " << qfactor << " " << p->GetParticleName() <<G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4BraggIonModel::SampleSecondaries(std::vector<G4DynamicParticle*>* vdp,
					const G4MaterialCutsCouple* couple,
					const G4DynamicParticle* dp,
					G4double xmin,
					G4double maxEnergy)
{
  G4double tmax = MaxSecondaryKinEnergy(dp);
  G4double xmax = std::min(tmax, maxEnergy);
  if(xmin >= xmax) { return; }

  G4double kineticEnergy = dp->GetKineticEnergy();
  G4double energy  = kineticEnergy + mass;
  G4double energy2 = energy*energy;
  G4double beta2   = kineticEnergy*(kineticEnergy + 2.0*mass)/energy2;
  G4double grej    = 1.0;
  G4double deltaKinEnergy, f;

  // sampling follows ...
  do {
    G4double q = G4UniformRand();
    deltaKinEnergy = xmin*xmax/(xmin*(1.0 - q) + xmax*q);

    f = 1.0 - beta2*deltaKinEnergy/tmax;

    if(f > grej) {
        G4cout << "G4BraggIonModel::SampleSecondary Warning! "
               << "Majorant " << grej << " < "
               << f << " for e= " << deltaKinEnergy
               << G4endl;
    }

  } while( grej*G4UniformRand() >= f );

  G4ThreeVector deltaDirection;

  if(UseAngularGeneratorFlag()) {
    const G4Material* mat =  couple->GetMaterial();
    G4int Z = SelectRandomAtomNumber(mat);

    deltaDirection = 
      GetAngularDistribution()->SampleDirection(dp, deltaKinEnergy, Z, mat);

  } else {
 
    G4double deltaMomentum =
      sqrt(deltaKinEnergy * (deltaKinEnergy + 2.0*electron_mass_c2));
    G4double cost = deltaKinEnergy * (energy + electron_mass_c2) /
      (deltaMomentum * dp->GetTotalMomentum());
    if(cost > 1.0) { cost = 1.0; }
    G4double sint = sqrt((1.0 - cost)*(1.0 + cost));

    G4double phi = twopi * G4UniformRand() ;

    deltaDirection.set(sint*cos(phi),sint*sin(phi), cost) ;
    deltaDirection.rotateUz(dp->GetMomentumDirection());
  }  

  // create G4DynamicParticle object for delta ray
  G4DynamicParticle* delta = 
    new G4DynamicParticle(theElectron,deltaDirection,deltaKinEnergy);

  vdp->push_back(delta);

  // Change kinematics of primary particle
  kineticEnergy -= deltaKinEnergy;
  G4ThreeVector finalP = dp->GetMomentum() - delta->GetMomentum();
  finalP               = finalP.unit();

  fParticleChange->SetProposedKineticEnergy(kineticEnergy);
  fParticleChange->SetProposedMomentumDirection(finalP);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::MaxSecondaryEnergy(const G4ParticleDefinition* pd,
					     G4double kinEnergy)
{
  if(pd != particle) { SetParticle(pd); }
  G4double tau  = kinEnergy/mass;
  G4double tmax = 2.0*electron_mass_c2*tau*(tau + 2.) /
                  (1. + 2.0*(tau + 1.)*ratio + ratio*ratio);
  return tmax;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4bool G4BraggIonModel::HasMaterial(const G4Material*)
{
  return false;
  /*
  G4String chFormula = material->GetChemicalFormula();
  if("" == chFormula) { return false; }

  // ICRU Report N49, 1993. Ziegler model for He.
  
  static const size_t numberOfMolecula = 11;
  static const G4String molName[numberOfMolecula] = {
    "CaF_2",  "Cellulose_Nitrate",  "LiF", "Policarbonate",  
    "(C_2H_4)_N-Polyethylene",  "(C_2H_4)_N-Polymethly_Methacralate",
    "Polysterene", "SiO_2", "NaI", "H_2O",
    "Graphite" };

  // Search for the material in the table
  for (size_t i=0; i<numberOfMolecula; ++i) {
    if (chFormula == molName[i]) {
      iASTAR = fASTAR->GetIndex(matName[i]);  
      break;
    }
  }
  return (iASTAR >= 0);
  */
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::StoppingPower(const G4Material* material,
					G4double kineticEnergy) 
{
  G4double ionloss = 0.0 ;

  if (iMolecula >= 0) {
  
    // The data and the fit from: 
    // ICRU Report N49, 1993. Ziegler's model for alpha
    // He energy in internal units of parametrisation formula (MeV)

    G4double T = kineticEnergy*rateMassHe2p/MeV ;

    static const G4double a[11][5] = {
       {9.43672, 0.54398, 84.341, 1.3705, 57.422},
       {67.1503, 0.41409, 404.512, 148.97, 20.99},
       {5.11203, 0.453,  36.718,  50.6,  28.058}, 
       {61.793, 0.48445, 361.537, 57.889, 50.674},
       {7.83464, 0.49804, 160.452, 3.192, 0.71922},
       {19.729, 0.52153, 162.341, 58.35, 25.668}, 
       {26.4648, 0.50112, 188.913, 30.079, 16.509},
       {7.8655, 0.5205, 63.96, 51.32, 67.775},
       {8.8965, 0.5148, 339.36, 1.7205, 0.70423},
       {2.959, 0.53255, 34.247, 60.655, 15.153}, 
       {3.80133, 0.41590, 12.9966, 117.83, 242.28} };   

    static const G4double atomicWeight[11] = {
       101.96128, 44.0098, 16.0426, 28.0536, 42.0804,
       104.1512, 44.665, 60.0843, 18.0152, 18.0152, 12.0};       

    G4int i = iMolecula;

    // Free electron gas model
    if ( T < 0.001 ) {
      G4double slow  = a[i][0] ;
      G4double shigh = G4Log( 1.0 + a[i][3]*1000.0 + a[i][4]*0.001 )
	 * a[i][2]*1000.0 ;
      ionloss  = slow*shigh / (slow + shigh) ;
      ionloss *= sqrt(T*1000.0) ;

      // Main parametrisation
    } else {
      G4double slow  = a[i][0] * G4Exp(G4Log(T*1000.0)*a[i][1]) ;
      G4double shigh = G4Log( 1.0 + a[i][3]/T + a[i][4]*T ) * a[i][2]/T ;
      ionloss = slow*shigh / (slow + shigh) ;
       /*
	 G4cout << "## " << i << ". T= " << T << " slow= " << slow
	 << " a0= " << a[i][0] << " a1= " << a[i][1] 
	 << " shigh= " << shigh 
	 << " dedx= " << ionloss << " q^2= " <<  HeEffChargeSquare(z, T*MeV)
	 << G4endl;
       */
    }
    if ( ionloss < 0.0) ionloss = 0.0 ;

    // He effective charge
    G4double aa = atomicWeight[iMolecula];
    ionloss /= (HeEffChargeSquare(0.5*aa, T)*aa);

  // pure material (normally not the case for this function)
  } else if(1 == (material->GetNumberOfElements())) {
    G4double z = material->GetZ() ;
    ionloss = ElectronicStoppingPower( z, kineticEnergy ) ;  
  }
  
  return ionloss;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::ElectronicStoppingPower(G4double z,
                                                  G4double kineticEnergy) const
{
  G4double ionloss ;
  G4int i = G4int(z)-1 ;  // index of atom
  if(i < 0)  i = 0 ;
  if(i > 91) i = 91 ;

  // The data and the fit from:
  // ICRU Report 49, 1993. Ziegler's type of parametrisations.
  // Proton kinetic energy for parametrisation (keV/amu)

   // He energy in internal units of parametrisation formula (MeV)
  G4double T = kineticEnergy*rateMassHe2p/MeV ;

  static const G4double a[92][5] = {
    {0.35485, 0.6456, 6.01525,  20.8933, 4.3515
   },{ 0.58,    0.59,   6.3,     130.0,   44.07
   },{ 1.42,    0.49,   12.25,    32.0,    9.161
   },{ 2.206,   0.51,   15.32,    0.25,    8.995 //Be Ziegler77
       // },{ 2.1895,  0.47183,7.2362,   134.30,  197.96 //Be from ICRU
   },{ 3.691,   0.4128, 18.48,    50.72,   9.0
   },{ 3.83523, 0.42993,12.6125,  227.41,  188.97
   },{ 1.9259,  0.5550, 27.15125, 26.0665, 6.2768
   },{ 2.81015, 0.4759, 50.0253,  10.556,  1.0382
   },{ 1.533,   0.531,  40.44,    18.41,   2.718
   },{ 2.303,   0.4861, 37.01,    37.96,   5.092
       // Z= 11-20
   },{ 9.894,   0.3081, 23.65,    0.384,   92.93
   },{ 4.3,     0.47,   34.3,     3.3,     12.74
   },{ 2.5,     0.625,  45.7,     0.1,     4.359
   },{ 2.1,     0.65,   49.34,    1.788,   4.133
   },{ 1.729,   0.6562, 53.41,    2.405,   3.845
   },{ 1.402,   0.6791, 58.98,    3.528,   3.211
   },{ 1.117,   0.7044, 69.69,    3.705,    2.156
   },{ 2.291,   0.6284, 73.88,    4.478,    2.066
   },{ 8.554,   0.3817, 83.61,    11.84,    1.875
   },{ 6.297,   0.4622, 65.39,    10.14,    5.036
       // Z= 21-30     
   },{ 5.307,   0.4918, 61.74,    12.4,    6.665
   },{ 4.71,    0.5087, 65.28,    8.806,    5.948
   },{ 6.151,   0.4524, 83.0,    18.31,    2.71
   },{ 6.57,    0.4322, 84.76,    15.53,    2.779
   },{ 5.738,   0.4492, 84.6,    14.18,    3.101
   },{ 5.013,   0.4707, 85.8,    16.55,    3.211
   },{ 4.32,    0.4947, 76.14,    10.85,    5.441
   },{ 4.652,   0.4571, 80.73,    22.0,    4.952
   },{ 3.114,   0.5236, 76.67,    7.62,    6.385
   },{ 3.114,   0.5236, 76.67,    7.62,    7.502
       // Z= 31-40
   },{ 3.114,   0.5236, 76.67,    7.62,    8.514
   },{ 5.746,   0.4662, 79.24,    1.185,    7.993
   },{ 2.792,   0.6346, 106.1,    0.2986,   2.331
   },{ 4.667,   0.5095, 124.3,    2.102,    1.667
   },{ 2.44,    0.6346, 105.0,    0.83,    2.851
   },{ 1.413,   0.7377, 147.9,    1.466,    1.016
   },{ 11.72,   0.3826, 102.8,    9.231,    4.371
   },{ 7.126,   0.4804, 119.3,    5.784,    2.454
   },{ 11.61,   0.3955, 146.7,    7.031,    1.423
   },{ 10.99,   0.41,   163.9,   7.1,      1.052
       // Z= 41-50
   },{ 9.241,   0.4275, 163.1,    7.954,    1.102
   },{ 9.276,   0.418,  157.1,   8.038,    1.29
   },{ 3.999,   0.6152, 97.6,    1.297,    5.792
   },{ 4.306,   0.5658, 97.99,    5.514,    5.754
   },{ 3.615,   0.6197, 86.26,    0.333,    8.689
   },{ 5.8,     0.49,   147.2,   6.903,    1.289
   },{ 5.6,     0.49,   130.0,   10.0,     2.844
   },{ 3.55,    0.6068, 124.7,    1.112,    3.119
   },{ 3.6,     0.62,   105.8,   0.1692,   6.026
   },{ 5.4,     0.53,   103.1,   3.931,    7.767
       // Z= 51-60
   },{ 3.97,    0.6459, 131.8,    0.2233,   2.723
   },{ 3.65,    0.64,   126.8,   0.6834,   3.411
   },{ 3.118,   0.6519, 164.9,    1.208,    1.51
   },{ 3.949,   0.6209, 200.5,    1.878,    0.9126
   },{ 14.4,    0.3923, 152.5,    8.354,    2.597
   },{ 10.99,   0.4599, 138.4,    4.811,    3.726
   },{ 16.6,    0.3773, 224.1,    6.28,    0.9121
   },{ 10.54,   0.4533, 159.3,   4.832,    2.529
   },{ 10.33,   0.4502, 162.0,   5.132,    2.444
   },{ 10.15,   0.4471, 165.6,   5.378,    2.328
       // Z= 61-70
   },{ 9.976,   0.4439, 168.0,   5.721,    2.258
   },{ 9.804,   0.4408, 176.2,   5.675,    1.997
   },{ 14.22,   0.363,  228.4,   7.024,    1.016
   },{ 9.952,   0.4318, 233.5,   5.065,    0.9244
   },{ 9.272,   0.4345, 210.0,   4.911,    1.258
   },{ 10.13,   0.4146, 225.7,   5.525,    1.055
   },{ 8.949,   0.4304, 213.3,   5.071,    1.221
   },{ 11.94,   0.3783, 247.2,   6.655,    0.849
   },{ 8.472,   0.4405, 195.5,   4.051,    1.604
   },{ 8.301,   0.4399, 203.7,   3.667,    1.459
       // Z= 71-80
   },{ 6.567,   0.4858, 193.0,   2.65,     1.66
   },{ 5.951,   0.5016, 196.1,   2.662,    1.589
   },{ 7.495,   0.4523, 251.4,   3.433,    0.8619
   },{ 6.335,   0.4825, 255.1,   2.834,    0.8228
   },{ 4.314,   0.5558, 214.8,   2.354,    1.263
   },{ 4.02,    0.5681, 219.9,   2.402,    1.191
   },{ 3.836,   0.5765, 210.2,   2.742,    1.305
   },{ 4.68,    0.5247, 244.7,   2.749,    0.8962
   },{ 2.892,   0.6204, 208.6,   2.415,    1.416 //Au Z77
       // },{ 3.223,   0.5883, 232.7,   2.954,    1.05  //Au ICRU
   },{ 2.892,   0.6204, 208.6,   2.415,    1.416
       // Z= 81-90
   },{ 4.728,   0.5522, 217.0,   3.091,    1.386
   },{ 6.18,    0.52,   170.0,   4.0,      3.224
   },{ 9.0,     0.47,   198.0,   3.8,      2.032
   },{ 2.324,   0.6997, 216.0,   1.599,    1.399
   },{ 1.961,   0.7286, 223.0,   1.621,    1.296
   },{ 1.75,    0.7427, 350.1,   0.9789,   0.5507
   },{ 10.31,   0.4613, 261.2,   4.738,    0.9899
   },{ 7.962,   0.519,  235.7,   4.347,    1.313
   },{ 6.227,   0.5645, 231.9,   3.961,    1.379
   },{ 5.246,   0.5947, 228.6,   4.027,    1.432
       // Z= 91-92
   },{ 5.408,   0.5811, 235.7,   3.961,    1.358
   },{ 5.218,   0.5828, 245.0,   3.838,    1.25}
  };

  // Free electron gas model
  if ( T < 0.001 ) {
    G4double slow  = a[i][0] ;
    G4double shigh = G4Log( 1.0 + a[i][3]*1000.0 + a[i][4]*0.001 )
                   * a[i][2]*1000.0 ;
    ionloss  = slow*shigh / (slow + shigh) ;
    ionloss *= sqrt(T*1000.0) ;

  // Main parametrisation
  } else {
    G4double slow  = a[i][0] * G4Exp(G4Log(T*1000.0)*a[i][1]) ;
    G4double shigh = G4Log( 1.0 + a[i][3]/T + a[i][4]*T ) * a[i][2]/T ;
    ionloss = slow*shigh / (slow + shigh) ;
    /*
    G4cout << "## " << i << ". T= " << T << " slow= " << slow
           << " a0= " << a[i][0] << " a1= " << a[i][1] 
           << " shigh= " << shigh 
           << " dedx= " << ionloss << " q^2= " <<  HeEffChargeSquare(z, T*MeV) 
	   << G4endl;
    */
  }
  if ( ionloss < 0.0) { ionloss = 0.0; }

  // He effective charge
  ionloss /= HeEffChargeSquare(z, T);

  return ionloss;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::DEDX(const G4Material* material,
                                     G4double kineticEnergy)
{
  G4double eloss = 0.0;
  // check DB
  if(material != currentMaterial) {
    currentMaterial = material;
    iASTAR    = -1;
    iMolecula = -1;
    if( !HasMaterial(material) ) { iASTAR = fASTAR->GetIndex(material); }
  }

  const G4int numberOfElements = material->GetNumberOfElements();
  const G4double* theAtomicNumDensityVector =
                                 material->GetAtomicNumDensityVector();

  if( iASTAR >= 0 ) {
    G4double T = kineticEnergy*rateMassHe2p;
    return fASTAR->GetElectronicDEDX(iASTAR, T)*material->GetDensity()/
      HeEffChargeSquare(fASTAR->GetEffectiveZ(iASTAR), T/MeV);

  } else if(iMolecula >= 0) {

    eloss = StoppingPower(material, kineticEnergy)*
      material->GetDensity()/amu;

  // pure material
  } else if(1 == numberOfElements) {

    G4double z = material->GetZ();
    eloss = ElectronicStoppingPower(z, kineticEnergy)
                               * (material->GetTotNbOfAtomsPerVolume());

  // Brugg's rule calculation
  } else {
    const G4ElementVector* theElementVector =
                           material->GetElementVector() ;

    //  loop for the elements in the material
    for (G4int i=0; i<numberOfElements; i++)
    {
      const G4Element* element = (*theElementVector)[i] ;
      eloss   += ElectronicStoppingPower(element->GetZ(), kineticEnergy)
                                   * theAtomicNumDensityVector[i];
    }
  }
  return eloss*theZieglerFactor;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggIonModel::HeEffChargeSquare(G4double z, 
                                            G4double kinEnergyHeInMeV) const
{
  // The aproximation of He effective charge from:
  // J.F.Ziegler, J.P. Biersack, U. Littmark
  // The Stopping and Range of Ions in Matter,
  // Vol.1, Pergamon Press, 1985

  static const G4double c[6] = {0.2865,  0.1266, -0.001429,
				0.02402,-0.01135, 0.001475};

  G4double e = std::max(0.0, G4Log(kinEnergyHeInMeV*massFactor));
  G4double x = c[0] ;
  G4double y = 1.0 ;
  for (G4int i=1; i<6; i++) {
    y *= e ;
    x += y * c[i] ;
  }

  G4double w = 7.6 -  e ;
  w = 1.0 + (0.007 + 0.00005*z) * G4Exp( -w*w ) ;
  w = 4.0 * (1.0 - G4Exp(-x)) * w * w ;

  return w;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

