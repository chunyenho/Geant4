#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//      G4ScreeningMottCrossSection.cc
//-------------------------------------------------------------------
//
// GEANT4 Class header file
//
// File name:    G4ScreeningMottCrossSection
//
// Author:      Cristina Consolandi
//
// Creation date: 20.10.2011  
//
// Modifications:
// 27-05-2012 Added Analytic Fitting to the Mott Cross Section by means of G4MottCoefficients class.
//
//
// Class Description:
//	Computation of electron Coulomb Scattering Cross Section.
//	Suitable for high energy electrons and light target materials. 
//
//      Reference:
//      M.J. Boschini et al.
//     "Non Ionizing Energy Loss induced by Electrons in the Space Environment"
//      Proc. of the 13th International Conference on Particle Physics and Advanced Technology 
//      (13th ICPPAT, Como 3-7/10/2011), World Scientific (Singapore).
//	Available at: http://arxiv.org/abs/1111.4042v4
//
//      1) Mott Differential Cross Section Approximation:  
//	   For Target material up to Z=92 (U):
//         As described in http://arxiv.org/abs/1111.4042v4 
//         par. 2.1 , eq. (16)-(17)
//         Else (Z>92):
//	   W. A. McKinley and H. Fashbach, Phys. Rev. 74, (1948) 1759.
//      2) Screening coefficient: 
//      vomn G. Moliere, Z. Naturforsh A2 (1947), 133-145; A3 (1948), 78.
//      3) Nuclear Form Factor: 
//      A.V. Butkevich et al. Nucl. Instr. and Meth. in Phys. Res. A 488 (2002), 282-294.
//
// -------------------------------------------------------------------------------------
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4ScreeningMottCrossSection.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4MottCoefficients.hh"
#include "Randomize.hh"
#include "G4Proton.hh"
#include "G4LossTableManager.hh"
#include "G4NucleiProperties.hh"
#include "G4Element.hh"
#include "G4UnitsTable.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......


using namespace std;

G4ScreeningMottCrossSection::G4ScreeningMottCrossSection():
   cosThetaMin(1.0),
   cosThetaMax(-1.0),
   alpha(fine_structure_const),
   htc2(hbarc_squared),
   e2(electron_mass_c2*classic_electr_radius) 
{
  TotalCross=0;

  fNistManager = G4NistManager::Instance();
  particle=0;

  spin = mass = mu_rel=0;
  tkinLab = momLab2 = invbetaLab2=0;
  tkin = mom2 = invbeta2=beta=gamma=0;

  Trec=targetZ = targetMass = As =0;
  etag = ecut = 0.0;

  targetA = 0;

  cosTetMinNuc=0;
  cosTetMaxNuc=0;

  for(G4int i=0 ; i<5; i++){
    for(G4int j=0; j< 6; j++){
      coeffb[i][j]=0;
    }
  }

  mottcoeff = new G4MottCoefficients();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ScreeningMottCrossSection::~G4ScreeningMottCrossSection()
{
  delete mottcoeff;
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4ScreeningMottCrossSection::Initialise(const G4ParticleDefinition* p,
                                          G4double CosThetaLim)
{
  SetupParticle(p);
  tkin = targetZ = mom2 = DBL_MIN;
  ecut = etag = DBL_MAX;
  particle = p;
  cosThetaMin = CosThetaLim; 

}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
void G4ScreeningMottCrossSection::SetScreeningCoefficient()
{
  G4double alpha2=alpha*alpha;
  //Bohr radius 
  G4double a0=  Bohr_radius  ;//0.529e-8*cm;
  //Thomas-Fermi screening length
  G4double aU=0.88534*a0/pow(targetZ,1./3.);
  G4double twoR2=aU*aU;

  G4double factor= 1.13 + 3.76*targetZ*targetZ*invbeta2*alpha2;
  As=0.25*(htc2)/(twoR2*mom2)*factor;
  //cout<<"0k .........................As  "<<As<<endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::GetScreeningAngle()
{
  SetScreeningCoefficient();

  //cout<<" As "<<As<<endl;
  if(As < 0.0) { As = 0.0; }
  else if(As > 1.0) { As = 1.0; }

  G4double screenangle=2.*asin(sqrt(As));
  //	cout<<"  screenangle  "<<  screenangle <<endl;
  if(screenangle>=pi) screenangle=pi;
	
  return screenangle;
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4ScreeningMottCrossSection::SetupKinematic(G4double ekin, G4double Z )
{
  //...Target
  G4int iz = G4int(Z);
  G4double A = fNistManager->GetAtomicMassAmu(iz);
  G4int ia = G4int(A);
  G4double mass2 = G4NucleiProperties::GetNuclearMass(ia, iz);

  targetZ = Z;
  targetA = fNistManager->GetAtomicMassAmu(iz);
  targetMass= mass2;

  mottcoeff->SetMottCoeff(targetZ, coeffb);

  //cout<<"......... targetA "<< targetA <<endl;
  //cout<<"......... targetMass "<< targetMass/MeV <<endl;

  // incident particle lab
  tkinLab = ekin;
  momLab2 = tkinLab*(tkinLab + 2.0*mass);
  invbetaLab2 = 1.0 +  mass*mass/momLab2;

  G4double etot = tkinLab + mass;
  G4double ptot = sqrt(momLab2);
  G4double m12  = mass*mass;
                
  // relativistic reduced mass from publucation
  // A.P. Martynenko, R.N. Faustov, Teoret. mat. Fiz. 64 (1985) 179
        
  //incident particle & target nucleus
  G4double Ecm=sqrt(m12 + mass2*mass2 + 2.0*etot*mass2);
  mu_rel=mass*mass2/Ecm;
  G4double momCM= ptot*mass2/Ecm;
  // relative system
  mom2 = momCM*momCM;
  invbeta2 = 1.0 +  mu_rel*mu_rel/mom2;
  tkin = momCM*sqrt(invbeta2) - mu_rel;//Ekin of mu_rel
  G4double  beta2=1./invbeta2;
  beta=std::sqrt(beta2) ;
  G4double gamma2= 1./(1.-beta2);
  gamma=std::sqrt(gamma2);

  //.........................................................

  G4double screenangle=GetScreeningAngle()/10.;
  //cout<<" screenangle [rad] "<<screenangle/rad <<endl;

  cosTetMinNuc =min( cosThetaMin ,cos(screenangle));
  cosTetMaxNuc =cosThetaMax;
	
  //cout<<"ok..............mu_rel "<<mu_rel<<endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::FormFactor2ExpHof(G4double angle)
{
  G4double M=targetMass; 
  G4double E=tkinLab;
  G4double Etot=E+mass;
  G4double Tmax=2.*M*E*(E+2.*mass)/(mass*mass+M*M+2.*M*Etot);
  G4double T=Tmax*pow(sin(angle/2.),2.);
  G4double q2=T*(T+2.*M);
  q2/=htc2;//1/cm2
  G4double RN=1.27e-13*pow(targetA,0.27)*cm;
  G4double xN= (RN*RN*q2);
  G4double den=(1.+xN/12.);
  G4double FN=1./(den*den);
  G4double form2=(FN*FN);

  return form2;

  //cout<<"..................... form2 "<< form2<<endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::McFcorrection(G4double angle )
{
  G4double  beta2=1./invbeta2;
  G4double sintmezzi=std::sin(angle/2.);
  G4double sin2tmezzi = sintmezzi*sintmezzi;
  G4double R=1.-beta2*sin2tmezzi + targetZ*alpha*beta*pi*sintmezzi*(1.-sintmezzi);
  return R;
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
G4double G4ScreeningMottCrossSection::RatioMottRutherford(G4double angle)
{
  G4double R=0;
  G4double fcost=std::sqrt((1. -cos(angle)));
  G4double a[5];
  G4double shift=0.7181228;
  G4double beta0= beta -shift;

  for(G4int j=0 ;j<=4;j++){
    a[j]=0;
  }

  for(G4int j=0 ;j<=4;j++){
    for(G4int k=0;k<=5;k++ ){  
      a[j]+=coeffb[j][k]*pow(beta0,k);
    }
  }

  for(G4int j=0 ;j<=4 ;j++){
    R+=a[j]* pow(fcost,j);
  }
  return R;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::NuclearCrossSection()
{
  if(cosTetMaxNuc >= cosTetMinNuc) return 0.0;

  TotalCross=0;

  G4double anglemin =std::acos(cosTetMinNuc);
  G4double anglemax =std::acos(cosTetMaxNuc); 

  static const G4double limit = 1.e-9;
  if(anglemin < limit) {
    anglemin = GetScreeningAngle()/10.;
    if(anglemin < limit) { anglemin = limit; }
  }

  //cout<<" anglemin  "<< anglemin <<endl;

  G4double loganglemin=log10(anglemin);
  G4double loganglemax=log10(anglemax);
  G4double logdangle=0.01;

  G4int bins=(G4int)((loganglemax-loganglemin)/logdangle);

  vector<G4double> angle;
  vector<G4double> tet;
  vector<G4double> dangle;
  vector<G4double> cross;

  for(G4int i=0; i<=bins; i++ ){
    tet.push_back(0);
    dangle.push_back(0);
    angle.push_back(pow(10.,loganglemin+logdangle*i));
    cross.push_back(0);
  }

  G4int  dim = tet.size();
  //cout<<"dim--- "<<dim<<endl;

  for(G4int i=0; i<dim;i++){

    if(i!=dim-1){
      dangle[i]=(angle[i+1]-angle[i]);
      tet[i]=(angle[i+1]+angle[i])/2.;
    }else if(i==dim-1){
      break;
    }

    G4double R=0;
    G4double F2=FormFactor2ExpHof(tet[i]);
			
    if (coeffb[0][0]!=0){
      //cout<<" Mott....targetZ "<< targetZ<<endl;	
      R=RatioMottRutherford(tet[i]);
    } else if (coeffb[0][0]==0){
      // cout<<" McF.... targetZ "<< targetZ<<endl;
      R=McFcorrection(tet[i]);
    }

    //cout<<"----------------- R "<<R<<" F2 "<<F2<<endl;
    //                cout<<"angle "<<tet[i] << " F2 "<<F2<<endl;

    G4double e4=e2*e2;
    G4double den=2.*As+2.*pow(sin(tet[i]/2.),2.);
    G4double func=1./(den*den);

    G4double fatt= targetZ/(mu_rel*gamma*beta*beta);
    G4double sigma=e4*fatt*fatt*func;
    cross[i]=F2*R*sigma;
    G4double pi2sintet=2.*pi*sin(tet[i]);

    TotalCross+=pi2sintet*cross[i]*dangle[i];
  }//end integral

  //cout<< "ok ......... TotalCross "<<TotalCross<<endl;
  return TotalCross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::AngleDistribution(G4double anglein)
{
  G4double total=TotalCross ;
  G4double fatt= e2*targetZ/(mu_rel*gamma*beta*beta);
  G4double fatt2=fatt*fatt;
  total/=fatt2;

  G4double R=0;
  if (coeffb[0][0]!=0){
    //   cout<<" Mott....targetZ "<< targetZ<<endl;      
    R=RatioMottRutherford(anglein);
  } else if (coeffb[0][0]==0){
    // cout<<" McF.... targetZ "<< targetZ<<endl;
    R=McFcorrection(anglein);
  }

  G4double y=2.*pi*sin(anglein)*R*FormFactor2ExpHof(anglein)/
    ((2*As+2.*pow(sin(anglein/2.),2.))*(2.*As+2.*pow(sin(anglein/2.),2.) ));

  return y/total;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ScreeningMottCrossSection::GetScatteringAngle()
{
  //cout<<" G4ScreeningMottCrossSection::SampleCosineTheta ............."<<endl;	
  if(cosTetMaxNuc >= cosTetMinNuc) return 0.0;

  G4double anglemin=std::acos(cosTetMinNuc);	
  G4double anglemax= std::acos(cosTetMaxNuc);

  static const G4double limit = 1.e-9;
  if(anglemin < limit) {
    anglemin = GetScreeningAngle()/10.;
    if(anglemin < limit) { anglemin = limit; }
  }

  //	cout<<"................ tkinLab  "<< G4BestUnit(tkinLab,"Energy") << " anglemin=  "<<anglemin<<endl;
  //cout<<"anglemax=  "<<anglemax<<endl;
  G4double r =G4UniformRand();

  G4double loganglemin=log10(anglemin);
  G4double loganglemax=log10(anglemax);
  G4double logdangle=0.01;

  G4int bins=(G4int)((loganglemax-loganglemin)/logdangle);

  std::vector<G4double> angle;
  std::vector<G4double> tet;
  std::vector<G4double> dangle;
  
  for(G4int i=0; i<=bins; i++ ){
    tet.push_back(0);
    dangle.push_back(0);
    angle.push_back(pow(10.,loganglemin+logdangle*i));
  }

  G4int  dim = tet.size();
  G4double scattangle=0;
  G4double y=0;
  G4double dy=0;
  G4double area=0;

  for(G4int i=0; i<dim;i++){
    
    if(i!=dim-1){
      dangle[i]=(angle[i+1]-angle[i]);
      tet[i]=(angle[i+1]+angle[i])/2.;
    }else if(i==dim-1){
      break;
    }

    y+=AngleDistribution(tet[i])*dangle[i];
    dy= y-area ;
    area=y;

    if(r >=y-dy && r<=y+dy ){	
      scattangle= angle[i] +G4UniformRand()*dangle[i];
      //cout<<"y "<<y <<" r  "<< r  << " y/r "<<y/r << endl;
      break;
    }			
  }
  return scattangle;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4ThreeVector G4ScreeningMottCrossSection::GetNewDirection(){

  G4ThreeVector dir(0.0,0.0,1.0);
	
  G4double z1=GetScatteringAngle();
  
  G4double sint = sin(z1);
  G4double cost = sqrt(1.0 - sint*sint);
  G4double phi  = twopi* G4UniformRand();
  G4double dirx = sint*cos(phi);
  G4double diry = sint*sin(phi);
  G4double dirz = cost;

  //.......set Trc
  G4double etot=tkinLab+mass;
  G4double mass2=targetMass;
  Trec=(1.0 - cost)* mass2*(etot*etot - mass*mass )/
    (mass*mass + mass2*mass2+ 2.*mass2*etot);
       
  dir.set(dirx,diry,dirz);

  return dir;
}


