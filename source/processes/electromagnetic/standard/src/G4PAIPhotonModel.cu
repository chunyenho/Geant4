#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4PAIPhotonModel.cc 87015 2014-11-21 16:24:10Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class 
// File name:     G4PAIPhotonModel.cc
//
// Author: Vladimir.Grichine@cern.ch based on G4PAIModel class
//
// Creation date: 20.05.2004
//
// Modifications:
//
// 17.08.04 V.Grichine, bug fixed for Tkin<=0 in SampleSecondary
// 16.08.04 V.Grichine, bug fixed in massRatio for DEDX, CrossSection, SampleSecondary
// 11.04.05 Major optimisation of internal interfaces (V.Ivantchenko)
// 12.06.13 V. Grichine Bug fixed in SampleSecondaries for scaled Tkin (fMass -> proton_mass_c2)
//
//

#include "G4PAIPhotonModel.hh"

#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

#include "G4Region.hh"
#include "G4PhysicsLogVector.hh"
#include "G4PhysicsFreeVector.hh"
#include "G4PhysicsTable.hh"
#include "G4ProductionCutsTable.hh"
#include "G4EmCalculator.hh"
#include "G4ProductionCuts.hh"
#include "G4MaterialCutsCouple.hh"
#include "G4MaterialTable.hh"

#include "G4SandiaTable.hh"
#include "G4PAIxSection.hh"

#include "Randomize.hh"
#include "G4Electron.hh"
#include "G4Positron.hh"
#include "G4Gamma.hh"
#include "G4Poisson.hh"
#include "G4Step.hh"
#include "G4Material.hh"
#include "G4DynamicParticle.hh"
#include "G4ParticleDefinition.hh"
#include "G4ParticleChangeForLoss.hh"
#include "G4GeometryTolerance.hh"

////////////////////////////////////////////////////////////////////////

using namespace std;

G4PAIPhotonModel::G4PAIPhotonModel(const G4ParticleDefinition* p, const G4String& nam)
  : G4VEmModel(nam),G4VEmFluctuationModel(nam),
  fLowestKineticEnergy(10.0*keV),
  fHighestKineticEnergy(100.*TeV),
  fTotBin(200),
  fMeanNumber(20),
  fParticle(0),
  fHighKinEnergy(100.*TeV),
  fLowKinEnergy(2.0*MeV),
  fTwoln10(2.0*log(10.0)),
  fBg2lim(0.0169),
  fTaulim(8.4146e-3)
{
  fVerbose  = 0;
  fGamma    = G4Gamma::Gamma();
  fElectron = G4Electron::Electron();
  fPositron = G4Positron::Positron();

  fProtonEnergyVector = new G4PhysicsLogVector(fLowestKineticEnergy,
					       fHighestKineticEnergy,
					       fTotBin);
  fPAItransferTable     = 0;
  fPAIphotonTable       = 0;
  fPAIplasmonTable      = 0;

  fPAIdEdxTable         = 0;
  fSandiaPhotoAbsCof    = 0;
  fdEdxVector           = 0;

  fLambdaVector         = 0;
  fdNdxCutVector        = 0;
  fdNdxCutPhotonVector  = 0;
  fdNdxCutPlasmonVector = 0;

  fSandiaIntervalNumber = 0;
  fMatIndex = 0;
  fCutCouple = 0;
  fMaterial = 0;

  fParticleChange = 0;

  if(p) { SetParticle(p); }
  else  { SetParticle(fElectron); }

  isInitialised      = false;
}

////////////////////////////////////////////////////////////////////////////

G4PAIPhotonModel::~G4PAIPhotonModel()
{
  //  if(fdEdxVector)         delete fdEdxVector;
  //  if ( fLambdaVector)     delete fLambdaVector;
  // if ( fdNdxCutVector)    delete fdNdxCutVector;

  if( fPAItransferTable )
  {
        delete fPAItransferTable;
  }
  if( fPAIphotonTable )
  {
        delete fPAIphotonTable;
  }
  if( fPAIplasmonTable )
  {
        delete fPAIplasmonTable;
  }
  if( fProtonEnergyVector )
  {
        delete fProtonEnergyVector;
  }
}

///////////////////////////////////////////////////////////////////////////////

void G4PAIPhotonModel::SetParticle(const G4ParticleDefinition* p)
{
  fParticle = p;
  fMass = fParticle->GetPDGMass();
  fSpin = fParticle->GetPDGSpin();
  G4double q = fParticle->GetPDGCharge()/eplus;
  fChargeSquare = q*q;
  fLowKinEnergy *= fMass/proton_mass_c2;
  fRatio = electron_mass_c2/fMass;
  fQc = fMass/fRatio;
}

////////////////////////////////////////////////////////////////////////////

void G4PAIPhotonModel::Initialise(const G4ParticleDefinition* p,
				  const G4DataVector&)
{
  //  G4cout<<"G4PAIPhotonModel::Initialise for "<<p->GetParticleName()<<G4endl;
  if(isInitialised) { return; }
  isInitialised = true;

  if(!fParticle) SetParticle(p);

  fParticleChange = GetParticleChangeForLoss();

  //const G4ProductionCutsTable* theCoupleTable =
  //      G4ProductionCutsTable::GetProductionCutsTable();
  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();
  size_t numOfMat   = G4Material::GetNumberOfMaterials();
  size_t numRegions = fPAIRegionVector.size();

  for(size_t iReg = 0; iReg < numRegions; ++iReg) // region loop
  {
    const G4Region* curReg = fPAIRegionVector[iReg];
    G4Region* reg = const_cast<G4Region*>(curReg); 

    for(size_t jMat = 0; jMat < numOfMat; ++jMat) // material loop
    {
      G4Material* material = (*theMaterialTable)[jMat];
      const G4MaterialCutsCouple* couple = reg->FindCouple(material);

      //	theCoupleTable->GetMaterialCutsCouple( material, 
      //	 curReg->GetProductionCuts() );

      if(couple) {
	//G4cout << "Reg <" <<curReg->GetName() << ">  mat <" 
	//       << material->GetName() << ">  fCouple= " 
	//       << couple<<"  " << p->GetParticleName() <<G4endl;

	fMaterialCutsCoupleVector.push_back(couple);

	fMatIndex = jMat;
	fMaterial = material;

	// ComputeSandiaPhotoAbsCof();
        fSandia.Initialize(material);
	BuildPAIonisationTable();
	/*
        if( fSandiaPhotoAbsCof ) // delete SANDIA cofs've been used for pai-xsc
        {
          for( G4int i = 0;i < fSandiaIntervalNumber;i++)
          {
            delete[] fSandiaPhotoAbsCof[i];
          }
          delete[] fSandiaPhotoAbsCof;
          fSandiaPhotoAbsCof = 0;
        }
	*/
	fPAIxscBank.push_back(fPAItransferTable);
	fPAIphotonBank.push_back(fPAIphotonTable);
	fPAIplasmonBank.push_back(fPAIplasmonTable);
	fPAIdEdxBank.push_back(fPAIdEdxTable);
	fdEdxTable.push_back(fdEdxVector);

	BuildLambdaVector(couple);

	fdNdxCutTable.push_back(fdNdxCutVector);
	fdNdxCutPhotonTable.push_back(fdNdxCutPhotonVector);
	fdNdxCutPlasmonTable.push_back(fdNdxCutPlasmonVector);
	fLambdaTable.push_back(fLambdaVector);
      }
    }
  }
}

//////////////////////////////////////////////////////////////////

void G4PAIPhotonModel::InitTest(const G4ParticleDefinition* p, G4MaterialCutsCouple* couple, 
				G4double phE, G4double eTkin)
{
  //  G4cout<<"G4PAIPhotonModel::InitTest for "<<p->GetParticleName()<<G4endl;
  if(isInitialised) { return; }
  isInitialised = true;

  if( !fParticle ) SetParticle(p);

  fParticleChange = GetParticleChangeForLoss();

  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();

  size_t jMat, numOfMat   = G4Material::GetNumberOfMaterials();


  // const G4MaterialCutsCouple* couple = new G4MaterialCutsCouple(material,cuts);

  if( couple ) 
  {
    const G4Material* material = couple->GetMaterial();

    fMaterialCutsCoupleVector.push_back(couple);

    for( jMat = 0; jMat < numOfMat; ++jMat ) // material loop
    {
      if( material->GetName() == (*theMaterialTable)[jMat]->GetName() ) break;
    }
    fMatIndex = jMat;
    G4Material* mat = (*theMaterialTable)[jMat];
	fMaterial = mat;


    // ComputeSandiaPhotoAbsCof();
        fSandia.Initialize(mat);
	BuildPAIonisationTable();
	/*
        if( fSandiaPhotoAbsCof ) // delete SANDIA cofs've been used for pai-xsc
        {
          for( G4int i = 0;i < fSandiaIntervalNumber;i++)
          {
            delete[] fSandiaPhotoAbsCof[i];
          }
          delete[] fSandiaPhotoAbsCof;
          fSandiaPhotoAbsCof = 0;
        }
	*/
	fPAIxscBank.push_back(fPAItransferTable);
	fPAIphotonBank.push_back(fPAIphotonTable);
	fPAIplasmonBank.push_back(fPAIplasmonTable);
	fPAIdEdxBank.push_back(fPAIdEdxTable);
	fdEdxTable.push_back(fdEdxVector);

	BuildLambdaVector(couple,phE,eTkin);

	fdNdxCutTable.push_back(fdNdxCutVector);
	fdNdxCutPhotonTable.push_back(fdNdxCutPhotonVector);
	fdNdxCutPlasmonTable.push_back(fdNdxCutPlasmonVector);
	fLambdaTable.push_back(fLambdaVector);
  }    
}

//////////////////////////////////////////////////////////////////

void G4PAIPhotonModel::ComputeSandiaPhotoAbsCof()
{
  G4int i, j, numberOfElements;
  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();

  G4SandiaTable thisMaterialSandiaTable(fMatIndex);
  numberOfElements = (*theMaterialTable)[fMatIndex]->
                                              GetNumberOfElements();
  G4int* thisMaterialZ = new G4int[numberOfElements];

  for(i=0;i<numberOfElements;i++)  
  {
    thisMaterialZ[i] = 
    (G4int)(*theMaterialTable)[fMatIndex]->GetElement(i)->GetZ();
  }  
  fSandiaIntervalNumber = thisMaterialSandiaTable.SandiaIntervals
                           (thisMaterialZ,numberOfElements);

  fSandiaIntervalNumber = thisMaterialSandiaTable.SandiaMixing
                           ( thisMaterialZ ,
                             (*theMaterialTable)[fMatIndex]->GetFractionVector() ,
        		     numberOfElements,fSandiaIntervalNumber);
   
  fSandiaPhotoAbsCof = new G4double*[fSandiaIntervalNumber];

  for(i=0;i<fSandiaIntervalNumber;i++)  fSandiaPhotoAbsCof[i] = new G4double[5];
   
  for( i = 0; i < fSandiaIntervalNumber; i++ )
  {
    fSandiaPhotoAbsCof[i][0] = thisMaterialSandiaTable.GetPhotoAbsorpCof(i+1,0); 

    for( j = 1; j < 5; j++ )
    {
      fSandiaPhotoAbsCof[i][j] = thisMaterialSandiaTable.
	                              GetPhotoAbsorpCof(i+1,j)*
                 (*theMaterialTable)[fMatIndex]->GetDensity();
    }
  }
  delete[] thisMaterialZ;
}

////////////////////////////////////////////////////////////////////////////
//
// Build tables for the ionization energy loss
//  the tables are built for MATERIALS
//                           *********

void
G4PAIPhotonModel::BuildPAIonisationTable()
{
  G4double LowEdgeEnergy , ionloss;
  G4double /*massRatio,*/ tau, Tmax, Tmin, Tkin, deltaLow, /*gamma,*/ bg2;
  /*
  if( fPAItransferTable )
  {
     fPAItransferTable->clearAndDestroy();
     delete fPAItransferTable;
  }
  */
  fPAItransferTable = new G4PhysicsTable(fTotBin);
  /*
  if( fPAIratioTable )
  {
     fPAIratioTable->clearAndDestroy();
     delete fPAIratioTable;
  }
  */
  fPAIphotonTable = new G4PhysicsTable(fTotBin);
  fPAIplasmonTable = new G4PhysicsTable(fTotBin);
  /*
  if( fPAIdEdxTable )
  {
     fPAIdEdxTable->clearAndDestroy();
     delete fPAIdEdxTable;
  }
  */
  fPAIdEdxTable = new G4PhysicsTable(fTotBin);

  //  if(fdEdxVector) delete fdEdxVector;
  fdEdxVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					 fHighestKineticEnergy,
					 fTotBin               );
  // Tmin     = fSandiaPhotoAbsCof[0][0];      // low energy Sandia interval
  Tmin = fSandia.GetSandiaMatTablePAI(0,0);      // low energy Sandia interval
  deltaLow = 100.*eV; // 0.5*eV;

  for (G4int i = 0; i <= fTotBin; i++)  //The loop for the kinetic energy
  {
    LowEdgeEnergy = fProtonEnergyVector->GetLowEdgeEnergy(i);
    tau = LowEdgeEnergy/proton_mass_c2;
    //    if(tau < 0.01)  tau = 0.01;
    //gamma = tau +1.;
    // G4cout<<"gamma = "<<gamma<<endl;
    bg2 = tau*(tau + 2. );
    //massRatio = electron_mass_c2/proton_mass_c2;
    Tmax = MaxSecondaryEnergy(fParticle, LowEdgeEnergy); 
    // G4cout<<"proton Tkin = "<<LowEdgeEnergy/MeV<<" MeV"
    // <<" Tmax = "<<Tmax/MeV<<" MeV"<<G4endl;
    // Tkin = DeltaCutInKineticEnergyNow;

    // if ( DeltaCutInKineticEnergyNow > Tmax)         // was <
    Tkin = Tmax;
    if ( Tkin < Tmin + deltaLow )  // low energy safety
    {
      Tkin = Tmin + deltaLow;
    }
    fPAIxSection.Initialize(fMaterial, Tkin, bg2, 
			    &fSandia);
    /*
    G4PAIxSection protonPAI( fMatIndex,
                             Tkin,
                             bg2,
                             fSandiaPhotoAbsCof,
                             fSandiaIntervalNumber  );

    */
    // G4cout<<"ionloss = "<<ionloss*cm/keV<<" keV/cm"<<endl;
    // G4cout<<"n1 = "<<fPAIxSection.GetIntegralPAIxSection(1)*cm<<" 1/cm"<<endl;
    // G4cout<<"fPAIxSection.GetSplineSize() = "<<
    //    fPAIxSection.GetSplineSize()<<G4endl<<G4endl;

    G4PhysicsFreeVector* transferVector = new
                             G4PhysicsFreeVector(fPAIxSection.GetSplineSize());
    G4PhysicsFreeVector* photonVector = new
                             G4PhysicsFreeVector(fPAIxSection.GetSplineSize());
    G4PhysicsFreeVector* plasmonVector = new
                             G4PhysicsFreeVector(fPAIxSection.GetSplineSize());
    G4PhysicsFreeVector* dEdxVector = new
                             G4PhysicsFreeVector(fPAIxSection.GetSplineSize());

    for( G4int k = 0; k < fPAIxSection.GetSplineSize(); k++ )
    {
      transferVector->PutValue( k ,
                                fPAIxSection.GetSplineEnergy(k+1),
                                fPAIxSection.GetIntegralPAIxSection(k+1) );
      photonVector->PutValue( k ,
                                fPAIxSection.GetSplineEnergy(k+1),
                                fPAIxSection.GetIntegralCerenkov(k+1) );
      plasmonVector->PutValue( k ,
                                fPAIxSection.GetSplineEnergy(k+1),
                                fPAIxSection.GetIntegralPlasmon(k+1) );
      dEdxVector->PutValue( k ,
                                fPAIxSection.GetSplineEnergy(k+1),
                                fPAIxSection.GetIntegralPAIdEdx(k+1) );
    }
    ionloss = fPAIxSection.GetMeanEnergyLoss();   //  total <dE/dx>
    if ( ionloss <= 0.)  ionloss = DBL_MIN;
    fdEdxVector->PutValue(i,ionloss);

    fPAItransferTable->insertAt(i,transferVector);
    fPAIphotonTable->insertAt(i,photonVector);
    fPAIplasmonTable->insertAt(i,plasmonVector);
    fPAIdEdxTable->insertAt(i,dEdxVector);

  }                                        // end of Tkin loop
  //  theLossTable->insert(fdEdxVector);
  // end of material loop
  // G4cout<<"G4PAIonisation::BuildPAIonisationTable() have been called"<<G4endl;
  // G4cout<<"G4PAIonisation::BuildLossTable() have been called"<<G4endl;
}

///////////////////////////////////////////////////////////////////////
//
// Build mean free path tables for the delta ray production process
//     tables are built for MATERIALS
//

void
G4PAIPhotonModel::BuildLambdaVector(const G4MaterialCutsCouple* matCutsCouple)
{
  G4int i;
  G4double dNdxCut,dNdxPhotonCut,dNdxPlasmonCut, lambda, deltaCutInKineticEnergyNow, photonCutInKineticEnergyNow;
  G4double kCarTolerance = G4GeometryTolerance::GetInstance()
                           ->GetSurfaceTolerance();

  G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();

  // G4EmCalculator converter;

  // const G4Material* material = matCutsCouple->GetMaterial();

  // G4double rangeGamma    = matCutsCouple->GetProductionCuts()->GetProductionCut(0); 
  // G4double rangeElectron = matCutsCouple->GetProductionCuts()->GetProductionCut(1); 

  size_t numOfCouples = theCoupleTable->GetTableSize();
  size_t jMatCC;

  for (jMatCC = 0; jMatCC < numOfCouples; jMatCC++ )
  {
    if( matCutsCouple == theCoupleTable->GetMaterialCutsCouple(jMatCC) ) break;
  }
  if( jMatCC == numOfCouples && jMatCC > 0 ) jMatCC--;

  const vector<G4double>*  deltaCutInKineticEnergy = theCoupleTable->GetEnergyCutsVector(idxG4ElectronCut);
  const vector<G4double>*  photonCutInKineticEnergy = theCoupleTable->GetEnergyCutsVector(idxG4GammaCut);

  if (fLambdaVector)         delete fLambdaVector;
  if (fdNdxCutVector)        delete fdNdxCutVector;
  if (fdNdxCutPhotonVector)  delete fdNdxCutPhotonVector;
  if (fdNdxCutPlasmonVector) delete fdNdxCutPlasmonVector;

  fLambdaVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutPhotonVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutPlasmonVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );

  deltaCutInKineticEnergyNow  = (*deltaCutInKineticEnergy)[jMatCC];
  photonCutInKineticEnergyNow = (*photonCutInKineticEnergy)[jMatCC];

  // deltaCutInKineticEnergyNow  = theCoupleTable->ConvertRangeToEnergy(fElectron,material,rangeElectron);
  // photonCutInKineticEnergyNow = theCoupleTable->ConvertRangeToEnergy(fGamma,material,rangeGamma);

  // photonCutInKineticEnergyNow = converter.GetKinEnergy(rangeGamma, fGamma,material);
  // deltaCutInKineticEnergyNow  = converter.GetKinEnergy(rangeElectron, fElectron,material);

  if(fVerbose > 0)
  {
    G4cout<<"PAIPhotonModel deltaCutInKineticEnergyNow = "
	  <<deltaCutInKineticEnergyNow/keV<<" keV"<<G4endl;
    G4cout<<"PAIPhotonModel photonCutInKineticEnergyNow = "
	  <<photonCutInKineticEnergyNow/keV<<" keV"<<G4endl;
  }
  for ( i = 0; i <= fTotBin; i++ )
  {
    dNdxPhotonCut  = GetdNdxPhotonCut(i,photonCutInKineticEnergyNow);
    dNdxPlasmonCut = GetdNdxPlasmonCut(i,deltaCutInKineticEnergyNow);

    dNdxCut        =  dNdxPhotonCut + dNdxPlasmonCut;
    lambda         = dNdxCut <= DBL_MIN ? DBL_MAX: 1.0/dNdxCut;

    if (lambda <= 1000*kCarTolerance) lambda = 1000*kCarTolerance; // Mmm ???

    fLambdaVector->PutValue(i, lambda);

    fdNdxCutVector->PutValue(i, dNdxCut);
    fdNdxCutPhotonVector->PutValue(i, dNdxPhotonCut);
    fdNdxCutPlasmonVector->PutValue(i, dNdxPlasmonCut);
  }
}

///////////////////////////////////////////////////////////////////////
//
// Build mean free path tables for the delta ray production process
//     tables are built for MATERIALS
//

void
G4PAIPhotonModel::BuildLambdaVector(const G4MaterialCutsCouple* matCutsCouple, G4double photEnergy, G4double eTkin)
{
  G4int i;
  G4double dNdxCut,dNdxPhotonCut,dNdxPlasmonCut, lambda, deltaCutInKineticEnergyNow, photonCutInKineticEnergyNow;
  G4double kCarTolerance = G4GeometryTolerance::GetInstance()
                           ->GetSurfaceTolerance();

  G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();


  //  const G4Material* material = matCutsCouple->GetMaterial();


  size_t numOfCouples = theCoupleTable->GetTableSize();
  size_t jMatCC;

  for (jMatCC = 0; jMatCC < numOfCouples; jMatCC++ )
  {
    if( matCutsCouple == theCoupleTable->GetMaterialCutsCouple(jMatCC) ) break;
  }
  if( jMatCC == numOfCouples && jMatCC > 0 ) jMatCC--;

  if (fLambdaVector)         delete fLambdaVector;
  if (fdNdxCutVector)        delete fdNdxCutVector;
  if (fdNdxCutPhotonVector)  delete fdNdxCutPhotonVector;
  if (fdNdxCutPlasmonVector) delete fdNdxCutPlasmonVector;

  fLambdaVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutPhotonVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );
  fdNdxCutPlasmonVector = new G4PhysicsLogVector( fLowestKineticEnergy,
					  fHighestKineticEnergy,
					  fTotBin                );


  photonCutInKineticEnergyNow = photEnergy;
  deltaCutInKineticEnergyNow  = eTkin;

  if(fVerbose > 0)
  {
    G4cout<<"PAIPhotonModel deltaCutInKineticEnergyNow = "
	  <<deltaCutInKineticEnergyNow/keV<<" keV"<<G4endl;
    G4cout<<"PAIPhotonModel photonCutInKineticEnergyNow = "
	  <<photonCutInKineticEnergyNow/keV<<" keV"<<G4endl;
  }
  for ( i = 0; i <= fTotBin; i++ )
  {
    dNdxPhotonCut  = GetdNdxPhotonCut(i,photonCutInKineticEnergyNow);
    dNdxPlasmonCut = GetdNdxPlasmonCut(i,deltaCutInKineticEnergyNow);

    dNdxCut        =  dNdxPhotonCut + dNdxPlasmonCut;
    lambda         = dNdxCut <= DBL_MIN ? DBL_MAX: 1.0/dNdxCut;

    if (lambda <= 1000*kCarTolerance) lambda = 1000*kCarTolerance; // Mmm ???

    fLambdaVector->PutValue(i, lambda);

    fdNdxCutVector->PutValue(i, dNdxCut);
    fdNdxCutPhotonVector->PutValue(i, dNdxPhotonCut);
    fdNdxCutPlasmonVector->PutValue(i, dNdxPlasmonCut);
  }
}

///////////////////////////////////////////////////////////////////////
//
// Returns integral PAI cross section for energy transfers >= transferCut

G4double  
G4PAIPhotonModel::GetdNdxCut( G4int iPlace, G4double transferCut)
{ 
  G4int iTransfer;
  G4double x1, x2, y1, y2, dNdxCut;
  // G4cout<<"iPlace = "<<iPlace<<"; "<<"transferCut = "<<transferCut<<G4endl;
  // G4cout<<"size = "<<G4int((*fPAItransferTable)(iPlace)->GetVectorLength())
  //           <<G4endl;  
  for( iTransfer = 0; 
       iTransfer < G4int((*fPAItransferTable)(iPlace)->GetVectorLength()); 
       iTransfer++)
  {
    if(transferCut <= (*fPAItransferTable)(iPlace)->GetLowEdgeEnergy(iTransfer))
    {
      break;
    }
  }  
  if ( iTransfer >= G4int((*fPAItransferTable)(iPlace)->GetVectorLength()) )
  {
      iTransfer = (*fPAItransferTable)(iPlace)->GetVectorLength() - 1;
  }
  if (iTransfer == 0) return (*(*fPAItransferTable)(iPlace))(iTransfer);
  y1 = (*(*fPAItransferTable)(iPlace))(iTransfer-1);
  y2 = (*(*fPAItransferTable)(iPlace))(iTransfer);
  // G4cout<<"y1 = "<<y1<<"; "<<"y2 = "<<y2<<G4endl;
  x1 = (*fPAItransferTable)(iPlace)->GetLowEdgeEnergy(iTransfer-1);
  x2 = (*fPAItransferTable)(iPlace)->GetLowEdgeEnergy(iTransfer);
  // G4cout<<"x1 = "<<x1<<"; "<<"x2 = "<<x2<<G4endl;

  if ( y1 == y2 )    dNdxCut = y2;
  else
  {
    //  if ( x1 == x2  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    //    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*0.5;
    else             dNdxCut = y1 + (transferCut - x1)*(y2 - y1)/(x2 - x1);      
  }
  //  G4cout<<""<<dNdxCut<<G4endl;
  return dNdxCut;
}

///////////////////////////////////////////////////////////////////////
//
// Returns integral PAI cherenkovcross section for energy transfers >= transferCut

G4double  
G4PAIPhotonModel::GetdNdxPhotonCut( G4int iPlace, G4double transferCut)
{ 
  G4int iTransfer;
  G4double x1, x2, y1, y2, dNdxCut;
  // G4cout<<"iPlace = "<<iPlace<<"; "<<"transferCut = "<<transferCut<<G4endl;
  // G4cout<<"size = "<<G4int((*fPAIphotonTable)(iPlace)->GetVectorLength())
  //           <<G4endl;  
  for( iTransfer = 0; 
       iTransfer < G4int((*fPAIphotonTable)(iPlace)->GetVectorLength()); 
       iTransfer++)
  {
    if(transferCut <= (*fPAIphotonTable)(iPlace)->GetLowEdgeEnergy(iTransfer))
    {
      break;
    }
  }  
  if ( iTransfer >= G4int((*fPAIphotonTable)(iPlace)->GetVectorLength()) )
  {
      iTransfer = (*fPAIphotonTable)(iPlace)->GetVectorLength() - 1;
  }
  if (iTransfer == 0) return (*(*fPAIphotonTable)(iPlace))(iTransfer);
  y1 = (*(*fPAIphotonTable)(iPlace))(iTransfer-1);
  y2 = (*(*fPAIphotonTable)(iPlace))(iTransfer);
  // G4cout<<"y1 = "<<y1<<"; "<<"y2 = "<<y2<<G4endl;
  x1 = (*fPAIphotonTable)(iPlace)->GetLowEdgeEnergy(iTransfer-1);
  x2 = (*fPAIphotonTable)(iPlace)->GetLowEdgeEnergy(iTransfer);
  // G4cout<<"x1 = "<<x1<<"; "<<"x2 = "<<x2<<G4endl;

  if ( y1 == y2 )    dNdxCut = y2;
  else
  {
    //  if ( x1 == x2  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    //    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*0.5;
    else             dNdxCut = y1 + (transferCut - x1)*(y2 - y1)/(x2 - x1);      
  }
  //  G4cout<<""<<dNdxPhotonCut<<G4endl;
  return dNdxCut;
}

///////////////////////////////////////////////////////////////////////
//
// Returns integral PAI cross section for energy transfers >= transferCut

G4double  
G4PAIPhotonModel::GetdNdxPlasmonCut( G4int iPlace, G4double transferCut)
{ 
  G4int iTransfer;
  G4double x1, x2, y1, y2, dNdxCut;

  // G4cout<<"iPlace = "<<iPlace<<"; "<<"transferCut = "<<transferCut<<G4endl;
  // G4cout<<"size = "<<G4int((*fPAIPlasmonTable)(iPlace)->GetVectorLength())
  //           <<G4endl;  
  for( iTransfer = 0; 
       iTransfer < G4int((*fPAIplasmonTable)(iPlace)->GetVectorLength()); 
       iTransfer++)
  {
    if(transferCut <= (*fPAIplasmonTable)(iPlace)->GetLowEdgeEnergy(iTransfer))
    {
      break;
    }
  }  
  if ( iTransfer >= G4int((*fPAIplasmonTable)(iPlace)->GetVectorLength()) )
  {
      iTransfer = (*fPAIplasmonTable)(iPlace)->GetVectorLength() - 1;
  }
  if (iTransfer == 0) return (*(*fPAIplasmonTable)(iPlace))(iTransfer);
  y1 = (*(*fPAIplasmonTable)(iPlace))(iTransfer-1);
  y2 = (*(*fPAIplasmonTable)(iPlace))(iTransfer);
  // G4cout<<"y1 = "<<y1<<"; "<<"y2 = "<<y2<<G4endl;
  x1 = (*fPAIplasmonTable)(iPlace)->GetLowEdgeEnergy(iTransfer-1);
  x2 = (*fPAIplasmonTable)(iPlace)->GetLowEdgeEnergy(iTransfer);
  // G4cout<<"x1 = "<<x1<<"; "<<"x2 = "<<x2<<G4endl;

  if ( y1 == y2 )    dNdxCut = y2;
  else
  {
    //  if ( x1 == x2  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    //    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*G4UniformRand();
    if ( std::abs(x1-x2) <= eV  ) dNdxCut = y1 + (y2 - y1)*0.5;
    else             dNdxCut = y1 + (transferCut - x1)*(y2 - y1)/(x2 - x1);      
  }
  //  G4cout<<""<<dNdxPlasmonCut<<G4endl;
  return dNdxCut;
}

///////////////////////////////////////////////////////////////////////
//
// Returns integral dEdx for energy transfers >= transferCut

G4double  
G4PAIPhotonModel::GetdEdxCut( G4int iPlace, G4double transferCut)
{ 
  G4int iTransfer;
  G4double x1, x2, y1, y2, dEdxCut;
  // G4cout<<"iPlace = "<<iPlace<<"; "<<"transferCut = "<<transferCut<<G4endl;
  // G4cout<<"size = "<<G4int((*fPAIdEdxTable)(iPlace)->GetVectorLength())
  //           <<G4endl;  
  for( iTransfer = 0; 
       iTransfer < G4int((*fPAIdEdxTable)(iPlace)->GetVectorLength()); 
       iTransfer++)
  {
    if(transferCut <= (*fPAIdEdxTable)(iPlace)->GetLowEdgeEnergy(iTransfer))
    {
      break;
    }
  }  
  if ( iTransfer >= G4int((*fPAIdEdxTable)(iPlace)->GetVectorLength()) )
  {
      iTransfer = (*fPAIdEdxTable)(iPlace)->GetVectorLength() - 1;
  }
  if (iTransfer == 0) return (*(*fPAIdEdxTable)(iPlace))(iTransfer);
  y1 = (*(*fPAIdEdxTable)(iPlace))(iTransfer-1);
  y2 = (*(*fPAIdEdxTable)(iPlace))(iTransfer);
  // G4cout<<"y1 = "<<y1<<"; "<<"y2 = "<<y2<<G4endl;
  x1 = (*fPAIdEdxTable)(iPlace)->GetLowEdgeEnergy(iTransfer-1);
  x2 = (*fPAIdEdxTable)(iPlace)->GetLowEdgeEnergy(iTransfer);
  // G4cout<<"x1 = "<<x1<<"; "<<"x2 = "<<x2<<G4endl;

  if ( y1 == y2 )    dEdxCut = y2;
  else
  {
    //  if ( x1 == x2  ) dEdxCut = y1 + (y2 - y1)*G4UniformRand();
    //    if ( std::abs(x1-x2) <= eV  ) dEdxCut = y1 + (y2 - y1)*G4UniformRand();
    if ( std::abs(x1-x2) <= eV  ) dEdxCut = y1 + (y2 - y1)*0.5;
    else             dEdxCut = y1 + (transferCut - x1)*(y2 - y1)/(x2 - x1);      
  }
  //  G4cout<<""<<dEdxCut<<G4endl;
  return dEdxCut;
}

//////////////////////////////////////////////////////////////////////////////

G4double G4PAIPhotonModel::ComputeDEDXPerVolume(const G4Material*,
						const G4ParticleDefinition* p,
						G4double kineticEnergy,
						G4double cutEnergy)
{
  G4int iTkin,iPlace;
  size_t jMat;

  //G4double cut = std::min(MaxSecondaryEnergy(p, kineticEnergy), cutEnergy);
  G4double cut = cutEnergy;

  G4double particleMass = p->GetPDGMass();
  G4double scaledTkin   = kineticEnergy*proton_mass_c2/particleMass;
  G4double charge       = p->GetPDGCharge()/eplus;
  G4double charge2      = charge*charge;
  G4double dEdx         = 0.;
  const G4MaterialCutsCouple* matCC = CurrentCouple();

  for( jMat = 0;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC == fMaterialCutsCoupleVector[jMat] ) break;
  }
  if(jMat == fMaterialCutsCoupleVector.size() && jMat > 0) jMat--;
  /*
  G4cout << "G4PAIPhotonModel::ComputeDEDXPerVolume: jMat= " << jMat 
  	 << " jMax= " << fMaterialCutsCoupleVector.size()
         << " matCC: " << matCC;
  if(matCC) G4cout << " mat: " << matCC->GetMaterial()->GetName();
  G4cout << G4endl;
  G4cout << fPAIdEdxTable << " " << fdEdxVector << " " 
	 << fProtonEnergyVector << G4endl;
  */
  fPAIdEdxTable = fPAIdEdxBank[jMat];
  fdEdxVector = fdEdxTable[jMat];
  for(iTkin = 0; iTkin <= fTotBin; iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin)) break;    
  }
  iPlace = iTkin - 1;
  if(iPlace < 0) iPlace = 0;
  else if(iPlace > fTotBin) iPlace = fTotBin;
  dEdx = charge2*( (*fdEdxVector)(iPlace) - GetdEdxCut(iPlace,cut) );  

  if( dEdx < 0.) dEdx = 0.;
  return dEdx;
}

//////////////////////////////////////////////////////////////////////
//
// Return xsc for mean interaction length

G4double G4PAIPhotonModel::CrossSectionPerVolume( const G4Material*,
						  const G4ParticleDefinition* p,
						  G4double kineticEnergy,
						  G4double cutEnergy,
						  G4double maxEnergy  ) 
{
  G4int iTkin,iPlace;
  size_t jMat, jMatCC;
  G4double tmax = std::min(MaxSecondaryEnergy(p, kineticEnergy), maxEnergy);
  if(cutEnergy >= tmax) return 0.0;
  G4double particleMass = p->GetPDGMass();
  G4double scaledTkin   = kineticEnergy*proton_mass_c2/particleMass;
  G4double charge       = p->GetPDGCharge();
  G4double charge2      = charge*charge, cross, cross1, cross2;
  G4double photon1, photon2, plasmon1, plasmon2;

  const G4MaterialCutsCouple* matCC = CurrentCouple();

  const G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();

  size_t numOfCouples = theCoupleTable->GetTableSize();

  for (jMatCC = 0; jMatCC < numOfCouples; jMatCC++ )
  {
    if( matCC == theCoupleTable->GetMaterialCutsCouple(jMatCC) ) break;
  }
  if( jMatCC == numOfCouples && jMatCC > 0 ) jMatCC--;

  const vector<G4double>*  photonCutInKineticEnergy = theCoupleTable->
                                GetEnergyCutsVector(idxG4GammaCut);

  G4double photonCut = (*photonCutInKineticEnergy)[jMatCC];

  for( jMat = 0;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC == fMaterialCutsCoupleVector[jMat] ) break;
  }
  if(jMat == fMaterialCutsCoupleVector.size() && jMat > 0) jMat--;

  fPAItransferTable = fPAIxscBank[jMat];
  fPAIphotonTable   = fPAIphotonBank[jMat];
  fPAIplasmonTable  = fPAIplasmonBank[jMat];

  for(iTkin = 0; iTkin <= fTotBin; iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin)) break;    
  }
  iPlace = iTkin - 1;
  if(iPlace < 0) iPlace = 0;

  // G4cout<<"iPlace = "<<iPlace<<"; tmax = "
  // <<tmax<<"; cutEnergy = "<<cutEnergy<<G4endl;  
  photon1 = GetdNdxPhotonCut(iPlace,tmax);  
  photon2 = GetdNdxPhotonCut(iPlace,photonCut); 
 
  plasmon1 = GetdNdxPlasmonCut(iPlace,tmax);  
  plasmon2 = GetdNdxPlasmonCut(iPlace,cutEnergy); 
 
  cross1 = photon1 + plasmon1;    
  // G4cout<<"cross1 = "<<cross1<<G4endl;  
  cross2 = photon2 + plasmon2;    
  // G4cout<<"cross2 = "<<cross2<<G4endl;  
  cross  = (cross2 - cross1)*charge2;
  // G4cout<<"cross = "<<cross<<G4endl;  

  if( cross < 0. ) cross = 0.;
  return cross;
}
//////////////////////////////////////////////////////////////////////
//
// Return xsc for mean interaction length in test

G4double G4PAIPhotonModel::GetXscPerVolume( const G4Material*,
					    const G4ParticleDefinition* p,
						  G4double kineticEnergy,
						  G4double photonCut,
						  G4double cutEnergy,
						  G4double maxEnergy  ) 
{
  G4int iTkin,iPlace;
  size_t jMat, jMatCC;
  G4double tmax = std::min(MaxSecondaryEnergy(p, kineticEnergy), maxEnergy);
  if(cutEnergy >= tmax) return 0.0;
  G4double particleMass = p->GetPDGMass();
  G4double scaledTkin   = kineticEnergy*proton_mass_c2/particleMass;
  G4double charge       = p->GetPDGCharge();
  G4double charge2      = charge*charge, cross, cross1, cross2;
  G4double photon1, photon2, plasmon1, plasmon2;

  const G4MaterialCutsCouple* matCC = CurrentCouple();

  const G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();

  size_t numOfCouples = theCoupleTable->GetTableSize();

  for (jMatCC = 0; jMatCC < numOfCouples; jMatCC++ )
  {
    if( matCC == theCoupleTable->GetMaterialCutsCouple(jMatCC) ) break;
  }
  if( jMatCC == numOfCouples && jMatCC > 0 ) jMatCC--;

  // const vector<G4double>*  photonCutInKineticEnergy = theCoupleTable->
  //                               GetEnergyCutsVector(idxG4GammaCut);
  // G4double photonCut = (*photonCutInKineticEnergy)[jMatCC];

  for( jMat = 0;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC == fMaterialCutsCoupleVector[jMat] ) break;
  }
  if(jMat == fMaterialCutsCoupleVector.size() && jMat > 0) jMat--;

  fPAItransferTable = fPAIxscBank[jMat];
  fPAIphotonTable   = fPAIphotonBank[jMat];
  fPAIplasmonTable  = fPAIplasmonBank[jMat];

  for(iTkin = 0; iTkin <= fTotBin; iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin)) break;    
  }
  iPlace = iTkin - 1;
  if(iPlace < 0) iPlace = 0;

  // G4cout<<"iPlace = "<<iPlace<<"; tmax = "
  // <<tmax<<"; cutEnergy = "<<cutEnergy<<G4endl;  
  photon1 = GetdNdxPhotonCut(iPlace,tmax);  
  photon2 = GetdNdxPhotonCut(iPlace,photonCut); 
 
  plasmon1 = GetdNdxPlasmonCut(iPlace,tmax);  
  plasmon2 = GetdNdxPlasmonCut(iPlace,cutEnergy); 
 
  cross1 = photon1 + plasmon1;    
  // G4cout<<"cross1 = "<<cross1<<G4endl;  
  cross2 = photon2 + plasmon2;    
  // G4cout<<"cross2 = "<<cross2<<G4endl;  
  cross  = (cross2 - cross1)*charge2;
  // G4cout<<"cross = "<<cross<<G4endl;  

  if( cross < 0. ) cross = 0.;
  return cross;
}

///////////////////////////////////////////////////////////////////////////
//
// It is analog of PostStepDoIt in terms of secondary electron or photon to
// be returned as G4Dynamicparticle*.
//

void G4PAIPhotonModel::SampleSecondaries(std::vector<G4DynamicParticle*>* vdp,
					 const G4MaterialCutsCouple* matCC,
					 const G4DynamicParticle* dp,
					 G4double tmin,
					 G4double maxEnergy)
{
  size_t jMat;
  for( jMat = 0;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC == fMaterialCutsCoupleVector[jMat] ) break;
  }
  if( jMat == fMaterialCutsCoupleVector.size() && jMat > 0 ) jMat--;

  fPAItransferTable = fPAIxscBank[jMat];
  fPAIphotonTable   = fPAIphotonBank[jMat];
  fPAIplasmonTable  = fPAIplasmonBank[jMat];

  fdNdxCutVector        = fdNdxCutTable[jMat];
  fdNdxCutPhotonVector  = fdNdxCutPhotonTable[jMat];
  fdNdxCutPlasmonVector = fdNdxCutPlasmonTable[jMat];

  G4double tmax = std::min(MaxSecondaryEnergy(dp->GetDefinition(),dp->GetKineticEnergy()), maxEnergy);
  if( tmin >= tmax && fVerbose > 0) 
  {
    G4cout<<"G4PAIPhotonModel::SampleSecondary: tmin >= tmax "<<G4endl;
  }

  G4ThreeVector direction = dp->GetMomentumDirection();
  G4double particleMass  = dp->GetMass();
  G4double kineticEnergy = dp->GetKineticEnergy();
  G4double scaledTkin    = kineticEnergy*proton_mass_c2/particleMass; // fMass
  G4double totalEnergy   = kineticEnergy + particleMass;
  G4double pSquare       = kineticEnergy*(totalEnergy+particleMass);

  G4int iTkin;
  for(iTkin=0;iTkin<=fTotBin;iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin))  break;
  }
  G4int iPlace = iTkin - 1;
  if(iPlace < 0) iPlace = 0;

  G4double dNdxPhotonCut  = (*fdNdxCutPhotonVector)(iPlace);  
  G4double dNdxPlasmonCut = (*fdNdxCutPlasmonVector)(iPlace);  
  G4double dNdxCut        = dNdxPhotonCut  + dNdxPlasmonCut;
 
  G4double ratio;
  if (dNdxCut > 0.) ratio = dNdxPhotonCut/dNdxCut;
  else              return; // ratio = 0.;

  if(ratio < G4UniformRand() ) // secondary e-
  {
    G4double deltaTkin     = GetPostStepTransfer(fPAIplasmonTable, fdNdxCutPlasmonVector,
                                                 iPlace, scaledTkin);

//  G4cout<<"PAIPhotonModel PlasmonPostStepTransfer = "<<deltaTkin/keV<<" keV"<<G4endl; 
 
    if( deltaTkin <= 0. ) 
    {
      G4cout<<"G4PAIPhotonModel::SampleSecondary e- deltaTkin = "<<deltaTkin<<G4endl;
    }
    if( deltaTkin <= 0.) return;

    if( deltaTkin >= kineticEnergy ) // stop primary
    {
      deltaTkin = kineticEnergy;
      kineticEnergy = 0.0;
    }
    G4double deltaTotalMomentum = sqrt(deltaTkin*(deltaTkin + 2. * electron_mass_c2 ));
    G4double totalMomentum      = sqrt(pSquare);
    G4double costheta           = deltaTkin*(totalEnergy + electron_mass_c2)
                                /(deltaTotalMomentum * totalMomentum);

    if( costheta > 0.99999 ) costheta = 0.99999;
    G4double sintheta = 0.0;
    G4double sin2 = 1. - costheta*costheta;
    if( sin2 > 0.) sintheta = sqrt(sin2);

    //  direction of the delta electron
  
    G4double phi = twopi*G4UniformRand(); 
    G4double dirx = sintheta*cos(phi), diry = sintheta*sin(phi), dirz = costheta;

    G4ThreeVector deltaDirection(dirx,diry,dirz);
    deltaDirection.rotateUz(direction);

    if( kineticEnergy > 0.) // primary change
    {
      kineticEnergy -= deltaTkin;
      G4ThreeVector dir = totalMomentum*direction - deltaTotalMomentum*deltaDirection;
      direction = dir.unit();
      fParticleChange->SetProposedKineticEnergy(kineticEnergy);
      fParticleChange->SetProposedMomentumDirection(direction);
    }
    else // stop primary
    {
      fParticleChange->ProposeTrackStatus(fStopAndKill);
      fParticleChange->SetProposedKineticEnergy(0.0);
    }

   // create G4DynamicParticle object for e- delta ray
 
    G4DynamicParticle* deltaRay = new G4DynamicParticle;
    deltaRay->SetDefinition(G4Electron::Electron());
    deltaRay->SetKineticEnergy( deltaTkin );
    deltaRay->SetMomentumDirection(deltaDirection); 
    vdp->push_back(deltaRay);

  }
  else    // secondary 'Cherenkov' photon
  { 
    G4double deltaTkin     = GetPostStepTransfer(fPAIphotonTable, fdNdxCutPhotonVector,
                                                 iPlace,scaledTkin);

    //  G4cout<<"PAIPhotonModel PhotonPostStepTransfer = "<<deltaTkin/keV<<" keV"<<G4endl; 

    if( deltaTkin <= 0. )
    {
      G4cout<<"G4PAIPhotonModel::SampleSecondary gamma deltaTkin = "<<deltaTkin<<G4endl;
    }
    if( deltaTkin <= 0.) return;

    if( deltaTkin >= kineticEnergy ) // stop primary
    {
      deltaTkin = kineticEnergy;
      kineticEnergy = 0.0;
    }
    G4double costheta = 0.; // G4UniformRand(); // VG: ??? for start only
    G4double sintheta = sqrt((1.+costheta)*(1.-costheta));

    //  direction of the 'Cherenkov' photon  
    G4double phi = twopi*G4UniformRand(); 
    G4double dirx = sintheta*cos(phi), diry = sintheta*sin(phi), dirz = costheta;

    G4ThreeVector deltaDirection(dirx,diry,dirz);
    deltaDirection.rotateUz(direction);

    if( kineticEnergy > 0.) // primary change
    {
      kineticEnergy -= deltaTkin;
      fParticleChange->SetProposedKineticEnergy(kineticEnergy);
    }
    else // stop primary
    {
      fParticleChange->ProposeTrackStatus(fStopAndKill);
      fParticleChange->SetProposedKineticEnergy(0.0);
    }
    // create G4DynamicParticle object for photon ray
 
    G4DynamicParticle* photonRay = new G4DynamicParticle;
    photonRay->SetDefinition( G4Gamma::Gamma() );
    photonRay->SetKineticEnergy( deltaTkin );
    photonRay->SetMomentumDirection(deltaDirection); 

    vdp->push_back(photonRay);
  }
}

///////////////////////////////////////////////////////////////////////////////
//
// test function for losses more than cut

G4double G4PAIPhotonModel::TestSecondaries( G4MaterialCutsCouple* matCC,  G4DynamicParticle* dp,
					 G4double tmin,
					 G4double maxEnergy)
{
  size_t jMat;
  for( jMat = 0;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC->GetMaterial()->GetName() == fMaterialCutsCoupleVector[jMat]->GetMaterial()->GetName() ) break;
  }
  if( jMat == fMaterialCutsCoupleVector.size() && jMat > 0 ) jMat--;

  fPAItransferTable = fPAIxscBank[jMat];
  fPAIphotonTable   = fPAIphotonBank[jMat];
  fPAIplasmonTable  = fPAIplasmonBank[jMat];

  fdNdxCutVector        = fdNdxCutTable[jMat];
  fdNdxCutPhotonVector  = fdNdxCutPhotonTable[jMat];
  fdNdxCutPlasmonVector = fdNdxCutPlasmonTable[jMat];

  G4double tmax = std::min(MaxSecondaryEnergy(dp->GetDefinition(),dp->GetKineticEnergy()), maxEnergy);
  if( tmin >= tmax && fVerbose > 0) 
  {
    G4cout<<"G4PAIPhotonModel::TestSecondaries: tmin >= tmax "<<G4endl;
  }

  G4ThreeVector direction = dp->GetMomentumDirection();
  G4double particleMass  = dp->GetMass();
  G4double kineticEnergy = dp->GetKineticEnergy();
  G4double scaledTkin    = kineticEnergy*proton_mass_c2/particleMass; // fMass
  G4double totalEnergy   = kineticEnergy + particleMass;
  G4double pSquare       = kineticEnergy*(totalEnergy+particleMass);

  G4int iTkin;
  for(iTkin=0;iTkin<=fTotBin;iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin))  break;
  }
  G4int iPlace = iTkin - 1;
  if(iPlace < 0) iPlace = 0;

  G4double dNdxPhotonCut  = (*fdNdxCutPhotonVector)(iPlace);  
  G4double dNdxPlasmonCut = (*fdNdxCutPlasmonVector)(iPlace);  
  G4double dNdxCut        = dNdxPhotonCut  + dNdxPlasmonCut;
 
  G4double ratio, deltaTkin;
  if (dNdxCut > 0.) ratio = dNdxPhotonCut/dNdxCut;
  else              ratio = 0.;

  if(ratio < G4UniformRand() ) // secondary e-
  {
    deltaTkin     = GetPostStepTransfer(fPAIplasmonTable, fdNdxCutPlasmonVector,
                                                 iPlace, scaledTkin);

//  G4cout<<"PAIPhotonModel PlasmonPostStepTransfer = "<<deltaTkin/keV<<" keV"<<G4endl; 
 
    if( deltaTkin <= 0. ) 
    {
      G4cout<<"G4PAIPhotonModel::SampleSecondary e- deltaTkin = "<<deltaTkin<<G4endl;
    }
    if( deltaTkin <= 0.) return 0.;

    G4double deltaTotalMomentum = sqrt(deltaTkin*(deltaTkin + 2. * electron_mass_c2 ));
    G4double totalMomentum      = sqrt(pSquare);
    G4double costheta           = deltaTkin*(totalEnergy + electron_mass_c2)
                                /(deltaTotalMomentum * totalMomentum);

    if( costheta > 0.99999 ) costheta = 0.99999;
    G4double sintheta = 0.0;
    G4double sin2 = 1. - costheta*costheta;
    if( sin2 > 0.) sintheta = sqrt(sin2);

    //  direction of the delta electron
  
    G4double phi = twopi*G4UniformRand(); 
    G4double dirx = sintheta*cos(phi), diry = sintheta*sin(phi), dirz = costheta;

    G4ThreeVector deltaDirection(dirx,diry,dirz);
    deltaDirection.rotateUz(direction);

    // primary change

    kineticEnergy -= deltaTkin;
    G4ThreeVector dir = totalMomentum*direction - deltaTotalMomentum*deltaDirection;
    direction = dir.unit();
    fParticleChange->SetProposedMomentumDirection(direction);

    // create G4DynamicParticle object for e- delta ray
 
    G4DynamicParticle* deltaRay = new G4DynamicParticle;
    deltaRay->SetDefinition(G4Electron::Electron());
    deltaRay->SetKineticEnergy( deltaTkin );
    deltaRay->SetMomentumDirection(deltaDirection); 

  }
  else    // secondary 'Cherenkov' photon
  { 
    deltaTkin     = GetPostStepTransfer(fPAIphotonTable, fdNdxCutPhotonVector,
                                                 iPlace,scaledTkin);

    //  G4cout<<"PAIPhotonModel PhotonPostStepTransfer = "<<deltaTkin/keV<<" keV"<<G4endl; 

    if( deltaTkin <= 0. )
    {
      G4cout<<"G4PAIPhotonModel::SampleSecondary gamma deltaTkin = "<<deltaTkin<<G4endl;
    }
    if( deltaTkin <= 0.) return 0.;

    G4double costheta = 0.; // G4UniformRand(); // VG: ??? for start only
    G4double sintheta = sqrt((1.+costheta)*(1.-costheta));

    //  direction of the 'Cherenkov' photon  
    G4double phi = twopi*G4UniformRand(); 
    G4double dirx = sintheta*cos(phi), diry = sintheta*sin(phi), dirz = costheta;

    G4ThreeVector deltaDirection(dirx,diry,dirz);
    deltaDirection.rotateUz(direction);

    // primary change
    kineticEnergy -= deltaTkin;

    // create G4DynamicParticle object for photon ray
 
    G4DynamicParticle* photonRay = new G4DynamicParticle;
    photonRay->SetDefinition( G4Gamma::Gamma() );
    photonRay->SetKineticEnergy( deltaTkin );
    photonRay->SetMomentumDirection(deltaDirection); 

  }
  fParticleChange->SetProposedKineticEnergy(kineticEnergy);

  return deltaTkin;
}


///////////////////////////////////////////////////////////////////////
//
// Returns post step PAI energy transfer > cut electron/photon energy according to passed 
// scaled kinetic energy of particle

G4double  
G4PAIPhotonModel::GetPostStepTransfer( G4PhysicsTable* pTable,
				       G4PhysicsLogVector* pVector,
                                       G4int iPlace, G4double scaledTkin )
{  
  // G4cout<<"G4PAIPhotonModel::GetPostStepTransfer"<<G4endl;

  G4int iTkin = iPlace+1, iTransfer;
  G4double transfer = 0.0, position, dNdxCut1, dNdxCut2, E1, E2, W1, W2, W;

  dNdxCut1 = (*pVector)(iPlace);  

  //  G4cout<<"iPlace = "<<iPlace<<endl;

  if(iTkin == fTotBin) // Fermi plato, try from left
  {
      position = dNdxCut1*G4UniformRand();

      for( iTransfer = 0;
 iTransfer < G4int((*pTable)(iPlace)->GetVectorLength()); iTransfer++ )
      {
        if(position >= (*(*pTable)(iPlace))(iTransfer)) break;
      }
      transfer = GetEnergyTransfer(pTable,iPlace,position,iTransfer);
  }
  else
  {
    dNdxCut2 = (*pVector)(iPlace+1);  
    if(iTkin == 0) // Tkin is too small, trying from right only
    {
      position = dNdxCut2*G4UniformRand();

      for( iTransfer = 0;
  iTransfer < G4int((*pTable)(iPlace+1)->GetVectorLength()); iTransfer++ )
      {
        if(position >= (*(*pTable)(iPlace+1))(iTransfer)) break;
      }
      transfer = GetEnergyTransfer(pTable,iPlace+1,position,iTransfer);
    } 
    else // general case: Tkin between two vectors of the material
    {
      E1 = fProtonEnergyVector->GetLowEdgeEnergy(iTkin - 1); 
      E2 = fProtonEnergyVector->GetLowEdgeEnergy(iTkin)    ;
      W  = 1.0/(E2 - E1);
      W1 = (E2 - scaledTkin)*W;
      W2 = (scaledTkin - E1)*W;

      position = ( dNdxCut1*W1 + dNdxCut2*W2 )*G4UniformRand();

        // G4cout<<position<<"\t";

      G4int iTrMax1, iTrMax2, iTrMax;

      iTrMax1 = G4int((*pTable)(iPlace)->GetVectorLength());
      iTrMax2 = G4int((*pTable)(iPlace+1)->GetVectorLength());

      if (iTrMax1 >= iTrMax2) iTrMax = iTrMax2;
      else                    iTrMax = iTrMax1;

      for( iTransfer = 0; iTransfer < iTrMax; iTransfer++ )
      {
          if( position >=
          ( (*(*pTable)(iPlace))(iTransfer)*W1 +
            (*(*pTable)(iPlace+1))(iTransfer)*W2) ) break;
      }
      transfer = GetEnergyTransfer(pTable, iPlace, position, iTransfer);
    }
  } 
  //  G4cout<<"PAIPhotonModel PostStepTransfer = "<<transfer/keV<<" keV"<<G4endl; 
  if( transfer < 0.0 ) transfer = 0.0;
  return transfer;
}

///////////////////////////////////////////////////////////////////////
//
// Returns random PAI energy transfer according to passed 
// indexes of particle 

G4double
G4PAIPhotonModel::GetEnergyTransfer( G4PhysicsTable* pTable, G4int iPlace, 
                                     G4double position, G4int iTransfer )
{ 
  G4int iTransferMax;
  G4double x1, x2, y1, y2, energyTransfer;

  if(iTransfer == 0)
  {
    energyTransfer = (*pTable)(iPlace)->GetLowEdgeEnergy(iTransfer);
  }  
  else
  {
    iTransferMax = G4int((*pTable)(iPlace)->GetVectorLength());

    if ( iTransfer >= iTransferMax)  iTransfer = iTransferMax - 1;
    
    y1 = (*(*pTable)(iPlace))(iTransfer-1);
    y2 = (*(*pTable)(iPlace))(iTransfer);

    x1 = (*pTable)(iPlace)->GetLowEdgeEnergy(iTransfer-1);
    x2 = (*pTable)(iPlace)->GetLowEdgeEnergy(iTransfer);

    if ( x1 == x2 )    energyTransfer = x2;
    else
    {
      if ( y1 == y2  ) energyTransfer = x1 + (x2 - x1)*G4UniformRand();
      else
      {
        energyTransfer = x1 + (position - y1)*(x2 - x1)/(y2 - y1);
      }
    }
  }
  return energyTransfer;
}

///////////////////////////////////////////////////////////////////////
//
// Works like AlongStepDoIt method of process family

G4double 
G4PAIPhotonModel::SampleFluctuations( const G4MaterialCutsCouple* matCC,
				      const G4DynamicParticle* aParticle,
				      G4double,
				      G4double step,
				      G4double eloss)
{
  size_t jMat = 0;
  for(;jMat < fMaterialCutsCoupleVector.size(); ++jMat )
  {
    if( matCC == fMaterialCutsCoupleVector[jMat] ) break;
  }
  if(jMat == fMaterialCutsCoupleVector.size()) { return eloss; }

  fPAItransferTable = fPAIxscBank[jMat];
  fPAIphotonTable = fPAIphotonBank[jMat];
  fPAIplasmonTable = fPAIplasmonBank[jMat];

  fdNdxCutVector   = fdNdxCutTable[jMat];
  fdNdxCutPhotonVector   = fdNdxCutPhotonTable[jMat];
  fdNdxCutPlasmonVector   = fdNdxCutPlasmonTable[jMat];

  G4int iTkin, iPlace ;

  // G4cout<<"G4PAIPhotonModel::SampleFluctuations"<<G4endl;

  G4double loss, photonLoss, plasmonLoss, charge2;
 

  G4double Tkin       = aParticle->GetKineticEnergy();
  G4double MassRatio  = proton_mass_c2/aParticle->GetDefinition()->GetPDGMass();
  G4double charge     = aParticle->GetDefinition()->GetPDGCharge();
  charge2             = charge*charge;
  G4double scaledTkin = Tkin*MassRatio;
  G4double cof        = step*charge2;

  for( iTkin = 0; iTkin <= fTotBin; iTkin++)
  {
    if(scaledTkin < fProtonEnergyVector->GetLowEdgeEnergy(iTkin))   break;
  }
  iPlace = iTkin - 1; 
  if( iPlace < 0 ) iPlace = 0;

  photonLoss = GetAlongStepTransfer(fPAIphotonTable,fdNdxCutPhotonVector,
iPlace,scaledTkin,step,cof);

  //  G4cout<<"PAIPhotonModel AlongStepPhotonLoss = "<<photonLoss/keV<<" keV"<<G4endl; 

  plasmonLoss = GetAlongStepTransfer(fPAIplasmonTable,fdNdxCutPlasmonVector,
iPlace,scaledTkin,step,cof);

  //  G4cout<<"PAIPhotonModel AlongStepPlasmonLoss = "<<plasmonLoss/keV<<" keV"<<G4endl; 

  loss = photonLoss + plasmonLoss;

  //  G4cout<<"PAIPhotonModel AlongStepLoss = "<<loss/keV<<" keV"<<G4endl; 

  return loss;
}

///////////////////////////////////////////////////////////////////////
//
// Returns along step PAI energy transfer < cut electron/photon energy according to passed 
// scaled kinetic energy of particle and cof = step*charge*charge

G4double  
G4PAIPhotonModel::GetAlongStepTransfer( G4PhysicsTable* pTable,
				        G4PhysicsLogVector* pVector,
                                        G4int iPlace, G4double scaledTkin,G4double step,
                                        G4double cof )
{  
  G4int iTkin = iPlace + 1, iTransfer;
  G4double loss = 0., position, E1, E2, W1, W2, W, dNdxCut1, dNdxCut2, meanNumber;
  G4double lambda, stepDelta, stepSum=0.;
  G4long numOfCollisions=0;
  G4bool numb = true;

  dNdxCut1 = (*pVector)(iPlace);  

  //  G4cout<<"iPlace = "<<iPlace<<endl;

  if(iTkin == fTotBin) // Fermi plato, try from left
  {
    meanNumber = ((*(*pTable)(iPlace))(0) - dNdxCut1)*cof;
    if(meanNumber < 0.) meanNumber = 0.;
    //  numOfCollisions = RandPoisson::shoot(meanNumber);
    if( meanNumber > 0.) lambda = step/meanNumber;
    else                 lambda = DBL_MAX;
    while(numb)
    {
      stepDelta = G4RandExponential::shoot(lambda);
      stepSum += stepDelta;
      if(stepSum >= step) break;
      numOfCollisions++;
    }   
    
    //     G4cout<<"numOfCollisions = "<<numOfCollisions<<G4endl;

    while(numOfCollisions)
    {
      position = dNdxCut1+
                 ((*(*pTable)(iPlace))(0) - dNdxCut1)*G4UniformRand();

      for( iTransfer = 0;
   iTransfer < G4int((*pTable)(iPlace)->GetVectorLength()); iTransfer++ )
      {
        if(position >= (*(*pTable)(iPlace))(iTransfer)) break;
      }
      loss += GetEnergyTransfer(pTable,iPlace,position,iTransfer);
      numOfCollisions--;
    }
  }
  else
  {
    dNdxCut2 = (*pVector)(iPlace+1); 
 
    if(iTkin == 0) // Tkin is too small, trying from right only
    {
      meanNumber = ((*(*pTable)(iPlace+1))(0) - dNdxCut2)*cof;
      if( meanNumber < 0. ) meanNumber = 0.;
      //  numOfCollisions = G4RandPoisson::shoot(meanNumber);
      if( meanNumber > 0.) lambda = step/meanNumber;
      else                 lambda = DBL_MAX;
      while(numb)
      {
        stepDelta = G4RandExponential::shoot(lambda);
        stepSum += stepDelta;
        if(stepSum >= step) break;
        numOfCollisions++;
      }   

      //  G4cout<<"numOfCollisions = "<<numOfCollisions<<G4endl;

      while(numOfCollisions)
      {
        position = dNdxCut2+
                   ((*(*pTable)(iPlace+1))(0) - dNdxCut2)*G4UniformRand();
   
        for( iTransfer = 0;
   iTransfer < G4int((*pTable)(iPlace+1)->GetVectorLength()); iTransfer++ )
        {
          if(position >= (*(*pTable)(iPlace+1))(iTransfer)) break;
        }
        loss += GetEnergyTransfer(pTable,iPlace+1,position,iTransfer);
        numOfCollisions--;
      }
    } 
    else // general case: Tkin between two vectors of the material
    {
      E1 = fProtonEnergyVector->GetLowEdgeEnergy(iTkin - 1); 
      E2 = fProtonEnergyVector->GetLowEdgeEnergy(iTkin)    ;
       W = 1.0/(E2 - E1);
      W1 = (E2 - scaledTkin)*W;
      W2 = (scaledTkin - E1)*W;

      // G4cout<<"(*(*pTable)(iPlace))(0) = "<<
      //   (*(*pTable)(iPlace))(0)<<G4endl;
      // G4cout<<"(*(*pTable)(iPlace+1))(0) = "<<
      //     (*(*pTable)(iPlace+1))(0)<<G4endl;

      meanNumber=( ((*(*pTable)(iPlace))(0)-dNdxCut1)*W1 + 
		   ((*(*pTable)(iPlace+1))(0)-dNdxCut2)*W2 )*cof;
      if(meanNumber<0.0) meanNumber = 0.0;
      //  numOfCollisions = G4RandPoisson::shoot(meanNumber);
      if( meanNumber > 0.) lambda = step/meanNumber;
      else                 lambda = DBL_MAX;
      while(numb)
      {
        stepDelta = G4RandExponential::shoot(lambda);
        stepSum += stepDelta;
        if(stepSum >= step) break;
        numOfCollisions++;
      }   

      //  G4cout<<"numOfCollisions = "<<numOfCollisions<<endl;

      while(numOfCollisions)
      {
        position = dNdxCut1*W1 + dNdxCut2*W2 +
                   ( ( (*(*pTable)(iPlace  ))(0) - dNdxCut1)*W1 + 
                    
                     ( (*(*pTable)(iPlace+1))(0) - dNdxCut2)*W2 )*G4UniformRand();

        // G4cout<<position<<"\t";

        for( iTransfer = 0;
    iTransfer < G4int((*pTable)(iPlace)->GetVectorLength()); iTransfer++ )
        {
          if( position >=
          ( (*(*pTable)(iPlace))(iTransfer)*W1 + 
            (*(*pTable)(iPlace+1))(iTransfer)*W2) )
          {
	      break;
	  }
        }
	// loss += (*pTable)(iPlace)->GetLowEdgeEnergy(iTransfer); 
        loss += GetEnergyTransfer(pTable,iPlace,position,iTransfer);
        numOfCollisions--;    
      }
    }
  } 

  return loss;

}

//////////////////////////////////////////////////////////////////////
//
// Returns the statistical estimation of the energy loss distribution variance
//


G4double G4PAIPhotonModel::Dispersion( const G4Material* material, 
				       const G4DynamicParticle* aParticle,
 				       G4double tmax, 
			               G4double step       )
{
  G4double particleMass  = aParticle->GetMass();
  G4double electronDensity = material->GetElectronDensity();
  G4double kineticEnergy = aParticle->GetKineticEnergy();
  G4double q = aParticle->GetCharge()/eplus;
  G4double etot = kineticEnergy + particleMass;
  G4double beta2 = kineticEnergy*(kineticEnergy + 2.0*particleMass)/(etot*etot);
  G4double siga  = (1.0/beta2 - 0.5) * twopi_mc2_rcl2 * tmax * step
                 * electronDensity * q * q;

  return siga;

  /*
  G4double loss, sumLoss=0., sumLoss2=0., sigma2, meanLoss=0.;
  for(G4int i = 0; i < fMeanNumber; i++)
  {
    loss      = SampleFluctuations(material,aParticle,tmax,step,meanLoss);
    sumLoss  += loss;
    sumLoss2 += loss*loss;
  }
  meanLoss = sumLoss/fMeanNumber;
  sigma2   = meanLoss*meanLoss + (sumLoss2-2*sumLoss*meanLoss)/fMeanNumber;
  return sigma2;
  */
}

/////////////////////////////////////////////////////////////////////

G4double G4PAIPhotonModel::MaxSecondaryEnergy( const G4ParticleDefinition* p,
                                                      G4double kinEnergy) 
{
  G4double tmax = kinEnergy;
  if(p == fElectron) tmax *= 0.5;
  else if(p != fPositron) 
 { 
    G4double mass = p->GetPDGMass();
    G4double ratio= electron_mass_c2/mass;
    G4double gamma= kinEnergy/mass + 1.0;
    tmax = 2.0*electron_mass_c2*(gamma*gamma - 1.) /
                  (1. + 2.0*gamma*ratio + ratio*ratio);
  }
  return tmax;
}

///////////////////////////////////////////////////////////////

void G4PAIPhotonModel::DefineForRegion(const G4Region* r) 
{
  fPAIRegionVector.push_back(r);
}


//
//
/////////////////////////////////////////////////






