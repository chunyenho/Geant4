#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4BraggModel.cc 83008 2014-07-24 14:49:52Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:   G4BraggModel
//
// Author:        Vladimir Ivanchenko
//
// Creation date: 03.01.2002
//
// Modifications: 
//
// 04-12-02 Fix problem of G4DynamicParticle constructor (V.Ivanchenko)
// 23-12-02 Change interface in order to move to cut per region (V.Ivanchenko)
// 27-01-03 Make models region aware (V.Ivanchenko)
// 13-02-03 Add name (V.Ivanchenko)
// 04-06-03 Fix compilation warnings (V.Ivanchenko)
// 12-09-04 Add lowestKinEnergy and change order of if in DEDX method (VI)
// 11-04-05 Major optimisation of internal interfaces (V.Ivantchenko)
// 16-06-05 Fix problem of chemical formula (V.Ivantchenko)
// 15-02-06 ComputeCrossSectionPerElectron, ComputeCrossSectionPerAtom (mma)
// 25-04-06 Add stopping data from PSTAR (V.Ivanchenko)
// 12-08-08 Added methods GetParticleCharge, GetChargeSquareRatio, 
//          CorrectionsAlongStep needed for ions(V.Ivanchenko)

// Class Description:
//
// Implementation of energy loss and delta-electron production by
// slow charged heavy particles

// -------------------------------------------------------------------
//


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "G4BraggModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"
#include "G4Electron.hh"
#include "G4ParticleChangeForLoss.hh"
#include "G4LossTableManager.hh"
#include "G4EmCorrections.hh"
#include "G4DeltaAngle.hh"
#include "G4Log.hh"
#include "G4Exp.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

using namespace std;

static const G4double invLog10 = 1.0/G4Log(10.);
G4PSTARStopping* G4BraggModel::fPSTAR = 0;

G4BraggModel::G4BraggModel(const G4ParticleDefinition* p, const G4String& nam)
  : G4VEmModel(nam),
    particle(0),
    currentMaterial(0),
    protonMassAMU(1.007276),
    iMolecula(-1),
    iPSTAR(-1),
    isIon(false),
    isInitialised(false)
{
  fParticleChange = 0;
  SetHighEnergyLimit(2.0*MeV);

  lowestKinEnergy  = 1.0*keV;
  theZieglerFactor = eV*cm2*1.0e-15;
  theElectron = G4Electron::Electron();
  expStopPower125 = 0.0;

  corr = G4LossTableManager::Instance()->EmCorrections();
  if(p) { SetParticle(p); }
  else  { SetParticle(theElectron); }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4BraggModel::~G4BraggModel()
{
  if(IsMaster()) { delete fPSTAR; fPSTAR = 0; }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4BraggModel::Initialise(const G4ParticleDefinition* p,
                              const G4DataVector&)
{
  if(p != particle) { SetParticle(p); }

  // always false before the run
  SetDeexcitationFlag(false);

  if(!isInitialised) {
    isInitialised = true;

    if(UseAngularGeneratorFlag() && !GetAngularDistribution()) {
      SetAngularDistribution(new G4DeltaAngle());
    }
    G4String pname = particle->GetParticleName();
    if(particle->GetParticleType() == "nucleus" && 
       pname != "deuteron" && pname != "triton" &&
       pname != "alpha+"   && pname != "helium" &&
       pname != "hydrogen") { isIon = true; }

    fParticleChange = GetParticleChangeForLoss();
    if(!fPSTAR) { fPSTAR = new G4PSTARStopping(); }
  }
  if(IsMaster() && particle->GetPDGMass() < GeV) { fPSTAR->Initialise(); }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::GetChargeSquareRatio(const G4ParticleDefinition* p,
					    const G4Material* mat,
					    G4double kineticEnergy)
{
  // this method is called only for ions
  G4double q2 = corr->EffectiveChargeSquareRatio(p,mat,kineticEnergy);
  GetModelOfFluctuations()->SetParticleAndCharge(p, q2);
  return q2*corr->EffectiveChargeCorrection(p,mat,kineticEnergy);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::GetParticleCharge(const G4ParticleDefinition* p,
					 const G4Material* mat,
					 G4double kineticEnergy)
{
  // this method is called only for ions, so no check if it is an ion 
  return corr->GetParticleCharge(p,mat,kineticEnergy);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::ComputeCrossSectionPerElectron(
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
                                                 G4double cutEnergy,
                                                 G4double maxKinEnergy)
{
  G4double cross     = 0.0;
  G4double tmax      = MaxSecondaryEnergy(p, kineticEnergy);
  G4double maxEnergy = std::min(tmax,maxKinEnergy);
  if(cutEnergy < maxEnergy) {

    G4double energy  = kineticEnergy + mass;
    G4double energy2 = energy*energy;
    G4double beta2   = kineticEnergy*(kineticEnergy + 2.0*mass)/energy2;
    cross = (maxEnergy - cutEnergy)/(cutEnergy*maxEnergy) 
      - beta2*G4Log(maxEnergy/cutEnergy)/tmax;

    if( 0.5 == spin ) { cross += 0.5*(maxEnergy - cutEnergy)/energy2; }

    cross *= twopi_mc2_rcl2*chargeSquare/beta2;
  }
 //   G4cout << "BR: e= " << kineticEnergy << " tmin= " << cutEnergy 
 //          << " tmax= " << tmax << " cross= " << cross << G4endl;
 
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::ComputeCrossSectionPerAtom(
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
						 G4double Z, G4double,
                                                 G4double cutEnergy,
                                                 G4double maxEnergy)
{
  G4double cross = Z*ComputeCrossSectionPerElectron
                                         (p,kineticEnergy,cutEnergy,maxEnergy);
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::CrossSectionPerVolume(
					   const G4Material* material,
                                           const G4ParticleDefinition* p,
                                                 G4double kineticEnergy,
                                                 G4double cutEnergy,
                                                 G4double maxEnergy)
{
  G4double eDensity = material->GetElectronDensity();
  G4double cross = eDensity*ComputeCrossSectionPerElectron
                                         (p,kineticEnergy,cutEnergy,maxEnergy);
  return cross;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::ComputeDEDXPerVolume(const G4Material* material,
					    const G4ParticleDefinition* p,
					    G4double kineticEnergy,
					    G4double cutEnergy)
{
  G4double tmax  = MaxSecondaryEnergy(p, kineticEnergy);
  G4double tkin  = kineticEnergy/massRate;
  G4double dedx  = 0.0;

  if(tkin < lowestKinEnergy) {
    dedx = DEDX(material, lowestKinEnergy)*sqrt(tkin/lowestKinEnergy);
  } else {
    dedx = DEDX(material, tkin); 
  }

  if (cutEnergy < tmax) {

    G4double tau   = kineticEnergy/mass;
    G4double gam   = tau + 1.0;
    G4double bg2   = tau * (tau+2.0);
    G4double beta2 = bg2/(gam*gam);
    G4double x     = cutEnergy/tmax;

    dedx += (G4Log(x) + (1.0 - x)*beta2) * twopi_mc2_rcl2
          * (material->GetElectronDensity())/beta2;
  }

  // now compute the total ionization loss

  if (dedx < 0.0) { dedx = 0.0; }

  dedx *= chargeSquare;

  //G4cout << "E(MeV)= " << tkin/MeV << " dedx= " << dedx 
  //	 << "  " << material->GetName() << G4endl;

  return dedx;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4BraggModel::SampleSecondaries(vector<G4DynamicParticle*>* vdp,
				     const G4MaterialCutsCouple* couple,
				     const G4DynamicParticle* dp,
				     G4double xmin,
				     G4double maxEnergy)
{
  G4double tmax = MaxSecondaryKinEnergy(dp);
  G4double xmax = std::min(tmax, maxEnergy);
  if(xmin >= xmax) { return; }

  G4double kineticEnergy = dp->GetKineticEnergy();
  G4double energy  = kineticEnergy + mass;
  G4double energy2 = energy*energy;
  G4double beta2   = kineticEnergy*(kineticEnergy + 2.0*mass)/energy2;
  G4double grej    = 1.0;
  G4double deltaKinEnergy, f;

  // sampling follows ...
  do {
    G4double q = G4UniformRand();
    deltaKinEnergy = xmin*xmax/(xmin*(1.0 - q) + xmax*q);

    f = 1.0 - beta2*deltaKinEnergy/tmax;

    if(f > grej) {
        G4cout << "G4BraggModel::SampleSecondary Warning! "
               << "Majorant " << grej << " < "
               << f << " for e= " << deltaKinEnergy
               << G4endl;
    }

  } while( grej*G4UniformRand() >= f );

  G4ThreeVector deltaDirection;

  if(UseAngularGeneratorFlag()) {
    const G4Material* mat =  couple->GetMaterial();
    G4int Z = SelectRandomAtomNumber(mat);

    deltaDirection = 
      GetAngularDistribution()->SampleDirection(dp, deltaKinEnergy, Z, mat);

  } else {
 
    G4double deltaMomentum =
      sqrt(deltaKinEnergy * (deltaKinEnergy + 2.0*electron_mass_c2));
    G4double cost = deltaKinEnergy * (energy + electron_mass_c2) /
      (deltaMomentum * dp->GetTotalMomentum());
    if(cost > 1.0) { cost = 1.0; }
    G4double sint = sqrt((1.0 - cost)*(1.0 + cost));

    G4double phi = twopi * G4UniformRand() ;

    deltaDirection.set(sint*cos(phi),sint*sin(phi), cost) ;
    deltaDirection.rotateUz(dp->GetMomentumDirection());
  }  

  // create G4DynamicParticle object for delta ray
  G4DynamicParticle* delta = 
    new G4DynamicParticle(theElectron,deltaDirection,deltaKinEnergy);

  // Change kinematics of primary particle
  kineticEnergy -= deltaKinEnergy;
  G4ThreeVector finalP = dp->GetMomentum() - delta->GetMomentum();
  finalP               = finalP.unit();
  
  fParticleChange->SetProposedKineticEnergy(kineticEnergy);
  fParticleChange->SetProposedMomentumDirection(finalP);

  vdp->push_back(delta);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::MaxSecondaryEnergy(const G4ParticleDefinition* pd,
					  G4double kinEnergy)
{
  if(pd != particle) { SetParticle(pd); }
  G4double tau  = kinEnergy/mass;
  G4double tmax = 2.0*electron_mass_c2*tau*(tau + 2.) /
                  (1. + 2.0*(tau + 1.)*ratio + ratio*ratio);
  return tmax;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4bool G4BraggModel::HasMaterial(const G4Material*)
{
  return false;
  /*
  G4String chFormula = material->GetChemicalFormula();
  if("" == chFormula) { return false; }

  // ICRU Report N49, 1993. Power's model for H
  static const size_t numberOfMolecula = 11;
  static const G4String molName[numberOfMolecula] = {
    "Al_2O_3",                 "CO_2",                      "CH_4",
    "(C_2H_4)_N-Polyethylene", "(C_2H_4)_N-Polypropylene",  "(C_8H_8)_N",
    "C_3H_8",                  "SiO_2",                     "H_2O",
    "H_2O-Gas",                "Graphite" } ;

  // Search for the material in the table
  for (size_t i=0; i<numberOfMolecula; ++i) {
    if (chFormula == molName[i]) {
      iPSTAR = fPSTAR->GetIndex(matName[i]);  
      break;
    }
  }
  return (iPSTAR >= 0);
  */
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::StoppingPower(const G4Material* material,
                                           G4double kineticEnergy) 
{
  G4double ionloss = 0.0 ;

  if (iMolecula >= 0) {
  
    // The data and the fit from: 
    // ICRU Report N49, 1993. Ziegler's model for protons.
    // Proton kinetic energy for parametrisation (keV/amu)  

    G4double T = kineticEnergy/(keV*protonMassAMU) ; 

    static const G4double a[11][5] = {
   {1.187E+1, 1.343E+1, 1.069E+4, 7.723E+2, 2.153E-2},
   {7.802E+0, 8.814E+0, 8.303E+3, 7.446E+2, 7.966E-3}, 
   {7.294E+0, 8.284E+0, 5.010E+3, 4.544E+2, 8.153E-3}, 
   {8.646E+0, 9.800E+0, 7.066E+3, 4.581E+2, 9.383E-3}, 
   {1.286E+1, 1.462E+1, 5.625E+3, 2.621E+3, 3.512E-2}, 
   {3.229E+1, 3.696E+1, 8.918E+3, 3.244E+3, 1.273E-1}, 
   {1.604E+1, 1.825E+1, 6.967E+3, 2.307E+3, 3.775E-2}, 
   {8.049E+0, 9.099E+0, 9.257E+3, 3.846E+2, 1.007E-2},
   {4.015E+0, 4.542E+0, 3.955E+3, 4.847E+2, 7.904E-3}, 
   {4.571E+0, 5.173E+0, 4.346E+3, 4.779E+2, 8.572E-3},
   {2.631E+0, 2.601E+0, 1.701E+3, 1.279E+3, 1.638E-2} };

    static const G4double atomicWeight[11] = {
    101.96128, 44.0098, 16.0426, 28.0536, 42.0804,
    104.1512, 44.665, 60.0843, 18.0152, 18.0152, 12.0};       

    if ( T < 10.0 ) {
      ionloss = a[iMolecula][0] * sqrt(T) ;
    
    } else if ( T < 10000.0 ) {
      G4double slow  = a[iMolecula][1] * G4Exp(G4Log(T)* 0.45);
      G4double shigh = G4Log( 1.0 + a[iMolecula][3]/T  
                     + a[iMolecula][4]*T ) * a[iMolecula][2]/T ;
      ionloss = slow*shigh / (slow + shigh) ;     
    } 

    if ( ionloss < 0.0) ionloss = 0.0 ;
    if ( 10 == iMolecula ) { 
      if (T < 100.0) {
	ionloss *= (1.0+0.023+0.0066*G4Log(T)*invLog10);  
      }
      else if (T < 700.0) {   
	ionloss *=(1.0+0.089-0.0248*G4Log(T-99.)*invLog10);
      } 
      else if (T < 10000.0) {    
	ionloss *=(1.0+0.089-0.0248*G4Log(700.-99.)*invLog10);
      }
    }
    ionloss /= atomicWeight[iMolecula];

  // pure material (normally not the case for this function)
  } else if(1 == (material->GetNumberOfElements())) {
    G4double z = material->GetZ() ;
    ionloss = ElectronicStoppingPower( z, kineticEnergy ) ;  
  }
  
  return ionloss;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::ElectronicStoppingPower(G4double z,
                                               G4double kineticEnergy) const
{
  G4double ionloss ;
  G4int i = G4int(z)-1 ;  // index of atom
  if(i < 0)  i = 0 ;
  if(i > 91) i = 91 ;
  
  // The data and the fit from: 
  // ICRU Report 49, 1993. Ziegler's type of parametrisations.
  // Proton kinetic energy for parametrisation (keV/amu)  

  G4double T = kineticEnergy/(keV*protonMassAMU) ; 
  
  static const G4double a[92][5] = {
   {1.254E+0, 1.440E+0, 2.426E+2, 1.200E+4, 1.159E-1},
   {1.229E+0, 1.397E+0, 4.845E+2, 5.873E+3, 5.225E-2},
   {1.411E+0, 1.600E+0, 7.256E+2, 3.013E+3, 4.578E-2},
   {2.248E+0, 2.590E+0, 9.660E+2, 1.538E+2, 3.475E-2},
   {2.474E+0, 2.815E+0, 1.206E+3, 1.060E+3, 2.855E-2},
   {2.631E+0, 2.601E+0, 1.701E+3, 1.279E+3, 1.638E-2},
   {2.954E+0, 3.350E+0, 1.683E+3, 1.900E+3, 2.513E-2},
   {2.652E+0, 3.000E+0, 1.920E+3, 2.000E+3, 2.230E-2},
   {2.085E+0, 2.352E+0, 2.157E+3, 2.634E+3, 1.816E-2},
   {1.951E+0, 2.199E+0, 2.393E+3, 2.699E+3, 1.568E-2},
       // Z= 11-20
   {2.542E+0, 2.869E+0, 2.628E+3, 1.854E+3, 1.472E-2},
   {3.791E+0, 4.293E+0, 2.862E+3, 1.009E+3, 1.397E-2},
   {4.154E+0, 4.739E+0, 2.766E+3, 1.645E+2, 2.023E-2},
   {4.914E+0, 5.598E+0, 3.193E+3, 2.327E+2, 1.419E-2},
   {3.232E+0, 3.647E+0, 3.561E+3, 1.560E+3, 1.267E-2},
   {3.447E+0, 3.891E+0, 3.792E+3, 1.219E+3, 1.211E-2},
   {5.301E+0, 6.008E+0, 3.969E+3, 6.451E+2, 1.183E-2},
   {5.731E+0, 6.500E+0, 4.253E+3, 5.300E+2, 1.123E-2},
   {5.152E+0, 5.833E+0, 4.482E+3, 5.457E+2, 1.129E-2},
   {5.521E+0, 6.252E+0, 4.710E+3, 5.533E+2, 1.112E-2},
       // Z= 21-30
   {5.201E+0, 5.884E+0, 4.938E+3, 5.609E+2, 9.995E-3},
   {4.858E+0, 5.489E+0, 5.260E+3, 6.511E+2, 8.930E-3},
   {4.479E+0, 5.055E+0, 5.391E+3, 9.523E+2, 9.117E-3},
   {3.983E+0, 4.489E+0, 5.616E+3, 1.336E+3, 8.413E-3},
   {3.469E+0, 3.907E+0, 5.725E+3, 1.461E+3, 8.829E-3},
   {3.519E+0, 3.963E+0, 6.065E+3, 1.243E+3, 7.782E-3},
   {3.140E+0, 3.535E+0, 6.288E+3, 1.372E+3, 7.361E-3},
   {3.553E+0, 4.004E+0, 6.205E+3, 5.551E+2, 8.763E-3},
   {3.696E+0, 4.194E+0, 4.649E+3, 8.113E+1, 2.242E-2},
   {4.210E+0, 4.750E+0, 6.953E+3, 2.952E+2, 6.809E-3},
       // Z= 31-40
   {5.041E+0, 5.697E+0, 7.173E+3, 2.026E+2, 6.725E-3},
   {5.554E+0, 6.300E+0, 6.496E+3, 1.100E+2, 9.689E-3},
   {5.323E+0, 6.012E+0, 7.611E+3, 2.925E+2, 6.447E-3},
   {5.874E+0, 6.656E+0, 7.395E+3, 1.175E+2, 7.684E-3},
   {6.658E+0, 7.536E+0, 7.694E+3, 2.223E+2, 6.509E-3},
   {6.413E+0, 7.240E+0, 1.185E+4, 1.537E+2, 2.880E-3},
   {5.694E+0, 6.429E+0, 8.478E+3, 2.929E+2, 6.087E-3},
   {6.339E+0, 7.159E+0, 8.693E+3, 3.303E+2, 6.003E-3},
   {6.407E+0, 7.234E+0, 8.907E+3, 3.678E+2, 5.889E-3},
   {6.734E+0, 7.603E+0, 9.120E+3, 4.052E+2, 5.765E-3},
       // Z= 41-50
   {6.901E+0, 7.791E+0, 9.333E+3, 4.427E+2, 5.587E-3},
   {6.424E+0, 7.248E+0, 9.545E+3, 4.802E+2, 5.376E-3},
   {6.799E+0, 7.671E+0, 9.756E+3, 5.176E+2, 5.315E-3},
   {6.109E+0, 6.887E+0, 9.966E+3, 5.551E+2, 5.151E-3},
   {5.924E+0, 6.677E+0, 1.018E+4, 5.925E+2, 4.919E-3},
   {5.238E+0, 5.900E+0, 1.038E+4, 6.300E+2, 4.758E-3},
   // {5.623,    6.354,    7160.0,   337.6,    0.013940}, // Ag Ziegler77
   {5.345E+0, 6.038E+0, 6.790E+3, 3.978E+2, 1.676E-2}, // Ag ICRU49
   {5.814E+0, 6.554E+0, 1.080E+4, 3.555E+2, 4.626E-3},
   {6.229E+0, 7.024E+0, 1.101E+4, 3.709E+2, 4.540E-3},
   {6.409E+0, 7.227E+0, 1.121E+4, 3.864E+2, 4.474E-3},
       // Z= 51-60
   {7.500E+0, 8.480E+0, 8.608E+3, 3.480E+2, 9.074E-3},
   {6.979E+0, 7.871E+0, 1.162E+4, 3.924E+2, 4.402E-3},
   {7.725E+0, 8.716E+0, 1.183E+4, 3.948E+2, 4.376E-3},
   {8.337E+0, 9.425E+0, 1.051E+4, 2.696E+2, 6.206E-3},
   {7.287E+0, 8.218E+0, 1.223E+4, 3.997E+2, 4.447E-3},
   {7.899E+0, 8.911E+0, 1.243E+4, 4.021E+2, 4.511E-3},
   {8.041E+0, 9.071E+0, 1.263E+4, 4.045E+2, 4.540E-3},
   {7.488E+0, 8.444E+0, 1.283E+4, 4.069E+2, 4.420E-3},
   {7.291E+0, 8.219E+0, 1.303E+4, 4.093E+2, 4.298E-3},
   {7.098E+0, 8.000E+0, 1.323E+4, 4.118E+2, 4.182E-3},
       // Z= 61-70
   {6.909E+0, 7.786E+0, 1.343E+4, 4.142E+2, 4.058E-3},
   {6.728E+0, 7.580E+0, 1.362E+4, 4.166E+2, 3.976E-3},
   {6.551E+0, 7.380E+0, 1.382E+4, 4.190E+2, 3.877E-3},
   {6.739E+0, 7.592E+0, 1.402E+4, 4.214E+2, 3.863E-3},
   {6.212E+0, 6.996E+0, 1.421E+4, 4.239E+2, 3.725E-3},
   {5.517E+0, 6.210E+0, 1.440E+4, 4.263E+2, 3.632E-3},
   {5.220E+0, 5.874E+0, 1.460E+4, 4.287E+2, 3.498E-3},
   {5.071E+0, 5.706E+0, 1.479E+4, 4.330E+2, 3.405E-3},
   {4.926E+0, 5.542E+0, 1.498E+4, 4.335E+2, 3.342E-3},
   {4.788E+0, 5.386E+0, 1.517E+4, 4.359E+2, 3.292E-3},
       // Z= 71-80
   {4.893E+0, 5.505E+0, 1.536E+4, 4.384E+2, 3.243E-3},
   {5.028E+0, 5.657E+0, 1.555E+4, 4.408E+2, 3.195E-3},
   {4.738E+0, 5.329E+0, 1.574E+4, 4.432E+2, 3.186E-3},
   {4.587E+0, 5.160E+0, 1.541E+4, 4.153E+2, 3.406E-3},
   {5.201E+0, 5.851E+0, 1.612E+4, 4.416E+2, 3.122E-3},
   {5.071E+0, 5.704E+0, 1.630E+4, 4.409E+2, 3.082E-3},
   {4.946E+0, 5.563E+0, 1.649E+4, 4.401E+2, 2.965E-3},
   {4.477E+0, 5.034E+0, 1.667E+4, 4.393E+2, 2.871E-3},
   //  {4.856,    5.460,    18320.0,  438.5,    0.002542}, //Ziegler77
   {4.844E+0, 5.458E+0, 7.852E+3, 9.758E+2, 2.077E-2}, //ICRU49
   {4.307E+0, 4.843E+0, 1.704E+4, 4.878E+2, 2.882E-3},
       // Z= 81-90
   {4.723E+0, 5.311E+0, 1.722E+4, 5.370E+2, 2.913E-3},
   {5.319E+0, 5.982E+0, 1.740E+4, 5.863E+2, 2.871E-3},
   {5.956E+0, 6.700E+0, 1.780E+4, 6.770E+2, 2.660E-3},
   {6.158E+0, 6.928E+0, 1.777E+4, 5.863E+2, 2.812E-3},
   {6.203E+0, 6.979E+0, 1.795E+4, 5.863E+2, 2.776E-3},
   {6.181E+0, 6.954E+0, 1.812E+4, 5.863E+2, 2.748E-3},
   {6.949E+0, 7.820E+0, 1.830E+4, 5.863E+2, 2.737E-3},
   {7.506E+0, 8.448E+0, 1.848E+4, 5.863E+2, 2.727E-3},
   {7.648E+0, 8.609E+0, 1.866E+4, 5.863E+2, 2.697E-3},
   {7.711E+0, 8.679E+0, 1.883E+4, 5.863E+2, 2.641E-3},
       // Z= 91-92
   {7.407E+0, 8.336E+0, 1.901E+4, 5.863E+2, 2.603E-3},
   {7.290E+0, 8.204E+0, 1.918E+4, 5.863E+2, 2.673E-3}
  };

  G4double fac = 1.0 ;

    // Carbon specific case for E < 40 keV
  if ( T < 40.0 && 5 == i) {
    fac = sqrt(T/40.0) ;
    T = 40.0 ;  

    // Free electron gas model
  } else if ( T < 10.0 ) { 
    fac = sqrt(T*0.1) ;
    T =10.0 ;
  }

  // Main parametrisation
  G4double slow  = a[i][1] * G4Exp(G4Log(T) * 0.45) ;
  G4double shigh = G4Log( 1.0 + a[i][3]/T + a[i][4]*T ) * a[i][2]/T ;
  ionloss = slow*shigh*fac / (slow + shigh) ;     
  
  if ( ionloss < 0.0) { ionloss = 0.0; }
  
  return ionloss;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::DEDX(const G4Material* material, G4double kineticEnergy) 
{
  G4double eloss = 0.0;

  // check DB
  if(material != currentMaterial) {
    currentMaterial = material;
    iPSTAR    = -1;
    iMolecula = -1;
    if( !HasMaterial(material) ) { iPSTAR = fPSTAR->GetIndex(material); }

    //G4cout << "%%% " <<material->GetName() << "  iMolecula= " 
    //	   << iMolecula << "  iPSTAR= " << iPSTAR << G4endl; 

  }

  const G4int numberOfElements = material->GetNumberOfElements();
  const G4double* theAtomicNumDensityVector =
                                 material->GetAtomicNumDensityVector();
  
  if( iPSTAR >= 0 ) {
    return 
      fPSTAR->GetElectronicDEDX(iPSTAR, kineticEnergy)*material->GetDensity();

  } else if(iMolecula >= 0) {

    eloss = StoppingPower(material, kineticEnergy)*
                          material->GetDensity()/amu;

  // Pure material ICRU49 paralmeterisation
  } else if(1 == numberOfElements) {

    G4double z = material->GetZ();
    eloss = ElectronicStoppingPower(z, kineticEnergy)
                               * (material->GetTotNbOfAtomsPerVolume());


  // Experimental data exist only for kinetic energy 125 keV
  } else if( MolecIsInZiegler1988(material) ) { 

    // Loop over elements - calculation based on Bragg's rule 
    G4double eloss125 = 0.0 ;
    const G4ElementVector* theElementVector =
                           material->GetElementVector();
  
    //  Loop for the elements in the material
    for (G4int i=0; i<numberOfElements; i++) {
      const G4Element* element = (*theElementVector)[i] ;
      G4double z = element->GetZ() ;
      eloss    += ElectronicStoppingPower(z,kineticEnergy)
                                    * theAtomicNumDensityVector[i] ;
      eloss125 += ElectronicStoppingPower(z,125.0*keV)
                                    * theAtomicNumDensityVector[i] ;
    }      

    // Chemical factor is taken into account
    eloss *= ChemicalFactor(kineticEnergy, eloss125) ;
 
  // Brugg's rule calculation
  } else {
    const G4ElementVector* theElementVector =
                           material->GetElementVector() ;
  
    //  loop for the elements in the material
    for (G4int i=0; i<numberOfElements; i++)
    {
      const G4Element* element = (*theElementVector)[i] ;
      eloss   += ElectronicStoppingPower(element->GetZ(), kineticEnergy)
                                   * theAtomicNumDensityVector[i];
    }      
  }
  return eloss*theZieglerFactor;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4bool G4BraggModel::MolecIsInZiegler1988(const G4Material* material) 
{
  // The list of molecules from
  // J.F.Ziegler and J.M.Manoyan, The stopping of ions in compaunds,
  // Nucl. Inst. & Meth. in Phys. Res. B35 (1988) 215-228.
  
  G4String myFormula = G4String(" ") ;
  const G4String chFormula = material->GetChemicalFormula() ;
  if (myFormula == chFormula ) { return false; }
  
  //  There are no evidence for difference of stopping power depended on
  //  phase of the compound except for water. The stopping power of the 
  //  water in gas phase can be predicted using Bragg's rule.
  //  
  //  No chemical factor for water-gas 
   
  myFormula = G4String("H_2O") ;
  const G4State theState = material->GetState() ;
  if( theState == kStateGas && myFormula == chFormula) return false ;
    

  // The coffecient from Table.4 of Ziegler & Manoyan
  static const G4double HeEff = 2.8735 ;
  
  static const size_t numberOfMolecula = 53;
  static const G4String nameOfMol[53] = {
    "H_2O",      "C_2H_4O",    "C_3H_6O",  "C_2H_2",             "C_H_3OH",
    "C_2H_5OH",  "C_3H_7OH",   "C_3H_4",   "NH_3",               "C_14H_10",
    "C_6H_6",    "C_4H_10",    "C_4H_6",   "C_4H_8O",            "CCl_4",
    "CF_4",      "C_6H_8",     "C_6H_12",  "C_6H_10O",           "C_6H_10",
    "C_8H_16",   "C_5H_10",    "C_5H_8",   "C_3H_6-Cyclopropane","C_2H_4F_2",
    "C_2H_2F_2", "C_4H_8O_2",  "C_2H_6",   "C_2F_6",             "C_2H_6O",
    "C_3H_6O",   "C_4H_10O",   "C_2H_4",   "C_2H_4O",            "C_2H_4S",
    "SH_2",      "CH_4",       "CCLF_3",   "CCl_2F_2",           "CHCl_2F",
    "(CH_3)_2S", "N_2O",       "C_5H_10O", "C_8H_6",             "(CH_2)_N",
    "(C_3H_6)_N","(C_8H_8)_N", "C_3H_8",   "C_3H_6-Propylene",   "C_3H_6O",
    "C_3H_6S",   "C_4H_4S",    "C_7H_8"
  };

  static const G4double expStopping[numberOfMolecula] = {
     66.1,  190.4, 258.7,  42.2, 141.5,
    210.9,  279.6, 198.8,  31.0, 267.5,
    122.8,  311.4, 260.3, 328.9, 391.3,
    206.6,  374.0, 422.0, 432.0, 398.0,
    554.0,  353.0, 326.0,  74.6, 220.5,
    197.4,  362.0, 170.0, 330.5, 211.3,
    262.3,  349.6,  51.3, 187.0, 236.9,
    121.9,   35.8, 247.0, 292.6, 268.0,
    262.3,   49.0, 398.9, 444.0,  22.91,
     68.0,  155.0,  84.0,  74.2, 254.7,
    306.8,  324.4, 420.0
  } ;

  static const G4double expCharge[53] = {
    HeEff, HeEff, HeEff,   1.0, HeEff,
    HeEff, HeEff, HeEff,   1.0,   1.0,
      1.0, HeEff, HeEff, HeEff, HeEff,
    HeEff, HeEff, HeEff, HeEff, HeEff,
    HeEff, HeEff, HeEff,   1.0, HeEff,
    HeEff, HeEff, HeEff, HeEff, HeEff,
    HeEff, HeEff,   1.0, HeEff, HeEff,
    HeEff,   1.0, HeEff, HeEff, HeEff,
    HeEff,   1.0, HeEff, HeEff,   1.0,
      1.0,   1.0,   1.0,   1.0, HeEff,
    HeEff, HeEff, HeEff
  } ;

  static const G4double numberOfAtomsPerMolecula[53] = {
    3.0,  7.0, 10.0,  4.0,  6.0,
    9.0, 12.0,  7.0,  4.0, 24.0,
    12.0, 14.0, 10.0, 13.0,  5.0,
    5.0, 14.0, 18.0, 17.0, 17.0,
    24.0, 15.0, 13.0,  9.0,  8.0,
    6.0, 14.0,  8.0,  8.0,  9.0,
    10.0, 15.0,  6.0,  7.0,  7.0,
    3.0,  5.0,  5.0,  5.0,  5.0,
    9.0,  3.0, 16.0, 14.0,  3.0,
    9.0, 16.0, 11.0,  9.0, 10.0,
    10.0,  9.0, 15.0
  } ;

  // Search for the compaund in the table
  for (size_t i=0; i<numberOfMolecula; i++)
    {
      if(chFormula == nameOfMol[i]) {
        G4double exp125 = expStopping[i] *
	                  (material->GetTotNbOfAtomsPerVolume()) /
	                  (expCharge[i] * numberOfAtomsPerMolecula[i]) ;
        SetExpStopPower125(exp125);
        return true;
      }
    }
  
  return false;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4double G4BraggModel::ChemicalFactor(G4double kineticEnergy, 
                                      G4double eloss125) const
{
  // Approximation of Chemical Factor according to
  // J.F.Ziegler and J.M.Manoyan, The stopping of ions in compaunds,
  // Nucl. Inst. & Meth. in Phys. Res. B35 (1988) 215-228.
  
  G4double gamma    = 1.0 + kineticEnergy/proton_mass_c2 ;    
  G4double gamma25  = 1.0 + 25.0*keV /proton_mass_c2 ;
  G4double gamma125 = 1.0 + 125.0*keV/proton_mass_c2 ;
  G4double beta     = sqrt(1.0 - 1.0/(gamma*gamma)) ;
  G4double beta25   = sqrt(1.0 - 1.0/(gamma25*gamma25)) ;
  G4double beta125  = sqrt(1.0 - 1.0/(gamma125*gamma125)) ;
  
  G4double factor = 1.0 + (expStopPower125/eloss125 - 1.0) *
                   (1.0 + G4Exp( 1.48 * ( beta125/beta25 - 7.0 ) ) ) /
                   (1.0 + G4Exp( 1.48 * ( beta/beta25    - 7.0 ) ) ) ;

  return factor ;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

