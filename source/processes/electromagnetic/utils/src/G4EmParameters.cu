#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4EmParameters.cc 69320 2013-04-30 15:59:36Z vnivanch $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4EmParameters
//
// Author:        Vladimir Ivanchenko
//
// Creation date: 18.05.2013
//
// Modifications:
//
//
//
// -------------------------------------------------------------------
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4EmParameters.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4EmParametersMessenger.hh"
#include "G4NistManager.hh"

G4EmParameters* G4EmParameters::theInstance = 0;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo.....

G4EmParameters* G4EmParameters::Instance()
{
  if(0 == theInstance) {
    static G4EmParameters manager;
    theInstance = &manager;
  }
  return theInstance;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo.....

G4EmParameters::~G4EmParameters()
{
  delete theMessenger;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo.....

G4EmParameters::G4EmParameters()
{
  G4NistManager::Instance();
  theMessenger = new G4EmParametersMessenger(this);

  lossFluctuation = true;
  buildCSDARange = false;
  flagLPM = true;
  spline = true;
  finalRange = false;
  applyCuts = false;
  fluo = false;
  auger = false;
  pixe = false;
  deexIgnoreCut = false;
  lateralDisplacement = true;
  muhadLateralDisplacement = true;
  latDisplacementBeyondSafety = false;
  useAngGeneratorForIonisation = false;

  minSubRange = DBL_MAX;
  minKinEnergy = 0.1*keV;
  maxKinEnergy = 10.0*TeV;
  maxKinEnergyCSDA = 1.0*GeV;
  linLossLimit = 0.01;
  bremsTh = DBL_MAX;
  lambdaFactor = 0.8;
  factorForAngleLimit = 1.0;
  thetaLimit = CLHEP::pi;
  rangeFactor = 0.04;
  geomFactor = 2.5;
  skin = 1.0;

  nbins  = 77;
  nbinsPerDecade = 7;
  verbose = 1;
  workerVerbose = 0;

  mscStepLimit = fUseSafety;
}

#include "G4AutoLock.hh"
namespace { G4Mutex EmParametersMutex = G4MUTEX_INITIALIZER; }

void G4EmParameters::SetLossFluctuations(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  lossFluctuation = val;
}

G4bool G4EmParameters::LossFluctuation() const
{
  return lossFluctuation;
}

void G4EmParameters::SetBuildCSDARange(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  buildCSDARange = val;
}

G4bool G4EmParameters::BuildCSDARange() const 
{
  return buildCSDARange;
}

void G4EmParameters::SetLPM(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  flagLPM = val;
}

G4bool G4EmParameters::LPM() const 
{
  return flagLPM;
}

void G4EmParameters::SetSpline(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  spline = val;
}

G4bool G4EmParameters::Spline() const
{
  return spline;
}

void G4EmParameters::SetUseCutAsFinalRange(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  finalRange = val;
}

G4bool G4EmParameters::UseCutAsFinalRange() const
{
  return finalRange;
}

void G4EmParameters::SetApplyCuts(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  applyCuts = val;
}

G4bool G4EmParameters::ApplyCuts() const
{
  return applyCuts;
}

void G4EmParameters::SetFluo(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  fluo = val;
}

G4bool G4EmParameters::Fluo() const
{
  return fluo;
}

void G4EmParameters::SetAuger(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  auger = val;
  if(val) { fluo = true; }
}

G4bool G4EmParameters::Auger() const
{
  return auger;
}

void G4EmParameters::SetPixe(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  pixe = val;
  if(val) { fluo = true; }
}

G4bool G4EmParameters::Pixe() const
{
  return pixe;
}

void G4EmParameters::SetDeexcitationIgnoreCut(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  deexIgnoreCut = val;
}

G4bool G4EmParameters::DeexcitationIgnoreCut() const
{
  return deexIgnoreCut;
}

void G4EmParameters::SetLateralDisplacement(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  lateralDisplacement = val;
}

G4bool G4EmParameters::LateralDisplacement() const
{
  return lateralDisplacement;
}

void G4EmParameters::SetMuHadLateralDisplacement(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  muhadLateralDisplacement = val;
}

G4bool G4EmParameters::MuHadLateralDisplacement() const
{
  return muhadLateralDisplacement;
}

void G4EmParameters::SetLatDisplacementBeyondSafety(G4bool val)
{
  G4AutoLock l(&EmParametersMutex);
  latDisplacementBeyondSafety = val;
}

G4bool G4EmParameters::LatDisplacementBeyondSafety() const
{
  return latDisplacementBeyondSafety;
}

void G4EmParameters::ActivateAngularGeneratorForIonisation(G4bool val)
{
  useAngGeneratorForIonisation = val;
}

G4bool G4EmParameters::UseAngularGeneratorForIonisation() const
{
  return useAngGeneratorForIonisation;
}

void G4EmParameters::SetMinSubRange(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0 && val < 1.0) {
    minSubRange = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of MinSubRange is out of range (0 - 1): " << val
       << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MinSubRange() const
{
  return minSubRange;
}

void G4EmParameters::SetMinEnergy(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 1.e-3*eV && val < maxKinEnergy) {
    minKinEnergy = val;
    nbins = nbinsPerDecade*G4lrint(std::log10(maxKinEnergy/minKinEnergy));
  } else {
    G4ExceptionDescription ed;
    ed << "Value of MinKinEnergy is out of range: " << val/MeV 
       << " MeV is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MinKinEnergy() const
{
  return minKinEnergy;
}

void G4EmParameters::SetMaxEnergy(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > minKinEnergy && val < 1.e+7*TeV) {
    maxKinEnergy = val;
    nbins = nbinsPerDecade*G4lrint(std::log10(maxKinEnergy/minKinEnergy));
  } else {
    G4ExceptionDescription ed;
    ed << "Value of MaxKinEnergy is out of range: " 
       << val/GeV << " GeV is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MaxKinEnergy() const
{
  return maxKinEnergy;
}

void G4EmParameters::SetMaxEnergyForCSDARange(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > minKinEnergy && val <= 100*TeV) {
    maxKinEnergyCSDA = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of MaxKinEnergyCSDA is out of range: " 
       << val/GeV << " GeV is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MaxEnergyForCSDARange() const
{
  return maxKinEnergyCSDA; 
}

void G4EmParameters::SetLinearLossLimit(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0 && val < 0.5) {
    linLossLimit = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of linLossLimit is out of range: " << val 
       << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::LinearLossLimit() const
{
  return linLossLimit;
}

void G4EmParameters::SetBremsstrahlungTh(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0) {
    bremsTh = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of bremsstrahlung threshold is out of range: " 
       << val/GeV << " GeV is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::BremsstrahlungTh() const 
{
  return bremsTh;
}

void G4EmParameters::SetLambdaFactor(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0 && val < 1.0) {
    lambdaFactor = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of lambda factor is out of range: " << val 
       << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::LambdaFactor() const 
{
  return lambdaFactor;
}

void G4EmParameters::SetFactorForAngleLimit(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0) {
    factorForAngleLimit = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of factor for enegry limit is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::FactorForAngleLimit() const 
{
  return factorForAngleLimit;
}

void G4EmParameters::SetMscThetaLimit(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val >= 0.0 && val <= pi) {
    thetaLimit = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of polar angle limit is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MscThetaLimit() const 
{
  return thetaLimit;
}

void G4EmParameters::SetMscRangeFactor(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val > 0.0 && val < 1.0) {
    rangeFactor = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of rangeFactor is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MscRangeFactor() const 
{
  return rangeFactor;
}

void G4EmParameters::SetMscGeomFactor(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val >= 1.0) {
    geomFactor = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of geomFactor is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MscGeomFactor() const 
{
  return geomFactor;
}

void G4EmParameters::SetMscSkin(G4double val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val >= 1.0) {
    skin = val;
  } else {
    G4ExceptionDescription ed;
    ed << "Value of skin is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4double G4EmParameters::MscSkin() const 
{
  return skin;
}

void G4EmParameters::SetNumberOfBins(G4int val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val >= 5 && val < 10000000) {
    nbins = val;
    nbinsPerDecade = G4lrint(nbins/std::log10(maxKinEnergy/minKinEnergy));
  } else {
    G4ExceptionDescription ed;
    ed << "Value of number of bins is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4int G4EmParameters::NumberOfBins() const 
{
  return nbins;
}

void G4EmParameters::SetNumberOfBinsPerDecade(G4int val)
{
  G4AutoLock l(&EmParametersMutex);
  if(val >= 5 && val < 1000000) {
    nbinsPerDecade = val;
    nbins = nbinsPerDecade*G4lrint(std::log10(maxKinEnergy/minKinEnergy));
  } else {
    G4ExceptionDescription ed;
    ed << "Value of number of bins per decade is out of range: " 
       << val << " is ignored"; 
    PrintWarning(ed);
  }
}

G4int G4EmParameters::NumberOfBinsPerDecade() const 
{
  return nbinsPerDecade; 
}

void G4EmParameters::SetVerbose(G4int val)
{
  G4AutoLock l(&EmParametersMutex);
  verbose = val;
  workerVerbose = std::min(workerVerbose, verbose);
}

G4int G4EmParameters::Verbose() const 
{
  return verbose;
}

void G4EmParameters::SetWorkerVerbose(G4int val)
{
  G4AutoLock l(&EmParametersMutex);
  workerVerbose = val;
}

G4int G4EmParameters::WorkerVerbose() const 
{
  return workerVerbose;
}

void G4EmParameters::SetMscStepLimitType(G4MscStepLimitType val)
{
  G4AutoLock l(&EmParametersMutex);
  mscStepLimit = val;
}

G4MscStepLimitType G4EmParameters::MscStepLimitType() const 
{
  return mscStepLimit;
}

void G4EmParameters::PrintWarning(G4ExceptionDescription& ed)
{
  G4Exception("G4EmParameters", "em0044", JustWarning, ed);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo.....

