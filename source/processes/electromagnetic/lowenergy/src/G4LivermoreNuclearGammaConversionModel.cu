#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4LivermoreNuclearGammaConversionModel.cc 66241 2012-12-13 18:34:42Z gunter $
//
// Authors: G.Depaola & F.Longo
//

#include "G4LivermoreNuclearGammaConversionModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4LivermoreNuclearGammaConversionModel::G4LivermoreNuclearGammaConversionModel(const G4ParticleDefinition*,
								 const G4String& nam)
  :G4VEmModel(nam),fParticleChange(0),smallEnergy(2.*MeV),
   isInitialised(false),
   crossSectionHandler(0),meanFreePathTable(0)
{
  lowEnergyLimit = 2.0*electron_mass_c2;
  highEnergyLimit = 100 * GeV;
  SetHighEnergyLimit(highEnergyLimit);
  	 
  verboseLevel= 0;
  // Verbosity scale:
  // 0 = nothing 
  // 1 = warning for energy non-conservation 
  // 2 = details of energy budget
  // 3 = calculation of cross sections, file openings, sampling of atoms
  // 4 = entering in methods

  if(verboseLevel > 0) {
    G4cout << "Livermore Nuclear Gamma conversion is constructed " << G4endl
	   << "Energy range: "
	   << lowEnergyLimit / MeV << " MeV - "
	   << highEnergyLimit / GeV << " GeV"
	   << G4endl;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4LivermoreNuclearGammaConversionModel::~G4LivermoreNuclearGammaConversionModel()
{  
  if (crossSectionHandler) delete crossSectionHandler;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void 
G4LivermoreNuclearGammaConversionModel::Initialise(const G4ParticleDefinition*,
					    const G4DataVector&)
{
  if (verboseLevel > 3)
    G4cout << "Calling G4LivermoreNuclearGammaConversionModel::Initialise()" << G4endl;

  if (crossSectionHandler)
  {
    crossSectionHandler->Clear();
    delete crossSectionHandler;
  }

  // Read data tables for all materials
  
  crossSectionHandler = new G4CrossSectionHandler();
  crossSectionHandler->Initialise(0,lowEnergyLimit,100.*GeV,400);
  G4String crossSectionFile = "pairdata/pp-pair-cs-"; // here only pair in nuclear field cs should be used
  crossSectionHandler->LoadData(crossSectionFile);

  //
  
  if (verboseLevel > 0) {
    G4cout << "Loaded cross section files for Livermore GammaConversion" << G4endl;
    G4cout << "To obtain the total cross section this should be used only " << G4endl 
	   << "in connection with G4ElectronGammaConversion " << G4endl;
  }

  if (verboseLevel > 0) { 
    G4cout << "Livermore Nuclear Gamma Conversion model is initialized " << G4endl
	   << "Energy range: "
	   << LowEnergyLimit() / MeV << " MeV - "
	   << HighEnergyLimit() / GeV << " GeV"
	   << G4endl;
  }

  if(isInitialised) return;
  fParticleChange = GetParticleChangeForGamma();
  isInitialised = true;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double 
G4LivermoreNuclearGammaConversionModel::ComputeCrossSectionPerAtom(const G4ParticleDefinition*,
							    G4double GammaEnergy,
							    G4double Z, G4double,
							    G4double, G4double)
{
  if (verboseLevel > 3) {
    G4cout << "Calling ComputeCrossSectionPerAtom() of G4LivermoreNuclearGammaConversionModel" 
	   << G4endl;
  }
  if (GammaEnergy < lowEnergyLimit || GammaEnergy > highEnergyLimit) return 0;

  G4double cs = crossSectionHandler->FindValue(G4int(Z), GammaEnergy);
  return cs;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4LivermoreNuclearGammaConversionModel::SampleSecondaries(std::vector<G4DynamicParticle*>* fvect,
					      const G4MaterialCutsCouple* couple,
					      const G4DynamicParticle* aDynamicGamma,
					      G4double,
					      G4double)
{

// The energies of the e+ e- secondaries are sampled using the Bethe - Heitler
// cross sections with Coulomb correction. A modified version of the random
// number techniques of Butcher & Messel is used (Nuc Phys 20(1960),15).

// Note 1 : Effects due to the breakdown of the Born approximation at low
// energy are ignored.
// Note 2 : The differential cross section implicitly takes account of
// pair creation in both nuclear and atomic electron fields. However triplet
// prodution is not generated.

  if (verboseLevel > 3)
    G4cout << "Calling SampleSecondaries() of G4LivermoreNuclearGammaConversionModel" << G4endl;

  G4double photonEnergy = aDynamicGamma->GetKineticEnergy();
  G4ParticleMomentum photonDirection = aDynamicGamma->GetMomentumDirection();

  G4double epsilon ;
  G4double epsilon0Local = electron_mass_c2 / photonEnergy ;

  // Do it fast if photon energy < 2. MeV
  if (photonEnergy < smallEnergy )
    {
      epsilon = epsilon0Local + (0.5 - epsilon0Local) * G4UniformRand();
    }
  else
    {
      // Select randomly one element in the current material
      //const G4Element* element = crossSectionHandler->SelectRandomElement(couple,photonEnergy);
      const G4ParticleDefinition* particle =  aDynamicGamma->GetDefinition();
      const G4Element* element = SelectRandomAtom(couple,particle,photonEnergy);

      if (element == 0)
	{
	  G4cout << "G4LivermoreNuclearGammaConversionModel::SampleSecondaries - element = 0" 
		 << G4endl;
	  return;
	}
      G4IonisParamElm* ionisation = element->GetIonisation();
      if (ionisation == 0)
	{
	  G4cout << "G4LivermoreNuclearGammaConversionModel::SampleSecondaries - ionisation = 0" 
		 << G4endl;
	  return;
	}

      // Extract Coulomb factor for this Element
      G4double fZ = 8. * (ionisation->GetlogZ3());
      if (photonEnergy > 50. * MeV) fZ += 8. * (element->GetfCoulomb());

      // Limits of the screening variable
      G4double screenFactor = 136. * epsilon0Local / (element->GetIonisation()->GetZ3()) ;
      G4double screenMax = std::exp ((42.24 - fZ)/8.368) - 0.952 ;
      G4double screenMin = std::min(4.*screenFactor,screenMax) ;

      // Limits of the energy sampling
      G4double epsilon1 = 0.5 - 0.5 * std::sqrt(1. - screenMin / screenMax) ;
      G4double epsilonMin = std::max(epsilon0Local,epsilon1);
      G4double epsilonRange = 0.5 - epsilonMin ;

      // Sample the energy rate of the created electron (or positron)
      G4double screen;
      G4double gReject ;

      G4double f10 = ScreenFunction1(screenMin) - fZ;
      G4double f20 = ScreenFunction2(screenMin) - fZ;
      G4double normF1 = std::max(f10 * epsilonRange * epsilonRange,0.);
      G4double normF2 = std::max(1.5 * f20,0.);

      do {
	if (normF1 / (normF1 + normF2) > G4UniformRand() )
	  {
	    epsilon = 0.5 - epsilonRange * std::pow(G4UniformRand(), 0.3333) ;
	    screen = screenFactor / (epsilon * (1. - epsilon));
	    gReject = (ScreenFunction1(screen) - fZ) / f10 ;
	  }
	else
	  {
	    epsilon = epsilonMin + epsilonRange * G4UniformRand();
	    screen = screenFactor / (epsilon * (1 - epsilon));
	    gReject = (ScreenFunction2(screen) - fZ) / f20 ;
	  }
      } while ( gReject < G4UniformRand() );

    }   //  End of epsilon sampling

  // Fix charges randomly

  G4double electronTotEnergy;
  G4double positronTotEnergy;

  if (G4int(2*G4UniformRand()))    
    {
      electronTotEnergy = (1. - epsilon) * photonEnergy;
      positronTotEnergy = epsilon * photonEnergy;
    }
  else
    {
      positronTotEnergy = (1. - epsilon) * photonEnergy;
      electronTotEnergy = epsilon * photonEnergy;
    }

  // Scattered electron (positron) angles. ( Z - axis along the parent photon)
  // Universal distribution suggested by L. Urban (Geant3 manual (1993) Phys211),
  // derived from Tsai distribution (Rev. Mod. Phys. 49, 421 (1977)

  G4double u;
  const G4double a1 = 0.625;
  G4double a2 = 3. * a1;
  //  G4double d = 27. ;

  //  if (9. / (9. + d) > G4UniformRand())
  if (0.25 > G4UniformRand())
    {
      u = - std::log(G4UniformRand() * G4UniformRand()) / a1 ;
    }
  else
    {
      u = - std::log(G4UniformRand() * G4UniformRand()) / a2 ;
    }

  G4double thetaEle = u*electron_mass_c2/electronTotEnergy;
  G4double thetaPos = u*electron_mass_c2/positronTotEnergy;
  G4double phi  = twopi * G4UniformRand();

  G4double dxEle= std::sin(thetaEle)*std::cos(phi),dyEle= std::sin(thetaEle)*std::sin(phi),dzEle=std::cos(thetaEle);
  G4double dxPos=-std::sin(thetaPos)*std::cos(phi),dyPos=-std::sin(thetaPos)*std::sin(phi),dzPos=std::cos(thetaPos);
  
  
  // Kinematics of the created pair:
  // the electron and positron are assumed to have a symetric angular 
  // distribution with respect to the Z axis along the parent photon
  
  G4double electronKineEnergy = std::max(0.,electronTotEnergy - electron_mass_c2) ;
  
  // SI - The range test has been removed wrt original G4LowEnergyGammaconversion class

  G4ThreeVector electronDirection (dxEle, dyEle, dzEle);
  electronDirection.rotateUz(photonDirection);
      
  G4DynamicParticle* particle1 = new G4DynamicParticle (G4Electron::Electron(),
							    electronDirection,
							    electronKineEnergy);

  // The e+ is always created (even with kinetic energy = 0) for further annihilation
  G4double positronKineEnergy = std::max(0.,positronTotEnergy - electron_mass_c2) ;

  // SI - The range test has been removed wrt original G4LowEnergyGammaconversion class

  G4ThreeVector positronDirection (dxPos, dyPos, dzPos);
  positronDirection.rotateUz(photonDirection);   
  
  // Create G4DynamicParticle object for the particle2 
  G4DynamicParticle* particle2 = new G4DynamicParticle(G4Positron::Positron(),
						       positronDirection, positronKineEnergy);
  // Fill output vector

  fvect->push_back(particle1);
  fvect->push_back(particle2);

  // kill incident photon
  fParticleChange->SetProposedKineticEnergy(0.);
  fParticleChange->ProposeTrackStatus(fStopAndKill);   

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4LivermoreNuclearGammaConversionModel::ScreenFunction1(G4double screenVariable)
{
  // Compute the value of the screening function 3*phi1 - phi2

  G4double value;
  
  if (screenVariable > 1.)
    value = 42.24 - 8.368 * std::log(screenVariable + 0.952);
  else
    value = 42.392 - screenVariable * (7.796 - 1.961 * screenVariable);
  
  return value;
} 

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4LivermoreNuclearGammaConversionModel::ScreenFunction2(G4double screenVariable)
{
  // Compute the value of the screening function 1.5*phi1 - 0.5*phi2
  
  G4double value;
  
  if (screenVariable > 1.)
    value = 42.24 - 8.368 * std::log(screenVariable + 0.952);
  else
    value = 41.405 - screenVariable * (5.828 - 0.8945 * screenVariable);
  
  return value;
} 

