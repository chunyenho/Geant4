#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4LivermorePolarizedComptonModel.cc 82874 2014-07-15 15:25:29Z gcosmo $
//
// Authors: G.Depaola & F.Longo
//
// History:
// --------
// 02 May 2009   S Incerti as V. Ivanchenko proposed in G4LivermoreComptonModel.cc
//
// Cleanup initialisation and generation of secondaries:
//                  - apply internal high-energy limit only in constructor 
//                  - do not apply low-energy limit (default is 0)
//                  - remove GetMeanFreePath method and table
//                  - added protection against numerical problem in energy sampling 
//                  - use G4ElementSelector

#include "G4LivermorePolarizedComptonModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4LivermorePolarizedComptonModel::G4LivermorePolarizedComptonModel(const G4ParticleDefinition*,
                                             const G4String& nam)
  :G4VEmModel(nam),fParticleChange(0),isInitialised(false),
   meanFreePathTable(0),scatterFunctionData(0),crossSectionHandler(0)
{ 
  verboseLevel= 0;
  // Verbosity scale:
  // 0 = nothing 
  // 1 = warning for energy non-conservation 
  // 2 = details of energy budget
  // 3 = calculation of cross sections, file openings, sampling of atoms
  // 4 = entering in methods

  if( verboseLevel>0 )  
    G4cout << "Livermore Polarized Compton is constructed " << G4endl;
        
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4LivermorePolarizedComptonModel::~G4LivermorePolarizedComptonModel()
{  
  if (meanFreePathTable)   delete meanFreePathTable;
  if (crossSectionHandler) delete crossSectionHandler;
  if (scatterFunctionData) delete scatterFunctionData;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4LivermorePolarizedComptonModel::Initialise(const G4ParticleDefinition* particle,
                                       const G4DataVector& cuts)
{
  if (verboseLevel > 3)
    G4cout << "Calling G4LivermorePolarizedComptonModel::Initialise()" << G4endl;

  if (crossSectionHandler)
  {
    crossSectionHandler->Clear();
    delete crossSectionHandler;
  }

  // Reading of data files - all materials are read
  
  crossSectionHandler = new G4CrossSectionHandler;
  crossSectionHandler->Clear();
  G4String crossSectionFile = "comp/ce-cs-";
  crossSectionHandler->LoadData(crossSectionFile);

  meanFreePathTable = 0;
  meanFreePathTable = crossSectionHandler->BuildMeanFreePathForMaterials();

  G4VDataSetAlgorithm* scatterInterpolation = new G4LogLogInterpolation;
  G4String scatterFile = "comp/ce-sf-";
  scatterFunctionData = new G4CompositeEMDataSet(scatterInterpolation, 1., 1.);
  scatterFunctionData->LoadData(scatterFile);

  // For Doppler broadening
  shellData.SetOccupancyData();
  G4String file = "/doppler/shell-doppler";
  shellData.LoadData(file);

  if (verboseLevel > 2) 
    G4cout << "Loaded cross section files for Livermore Polarized Compton model" << G4endl;

  InitialiseElementSelectors(particle,cuts);

  if(  verboseLevel>0 ) { 
    G4cout << "Livermore Polarized Compton model is initialized " << G4endl
         << "Energy range: "
         << LowEnergyLimit() / eV << " eV - "
         << HighEnergyLimit() / GeV << " GeV"
         << G4endl;
  }
  
  //
    
  if(isInitialised) return;
  fParticleChange = GetParticleChangeForGamma();
  isInitialised = true;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4LivermorePolarizedComptonModel::ComputeCrossSectionPerAtom(
                                       const G4ParticleDefinition*,
                                             G4double GammaEnergy,
                                             G4double Z, G4double,
                                             G4double, G4double)
{
  if (verboseLevel > 3)
    G4cout << "Calling ComputeCrossSectionPerAtom() of G4LivermorePolarizedComptonModel" << G4endl;

  if (GammaEnergy < LowEnergyLimit()) 
    return 0.0;

  G4double cs = crossSectionHandler->FindValue(G4int(Z), GammaEnergy);
  return cs;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4LivermorePolarizedComptonModel::SampleSecondaries(std::vector<G4DynamicParticle*>* fvect,
					      const G4MaterialCutsCouple* couple,
					      const G4DynamicParticle* aDynamicGamma,
					      G4double,
					      G4double)
{
  // The scattered gamma energy is sampled according to Klein - Nishina formula.
  // The random number techniques of Butcher & Messel are used (Nuc Phys 20(1960),15).
  // GEANT4 internal units
  //
  // Note : Effects due to binding of atomic electrons are negliged.

  if (verboseLevel > 3)
    G4cout << "Calling SampleSecondaries() of G4LivermorePolarizedComptonModel" << G4endl;

  G4double gammaEnergy0 = aDynamicGamma->GetKineticEnergy();
 
  // do nothing below the threshold
  // should never get here because the XS is zero below the limit
  if (gammaEnergy0 < LowEnergyLimit())     
    return ; 


  G4ThreeVector gammaPolarization0 = aDynamicGamma->GetPolarization();

  // Protection: a polarisation parallel to the
  // direction causes problems;
  // in that case find a random polarization

  G4ThreeVector gammaDirection0 = aDynamicGamma->GetMomentumDirection();

  // Make sure that the polarization vector is perpendicular to the
  // gamma direction. If not

  if(!(gammaPolarization0.isOrthogonal(gammaDirection0, 1e-6))||(gammaPolarization0.mag()==0))
    { // only for testing now
      gammaPolarization0 = GetRandomPolarization(gammaDirection0);
    }
  else
    {
      if ( gammaPolarization0.howOrthogonal(gammaDirection0) != 0)
	{
	  gammaPolarization0 = GetPerpendicularPolarization(gammaDirection0, gammaPolarization0);
	}
    }

  // End of Protection

  G4double E0_m = gammaEnergy0 / electron_mass_c2 ;

  // Select randomly one element in the current material
  //G4int Z = crossSectionHandler->SelectRandomAtom(couple,gammaEnergy0);
  const G4ParticleDefinition* particle =  aDynamicGamma->GetDefinition();
  const G4Element* elm = SelectRandomAtom(couple,particle,gammaEnergy0);
  G4int Z = (G4int)elm->GetZ();

  // Sample the energy and the polarization of the scattered photon

  G4double epsilon, epsilonSq, onecost, sinThetaSqr, greject ;

  G4double epsilon0Local = 1./(1. + 2*E0_m);
  G4double epsilon0Sq = epsilon0Local*epsilon0Local;
  G4double alpha1   = - std::log(epsilon0Local);
  G4double alpha2 = 0.5*(1.- epsilon0Sq);

  G4double wlGamma = h_Planck*c_light/gammaEnergy0;
  G4double gammaEnergy1;
  G4ThreeVector gammaDirection1;

  do {
    if ( alpha1/(alpha1+alpha2) > G4UniformRand() )
      {
	epsilon   = std::exp(-alpha1*G4UniformRand());  
	epsilonSq = epsilon*epsilon; 
      }
    else 
      {
	epsilonSq = epsilon0Sq + (1.- epsilon0Sq)*G4UniformRand();
	epsilon   = std::sqrt(epsilonSq);
      }

    onecost = (1.- epsilon)/(epsilon*E0_m);
    sinThetaSqr   = onecost*(2.-onecost);

    // Protection
    if (sinThetaSqr > 1.)
      {
	G4cout
	  << " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	  << "sin(theta)**2 = "
	  << sinThetaSqr
	  << "; set to 1"
	  << G4endl;
	sinThetaSqr = 1.;
      }
    if (sinThetaSqr < 0.)
      {
	G4cout
	  << " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	  << "sin(theta)**2 = "
	  << sinThetaSqr
	  << "; set to 0"
	  << G4endl;
	sinThetaSqr = 0.;
      }
    // End protection

    G4double x =  std::sqrt(onecost/2.) / (wlGamma/cm);;
    G4double scatteringFunction = scatterFunctionData->FindValue(x,Z-1);
    greject = (1. - epsilon*sinThetaSqr/(1.+ epsilonSq))*scatteringFunction;

  } while(greject < G4UniformRand()*Z);


  // ****************************************************
  //		Phi determination
  // ****************************************************

  G4double phi = SetPhi(epsilon,sinThetaSqr);

  //
  // scattered gamma angles. ( Z - axis along the parent gamma)
  //

  G4double cosTheta = 1. - onecost;

  // Protection

  if (cosTheta > 1.)
    {
      G4cout
	<< " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	<< "cosTheta = "
	<< cosTheta
	<< "; set to 1"
	<< G4endl;
      cosTheta = 1.;
    }
  if (cosTheta < -1.)
    {
      G4cout 
	<< " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	<< "cosTheta = " 
	<< cosTheta
	<< "; set to -1"
	<< G4endl;
      cosTheta = -1.;
    }
  // End protection      
  
  
  G4double sinTheta = std::sqrt (sinThetaSqr);
  
  // Protection
  if (sinTheta > 1.)
    {
      G4cout 
	<< " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	<< "sinTheta = " 
	<< sinTheta
	<< "; set to 1"
	<< G4endl;
      sinTheta = 1.;
    }
  if (sinTheta < -1.)
    {
      G4cout 
	<< " -- Warning -- G4LivermorePolarizedComptonModel::SampleSecondaries "
	<< "sinTheta = " 
	<< sinTheta
	<< "; set to -1" 
	<< G4endl;
      sinTheta = -1.;
    }
  // End protection
  
      
  G4double dirx = sinTheta*std::cos(phi);
  G4double diry = sinTheta*std::sin(phi);
  G4double dirz = cosTheta ;
  

  // oneCosT , eom

  // Doppler broadening -  Method based on:
  // Y. Namito, S. Ban and H. Hirayama, 
  // "Implementation of the Doppler Broadening of a Compton-Scattered Photon Into the EGS4 Code" 
  // NIM A 349, pp. 489-494, 1994
  
  // Maximum number of sampling iterations

  G4int maxDopplerIterations = 1000;
  G4double bindingE = 0.;
  G4double photonEoriginal = epsilon * gammaEnergy0;
  G4double photonE = -1.;
  G4int iteration = 0;
  G4double eMax = gammaEnergy0;

  do
    {
      iteration++;
      // Select shell based on shell occupancy
      G4int shell = shellData.SelectRandomShell(Z);
      bindingE = shellData.BindingEnergy(Z,shell);
      
      eMax = gammaEnergy0 - bindingE;
     
      // Randomly sample bound electron momentum (memento: the data set is in Atomic Units)
      G4double pSample = profileData.RandomSelectMomentum(Z,shell);
      // Rescale from atomic units
      G4double pDoppler = pSample * fine_structure_const;
      G4double pDoppler2 = pDoppler * pDoppler;
      G4double var2 = 1. + onecost * E0_m;
      G4double var3 = var2*var2 - pDoppler2;
      G4double var4 = var2 - pDoppler2 * cosTheta;
      G4double var = var4*var4 - var3 + pDoppler2 * var3;
      if (var > 0.)
	{
	  G4double varSqrt = std::sqrt(var);        
	  G4double scale = gammaEnergy0 / var3;  
          // Random select either root
 	  if (G4UniformRand() < 0.5) photonE = (var4 - varSqrt) * scale;               
	  else photonE = (var4 + varSqrt) * scale;
	} 
      else
	{
	  photonE = -1.;
	}
   } while ( iteration <= maxDopplerIterations && 
	     (photonE < 0. || photonE > eMax || photonE < eMax*G4UniformRand()) );
 
  // End of recalculation of photon energy with Doppler broadening
  // Revert to original if maximum number of iterations threshold has been reached
  if (iteration >= maxDopplerIterations)
    {
      photonE = photonEoriginal;
      bindingE = 0.;
    }

  gammaEnergy1 = photonE;
 
  //
  // update G4VParticleChange for the scattered photon 
  //

  //  gammaEnergy1 = epsilon*gammaEnergy0;


  // New polarization

  G4ThreeVector gammaPolarization1 = SetNewPolarization(epsilon,
							sinThetaSqr,
							phi,
							cosTheta);

  // Set new direction
  G4ThreeVector tmpDirection1( dirx,diry,dirz );
  gammaDirection1 = tmpDirection1;

  // Change reference frame.

  SystemOfRefChange(gammaDirection0,gammaDirection1,
		    gammaPolarization0,gammaPolarization1);

  if (gammaEnergy1 > 0.)
    {
      fParticleChange->SetProposedKineticEnergy( gammaEnergy1 ) ;
      fParticleChange->ProposeMomentumDirection( gammaDirection1 );
      fParticleChange->ProposePolarization( gammaPolarization1 );
    }
  else
    {
      gammaEnergy1 = 0.;
      fParticleChange->SetProposedKineticEnergy(0.) ;
      fParticleChange->ProposeTrackStatus(fStopAndKill);
    }

  //
  // kinematic of the scattered electron
  //

  G4double ElecKineEnergy = gammaEnergy0 - gammaEnergy1 -bindingE;

  // SI -protection against negative final energy: no e- is created
  // like in G4LivermoreComptonModel.cc
  if(ElecKineEnergy < 0.0) {
    fParticleChange->ProposeLocalEnergyDeposit(gammaEnergy0 - gammaEnergy1);
    return;
  }
 
  // SI - Removed range test
  
  G4double ElecMomentum = std::sqrt(ElecKineEnergy*(ElecKineEnergy+2.*electron_mass_c2));

  G4ThreeVector ElecDirection((gammaEnergy0 * gammaDirection0 -
				   gammaEnergy1 * gammaDirection1) * (1./ElecMomentum));

  fParticleChange->ProposeLocalEnergyDeposit(bindingE);
  
  G4DynamicParticle* dp = new G4DynamicParticle (G4Electron::Electron(),ElecDirection.unit(),ElecKineEnergy) ;
  fvect->push_back(dp);

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4LivermorePolarizedComptonModel::SetPhi(G4double energyRate,
					     G4double sinSqrTh)
{
  G4double rand1;
  G4double rand2;
  G4double phiProbability;
  G4double phi;
  G4double a, b;

  do
    {
      rand1 = G4UniformRand();
      rand2 = G4UniformRand();
      phiProbability=0.;
      phi = twopi*rand1;
      
      a = 2*sinSqrTh;
      b = energyRate + 1/energyRate;
      
      phiProbability = 1 - (a/b)*(std::cos(phi)*std::cos(phi));

      
 
    }
  while ( rand2 > phiProbability );
  return phi;
}


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4LivermorePolarizedComptonModel::SetPerpendicularVector(G4ThreeVector& a)
{
  G4double dx = a.x();
  G4double dy = a.y();
  G4double dz = a.z();
  G4double x = dx < 0.0 ? -dx : dx;
  G4double y = dy < 0.0 ? -dy : dy;
  G4double z = dz < 0.0 ? -dz : dz;
  if (x < y) {
    return x < z ? G4ThreeVector(-dy,dx,0) : G4ThreeVector(0,-dz,dy);
  }else{
    return y < z ? G4ThreeVector(dz,0,-dx) : G4ThreeVector(-dy,dx,0);
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4LivermorePolarizedComptonModel::GetRandomPolarization(G4ThreeVector& direction0)
{
  G4ThreeVector d0 = direction0.unit();
  G4ThreeVector a1 = SetPerpendicularVector(d0); //different orthogonal
  G4ThreeVector a0 = a1.unit(); // unit vector

  G4double rand1 = G4UniformRand();
  
  G4double angle = twopi*rand1; // random polar angle
  G4ThreeVector b0 = d0.cross(a0); // cross product
  
  G4ThreeVector c;
  
  c.setX(std::cos(angle)*(a0.x())+std::sin(angle)*b0.x());
  c.setY(std::cos(angle)*(a0.y())+std::sin(angle)*b0.y());
  c.setZ(std::cos(angle)*(a0.z())+std::sin(angle)*b0.z());
  
  G4ThreeVector c0 = c.unit();

  return c0;
  
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4LivermorePolarizedComptonModel::GetPerpendicularPolarization
(const G4ThreeVector& gammaDirection, const G4ThreeVector& gammaPolarization) const
{

  // 
  // The polarization of a photon is always perpendicular to its momentum direction.
  // Therefore this function removes those vector component of gammaPolarization, which
  // points in direction of gammaDirection
  //
  // Mathematically we search the projection of the vector a on the plane E, where n is the
  // plains normal vector.
  // The basic equation can be found in each geometry book (e.g. Bronstein):
  // p = a - (a o n)/(n o n)*n
  
  return gammaPolarization - gammaPolarization.dot(gammaDirection)/gammaDirection.dot(gammaDirection) * gammaDirection;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ThreeVector G4LivermorePolarizedComptonModel::SetNewPolarization(G4double epsilon,
							      G4double sinSqrTh, 
							      G4double phi,
							      G4double costheta) 
{
  G4double rand1;
  G4double rand2;
  G4double cosPhi = std::cos(phi);
  G4double sinPhi = std::sin(phi);
  G4double sinTheta = std::sqrt(sinSqrTh);
  G4double cosSqrPhi = cosPhi*cosPhi;
  //  G4double cossqrth = 1.-sinSqrTh;
  //  G4double sinsqrphi = sinPhi*sinPhi;
  G4double normalisation = std::sqrt(1. - cosSqrPhi*sinSqrTh);
 

  // Determination of Theta 
  
  // ---- MGP ---- Commented out the following 3 lines to avoid compilation 
  // warnings (unused variables)
  // G4double thetaProbability;
  G4double theta;
  // G4double a, b;
  // G4double cosTheta;

  /*

  depaola method
  
  do
  {
      rand1 = G4UniformRand();
      rand2 = G4UniformRand();
      thetaProbability=0.;
      theta = twopi*rand1;
      a = 4*normalisation*normalisation;
      b = (epsilon + 1/epsilon) - 2;
      thetaProbability = (b + a*std::cos(theta)*std::cos(theta))/(a+b);
      cosTheta = std::cos(theta);
    }
  while ( rand2 > thetaProbability );
  
  G4double cosBeta = cosTheta;

  */


  // Dan Xu method (IEEE TNS, 52, 1160 (2005))

  rand1 = G4UniformRand();
  rand2 = G4UniformRand();

  if (rand1<(epsilon+1.0/epsilon-2)/(2.0*(epsilon+1.0/epsilon)-4.0*sinSqrTh*cosSqrPhi))
    {
      if (rand2<0.5)
	theta = pi/2.0;
      else
	theta = 3.0*pi/2.0;
    }
  else
    {
      if (rand2<0.5)
	theta = 0;
      else
	theta = pi;
    }
  G4double cosBeta = std::cos(theta);
  G4double sinBeta = std::sqrt(1-cosBeta*cosBeta);
  
  G4ThreeVector gammaPolarization1;

  G4double xParallel = normalisation*cosBeta;
  G4double yParallel = -(sinSqrTh*cosPhi*sinPhi)*cosBeta/normalisation;
  G4double zParallel = -(costheta*sinTheta*cosPhi)*cosBeta/normalisation;
  G4double xPerpendicular = 0.;
  G4double yPerpendicular = (costheta)*sinBeta/normalisation;
  G4double zPerpendicular = -(sinTheta*sinPhi)*sinBeta/normalisation;

  G4double xTotal = (xParallel + xPerpendicular);
  G4double yTotal = (yParallel + yPerpendicular);
  G4double zTotal = (zParallel + zPerpendicular);
  
  gammaPolarization1.setX(xTotal);
  gammaPolarization1.setY(yTotal);
  gammaPolarization1.setZ(zTotal);
  
  return gammaPolarization1;

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4LivermorePolarizedComptonModel::SystemOfRefChange(G4ThreeVector& direction0,
						    G4ThreeVector& direction1,
						    G4ThreeVector& polarization0,
						    G4ThreeVector& polarization1)
{
  // direction0 is the original photon direction ---> z
  // polarization0 is the original photon polarization ---> x
  // need to specify y axis in the real reference frame ---> y 
  G4ThreeVector Axis_Z0 = direction0.unit();
  G4ThreeVector Axis_X0 = polarization0.unit();
  G4ThreeVector Axis_Y0 = (Axis_Z0.cross(Axis_X0)).unit(); // to be confirmed;

  G4double direction_x = direction1.getX();
  G4double direction_y = direction1.getY();
  G4double direction_z = direction1.getZ();
  
  direction1 = (direction_x*Axis_X0 + direction_y*Axis_Y0 + direction_z*Axis_Z0).unit();
  G4double polarization_x = polarization1.getX();
  G4double polarization_y = polarization1.getY();
  G4double polarization_z = polarization1.getZ();

  polarization1 = (polarization_x*Axis_X0 + polarization_y*Axis_Y0 + polarization_z*Axis_Z0).unit();

}


