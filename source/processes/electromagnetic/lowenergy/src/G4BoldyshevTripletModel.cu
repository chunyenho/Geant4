#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4BoldyshevTripletModel.cc 74822 2013-10-22 14:42:13Z gcosmo $
// GEANT4 tag $Name:  $
//
//
// Author: Gerardo Depaola & Francesco Longo
//
// History:
// --------
//   23-06-2010 First implementation as model 


#include "G4BoldyshevTripletModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4BoldyshevTripletModel::G4BoldyshevTripletModel(const G4ParticleDefinition*,
								 const G4String& nam)
  :G4VEmModel(nam),fParticleChange(0),smallEnergy(4.*MeV),isInitialised(false),
   crossSectionHandler(0),meanFreePathTable(0)
{
  lowEnergyLimit = 4.0*electron_mass_c2;
  highEnergyLimit = 100 * GeV;
  SetHighEnergyLimit(highEnergyLimit);
  	 
  verboseLevel= 0;
  // Verbosity scale:
  // 0 = nothing 
  // 1 = warning for energy non-conservation 
  // 2 = details of energy budget
  // 3 = calculation of cross sections, file openings, sampling of atoms
  // 4 = entering in methods

  if(verboseLevel > 0) {
    G4cout << "Triplet Gamma conversion is constructed " << G4endl
	   << "Energy range: "
	   << lowEnergyLimit / MeV << " MeV - "
	   << highEnergyLimit / GeV << " GeV"
	   << G4endl;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4BoldyshevTripletModel::~G4BoldyshevTripletModel()
{  
  if (crossSectionHandler) delete crossSectionHandler;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void 
G4BoldyshevTripletModel::Initialise(const G4ParticleDefinition*,
					    const G4DataVector&)
{
  if (verboseLevel > 3)
    G4cout << "Calling G4BoldyshevTripletModel::Initialise()" << G4endl;

  if (crossSectionHandler)
  {
    crossSectionHandler->Clear();
    delete crossSectionHandler;
  }

  // Read data tables for all materials
  
  crossSectionHandler = new G4CrossSectionHandler();
  crossSectionHandler->Initialise(0,lowEnergyLimit,100.*GeV,400);
  G4String crossSectionFile = "tripdata/pp-trip-cs-"; // here only pair in electron field cs should be used
  crossSectionHandler->LoadData(crossSectionFile);

  //
  
  if (verboseLevel > 0) {
    G4cout << "Loaded cross section files for Livermore GammaConversion" << G4endl;
    G4cout << "To obtain the total cross section this should be used only " << G4endl 
	   << "in connection with G4NuclearGammaConversion " << G4endl;
  }

  if (verboseLevel > 0) { 
    G4cout << "Livermore Electron Gamma Conversion model is initialized " << G4endl
	   << "Energy range: "
	   << LowEnergyLimit() / MeV << " MeV - "
	   << HighEnergyLimit() / GeV << " GeV"
	   << G4endl;
  }

  if(isInitialised) return;
  fParticleChange = GetParticleChangeForGamma();
  isInitialised = true;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double 
G4BoldyshevTripletModel::ComputeCrossSectionPerAtom(const G4ParticleDefinition*,
							    G4double GammaEnergy,
							    G4double Z, G4double,
							    G4double, G4double)
{
  if (verboseLevel > 3) {
    G4cout << "Calling ComputeCrossSectionPerAtom() of G4BoldyshevTripletModel" 
	   << G4endl;
  }
  if (GammaEnergy < lowEnergyLimit || GammaEnergy > highEnergyLimit) return 0;

  G4double cs = crossSectionHandler->FindValue(G4int(Z), GammaEnergy);
  return cs;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4BoldyshevTripletModel::SampleSecondaries(std::vector<G4DynamicParticle*>* fvect,
					      const G4MaterialCutsCouple* ,
					      const G4DynamicParticle* aDynamicGamma,
					      G4double,
					      G4double)
{

// The energies of the secondary particles are sampled using
// a modified Wheeler-Lamb model (see PhysRevD 7 (1973), 26) 

  if (verboseLevel > 3)
    G4cout << "Calling SampleSecondaries() of G4BoldyshevTripletModel" << G4endl;

  G4double photonEnergy = aDynamicGamma->GetKineticEnergy();
  G4ParticleMomentum photonDirection = aDynamicGamma->GetMomentumDirection();

  G4double epsilon ;
  G4double p0 = electron_mass_c2; 
  
  G4double positronTotEnergy, electronTotEnergy, thetaEle, thetaPos;
  G4double ener_re=0., theta_re, phi_re, phi;

  // Calculo de theta - elecron de recoil
  
  G4double energyThreshold = sqrt(2.)*electron_mass_c2; // -> momentumThreshold_N = 1
  energyThreshold = 1.1*electron_mass_c2;
  // G4cout << energyThreshold << G4endl; 
 
  G4double momentumThreshold_c = sqrt(energyThreshold * energyThreshold - electron_mass_c2*electron_mass_c2); // momentun in MeV/c unit
  G4double momentumThreshold_N = momentumThreshold_c/electron_mass_c2; // momentun in mc unit
  
  // Calculation of recoil electron production 
  
  G4double SigmaTot = (28./9.) * std::log ( 2.* photonEnergy / electron_mass_c2 ) - 218. / 27. ; 
  G4double X_0 = 2. * ( sqrt(momentumThreshold_N*momentumThreshold_N + 1) -1 );
  G4double SigmaQ = (82./27. - (14./9.) * log (X_0) + 4./15.*X_0 - 0.0348 * X_0 * X_0); 
  G4double recoilProb = G4UniformRand();
  //G4cout << "SIGMA TOT " << SigmaTot <<  " " << "SigmaQ " << SigmaQ << " " << SigmaQ/SigmaTot << " " << recoilProb << G4endl;
  
  if (recoilProb >= SigmaQ/SigmaTot) // create electron recoil 
    { 
      
      G4double cosThetaMax = (  ( energyThreshold - electron_mass_c2 ) / (momentumThreshold_c) + electron_mass_c2*
	    			( energyThreshold + electron_mass_c2 ) / (photonEnergy*momentumThreshold_c) );
      
      if (cosThetaMax > 1) G4cout << "ERRORE " << G4endl;
      
      G4double r1;
      G4double r2;
      G4double are, bre, loga, f1_re, greject, cost;
      
      do {
	r1 = G4UniformRand();
	r2 = G4UniformRand();
	//	cost = (pow(4./enern,0.5*r1)) ;
	cost = pow(cosThetaMax,r1);
	theta_re = acos(cost);
	are = 1./(14.*cost*cost);
	bre = (1.-5.*cost*cost)/(2.*cost);
	loga = log((1.+ cost)/(1.- cost));
	f1_re = 1. - bre*loga;
	
	if ( theta_re >= 4.47*CLHEP::pi/180.)
	  {
	    greject = are*f1_re;
	  } else {
	  greject = 1. ;
	}
      } while(greject < r2);
      
      // Calculo de phi - elecron de recoil
      
      G4double r3, r4, rt;
      
      do {
	
	r3 = G4UniformRand();
	r4 = G4UniformRand();
	phi_re = twopi*r3 ;
	G4double sint2 = 1. - cost*cost ;
	G4double fp = 1. - sint2*loga/(2.*cost) ;
	rt = (1.-cos(2.*phi_re)*fp/f1_re)/(2.*pi) ;
	
      } while(rt < r4);
      
      // Calculo de la energia - elecron de recoil - relacion momento maximo <-> angulo
      
      G4double S = electron_mass_c2*(2.* photonEnergy + electron_mass_c2);
      G4double D2 = 4.*S * electron_mass_c2*electron_mass_c2
	+ (S - electron_mass_c2*electron_mass_c2)
	*(S - electron_mass_c2*electron_mass_c2)*sin(theta_re)*sin(theta_re);
      ener_re = electron_mass_c2 * (S + electron_mass_c2*electron_mass_c2)/sqrt(D2);
      
      // New Recoil energy calculation 

      G4double momentum_recoil = 2* (electron_mass_c2) * (std::cos(theta_re)/(std::sin(phi_re)*std::sin(phi_re)));
      G4double ener_recoil = sqrt( momentum_recoil*momentum_recoil + electron_mass_c2*electron_mass_c2);
      ener_re = ener_recoil;

      //      G4cout << "electron de retroceso " << ener_re << " " << theta_re << " " << phi_re << G4endl;
      
      // Recoil electron creation
      G4double dxEle_re=sin(theta_re)*std::cos(phi_re),dyEle_re=sin(theta_re)*std::sin(phi_re), dzEle_re=cos(theta_re);
      
      G4double electronRKineEnergy = std::max(0.,ener_re - electron_mass_c2) ;
      
      G4ThreeVector electronRDirection (dxEle_re, dyEle_re, dzEle_re);
      electronRDirection.rotateUz(photonDirection);
      
      G4DynamicParticle* particle3 = new G4DynamicParticle (G4Electron::Electron(),
							    electronRDirection,
							    electronRKineEnergy);
      fvect->push_back(particle3);	    
      
    }
  else
    {
      // deposito la energia  ener_re - electron_mass_c2
      // G4cout << "electron de retroceso " << ener_re << G4endl;
      fParticleChange->ProposeLocalEnergyDeposit(ener_re - electron_mass_c2);
    }
  
  // Depaola (2004) suggested distribution for e+e- energy
  
  //  G4double t = 0.5*asinh(momentumThreshold_N);
  G4double t = 0.5*log(momentumThreshold_N + sqrt(momentumThreshold_N*momentumThreshold_N+1));
  
  G4cout << 0.5*asinh(momentumThreshold_N) << "  " << t << G4endl;

  G4double J1 = 0.5*(t*cosh(t)/sinh(t) - log(2.*sinh(t)));
  G4double J2 = (-2./3.)*log(2.*sinh(t)) + t*cosh(t)/sinh(t) + (sinh(t)-t*pow(cosh(t),3))/(3.*pow(sinh(t),3));
  G4double b = 2.*(J1-J2)/J1;
  
  G4double n = 1 - b/6.;
  G4double re=0.;
  re = G4UniformRand();
  G4double a = 0.;
  G4double b1 =  16. - 3.*b - 36.*b*re*n + 36.*b*pow(re,2.)*pow(n,2.) + 
    6.*pow(b,2.)*re*n;
  a = pow((b1/b),0.5);
  G4double c1 = (-6. + 12.*re*n + b + 2*a)*pow(b,2.);
  epsilon = (pow(c1,1./3.))/(2.*b) + (b-4.)/(2.*pow(c1,1./3.))+0.5;
  
  G4double photonEnergy1 = photonEnergy - ener_re ; // resto al foton la energia del electron de retro.
  positronTotEnergy = epsilon*photonEnergy1;
  electronTotEnergy = photonEnergy1 - positronTotEnergy; // temporarly
  
  G4double momento_e = sqrt(electronTotEnergy*electronTotEnergy - 
			    electron_mass_c2*electron_mass_c2) ;
  G4double momento_p = sqrt(positronTotEnergy*positronTotEnergy - 
			    electron_mass_c2*electron_mass_c2) ;
  
  thetaEle = acos((sqrt(p0*p0/(momento_e*momento_e) +1.)- p0/momento_e)) ;
  thetaPos = acos((sqrt(p0*p0/(momento_p*momento_p) +1.)- p0/momento_p)) ;
  phi  = twopi * G4UniformRand();
  
  G4double dxEle= std::sin(thetaEle)*std::cos(phi),dyEle= std::sin(thetaEle)*std::sin(phi),dzEle=std::cos(thetaEle);
  G4double dxPos=-std::sin(thetaPos)*std::cos(phi),dyPos=-std::sin(thetaPos)*std::sin(phi),dzPos=std::cos(thetaPos);
  
  
  // Kinematics of the created pair:
  // the electron and positron are assumed to have a symetric angular 
  // distribution with respect to the Z axis along the parent photon
  
  G4double electronKineEnergy = std::max(0.,electronTotEnergy - electron_mass_c2) ;
  
  // SI - The range test has been removed wrt original G4LowEnergyGammaconversion class
  
  G4ThreeVector electronDirection (dxEle, dyEle, dzEle);
  electronDirection.rotateUz(photonDirection);
  
  G4DynamicParticle* particle1 = new G4DynamicParticle (G4Electron::Electron(),
							electronDirection,
							electronKineEnergy);
  
  // The e+ is always created (even with kinetic energy = 0) for further annihilation
  G4double positronKineEnergy = std::max(0.,positronTotEnergy - electron_mass_c2) ;
  
  // SI - The range test has been removed wrt original G4LowEnergyGammaconversion class
  
  G4ThreeVector positronDirection (dxPos, dyPos, dzPos);
  positronDirection.rotateUz(photonDirection);   
  
  // Create G4DynamicParticle object for the particle2 
  G4DynamicParticle* particle2 = new G4DynamicParticle(G4Positron::Positron(),
						       positronDirection, positronKineEnergy);
  // Fill output vector
  
  
  fvect->push_back(particle1);
  fvect->push_back(particle2);
  

  
  
  // kill incident photon
  fParticleChange->SetProposedKineticEnergy(0.);
  fParticleChange->ProposeTrackStatus(fStopAndKill);   
  
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....




