#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4ePolarizedIonisation.cc 85018 2014-10-23 09:51:37Z gcosmo $
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4ePolarizedIonisation
//
// Author:        A.Schaelicke on base of Vladimir Ivanchenko code
//
// Creation date: 10.11.2005
//
// Modifications:
//
// 10-11-05, include polarization description (A.Schaelicke)
// , create asymmetry table and determine interactionlength 
// , update polarized differential cross section 
//
// 20-08-06, modified interface (A.Schaelicke)
// 11-06-07, add PostStepGetPhysicalInteractionLength (A.Schalicke)
//
// Class Description:
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4ePolarizedIonisation.hh"
#include "G4Electron.hh"
#include "G4UniversalFluctuation.hh"
#include "G4BohrFluctuations.hh"
#include "G4UnitsTable.hh"

#include "G4PolarizedMollerBhabhaModel.hh"
#include "G4ProductionCutsTable.hh"
#include "G4PolarizationManager.hh"
#include "G4PolarizationHelper.hh"
#include "G4StokesVector.hh"
#include "G4EmParameters.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ePolarizedIonisation::G4ePolarizedIonisation(const G4String& name)
  : G4VEnergyLossProcess(name),
    theElectron(G4Electron::Electron()),
    isElectron(true),
    isInitialised(false),
    theAsymmetryTable(NULL),
    theTransverseAsymmetryTable(NULL)
{
  verboseLevel=0;
  SetProcessSubType(fIonisation);
  SetSecondaryParticle(theElectron);
  flucModel = 0;
  emModel = 0; 
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4ePolarizedIonisation::~G4ePolarizedIonisation()
{
  delete theAsymmetryTable;
  delete theTransverseAsymmetryTable;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4ePolarizedIonisation::InitialiseEnergyLossProcess(
		    const G4ParticleDefinition* part,
		    const G4ParticleDefinition* /*part2*/)
{
  if(!isInitialised) {

    if(part == G4Positron::Positron()) { isElectron = false; }

    if (!FluctModel()) { SetFluctModel(new G4UniversalFluctuation()); }
    flucModel = FluctModel();

    emModel = new  G4PolarizedMollerBhabhaModel();
    SetEmModel(emModel, 1);
    G4EmParameters* param = G4EmParameters::Instance();
    emModel->SetLowEnergyLimit(param->MinKinEnergy());
    emModel->SetHighEnergyLimit(param->MaxKinEnergy());
    AddEmModel(1, emModel, flucModel);

    isInitialised = true;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4ePolarizedIonisation::PrintInfo()
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ePolarizedIonisation::GetMeanFreePath(const G4Track& track,
						 G4double step,
						 G4ForceCondition* cond)
{
  // *** get unploarised mean free path from lambda table ***
  G4double mfp = G4VEnergyLossProcess::GetMeanFreePath(track, step, cond);


  // *** get asymmetry, if target is polarized ***
  G4VPhysicalVolume*  aPVolume  = track.GetVolume();
  G4LogicalVolume*    aLVolume  = aPVolume->GetLogicalVolume();

  G4PolarizationManager * polarizationManger = G4PolarizationManager::GetInstance();
  G4bool volumeIsPolarized = polarizationManger->IsPolarized(aLVolume);
  const G4StokesVector ePolarization = track.GetPolarization();

  if (mfp != DBL_MAX &&  volumeIsPolarized && !ePolarization.IsZero()) {
    const G4DynamicParticle* aDynamicElectron = track.GetDynamicParticle();
    G4double eEnergy = aDynamicElectron->GetKineticEnergy();
    const G4ParticleMomentum eDirection0 = aDynamicElectron->GetMomentumDirection();

    G4StokesVector volumePolarization = polarizationManger->GetVolumePolarization(aLVolume);

    G4bool isOutRange;
    size_t idx = CurrentMaterialCutsCoupleIndex();
    G4double lAsymmetry = (*theAsymmetryTable)(idx)->
                                  GetValue(eEnergy, isOutRange);
    G4double tAsymmetry = (*theTransverseAsymmetryTable)(idx)->
                                  GetValue(eEnergy, isOutRange);

    // calculate longitudinal spin component
    G4double polZZ = ePolarization.z()*
			volumePolarization*eDirection0;
    // calculate transvers spin components
    G4double polXX = ePolarization.x()*
			volumePolarization*G4PolarizationHelper::GetParticleFrameX(eDirection0);
    G4double polYY = ePolarization.y()*
			volumePolarization*G4PolarizationHelper::GetParticleFrameY(eDirection0);


    G4double impact = 1. + polZZ*lAsymmetry + (polXX + polYY)*tAsymmetry;
    // determine polarization dependent mean free path
    mfp /= impact;
    if (mfp <=0.) {
     G4cout <<"PV impact ( "<<polXX<<" , "<<polYY<<" , "<<polZZ<<" )"<<G4endl;
     G4cout << " impact on MFP is "<< impact <<G4endl;
     G4cout<<" lAsymmetry= "<<lAsymmetry<<" ("<<std::fabs(lAsymmetry)-1.<<")\n";
     G4cout<<" tAsymmetry= "<<tAsymmetry<<" ("<<std::fabs(tAsymmetry)-1.<<")\n";
    }
  }

  return mfp;
}

G4double G4ePolarizedIonisation::PostStepGetPhysicalInteractionLength(const G4Track& track,
                                              G4double step,
                                              G4ForceCondition* cond)
{
  // *** get unploarised mean free path from lambda table ***
  G4double mfp = G4VEnergyLossProcess::PostStepGetPhysicalInteractionLength(track, step, cond);


  // *** get asymmetry, if target is polarized ***
  G4VPhysicalVolume*  aPVolume  = track.GetVolume();
  G4LogicalVolume*    aLVolume  = aPVolume->GetLogicalVolume();

  G4PolarizationManager * polarizationManger = G4PolarizationManager::GetInstance();
  G4bool volumeIsPolarized = polarizationManger->IsPolarized(aLVolume);
  const G4StokesVector ePolarization = track.GetPolarization();

  if (mfp != DBL_MAX &&  volumeIsPolarized && !ePolarization.IsZero()) {
    const G4DynamicParticle* aDynamicElectron = track.GetDynamicParticle();
    G4double eEnergy = aDynamicElectron->GetKineticEnergy();
    const G4ParticleMomentum eDirection0 = aDynamicElectron->GetMomentumDirection();

    G4StokesVector volumePolarization = polarizationManger->GetVolumePolarization(aLVolume);

    size_t idx = CurrentMaterialCutsCoupleIndex();
    G4double lAsymmetry = (*theAsymmetryTable)(idx)->Value(eEnergy);
    G4double tAsymmetry = (*theTransverseAsymmetryTable)(idx)->Value(eEnergy);

    // calculate longitudinal spin component
    G4double polZZ = ePolarization.z()*
			volumePolarization*eDirection0;
    // calculate transvers spin components
    G4double polXX = ePolarization.x()*
			volumePolarization*G4PolarizationHelper::GetParticleFrameX(eDirection0);
    G4double polYY = ePolarization.y()*
			volumePolarization*G4PolarizationHelper::GetParticleFrameY(eDirection0);


    G4double impact = 1. + polZZ*lAsymmetry + (polXX + polYY)*tAsymmetry;
    // determine polarization dependent mean free path
    mfp /= impact;
    if (mfp <=0.) {
     G4cout <<"PV impact ( "<<polXX<<" , "<<polYY<<" , "<<polZZ<<" )"<<G4endl;
     G4cout << " impact on MFP is "<< impact <<G4endl;
     G4cout<<" lAsymmetry= "<<lAsymmetry<<" ("<<std::fabs(lAsymmetry)-1.<<")\n";
     G4cout<<" tAsymmetry= "<<tAsymmetry<<" ("<<std::fabs(tAsymmetry)-1.<<")\n";
    }
  }

  return mfp;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
void G4ePolarizedIonisation::BuildPhysicsTable(const G4ParticleDefinition& part)
{
  // *** build DEDX and (unpolarized) cross section tables
  G4VEnergyLossProcess::BuildPhysicsTable(part);
  //  G4PhysicsTable* pt =
  //  BuildDEDXTable();


  // *** build asymmetry-table
  if (theAsymmetryTable) {
    theAsymmetryTable->clearAndDestroy(); delete theAsymmetryTable;}
  if (theTransverseAsymmetryTable) {
    theTransverseAsymmetryTable->clearAndDestroy(); delete theTransverseAsymmetryTable;}

  const G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();
  size_t numOfCouples = theCoupleTable->GetTableSize();

  theAsymmetryTable = new G4PhysicsTable(numOfCouples);
  theTransverseAsymmetryTable = new G4PhysicsTable(numOfCouples);

  for (size_t j=0 ; j < numOfCouples; j++ ) {
    // get cut value
    const G4MaterialCutsCouple* couple = theCoupleTable->GetMaterialCutsCouple(j);

    G4double cut = (*theCoupleTable->GetEnergyCutsVector(1))[j];

    //create physics vectors then fill it (same parameters as lambda vector)
    G4PhysicsVector * ptrVectorA = LambdaPhysicsVector(couple,cut);
    G4PhysicsVector * ptrVectorB = LambdaPhysicsVector(couple,cut);
    size_t bins = ptrVectorA->GetVectorLength();

    for (size_t i = 0 ; i < bins ; i++ ) {
      G4double lowEdgeEnergy = ptrVectorA->Energy(i);
      G4double tasm=0.;
      G4double asym = ComputeAsymmetry(lowEdgeEnergy, couple, part, cut, tasm);
      ptrVectorA->PutValue(i,asym);
      ptrVectorB->PutValue(i,tasm);
    }
    theAsymmetryTable->insertAt( j , ptrVectorA ) ;
    theTransverseAsymmetryTable->insertAt( j , ptrVectorB ) ;
  }

}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4ePolarizedIonisation::ComputeAsymmetry(G4double energy,
					 const G4MaterialCutsCouple* couple,
					       const G4ParticleDefinition& aParticle,
					       G4double cut,
					       G4double & tAsymmetry)
{
  G4double lAsymmetry = 0.0;
  	   tAsymmetry = 0.0;
  if (isElectron) {lAsymmetry = tAsymmetry = -1.0;}

  // calculate polarized cross section
  theTargetPolarization=G4ThreeVector(0.,0.,1.);
  emModel->SetTargetPolarization(theTargetPolarization);
  emModel->SetBeamPolarization(theTargetPolarization);
  G4double sigma2=emModel->CrossSection(couple,&aParticle,energy,cut,energy);

  // calculate transversely polarized cross section
  theTargetPolarization=G4ThreeVector(1.,0.,0.);
  emModel->SetTargetPolarization(theTargetPolarization);
  emModel->SetBeamPolarization(theTargetPolarization);
  G4double sigma3=emModel->CrossSection(couple,&aParticle,energy,cut,energy);

  // calculate unpolarized cross section
  theTargetPolarization=G4ThreeVector();
  emModel->SetTargetPolarization(theTargetPolarization);
  emModel->SetBeamPolarization(theTargetPolarization);
  G4double sigma0=emModel->CrossSection(couple,&aParticle,energy,cut,energy);
  // determine assymmetries
  if (sigma0>0.) {
    lAsymmetry=sigma2/sigma0-1.;
    tAsymmetry=sigma3/sigma0-1.;
  }
  if (std::fabs(lAsymmetry)>1.) {
    G4cout<<" energy="<<energy<<"\n";
    G4cout<<"WARNING lAsymmetry= "<<lAsymmetry<<" ("<<std::fabs(lAsymmetry)-1.<<")\n";
  }
  if (std::fabs(tAsymmetry)>1.) {
    G4cout<<" energy="<<energy<<"\n";
    G4cout<<"WARNING tAsymmetry= "<<tAsymmetry<<" ("<<std::fabs(tAsymmetry)-1.<<")\n";
  }
//   else {
//     G4cout<<"        tAsymmetry= "<<tAsymmetry<<" ("<<std::fabs(tAsymmetry)-1.<<")\n";
//   }
  return lAsymmetry;
}


