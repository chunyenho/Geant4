#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4eplusPolarizedAnnihilation.cc 76472 2013-11-11 10:34:07Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4eplusPolarizedAnnihilation
//
// Author:        A. Schaelicke on base of Vladimir Ivanchenko / Michel Maire code
//
// Creation date: 02.07.2006
//
// Modifications:
// 26-07-06 modified cross section  (P. Starovoitov)
// 21-08-06 interface updated   (A. Schaelicke)
// 11-06-07, add PostStepGetPhysicalInteractionLength (A.Schalicke)
// 02-10-07, enable AtRest (V.Ivanchenko)
//
//
// Class Description:
//
// Polarized process of e+ annihilation into 2 gammas
//

//
// -------------------------------------------------------------------
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4eplusPolarizedAnnihilation.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4MaterialCutsCouple.hh"
#include "G4Gamma.hh"
#include "G4PhysicsVector.hh"
#include "G4PhysicsLogVector.hh"


#include "G4PolarizedAnnihilationModel.hh"
#include "G4PhysicsTableHelper.hh"
#include "G4ProductionCutsTable.hh"
#include "G4PolarizationManager.hh"
#include "G4PolarizationHelper.hh"
#include "G4StokesVector.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4eplusPolarizedAnnihilation::G4eplusPolarizedAnnihilation(const G4String& name)
  : G4VEmProcess(name), isInitialised(false),
    theAsymmetryTable(NULL),
    theTransverseAsymmetryTable(NULL)
{
  enableAtRestDoIt = true;
  SetProcessSubType(fAnnihilation);
  emModel = 0; 
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4eplusPolarizedAnnihilation::~G4eplusPolarizedAnnihilation()
{
  delete theAsymmetryTable;
  delete theTransverseAsymmetryTable;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eplusPolarizedAnnihilation::InitialiseProcess(const G4ParticleDefinition*)
{
  if(!isInitialised) {
    isInitialised = true;
    //    SetVerboseLevel(3);
    SetBuildTableFlag(true);
    SetStartFromNullFlag(false);
    SetSecondaryParticle(G4Gamma::Gamma());
    G4double emin = 0.1*keV;
    G4double emax = 100.*TeV;
    SetLambdaBinning(120);
    SetMinKinEnergy(emin);
    SetMaxKinEnergy(emax);
    emModel = new G4PolarizedAnnihilationModel();
    emModel->SetLowEnergyLimit(emin);
    emModel->SetHighEnergyLimit(emax);
    AddEmModel(1, emModel);
  }
}


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

  // for polarization

G4double G4eplusPolarizedAnnihilation::GetMeanFreePath(const G4Track& track,
                              G4double previousStepSize,
                              G4ForceCondition* condition)
{
  G4double mfp = G4VEmProcess::GetMeanFreePath(track, previousStepSize, condition);

  if (theAsymmetryTable) {

    G4Material*         aMaterial = track.GetMaterial();
    G4VPhysicalVolume*  aPVolume  = track.GetVolume();
    G4LogicalVolume*    aLVolume  = aPVolume->GetLogicalVolume();
    
    //   G4Material* bMaterial = aLVolume->GetMaterial();
    G4PolarizationManager * polarizationManger = G4PolarizationManager::GetInstance();
    
    const G4bool volumeIsPolarized = polarizationManger->IsPolarized(aLVolume);
    G4StokesVector electronPolarization = polarizationManger->GetVolumePolarization(aLVolume);

    if (!volumeIsPolarized || mfp == DBL_MAX) return mfp;
     
    // *** get asymmetry, if target is polarized ***
    const G4DynamicParticle* aDynamicPositron = track.GetDynamicParticle();
    const G4double positronEnergy = aDynamicPositron->GetKineticEnergy();
    const G4StokesVector positronPolarization = track.GetPolarization();
    const G4ParticleMomentum positronDirection0 = aDynamicPositron->GetMomentumDirection();

    if (verboseLevel>=2) {
      
      G4cout << " Mom " << positronDirection0  << G4endl;
      G4cout << " Polarization " << positronPolarization  << G4endl;
      G4cout << " MaterialPol. " << electronPolarization  << G4endl;
      G4cout << " Phys. Volume " << aPVolume->GetName() << G4endl;
      G4cout << " Log. Volume  " << aLVolume->GetName() << G4endl;
      G4cout << " Material     " << aMaterial          << G4endl;
    }
    
    G4bool isOutRange;
    G4int idx= CurrentMaterialCutsCoupleIndex();
    G4double lAsymmetry = (*theAsymmetryTable)(idx)->
                                  GetValue(positronEnergy, isOutRange);
    G4double tAsymmetry = (*theTransverseAsymmetryTable)(idx)->
                                  GetValue(positronEnergy, isOutRange);

    G4double polZZ = positronPolarization.z()*
      electronPolarization*positronDirection0;
    G4double polXX = positronPolarization.x()*
      electronPolarization*G4PolarizationHelper::GetParticleFrameX(positronDirection0);
    G4double polYY = positronPolarization.y()*
      electronPolarization*G4PolarizationHelper::GetParticleFrameY(positronDirection0);

    G4double impact = 1. + polZZ*lAsymmetry + (polXX + polYY)*tAsymmetry;

    mfp *= 1. / impact;

    if (verboseLevel>=2) {
      G4cout << " MeanFreePath:  " << mfp / mm << " mm " << G4endl;
      G4cout << " Asymmetry:     " << lAsymmetry << ", " << tAsymmetry  << G4endl;
      G4cout << " PolProduct:    " << polXX << ", " << polYY << ", " << polZZ << G4endl;
    }
  }
  
  return mfp;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4eplusPolarizedAnnihilation::PostStepGetPhysicalInteractionLength(
                              const G4Track& track,
                              G4double previousStepSize,
                              G4ForceCondition* condition)
{
  G4double mfp = G4VEmProcess::PostStepGetPhysicalInteractionLength(track, previousStepSize, condition);

  if (theAsymmetryTable) {

    G4Material*         aMaterial = track.GetMaterial();
    G4VPhysicalVolume*  aPVolume  = track.GetVolume();
    G4LogicalVolume*    aLVolume  = aPVolume->GetLogicalVolume();
    
    //   G4Material* bMaterial = aLVolume->GetMaterial();
    G4PolarizationManager * polarizationManger = G4PolarizationManager::GetInstance();
    
    const G4bool volumeIsPolarized = polarizationManger->IsPolarized(aLVolume);
    G4StokesVector electronPolarization = polarizationManger->GetVolumePolarization(aLVolume);

    if (!volumeIsPolarized || mfp == DBL_MAX) return mfp;
     
    // *** get asymmetry, if target is polarized ***
    const G4DynamicParticle* aDynamicPositron = track.GetDynamicParticle();
    const G4double positronEnergy = aDynamicPositron->GetKineticEnergy();
    const G4StokesVector positronPolarization = track.GetPolarization();
    const G4ParticleMomentum positronDirection0 = aDynamicPositron->GetMomentumDirection();

    if (verboseLevel>=2) {
      
      G4cout << " Mom " << positronDirection0  << G4endl;
      G4cout << " Polarization " << positronPolarization  << G4endl;
      G4cout << " MaterialPol. " << electronPolarization  << G4endl;
      G4cout << " Phys. Volume " << aPVolume->GetName() << G4endl;
      G4cout << " Log. Volume  " << aLVolume->GetName() << G4endl;
      G4cout << " Material     " << aMaterial          << G4endl;
    }
    
    G4bool isOutRange;
    G4int idx= CurrentMaterialCutsCoupleIndex();
    G4double lAsymmetry = (*theAsymmetryTable)(idx)->
                                  GetValue(positronEnergy, isOutRange);
    G4double tAsymmetry = (*theTransverseAsymmetryTable)(idx)->
                                  GetValue(positronEnergy, isOutRange);

    G4double polZZ = positronPolarization.z()*
      electronPolarization*positronDirection0;
    G4double polXX = positronPolarization.x()*
      electronPolarization*G4PolarizationHelper::GetParticleFrameX(positronDirection0);
    G4double polYY = positronPolarization.y()*
      electronPolarization*G4PolarizationHelper::GetParticleFrameY(positronDirection0);

    G4double impact = 1. + polZZ*lAsymmetry + (polXX + polYY)*tAsymmetry;

    mfp *= 1. / impact;

    if (verboseLevel>=2) {
      G4cout << " MeanFreePath:  " << mfp / mm << " mm " << G4endl;
      G4cout << " Asymmetry:     " << lAsymmetry << ", " << tAsymmetry  << G4endl;
      G4cout << " PolProduct:    " << polXX << ", " << polYY << ", " << polZZ << G4endl;
    }
  }
  
  return mfp;
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eplusPolarizedAnnihilation::BuildPhysicsTable(const G4ParticleDefinition& pd) 
{
  G4VEmProcess::BuildPhysicsTable(pd);
  BuildAsymmetryTable(pd);  
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eplusPolarizedAnnihilation::PreparePhysicsTable(const G4ParticleDefinition& pd)
{
  G4VEmProcess::PreparePhysicsTable(pd);
  theAsymmetryTable = G4PhysicsTableHelper::PreparePhysicsTable(theAsymmetryTable);
  theTransverseAsymmetryTable = G4PhysicsTableHelper::PreparePhysicsTable(theTransverseAsymmetryTable);
}

void G4eplusPolarizedAnnihilation::BuildAsymmetryTable(const G4ParticleDefinition& part)
{
  // Access to materials
  const G4ProductionCutsTable* theCoupleTable=
        G4ProductionCutsTable::GetProductionCutsTable();
  size_t numOfCouples = theCoupleTable->GetTableSize();
  G4cout<<" annih-numOfCouples="<<numOfCouples<<"\n";
  for(size_t i=0; i<numOfCouples; ++i) {
    G4cout<<"annih- "<<i<<"/"<<numOfCouples<<"\n";
    if (!theAsymmetryTable) break;
    G4cout<<"annih- "<<theAsymmetryTable->GetFlag(i)<<"\n";
    if (theAsymmetryTable->GetFlag(i)) {
     G4cout<<" building pol-annih ... \n";

      // create physics vector and fill it
      const G4MaterialCutsCouple* couple = theCoupleTable->GetMaterialCutsCouple(i);

      // use same parameters as for lambda
      G4PhysicsVector* aVector = LambdaPhysicsVector(couple);
      G4PhysicsVector* tVector = LambdaPhysicsVector(couple);

      for (G4int j = 0 ; j < LambdaBinning() ; ++j ) {
	G4double lowEdgeEnergy = aVector->GetLowEdgeEnergy(j);
	G4double tasm=0.;
	G4double asym = ComputeAsymmetry(lowEdgeEnergy, couple, part, 0., tasm);
	aVector->PutValue(j,asym);
	tVector->PutValue(j,tasm);
      }

      G4PhysicsTableHelper::SetPhysicsVector(theAsymmetryTable, i, aVector);
      G4PhysicsTableHelper::SetPhysicsVector(theTransverseAsymmetryTable, i, tVector);
    }
  }

}


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4double G4eplusPolarizedAnnihilation::ComputeAsymmetry(G4double energy,
			    const G4MaterialCutsCouple* couple,
			    const G4ParticleDefinition& aParticle,
			    G4double cut,
			    G4double &tAsymmetry)
{
 G4double lAsymmetry = 0.0;
 	  tAsymmetry = 0.0;

 // calculate polarized cross section
 theTargetPolarization=G4ThreeVector(0.,0.,1.);
 emModel->SetTargetPolarization(theTargetPolarization);
 emModel->SetBeamPolarization(theTargetPolarization);
 G4double sigma2=emModel->CrossSection(couple,&aParticle,energy,cut,energy);

 // calculate transversely polarized cross section
 theTargetPolarization=G4ThreeVector(1.,0.,0.);
 emModel->SetTargetPolarization(theTargetPolarization);
 emModel->SetBeamPolarization(theTargetPolarization);
 G4double sigma3=emModel->CrossSection(couple,&aParticle,energy,cut,energy);

 // calculate unpolarized cross section
 theTargetPolarization=G4ThreeVector();
 emModel->SetTargetPolarization(theTargetPolarization);
 emModel->SetBeamPolarization(theTargetPolarization);
 G4double sigma0=emModel->CrossSection(couple,&aParticle,energy,cut,energy);

 // determine assymmetries
  if (sigma0>0.) {
    lAsymmetry=sigma2/sigma0-1.;
    tAsymmetry=sigma3/sigma0-1.;
                 }
 return lAsymmetry;

}



//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4eplusPolarizedAnnihilation::PrintInfo()
{
  G4cout << "      Polarized model for annihilation into 2 photons"
         << G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4VParticleChange* G4eplusPolarizedAnnihilation::AtRestDoIt(const G4Track& aTrack,
                                                     const G4Step& )
//
// Performs the e+ e- annihilation when both particles are assumed at rest.
// It generates two back to back photons with energy = electron_mass.
// The angular distribution is isotropic.
// GEANT4 internal units
//
// Note : Effects due to binding of atomic electrons are negliged.
{
  fParticleChange.InitializeForPostStep(aTrack);

  fParticleChange.SetNumberOfSecondaries(2);

  G4double cosTeta = 2.*G4UniformRand()-1. , sinTeta = std::sqrt(1.-cosTeta*cosTeta);
  G4double phi     = twopi * G4UniformRand();
  G4ThreeVector direction (sinTeta*std::cos(phi), sinTeta*std::sin(phi), cosTeta);
  fParticleChange.AddSecondary( new G4DynamicParticle (G4Gamma::Gamma(),
                                            direction, electron_mass_c2) );
  fParticleChange.AddSecondary( new G4DynamicParticle (G4Gamma::Gamma(),
                                           -direction, electron_mass_c2) );
  // Kill the incident positron
  //
  fParticleChange.ProposeTrackStatus(fStopAndKill);
  return &fParticleChange;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
