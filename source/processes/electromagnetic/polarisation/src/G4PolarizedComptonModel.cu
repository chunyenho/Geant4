#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4PolarizedComptonModel.cc 82755 2014-07-08 14:07:29Z gcosmo $
//
// -------------------------------------------------------------------
//
// GEANT4 Class file
//
//
// File name:     G4PolarizedComptonModel
//
// Author:        Andreas Schaelicke
//
// Creation date: 01.05.2005
//
// Modifications:
// 18-07-06 use newly calculated cross sections (P. Starovoitov)
// 21-08-05 update interface (A. Schaelicke)
//
// Class Description:
//
// -------------------------------------------------------------------
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4PolarizedComptonModel.hh"
#include "G4PhysicalConstants.hh"
#include "G4Electron.hh"
#include "G4Gamma.hh"
#include "Randomize.hh"
#include "G4DataVector.hh"
#include "G4ParticleChangeForGamma.hh"


#include "G4StokesVector.hh"
#include "G4PolarizationManager.hh"
#include "G4PolarizationHelper.hh"
#include "G4PolarizedComptonCrossSection.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4PolarizedComptonModel::G4PolarizedComptonModel(const G4ParticleDefinition*,
						 const G4String& nam)
  : G4KleinNishinaCompton(0,nam),
    verboseLevel(0)
{
  crossSectionCalculator = new G4PolarizedComptonCrossSection();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

G4PolarizedComptonModel::~G4PolarizedComptonModel()
{
  delete crossSectionCalculator;
}

G4double G4PolarizedComptonModel::ComputeAsymmetryPerAtom
                       (G4double gammaEnergy, G4double /*Z*/)
 
{
  G4double asymmetry = 0.0 ;

  G4double k0 = gammaEnergy / electron_mass_c2 ;
  G4double k1 = 1 + 2*k0 ;

  asymmetry = -k0;
  asymmetry *= (k0 + 1.)*sqr(k1)*std::log(k1) - 2.*k0*(5.*sqr(k0) + 4.*k0 + 1.);
  asymmetry /= ((k0 - 2.)*k0  -2.)*sqr(k1)*std::log(k1) + 2.*k0*(k0*(k0 + 1.)*(k0 + 8.) + 2.);		

  // G4cout<<"energy = "<<GammaEnergy<<"  asymmetry = "<<asymmetry<<"\t\t GAM = "<<k0<<G4endl;
  if (asymmetry>1.) G4cout<<"ERROR in G4PolarizedComptonModel::ComputeAsymmetryPerAtom"<<G4endl;

  return asymmetry;
}


G4double G4PolarizedComptonModel::ComputeCrossSectionPerAtom(
                                const G4ParticleDefinition* pd,
                                      G4double kinEnergy, 
                                      G4double Z, 
                                      G4double A, 
                                      G4double cut,
                                      G4double emax)
{
  double xs = 
    G4KleinNishinaCompton::ComputeCrossSectionPerAtom(pd,kinEnergy,
						      Z,A,cut,emax);
  G4double polzz = theBeamPolarization.p3()*theTargetPolarization.z();
  if (polzz > 0.0) {
    G4double asym = ComputeAsymmetryPerAtom(kinEnergy, Z);  
    xs*=(1.+polzz*asym);
  }
  return xs;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void G4PolarizedComptonModel::SampleSecondaries(
                              std::vector<G4DynamicParticle*>* fvect,
                              const G4MaterialCutsCouple*,
			      const G4DynamicParticle* aDynamicGamma,
			      G4double, G4double)
{
  // do nothing below the threshold
  if(aDynamicGamma->GetKineticEnergy() <= LowEnergyLimit()) { return; }

  const G4Track * aTrack = fParticleChange->GetCurrentTrack();
  G4VPhysicalVolume*  aPVolume  = aTrack->GetVolume();
  G4LogicalVolume*    aLVolume  = aPVolume->GetLogicalVolume();

  if (verboseLevel >= 1) {
    G4cout<<"G4PolarizedComptonModel::SampleSecondaries in "
          <<  aLVolume->GetName() <<G4endl;
  }
  G4PolarizationManager * polarizationManager = 
    G4PolarizationManager::GetInstance();

  // obtain polarization of the beam
  theBeamPolarization =  aDynamicGamma->GetPolarization();
  theBeamPolarization.SetPhoton();

  // obtain polarization of the media
  G4bool targetIsPolarized = polarizationManager->IsPolarized(aLVolume);
  theTargetPolarization = 
    polarizationManager->GetVolumePolarization(aLVolume);

  // if beam is linear polarized or target is transversely polarized 
  // determine the angle to x-axis
  // (assumes same PRF as in the polarization definition)

  G4ThreeVector gamDirection0 = aDynamicGamma->GetMomentumDirection();

  // transfere theTargetPolarization 
  // into the gamma frame (problem electron is at rest)
  if (targetIsPolarized) {
    theTargetPolarization.rotateUz(gamDirection0);
  }
  // The scattered gamma energy is sampled according to 
  // Klein - Nishina formula.
  // The random number techniques of Butcher & Messel are used 
  // (Nuc Phys 20(1960),15).
  // Note : Effects due to binding of atomic electrons are negliged.
 
  G4double gamEnergy0 = aDynamicGamma->GetKineticEnergy();
  G4double E0_m = gamEnergy0 / electron_mass_c2 ;

  //
  // sample the energy rate of the scattered gamma 
  //

  G4double epsilon, epsilonsq, onecost, sint2, greject ;

  G4double eps0       = 1./(1. + 2.*E0_m);
  G4double epsilon0sq = eps0*eps0;
  G4double alpha1     = - std::log(eps0);
  G4double alpha2     = 0.5*(1.- epsilon0sq);

  G4double polarization = 
    theBeamPolarization.p3()*theTargetPolarization.p3();

  G4int nloop = 0;
  do {
    ++nloop;
    // false interaction if too many iterations
    if(nloop > 1000) { return; }

    if ( alpha1/(alpha1+alpha2) > G4UniformRand() ) {
      epsilon   = std::exp(-alpha1*G4UniformRand());   // epsilon0**r
      epsilonsq = epsilon*epsilon; 

    } else {
      epsilonsq = epsilon0sq + (1.- epsilon0sq)*G4UniformRand();
      epsilon   = std::sqrt(epsilonsq);
    }

    onecost = (1.- epsilon)/(epsilon*E0_m);
    sint2   = onecost*(2.-onecost);

    G4double gdiced = 2.*(1./epsilon+epsilon);
    G4double gdist  = 1./epsilon + epsilon - sint2 
      - polarization*(1./epsilon-epsilon)*(1.-onecost);

    greject = gdist/gdiced;

    if (greject>1) {
      G4cout<<"ERROR in PolarizedComptonScattering::PostStepDoIt\n"
	    <<" costh rejection does not work properly: "<<greject
	    <<G4endl;
    }
  } while (greject < G4UniformRand());
 
  //
  // scattered gamma angles. ( Z - axis along the parent gamma)
  //

  G4double cosTeta = 1. - onecost; 
  G4double sinTeta = std::sqrt (sint2);
  G4double Phi;
  do {
    ++nloop;
    // false interaction if too many iterations
    if(nloop > 1000) { return; }

    Phi = twopi * G4UniformRand();
    G4double gdiced = 1./epsilon + epsilon - sint2 
       + std::abs(theBeamPolarization.p3())*
       ( std::abs((1./epsilon-epsilon)*cosTeta*theTargetPolarization.p3())
	+(1.-epsilon)*sinTeta*(std::sqrt(sqr(theTargetPolarization.p1()) 
				    + sqr(theTargetPolarization.p2()))))
       +sint2*(std::sqrt(sqr(theBeamPolarization.p1()) + 
			 sqr(theBeamPolarization.p2())));

    G4double gdist = 1./epsilon + epsilon - sint2 
       + theBeamPolarization.p3()*
       ((1./epsilon-epsilon)*cosTeta*theTargetPolarization.p3()
	+(1.-epsilon)*sinTeta*(std::cos(Phi)*theTargetPolarization.p1()+
			       std::sin(Phi)*theTargetPolarization.p2()))
       -sint2*(std::cos(2.*Phi)*theBeamPolarization.p1()
	       +std::sin(2.*Phi)*theBeamPolarization.p2());
    greject = gdist/gdiced;

    if (greject>1.+1.e-10 || greject<0) {
      G4cout<<"ERROR in PolarizedComptonScattering::PostStepDoIt\n"
	    <<" phi rejection does not work properly: "<<greject<<G4endl;
    }
    if (greject<1.e-3) {
      G4cout<<"ERROR in PolarizedComptonScattering::PostStepDoIt\n"
	    <<" phi rejection does not work properly: "<<greject<<"\n";
      G4cout<<" greject="<<greject<<"  phi="<<Phi<<"   cost="<<cosTeta<<"\n";
      G4cout<<" gdiced="<<gdiced<<"   gdist="<<gdist<<"\n";
      G4cout<<" eps="<<epsilon<<"    1/eps="<<1./epsilon<<"\n";
    }
     
  } while (greject < G4UniformRand());
  G4double dirx = sinTeta*std::cos(Phi), diry = sinTeta*std::sin(Phi), 
    dirz = cosTeta;

  //
  // update G4VParticleChange for the scattered gamma
  //
   
  G4ThreeVector gamDirection1 ( dirx,diry,dirz );
  gamDirection1.rotateUz(gamDirection0);
  G4double gamEnergy1 = epsilon*gamEnergy0;

  G4double edep = 0.0;
  if(gamEnergy1 > lowestSecondaryEnergy) {
    fParticleChange->ProposeMomentumDirection(gamDirection1);
    fParticleChange->SetProposedKineticEnergy(gamEnergy1);
  } else { 
    fParticleChange->ProposeTrackStatus(fStopAndKill);
    fParticleChange->SetProposedKineticEnergy(0.0);
    edep = gamEnergy1;
  }
 
  // 
  // calculate Stokesvector of final state photon and electron
  //
  G4ThreeVector  nInteractionFrame = 
    G4PolarizationHelper::GetFrame(gamDirection1,gamDirection0);

  // transfere theBeamPolarization and theTargetPolarization 
  // into the interaction frame (note electron is in gamma frame)
  if (verboseLevel>=1) {
    G4cout << "========================================\n";
    G4cout << " nInteractionFrame = " <<nInteractionFrame<<"\n";
    G4cout << " GammaDirection0 = " <<gamDirection0<<"\n";
    G4cout << " gammaPolarization = " <<theBeamPolarization<<"\n";
    G4cout << " electronPolarization = " <<theTargetPolarization<<"\n";
  }

  theBeamPolarization.InvRotateAz(nInteractionFrame,gamDirection0);
  theTargetPolarization.InvRotateAz(nInteractionFrame,gamDirection0);

  if (verboseLevel>=1) {
    G4cout << "----------------------------------------\n";
    G4cout << " gammaPolarization = " <<theBeamPolarization<<"\n";
    G4cout << " electronPolarization = " <<theTargetPolarization<<"\n";
    G4cout << "----------------------------------------\n";
  }

  // initialize the polarization transfer matrix
  crossSectionCalculator->Initialize(epsilon,E0_m,0.,
				     theBeamPolarization,
				     theTargetPolarization,2);
  
  if(gamEnergy1 > lowestSecondaryEnergy) {
 
    // in interaction frame
    // calculate polarization transfer to the photon (in interaction plane)
    finalGammaPolarization = crossSectionCalculator->GetPol2();
    if (verboseLevel>=1) {
      G4cout << " gammaPolarization1 = " <<finalGammaPolarization<<"\n";
    }
    finalGammaPolarization.SetPhoton();

    // translate polarization into particle reference frame
    finalGammaPolarization.RotateAz(nInteractionFrame,gamDirection1);
    //store polarization vector
    fParticleChange->ProposePolarization(finalGammaPolarization);
    if (finalGammaPolarization.mag() > 1.+1.e-8){
      G4cout<<"ERROR in Polarizaed Compton Scattering !"<<G4endl;
      G4cout<<"Polarization of final photon more than 100%"<<G4endl;
      G4cout<<finalGammaPolarization<<" mag = "
	    <<finalGammaPolarization.mag()<<G4endl;
    }
    if (verboseLevel>=1) {
      G4cout << " gammaPolarization1 = " <<finalGammaPolarization<<"\n";
      G4cout << " GammaDirection1 = " <<gamDirection1<<"\n";
    }
  }

  //
  // kinematic of the scattered electron
  //
  G4double eKinEnergy = gamEnergy0 - gamEnergy1;

  if (eKinEnergy > lowestSecondaryEnergy) {
  
    G4ThreeVector eDirection = 
      gamEnergy0*gamDirection0 - gamEnergy1*gamDirection1;
    eDirection = eDirection.unit();

    finalElectronPolarization = crossSectionCalculator->GetPol3();
    if (verboseLevel>=1) {
      G4cout << " electronPolarization1 = " 
	     <<finalElectronPolarization<<"\n";
    }
    // transfer into particle reference frame
    finalElectronPolarization.RotateAz(nInteractionFrame,eDirection);
    if (verboseLevel>=1) {
      G4cout << " electronPolarization1 = " 
	     <<finalElectronPolarization<<"\n";
      G4cout << " ElecDirection = " <<eDirection<<"\n";
    }

    // create G4DynamicParticle object for the electron.
    G4DynamicParticle* aElectron = 
      new G4DynamicParticle(theElectron,eDirection,eKinEnergy);
    //store polarization vector
    if (finalElectronPolarization.mag() > 1.+1.e-8){
      G4cout<<"ERROR in Polarizaed Compton Scattering !"<<G4endl;
      G4cout<<"Polarization of final electron more than 100%"<<G4endl;
      G4cout<<finalElectronPolarization<<" mag = "
	    <<finalElectronPolarization.mag()<<G4endl;
    }
    aElectron->SetPolarization(finalElectronPolarization.p1(),
			       finalElectronPolarization.p2(),
			       finalElectronPolarization.p3());
    fvect->push_back(aElectron);
  } else {
    edep += eKinEnergy;  
  }
  // energy balance
  if(edep > 0.0) { 
    fParticleChange->ProposeLocalEnergyDeposit(edep);
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......


