#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4NuclideTable.cc
//
// Date:                10/10/13
// Author:              T.Koi
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// HISTORY
// Based on G4IsomerTable
////////////////////////////////////////////////////////////////////////////////
//
#include "G4NuclideTable.hh"

#include "G4ios.hh"
#include "globals.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include <iomanip>
#include <fstream>
#include <sstream>

const G4double G4NuclideTable::levelTolerance = 1.0*eV;
// const G4double G4NuclideTable::levelTolerance = 1.0e-3*eV;
//  torelance for excitation energy
  
 
///////////////////////////////////////////////////////////////////////////////
G4NuclideTable* G4NuclideTable::GetInstance() {
   static G4NuclideTable instance;
   return &instance;
}

///////////////////////////////////////////////////////////////////////////////
G4NuclideTable::G4NuclideTable()
  :G4VIsotopeTable("Isomer"),
   threshold_of_half_life(1000.0*ns),
   fUserDefinedList(NULL), 
   fIsotopeList(0) 
{
  //SetVerboseLevel(G4ParticleTable::GetParticleTable()->GetVerboseLevel());
  FillHardCodeList();
}

///////////////////////////////////////////////////////////////////////////////
G4NuclideTable::~G4NuclideTable()
{
  if (fIsotopeList!=0) {
    for (size_t i = 0 ; i<fIsotopeList->size(); i++) {
      delete (*fIsotopeList)[i];
    }
    fIsotopeList->clear();
    delete fIsotopeList;
    fIsotopeList = 0;
  }

  for ( std::map< G4int , std::multimap< G4double , G4IsotopeProperty* > >::iterator 
     it = map_pre_load_list.begin(); it != map_pre_load_list.end(); it++ ) {
     it->second.clear();
  }
  map_pre_load_list.clear();

  for ( std::map< G4int , std::multimap< G4double , G4IsotopeProperty* > >::iterator 
     it = map_hard_code_list.begin(); it != map_hard_code_list.end(); it++ ) {
     for ( std::multimap< G4double , G4IsotopeProperty* >::iterator 
        itt = it->second.begin(); itt != it->second.end(); itt++ ) {
        delete itt->second;
     }
     it->second.clear();
  }
  map_hard_code_list.clear();

  for ( std::map< G4int , std::multimap< G4double , G4IsotopeProperty* > >::iterator 
     it = map_full_list.begin(); it != map_full_list.end(); it++ ) {
     for ( std::multimap< G4double , G4IsotopeProperty* >::iterator 
        itt = it->second.begin(); itt != it->second.end(); itt++ ) {
        delete itt->second;
     }
     it->second.clear();
  }
  map_full_list.clear();
}

///////////////////////////////////////////////////////////////////////////////
//
G4IsotopeProperty* G4NuclideTable::GetIsotope(G4int Z, G4int A, G4double E)
{

   G4IsotopeProperty* fProperty = 0;
   G4int ionCode = 1000*Z + A;

   //Serching pre-load
   //Note: isomer level is properly set only for pre_load_list.
   if ( map_pre_load_list.find( ionCode ) !=  map_pre_load_list.end() ) {

     std::multimap< G4double , G4IsotopeProperty* >::iterator lower_bound_itr = 
       map_pre_load_list.find( ionCode ) -> second.lower_bound ( E - levelTolerance/2 );
     
     //std::multimap< G4double , G4IsotopeProperty* >::iterator upper_bound_itr = 
     //map_pre_load_list.find( ionCode ) -> second.upper_bound ( E );
     
     G4double levelE = DBL_MAX;
     if ( lower_bound_itr !=  map_pre_load_list.find( ionCode ) -> second.end() ) {
       levelE = lower_bound_itr->first;
       if ( levelE - levelTolerance/2 <= E && E < levelE + levelTolerance/2 ) {
         return lower_bound_itr->second; // found
       } 
     }
   }
     
   //Searching hard-code
   if ( map_hard_code_list.find( ionCode ) !=  map_hard_code_list.end() ) {
      std::multimap< G4double , G4IsotopeProperty* >::iterator lower_bound_itr = 
      map_hard_code_list.find( ionCode ) -> second.lower_bound ( E - levelTolerance/2 );

      //std::multimap< G4double , G4IsotopeProperty* >::iterator upper_bound_itr = 
      //map_pre_load_list.find( ionCode ) -> second.upper_bound ( E );
      
      G4double levelE = DBL_MAX;
      if ( lower_bound_itr !=  map_hard_code_list.find( ionCode ) -> second.end() ) {
         levelE = lower_bound_itr->first;
	 if ( levelE - levelTolerance/2 <= E && E < levelE + levelTolerance/2 ) {
	   return lower_bound_itr->second; // found
	 }
      }
   }

   //Searching big-list 
   char* path = getenv("G4ENSDFSTATEDATA");

   if ( !path ) {
      return fProperty; // not found;
   }

   if ( map_full_list.find( ionCode ) ==  map_full_list.end() ) {

      std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
      map_full_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );

      std::fstream ifs;
      G4String filename(path);
      filename += "/ENSDFSTATE.dat";
      ifs.open( filename.c_str() );

      G4bool reading_target = false; 

      G4int ionZ;
      G4int ionA;
      G4double ionE;
      G4double ionLife;
      G4int ionJ;
      G4double ionMu;
      
      ifs >> ionZ >> ionA >> ionE >> ionLife >> ionJ >> ionMu;

      while ( ifs.good() ) {

         if ( ionZ == Z && ionA == A ) {

            reading_target = true;

            ionE *= keV;
            ionLife *= ns;
            ionMu *= (joule/tesla);

            G4IsotopeProperty* property = new G4IsotopeProperty(); 

            G4int iLevel=9;
            property->SetAtomicNumber(ionZ);
            property->SetAtomicMass(ionA);
            property->SetIsomerLevel(iLevel);
            property->SetEnergy(ionE);
            property->SetiSpin(ionJ);
            property->SetLifeTime(ionLife);
            property->SetMagneticMoment(ionMu);
       
            map_full_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , property ) );

         } else if ( reading_target == true ) {
            ifs.close();
            break;
         }
         
         ifs >> ionZ >> ionA >> ionE >> ionLife >> ionJ >> ionMu;
      }

      ifs.close();
   }


   if ( map_full_list.find( ionCode ) !=  map_full_list.end() ) {

      std::multimap< G4double , G4IsotopeProperty* >::iterator lower_bound_itr = 
      map_full_list.find( ionCode ) -> second.lower_bound ( E - levelTolerance/2 );

      //std::multimap< G4double , G4IsotopeProperty* >::iterator upper_bound_itr = 
      //map_full_list.find( ionCode ) -> second.upper_bound ( E - levelTolerance/2 );
      
      G4double levelE = DBL_MAX;
      if ( lower_bound_itr !=  map_full_list.find( ionCode ) -> second.end() ) {
         levelE = lower_bound_itr->first;
	 if ( levelE - levelTolerance/2 < E && E < levelE + levelTolerance/2 ) {
	   return lower_bound_itr->second; // found
	 }
      }
   }

   return fProperty; // not found;
}

///////////////////////////////////////////////////////////////////////
G4IsotopeProperty* 
 G4NuclideTable::GetIsotopeByIsoLvl(G4int Z, G4int A, G4int lvl)
{
  if(lvl==0) return GetIsotope(Z,A,0.0);
  return (G4IsotopeProperty*)0;
}

///////////////////////////////////////////////////////////////////////////////
void G4NuclideTable::FillHardCodeList()
{
   for (size_t i=0; i<nEntries_ground_state; i++) {

      G4int    ionZ     = (G4int)groundStateTable[i][idxZ];
      G4int    ionA     = (G4int)groundStateTable[i][idxA];
      G4int    lvl      = 0; // ground state
      G4double ionE     = groundStateTable[i][idxEnergy]*keV;
      G4double ionLife  = groundStateTable[i][idxLife]*ns;
      G4int    ionJ     = (G4int)(groundStateTable[i][idxSpin]);
      G4double ionMu    = groundStateTable[i][idxMu]*(joule/tesla);

      G4int ionCode = 1000*ionZ + ionA;

      G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

      // Set Isotope Property
      fProperty->SetAtomicNumber(ionZ);
      fProperty->SetAtomicMass(ionA);
      fProperty->SetIsomerLevel(lvl);
      fProperty->SetEnergy(ionE);
      fProperty->SetiSpin(ionJ);
      fProperty->SetLifeTime(ionLife);
      fProperty->SetDecayTable(0);
      fProperty->SetMagneticMoment(ionMu);

      if ( map_hard_code_list.find ( ionCode ) == map_hard_code_list.end() ) {
         std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
         map_hard_code_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );
      }
      map_hard_code_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , fProperty ) );

   }

   for (size_t i=0; i<nEntries_excite_state; i++) {

      G4int    ionZ     = (G4int)exciteStateTable[i][idxZ];
      G4int    ionA     = (G4int)exciteStateTable[i][idxA];
      G4double ionE     = exciteStateTable[i][idxEnergy]*keV;
      G4double ionLife  = exciteStateTable[i][idxLife]*ns;
      G4int    ionJ     = (G4int)(exciteStateTable[i][idxSpin]);
      G4double ionMu    = exciteStateTable[i][idxMu]*(joule/tesla);

      G4int ionCode = 1000*ionZ + ionA;

      G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

      // Set Isotope Property
      fProperty->SetAtomicNumber(ionZ);
      fProperty->SetAtomicMass(ionA);
      fProperty->SetIsomerLevel(9);
      fProperty->SetEnergy(ionE);
      fProperty->SetiSpin(ionJ);
      fProperty->SetLifeTime(ionLife);
      fProperty->SetDecayTable(0);
      fProperty->SetMagneticMoment(ionMu);

      if ( map_hard_code_list.find ( ionCode ) == map_hard_code_list.end() ) {
         std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
         map_hard_code_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );
      }
      map_hard_code_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , fProperty ) );

   }
}

///////////////////////////////////////////////////////////////////////////////
void G4NuclideTable::GenerateNuclide()
{

   if( fIsotopeList !=0 ) return;
   fIsotopeList = new G4IsotopeList();

   for (size_t i=0; i<nEntries_ground_state; i++) {

      G4int    ionZ     = (G4int)groundStateTable[i][idxZ];
      G4int    ionA     = (G4int)groundStateTable[i][idxA];
      G4int    lvl      = 0; // ground state
      G4double ionE     = groundStateTable[i][idxEnergy]*keV;
      G4double ionLife  = groundStateTable[i][idxLife]*ns;
      G4int    ionJ     = (G4int)(groundStateTable[i][idxSpin]);
      G4double ionMu    = groundStateTable[i][idxMu]*(joule/tesla);

      if ( ionLife < 0.0 || ionLife*std::log(2.0) > threshold_of_half_life ) {

         G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

         // Set Isotope Property
         fProperty->SetAtomicNumber(ionZ);
         fProperty->SetAtomicMass(ionA);
         fProperty->SetIsomerLevel(lvl);
         fProperty->SetEnergy(ionE);
         fProperty->SetiSpin(ionJ);
         fProperty->SetLifeTime(ionLife);
         fProperty->SetDecayTable(0);
         fProperty->SetMagneticMoment(ionMu);
    
         //G4cout << ionZ << " " << ionA << " " << lvl << " " << ionE/keV << " [keV]" << G4endl;
         fIsotopeList->push_back(fProperty);

         G4int ionCode = 1000*ionZ + ionA;
         if ( map_pre_load_list.find ( ionCode ) == map_pre_load_list.end() ) {
            std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
            map_pre_load_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );
         }
         map_pre_load_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , fProperty ) );

      }
   }

   if ( threshold_of_half_life >= 1.0*ns ) {

      G4int ionCode=0;
      G4int iLevel=0;
      G4double previousE=0.0;
      
      for (size_t i=0; i<nEntries_excite_state; i++) {

         G4int    ionZ     = (G4int)exciteStateTable[i][idxZ];
         G4int    ionA     = (G4int)exciteStateTable[i][idxA];
         if ( ionCode != 1000*ionZ + ionA ) {
            previousE=0.0;
            iLevel = 0;
            ionCode = 1000*ionZ + ionA;
         } 

         G4double ionE     = exciteStateTable[i][idxEnergy]*keV;
         G4double ionLife  = exciteStateTable[i][idxLife]*ns;
         G4int    ionJ     = (G4int)(exciteStateTable[i][idxSpin]);
         G4double ionMu    = exciteStateTable[i][idxMu]*(joule/tesla);

         if (( ionLife < 0.0 || ionLife*ionLife*std::log(2.0) > threshold_of_half_life )
           && (ionE > levelTolerance+previousE)) {
            previousE = ionE;
            iLevel++;
            if ( iLevel > 9 ) iLevel=9;
         //G4cout << ionZ << " " << ionA << " " << iLevel << " " << ionE/keV << " [keV]" << G4endl;

            G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

            // Set Isotope Property
            fProperty->SetAtomicNumber(ionZ);
            fProperty->SetAtomicMass(ionA);
            fProperty->SetIsomerLevel(iLevel);
            fProperty->SetEnergy(ionE);
            fProperty->SetiSpin(ionJ);
            fProperty->SetLifeTime(ionLife);
            fProperty->SetDecayTable(0);
            fProperty->SetMagneticMoment(ionMu);
       
            fIsotopeList->push_back(fProperty);

            if ( map_pre_load_list.find ( ionCode ) == map_pre_load_list.end() ) {
               std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
               map_pre_load_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );
            }
            map_pre_load_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , fProperty ) );

         }
      }
   } else {

      char* path = getenv("G4ENSDFSTATEDATA");

      if ( !path ) {
         G4Exception("G4NuclideTable", "PART70000",
                  FatalException, "G4ENSDFSTATEDATA environment variable must be set");
	 return;
      }
   
      std::fstream ifs;
      G4String filename(path);
      filename += "/ENSDFSTATE.dat";

      ifs.open( filename.c_str() );
     
      if ( !ifs.good() ) {
         G4Exception("G4NuclideTable", "PART70001",
                  FatalException, "ENSDFSTATE.dat is not found.");
	 return;
      }
     

      G4int ionCode=0;
      G4int iLevel=0;

      G4int ionZ;
      G4int ionA;
      G4double ionE;
      G4double ionLife;
      G4int ionJ;
      G4double ionMu;
      
      ifs >> ionZ >> ionA >> ionE >> ionLife >> ionJ >> ionMu;

      while ( ifs.good() ) {

         if ( ionCode != 1000*ionZ + ionA ) {
            iLevel = 0;
            ionCode = 1000*ionZ + ionA;
         } 

         ionE *= keV;
         ionLife *= ns;
         ionMu *= (joule/tesla);

         //if ( ionLife == -1 || ionLife > threshold_of_half_life ) {
         if ( ionLife*std::log(2.0) > threshold_of_half_life && ionE != 0 ) {

            iLevel++;
            if ( iLevel > 9 ) iLevel=9;
            //G4cout << ionZ << " " << ionA << " " << iLevel << " " << ionE/keV << " [keV]" << G4endl;

            G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

            // Set Isotope Property
            fProperty->SetAtomicNumber(ionZ);
            fProperty->SetAtomicMass(ionA);
            fProperty->SetIsomerLevel(iLevel);
            fProperty->SetEnergy(ionE);
            fProperty->SetiSpin(ionJ);
            fProperty->SetLifeTime(ionLife);
            fProperty->SetDecayTable(0);
            fProperty->SetMagneticMoment(ionMu);
       
            fIsotopeList->push_back(fProperty);

            if ( map_pre_load_list.find ( ionCode ) == map_pre_load_list.end() ) {
               std::multimap<G4double, G4IsotopeProperty*> aMultiMap;
               map_pre_load_list.insert( std::pair< G4int , std::multimap< G4double , G4IsotopeProperty* > > ( ionCode , aMultiMap ) );
            }
            map_pre_load_list.find ( ionCode ) -> second.insert( std::pair< G4double, G4IsotopeProperty* >( ionE , fProperty ) );

         }

         ifs >> ionZ >> ionA >> ionE >> ionLife >> ionJ >> ionMu;
      }

   }

   if ( fUserDefinedList != NULL ) {
      for ( G4IsotopeList::iterator it = fUserDefinedList->begin() ; it != fUserDefinedList->end() ; it++ ) {
         fIsotopeList->push_back( *it );
      }
   }

}

void G4NuclideTable::AddState( G4int ionZ, G4int ionA, G4double ionE, G4double ionLife, G4int ionJ=0, G4double ionMu=0.0)
{
   if ( fUserDefinedList == NULL ) fUserDefinedList = new G4IsotopeList();

            G4IsotopeProperty* fProperty = new G4IsotopeProperty(); 

            // Set Isotope Property
            fProperty->SetAtomicNumber(ionZ);
            fProperty->SetAtomicMass(ionA);
            fProperty->SetIsomerLevel(9);
            fProperty->SetEnergy(ionE);
            fProperty->SetiSpin(ionJ);
            fProperty->SetLifeTime(ionLife);
            fProperty->SetDecayTable(0);
            fProperty->SetMagneticMoment(ionMu);
       
            fUserDefinedList->push_back(fProperty);

}

