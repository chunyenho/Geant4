#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// ------------------------------------------------------------
//      GEANT 4 class header file
//
//      History:
//               01 August 2007 P.Gumplinger
//               Reference: TRIUMF PIENU Technote:
//                          M. Blecher - "Inclusion of pi->enug in MC "
//                              Rate is for gammas > 100keV
//
// ------------------------------------------------------------
//
//
//

#include "G4PionRadiativeDecayChannel.hh"

#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"
#include "G4DecayProducts.hh"
#include "G4LorentzVector.hh"

G4PionRadiativeDecayChannel::G4PionRadiativeDecayChannel()
  : G4VDecayChannel(),
    beta(0.),cib(0.),csdp(0.),csdm(0.),cif(0.),cig(0.),
    xl(0.), yl(0.), xu(0.), yu(0.), d2wmax(0.)
{
}

G4PionRadiativeDecayChannel::
           G4PionRadiativeDecayChannel(const G4String& theParentName,
                                       G4double        theBR)
                            : G4VDecayChannel("Radiative Pion Decay",1)
{
  // set names for daughter particles
  if (theParentName == "pi+") {
    SetBR(theBR);
    SetParent("pi+");
    SetNumberOfDaughters(3);
    SetDaughter(0, "e+");
    SetDaughter(1, "gamma");
    SetDaughter(2, "nu_e");
  } else if (theParentName == "pi-") {
    SetBR(theBR);
    SetParent("pi-");
    SetNumberOfDaughters(3);
    SetDaughter(0, "e-");
    SetDaughter(1, "gamma");
    SetDaughter(2, "anti_nu_e");
  } else {
#ifdef G4VERBOSE
    if (GetVerboseLevel()>0) {
      G4cout << "G4RadiativePionDecayChannel:: constructor :";
      G4cout << " parent particle is not charged pion but ";
      G4cout << theParentName << G4endl;
    }
#endif
  }

  beta = 3.6612e-03;

  cib  = 1.16141e-03;
  csdp = 3.45055e-02;
  csdm = 5.14122e-03;
  cif  = 4.63543e-05;
  cig  = 1.78928e-05;

  xl = 2.*0.1*MeV/139.57*MeV;
  yl = ((1.-xl) + std::sqrt((1-xl)*(1-xl)+4*beta*beta))/2.;

  xu = 1. - (yl - std::sqrt(yl*yl-4.*beta*beta))/2.;
  yu = 1. + beta*beta;

  d2wmax = D2W(xl,yl);

}

G4PionRadiativeDecayChannel::~G4PionRadiativeDecayChannel()
{
}
G4PionRadiativeDecayChannel::G4PionRadiativeDecayChannel(const G4PionRadiativeDecayChannel &right)
  :G4VDecayChannel(right),
   beta(right.beta),cib(right.cib),csdp(right.csdp),
   csdm(right.csdm),cif(right.cif),cig(right.cig),
   xl(right.xl), yl(right.yl), xu(right.xu), yu(right.yu), 
   d2wmax(right.d2wmax)
{
}

G4PionRadiativeDecayChannel & G4PionRadiativeDecayChannel::operator=(const G4PionRadiativeDecayChannel & right)
{
  if (this != &right) { 
    kinematics_name = right.kinematics_name;
    verboseLevel = right.verboseLevel;
    rbranch = right.rbranch;

    // copy parent name
    parent_name = new G4String(*right.parent_name);

    // clear daughters_name array
    ClearDaughtersName();

    // recreate array
    numberOfDaughters = right.numberOfDaughters;
    if ( numberOfDaughters >0 ) {
      if (daughters_name !=0) ClearDaughtersName();
      daughters_name = new G4String*[numberOfDaughters];
      //copy daughters name
      for (G4int index=0; index < numberOfDaughters; index++) {
          daughters_name[index] = new G4String(*right.daughters_name[index]);
      }
    }
    beta = right.beta;
    cib  = right.cib;
    csdp = right.csdp;
    csdm = right.csdm;
    cif  = right.cif;
    cig  = right.cig;
    xl   = right.xl;
    yl   = right.yl;
    xu   = right.xu;
    yu   = right.yu; 
    d2wmax = right.d2wmax;
  }
  return *this;
}

G4DecayProducts *G4PionRadiativeDecayChannel::DecayIt(G4double) 
{

#ifdef G4VERBOSE
  if (GetVerboseLevel()>1) 
                 G4cout << "G4PionRadiativeDecayChannel::DecayIt ";
#endif

  if (G4MT_parent == 0) FillParent();  
  if (G4MT_daughters == 0) FillDaughters();

  // parent mass
  G4double parentmass = G4MT_parent->GetPDGMass();

  G4double EMPI = parentmass;

  //daughters'mass
  G4double daughtermass[3]; 
  G4double sumofdaughtermass = 0.0;
  for (G4int index=0; index<3; index++){
    daughtermass[index] = G4MT_daughters[index]->GetPDGMass();
    sumofdaughtermass += daughtermass[index];
  }

  G4double EMASS = daughtermass[0];

  //create parent G4DynamicParticle at rest
  G4ThreeVector dummy;
  G4DynamicParticle * parentparticle = 
                               new G4DynamicParticle( G4MT_parent, dummy, 0.0);
  //create G4Decayproducts
  G4DecayProducts *products = new G4DecayProducts(*parentparticle);
  delete parentparticle;

  G4double x, y, d2w;

  do {

     do {

        x = xl + G4UniformRand()*(xu-xl);
        y = yl + G4UniformRand()*(yu-yl);

     } while (x+y <= 1.);

     d2w = D2W(x,y);

  } while (d2w <= G4UniformRand()*d2wmax);

//-----------------------------------------------------------------------
//
//      Calculate the angle between positron and photon (cosine)
//
  G4double cthetaGE =  (y*(x-2.)+2.*(1.-x+beta*beta)) /
                       (x*std::sqrt(y*y-4.*beta*beta));

//
//-----------------------------------------------------------------------
//
  G4double G = x * EMPI/2.;
  G4double E = y * EMPI/2.;
//
//-----------------------------------------------------------------------
//

  if (E < EMASS) E = EMASS;

  // calculate daughter momentum
  G4double daughtermomentum[2];

  daughtermomentum[0] = std::sqrt(E*E - EMASS*EMASS);

  G4double cthetaE = 2.*G4UniformRand()-1.;
  G4double sthetaE = std::sqrt(1.-cthetaE*cthetaE);

  G4double phiE = twopi*G4UniformRand()*rad;
  G4double cphiE = std::cos(phiE);
  G4double sphiE = std::sin(phiE);

  //Coordinates of the decay positron

  G4double px = sthetaE*cphiE;
  G4double py = sthetaE*sphiE;
  G4double pz = cthetaE;

  G4ThreeVector direction0(px,py,pz);

  G4DynamicParticle * daughterparticle0 
    = new G4DynamicParticle( G4MT_daughters[0], daughtermomentum[0]*direction0);

  products->PushProducts(daughterparticle0);

  daughtermomentum[1] = G;

  G4double sthetaGE = std::sqrt(1.-cthetaGE*cthetaGE);

  G4double phiGE = twopi*G4UniformRand()*rad;
  G4double cphiGE = std::cos(phiGE);
  G4double sphiGE = std::sin(phiGE);

  //Coordinates of the decay gamma with respect to the decay positron

  px = sthetaGE*cphiGE;
  py = sthetaGE*sphiGE;
  pz = cthetaGE;

  G4ThreeVector direction1(px,py,pz);

  direction1.rotateUz(direction0);

  G4DynamicParticle * daughterparticle1
    = new G4DynamicParticle( G4MT_daughters[1], daughtermomentum[1]*direction1);

  products->PushProducts(daughterparticle1);

// output message
#ifdef G4VERBOSE
  if (GetVerboseLevel()>1) {
    G4cout << "G4PionRadiativeDecayChannel::DecayIt ";
    G4cout << "  create decay products in rest frame " <<G4endl;
    products->DumpInfo();
  }
#endif

  return products;

}
