#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: G4MaterialPropertiesTable.cc 80573 2014-04-29 07:40:08Z gcosmo $
//
// 
////////////////////////////////////////////////////////////////////////
// G4MaterialPropertiesTable Implementation
////////////////////////////////////////////////////////////////////////
//
// File: G4MaterialPropertiesTable.cc 
// Version:     1.0
// Created:     1996-02-08
// Author:      Juliet Armstrong
// Updated:     2005-05-12 add SetGROUPVEL(), courtesy of
//              Horton-Smith (bug report #741), by P. Gumplinger
//              2002-11-05 add named material constants by P. Gumplinger
//              1999-11-05 Migration from G4RWTPtrHashDictionary to STL
//                         by John Allison
//              1997-03-26 by Peter Gumplinger
//              > cosmetics (only)
// mail:        gum@triumf.ca
//
////////////////////////////////////////////////////////////////////////

#include "globals.hh"
#include "G4MaterialPropertiesTable.hh"
#include "G4PhysicalConstants.hh"

/////////////////
// Constructors
/////////////////

G4MaterialPropertiesTable::G4MaterialPropertiesTable()
{
}

////////////////
// Destructor
////////////////

G4MaterialPropertiesTable::~G4MaterialPropertiesTable()
{
  MPTiterator i;
  for (i = MPT.begin(); i != MPT.end(); ++i)
  {
    delete (*i).second;
  }
  MPT.clear();
  MPTC.clear();
}

////////////
// Methods
////////////

void G4MaterialPropertiesTable::DumpTable()
{
  MPTiterator i;
  for (i = MPT.begin(); i != MPT.end(); ++i)
  {
    G4cout << (*i).first << G4endl;
    if ( (*i).second != 0 )
    {
      (*i).second->DumpValues();
    }
    else
    {
      G4Exception("G4MaterialPropertiesTable::DumpTable()", "mat204",
                  JustWarning, "NULL Material Property Vector Pointer.");
    }
  }
  MPTCiterator j;
  for (j = MPTC.begin(); j != MPTC.end(); ++j)
  {
    G4cout << j->first << G4endl;
    if ( j->second != 0 )
    {
      G4cout << j->second << G4endl;
    }
    else
    {
      G4Exception("G4MaterialPropertiesTable::DumpTable()", "mat202",
                  JustWarning, "No Material Constant Property.");
    }
  }
}

#ifdef G4MULTITHREADED
#include "G4AutoLock.hh"
namespace {
 G4Mutex materialPropertyTableMutex = G4MUTEX_INITIALIZER;
}
#endif

G4MaterialPropertyVector* G4MaterialPropertiesTable::SetGROUPVEL()
{
#ifdef G4MULTITHREADED
  G4AutoLock mptm(&materialPropertyTableMutex);
#endif

  // check if "GROUPVEL" already exists
  MPTiterator itr;
  itr = MPT.find("GROUPVEL");
  if(itr != MPT.end()) return itr->second;

  // fetch RINDEX data, give up if unavailable
  //
  G4MaterialPropertyVector *rindex = this->GetProperty("RINDEX");
  if (rindex==0)  { return 0; }

  // RINDEX exists but has no entries, give up
  //
  if ( rindex->GetVectorLength() == 0 ) { return 0; }

  // add GROUPVEL vector
  //
  G4MaterialPropertyVector* groupvel = new G4MaterialPropertyVector();

  // fill GROUPVEL vector using RINDEX values
  // rindex built-in "iterator" was advanced to first entry above
  //
  G4double E0 = rindex->Energy(0);
  G4double n0 = (*rindex)[0];

  if (E0 <= 0.)
  {
    G4Exception("G4MaterialPropertiesTable::SetGROUPVEL()", "mat205",
                FatalException, "Optical Photon Energy <= 0");
  }
                                                                                
  if ( rindex->GetVectorLength() >= 2 )
  {
    // good, we have at least two entries in RINDEX
    // get next energy/value pair

    G4double E1 = rindex->Energy(1);
    G4double n1 = (*rindex)[1];

    if (E1 <= 0.)
    {
      G4Exception("G4MaterialPropertiesTable::SetGROUPVEL()", "mat205",
                  FatalException, "Optical Photon Energy <= 0");
    }

    G4double vg;

    // add entry at first photon energy
    //
    vg = c_light/(n0+(n1-n0)/std::log(E1/E0));

    // allow only for 'normal dispersion' -> dn/d(logE) > 0
    //
    if((vg<0) || (vg>c_light/n0))  { vg = c_light/n0; }

    groupvel->InsertValues( E0, vg );

    // add entries at midpoints between remaining photon energies
    //

    for (size_t i = 2; i < rindex->GetVectorLength(); i++)
    {
      vg = c_light/( 0.5*(n0+n1)+(n1-n0)/std::log(E1/E0));

      // allow only for 'normal dispersion' -> dn/d(logE) > 0
      //
      if((vg<0) || (vg>c_light/(0.5*(n0+n1))))  { vg = c_light/(0.5*(n0+n1)); }
      groupvel->InsertValues( 0.5*(E0+E1), vg );

      // get next energy/value pair, or exit loop
      //
      E0 = E1;
      n0 = n1;
      E1 = rindex->Energy(i);
      n1 = (*rindex)[i];

      if (E1 <= 0.)
      {
        G4Exception("G4MaterialPropertiesTable::SetGROUPVEL()", "mat205",
                    FatalException, "Optical Photon Energy <= 0");
      }
    }

    // add entry at last photon energy
    //
    vg = c_light/(n1+(n1-n0)/std::log(E1/E0));

    // allow only for 'normal dispersion' -> dn/d(logE) > 0
    //
    if((vg<0) || (vg>c_light/n1))  { vg = c_light/n1; }
    groupvel->InsertValues( E1, vg );
  }
  else // only one entry in RINDEX -- weird!
  {
    groupvel->InsertValues( E0, c_light/n0 );
  }
                                                                                
  this->AddProperty( "GROUPVEL", groupvel );
                                                                                
  return groupvel;
}
