//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo...... 

#include "G4RadioactiveDecayPhysics.hh"

#include "G4RadioactiveDecay.hh"
#include "G4GenericIon.hh"
#include "globals.hh"
#include "G4PhysicsListHelper.hh"

// factory
#include "G4PhysicsConstructorFactory.hh"
//
G4_DECLARE_PHYSCONSTR_FACTORY(G4RadioactiveDecayPhysics);


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4RadioactiveDecayPhysics::G4RadioactiveDecayPhysics(G4int)
:  G4VPhysicsConstructor("G4RadioactiveDecay")//, theRadioactiveDecay(0)
{}

G4RadioactiveDecayPhysics::G4RadioactiveDecayPhysics(const G4String& name)
:  G4VPhysicsConstructor(name)//, theRadioactiveDecay(0)
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4RadioactiveDecayPhysics::~G4RadioactiveDecayPhysics()
{
  //delete theRadioactiveDecay;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4RadioactiveDecayPhysics::ConstructParticle()
{
  G4GenericIon::GenericIon();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4RadioactiveDecayPhysics::ConstructProcess()
{
  G4PhysicsListHelper::GetPhysicsListHelper()->
    RegisterProcess(new G4RadioactiveDecay(), G4GenericIon::GenericIon());
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

