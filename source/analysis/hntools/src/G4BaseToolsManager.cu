#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$

// Author: Ivana Hrivnacova, 24/07/2014  (ivana@ipno.in2p3.fr)

#include "G4BaseToolsManager.hh"

#include "tools/histo/axis"


// static data

const G4int G4BaseToolsManager::kX = 0;
const G4int G4BaseToolsManager::kY = 1;
const G4int G4BaseToolsManager::kZ = 2;

//
// Constructors, destructor
//

//_____________________________________________________________________________
G4BaseToolsManager::G4BaseToolsManager(const G4String& hnType)
 : fHnType(hnType)
{
}

//_____________________________________________________________________________
G4BaseToolsManager::~G4BaseToolsManager()
{  
}

//_____________________________________________________________________________
G4int G4BaseToolsManager::GetNbins(const G4ToolsBaseHisto& baseHisto, 
                                   G4int dimension) const
{
  return baseHisto.get_axis(dimension).bins();
}  

//_____________________________________________________________________________
G4double G4BaseToolsManager::GetMin(const G4ToolsBaseHisto& baseHisto, 
                                    G4int dimension) const
{
// Returns min data value

  return baseHisto.get_axis(dimension).lower_edge();
}  

//_____________________________________________________________________________
G4double G4BaseToolsManager::GetMax(const G4ToolsBaseHisto& baseHisto, 
                                    G4int dimension) const
{
// Returns max data value

  return baseHisto.get_axis(dimension).upper_edge();
}  

//_____________________________________________________________________________
G4double G4BaseToolsManager::GetWidth(const G4ToolsBaseHisto& baseHisto, 
                                      G4int dimension) const
{
  G4int nbins = baseHisto.get_axis(dimension).bins();
  if ( ! nbins ) {
    G4String functionName = "G4BaseToolsManager::Get";
    functionName += fHnType;
    functionName += "Width";
    G4ExceptionDescription description;
    description << "    nbins = 0 (for " << fHnType << ").";
    G4Exception(functionName, "Analysis_W014", JustWarning, description);
    return 0;
  }              
  
  return ( baseHisto.get_axis(dimension).upper_edge() 
           - baseHisto.get_axis(dimension).lower_edge() )/nbins;
}  

//_____________________________________________________________________________
G4bool G4BaseToolsManager::SetTitle(G4ToolsBaseHisto& baseHisto, 
                                    const G4String& title)
{
  return baseHisto.set_title(title);
}  

//_____________________________________________________________________________
G4bool G4BaseToolsManager::SetAxisTitle(G4ToolsBaseHisto& baseHisto, 
                                        G4int dimension, const G4String& title)
{
  if ( dimension == 0 ) {
    baseHisto.add_annotation(tools::histo::key_axis_x_title(), title);
  }
  else if ( dimension == 1 ) {  
    baseHisto.add_annotation(tools::histo::key_axis_y_title(), title);
  }
  else if ( dimension == 2 ) {  
    baseHisto.add_annotation(tools::histo::key_axis_z_title(), title);
  }
  
  return true;
}  

//_____________________________________________________________________________
G4String G4BaseToolsManager::GetTitle(const G4ToolsBaseHisto& baseHisto) const
{
  return baseHisto.title();
}  


//_____________________________________________________________________________
G4String G4BaseToolsManager::GetAxisTitle(const G4ToolsBaseHisto& baseHisto, 
                                          G4int dimension) const 
{
  G4String title;
  G4bool result = false;
  if ( dimension == 0 ) {
    result = baseHisto.annotation(tools::histo::key_axis_x_title(), title);
  }  
  else if ( dimension == 1 ) {  
    result = baseHisto.annotation(tools::histo::key_axis_y_title(), title);
  }
  else if ( dimension == 2 ) {  
    result = baseHisto.annotation(tools::histo::key_axis_z_title(), title);
  }

  if ( ! result ) {
    G4String axes("xyz");
    G4String axis = axes(dimension, 1);
    G4String functionName = "G4BaseToolsManager::Get";
    functionName += fHnType;
    functionName += axis;
    functionName += "Title";
    G4ExceptionDescription description;
    description << "    Failed to get " << axis << " axis " << fHnType << " title.";
    G4Exception(functionName, "Analysis_W014", JustWarning, description);
    return "";
  }
  
  return title;              
}  
