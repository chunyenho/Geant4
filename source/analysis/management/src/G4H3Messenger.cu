#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$

// Author: Ivana Hrivnacova, 24/07/2014  (ivana@ipno.in2p3.fr)

#include "G4H3Messenger.hh"
#include "G4VAnalysisManager.hh"
#include "G4AnalysisUtilities.hh"

#include "G4UIdirectory.hh"
#include "G4UIcommand.hh"
#include "G4UIparameter.hh"

#include <iostream>

using namespace G4Analysis;

namespace {

void Exception(G4UIcommand* command, G4int nofParameters)
{
  G4ExceptionDescription description;
  description 
    << "Got wrong number of \"" << command->GetCommandName() 
    << "\" parameters: " << nofParameters
    << " instead of " << command->GetParameterEntries() 
    << " expected" << G4endl;
  G4Exception("G4H3Messenger::SetNewValue",
              "Analysis_W013", JustWarning, description);
}

}                  


//_____________________________________________________________________________
G4H3Messenger::G4H3Messenger(G4VAnalysisManager* manager)
  : G4UImessenger(),
    fManager(manager),
    fH3Dir(0),  
    fCreateH3Cmd(0),
    fSetH3Cmd(0),
    fSetH3TitleCmd(0), 
    fSetH3XAxisCmd(0), 
    fSetH3YAxisCmd(0)
{  
  fH3Dir = new G4UIdirectory("/analysis/h3/");
  fH3Dir->SetGuidance("3D histograms control");

  CreateH3Cmd();
  SetH3Cmd();
  
  SetH3TitleCmd();
  SetH3XAxisCmd();
  SetH3YAxisCmd();
  SetH3ZAxisCmd();
}

//_____________________________________________________________________________
G4H3Messenger::~G4H3Messenger()
{
  delete fCreateH3Cmd;
  delete fSetH3Cmd;
  delete fSetH3TitleCmd;  
  delete fSetH3XAxisCmd;  
  delete fSetH3YAxisCmd;  
  delete fSetH3ZAxisCmd;  
  delete fH3Dir;
}

//
// private functions
//

//_____________________________________________________________________________
void G4H3Messenger::CreateH3Cmd()
{
  G4UIparameter* h3Name = new G4UIparameter("name", 's', false);
  h3Name->SetGuidance("Histogram name (label)");
  
  G4UIparameter* h3Title = new G4UIparameter("title", 's', false);
  h3Title->SetGuidance("Histogram title");

  G4UIparameter* h3xNbins0 = new G4UIparameter("xnbins0", 'i', true);
  h3xNbins0->SetGuidance("Number of x-bins (default = 100)");
  h3xNbins0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3xNbins0->SetDefaultValue(100);
  
  G4UIparameter* h3xValMin0 = new G4UIparameter("xvalMin0", 'd', true);
  h3xValMin0->SetGuidance("Minimum x-value, expressed in unit (default = 0.)");
  h3xValMin0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3xValMin0->SetDefaultValue(0.);
  
  G4UIparameter* h3xValMax0 = new G4UIparameter("xvalMax0", 'd', true);
  h3xValMax0->SetGuidance("Maximum x-value, expressed in unit (default = 1.)");
  h3xValMax0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3xValMax0->SetDefaultValue(1.);

  G4UIparameter* h3xValUnit0 = new G4UIparameter("xvalUnit0", 's', true);
  h3xValUnit0->SetGuidance("The unit applied to filled x-values and xvalMin0, xvalMax0");
  h3xValUnit0->SetDefaultValue("none");
  
  G4UIparameter* h3xValFcn0 = new G4UIparameter("xvalFcn0", 's', true);
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  h3xValFcn0->SetGuidance(fcnxGuidance);
  h3xValFcn0->SetParameterCandidates("log log10 exp none");
  h3xValFcn0->SetDefaultValue("none");
  
  G4UIparameter* h3xValBinScheme0 = new G4UIparameter("xvalBinScheme0", 's', true);
  G4String xbinSchemeGuidance = "The binning scheme (linear, log).";
  h3xValBinScheme0->SetParameterCandidates("linear log");
  h3xValBinScheme0->SetGuidance(xbinSchemeGuidance);
  h3xValBinScheme0->SetDefaultValue("linear");
  
  G4UIparameter* h3yNbins0 = new G4UIparameter("ynbins0", 'i', true);
  h3yNbins0->SetGuidance("Number of y-bins (default = 100)");
  h3yNbins0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3yNbins0->SetDefaultValue(100);
  
  G4UIparameter* h3yValMin0 = new G4UIparameter("yvalMin0", 'd', true);
  h3yValMin0->SetGuidance("Minimum y-value, expressed in unit (default = 0.)");
  h3yValMin0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3yValMin0->SetDefaultValue(0.);
  
  G4UIparameter* h3yValMax0 = new G4UIparameter("yvalMax0", 'd', true);
  h3yValMax0->SetGuidance("Maximum y-value, expressed in unit (default = 1.)");
  h3yValMax0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3yValMax0->SetDefaultValue(1.);

  G4UIparameter* h3yValUnit0 = new G4UIparameter("yvalUnit0", 's', true);
  h3yValUnit0->SetGuidance("The unit applied to filled y-values and yvalMin0, yvalMax0");
  h3yValUnit0->SetDefaultValue("none");
  
  G4UIparameter* h3yValFcn0 = new G4UIparameter("yvalFcn0", 's', true);
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  h3yValFcn0->SetGuidance(fcnyGuidance);
  h3yValFcn0->SetParameterCandidates("log log10 exp none");
  h3yValFcn0->SetDefaultValue("none");

  G4UIparameter* h3yValBinScheme0 = new G4UIparameter("yvalBinScheme0", 's', true);
  G4String ybinSchemeGuidance = "The binning scheme (linear, log).";
  h3yValBinScheme0->SetParameterCandidates("linear log");
  h3yValBinScheme0->SetGuidance(ybinSchemeGuidance);
  h3yValBinScheme0->SetDefaultValue("linear");
  
  G4UIparameter* h3zNbins0 = new G4UIparameter("znbins0", 'i', true);
  h3zNbins0->SetGuidance("Number of z-bins (default = 100)");
  h3zNbins0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3zNbins0->SetDefaultValue(100);
  
  G4UIparameter* h3zValMin0 = new G4UIparameter("zvalMin0", 'd', true);
  h3zValMin0->SetGuidance("Minimum z-value, expressed in unit (default = 0.)");
  h3zValMin0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3zValMin0->SetDefaultValue(0.);
  
  G4UIparameter* h3zValMax0 = new G4UIparameter("zvalMax0", 'd', true);
  h3zValMax0->SetGuidance("Maximum z-value, expressed in unit (default = 1.)");
  h3zValMax0->SetGuidance("Can be reset with /analysis/h3/set command");
  h3zValMax0->SetDefaultValue(1.);

  G4UIparameter* h3zValUnit0 = new G4UIparameter("zvalUnit0", 's', true);
  h3zValUnit0->SetGuidance("The unit applied to filled z-values and zvalMin0, zvalMax0");
  h3zValUnit0->SetDefaultValue("none");
  
  G4UIparameter* h3zValFcn0 = new G4UIparameter("zvalFcn0", 's', true);
  G4String fcnzGuidance = "The function applied to filled z-values (log, log10, exp, none).";
  h3zValFcn0->SetGuidance(fcnzGuidance);
  h3zValFcn0->SetParameterCandidates("log log10 exp none");
  h3zValFcn0->SetDefaultValue("none");

  G4UIparameter* h3zValBinScheme0 = new G4UIparameter("zvalBinScheme0", 's', true);
  G4String zbinSchemeGuidance = "The binning scheme (linear, log).";
  h3zValBinScheme0->SetParameterCandidates("linear log");
  h3zValBinScheme0->SetGuidance(zbinSchemeGuidance);
  h3zValBinScheme0->SetDefaultValue("linear");
  
  fCreateH3Cmd = new G4UIcommand("/analysis/h3/create", this);
  fCreateH3Cmd->SetGuidance("Create 3D histogram");
  fCreateH3Cmd->SetParameter(h3Name);
  fCreateH3Cmd->SetParameter(h3Title);
  fCreateH3Cmd->SetParameter(h3xNbins0);
  fCreateH3Cmd->SetParameter(h3xValMin0);
  fCreateH3Cmd->SetParameter(h3xValMax0);
  fCreateH3Cmd->SetParameter(h3xValUnit0);
  fCreateH3Cmd->SetParameter(h3xValFcn0);
  fCreateH3Cmd->SetParameter(h3xValBinScheme0);
  fCreateH3Cmd->SetParameter(h3yNbins0);
  fCreateH3Cmd->SetParameter(h3yValMin0);
  fCreateH3Cmd->SetParameter(h3yValMax0);
  fCreateH3Cmd->SetParameter(h3yValUnit0);
  fCreateH3Cmd->SetParameter(h3yValFcn0);
  fCreateH3Cmd->SetParameter(h3yValBinScheme0);
  fCreateH3Cmd->SetParameter(h3zNbins0);
  fCreateH3Cmd->SetParameter(h3zValMin0);
  fCreateH3Cmd->SetParameter(h3zValMax0);
  fCreateH3Cmd->SetParameter(h3zValUnit0);
  fCreateH3Cmd->SetParameter(h3zValFcn0);
  fCreateH3Cmd->SetParameter(h3zValBinScheme0);
  fCreateH3Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  


//_____________________________________________________________________________
void G4H3Messenger::SetH3Cmd()
{
  G4UIparameter* h3Id = new G4UIparameter("id", 'i', false);
  h3Id->SetGuidance("Histogram id");
  h3Id->SetParameterRange("id>=0");
  
  G4UIparameter* h3xNbins = new G4UIparameter("xnbins", 'i', false);
  h3xNbins->SetGuidance("Number of x-bins");
  
  G4UIparameter* h3xValMin = new G4UIparameter("xvalMin", 'd', false);
  h3xValMin->SetGuidance("Minimum x-value, expressed in unit");
  
  G4UIparameter* h3xValMax = new G4UIparameter("xvalMax", 'd', false);
  h3xValMax->SetGuidance("Maximum x-value, expressed in unit");
  
  G4UIparameter* h3xValUnit = new G4UIparameter("xvalUnit", 's', false);
  h3xValUnit->SetGuidance("The unit applied to filled x-values and xvalMin0, xvalMax0");
  h3xValUnit->SetDefaultValue("none");
 
  G4UIparameter* h3xValFcn = new G4UIparameter("xvalFcn", 's', false);
  h3xValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  h3xValFcn->SetGuidance(fcnxGuidance);
  h3xValFcn->SetDefaultValue("none");
 
  G4UIparameter* h3xValBinScheme = new G4UIparameter("xvalBinScheme", 's', true);
  G4String xbinSchemeGuidance = "The binning scheme (linear, log).";
  h3xValBinScheme->SetParameterCandidates("linear log");
  h3xValBinScheme->SetGuidance(xbinSchemeGuidance);
  h3xValBinScheme->SetDefaultValue("linear");
  
  G4UIparameter* h3yNbins = new G4UIparameter("nybins", 'i', false);
  h3yNbins->SetGuidance("Number of y-bins");
  
  G4UIparameter* h3yValMin = new G4UIparameter("yvalMin", 'd', false);
  h3yValMin->SetGuidance("Minimum y-value, expressed in unit");
  
  G4UIparameter* h3yValMax = new G4UIparameter("yvalMax", 'd', false);
  h3yValMax->SetGuidance("Maximum y-value, expressed in unit");
  
  G4UIparameter* h3yValUnit = new G4UIparameter("yvalUnit", 's', true);
  h3yValUnit->SetGuidance("The unit applied to filled y-values and yvalMin0, yvalMax0");
  h3yValUnit->SetDefaultValue("none");
 
  G4UIparameter* h3yValFcn = new G4UIparameter("yvalFcn", 's', false);
  h3yValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  h3yValFcn->SetGuidance(fcnyGuidance);
  h3yValFcn->SetDefaultValue("none");
 
  G4UIparameter* h3yValBinScheme = new G4UIparameter("yvalBinScheme", 's', true);
  G4String ybinSchemeGuidance = "The binning scheme (linear, log).";
  h3yValBinScheme->SetParameterCandidates("linear log");
  h3yValBinScheme->SetGuidance(ybinSchemeGuidance);
  h3yValBinScheme->SetDefaultValue("linear");
 
  G4UIparameter* h3zNbins = new G4UIparameter("nzbins", 'i', false);
  h3zNbins->SetGuidance("Number of z-bins");
  
  G4UIparameter* h3zValMin = new G4UIparameter("zvalMin", 'd', false);
  h3zValMin->SetGuidance("Minimum z-value, expressed in unit");
  
  G4UIparameter* h3zValMax = new G4UIparameter("zvalMax", 'd', false);
  h3zValMax->SetGuidance("Maximum z-value, expressed in unit");
  
  G4UIparameter* h3zValUnit = new G4UIparameter("zvalUnit", 's', true);
  h3zValUnit->SetGuidance("The unit applied to filled z-values and zvalMin0, zvalMax0");
  h3zValUnit->SetDefaultValue("none");
 
  G4UIparameter* h3zValFcn = new G4UIparameter("zvalFcn", 's', false);
  h3zValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnzGuidance = "The function applied to filled z-values (log, log10, exp, none).";
  h3zValFcn->SetGuidance(fcnzGuidance);
  h3zValFcn->SetDefaultValue("none");
 
  G4UIparameter* h3zValBinScheme = new G4UIparameter("zvalBinScheme", 's', true);
  G4String zbinSchemeGuidance = "The binning scheme (linear, log).";
  h3zValBinScheme->SetParameterCandidates("linear log");
  h3zValBinScheme->SetGuidance(zbinSchemeGuidance);
  h3zValBinScheme->SetDefaultValue("linear");
 
  fSetH3Cmd = new G4UIcommand("/analysis/h3/set", this);
  fSetH3Cmd->SetGuidance("Set parameters for the 3D histogram of #Id :");
  fSetH3Cmd->SetGuidance("  nxbins; xvalMin; xvalMax; xunit; xfunction; xbinScheme");
  fSetH3Cmd->SetGuidance("  nybins; yvalMin; yvalMax; yunit; yfunction; ybinScheme");
  fSetH3Cmd->SetGuidance("  nzbins; zvalMin; zvalMax; zunit; zfunction; zbinScheme");
  fSetH3Cmd->SetParameter(h3Id);
  fSetH3Cmd->SetParameter(h3xNbins);
  fSetH3Cmd->SetParameter(h3xValMin);
  fSetH3Cmd->SetParameter(h3xValMax);
  fSetH3Cmd->SetParameter(h3xValUnit);
  fSetH3Cmd->SetParameter(h3xValFcn);
  fSetH3Cmd->SetParameter(h3xValBinScheme);
  fSetH3Cmd->SetParameter(h3yNbins);
  fSetH3Cmd->SetParameter(h3yValMin);
  fSetH3Cmd->SetParameter(h3yValMax);
  fSetH3Cmd->SetParameter(h3yValUnit);
  fSetH3Cmd->SetParameter(h3yValFcn);
  fSetH3Cmd->SetParameter(h3yValBinScheme);
  fSetH3Cmd->SetParameter(h3zNbins);
  fSetH3Cmd->SetParameter(h3zValMin);
  fSetH3Cmd->SetParameter(h3zValMax);
  fSetH3Cmd->SetParameter(h3zValUnit);
  fSetH3Cmd->SetParameter(h3zValFcn);
  fSetH3Cmd->SetParameter(h3zValBinScheme);
  fSetH3Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H3Messenger::SetH3TitleCmd()
{
  G4UIparameter* h3Id = new G4UIparameter("idTitle", 'i', false);
  h3Id->SetGuidance("Histogram id");
  h3Id->SetParameterRange("idTitle>=0");

  G4UIparameter* h3Title = new G4UIparameter("h3Title", 's', true);
  h3Title->SetGuidance("Histogram title");
  h3Title->SetDefaultValue("none");

  fSetH3TitleCmd = new G4UIcommand("/analysis/h3/setTitle", this);
  fSetH3TitleCmd->SetGuidance("Set title for the 3D histogram of #Id");
  fSetH3TitleCmd->SetParameter(h3Id);
  fSetH3TitleCmd->SetParameter(h3Title);
  fSetH3TitleCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H3Messenger::SetH3XAxisCmd()
{
  G4UIparameter* h3Id = new G4UIparameter("idXaxis", 'i', false);
  h3Id->SetGuidance("Histogram id");
  h3Id->SetParameterRange("idXaxis>=0");

  G4UIparameter* h3XAxis = new G4UIparameter("h3Xaxis", 's', true);
  h3XAxis->SetGuidance("Histogram x-axis title");
  h3XAxis->SetDefaultValue("none");

  fSetH3XAxisCmd = new G4UIcommand("/analysis/h3/setXaxis", this);
  fSetH3XAxisCmd->SetGuidance("Set x-axis title for the 3D histogram of #Id");
  fSetH3XAxisCmd->SetParameter(h3Id);
  fSetH3XAxisCmd->SetParameter(h3XAxis);
  fSetH3XAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H3Messenger::SetH3YAxisCmd()
{
  G4UIparameter* h3Id = new G4UIparameter("idYaxis", 'i', false);
  h3Id->SetGuidance("Histogram id");
  h3Id->SetParameterRange("idYaxis>=0");

  G4UIparameter* h3YAxis = new G4UIparameter("h3Yaxis", 's', true);
  h3YAxis->SetGuidance("Histogram y-axis title");
  h3YAxis->SetDefaultValue("none");

  fSetH3YAxisCmd = new G4UIcommand("/analysis/h3/setYaxis", this);
  fSetH3YAxisCmd->SetGuidance("Set y-axis title for the 3D histogram of #Id");
  fSetH3YAxisCmd->SetParameter(h3Id);
  fSetH3YAxisCmd->SetParameter(h3YAxis);
  fSetH3YAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H3Messenger::SetH3ZAxisCmd()
{
  G4UIparameter* h3Id = new G4UIparameter("idYaxis", 'i', false);
  h3Id->SetGuidance("Histogram id");
  h3Id->SetParameterRange("idYaxis>=0");

  G4UIparameter* h3YAxis = new G4UIparameter("h3Yaxis", 's', true);
  h3YAxis->SetGuidance("Histogram y-axis title");
  h3YAxis->SetDefaultValue("none");

  fSetH3ZAxisCmd = new G4UIcommand("/analysis/h3/setZaxis", this);
  fSetH3ZAxisCmd->SetGuidance("Set y-axis title for the 3D histogram of #Id");
  fSetH3ZAxisCmd->SetParameter(h3Id);
  fSetH3ZAxisCmd->SetParameter(h3YAxis);
  fSetH3ZAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//
// public functions
//

//_____________________________________________________________________________
void G4H3Messenger::SetNewValue(G4UIcommand* command, G4String newValues)
{
  // tokenize parameters in a vector
  std::vector<G4String> parameters;
  G4Analysis::Tokenize(newValues, parameters);
  // check consistency
  if ( G4int(parameters.size()) != command->GetParameterEntries() ) {
    // Should never happen but let's check anyway for consistency
    Exception(command, parameters.size());
    return;
  }  

  if ( command == fCreateH3Cmd ) { 
    G4int counter = 0;
    G4String name = parameters[counter++];
    G4String title = parameters[counter++];
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    G4int znbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double zvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double zvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String zsunit = parameters[counter++];
    G4String zsfcn = parameters[counter++];
    G4String zsbinScheme = parameters[counter++];
    G4double zunit = GetUnitValue(zsunit);
    fManager->CreateH3(name, title, 
                       xnbins, xvmin*xunit, xvmax*xunit,
                       ynbins, yvmin*yunit, yvmax*yunit, 
                       znbins, zvmin*zunit, zvmax*zunit, 
                       xsunit, ysunit, zsunit, xsfcn, ysfcn, zsfcn,
                       xsbinScheme, ysbinScheme, zsbinScheme);     
  }
  else if ( command == fSetH3Cmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]);
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    G4int znbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double zvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double zvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String zsunit = parameters[counter++];
    G4String zsfcn = parameters[counter++];
    G4String zsbinScheme = parameters[counter++];
    G4double zunit = GetUnitValue(zsunit);
    fManager->SetH3(id, 
                    xnbins, xvmin*xunit, xvmax*xunit,
                    ynbins, yvmin*yunit, yvmax*yunit, 
                    znbins, zvmin*zunit, zvmax*zunit, 
                    xsunit, ysunit, zsunit, xsfcn, ysfcn, zsfcn,
                    xsbinScheme, ysbinScheme, zsbinScheme);     
  }
  else if ( command == fSetH3TitleCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String title = parameters[counter++];
    fManager->SetH3Title(id, title);     
  }
  else if ( command == fSetH3XAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String xaxis = parameters[counter++];
    fManager->SetH3XAxisTitle(id, xaxis);     
  }
  else if ( command == fSetH3YAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String yaxis = parameters[counter++];
    fManager->SetH3YAxisTitle(id, yaxis);     
  }
  else if ( command == fSetH3ZAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String zaxis = parameters[counter++];
    fManager->SetH3ZAxisTitle(id, zaxis);     
  }
}  
