#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$

// Author: Ivana Hrivnacova, 24/07/2014  (ivana@ipno.in2p3.fr)

#include "G4P2Messenger.hh"
#include "G4VAnalysisManager.hh"
#include "G4AnalysisUtilities.hh"

#include "G4UIdirectory.hh"
#include "G4UIcommand.hh"
#include "G4UIparameter.hh"

#include <iostream>

using namespace G4Analysis;

namespace {

void Exception(G4UIcommand* command, G4int nofParameters)
{
  G4ExceptionDescription description;
  description 
    << "Got wrong number of \"" << command->GetCommandName() 
    << "\" parameters: " << nofParameters
    << " instead of " << command->GetParameterEntries() 
    << " expected" << G4endl;
  G4Exception("G4P2Messenger::SetNewValue",
              "Analysis_W013", JustWarning, description);
}

}                  


//_____________________________________________________________________________
G4P2Messenger::G4P2Messenger(G4VAnalysisManager* manager)
  : G4UImessenger(),
    fManager(manager),
    fP2Dir(0),  
    fCreateP2Cmd(0),
    fSetP2Cmd(0),
    fSetP2TitleCmd(0), 
    fSetP2XAxisCmd(0), 
    fSetP2YAxisCmd(0)
{  
  fP2Dir = new G4UIdirectory("/analysis/p2/");
  fP2Dir->SetGuidance("2D profiles control");

  CreateP2Cmd();
  SetP2Cmd();
  
  SetP2TitleCmd();
  SetP2XAxisCmd();
  SetP2YAxisCmd();
  SetP2ZAxisCmd();
}

//_____________________________________________________________________________
G4P2Messenger::~G4P2Messenger()
{
  delete fCreateP2Cmd;
  delete fSetP2Cmd;
  delete fSetP2TitleCmd;  
  delete fSetP2XAxisCmd;  
  delete fSetP2YAxisCmd;  
  delete fSetP2ZAxisCmd;  
  delete fP2Dir;
}

//
// private functions
//

//_____________________________________________________________________________
void G4P2Messenger::CreateP2Cmd()
{
  G4UIparameter* p2Name = new G4UIparameter("name", 's', false);
  p2Name->SetGuidance("Profile name (label)");
  
  G4UIparameter* p2Title = new G4UIparameter("title", 's', false);
  p2Title->SetGuidance("Profile title");

  G4UIparameter* p2xNbins0 = new G4UIparameter("xnbins0", 'i', true);
  p2xNbins0->SetGuidance("Number of x-bins (default = 100)");
  p2xNbins0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2xNbins0->SetDefaultValue(100);
  
  G4UIparameter* p2xValMin0 = new G4UIparameter("xvalMin0", 'd', true);
  p2xValMin0->SetGuidance("Minimum x-value, expressed in unit (default = 0.)");
  p2xValMin0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2xValMin0->SetDefaultValue(0.);
  
  G4UIparameter* p2xValMax0 = new G4UIparameter("xvalMax0", 'd', true);
  p2xValMax0->SetGuidance("Maximum x-value, expressed in unit (default = 1.)");
  p2xValMax0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2xValMax0->SetDefaultValue(1.);

  G4UIparameter* p2xValUnit0 = new G4UIparameter("xvalUnit0", 's', true);
  p2xValUnit0->SetGuidance("The unit applied to filled x-values and xvalMin0, xvalMax0");
  p2xValUnit0->SetDefaultValue("none");
  
  G4UIparameter* p2xValFcn0 = new G4UIparameter("xvalFcn0", 's', true);
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  p2xValFcn0->SetGuidance(fcnxGuidance);
  p2xValFcn0->SetParameterCandidates("log log10 exp none");
  p2xValFcn0->SetDefaultValue("none");
    
  G4UIparameter* p2xValBinScheme0 = new G4UIparameter("xvalBinScheme0", 's', true);
  G4String binSchemeGuidance = "The binning scheme (linear, log).";
  p2xValBinScheme0->SetParameterCandidates("linear log");
  p2xValBinScheme0->SetGuidance(binSchemeGuidance);
  p2xValBinScheme0->SetDefaultValue("linear");
  
  G4UIparameter* p2yNbins0 = new G4UIparameter("ynbins0", 'i', true);
  p2yNbins0->SetGuidance("Number of y-bins (default = 100)");
  p2yNbins0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2yNbins0->SetDefaultValue(100);
  
  G4UIparameter* p2yValMin0 = new G4UIparameter("yvalMin0", 'd', true);
  p2yValMin0->SetGuidance("Minimum y-value, expressed in unit (default = 0.)");
  p2yValMin0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2yValMin0->SetDefaultValue(0.);
  
  G4UIparameter* p2yValMax0 = new G4UIparameter("yvalMax0", 'd', true);
  p2yValMax0->SetGuidance("Maximum y-value, expressed in unit (default = 1.)");
  p2yValMax0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2yValMax0->SetDefaultValue(1.);

  G4UIparameter* p2yValUnit0 = new G4UIparameter("yvalUnit0", 's', true);
  p2yValUnit0->SetGuidance("The unit applied to filled y-values and yvalMin0, yvalMax0");
  p2yValUnit0->SetDefaultValue("none");
  
  G4UIparameter* p2yValFcn0 = new G4UIparameter("yvalFcn0", 's', true);
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  p2yValFcn0->SetGuidance(fcnyGuidance);
  p2yValFcn0->SetParameterCandidates("log log10 exp none");
  p2yValFcn0->SetDefaultValue("none");
    
  G4UIparameter* p2yValBinScheme0 = new G4UIparameter("yvalBinScheme0", 's', true);
  p2yValBinScheme0->SetParameterCandidates("linear log");
  p2yValBinScheme0->SetGuidance(binSchemeGuidance);
  p2yValBinScheme0->SetDefaultValue("linear");
  
  G4UIparameter* p2zValMin0 = new G4UIparameter("zvalMin0", 'd', true);
  p2zValMin0->SetGuidance("Minimum z-value, expressed in unit (default = 0.)");
  p2zValMin0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2zValMin0->SetDefaultValue(0.);
  
  G4UIparameter* p2zValMax0 = new G4UIparameter("zvalMax0", 'd', true);
  p2zValMax0->SetGuidance("Maximum z-value, expressed in unit (default = 1.)");
  p2zValMax0->SetGuidance("Can be reset with /analysis/p2/set command");
  p2zValMax0->SetDefaultValue(1.);

  G4UIparameter* p2zValUnit0 = new G4UIparameter("zvalUnit0", 's', true);
  p2zValUnit0->SetGuidance("The unit applied to filled z-values and zvalMin0, zvalMax0");
  p2zValUnit0->SetDefaultValue("none");
  
  G4UIparameter* p2zValFcn0 = new G4UIparameter("zvalFcn0", 's', true);
  G4String fcnzGuidance = "The function applied to filled z-values (log, log10, exp, none).";
  p2zValFcn0->SetGuidance(fcnzGuidance);
  p2zValFcn0->SetParameterCandidates("log log10 exp none");
  p2zValFcn0->SetDefaultValue("none");
  
  fCreateP2Cmd = new G4UIcommand("/analysis/p2/create", this);
  fCreateP2Cmd->SetGuidance("Create 2D profile");
  fCreateP2Cmd->SetParameter(p2Name);
  fCreateP2Cmd->SetParameter(p2Title);
  fCreateP2Cmd->SetParameter(p2xNbins0);
  fCreateP2Cmd->SetParameter(p2xValMin0);
  fCreateP2Cmd->SetParameter(p2xValMax0);
  fCreateP2Cmd->SetParameter(p2xValUnit0);
  fCreateP2Cmd->SetParameter(p2xValFcn0);
  fCreateP2Cmd->SetParameter(p2xValBinScheme0);
  fCreateP2Cmd->SetParameter(p2yNbins0);
  fCreateP2Cmd->SetParameter(p2yValMin0);
  fCreateP2Cmd->SetParameter(p2yValMax0);
  fCreateP2Cmd->SetParameter(p2yValUnit0);
  fCreateP2Cmd->SetParameter(p2yValFcn0);
  fCreateP2Cmd->SetParameter(p2yValBinScheme0);
  fCreateP2Cmd->SetParameter(p2zValMin0);
  fCreateP2Cmd->SetParameter(p2zValMax0);
  fCreateP2Cmd->SetParameter(p2zValUnit0);
  fCreateP2Cmd->SetParameter(p2zValFcn0);
  fCreateP2Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  


//_____________________________________________________________________________
void G4P2Messenger::SetP2Cmd()
{
  G4UIparameter* p2Id = new G4UIparameter("id", 'i', false);
  p2Id->SetGuidance("Profile id");
  p2Id->SetParameterRange("id>=0");
  
  G4UIparameter* p2xNbins = new G4UIparameter("xnbins", 'i', false);
  p2xNbins->SetGuidance("Number of x-bins");
  
  G4UIparameter* p2xValMin = new G4UIparameter("xvalMin", 'd', false);
  p2xValMin->SetGuidance("Minimum x-value, expressed in unit");
  
  G4UIparameter* p2xValMax = new G4UIparameter("xvalMax", 'd', false);
  p2xValMax->SetGuidance("Maximum x-value, expressed in unit");
  
  G4UIparameter* p2xValUnit = new G4UIparameter("xvalUnit", 's', false);
  p2xValUnit->SetGuidance("The unit applied to filled x-values and xvalMin0, xvalMax0");
  p2xValUnit->SetDefaultValue("none");
 
  G4UIparameter* p2xValFcn = new G4UIparameter("xvalFcn", 's', false);
  p2xValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  p2xValFcn->SetGuidance(fcnxGuidance);
  p2xValFcn->SetDefaultValue("none");
    
  G4UIparameter* p2xValBinScheme = new G4UIparameter("xvalBinScheme", 's', true);
  G4String binSchemeGuidance = "The binning scheme (linear, log).";
  p2xValBinScheme->SetParameterCandidates("linear log");
  p2xValBinScheme->SetGuidance(binSchemeGuidance);
  p2xValBinScheme->SetDefaultValue("linear");
 
  G4UIparameter* p2yNbins = new G4UIparameter("nybins", 'i', false);
  p2yNbins->SetGuidance("Number of y-bins");
  
  G4UIparameter* p2yValMin = new G4UIparameter("yvalMin", 'd', false);
  p2yValMin->SetGuidance("Minimum y-value, expressed in unit");
  
  G4UIparameter* p2yValMax = new G4UIparameter("yvalMax", 'd', false);
  p2yValMax->SetGuidance("Maximum y-value, expressed in unit");
  
  G4UIparameter* p2yValUnit = new G4UIparameter("yvalUnit", 's', true);
  p2yValUnit->SetGuidance("The unit applied to filled y-values and yvalMin0, yvalMax0");
  p2yValUnit->SetDefaultValue("none");
 
  G4UIparameter* p2yValFcn = new G4UIparameter("yvalFcn", 's', false);
  p2yValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  p2yValFcn->SetGuidance(fcnyGuidance);
  p2yValFcn->SetDefaultValue("none");
    
  G4UIparameter* p2yValBinScheme = new G4UIparameter("yvalBinScheme", 's', true);
  p2yValBinScheme->SetParameterCandidates("linear log");
  p2yValBinScheme->SetGuidance(binSchemeGuidance);
  p2yValBinScheme->SetDefaultValue("linear");
 
  G4UIparameter* p2zValMin = new G4UIparameter("zvalMin", 'd', false);
  p2zValMin->SetGuidance("Minimum z-value, expressed in unit");
  
  G4UIparameter* p2zValMax = new G4UIparameter("zvalMax", 'd', false);
  p2zValMax->SetGuidance("Maximum z-value, expressed in unit");
  
  G4UIparameter* p2zValUnit = new G4UIparameter("zvalUnit", 's', true);
  p2zValUnit->SetGuidance("The unit applied to filled z-values and zvalMin0, zvalMax0");
  p2zValUnit->SetDefaultValue("none");
 
  G4UIparameter* p2zValFcn = new G4UIparameter("zvalFcn", 's', false);
  p2zValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnzGuidance = "The function applied to filled z-values (log, log10, exp, none).";
  p2zValFcn->SetGuidance(fcnzGuidance);
  p2zValFcn->SetDefaultValue("none");
 
  fSetP2Cmd = new G4UIcommand("/analysis/p2/set", this);
  fSetP2Cmd->SetGuidance("Set parameters for the 2D profile of #Id :");
  fSetP2Cmd->SetGuidance("  nxbins; xvalMin; xvalMax; xunit; xbinScheme");
  fSetP2Cmd->SetGuidance("  nybins; yvalMin; yvalMax; yunit; ybinScheme");
  fSetP2Cmd->SetGuidance("  zvalMin; zvalMax; zunit; zfunction");
  fSetP2Cmd->SetParameter(p2Id);
  fSetP2Cmd->SetParameter(p2xNbins);
  fSetP2Cmd->SetParameter(p2xValMin);
  fSetP2Cmd->SetParameter(p2xValMax);
  fSetP2Cmd->SetParameter(p2xValUnit);
  fSetP2Cmd->SetParameter(p2xValFcn);
  fSetP2Cmd->SetParameter(p2xValBinScheme);
  fSetP2Cmd->SetParameter(p2yNbins);
  fSetP2Cmd->SetParameter(p2yValMin);
  fSetP2Cmd->SetParameter(p2yValMax);
  fSetP2Cmd->SetParameter(p2yValUnit);
  fSetP2Cmd->SetParameter(p2yValFcn);
  fSetP2Cmd->SetParameter(p2yValBinScheme);
  fSetP2Cmd->SetParameter(p2zValMin);
  fSetP2Cmd->SetParameter(p2zValMax);
  fSetP2Cmd->SetParameter(p2zValUnit);
  fSetP2Cmd->SetParameter(p2zValFcn);
  fSetP2Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4P2Messenger::SetP2TitleCmd()
{
  G4UIparameter* p2Id = new G4UIparameter("idTitle", 'i', false);
  p2Id->SetGuidance("Profile id");
  p2Id->SetParameterRange("idTitle>=0");

  G4UIparameter* p2Title = new G4UIparameter("p2Title", 's', true);
  p2Title->SetGuidance("Profile title");
  p2Title->SetDefaultValue("none");

  fSetP2TitleCmd = new G4UIcommand("/analysis/p2/setTitle", this);
  fSetP2TitleCmd->SetGuidance("Set title for the 2D profile of #Id");
  fSetP2TitleCmd->SetParameter(p2Id);
  fSetP2TitleCmd->SetParameter(p2Title);
  fSetP2TitleCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4P2Messenger::SetP2XAxisCmd()
{
  G4UIparameter* p2Id = new G4UIparameter("idXaxis", 'i', false);
  p2Id->SetGuidance("Profile id");
  p2Id->SetParameterRange("idXaxis>=0");

  G4UIparameter* p2XAxis = new G4UIparameter("p2Xaxis", 's', true);
  p2XAxis->SetGuidance("Profile x-axis title");
  p2XAxis->SetDefaultValue("none");

  fSetP2XAxisCmd = new G4UIcommand("/analysis/p2/setXaxis", this);
  fSetP2XAxisCmd->SetGuidance("Set x-axis title for the 2D profile of #Id");
  fSetP2XAxisCmd->SetParameter(p2Id);
  fSetP2XAxisCmd->SetParameter(p2XAxis);
  fSetP2XAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4P2Messenger::SetP2YAxisCmd()
{
  G4UIparameter* p2Id = new G4UIparameter("idYaxis", 'i', false);
  p2Id->SetGuidance("Profile id");
  p2Id->SetParameterRange("idYaxis>=0");

  G4UIparameter* p2YAxis = new G4UIparameter("p2Yaxis", 's', true);
  p2YAxis->SetGuidance("Profile y-axis title");
  p2YAxis->SetDefaultValue("none");

  fSetP2YAxisCmd = new G4UIcommand("/analysis/p2/setYaxis", this);
  fSetP2YAxisCmd->SetGuidance("Set y-axis title for the 2D profile of #Id");
  fSetP2YAxisCmd->SetParameter(p2Id);
  fSetP2YAxisCmd->SetParameter(p2YAxis);
  fSetP2YAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4P2Messenger::SetP2ZAxisCmd()
{
  G4UIparameter* p2Id = new G4UIparameter("idZaxis", 'i', false);
  p2Id->SetGuidance("Profile id");
  p2Id->SetParameterRange("idZaxis>=0");

  G4UIparameter* p2ZAxis = new G4UIparameter("p2Zaxis", 's', true);
  p2ZAxis->SetGuidance("Profile z-axis title");
  p2ZAxis->SetDefaultValue("none");

  fSetP2ZAxisCmd = new G4UIcommand("/analysis/p2/setZaxis", this);
  fSetP2ZAxisCmd->SetParameter(p2Id);
  fSetP2ZAxisCmd->SetParameter(p2ZAxis);
  fSetP2ZAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//
// public functions
//

//_____________________________________________________________________________
void G4P2Messenger::SetNewValue(G4UIcommand* command, G4String newValues)
{
  // tokenize parameters in a vector
  std::vector<G4String> parameters;
  G4Analysis::Tokenize(newValues, parameters);
  // check consistency
  if ( G4int(parameters.size()) != command->GetParameterEntries() ) {
    // Should never happen but let's check anyway for consistency
    Exception(command, parameters.size());
    return;
  }  

  if ( command == fCreateP2Cmd ) { 
    G4int counter = 0;
    G4String name = parameters[counter++];
    G4String title = parameters[counter++];
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    G4double zvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double zvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String zsunit = parameters[counter++];
    G4String zsfcn = parameters[counter++];
    G4double zunit = GetUnitValue(zsunit);
    fManager->CreateP2(name, title, 
                       xnbins, xvmin*xunit, xvmax*xunit,
                       ynbins, yvmin*yunit, yvmax*yunit, 
                       zvmin*zunit, zvmax*zunit, 
                       xsunit, ysunit, zsunit, xsfcn, ysfcn, zsfcn,
                       xsbinScheme, ysbinScheme);     
  }
  else if ( command == fSetP2Cmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]);
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    G4double zvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double zvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String zsunit = parameters[counter++];
    G4String zsfcn = parameters[counter++];
    G4double zunit = GetUnitValue(zsunit);
    fManager->SetP2(id, 
                    xnbins, xvmin*xunit, xvmax*xunit,
                    ynbins, yvmin*yunit, yvmax*yunit, 
                    zvmin*zunit, zvmax*zunit, 
                    xsunit, ysunit, zsunit, xsfcn, ysfcn, zsfcn,     
                    xsbinScheme, ysbinScheme);     
  }
  else if ( command == fSetP2TitleCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String title = parameters[counter++];
    fManager->SetP2Title(id, title);     
  }
  else if ( command == fSetP2XAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String xaxis = parameters[counter++];
    fManager->SetP2XAxisTitle(id, xaxis);     
  }
  else if ( command == fSetP2YAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String yaxis = parameters[counter++];
    fManager->SetP2YAxisTitle(id, yaxis);     
  }
  else if ( command == fSetP2ZAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String zaxis = parameters[counter++];
    fManager->SetP2ZAxisTitle(id, zaxis);     
  }
}  
