#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id:$

// Author: Ivana Hrivnacova, 22/08/2013  (ivana@ipno.in2p3.fr)

#include "G4BinScheme.hh"

namespace G4Analysis
{

//_____________________________________________________________________________
G4BinScheme GetBinScheme(const G4String& binSchemeName)
{
  G4BinScheme binScheme = kLinearBinScheme;
  if ( binSchemeName != "linear" ) {
    if  ( binSchemeName == "log" )  
      binScheme = kLogBinScheme;
    else {
      // There is no name associated with kUserBinScheme
      G4ExceptionDescription description;
      description 
        << "    \"" << binScheme << "\" binning scheme is not supported." << G4endl
        << "    " << "Linear binning will be applied.";
      G4Exception("G4Analysis::GetBinScheme",
                "Analysis_W013", JustWarning, description);
    }              
  }
  return binScheme;            
}

//_____________________________________________________________________________
void ComputeEdges(G4int nbins, G4double xmin, G4double xmax, 
                  G4double unit, G4Fcn fcn, G4BinScheme binScheme,
                  std::vector<G4double>& edges)
{
// Compute edges from parameters

  // Apply units
  G4double xumin = xmin/unit;
  G4double xumax = xmax/unit;

  if ( binScheme == kLinearBinScheme ) {
    G4double dx = (fcn(xumax) - fcn(xumin) ) / nbins;
    G4double binValue = fcn(xumin);
    while ( G4int(edges.size()) <= nbins ) {
      edges.push_back(binValue);
      binValue += dx;
    }
  }  
  else if ( binScheme == kLogBinScheme ) {
    // do not apply fcn 
    G4double dlog 
      = (std::log10(xumax) - std::log10(xumin))/ nbins;
    G4double dx = std::pow(10, dlog);
    G4double binValue = xumin;
    while ( G4int(edges.size()) <= nbins ) {
      edges.push_back(binValue);
      binValue *= dx;
    }
  }
  else if ( binScheme == kUserBinScheme ) {  
    // This should never happen, but let's make sure about it
    // by issuing a warning
    G4ExceptionDescription description;
    description 
      << "    User binning scheme setting was ignored." << G4endl
      << "    Linear binning will be applied with given (nbins, xmin, xmax) values";
    G4Exception("G4Analysis::ComputeEdges",
              "Analysis_W013", JustWarning, description);
  }              
}                                          

//_____________________________________________________________________________
void ComputeEdges(const std::vector<G4double>& edges, 
                  G4double unit, G4Fcn fcn, 
                  std::vector<G4double>& newBins)
{
// Apply function to defined edges
  std::vector<G4double>::const_iterator it;
  for (it = edges.begin(); it != edges.end(); it++ ) {
    newBins.push_back(fcn((*it)/unit));
  }
}
    
}
