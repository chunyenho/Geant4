#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: G4H2Messenger.cc 66310 2012-12-17 11:56:35Z ihrivnac $

// Author: Ivana Hrivnacova, 18/06/2013  (ivana@ipno.in2p3.fr)

#include "G4H2Messenger.hh"
#include "G4VAnalysisManager.hh"
#include "G4AnalysisUtilities.hh"

#include "G4UIdirectory.hh"
#include "G4UIcommand.hh"
#include "G4UIparameter.hh"

#include <iostream>

using namespace G4Analysis;

namespace {

void Exception(G4UIcommand* command, G4int nofParameters)
{
  G4ExceptionDescription description;
  description 
    << "Got wrong number of \"" << command->GetCommandName() 
    << "\" parameters: " << nofParameters
    << " instead of " << command->GetParameterEntries() 
    << " expected" << G4endl;
  G4Exception("G4H2Messenger::SetNewValue",
              "Analysis_W013", JustWarning, description);
}

}                  


//_____________________________________________________________________________
G4H2Messenger::G4H2Messenger(G4VAnalysisManager* manager)
  : G4UImessenger(),
    fManager(manager),
    fH2Dir(0),  
    fCreateH2Cmd(0),
    fSetH2Cmd(0),
    fSetH2TitleCmd(0), 
    fSetH2XAxisCmd(0), 
    fSetH2YAxisCmd(0)
{  
  fH2Dir = new G4UIdirectory("/analysis/h2/");
  fH2Dir->SetGuidance("2D histograms control");

  CreateH2Cmd();
  SetH2Cmd();
  
  SetH2TitleCmd();
  SetH2XAxisCmd();
  SetH2YAxisCmd();
  SetH2ZAxisCmd();
}

//_____________________________________________________________________________
G4H2Messenger::~G4H2Messenger()
{
  delete fCreateH2Cmd;
  delete fSetH2Cmd;
  delete fSetH2TitleCmd;  
  delete fSetH2XAxisCmd;  
  delete fSetH2YAxisCmd;  
  delete fSetH2ZAxisCmd;  
  delete fH2Dir;
}

//
// private functions
//

//_____________________________________________________________________________
void G4H2Messenger::CreateH2Cmd()
{
  G4UIparameter* h2Name = new G4UIparameter("name", 's', false);
  h2Name->SetGuidance("Histogram name (label)");
  
  G4UIparameter* h2Title = new G4UIparameter("title", 's', false);
  h2Title->SetGuidance("Histogram title");

  G4UIparameter* h2xNbins0 = new G4UIparameter("xnbins0", 'i', true);
  h2xNbins0->SetGuidance("Number of x-bins (default = 100)");
  h2xNbins0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2xNbins0->SetDefaultValue(100);
  
  G4UIparameter* h2xValMin0 = new G4UIparameter("xvalMin0", 'd', true);
  h2xValMin0->SetGuidance("Minimum x-value, expressed in unit (default = 0.)");
  h2xValMin0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2xValMin0->SetDefaultValue(0.);
  
  G4UIparameter* h2xValMax0 = new G4UIparameter("xvalMax0", 'd', true);
  h2xValMax0->SetGuidance("Maximum x-value, expressed in unit (default = 1.)");
  h2xValMax0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2xValMax0->SetDefaultValue(1.);

  G4UIparameter* h2xValUnit0 = new G4UIparameter("xvalUnit0", 's', true);
  h2xValUnit0->SetGuidance("The unit applied to filled x-values and xvalMin0, xvalMax0");
  h2xValUnit0->SetDefaultValue("none");
  
  G4UIparameter* h2xValFcn0 = new G4UIparameter("xvalFcn0", 's', true);
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  h2xValFcn0->SetGuidance(fcnxGuidance);
  h2xValFcn0->SetParameterCandidates("log log10 exp none");
  h2xValFcn0->SetDefaultValue("none");
  
  G4UIparameter* h2xValBinScheme0 = new G4UIparameter("xvalBinScheme0", 's', true);
  G4String xbinSchemeGuidance = "The binning scheme (linear, log).";
  h2xValBinScheme0->SetParameterCandidates("linear log");
  h2xValBinScheme0->SetGuidance(xbinSchemeGuidance);
  h2xValBinScheme0->SetDefaultValue("linear");
  
  G4UIparameter* h2yNbins0 = new G4UIparameter("ynbins0", 'i', true);
  h2yNbins0->SetGuidance("Number of y-bins (default = 100)");
  h2yNbins0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2yNbins0->SetDefaultValue(100);
  
  G4UIparameter* h2yValMin0 = new G4UIparameter("yvalMin0", 'd', true);
  h2yValMin0->SetGuidance("Minimum y-value, expressed in unit (default = 0.)");
  h2yValMin0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2yValMin0->SetDefaultValue(0.);
  
  G4UIparameter* h2yValMax0 = new G4UIparameter("yvalMax0", 'd', true);
  h2yValMax0->SetGuidance("Maximum y-value, expressed in unit (default = 1.)");
  h2yValMax0->SetGuidance("Can be reset with /analysis/h2/set command");
  h2yValMax0->SetDefaultValue(1.);

  G4UIparameter* h2yValUnit0 = new G4UIparameter("yvalUnit0", 's', true);
  h2yValUnit0->SetGuidance("The unit applied to filled y-values and yvalMin0, yvalMax0");
  h2yValUnit0->SetDefaultValue("none");
  
  G4UIparameter* h2yValFcn0 = new G4UIparameter("yvalFcn0", 's', true);
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  h2yValFcn0->SetGuidance(fcnyGuidance);
  h2yValFcn0->SetParameterCandidates("log log10 exp none");
  h2yValFcn0->SetDefaultValue("none");

  G4UIparameter* h2yValBinScheme0 = new G4UIparameter("yvalBinScheme0", 's', true);
  G4String ybinSchemeGuidance = "The binning scheme (linear, log).";
  h2yValBinScheme0->SetParameterCandidates("linear log");
  h2yValBinScheme0->SetGuidance(ybinSchemeGuidance);
  h2yValBinScheme0->SetDefaultValue("linear");
  
  fCreateH2Cmd = new G4UIcommand("/analysis/h2/create", this);
  fCreateH2Cmd->SetGuidance("Create 2D histogram");
  fCreateH2Cmd->SetParameter(h2Name);
  fCreateH2Cmd->SetParameter(h2Title);
  fCreateH2Cmd->SetParameter(h2xNbins0);
  fCreateH2Cmd->SetParameter(h2xValMin0);
  fCreateH2Cmd->SetParameter(h2xValMax0);
  fCreateH2Cmd->SetParameter(h2xValUnit0);
  fCreateH2Cmd->SetParameter(h2xValFcn0);
  fCreateH2Cmd->SetParameter(h2xValBinScheme0);
  fCreateH2Cmd->SetParameter(h2yNbins0);
  fCreateH2Cmd->SetParameter(h2yValMin0);
  fCreateH2Cmd->SetParameter(h2yValMax0);
  fCreateH2Cmd->SetParameter(h2yValUnit0);
  fCreateH2Cmd->SetParameter(h2yValFcn0);
  fCreateH2Cmd->SetParameter(h2yValBinScheme0);
  fCreateH2Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  


//_____________________________________________________________________________
void G4H2Messenger::SetH2Cmd()
{
  G4UIparameter* h2Id = new G4UIparameter("id", 'i', false);
  h2Id->SetGuidance("Histogram id");
  h2Id->SetParameterRange("id>=0");
  
  G4UIparameter* h2xNbins = new G4UIparameter("xnbins", 'i', false);
  h2xNbins->SetGuidance("Number of x-bins");
  
  G4UIparameter* h2xValMin = new G4UIparameter("xvalMin", 'd', false);
  h2xValMin->SetGuidance("Minimum x-value, expressed in unit");
  
  G4UIparameter* h2xValMax = new G4UIparameter("xvalMax", 'd', false);
  h2xValMax->SetGuidance("Maximum x-value, expressed in unit");
  
  G4UIparameter* h2xValUnit = new G4UIparameter("xvalUnit", 's', false);
  h2xValUnit->SetGuidance("The unit applied to filled x-values and xvalMin, xvalMax");
  h2xValUnit->SetDefaultValue("none");
 
  G4UIparameter* h2xValFcn = new G4UIparameter("xvalFcn", 's', false);
  h2xValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnxGuidance = "The function applied to filled x-values (log, log10, exp, none).";
  h2xValFcn->SetGuidance(fcnxGuidance);
  h2xValFcn->SetDefaultValue("none");
 
  G4UIparameter* h2xValBinScheme = new G4UIparameter("xvalBinScheme", 's', true);
  G4String xbinSchemeGuidance = "The binning scheme (linear, log).";
  h2xValBinScheme->SetParameterCandidates("linear log");
  h2xValBinScheme->SetGuidance(xbinSchemeGuidance);
  h2xValBinScheme->SetDefaultValue("linear");
  
  G4UIparameter* h2yNbins = new G4UIparameter("nybins", 'i', false);
  h2yNbins->SetGuidance("Number of y-bins");
  
  G4UIparameter* h2yValMin = new G4UIparameter("yvalMin", 'd', false);
  h2yValMin->SetGuidance("Minimum y-value, expressed in unit");
  
  G4UIparameter* h2yValMax = new G4UIparameter("yvalMax", 'd', false);
  h2yValMax->SetGuidance("Maximum y-value, expressed in unit");
  
  G4UIparameter* h2yValUnit = new G4UIparameter("yvalUnit", 's', true);
  h2yValUnit->SetGuidance("The unit applied to filled y-values and yvalMin, yvalMax");
  h2yValUnit->SetDefaultValue("none");
 
  G4UIparameter* h2yValFcn = new G4UIparameter("yvalFcn", 's', false);
  h2yValFcn->SetParameterCandidates("log log10 exp none");
  G4String fcnyGuidance = "The function applied to filled y-values (log, log10, exp, none).";
  h2yValFcn->SetGuidance(fcnyGuidance);
  h2yValFcn->SetDefaultValue("none");
 
  G4UIparameter* h2yValBinScheme = new G4UIparameter("yvalBinScheme", 's', true);
  G4String ybinSchemeGuidance = "The binning scheme (linear, log).";
  h2yValBinScheme->SetParameterCandidates("linear log");
  h2yValBinScheme->SetGuidance(ybinSchemeGuidance);
  h2yValBinScheme->SetDefaultValue("linear");

  fSetH2Cmd = new G4UIcommand("/analysis/h2/set", this);
  fSetH2Cmd->SetGuidance("Set parameters for the 2D histogram of #Id :");
  fSetH2Cmd->SetGuidance("  nxbins; xvalMin; xvalMax; xunit; xfunction; xbinScheme");
  fSetH2Cmd->SetGuidance("  nybins; yvalMin; yvalMax; yunit; yfunction; ybinScheme");
  fSetH2Cmd->SetParameter(h2Id);
  fSetH2Cmd->SetParameter(h2xNbins);
  fSetH2Cmd->SetParameter(h2xValMin);
  fSetH2Cmd->SetParameter(h2xValMax);
  fSetH2Cmd->SetParameter(h2xValUnit);
  fSetH2Cmd->SetParameter(h2xValFcn);
  fSetH2Cmd->SetParameter(h2xValBinScheme);
  fSetH2Cmd->SetParameter(h2yNbins);
  fSetH2Cmd->SetParameter(h2yValMin);
  fSetH2Cmd->SetParameter(h2yValMax);
  fSetH2Cmd->SetParameter(h2yValUnit);
  fSetH2Cmd->SetParameter(h2yValFcn);
  fSetH2Cmd->SetParameter(h2yValBinScheme);
  fSetH2Cmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H2Messenger::SetH2TitleCmd()
{
  G4UIparameter* h2Id = new G4UIparameter("idTitle", 'i', false);
  h2Id->SetGuidance("Histogram id");
  h2Id->SetParameterRange("idTitle>=0");

  G4UIparameter* h2Title = new G4UIparameter("h2Title", 's', true);
  h2Title->SetGuidance("Histogram title");
  h2Title->SetDefaultValue("none");

  fSetH2TitleCmd = new G4UIcommand("/analysis/h2/setTitle", this);
  fSetH2TitleCmd->SetGuidance("Set title for the 2D histogram of #Id");
  fSetH2TitleCmd->SetParameter(h2Id);
  fSetH2TitleCmd->SetParameter(h2Title);
  fSetH2TitleCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H2Messenger::SetH2XAxisCmd()
{
  G4UIparameter* h2Id = new G4UIparameter("idXaxis", 'i', false);
  h2Id->SetGuidance("Histogram id");
  h2Id->SetParameterRange("idXaxis>=0");

  G4UIparameter* h2XAxis = new G4UIparameter("h2Xaxis", 's', true);
  h2XAxis->SetGuidance("Histogram x-axis title");
  h2XAxis->SetDefaultValue("none");

  fSetH2XAxisCmd = new G4UIcommand("/analysis/h2/setXaxis", this);
  fSetH2XAxisCmd->SetGuidance("Set x-axis title for the 2D histogram of #Id");
  fSetH2XAxisCmd->SetParameter(h2Id);
  fSetH2XAxisCmd->SetParameter(h2XAxis);
  fSetH2XAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H2Messenger::SetH2YAxisCmd()
{
  G4UIparameter* h2Id = new G4UIparameter("idYaxis", 'i', false);
  h2Id->SetGuidance("Histogram id");
  h2Id->SetParameterRange("idYaxis>=0");

  G4UIparameter* h2YAxis = new G4UIparameter("h2Yaxis", 's', true);
  h2YAxis->SetGuidance("Histogram y-axis title");
  h2YAxis->SetDefaultValue("none");

  fSetH2YAxisCmd = new G4UIcommand("/analysis/h2/setYaxis", this);
  fSetH2YAxisCmd->SetGuidance("Set y-axis title for the 2D histogram of #Id");
  fSetH2YAxisCmd->SetParameter(h2Id);
  fSetH2YAxisCmd->SetParameter(h2YAxis);
  fSetH2YAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//_____________________________________________________________________________
void G4H2Messenger::SetH2ZAxisCmd()
{
  G4UIparameter* h2Id = new G4UIparameter("idYaxis", 'i', false);
  h2Id->SetGuidance("Histogram id");
  h2Id->SetParameterRange("idYaxis>=0");

  G4UIparameter* h2YAxis = new G4UIparameter("h2Yaxis", 's', true);
  h2YAxis->SetGuidance("Histogram y-axis title");
  h2YAxis->SetDefaultValue("none");

  fSetH2ZAxisCmd = new G4UIcommand("/analysis/h2/setYaxis", this);
  fSetH2ZAxisCmd->SetGuidance("Set y-axis title for the 2D histogram of #Id");
  fSetH2ZAxisCmd->SetParameter(h2Id);
  fSetH2ZAxisCmd->SetParameter(h2YAxis);
  fSetH2ZAxisCmd->AvailableForStates(G4State_PreInit, G4State_Idle);
}  

//
// public functions
//

//_____________________________________________________________________________
void G4H2Messenger::SetNewValue(G4UIcommand* command, G4String newValues)
{
  // tokenize parameters in a vector
  std::vector<G4String> parameters;
  G4Analysis::Tokenize(newValues, parameters);
  // check consistency
  if ( G4int(parameters.size()) != command->GetParameterEntries() ) {
    // Should never happen but let's check anyway for consistency
    Exception(command, parameters.size());
    return;
  }  

  if ( command == fCreateH2Cmd ) { 
    G4int counter = 0;
    G4String name = parameters[counter++];
    G4String title = parameters[counter++];
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    fManager->CreateH2(name, title, 
                       xnbins, xvmin*xunit, xvmax*xunit, 
                       ynbins, yvmin*yunit, yvmax*yunit, 
                       xsunit, ysunit, xsfcn, ysfcn, xsbinScheme, ysbinScheme);     
  }
  else if ( command == fSetH2Cmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]);
    G4int xnbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double xvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double xvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String xsunit = parameters[counter++];
    G4String xsfcn = parameters[counter++];
    G4String xsbinScheme = parameters[counter++];
    G4double xunit = GetUnitValue(xsunit);
    G4int ynbins = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4double yvmin = G4UIcommand::ConvertToDouble(parameters[counter++]); 
    G4double yvmax = G4UIcommand::ConvertToDouble(parameters[counter++]); ; 
    G4String ysunit = parameters[counter++];
    G4String ysfcn = parameters[counter++];
    G4String ysbinScheme = parameters[counter++];
    G4double yunit = GetUnitValue(ysunit);
    fManager->SetH2(id, 
                    xnbins, xvmin*xunit, xvmax*xunit,
                    ynbins, yvmin*yunit, yvmax*yunit, 
                    xsunit, ysunit, xsfcn, ysfcn, xsbinScheme, ysbinScheme);     
  }
  else if ( command == fSetH2TitleCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String title = parameters[counter++];
    fManager->SetH2Title(id, title);     
  }
  else if ( command == fSetH2XAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String xaxis = parameters[counter++];
    fManager->SetH2XAxisTitle(id, xaxis);     
  }
  else if ( command == fSetH2YAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String yaxis = parameters[counter++];
    fManager->SetH2YAxisTitle(id, yaxis);     
  }
  else if ( command == fSetH2ZAxisCmd ) {
    G4int counter = 0;
    G4int id = G4UIcommand::ConvertToInt(parameters[counter++]); 
    G4String zaxis = parameters[counter++];
    fManager->SetH2ZAxisTitle(id, zaxis);     
  }
}  
