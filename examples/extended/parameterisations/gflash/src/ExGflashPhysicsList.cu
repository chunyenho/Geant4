//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: ExGflashPhysicsList.cc 70923 2013-06-07 13:10:39Z gcosmo $
//
/// \file parameterisations/gflash/src/ExGflashPhysicsList.cc
/// \brief Implementation of the ExGflashPhysicsList class
//
#include "ExGflashPhysicsList.hh"

#include "globals.hh"
#include "G4ParticleDefinition.hh"
#include "G4ParticleWithCuts.hh"
#include "G4ProcessManager.hh"
#include "G4ProcessVector.hh"
#include "G4ParticleTypes.hh"
#include "G4ParticleTable.hh"
#include "G4Material.hh"
#include "G4MaterialTable.hh"
#include "G4Region.hh"
#include "G4RegionStore.hh"
#include "G4ios.hh"
#include <iomanip>   

#include "G4FastSimulationManagerProcess.hh"

using namespace std;

ExGflashPhysicsList::ExGflashPhysicsList():  G4VUserPhysicsList()
{
  SetVerboseLevel(0);
}

ExGflashPhysicsList::~ExGflashPhysicsList()
{
}

void ExGflashPhysicsList::ConstructParticle()
{
  // In this method, static member functions should be called
  // for all particles which you want to use.
  // This ensures that objects of these particle types will be
  // created in the program. 

  std::cout<<"start construct particle"<<std::endl;
  ConstructBosons();
  ConstructLeptons();
  ConstructMesons();
  ConstructBaryons();
  ConstructIons();
  std::cout<<"end construct particle"<<std::endl;
}

void ExGflashPhysicsList::ConstructBosons()
{
  // pseudo-particles
  G4Geantino::GeantinoDefinition();
  G4ChargedGeantino::ChargedGeantinoDefinition();

  // gamma
  G4Gamma::GammaDefinition();

  // optical photon
  G4OpticalPhoton::OpticalPhotonDefinition();
}

#include "G4LeptonConstructor.hh"
void ExGflashPhysicsList::ConstructLeptons()
{
  // Construct all leptons
  G4LeptonConstructor pConstructor;
  pConstructor.ConstructParticle();
}

#include "G4MesonConstructor.hh"
void ExGflashPhysicsList::ConstructMesons()
{
  //  Construct all mesons
  G4MesonConstructor pConstructor;
  pConstructor.ConstructParticle();
}

#include "G4BaryonConstructor.hh"
void ExGflashPhysicsList::ConstructBaryons()
{
  //  Construct all barions
  G4BaryonConstructor  pConstructor;
  pConstructor.ConstructParticle();  
}

#include "G4IonConstructor.hh"
void ExGflashPhysicsList::ConstructIons()
{
  //  Construct light ions
  G4IonConstructor pConstructor;
  pConstructor.ConstructParticle();  
}

void ExGflashPhysicsList::ConstructProcess()
{
  AddTransportation();
  AddParameterisation();
  std::cout<<"AddParameterisation"<<std::endl;
 
  ConstructEM();
  std::cout<<"ConstructEM"<<std::endl;
  ConstructGeneral();
}


void ExGflashPhysicsList::AddTransportation()
{
  G4VUserPhysicsList::AddTransportation();
}

#include "G4ComptonScattering.hh"
#include "G4GammaConversion.hh"
#include "G4PhotoElectricEffect.hh"

#include "G4eMultipleScattering.hh"
#include "G4MuMultipleScattering.hh"
#include "G4hMultipleScattering.hh"

#include "G4eIonisation.hh"
#include "G4eBremsstrahlung.hh"
#include "G4eplusAnnihilation.hh"

#include "G4UserLimits.hh"

#include "G4MuIonisation.hh"
#include "G4MuBremsstrahlung.hh"
#include "G4MuPairProduction.hh"

#include "G4hIonisation.hh"
void ExGflashPhysicsList::ConstructEM()
{

   G4cout<<"Physics List constructor"<<G4endl;
   SetCuts();
  theParticleIterator->reset();
  while( (*theParticleIterator)() ){
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    G4String particleName = particle->GetParticleName();
     
    if (particleName == "gamma") {
    // gamma
      // Construct processes for gamma
      G4VProcess* theGammaConversion = new G4GammaConversion();
      G4VProcess* theComptonScattering = new G4ComptonScattering();
      G4VProcess* thePhotoElectricEffect = new G4PhotoElectricEffect();     
      // G4VProcess* thegammacut = new G4UserLimits();
      //      thegammacut->SetUserMinEkine(1.0*MeV);

      pmanager->AddDiscreteProcess(theGammaConversion);
      pmanager->AddDiscreteProcess(theComptonScattering);      
      pmanager->AddDiscreteProcess(thePhotoElectricEffect);
      //  pmanager->AddProcess(thegammacut);
      //   G4cout <<"theGammaConversion" << theGammaConversion <<endl;
      //G4cout <<"theComptonScattering" << theComptonScattering <<endl;
      //G4cout <<"thePhotoElectricEffect" << thePhotoElectricEffect <<endl;

    } else if (particleName == "e-") {
    //electron
      // Construct processes for electron
      G4VProcess* theeminusMultipleScattering = new G4eMultipleScattering();
      G4VProcess* theeminusIonisation = new G4eIonisation();
      G4VProcess* theeminusBremsstrahlung = new G4eBremsstrahlung();
      //      G4VProcess* theeminuscut = new G4UserLimits();
      //  theeminuscut->SetUserMinEkine(1.0*MeV);
      // add processes
      pmanager->AddProcess(theeminusMultipleScattering);
      pmanager->AddProcess(theeminusIonisation);
      pmanager->AddProcess(theeminusBremsstrahlung);      
     
      //  pmanager->AddProcess( theeminuscut);


  
 // set ordering for AlongStepDoIt
      pmanager->SetProcessOrdering(theeminusMultipleScattering, idxAlongStep,  1);
      pmanager->SetProcessOrdering(theeminusIonisation, idxAlongStep,  2);
      // set ordering for PostStepDoIt
      pmanager->SetProcessOrdering(theeminusMultipleScattering, idxPostStep, 1);
      pmanager->SetProcessOrdering(theeminusIonisation, idxPostStep, 2);
      pmanager->SetProcessOrdering(theeminusBremsstrahlung, idxPostStep, 3);

      //G4cout <<"theeminusMultipleScattering" << theeminusMultipleScattering <<endl;
      //G4cout <<"theeminusIonisation" << theeminusIonisation <<endl;
      //G4cout <<"theeminusBremsstrahlung" << theeminusBremsstrahlung <<endl;

    } else if (particleName == "e+") {
    //positron
      // Construct processes for positron
      G4VProcess* theeplusMultipleScattering = new G4eMultipleScattering();
      G4VProcess* theeplusIonisation = new G4eIonisation();
      G4VProcess* theeplusBremsstrahlung = new G4eBremsstrahlung();
      G4VProcess* theeplusAnnihilation = new G4eplusAnnihilation();
      // add processes
      pmanager->AddProcess(theeplusMultipleScattering);
      pmanager->AddProcess(theeplusIonisation);
      pmanager->AddProcess(theeplusBremsstrahlung);
      pmanager->AddProcess(theeplusAnnihilation);
      // set ordering for AtRestDoIt
      pmanager->SetProcessOrderingToFirst(theeplusAnnihilation, idxAtRest);
      // set ordering for AlongStepDoIt
      pmanager->SetProcessOrdering(theeplusMultipleScattering, idxAlongStep,  1);
      pmanager->SetProcessOrdering(theeplusIonisation, idxAlongStep,  2);
      // set ordering for PostStepDoIt
      pmanager->SetProcessOrdering(theeplusMultipleScattering, idxPostStep, 1);
      pmanager->SetProcessOrdering(theeplusIonisation, idxPostStep, 2);
      pmanager->SetProcessOrdering(theeplusBremsstrahlung, idxPostStep, 3);
      pmanager->SetProcessOrdering(theeplusAnnihilation, idxPostStep, 4);
  
      //G4cout <<"theeplusMultipleScattering" << theeplusMultipleScattering <<endl;
      //G4cout <<"theeplusIonisation" << theeplusIonisation <<endl;
      //G4cout <<"theeplusBremsstrahlung" << theeplusBremsstrahlung <<endl;

    } else if( particleName == "mu+" || 
               particleName == "mu-"    ) {
    //muon  
     // Construct processes for muon+
     G4VProcess* aMultipleScattering = new G4MuMultipleScattering();
     G4VProcess* aBremsstrahlung = new G4MuBremsstrahlung();
     G4VProcess* aPairProduction = new G4MuPairProduction();
     G4VProcess* anIonisation = new G4MuIonisation();
      // add processes
     pmanager->AddProcess(anIonisation);
     pmanager->AddProcess(aMultipleScattering);
     pmanager->AddProcess(aBremsstrahlung);
     pmanager->AddProcess(aPairProduction);
     // set ordering for AlongStepDoIt
     pmanager->SetProcessOrdering(aMultipleScattering, idxAlongStep,  1);
     pmanager->SetProcessOrdering(anIonisation, idxAlongStep,  2);
     // set ordering for PostStepDoIt
     pmanager->SetProcessOrdering(aMultipleScattering, idxPostStep, 1);
     pmanager->SetProcessOrdering(anIonisation, idxPostStep, 2);
     pmanager->SetProcessOrdering(aBremsstrahlung, idxPostStep, 3);
     pmanager->SetProcessOrdering(aPairProduction, idxPostStep, 4);
     
   } else if ((!particle->IsShortLived()) &&
        (particle->GetPDGCharge() != 0.0) && 
        (particle->GetParticleName() != "chargedgeantino")) {
     // all others charged particles except geantino
     G4VProcess* aMultipleScattering = new G4hMultipleScattering();
     G4VProcess* anIonisation = new G4hIonisation();
     // add processes
     pmanager->AddProcess(anIonisation);
     pmanager->AddProcess(aMultipleScattering);
     // set ordering for AlongStepDoIt
     pmanager->SetProcessOrdering(aMultipleScattering, idxAlongStep,  1);
     pmanager->SetProcessOrdering(anIonisation, idxAlongStep,  2);
     // set ordering for PostStepDoIt
     pmanager->SetProcessOrdering(aMultipleScattering, idxPostStep, 1);
     pmanager->SetProcessOrdering(anIonisation, idxPostStep, 2);
    }
  }
}


#include "G4Decay.hh"
void ExGflashPhysicsList::ConstructGeneral()
{
  // Add Decay Process
  G4Decay* theDecayProcess = new G4Decay();
  //G4cout << "decay" <<theDecayProcess<<endl;
  theParticleIterator->reset();
  while( (*theParticleIterator)() ){
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    if (theDecayProcess->IsApplicable(*particle)) { 
      pmanager ->AddProcess(theDecayProcess);
      // set ordering for PostStepDoIt and AtRestDoIt
      pmanager ->SetProcessOrdering(theDecayProcess, idxPostStep);
      pmanager ->SetProcessOrdering(theDecayProcess, idxAtRest);
    }
  }  
}

//  WARNING: This methode is mandatory if U want to use GFLASH
void ExGflashPhysicsList::AddParameterisation()
{
  G4FastSimulationManagerProcess* 
    theFastSimulationManagerProcess = 
      new G4FastSimulationManagerProcess();
  G4cout << "FastSimulationManagerProcess" <<G4endl;
  theParticleIterator->reset();
  //std::cout<<"---"<<std::endl;
  while( (*theParticleIterator)() ){
    //std::cout<<"+++"<<std::endl;
  
    G4ParticleDefinition* particle = theParticleIterator->value();
    // std::cout<<"--- particle "<<particle->GetParticleName()<<std::endl;
    G4ProcessManager* pmanager = particle->GetProcessManager();
    // The fast simulation process becomes a discrete process only since 9.0:
    pmanager->AddDiscreteProcess(theFastSimulationManagerProcess);
  }
}

void ExGflashPhysicsList::SetCuts()
{
  if (verboseLevel >1){
    G4cout << "ExGflashPhysicsList::SetCuts:";
  }  
  //  " G4VUserPhysicsList::SetCutsWithDefault" method sets 
  //   the default cut value for all particle types 
  DumpCutValuesTable();
   SetCutsWithDefault();
   //  SetCutValue(100*mm, "gamma");
//   SetCutValue(0*mm, "e-");
//   SetCutValue(0*mm, "e+");

//   SetCutValue(62*mm, "gamma");
//   SetCutValue(0.73*mm, "e-");
//   SetCutValue(0.78*mm, "e+");



 
 
  DumpCutValuesTable();
// set cuts for region crystals with default Cuts
  G4Region* region = G4RegionStore::GetInstance()->GetRegion("crystals");
  region->SetProductionCuts(
          G4ProductionCutsTable::GetProductionCutsTable()->GetDefaultProductionCuts());
}









