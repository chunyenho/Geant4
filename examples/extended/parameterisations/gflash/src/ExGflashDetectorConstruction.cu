#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: ExGflashDetectorConstruction.cc 73006 2013-08-15 08:17:11Z gcosmo $
//
/// \file parameterisations/gflash/src/ExGflashDetectorConstruction.cc
/// \brief Implementation of the ExGflashDetectorConstruction class
//
// Created by Joanna Weng 26.11.2004

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include <iostream>
// G4 Classes
#include "globals.hh"
#include "G4ThreeVector.hh"
#include "G4PVPlacement.hh"
#include "G4VPhysicalVolume.hh"
#include "G4LogicalVolume.hh"
#include "G4VisAttributes.hh"
#include "G4Box.hh"
#include "G4NistManager.hh"
#include "G4SDManager.hh"
#include "G4Material.hh"
#include "G4GeometryManager.hh"
#include "G4PhysicalVolumeStore.hh"
#include "G4LogicalVolumeStore.hh"
#include "G4SolidStore.hh"
#include "G4Colour.hh"
#include "G4SystemOfUnits.hh"

// User Classes
#include "ExGflashDetectorConstruction.hh"
#include "ExGflashSensitiveDetector.hh"

//fast simulation
#include "GFlashHomoShowerParameterisation.hh"
#include "G4FastSimulationManager.hh"
#include "GFlashShowerModel.hh"
#include "GFlashHitMaker.hh"
#include "GFlashParticleBounds.hh"

using namespace std;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

ExGflashDetectorConstruction::ExGflashDetectorConstruction()
  :fExperimentalHall_log(0), 
   fCalo_log(0), 
   fExperimentalHall_phys(0), 
   fCalo_phys(0),
   fTheParameterisation(0),
   fTheHMaker(0),
   fTheParticleBounds(0),
   fTheFastShowerModel(0)
{
  G4cout<<"ExGflashDetectorConstruction::Detector constructor"<<G4endl;
  
  // Simplified `CMS-like` PbWO4 crystal calorimeter  
  fCalo_xside=31*cm;
  fCalo_yside=31*cm;
  fCalo_zside=24*cm; 
  
  // GlashStuff
  //Energy Cuts to kill particles
//  fTheParticleBounds  = new GFlashParticleBounds();
  // Makes the EnergieSpots
//  fTheHMaker          = new GFlashHitMaker();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

ExGflashDetectorConstruction::~ExGflashDetectorConstruction()
{ 
  //@@@ ExGflashDetectorConstruction::Soll ich alles dlete
  
  // -- !! this is not properly deleted in MT where
  // -- !! as there is one parameterisation/thread
  // -- !! and only the last one is remembered.
  if ( fTheParameterisation ) delete fTheParameterisation;
  if ( fTheParticleBounds )   delete fTheParticleBounds;
  if ( fTheHMaker )           delete fTheHMaker;
  if ( fTheFastShowerModel )  delete fTheFastShowerModel;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4VPhysicalVolume* ExGflashDetectorConstruction::Construct()
{
  //--------- Definitions of Solids, Logical Volumes, Physical Volumes ---------
  G4cout << "Defining the materials" << G4endl;
  // Get nist material manager
  G4NistManager* nistManager = G4NistManager::Instance();
  // Build materials
  G4Material* air   = nistManager->FindOrBuildMaterial("G4_AIR");
  G4Material* pbWO4 = nistManager->FindOrBuildMaterial("G4_PbWO4");
  
  
  /*******************************
   * The Experimental Hall       *
   *******************************/
  fExperimentalHall_x=1000.*cm;
  fExperimentalHall_y=1000.*cm;
  fExperimentalHall_z=1000.*cm;
  
  fExperimentalHall_box = new G4Box("expHall_box",              // World Volume
                                    fExperimentalHall_x,        // x size
                                    fExperimentalHall_y,        // y size
                                    fExperimentalHall_z);       // z size
  
  fExperimentalHall_log = new G4LogicalVolume(fExperimentalHall_box,
                                              air,
                                              "expHall_log",
                                              0,       //opt: fieldManager
                                              0,       //opt: SensitiveDetector
                                              0);      //opt: UserLimits
  fExperimentalHall_phys = new G4PVPlacement(0,
                                             G4ThreeVector(),   //at (0,0,0)
                                             "expHall",
                                             fExperimentalHall_log,
                                             0,
                                             false,
                                             0);
  
  
  //------------------------------ 
  // Calorimeter segments
  //------------------------------
  // Simplified `CMS-like` PbWO4 crystal calorimeter  
  
  fNbOfCrystals = 10;  // this are the crystals PER ROW in this example 
                       // cube of 10 x 10 crystals 
                       // don't change it @the moment, since 
                       // the readout in event action assumes this 
                       // dimensions and is not automatically adapted
                       // in this version of the example :-( 
  fCrystalWidth = 3*cm;
  fCrystalLenght= 24*cm;
  fCalo_xside=(fCrystalWidth*fNbOfCrystals)+1*cm;
  fCalo_yside=(fCrystalWidth*fNbOfCrystals)+1*cm;
  fCalo_zside=fCrystalLenght;
  
  G4Box *calo_box= new G4Box("CMS calorimeter",  // its name
                             fCalo_xside/2.,     // size
                             fCalo_yside/2.,
                             fCalo_zside/2.);
  fCalo_log = new G4LogicalVolume(calo_box,      // its solid
                                  air,           // its material
                                  "calo log",    // its name
                                  0,             // opt: fieldManager
                                  0,             // opt: SensitiveDetector 
                                  0);            // opt: UserLimit
  
  G4double Xpos = 0.0;
  G4double Ypos = 0.0;
  G4double Zpos = 100.0*cm;
  
  fCalo_phys = new G4PVPlacement(0,
                                 G4ThreeVector(Xpos,Ypos,Zpos),
                                 fCalo_log,
                                 "calorimeter",
                                 fExperimentalHall_log,
                                 false,
                                 1);
  //Visibility
  for (int i=0; i<fNbOfCrystals;i++)
    {
      
      for (int j=0; j<fNbOfCrystals;j++)
        {  
          int n =  i*10+j;
          fCrystal[n]= new G4Box("Crystal",                     // its name
                                 fCrystalWidth/2,
                                 fCrystalWidth/2,
                                 fCrystalLenght/2);             // size
          fCrystal_log[n] = new G4LogicalVolume(fCrystal[n],    // its solid
                                                pbWO4,          // its material
                                                "Crystal_log"); // its name
          G4ThreeVector crystalPos((i*fCrystalWidth)-135,
                                   (j*fCrystalWidth)-135,0 );
          fCrystal_phys[n] = new G4PVPlacement(0,               // no rotation
                                               crystalPos,      // translation
                                               fCrystal_log[n],
                                               "crystal",       // its name
                                               fCalo_log,
                                               false,
                                               1);
        }
    }  
  G4cout << "There are " << fNbOfCrystals <<
    " crystals per row in the calorimeter, so in total "<<
    fNbOfCrystals*fNbOfCrystals << " crystals" << G4endl;  
  G4cout << "The have widthof  " << fCrystalWidth /cm <<
    "  cm and a lenght of  " <<  fCrystalLenght /cm
         <<" cm. The Material is "<< pbWO4 << G4endl;
  
  
  fExperimentalHall_log->SetVisAttributes(G4VisAttributes::Invisible);
  G4VisAttributes* CaloVisAtt = new G4VisAttributes(G4Colour(1.0,1.0,1.0));
  G4VisAttributes* CrystalVisAtt = new G4VisAttributes(G4Colour(1.0,1.0,0.0));
  fCalo_log->SetVisAttributes(CaloVisAtt);
  for (int i=0; i<100;i++)
    {
      fCrystal_log[i]->SetVisAttributes(CrystalVisAtt);
    }
  // define the parameterisation region
  fRegion = new G4Region("crystals");
  fCalo_log->SetRegion(fRegion);
  fRegion->AddRootLogicalVolume(fCalo_log);
  
  return fExperimentalHall_phys;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void ExGflashDetectorConstruction::ConstructSDandField()
{
  // -- sensitive detectors:
  G4SDManager* SDman = G4SDManager::GetSDMpointer();
  ExGflashSensitiveDetector* CaloSD=
    new ExGflashSensitiveDetector("Calorimeter",this);
  SDman->AddNewDetector(CaloSD);
  
  for (int i=0; i<100;i++)
    {
      fCrystal_log[i]->SetSensitiveDetector(CaloSD);
    }
  
  // Get nist material manager
  G4NistManager* nistManager = G4NistManager::Instance();
  G4Material*          pbWO4 = nistManager->FindOrBuildMaterial("G4_PbWO4");
  // -- fast simulation models:
  // **********************************************
  // * Initializing shower modell
  // ***********************************************
  G4cout << "Creating shower parameterization models" << G4endl;
  GFlashShowerModel* lTheFastShowerModel =
                            new GFlashShowerModel("fastShowerModel",fRegion);
  GFlashHomoShowerParameterisation* lTheParameterisation =
    new GFlashHomoShowerParameterisation(pbWO4);
  lTheFastShowerModel->SetParameterisation(*lTheParameterisation);
  // Energy Cuts to kill particles:
  GFlashParticleBounds* lTheParticleBounds  = new GFlashParticleBounds();
  lTheFastShowerModel->SetParticleBounds(*lTheParticleBounds);
  // Makes the EnergieSpots
  GFlashHitMaker* lTheHMaker          = new GFlashHitMaker();
  lTheFastShowerModel->SetHitMaker(*lTheHMaker);
  G4cout<<"end shower parameterization."<<G4endl;
  // **********************************************
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
