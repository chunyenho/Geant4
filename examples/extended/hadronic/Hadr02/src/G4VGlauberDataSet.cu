
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// *                                                                  *
// * Parts of this code which have been  developed by QinetiQ Ltd     *
// * under contract to the European Space Agency (ESA) are the        *
// * intellectual property of ESA. Rights to use, copy, modify and    *
// * redistribute this software for general public use are granted    *
// * in compliance with any licensing, distribution and development   *
// * policy adopted by the Geant4 Collaboration. This code has been   *
// * written by QinetiQ Ltd for the European Space Agency, under ESA  *
// * contract 19770/06/NL/JD (Technology Research Programme).         *
// *                                                                  *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file hadronic/Hadr02/src/G4VGlauberDataSet.cc
/// \brief Implementation of the G4VGlauberDataSet class
//
// $Id: G4VGlauberDataSet.cc 77519 2013-11-25 10:54:57Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4VGlauberDataSet.cc
//
// Version:             0.B
// Date:                02/04/08
// Author:              P R Truscott
// Organisation:        QinetiQ Ltd, UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            19770/06/NL/JD
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#ifdef G4_USE_DPMJET


#include "G4VGlauberDataSet.hh"

#include "G4DPMJET2_5Interface.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

#include <iomanip>
#include <iostream>

///////////////////////////////////////////////////////////////////////////////
//
// G4G4VGlauberDataSet
//
// Constructor simply resets all variables to zero.
//
G4VGlauberDataSet::G4VGlauberDataSet()
{
  rproj = 0.0;
  rtarg = 0.0;
  bstep = 0.0;
  bmax  = 0.0;
  AP    = 0;
  ZP    = 0;
  AT    = 0;
  ZT    = 0;
  
  maxArray = 200;
  maxig    = 24;

  glauberDataSetType = -1;
  
  verboseLevel = 0;
}
///////////////////////////////////////////////////////////////////////////////
//
// ~G4G4VGlauberDataSet
//
// If you thought the contructor was boring, the destructor is even worse!.
// It doesn't do anything.
//
G4VGlauberDataSet::~G4VGlauberDataSet()
{}
////////////////////////////////////////////////////////////////////////////////
//
G4int G4VGlauberDataSet::GetAP () const
{
  return AP;
}
////////////////////////////////////////////////////////////////////////////////
//
G4int G4VGlauberDataSet::GetZP () const
{
  return ZP;
}
////////////////////////////////////////////////////////////////////////////////
//
G4int G4VGlauberDataSet::GetAT () const
{
  return AT;
}
////////////////////////////////////////////////////////////////////////////////
//
G4int G4VGlauberDataSet::GetZT () const
{
  return ZT;
}
////////////////////////////////////////////////////////////////////////////////
//
void G4VGlauberDataSet::SetArrayPointer (const G4int i)
{
  if (i<0 || i>=maxArray) {
    G4cerr <<"WARNING G4G4VGlauberDataSet::SetArrayPointer" <<G4endl;
    G4cerr <<"ATTEMPT TO SET POINTER TO VALUE OUTSIDE [0,"
           <<maxArray-1
           <<"]"
           <<G4endl;
  }
  else {
    arrayPtrn = baseArrayPtrn + i;
    arrayPtrm = baseArrayPtrm + i;
  }
}
////////////////////////////////////////////////////////////////////////////////
//
G4double * G4VGlauberDataSet::GetArrayPointerN (const G4double ppn)
{
  if (ppn < 1.0E-10) return baseArrayPtrn;
  else               return arrayPtrn;
}
////////////////////////////////////////////////////////////////////////////////
//
G4double * G4VGlauberDataSet::GetArrayPointerM (const G4double ppn)
{
  if (ppn < 1.0E-10) return baseArrayPtrm;
  else               return arrayPtrm;
}
////////////////////////////////////////////////////////////////////////////////
//
G4int G4VGlauberDataSet::GetGlauberDataSetType () const
{
  return glauberDataSetType;
}
////////////////////////////////////////////////////////////////////////////////
//
std::ofstream & G4VGlauberDataSet::WriteDataToFile (std::ofstream &File) const
{
//
//
// Dummy member function;
//
  return File;
}
std::ifstream & G4VGlauberDataSet::ReadDataFromFile (std::ifstream &File)
{
//
//
// Dummy member function;
//
  return File;
}
///////////////////////////////////////////////////////////////////////////////
//
// operator <<
//
// Output file-stream operator.  This is intended to match the standard
// GLAUBER data file format.
//
std::ofstream & operator << (std::ofstream &File, const G4VGlauberDataSet &q)
{
  File.unsetf(std::ios::scientific);
  File.setf(std::ios::fixed|std::ios::right|std::ios::adjustfield);
  File.precision(0);
  File <<std::setw(1) <<q.glauberDataSetType
       <<"NUCLEUS  "
       <<std::setw(10) <<q.AT
       <<std::setw(10) <<q.ZT
       <<std::setw(10) <<q.AP
       <<std::setw(10) <<q.ZP
       <<G4endl;

  File.unsetf(std::ios::fixed);
  File.setf(std::ios::fixed|std::ios::right|std::ios::adjustfield);
  File.precision(5);

  File <<std::setw(10) <<q.bmax
       <<std::setw(10) <<q.bstep
       <<std::setw(10) <<q.rproj
       <<std::setw(10) <<q.rtarg
       <<G4endl;

  File.precision(8);

  return q.WriteDataToFile (File);
}
///////////////////////////////////////////////////////////////////////////////
//
// operator >>
//
// Input file-stream operator.  This is assumed to the format matches the
// standard GLAUBER data file format.
//
std::ifstream & operator >> (std::ifstream &File, G4VGlauberDataSet &q)
{
  G4String dummy;
  File >>dummy
       >>q.AT
       >>q.ZT
       >>q.AP
       >>q.ZP;

  File >>q.bmax
       >>q.bstep
       >>q.rproj
       >>q.rtarg;

  return q.ReadDataFromFile (File);
}
#endif
