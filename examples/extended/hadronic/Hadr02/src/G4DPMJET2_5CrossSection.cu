
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// *                                                                  *
// * Parts of this code which have been  developed by QinetiQ Ltd     *
// * under contract to the European Space Agency (ESA) are the        *
// * intellectual property of ESA. Rights to use, copy, modify and    *
// * redistribute this software for general public use are granted    *
// * in compliance with any licensing, distribution and development   *
// * policy adopted by the Geant4 Collaboration. This code has been   *
// * written by QinetiQ Ltd for the European Space Agency, under ESA  *
// * contract 19770/06/NL/JD (Technology Research Programme).         *
// *                                                                  *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file hadronic/Hadr02/src/G4DPMJET2_5CrossSection.cc
/// \brief Implementation of the G4DPMJET2_5CrossSection class
//
// $Id: G4DPMJET2_5CrossSection.cc 81932 2014-06-06 15:39:45Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4DPMJET2_5CrossSection.cc
//
// Version:             0.A
// Date:                02/04/08
// Author:              P R Truscott
// Organisation:        QinetiQ Ltd, UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            19770/06/NL/JD
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#ifdef G4_USE_DPMJET


#include "G4DPMJET2_5CrossSection.hh"
#include "G4DynamicParticle.hh"
#include "G4NucleiProperties.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "G4HadronicException.hh"

#include "globals.hh"

#include <iomanip>
#include <fstream>
#include <sstream>

#include "G4DynamicParticle.hh"

using namespace std;

///////////////////////////////////////////////////////////////////////////////
//
G4DPMJET2_5CrossSection::G4DPMJET2_5CrossSection ():
  upperLimit ( 1000.0 * TeV ), lowerLimit ( 5.0 * GeV ), maxA(240)
{
  theCrossSectionIndex.clear();
  Initialise();
  //
  //
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
  // This next bit is provisional, stating that this cross-section estimator
  // is applicable to hydrogen targets.  However, the cross-section will be
  // set to zero.
  //
  ATmin = 1;
  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  //
}
///////////////////////////////////////////////////////////////////////////////
//
G4DPMJET2_5CrossSection::~G4DPMJET2_5CrossSection ()
{
  //
  // Go through the list of cross-section fit parameters and delete the arrays.
  //
  G4cout << "G4DPMJET2_5CrossSection::~G4DPMJET2_5CrossSection" << G4endl;
  G4cout << "Size: " << theCrossSectionIndex.size() << G4endl;
  /*  
  if(theCrossSectionIndex.size() > 0) {

    G4DPMJET2_5CrossSectionIndex::iterator it;
    for (it=theCrossSectionIndex.begin(); it!=theCrossSectionIndex.end(); ++it)
      {
        G4DPMJET2_5CrossSectionParamSet *ptr = it->second;
        for (G4DPMJET2_5CrossSectionParamSet *ptr1=ptr; ptr1<ptr+maxA; ptr1++)
          { delete ptr1; }
      }
  }
  */
}
///////////////////////////////////////////////////////////////////////////////
//
G4bool 
G4DPMJET2_5CrossSection::IsIsoApplicable(const G4DynamicParticle* theProjectile, 
                                         G4int, G4int AT,
                                         const G4Element*, const G4Material*)
{
  G4bool result = false;
  G4int AP = theProjectile->GetDefinition()->GetBaryonNumber();
  if(AP >= 1) {
    G4double EPN = theProjectile->GetKineticEnergy()/G4double(AP);
    result = (EPN >= lowerLimit && EPN <= upperLimit &&
              AT  >= ATmin      && AT  <= ATmax &&
              AP  >= APmin      && AP  <= APmax);
  }
  return result;
}

///////////////////////////////////////////////////////////////////////////////
//
G4double 
G4DPMJET2_5CrossSection::GetIsoCrossSection(
    const G4DynamicParticle* theProjectile, 
    G4int ZZ, G4int AT, const G4Isotope*,
    const G4Element*, const G4Material*)
{
  //
  // Initialise the result.
  G4double result = 0.0;
  //
  //
  // Get details of the projectile and target (nucleon number, atomic number,
  // kinetic enery and energy/nucleon.
  //
  G4int AP     = theProjectile->GetDefinition()->GetBaryonNumber();
  G4double TP  = theProjectile->GetKineticEnergy();
  G4double EPN = TP /G4double(AP);

  if (AT < ATmin || AT > ATmax || AP < APmin || AP > APmax ||
      EPN < lowerLimit || EPN > upperLimit)
  {
    G4cout <<G4endl;
    G4cout <<"ERROR IN G4DPMJET2_5CrossSection::GetIsoZACrossSection" <<G4endl;
    G4cout <<"ATTEMPT TO USE CROSS-SECTION OUTSIDE OF RANGE"          <<G4endl;
    G4cout <<"NUCLEON NUMBER OF PROJECTILE = " <<AP                   <<G4endl;
    G4cout <<"NUCLEON NUMBER OF TARGET     = " <<AT                   <<G4endl;
    G4cout <<"ENERGY PER NUCLEON           = " <<EPN*MeV              <<G4endl;
    G4cout <<"ACCEPTABLE RANGE FOR AP      = " <<APmin
           <<" TO "                            <<APmax                <<G4endl;
    G4cout <<"ACCEPTABLE RANGE FOR AT      = " <<ATmin
           <<" TO "                            <<ATmax                <<G4endl;
    G4cout <<"ACCEPTABLE RANGE FOR ENERGY  = " <<lowerLimit
           <<" MeV/n TO "                      <<upperLimit
           <<" MeV/n" <<G4endl;
    G4cout <<G4endl;
    return result;
  }
  //
  //
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
  // This next bit is provisional, stating that this cross-section hydrogen
  // targets is zero.
  //
  if ( AT == 1 ) return 0.0;
  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  //
  //
  //
  // Results are parameterised as a function of the natural logarithm of the
  // centre of mass energy of the projectile and target system.
  //
  G4double sigma = 0.0;
  G4double mT    = G4NucleiProperties::GetNuclearMass(AT, ZZ);
  G4double EP    = theProjectile->GetTotalEnergy();
  G4double mP    = theProjectile->GetDefinition()->GetPDGMass();
  G4double lnECM = std::log(std::sqrt(mP*mP + mT*mT + 2.0*mT*EP));
  G4DPMJET2_5CrossSectionIndex::iterator it = theCrossSectionIndex.find(AT);
  if (it != theCrossSectionIndex.end())
  {
    G4DPMJET2_5CrossSectionParamSet *ptr = (it->second) + AP;
    G4double cc0 = (*ptr)[0];
    G4double cc1 = (*ptr)[1];
    G4double cc2 = (*ptr)[2];
    sigma = cc0 + cc1*lnECM + cc2*lnECM*lnECM;
    sigma = sigma * millibarn;
    if (verboseLevel >= 2) {
      G4cout <<"***************************************************************"
             <<G4endl;
      G4cout <<"G4DPMJET2_5CrossSection::GetIsoCrossSection" <<G4endl;
      G4cout <<"PROJECTILE    = "
             <<theProjectile->GetDefinition()->GetParticleName() <<G4endl;
      G4cout <<"TARGET (A,Z)  = (" <<AT <<"," <<ZZ <<")" <<G4endl;
      G4cout <<"K. ENERGY/NUC = " <<EPN/MeV <<" MeV/n" <<G4endl;
      G4cout <<"CROSS SECTION = " <<sigma/millibarn <<" MILLIBARNS" <<G4endl;
      G4cout <<"***************************************************************"
             <<G4endl;
    }
  }
  else
  {
    G4cout <<G4endl;
    G4cout <<"ERROR IN G4DPMJET2_5CrossSection::GetIsoCrossSection" <<G4endl;
    G4cout <<"NO CROSS-SECTION FIT DATA LOADED FOR AT = " <<AT        <<G4endl;
    G4cout <<G4endl;
  }
  
  return sigma;
}
///////////////////////////////////////////////////////////////////////////////
//
void G4DPMJET2_5CrossSection::Initialise ()
{
  //verboseLevel = 2;
  //
  //
  // Determine first if the environment variable G4DPMJET2_5DATA is set.  If not
  // then ask for it to be set and call exception.
  //
  if ( !getenv("G4DPMJET2_5DATA") )
  {
    G4cout <<"ENVIRONMENT VARIABLE G4DPMJET2_5DATA NOT SET " <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, 
      "Please setenv G4DPMJET2_5DATA to point to the dpmjet2.5 data files.");
  }

  G4String filename = G4String(getenv("G4DPMJET2_5DATA")) + "/" +
    "GlauberCrossSections.dat";

  std::ifstream glauberXSFile(filename);
  if (glauberXSFile) {
    //
    //
    // Glaubercross-section file does exist, so read in maximum and minimum A
    // for target and projectile.
    //
    glauberXSFile >>APmin >>APmax >>ATmin >>ATmax;
    //
    //
    // Determine the list of targets based on the G4ElementList.  The list of
    // target nucleon numbers is stored as a ket to the map theCrossSectionIndex.
    // G4double[240][3] array objects are created to allow storage of the 
    // cross-section fit parameters.
    //
    const G4ElementTable *theElementTable = G4Element::GetElementTable();
    G4ElementTable::const_iterator it;
    for (it=theElementTable->begin(); it!=theElementTable->end(); it++)
      {
        G4int nIso = (*it)->GetNumberOfIsotopes();
        G4IsotopeVector* isoVector = (*it)->GetIsotopeVector();
        for (G4int i = 0; i < nIso; i++)
          {
            G4int AA = (*isoVector)[i]->GetN();
            if (theCrossSectionIndex.count(AA) == 0 && 
                AA >= ATmin && AA <= ATmax)
              {
//
//
// Whilst the use of std::map should eliminate duplication of keys, we need to
// know whether isotope's with the same nucleon number have been declared before
// creating the large arrays, hence the use of the "count" member function.
//
                G4DPMJET2_5CrossSectionParamSet *a =
                  new G4DPMJET2_5CrossSectionParamSet[maxA];
                theCrossSectionIndex.insert(
                  G4DPMJET2_5CrossSectionIndex::value_type(AA,a));
              }
          }
      }
    
    //
    //
    // Now proceed to read in the remainder of the GlauberCrossSection.dat file,
    // loading into theCrossSectionIndex any relevant fitting parameters to the
    // target nuclei.
    //
    char inputChars[80]={' '};
    G4String inputLine;
    while (-glauberXSFile.getline(inputChars, 80).eof() != EOF)
      {
        inputLine = inputChars;
        if (inputLine.length() != 0)
          {
            std::istringstream tmpStream(inputLine);
            G4int AP, AT;
            G4double cc0, cc1, cc2;
            tmpStream >>AP >>AT >>cc0 >>cc1 >>cc2;
            G4DPMJET2_5CrossSectionIndex::iterator IT = 
              theCrossSectionIndex.find(AT);
            if (IT != theCrossSectionIndex.end())
              {
                G4DPMJET2_5CrossSectionParamSet *ptr = (IT->second) + AP;
                *ptr = G4DPMJET2_5CrossSectionParamSet(cc0,cc1,cc2);
              }
          }
      }

    glauberXSFile.close();
    G4cout << "G4DPMJET2_5CrossSection::Initialise () done!" << G4endl;
  } else {
    G4cout <<"GlauberCrossSections.dat DOES NOT EXIST" <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, 
  "GlauberCrossSections.dat should be located in $G4DPMJET2_5DATA directory.");
  }
}
///////////////////////////////////////////////////////////////////////////////
//
void G4DPMJET2_5CrossSection::BuildPhysicsTable(const G4ParticleDefinition&)
{;}
///////////////////////////////////////////////////////////////////////////////
//
void G4DPMJET2_5CrossSection::DumpPhysicsTable(const G4ParticleDefinition 
  &theProjectile)
{
  const G4int AP    = G4lrint(theProjectile.GetBaryonNumber());
  G4cout <<G4endl;
  G4cout <<"G4DPMJET2_5CrossSection::DumpPhysicsTable" <<G4endl;
  G4cout <<"DUMPING CROSS-SECTION FITTING COEFFICIENTS FOR AP = "
         <<AP <<G4endl;
  G4cout <<G4endl;
  G4cout <<"   AT"
         <<"             c0"
         <<"             c1"
         <<"             c2"
         <<G4endl;
  G4DPMJET2_5CrossSectionIndex::iterator it;
  for (it=theCrossSectionIndex.begin(); it!=theCrossSectionIndex.end(); it++)
  {
    G4cout.unsetf(std::ios::scientific);
    G4cout.setf(std::ios::fixed|std::ios::right|std::ios::adjustfield);
    G4cout.precision(0);
    G4cout <<std::setw(5)  <<it->first;

    G4cout.unsetf(std::ios::fixed);
    G4cout.setf(std::ios::scientific|std::ios::right|std::ios::adjustfield);
    G4cout.precision(7);
    G4DPMJET2_5CrossSectionParamSet *ptr = (it->second) + AP;
    G4cout <<std::setw(15) <<(*ptr)[0]
           <<std::setw(15) <<(*ptr)[1]
           <<std::setw(15) <<(*ptr)[2]
           <<G4endl;
  }
  G4cout.setf(std::ios::fixed);
}
#endif
