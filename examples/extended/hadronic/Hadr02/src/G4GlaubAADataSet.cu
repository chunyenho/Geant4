
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// *                                                                  *
// * Parts of this code which have been  developed by QinetiQ Ltd     *
// * under contract to the European Space Agency (ESA) are the        *
// * intellectual property of ESA. Rights to use, copy, modify and    *
// * redistribute this software for general public use are granted    *
// * in compliance with any licensing, distribution and development   *
// * policy adopted by the Geant4 Collaboration. This code has been   *
// * written by QinetiQ Ltd for the European Space Agency, under ESA  *
// * contract 19770/06/NL/JD (Technology Research Programme).         *
// *                                                                  *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file hadronic/Hadr02/src/G4GlaubAADataSet.cc
/// \brief Implementation of the G4GlaubAADataSet class
//
// $Id: G4GlaubAADataSet.cc 77519 2013-11-25 10:54:57Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4GlaubAADataSet.cc
//
// Version:             0.B
// Date:                02/04/08
// Author:              P R Truscott
// Organisation:        QinetiQ Ltd, UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            19770/06/NL/JD
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#ifdef G4_USE_DPMJET


#include "G4GlaubAADataSet.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

#include "G4DPMJET2_5Interface.hh"

#include <iomanip>
#include <iostream>

///////////////////////////////////////////////////////////////////////////////
//
// G4GlaubAADataSet
//
// Constructor simply resets all variables to zero.
//
G4GlaubAADataSet::G4GlaubAADataSet() : G4VGlauberDataSet()
{
  ZP    = -1;
  ZT    = -1;

  DefineAZStabilityLine();
}
///////////////////////////////////////////////////////////////////////////////
//
// ~G4GlaubAADataSet
//
// If you thought the contructor was boring, the destructor is even worse!.
// It doesn't do anything.
//
G4GlaubAADataSet::~G4GlaubAADataSet()
{}
////////////////////////////////////////////////////////////////////////////////
//
G4bool G4GlaubAADataSet::CreateGlauberData (const G4int , const G4int )
{
//
//
// Dummy member function.
//
  return false;
}
void G4GlaubAADataSet::DefineAZStabilityLine ()
{
        stabZ[0]  = 0;
        stabZ[1]  = 1;
        stabZ[2]  = 1;
        stabZ[3]  = 2;
        stabZ[4]  = 2;
        stabZ[5]  = 3;
        stabZ[6]  = 3;
        stabZ[7]  = 3;
        stabZ[8]  = 3;
        stabZ[9]  = 4;
        stabZ[10] = 5;
        stabZ[11] = 5;
        stabZ[12] = 6;
        stabZ[13] = 6;
        stabZ[14] = 7;
        stabZ[15] = 7;
        stabZ[16] = 8;
        stabZ[17] = 8;
        stabZ[18] = 8;
        stabZ[19] = 9;
        stabZ[20] = 10;
        stabZ[21] = 10;
        stabZ[22] = 10;
        stabZ[23] = 11;
        stabZ[24] = 12;
        stabZ[25] = 12;
        stabZ[26] = 12;
        stabZ[27] = 13;
        stabZ[28] = 13;
        stabZ[29] = 14;
        stabZ[30] = 14;
        stabZ[31] = 15;
        stabZ[32] = 16;
        stabZ[33] = 16;
        stabZ[34] = 16;
        stabZ[35] = 17;
        stabZ[36] = 17;
        stabZ[37] = 17;
        stabZ[38] = 18;
        stabZ[39] = 19;
        stabZ[40] = 19;
        stabZ[41] = 19;
        stabZ[42] = 20;
        stabZ[43] = 20;
        stabZ[44] = 20;
        stabZ[45] = 21;
        stabZ[46] = 21;
        stabZ[47] = 22;
        stabZ[48] = 21;
        stabZ[49] = 22;
        stabZ[50] = 23;
        stabZ[51] = 23;
        stabZ[52] = 24;
        stabZ[53] = 24;
        stabZ[54] = 25;
        stabZ[55] = 25;
        stabZ[56] = 26;
        stabZ[57] = 26;
        stabZ[58] = 27;
        stabZ[59] = 27;
        stabZ[60] = 28;
        stabZ[61] = 28;
        stabZ[62] = 28;
        stabZ[63] = 29;
        stabZ[64] = 29;
        stabZ[65] = 29;
        stabZ[66] = 30;
        stabZ[67] = 30;
        stabZ[68] = 30;
        stabZ[69] = 31;
        stabZ[70] = 31;
        stabZ[71] = 31;
        stabZ[72] = 32;
        stabZ[73] = 32;
        stabZ[74] = 32;
        stabZ[75] = 33;
        stabZ[76] = 33;
        stabZ[77] = 34;
        stabZ[78] = 35;
        stabZ[79] = 35;
        stabZ[80] = 35;
        stabZ[81] = 35;
        stabZ[82] = 35;
        stabZ[83] = 36;
        stabZ[84] = 37;
        stabZ[85] = 37;
        stabZ[86] = 37;
        stabZ[87] = 37;
        stabZ[88] = 38;
        stabZ[89] = 39;
        stabZ[90] = 40;
        stabZ[91] = 40;
        stabZ[92] = 41;
        stabZ[93] = 41;
        stabZ[94] = 41;
        stabZ[95] = 42;
        stabZ[96] = 42;
        stabZ[97] = 42;
        stabZ[98] = 43;
        stabZ[99] = 44;
        stabZ[100]= 43;
        stabZ[101]= 44;
        stabZ[102]= 45;
        stabZ[103]= 45;
        stabZ[104]= 45;
        stabZ[105]= 46;
        stabZ[106]= 47;
        stabZ[107]= 47;
        stabZ[108]= 47;
        stabZ[109]= 47;
        stabZ[110]= 47;
        stabZ[111]= 48;
        stabZ[112]= 49;
        stabZ[113]= 49;
        stabZ[114]= 49;
        stabZ[115]= 49;
        stabZ[116]= 49;
        stabZ[117]= 50;
        stabZ[118]= 50;
        stabZ[119]= 50;
        stabZ[120]= 51;
        stabZ[121]= 51;
        stabZ[122]= 51;
        stabZ[123]= 51;
        stabZ[124]= 52;
        stabZ[125]= 52;
        stabZ[126]= 53;
        stabZ[127]= 53;
        stabZ[128]= 53;
        stabZ[129]= 54;
        stabZ[130]= 54;
        stabZ[131]= 54;
        stabZ[132]= 55;
        stabZ[133]= 55;
        stabZ[134]= 55;
        stabZ[135]= 56;
        stabZ[136]= 55;
        stabZ[137]= 56;
        stabZ[138]= 57;
        stabZ[139]= 57;
        stabZ[140]= 58;
        stabZ[141]= 59;
        stabZ[142]= 59;
        stabZ[143]= 60;
        stabZ[144]= 61;
        stabZ[145]= 60;
        stabZ[146]= 60;
        stabZ[147]= 62;
        stabZ[148]= 61;
        stabZ[149]= 62;
        stabZ[150]= 61;
        stabZ[151]= 63;
        stabZ[152]= 63;
        stabZ[153]= 63;
        stabZ[154]= 63;
        stabZ[155]= 64;
        stabZ[156]= 64;
        stabZ[157]= 64;
        stabZ[158]= 65;
        stabZ[159]= 65;
        stabZ[160]= 65;
        stabZ[161]= 66;
        stabZ[162]= 67;
        stabZ[163]= 66;
        stabZ[164]= 67;
        stabZ[165]= 67;
        stabZ[166]= 68;
        stabZ[167]= 68;
        stabZ[168]= 69;
        stabZ[169]= 69;
        stabZ[170]= 69;
        stabZ[171]= 70;
        stabZ[172]= 70;
        stabZ[173]= 70;
        stabZ[174]= 70;
        stabZ[175]= 71;
        stabZ[176]= 71;
        stabZ[177]= 72;
        stabZ[178]= 72;
        stabZ[179]= 72;
        stabZ[180]= 73;
        stabZ[181]= 73;
        stabZ[182]= 74;
        stabZ[183]= 74;
        stabZ[184]= 75;
        stabZ[185]= 75;
        stabZ[186]= 75;
        stabZ[187]= 76;
        stabZ[188]= 76;
        stabZ[189]= 76;
        stabZ[190]= 77;
        stabZ[191]= 77;
        stabZ[192]= 77;
        stabZ[193]= 77;
        stabZ[194]= 78;
        stabZ[195]= 78;
        stabZ[196]= 79;
        stabZ[197]= 79;
        stabZ[198]= 79;
        stabZ[199]= 80;
        stabZ[200]= 80;
        stabZ[201]= 80;
        stabZ[202]= 80;
        stabZ[203]= 81;
        stabZ[204]= 81;
        stabZ[205]= 81;
        stabZ[206]= 82;
        stabZ[207]= 82;
        stabZ[208]= 82;
        stabZ[209]= 83;
        stabZ[210]= 83;
}

#endif
