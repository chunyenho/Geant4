
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// *                                                                  *
// * Parts of this code which have been  developed by QinetiQ Ltd     *
// * under contract to the European Space Agency (ESA) are the        *
// * intellectual property of ESA. Rights to use, copy, modify and    *
// * redistribute this software for general public use are granted    *
// * in compliance with any licensing, distribution and development   *
// * policy adopted by the Geant4 Collaboration. This code has been   *
// * written by QinetiQ Ltd for the European Space Agency, under ESA  *
// * contract 19770/06/NL/JD (Technology Research Programme).         *
// *                                                                  *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file hadronic/Hadr02/src/G4Type1GlauberParameterisation.cc
/// \brief Implementation of the G4Type1GlauberParameterisation class
//
// $Id: G4Type1GlauberParameterisation.cc 81932 2014-06-06 15:39:45Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4Type1GlauberParameterisation.cc
//
// Version:             0.B
// Date:                02/04/08
// Author:              P R Truscott
// Organisation:        QinetiQ Ltd, UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            19770/06/NL/JD
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#ifdef G4_USE_DPMJET

#include "G4Type1GlauberParameterisation.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

using namespace std;

//
//
// The following function performs a polynomial least squares fit to double
// precision data.  It forms part of the CERN mathlib.  This is a really
// beyond standard policy and will eventually be changed to a C++ function.
//
extern "C" {void dlsqpm_ (int*,double*,double*,int*,double*,double*,int*);}

////////////////////////////////////////////////////////////////////////////////
//
G4Type1GlauberParameterisation::G4Type1GlauberParameterisation ()
{
  limit1    = 0.1;
  limit2    = 0.60;
  limit3    = 0.95;
  limit4    = 0.9999;
  
  maxArrayp = 200;
  maxigp    = 24;
  
  for (G4int ig=0; ig<maxigp; ig++) {
    for (G4int j=0; j<10; j++) {
      paramn[ig][j] = 0.0;
      paramn[ig][j] = 0.0;
    }
    mun1[ig] = 0.0;
    mun2[ig] = 0.0;
    cn[ig]   = 0.0;
    mum1[ig] = 0.0;
    mum2[ig] = 0.0;
    cm[ig]   = 0.0;
  }
}
////////////////////////////////////////////////////////////////////////////////
//
G4Type1GlauberParameterisation::~G4Type1GlauberParameterisation ()
{;}
////////////////////////////////////////////////////////////////////////////////
//
G4double G4Type1GlauberParameterisation::GetFitParameters
  (const G4double *bsite, G4double *p) const
{
//
//
// Initialise parameters.
//
  G4int pt1 = -1;
  G4int pt2 = -1;
  G4int pt3 = -1;
  G4int pt4 = -1;
//
//
// Locate transitions between different parts of the curve where different
// fits are used.
//
  G4double ib[200];
  G4double lnib[200];
  G4double lnbsite[200];
  for (G4int i=0; i<maxArrayp; i++) {
    ib[i]   = (G4double) i;
    lnib[i] = std::log(ib[i]);
    if (bsite[i] < 1.0E-10) lnbsite[i] = -23.02585093;
    else                    lnbsite[i] = std::log(bsite[i]);
    if (pt1 == -1) {
      if (bsite[i] >= limit1) pt1 = i;
    }
    else if (pt2 == -1) {
      if (bsite[i] >= limit2) pt2 = i;
    }
    else if (pt3 == -1) {
      if (bsite[i] >= limit3) pt3 = i;
    }
    else if (pt4 == -1) {
      if (bsite[i] >= limit4) pt4 = i;
    }
  }
//
//
// First section determines the power-law fits for the low and intermediate b
// values;
//
  G4int deltan = pt2 - pt1;
  G4int M      = 1;
  G4double a1[2];
  G4double sd;
  G4int ifail;
  dlsqpm_ (&deltan,&lnbsite[pt1],&lnib[pt1],&M,&a1[0],&sd,&ifail);
  
  G4double c1 = a1[0];
  G4double M1 = a1[1];

//  G4double M2 = (lnib[3] - lnib[2]) / (lnbsite[3] - lnbsite[2]);
//  G4double c2 = lnib[2] - M2*lnbsite[2];
  deltan = 3;
  dlsqpm_ (&deltan,&lnbsite[1],&lnib[1],&M,&a1[0],&sd,&ifail);
  
  G4double c2 = a1[0];
  G4double M2 = a1[1];
  
  p[0] = std::exp(c2);
  p[1] = M2;
  p[2] = std::exp(c1);
  p[3] = M1;
  if (std::abs(M2-M1) > 1.0E-10) {
    p[4] = exp(-(c2-c1)/(M2-M1));
  }
  else {
    p[4] = limit2 / 2.0;
  }
//
//
// This next bit solves for gamma to determine the inflection at high-b values.
// The algorthM used is EXTREEEEEMELY crude .... but practical and robust.
// It's a linear search.
//
  G4double delta = 1.0E+99;
  G4double gam   = 0.0;
  for (G4int ig = 120; ig < 1000; ig++) {
    G4double DELTA = 0.0;
    G4double GAMMA = (G4double) ig / 100.0;
    G4double EXPON = p[3] / GAMMA;
    for (G4int i = pt2; i<pt3; i++) {
      G4double f  = bsite[i];
      G4double B  = p[2] * std::pow(f,p[3]) / 
        std::pow((1.0-std::pow(f,GAMMA)),EXPON);
      G4double epsilon = std::abs((ib[i]-B)/ib[i]);
      if (epsilon > DELTA) DELTA = epsilon;
    }
    if (DELTA < delta) {
      gam   = GAMMA;
      delta = DELTA;
    }
  }
  p[5] = gam;
//
//
// For the final part of the curve, we use a cubic polynomial 
// fit to -ln(1-bsite)
// versus ib.  This does seem to work quite well.
//
  G4double phi[200];
  for (G4int i = pt3; i<pt4; i++) {
    phi[i] = -std::log(1.0 - bsite[i]);
  }
  deltan = pt4-pt3;
  M      = 3;
  G4double a2[4];
  
  dlsqpm_ (&deltan,&phi[pt3],&ib[pt3],&M,&a2[0],&sd,&ifail);
  
  p[6] = a2[0];
  p[7] = a2[1];
  p[8] = a2[2];
  p[9] = a2[3];
  
  return 0.0;
}
////////////////////////////////////////////////////////////////////////////////
//
// GetValueN
//
G4double 
G4Type1GlauberParameterisation::GetParameterisedValueN(const G4double f,
                                                       const G4double ppn) const
{
  G4int ig = 0;
  if (ppn < 1.0E-10) {
    return 0;
  }
  else {
    ig = G4int(2.0*std::log10(ppn)) - 2;
  }
  if      (ig < 0)  ig = 0;
  else if (ig > 23) ig = 23;
    
  G4double v = 0.0;
  if (f <= paramn[ig][4]) {
    v = paramn[ig][0] * std::pow(f,paramn[ig][1]);
  }
  else if (f <= limit3) {
    v = paramn[ig][2] * std::pow(f,paramn[ig][3]) /
      std::pow((1.0 - std::pow(f,paramn[ig][5])),paramn[ig][3]/paramn[ig][5]);
  }
  else {
    G4double l = -std::log(1.0-f);
    v = paramn[ig][6] +
        paramn[ig][7]*l +
        paramn[ig][8]*l*l +
        paramn[ig][9]*std::pow(l,3.0);
  }

  return v;
}
////////////////////////////////////////////////////////////////////////////////
//
// GetValueM
//
G4double 
G4Type1GlauberParameterisation::GetParameterisedValueM(const G4double f,
                                                       const G4double ppn) const
{
  G4int ig = 0;
  if (ppn < 1.0E-10) {
    return 0;
  }
  else {
    ig = G4int(2.0*std::log10(ppn)) - 2;
  }
  if      (ig < 0)  ig = 0;
  else if (ig > 23) ig = 23;
  
  G4double v = 0.0;
  if (f <= paramm[ig][4]) {
    v = paramm[ig][0] * std::pow(f,paramm[ig][1]);
  }
  else if (f <= limit3) {
    v = paramm[ig][2] * std::pow(f,paramm[ig][3]) /
      std::pow((1.0 - std::pow(f,paramm[ig][5])),paramm[ig][3]/paramm[ig][5]);
  }
  else {
    G4double l = -std::log(1.0-f);
    v = paramn[ig][6] +
        paramn[ig][7]*l +
        paramn[ig][8]*l*l +
        paramn[ig][9]*std::pow(l,3.0);
  }
  
  return v;
}
////////////////////////////////////////////////////////////////////////////////
//
// GetInverseValueN
//
/*G4double G4Type1GlauberParameterisation::GetInverseValueN (const G4int b,
  const G4double ppn) const
{
  G4int ig = 0;
  if (ppn < 1.0E-10) {
    return 0;
  }
  else {
    ig = G4int(2.0*std::log10(ppn)) - 2;
  }
  if (ig > 23) ig = 23; 
  
  return GetInverseValueN(b,ig);
}*/
////////////////////////////////////////////////////////////////////////////////
//
// GetInverseValueM
//
/*G4double G4Type1GlauberParameterisation::GetInverseValueM (const G4int b,
  const G4double ppn) const
{
  G4int ig = 0;
  if (ppn < 1.0E-10) {
    return 0;
  }
  else {
    ig = G4int(2.0*std::log10(ppn)) - 2;
  }
  if (ig > 23) ig = 23; 
  
  return GetInverseValueN(b,ig);
}*/
////////////////////////////////////////////////////////////////////////////////
//
// GetInverseValueN
//
/*G4double G4Type1GlauberParameterisation::GetInverseValueN (const G4int b,
  const G4int ig) const
{
  G4double v = 0.0;
  G4double x = (G4double) b;
  if (b <= 1) {
    v = 0.0;
  }
  else {
    G4double f1 = 1.0 + std::pow(paramn[ig][0]/x,mun1[ig]);
    G4double f2 = 1.0 + std::pow(cn[ig]/x,mun2[ig]);
    v = std::pow(f1*f2,1.0/paramn[ig][5]);
  }
  
  return v;
}*/
////////////////////////////////////////////////////////////////////////////////
//
// GetInverseValueM
//
/*G4double G4Type1GlauberParameterisation::GetInverseValueM (const G4int b,
  const G4int ig) const
{
  G4double v = 0.0;
  G4double x = (G4double) b;
  if (b <= 1) {
    v = 0.0;
  }
  else {
    G4double f1 = 1.0 + std::pow(paramm[ig][0]/x,mum1[ig]);
    G4double f2 = 1.0 + std::pow(cm[ig]/x,mum2[ig]);
    v = std::pow(f1*f2,1.0/paramm[ig][5]);
  }
  
  return v;
}*/
////////////////////////////////////////////////////////////////////////////////
//
#endif
