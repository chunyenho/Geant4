
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// *                                                                  *
// * Parts of this code which have been  developed by QinetiQ Ltd     *
// * under contract to the European Space Agency (ESA) are the        *
// * intellectual property of ESA. Rights to use, copy, modify and    *
// * redistribute this software for general public use are granted    *
// * in compliance with any licensing, distribution and development   *
// * policy adopted by the Geant4 Collaboration. This code has been   *
// * written by QinetiQ Ltd for the European Space Agency, under ESA  *
// * contract 19770/06/NL/JD (Technology Research Programme).         *
// *                                                                  *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file hadronic/Hadr02/src/G4DPMJET2_5Model.cc
/// \brief Implementation of the G4DPMJET2_5Model class
//
// $Id: G4DPMJET2_5Model.cc 81932 2014-06-06 15:39:45Z gcosmo $
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//
// MODULE:              G4DPMJET2_5Model.cc
//
// Version:             0.B
// Date:                02/04/08
// Author:              P R Truscott
// Organisation:        QinetiQ Ltd, UK
// Customer:            ESA/ESTEC, NOORDWIJK
// Contract:            19770/06/NL/JD
//
// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
///////////////////////////////////////////////////////////////////////////////
//
#ifdef G4_USE_DPMJET


#include "G4DPMJET2_5Model.hh"
#include "G4GlaubAADataSetHandler.hh"

#include "G4ExcitationHandler.hh"
#include "G4Evaporation.hh"
#include "G4FermiBreakUp.hh"
#include "G4PhotonEvaporation.hh"
#include "G4PreCompoundModel.hh"
#include "G4ParticleDefinition.hh"
#include "G4ParticleTable.hh"
#include "G4DynamicParticle.hh"
#include "Randomize.hh"
#include "G4Fragment.hh"
#include "G4VNuclearDensity.hh"
#include "G4NuclearShellModelDensity.hh"
#include "G4NuclearFermiDensity.hh"
#include "G4FermiMomentum.hh"
#include "G4ReactionProductVector.hh"
#include "G4LorentzVector.hh"
#include "G4ParticleMomentum.hh"
#include "G4Poisson.hh"
#include "G4ParticleTable.hh"
#include "G4IonTable.hh"
#include "G4LorentzVector.hh"
#include "G4HadTmpUtil.hh"
#include "globals.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

#include <fstream>

#include "G4DPMJET2_5Interface.hh"

/////////////////////////////////////////////////////////////////////////////////
//
// Constructor without arguments
//
// This constructor uses a default pre-compound.  It initialises the
// variables (including the de-excitation), but note that much of the work is
// done in the member function Initialise(), which is dedicated to
// initialising variables in DPMJET-II.5 to class-default values.
//

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4DPMJET2_5Model::G4DPMJET2_5Model () : G4HadronicInteraction("DPMJET2_5")
{
//
// Set the default verbose level to 0 - no output.
//
  SetVerboseLevel(1);
//
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"G4DPMJET2_5Model default constructor" <<G4endl;
  }
#endif
//
//
// Send message to stdout to advise that the G4DPMJET2_5Model model 
// is being used.
//
  theInitType = DEFAULT;
  PrintWelcomeMessage();
//
// Use the precompound model for nuclear de-excitation.
//
  theExcitationHandler = 0;
  SetDefaultPreCompoundModel();
//
//
// Set the minimum and maximum range for the model (despite nomanclature, this
// is in energy per nucleon number).
//
  SetMinEnergy(5.0*GeV);
  SetMaxEnergy(1000.0*TeV);

  SetEnergyMomentumCheckLevels(100*perCent, 1*TeV);

//
//
// Initialise the DPMJET model - this effectively does what the DPMJET
// subroutine DMINIT does without reading an input file.
//
  debug       = false;
  debug_level = 0;
  lunber      = 14;
  dpmver      = 2.5;
  
  LFALSE = 0;
  LTRUE  = 1;

  Initialise ();
//
//
// Next bit directs how and how many Glauber data sets are loaded
// or created.
//
  theGlauberDataSetHandler = G4GlaubAADataSetHandler::getInstance();
  theGlauberDataSetHandler->SetMaxGlauberDataSets(-1);
  
  theParticleTable = G4ParticleTable::GetParticleTable();
  theIonTable = const_cast <G4IonTable *> (theParticleTable->GetIonTable());
//
//
}

const std::pair<G4double, G4double> 
G4DPMJET2_5Model::GetFatalEnergyCheckLevels() const
{
  // default level of Check
  return std::pair<G4double, G4double>(100.*perCent, 500 * GeV);
}

/////////////////////////////////////////////////////////////////////////////////
//
// Constructor with DPMJET-II.5 initialisation type
//
// This constructor uses a default pre-compound.  It initialises the
// variables (including the de-excitation), but note that much of the work is
// done in the member function Initialise(), which is dedicated to
// initialising variables in DPMJET-II.5.  The user is able to define whether to
// use the default values, DPMJET-II.5 settings or DPMJET-III settings.
//
G4DPMJET2_5Model::G4DPMJET2_5Model (const G4DPMJET2_5InitialisationType initType)
{
//
// Set the default verbose level to 0 - no output.
//
  SetVerboseLevel(1);
//
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"G4DPMJET2_5Model constructor #1 " <<G4endl;
  }
#endif
//
//
// Send message to stdout to advise that the G4DPMJET2_5Model model 
// is being used.
//
  theInitType = initType;
  PrintWelcomeMessage();
//
// Use the precompound model for nuclear de-excitation.
//
  theExcitationHandler = 0;
  SetDefaultPreCompoundModel();
//
//
// Set the minimum and maximum range for the model (despite nomanclature, this
// is in energy per nucleon number).
//
  SetMinEnergy(5.0*GeV);
  SetMaxEnergy(1000.0*TeV);
//
//
// Initialise the DPMJET model - this effectively does what the DPMJET
// subroutine DMINIT does without reading an input file.
//
  debug       = false;
  debug_level = 0;
  lunber      = 14;
  dpmver      = 2.5;
  
  LFALSE = 0;
  LTRUE  = 1;

  Initialise ();
//
//
// Next bit directs how and how many Glauber data sets are loaded
// or created.
//
  theGlauberDataSetHandler = G4GlaubAADataSetHandler::getInstance();
  theGlauberDataSetHandler->SetMaxGlauberDataSets (-1);
  
  theParticleTable = G4ParticleTable::GetParticleTable();
  theIonTable      = theParticleTable->GetIonTable();
//
//
}
////////////////////////////////////////////////////////////////////////////////
//
// Constructor with de-excitation handler.
//
// This constructor uses the user-provided de-excitation handler.  However, it
// is possible for the use to provide a NULL pointer, in which case, it is
// assumed that the user doesn't want to simulate de-excitation - USER, BEWARE!
//
// The member function initialises the variables (including the de-excitation),
// but note that much of the work is done in the member function Initialise(),
// which is dedicated to initialising variables in DPMJET-II.5.
//
G4DPMJET2_5Model::G4DPMJET2_5Model (G4ExcitationHandler *aExcitationHandler,
  const G4DPMJET2_5InitialisationType initType) 
{
//
// Set the default verbose level to 0 - no output.
//
  SetVerboseLevel(1);
//
// Send message to stdout to advise that the G4DPMJET2_5Model model 
// is being used.
//
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"G4DPMJET2_5Model constructor #2 " <<G4endl;
  }
#endif
  theInitType = initType;
  PrintWelcomeMessage();
//
// The user is able to provide the excitation handler.
//
  theExcitationHandler = aExcitationHandler;
  thePreComp           = 0;
//
//
// Set the minimum and maximum range for the model (despite nomanclature, this
// is in energy per nucleon number).  
//
  SetMinEnergy(5.0*GeV);
  SetMaxEnergy(1000.0*TeV);
//
//
// Initialise the DPMJET model - this effectively does what the DPMJET
// subroutine DMINIT does without reading an input file.
//
  debug       = false;
  debug_level = 0;
  lunber      = 14;
  dpmver      = 2.5;
  
  LFALSE = 0;
  LTRUE  = 1;
  
  Initialise ();
//
//
// Next bit directs how and how many Glauber data sets are loaded
// or created.
//
  theGlauberDataSetHandler = G4GlaubAADataSetHandler::getInstance();
  theGlauberDataSetHandler->SetMaxGlauberDataSets (-1);

  theParticleTable = G4ParticleTable::GetParticleTable();
  theIonTable      = theParticleTable->GetIonTable();
//
//
}
////////////////////////////////////////////////////////////////////////////////
//
// Constructor with pre-compound model.
//
// This constructor uses the user-provided pre-equilibrium model.  However, it
// is possible for the use to provide a NULL pointer, in which case, it is
// assumed that the user doesn't want to simulate pre-equilibrium. - USER, 
// BEWARE!
//
// The member function initialises the variables (including the de-excitation),
// but note that much of the work is done in the member function Initialise(),
// which is dedicated to initialising variables in DPMJET-II.5.
//
G4DPMJET2_5Model::G4DPMJET2_5Model (G4VPreCompoundModel *aPreComp,
  const G4DPMJET2_5InitialisationType initType) 
{
//
// Set the default verbose level to 0 - no output.
//
  SetVerboseLevel(1);
//
// Send message to stdout to advise that the G4DPMJET2_5Model model 
// is being used.
//
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"G4DPMJET2_5Model constructor #3 " <<G4endl;
  }
#endif
  theInitType = initType;
  PrintWelcomeMessage();
//
// The user is able to provide the pre-compound model.
//
  theExcitationHandler = 0;
  thePreComp           = aPreComp;
//
//
// Set the minimum and maximum range for the model (despite nomanclature, this
// is in energy per nucleon number).  
//
  SetMinEnergy(5.0*GeV);
  SetMaxEnergy(1000.0*TeV);
//
//
// Initialise the DPMJET model - this effectively does what the DPMJET
// subroutine DMINIT does without reading an input file.
//
  debug       = false;
  debug_level = 0;
  lunber      = 14;
  dpmver      = 2.5;
  
  LFALSE = 0;
  LTRUE  = 1;
  
  Initialise ();
//
//
// Next bit directs how and how many Glauber data sets are loaded
// or created.
//
  theGlauberDataSetHandler = G4GlaubAADataSetHandler::getInstance();
  theGlauberDataSetHandler->SetMaxGlauberDataSets (-1);

  theParticleTable = G4ParticleTable::GetParticleTable();
  theIonTable      = theParticleTable->GetIonTable();
//
//
}
////////////////////////////////////////////////////////////////////////////////
//
// Destructor
//
G4DPMJET2_5Model::~G4DPMJET2_5Model ()
{
//
//
// The destructor doesn't have to do a great deal!
//
  if (theExcitationHandler) delete theExcitationHandler;
  if (thePreComp)           delete thePreComp;
//  delete theGlauberDataSetHandler;
  theGlauberDataSetHandler->UnloadAllGlauberData();
}
////////////////////////////////////////////////////////////////////////////////
//
// IsApplicable
//
// This member function simply determines whether there is relevant information
// in Glauber data for this projectile and target, and if the nucleus is
// sensible.
//
G4bool G4DPMJET2_5Model::IsApplicable (
  const G4HadProjectile &theTrack, G4Nucleus &theTarget)
{
//
//
// Get relevant information about the projectile and target (A, Z)
//
  const G4ParticleDefinition *definitionP = theTrack.GetDefinition();
  G4int AP   = definitionP->GetBaryonNumber();
  G4int ZP   = G4int(definitionP->GetPDGCharge()/eplus + 0.5);
  G4int AT   = theTarget.GetA_asInt();
  G4int ZT   = theTarget.GetZ_asInt();
  
  if (AP >= 2 && ZP >= 1 && AT >= 2 && ZT >=1) {
    return theGlauberDataSetHandler->IsGlauberDataSetAvailable(AP,AT);
  }
  else {
    return false;
  }
}
////////////////////////////////////////////////////////////////////////////////
//
// ApplyYourself
//
// Member function to process an event, and get information about the products.
// The phases are:
//
// (1) Determine the information about the projectile and target.
//
// (2) Identify to the Glauber data set handler which data need to be used.  If 
// the GDSH finds there are no Glauber profile data for the collision,the 
// product is the unchanged projectile.
//
// (3) Initialise further common-block variables in DPMJET-II.5, and perform
// FORTRAN calls (note this is taken largely from an interpretation of CORSIKA
// and DPMJET-II.5 FORTRAN ... I think there's duplication in some of the
// initialisation on top of what Initialise() does, but CORSIKA has similar
// duplication.  I'm not confident at removing this out at the moment.
//
// (4) Call the DPMJET-II.5 FORTRAN subroutine DPMEVT.
//
// (5) Pick out the final state particles and nuclei.  In the case of nuclei
// use the de-excitation handler if one has been defined.  Transfer all these
// particles to the final-state vector.
//
// (6) if very-verbose output is demanded by the user, there is a print-out
// of the total energy and total momentum before and after collision.
//
G4HadFinalState *G4DPMJET2_5Model::ApplyYourself (
  const G4HadProjectile &theTrack, G4Nucleus &theTarget)
{
//
//
// The secondaries will be returned in G4HadFinalState &theParticleChange -
// initialise this.  The original track will always be discontinued and
// secondaries followed.
//
  theParticleChange.Clear();
  theParticleChange.SetStatusChange(stopAndKill);
//
//
// Get relevant information about the projectile and target (A, Z, energy/nuc,
// momentum, etc).
//
  const G4ParticleDefinition *definitionP = theTrack.GetDefinition();
  G4int AP   = definitionP->GetBaryonNumber();
  G4int ZP   = G4int(definitionP->GetPDGCharge()/eplus+0.5);
  G4double M          = definitionP->GetPDGMass();
  G4ThreeVector pP    = theTrack.Get4Momentum().vect();
  G4double T          = theTrack.GetKineticEnergy()/G4double(AP);   
  // Units are MeV/nuc 
  G4double E          = theTrack.GetTotalEnergy()/G4double(AP);            
  // Units are MeV/nuc
  G4int AT         = theTarget.GetA_asInt();
  G4int ZT         = theTarget.GetZ_asInt();
  G4double mpnt  = theTarget.AtomicMass(AT, ZT);
  G4double TotalEPre  = theTrack.GetTotalEnergy() + mpnt;
    //    theTarget.AtomicMass(AT, ZT) + theTarget.GetEnergyDeposit();
//  G4LorentzRotation transformToLab =
//          (const_cast <G4HadProjectile*> (&theTrack))->GetTrafoToLab();
//
//
// Output relevant information on initial conditions if verbose.  Note that
// most of the verbsse output is dealt with through private member function
// calls.
//
  if (verboseLevel >= 2)
  {
    G4cout <<"########################################"
           <<"########################################"
           <<G4endl;
    G4cout.precision(6);
    G4cout <<"IN G4DPMJET2_5Model::ApplyYourself" <<G4endl;
    G4cout <<"START OF EVENT" <<G4endl;
    G4cout <<"Initial projectile (A,Z) = (" <<AP <<", " <<ZP <<")" <<G4endl;
    G4cout <<"Initial target     (A,Z) = (" <<AT <<", " <<ZT <<")" <<G4endl;
    G4cout <<"Projectile momentum      = " <<pP/MeV <<" MeV/c" <<G4endl;
    G4cout <<"Total energy             = " <<TotalEPre/MeV <<" MeV" <<G4endl;
    G4cout <<"Kinetic energy/nuc       = " <<T/MeV <<" MeV" <<G4endl;
  }
//
//
// Setup variables and call the DPMJET model.  There is a significant amount of
// initialisation which still needs to be done, based on DPMJET-II.5 main
// program and card reader subroutine, and the CORSIKA implementation.
//
  G4int AP1 = AP;
  G4int ZP1 = ZP;
  G4int AT1 = AT;
  G4int ZT1 = ZT;
//
//
// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
// ***** WARNING *****
// The following is a provisional "catch" for ions with A==Z.  The de-excitation
// and precompound model can produce such nuclei, although they should decay
// into free protons.  For the moment, DPMJET-II.5 doesn't treat them ... i.e.
// the FORTRAN code would crash.  Therefore, return such ions without 
// nuclear interactions.
//
  if (AP1 > 1 && AP1 == ZP1) {
    theParticleChange.SetStatusChange(isAlive);
    theParticleChange.SetEnergyChange(theTrack.GetKineticEnergy());
    theParticleChange.SetMomentumChange(theTrack.Get4Momentum().vect().unit());
    if (verboseLevel >= 2) {
      G4cout <<"PROJECTILE WITH AP = " <<AP1 <<"  ==  ZP = " <<ZP1 
             <<" REJECTED" <<G4endl;
      G4cout <<"########################################"
             <<"########################################"
             <<G4endl;
    }
    return &theParticleChange;
  }
// ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  
  nucc_.ibproj   = 1;                           // IBPROJ = 1
  nucc_.ijproj   = 1;                           // IJPROJ = 1
  collis_.ijprox = 1;                           // IJPROX = 1
  nucc_.ip       = AP1;                         // IP     = IP_P
  nucc_.ipz      = ZP1;                         // IPZ    = IPZ_P
  nucc_.it       = AT1;                         // IT     = IT_P
  nucc_.itz      = ZT1;                         // ITZ    = ITZ_P
  collis_.ijtar  = 1;                           // IJTAR=1
//
//
// Note that epn is deliberately given units of GeV/nuc, because of the units
// used in DPMJET-II.5.
//
  G4double epn   = E / GeV;
  G4double mpn   = M / (GeV*AP); // Projectile mass per nucleon in GeV/(c2*nuc)
//  G4double gamma = epn / mpn;
//  G4double elab  = epn * M / GeV;             // ELAB   = EPN * AMPRO_P

  diffra_.isingd = ISINGD;
  user2_.isingx  = ISINGX;
  user2_.idubld  = IDUBLD;
  user2_.sdfrac  = SDFRAC;

//  G4double amu_c2GeV = amu_c2 / GeV;
  G4double ppn   = std::sqrt((epn-mpn)*(epn+mpn));
// Units of GeV/(c*nuc)
// PPN    = SQRT( (EPN-AMPROJ)*(EPN+AMPROJ) )
//
//
// Before setting the remainder of the variables for DPMJET-II.5, check for
// appropriate Glauber data.  If the value returned is false, then set the 
// change object to the source particle.
//
  if (!(theGlauberDataSetHandler->SetCurrentGlauberDataSet(AP1,AT1,ppn))) {
    theParticleChange.SetStatusChange(isAlive);
    theParticleChange.SetEnergyChange(theTrack.GetKineticEnergy());
    theParticleChange.SetMomentumChange(theTrack.Get4Momentum().vect().unit());
    return &theParticleChange;
  }
  dtumat_.ntaxx[0]  = AT1;
  dtumat_.nztaxx[0] = ZT1;
  dtumat_.nprxx[0]  = AP1;
  dtumat_.nzprxx[0] = ZP1;
//
//
// Set the remainder of the variables for DPMJET-II.5 FORTRAN.
//
  nncms_.pproj   = ppn;                         // PPROJ  = PPN
  nncms_.eproj   = epn;                         // EPROJ  = EPN
  mpnt          /= (AT * GeV); // Mass per nuclon of target in GeV/(c2*nuc)
  nncms_.umo     = std::sqrt(mpn*mpn + mpnt*mpnt + 2.0*mpnt*epn);
// UMO    = SQRT( AMPROJ**2 + AMTAR**2 +2.D0*AMTAR*EPROJ )
// Note I believe this equation is only correct
// if the subsequent equations (for pTthr) 
// needs the Ecm for the NUCLEON-NUCLEON system 
  user2_.cmener  = nncms_.umo;                  // CMENER = UMO
  collis_.s      = nncms_.umo * nncms_.umo;
                                                // SS     = UMO**2
  collis_.ptthr  = 3.0;
  if (strufu_.istrut == 1)
  {
    collis_.ptthr = 2.1 + 0.15*std::pow(std::log10(user2_.cmener/50.),3.0);
    // PTTHR  = 2.1D0+0.15D0*(LOG10(CMENER/50.))**3
  }
  else if (strufu_.istrut == 2)
  {
    collis_.ptthr = 2.5 + 0.12*std::pow(std::log10(user2_.cmener/50.),3.0);
    // PTTHR  = 2.5D0+0.12D0*(LOG10(CMENER/50.))**3
  }
  collis_.ptthr2 = collis_.ptthr;               // PTTHR2 = PTTHR
  nncms_.gamcm   = (epn + mpnt) / nncms_.umo;
  // GAMCM  = (EPROJ+AMTAR)/UMO
  // Note I believe this equation is only correct
  // if the subsequent equations (for pTthr)
  // need the Ecm for the NUCLEON-NUCLEON system 
  nncms_.bgcm    = ppn / nncms_.umo;                // PPROJ/UMO
  nncms_.pcm     = nncms_.gamcm*ppn - nncms_.bgcm*epn;
  // PCM    = GAMCM*PPROJ - BGCM*EPROJ
  sigma_.sigsof  = 37.8 * std::pow(collis_.s,0.076);
  // ALFA   = 1.076D0
  // A      = 37.8D0
  // SIGSOF = A * SS**(ALFA-1.D0)
  seasu3_.seasq  = SEASQ;                       // SEASQ  = 0.50D0
  xseadi_.ssmima = SSMIMA;                      // SSMIMA = 1.201D0
  xseadi_.ssmimq = xseadi_.ssmima * xseadi_.ssmima;
                                                // SSMIMQ = SSMIMA**2
  taufo_.taufor  = TAUFOR;
  taufo_.ktauge  = KTAUGE;
  
  if ( theInitType == DEFAULT ) {
    final_.ifinal  = 0;                         // IFINAL = 1
    evappp_.ievap  = 0;                         // IEVAP  = 0
    parevt_.levprt = LTRUE;                     // LEVPRT = .FALSE.
    parevt_.ilvmod = 1;                         // ILVMOD = 1
    parevt_.ldeexg = LFALSE;                    // LDEEXG = .FALSE.
    parevt_.lheavy = LFALSE;                    // LHEAVY = .FALSE.
    frbkcm_.lfrmbk = LFALSE;                    // LFRMBK = .FALSE.
    inpflg_.ifiss  = 0;                         // IFISS  = 0
  } else if ( theInitType == CORSIKA ) {
    final_.ifinal  = 0;                         // IFINAL = 1
    evappp_.ievap  = 0;                         // IEVAP  = 0
    parevt_.levprt = LTRUE;                     // LEVPRT = .FALSE.
    parevt_.ilvmod = 1;                         // ILVMOD = 1
    parevt_.ldeexg = LFALSE;                    // LDEEXG = .FALSE.
    parevt_.lheavy = LTRUE;                     // LHEAVY = .FALSE.
    frbkcm_.lfrmbk = LFALSE;                    // LFRMBK = .FALSE.
    inpflg_.ifiss  = 0;                         // IFISS  = 0
  }
  else if ( theInitType == DPM2_5 ) {
    final_.ifinal  = 0;                         // IFINAL = 0
    evappp_.ievap  = 0;                         // IEVAP  = 0
    parevt_.levprt = LTRUE;                     
// LEVPRT = .TRUE. NOTE: THIS IS AT ODDS WITH WHAT'S IN DPMJET-II.5, 
// BUT IF NOT SET, ALL EVENTS GET REJECTED.
    parevt_.ilvmod = 1;                         // ILVMOD = 1
    parevt_.ldeexg = LFALSE;                    // LDEEXG = .FALSE.
    parevt_.lheavy = LFALSE;                    // LHEAVY = .FALSE.
    frbkcm_.lfrmbk = LFALSE;                    // LFRMBK = .FALSE.
    inpflg_.ifiss  = 0;                         // IFISS  = 0
  }
  else if ( theInitType == DPM3 ) {
    final_.ifinal  = 0;                         // IFINAL = 0
    evappp_.ievap  = 0;                         // IEVAP  = 0
    parevt_.levprt = LTRUE;                     
// LEVPRT = .TRUE. NOTE: THIS IS AT ODDS WITH WHAT'S IN DPMJET-II.5, 
// BUT IF NOT SET, ALL EVENTS GET REJECTED.
    parevt_.ilvmod = 1;                         // ILVMOD = 1
    parevt_.ldeexg = LFALSE;                    // LDEEXG = .FALSE.
    parevt_.lheavy = LFALSE;                    // LHEAVY = .FALSE.
    frbkcm_.lfrmbk = LFALSE;                    // LFRMBK = .FALSE.
    inpflg_.ifiss  = 0;                         // IFISS  = 0
  }
  xsecpt_.ptcut  = collis_.ptthr;               // PTCUT = PTTHR
  
  G4double dsig1[maxpro+1];
  csj1mi_ (&xsecpt_.ptcut, &dsig1[0]);          // CALL CSJ1MI(PTCUT,DSIG1)
  xsecpt_.dsigh   = dsig1[0];                   // SIG1  = DSIG1(0)
                                                // DSIGH = SIG1
  G4int i         = 0;
  G4double pt     = 0.0;
  samppt_ (&i,&pt);                             // SAMPPT(0,PT)
  collap_.s3      = collis_.s;                  // S3      = SS
  collap_.ijproj1 = collis_.ijprox;             // IJPROJ1 = IJPROX
  collap_.ijtar1  = collis_.ijtar;              // IJTAR1  = IJTAR
  collap_.ptthr1  = collis_.ptthr;              // PTTHR1  = PTTHR
  collap_.iophrd1 = collis_.iophrd;             // IOPHRD1 = IOPHRD
  collap_.ijprlu1 = collis_.ijprlu;             // IJPRLU1 = IJPRLU
  collap_.ijtalu1 = collis_.ijtalu;             // IJTALU1 = IJTALU
  collap_.ptthr3  = collis_.ptthr2;             // PTTHR3  = PTTHR2

  G4int iiipro          = nucc_.ijproj;         
  // IIPROJ = IJPROJ & IIIPRO = IIPROJ
  G4int iiitar          = nucc_.ijtarg;         // IITARG = IJTARG
  G4int kkmat           = 1;
  G4int nhkkh1          = 1;                    // NHKKH1 = 1
  
  for (i=0; i<8; i++) user1_.projty[i] = paname_.btype[iiipro][i];
                                                // PROJTY=BTYPE(IPROJ)
  G4int irej   = 1;
  G4int evtcnt = 0;
  
  do {
/*    bufueh_.annvv   = 0.001;                   // ANNVV = 0.001
    bufueh_.annss   = 0.001;                    // ANNSS = 0.001
    bufueh_.annsv   = 0.001;                    // ANNSV = 0.001
    bufueh_.annvs   = 0.001;                    // ANNVS = 0.001
    bufueh_.anncc   = 0.001;                    // ANNCC = 0.001
    bufueh_.anndv   = 0.001;                    // ANNDV = 0.001
    bufueh_.annvd   = 0.001;                    // ANNVD = 0.001
    bufueh_.annds   = 0.001;                    // ANNDS = 0.001
    bufueh_.annsd   = 0.001;                    // ANNSD = 0.001
    bufueh_.annhh   = 0.001;                    // ANNHH = 0.001
    bufueh_.annzz   = 0.001;                    // ANNZZ = 0.001
    bufueh_.anndi   = 0.001;                    // ANNDI = 0.001
    bufueh_.annzd   = 0.001;                    // ANNZD = 0.001
    bufueh_.anndz   = 0.001;                    // ANNDZ = 0.001
    bufueh_.ptvv    = 0.0;                      // PTVV = 0.
    bufueh_.ptss    = 0.0;                      // PTSS = 0.
    bufueh_.ptsv    = 0.0;                      // PTSV = 0.
    bufueh_.ptvs    = 0.0;                      // PTVS = 0.
    bufueh_.ptcc    = 0.0;                      // PTCC = 0.
    bufueh_.ptdv    = 0.0;                      // PTDV = 0.
    bufueh_.ptvd    = 0.0;                      // PTVD = 0.
    bufueh_.ptds    = 0.0;                      // PTDS = 0.
    bufueh_.ptsd    = 0.0;                      // PTSD = 0.
    bufueh_.pthh    = 0.0;                      // PTHH = 0.
    bufueh_.ptzz    = 0.0;                      // PTZZ = 0.
    bufueh_.ptdi    = 0.0;                      // PTDI = 0.
    bufueh_.ptzd    = 0.0;                      // PTZD = 0.
    bufueh_.ptdz    = 0.0;                      // PTDZ = 0.
    bufueh_.eevv    = 0.0;                      // EEVV = 0.
    bufueh_.eess    = 0.0;                      // EESS = 0.
    bufueh_.eesv    = 0.0;                      // EESV = 0.
    bufueh_.eevs    = 0.0;                      // EEVS = 0.
    bufueh_.eecc    = 0.0;                      // EECC = 0.
    bufueh_.eedv    = 0.0;                      // EEDV = 0.
    bufueh_.eevd    = 0.0;                      // EEVD = 0.
    bufueh_.eeds    = 0.0;                      // EEDS = 0.
    bufueh_.eesd    = 0.0;                      // EESD = 0.
    bufueh_.eehh    = 0.0;                      // EEHH = 0.
    bufueh_.eezz    = 0.0;                      // EEZZ = 0.
    bufueh_.eedi    = 0.0;                      // EEDI = 0.
    bufueh_.eezd    = 0.0;                      // EEZD = 0.
    bufueh_.eedz    = 0.0;                      // EEDZ = 0.
    ncouch_.acouvv  = 0.0;                      // ACOUVV = 0.
    ncouch_.acouss  = 0.0;                      // ACOUSS = 0.
    ncouch_.acousv  = 0.0;                      // ACOUSV = 0.
    ncouch_.acouvs  = 0.0;                      // ACOUVS = 0.
    ncouch_.acouzz  = 0.0;                      // ACOUZZ = 0.
    ncouch_.acouhh  = 0.0;                      // ACOUHH = 0.
    ncouch_.acouds  = 0.0;                      // ACOUDS = 0.
    ncouch_.acousd  = 0.0;                      // ACOUSD = 0.
    ncouch_.acoudz  = 0.0;                      // ACOUDZ = 0.
    ncouch_.acouzd  = 0.0;                      // ACOUZD = 0.
    ncouch_.acoudi  = 0.0;                      // ACOUDI = 0.
    ncouch_.acoudv  = 0.0;                      // ACOUDV = 0.
    ncouch_.acouvd  = 0.0;                      // ACOUVD = 0.
    ncouch_.acoucc  = 0.0;                      // ACOUCC = 0.*/
    if (evtcnt > 0)
      G4cout <<"REJECTED KKINC EVENT.  RETRY # = " <<evtcnt <<G4endl;
//
//
// Generate an event using DPMJET II-5.  NOTE this is a call to dpmevt, and
// could possibly be replaced by call to kkinc_.  After the call,
// ResetCurrentGlauberDataSet is used to delete any temporary Glauber data 
// set if one had to be set up.
//
    //G4cout << "Call to kkinc_" << G4endl;
    kkinc_ (&epn, &AT1, &ZT1, &AP1, &ZP1, &iiipro, &kkmat, &iiitar, &nhkkh1,
      &irej);
                        //      CALL KKINC(EPN,IIT,IITZ,IIP,IIPZ,IIPROJ,KKMAT,
                        //     * IITARG,NHKKH1,IREJ)
//    dpmevt_ (&elabt, &iiipro, &AP1, &ZP1, &AT1, &ZT1, &kkmat, &nhkkh1);
  } while (irej == 1 && ++evtcnt <100);
  //G4cout << "Call to reset G-data" << G4endl;

  theGlauberDataSetHandler->ResetCurrentGlauberDataSet();
//
//
// If the event has been rejected more than 100 times, then set the track
// as still active and return to the calling routine.
//
  if (irej == 1) {
    theParticleChange.SetStatusChange(isAlive);
    theParticleChange.SetEnergyChange(theTrack.GetKineticEnergy());
    theParticleChange.SetMomentumChange(theTrack.Get4Momentum().vect().unit());
    if (verboseLevel >= 2) {
      G4cout <<"Event rejected and original track maintained" <<G4endl;
      G4cout <<"########################################"
             <<"########################################"
             <<G4endl;
    }
    return &theParticleChange;
  }
//
//
// Determine number of final state particles (including nuclear fragments,
// and load into the particle change if you can identify the particles.
//  
  G4int n              = hkkevt_.nhkk;
  G4int M1             = 0;
  G4Fragment *fragment = 0;
  if (verboseLevel >= 2) DumpVerboseInformation1 (n);
//
//
// Now go through each of the secondaries and add to theParticleChange.
//
  for (G4int ii=0; ii<n; ii++)
  {
    if (hkkevt_.isthkk[ii]==1 || hkkevt_.isthkk[ii]==-1)
    {
//
// Particle is a final state secondary and not a nucleus.
// Determine what this secondary particle is, and if valid, load dynamic
// parameters.
//
      G4ParticleDefinition* theParticle =
        theParticleTable->FindParticle(hkkevt_.idhkk[i]);
      if (theParticle)
      {
        G4double px        = hkkevt_.phkk[i][0] * GeV;
        G4double py        = hkkevt_.phkk[i][1] * GeV;
        G4double pz        = hkkevt_.phkk[i][2] * GeV;
        G4double et        = hkkevt_.phkk[i][3] * GeV;
//        G4LorentzVector lv = transformToLab * G4LorentzVector(px,py,pz,et);
        G4LorentzVector lv = G4LorentzVector(px,py,pz,et);
        
        G4DynamicParticle *theDynamicParticle = 
          new G4DynamicParticle(theParticle,lv);
        theParticleChange.AddSecondary (theDynamicParticle);
        
        if (verboseLevel >= 2)
          DumpVerboseInformation2 (theParticle->GetParticleName(),
          lv.vect(), et, theDynamicParticle->GetKineticEnergy(),pP);
      }
    }
    else if (hkkevt_.idhkk[i]==80000 && hkkevt_.isthkk[i]==1001)
    {
//
//
// Particle is a secondary nucleus. Determine the details of the nuclear
// fragment prior to de-excitation. (Note that the 1 eV in the total energy
// is a safety factor to avoid any possibility of negative rest mass energy.)
// Note also that we don't full trust the energy provided by the DPMJET-II.5,
// and there it's based on the Geant4-determined ion rest-mass.
//
      G4int nucA = extevt_.idres[i];
      G4int nucZ = extevt_.idxres[i];
      if (nucA>0 && nucZ>0) {
        M1++;
        fragment           = 0;
        G4double px        = hkkevt_.phkk[i][0] * GeV;
        G4double py        = hkkevt_.phkk[i][1] * GeV;
        G4double pz        = hkkevt_.phkk[i][2] * GeV;
        G4double ionMass   = theIonTable->GetIonMass(nucZ,nucA);
        //GetIonMass(nucZ,nucA) + nucex[i];  // check how to get this energy
        G4double dpmMass   = hkkevt_.phkk[i][4] * GeV;
        if (dpmMass > ionMass) ionMass = dpmMass;
        G4double et        = std::sqrt(px*px + py*py + pz*pz + ionMass*ionMass);
        G4LorentzVector lv = G4LorentzVector(px,py,pz,et+1.0*eV);
        fragment           = new G4Fragment(nucA, nucZ, lv);
        if (verboseLevel >= 2)
          DumpVerboseInformation3 (M1, nucA, nucZ, lv.vect(), 
                                   et, et-ionMass, pP);
//
//
// Now we can decay the nuclear fragment if present.  The secondaries are
// collected and boosted as well.  The priority is to use a pre-compound
// de-excitation, otherwise the standard excitation-handler is used.
//
        if (fragment && (thePreComp || theExcitationHandler))
        {
          G4ReactionProductVector *products = 0;
          if (thePreComp && fragment->GetA() > 1.5)
            products = thePreComp->DeExcite(*fragment);
          else
            products = theExcitationHandler->BreakItUp(*fragment);
          delete fragment;
          fragment = 0;
          G4ReactionProductVector::iterator iter;
          for (iter = products->begin(); iter != products->end(); ++iter)
          {
            G4DynamicParticle *secondary =
              new G4DynamicParticle((*iter)->GetDefinition(),
              (*iter)->GetTotalEnergy(), (*iter)->GetMomentum());
            theParticleChange.AddSecondary (secondary);
            G4String particleName = 
              (*iter)->GetDefinition()->GetParticleName();
            delete (*iter);
            if (verboseLevel >= 2) {
              if (particleName.find("[",0) < particleName.size())
                DumpVerboseInformation4 (m, particleName, 
                                         secondary->GetMomentum(),
                                         secondary->GetTotalEnergy(),
                                         secondary->GetKineticEnergy(), pP);
              else DumpVerboseInformation2 (particleName, 
                                            secondary->GetMomentum(),
                                            secondary->GetTotalEnergy(),
                                            secondary->GetKineticEnergy(), 
                                            pP);
            }
          }
          delete products;
        }
        if (fragment != 0)
        {
//
//
// Add the excited fragment to the product vector.  Note that this is temporary
// since we should at the atomic excitation to strip all electrons ... i.e. it's
// actually a bare nucleus not an atom in the ground state.
//
          G4ParticleDefinition *theParticleDefinition = theIonTable->
            GetIon(nucZ,nucA);
          G4DynamicParticle *theDynamicParticle = 
            new G4DynamicParticle(theParticleDefinition,lv);
          theParticleChange.AddSecondary (theDynamicParticle);
          delete fragment;
          fragment = 0;
        }
      }
    }
  }
  
  if (verboseLevel >= 3) {
//
//
// Calculate and display the energy and momenta before and after the collision.
// Everything is calculated for the lab frame.
//
    G4double TotalEPost = 0.0;
    G4ThreeVector TotalPPost;
    G4double charge     = 0.0;
    G4int baryon        = 0;
    G4int lepton        = 0;
//    G4int parity        = 0;
    G4int nSecondaries  = theParticleChange.GetNumberOfSecondaries();
    for (G4int j=0; j<nSecondaries; j++) {
      TotalEPost += theParticleChange.GetSecondary(j)->
        GetParticle()->GetTotalEnergy();
      TotalPPost += theParticleChange.GetSecondary(j)->
        GetParticle()->GetMomentum();
      G4ParticleDefinition *theParticle = theParticleChange.GetSecondary(j)->
        GetParticle()->GetDefinition();
      charge += theParticle->GetPDGCharge();
      baryon += theParticle->GetBaryonNumber();
      lepton += theParticle->GetLeptonNumber();
//      parity += theParticle->GetPDGiParity();
    }
    G4cout <<"----------------------------------------"
           <<"----------------------------------------"
           <<G4endl;
    G4cout <<"Total energy before collision   = " <<TotalEPre/MeV
           <<" MeV" <<G4endl;
    G4cout <<"Total energy after collision    = " <<TotalEPost/MeV
           <<" MeV" <<G4endl;
    G4cout <<"Total momentum before collision = " <<pP/MeV
           <<" MeV/c" <<G4endl;
    G4cout <<"Total momentum after collision  = " <<TotalPPost/MeV
           <<" MeV/c" <<G4endl;
    if (verboseLevel >= 4) {
      G4cout <<"Total charge before collision   = " <<(ZP+ZT)*eplus
             <<G4endl;
      G4cout <<"Total charge after collision    = " <<charge
             <<G4endl;
      G4cout <<"Total baryon number before collision = "<<AP+AT
             <<G4endl;
      G4cout <<"Total baryon number after collision  = "<<baryon
             <<G4endl;
      G4cout <<"Total lepton number before collision = 0"
             <<G4endl;
      G4cout <<"Total lepton number after collision  = "<<lepton
             <<G4endl;
    }
    G4cout <<"----------------------------------------"
           <<"----------------------------------------"
           <<G4endl;
  }
  
  if (verboseLevel >= 2)
     G4cout <<"########################################"
            <<"########################################"
            <<G4endl;
  
  return &theParticleChange;
}
////////////////////////////////////////////////////////////////////////////////
//
// SetNoDeexcitation
//
// Deletes an exiting de-excitation handlers and zeros the pointer.  This
// allows the simulation to run without any de-excitation, or just pre-compound
// model only.
//
// Note that you need to separately SetNoPreCompoundModel, if you REALLY don't
// want any nuclear de-excitation.  But please only do this to understand the
// contribution of de-excitation/pre-equilibrium to the full simulation.
// Running without de-excitation or pre-compound is physically unrealistic!
//
//
void G4DPMJET2_5Model::SetNoDeexcitation ()
{
  if (theExcitationHandler)
  {
    delete theExcitationHandler;
    theExcitationHandler = 0;
  }
}
////////////////////////////////////////////////////////////////////////////////
//
// SetNoPreCompoundModel
//
// Deletes an exiting pre-equilibrium model and zeros the pointer.  This
// allows the simulation to run without any pre-compound, or just -de-excitation
// model only.
//
// Note that you need to separately SetNoDeexcitation, if you REALLY don't want
// any nuclear de-excitation.  But please only do this to understand the
// contribution of de-excitation/pre-equilibrium to the full simulation.
// Running without de-excitation or pre-compound is physically unrealistic!
//
//
void G4DPMJET2_5Model::SetNoPreCompoundModel ()
{
  if (thePreComp)
  {
    delete thePreComp;
    thePreComp = 0;
  }
}
////////////////////////////////////////////////////////////////////////////////
//
// SetDefaultDeexcitation
//
// Note that this is used by the default constructor, as well as can be called
// directly by the user.
//
void G4DPMJET2_5Model::SetDefaultDeexcitation ()
{
//  SetNoDeexcitation();
  
  theExcitationHandler               = new G4ExcitationHandler;
  G4Evaporation * theEvaporation     = new G4Evaporation;
  G4FermiBreakUp * theFermiBreakUp   = new G4FermiBreakUp;
  G4PhotonEvaporation* thePhotonEvap = new G4PhotonEvaporation;
  theExcitationHandler->SetEvaporation(theEvaporation);
  theExcitationHandler->SetFermiModel(theFermiBreakUp);
  theExcitationHandler->SetMaxAandZForFermiBreakUp(17, 9);
  theExcitationHandler->SetFermiModel(theFermiBreakUp);
  theExcitationHandler->SetPhotonEvaporation(thePhotonEvap);
}
////////////////////////////////////////////////////////////////////////////////
//
// SetDefaultPreCompoundModel
//
// Note that this is used by the default constructor, as well as can be called
// directly by the user.
//
void G4DPMJET2_5Model::SetDefaultPreCompoundModel ()
{
//  SetNoPreCompoundModel();
  
  G4ExcitationHandler *anExcitationHandler = new G4ExcitationHandler;
  G4Evaporation * theEvaporation           = new G4Evaporation;
  G4FermiBreakUp * theFermiBreakUp         = new G4FermiBreakUp;
  G4PhotonEvaporation* thePhotonEvap       = new G4PhotonEvaporation;
  anExcitationHandler->SetEvaporation(theEvaporation);
  anExcitationHandler->SetFermiModel(theFermiBreakUp);
  anExcitationHandler->SetMaxAandZForFermiBreakUp(17, 9);
  anExcitationHandler->SetPhotonEvaporation(thePhotonEvap);
  
  thePreComp = new G4PreCompoundModel(anExcitationHandler);
}
////////////////////////////////////////////////////////////////////////////////
//
// SetVerboseFortranOutput
//
G4bool G4DPMJET2_5Model::SetVerboseFortranOutput (const G4String filename)
{
  g4dpmjet_close_fort6_ ();
  if (filename == ""       || filename == "stdo" ||
      filename == "stdout" || filename == "std::out" )
  {
    verboseFortranFile = "std::out";
    return true;
  }
  else
  {
    G4int namelen     = filename.length();
    char *ptr         = new char[namelen+1];
    filename.copy(ptr,namelen,0);
    ptr[namelen]      = '\0';
//    char *ptr         = 0;
//    ptr               = const_cast<char*> (filename.c_str());
    ftnlogical opened = LFALSE; 
    g4dpmjet_open_fort6_ (&namelen, &opened, ptr);
    delete [] ptr;
    if (opened == LTRUE) {
      verboseFortranFile = filename;
      return true;
    } else {
      verboseFortranFile = "std::out";
      return false;
    }
  }
}
////////////////////////////////////////////////////////////////////////////////
//
// PrintWelcomeMessage
//
void G4DPMJET2_5Model::PrintWelcomeMessage () const
{
  G4cout <<G4endl;
  G4cout <<" *****************************************************************"
         <<G4endl;
  G4cout <<" Interface to DPMJET2.5 for nuclear-nuclear interactions activated"
         <<G4endl;
  G4cout <<" Version number : 00.00.0B          File date : 23/05/08" <<G4endl;
  G4cout <<" (Interface written by QinetiQ Ltd for the European Space Agency)"
         <<G4endl;
  G4cout <<G4endl;
  G4cout <<" Initialisation of DPMJET-II.5 variables will be according to "
         <<theInitType <<G4endl;
  G4cout <<" *****************************************************************"
         <<G4endl;
  G4cout << G4endl;

  return;
}
////////////////////////////////////////////////////////////////////////////////
//
// DumpVerboseInformation1
//
// Dumps raw information about the DPMJET-II.5 simulation if verbosity set
// to 4 or more.
//
void G4DPMJET2_5Model::DumpVerboseInformation1 (const G4int n) const
{
  G4cout <<"----------------------------------------"
         <<"----------------------------------------" <<G4endl;
  G4cout <<n <<" INTERMEDIATE AND FINAL-STATE SECONDARIES PRODUCED" <<G4endl;
  if (verboseLevel >= 4)
  {
    G4cout <<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"
           <<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~" <<G4endl;
    G4cout <<"ORIGINAL DPMJET-II.5 OUTPUT FOR EVENT:" <<G4endl;
    G4cout <<"Note that (1) the particles are yet to be transformed according"
           <<G4endl;
    G4cout <<"              to incident particle direction" <<G4endl;
    G4cout <<"          (2) the units of energy, momentum and mass are GeV,"
           <<G4endl;
    G4cout <<"              GeV/c and GeV/c^2 respectively" <<G4endl;
    G4cout <<"    I"
           <<"    ISTHKK"
           <<"     IDHKK"
           <<"     IDRES"
           <<"    IDXRES"
           <<"             PX"
           <<"             PY"
           <<"             PZ"
           <<"   TOTAL ENERGY"
           <<"           MASS"
           <<G4endl;
    for (G4int i=0; i<n; i++)
    {
      G4cout.unsetf(std::ios::scientific);
      G4cout.setf(std::ios::fixed|std::ios::right|std::ios::adjustfield);
      G4cout.precision(0);
      G4cout <<std::setw(5)  <<i
             <<std::setw(10) <<hkkevt_.isthkk[i]
             <<std::setw(10) <<hkkevt_.idhkk[i]
             <<std::setw(10) <<extevt_.idres[i]
             <<std::setw(10) <<extevt_.idxres[i];
      G4cout.unsetf(std::ios::fixed);
      G4cout.setf(std::ios::scientific|std::ios::right|std::ios::adjustfield);
      G4cout.precision(7);
      G4cout <<std::setw(15) <<hkkevt_.phkk[i][0]
             <<std::setw(15) <<hkkevt_.phkk[i][1]
             <<std::setw(15) <<hkkevt_.phkk[i][2]
             <<std::setw(15) <<hkkevt_.phkk[i][3]
             <<std::setw(15) <<hkkevt_.phkk[i][4]
             <<G4endl;
    }
    G4cout <<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~"
           <<"~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~" <<G4endl;
  }
  G4cout.setf(std::ios::fixed);
  G4cout <<" THE FOLLOWING LISTS ONLY THE FINAL-STATE SECONDARIES" <<G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4DPMJET2_5Model::DumpVerboseInformation2
  (const G4String particleName, const G4ThreeVector p,
  const G4double E, const G4double T, const G4ThreeVector pinit) const
{
  G4cout <<"Name = " <<particleName <<G4endl;
  G4cout <<"            Momentum          = " <<p/MeV <<" MeV/c" <<G4endl;
  G4cout <<"            T. Energy         = " <<E/MeV <<" MeV"   <<G4endl;
  G4cout <<"            K. Energy         = " <<T/MeV <<" MeV"   <<G4endl;
  if (verboseLevel >= 3)
  {
    G4ThreeVector axis = pinit.unit();
    G4double pz = p.dot(axis);
    G4cout <<"            Transverse mass   = " <<std::sqrt(E*E-pz*pz)/MeV 
           <<" MeV"
           <<G4endl;
    G4cout <<"            Rapidity          = "
           <<0.5*std::log((E+pz)/(E-pz)) <<G4endl;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4DPMJET2_5Model::DumpVerboseInformation3 (const G4int i,
  const G4int A, const G4int Z, const G4ThreeVector p,
  const G4double E, const G4double T, const G4ThreeVector pinit) const
{
  G4cout <<"----------------------------------------" 
         <<"----------------------------------------" <<G4endl;
  G4cout <<"The nuclear fragment #" <<i <<" before" <<G4endl;
  G4cout <<"----------------------------------------"
         <<"----------------------------------------" <<G4endl;

  std::ostringstream tmpStream;
  tmpStream <<"(A = " <<A <<", Z = " <<Z <<")";
  G4String AZ = tmpStream.str();
  
  DumpVerboseInformation2(AZ, p, E, T, pinit);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void G4DPMJET2_5Model::DumpVerboseInformation4 (const G4int i,
  const G4String particleName, const G4ThreeVector p,
  const G4double E, const G4double T, const G4ThreeVector pinit) const
{
  G4cout <<"----------------------------------------" 
         <<"----------------------------------------" <<G4endl;
  G4cout <<"The nuclear fragment #" <<i <<" after" <<G4endl;
  G4cout <<"----------------------------------------" 
         <<"----------------------------------------" <<G4endl;

  DumpVerboseInformation2(particleName, p, E, T, pinit);
}

////////////////////////////////////////////////////////////////////////////////
//
// Initialise
// This is intended to do exactly what the main program and subroutine DMINIT
// attempt to do with variable initialisation.  i've included most of the
// FORTRAN source code for reference.  Note that I'm certain that many of these
// lines relate to tallying of events for output by the standalone version
// of DPMJET-II.5, but for the moment, I'd rather initialise those variables,
// just in case not doing so has an adverse effect.
//
void G4DPMJET2_5Model::Initialise ()
{
//
//
// This first line is intended to make sure the block data statements are
// executed, since we're not running from a FORTRAN main program.
//
  g4dpmjet_initialise_block_data_ ();
//
//
  dpar_.aam[4]   = 0.001;                        // AAM(5)=0.001D0
  dpar_.aam[5]   = 0.001;                        // AAM(6)=0.001D0
  dpar_.aam[132] = 0.001;                        // AAM(133)=0.001D0
  dpar_.aam[133] = 0.001;                        // AAM(134)=0.001D0
  dpar_.aam[134] = 0.001;                        // AAM(135)=0.001D0
  dpar_.aam[135] = 0.001;                        // AAM(136)=0.001D0

  vxsvd_.nxsp  = 0;                                // NXSP=0
  vxsvd_.nxst  = 0;                                // NXST=0
  vxsvd_.nxsap = 0;                                // NXSAP=0
  vxsvd_.nxsat = 0;                                // NXSAT=0
  vxsvd_.nxvp  = 0;                                // NXVP=0
  vxsvd_.nxvt  = 0;                                // NXVT=0
  vxsvd_.nxdp  = 0;                                // NXDP=0
  vxsvd_.nxdt  = 0;                                // NXDT=0

  for (G4int i=0; i<50; i++)
  {
    vxsvd_.vxsp[i]  = 1.0E-08;                        // VXSP(II)=1.D-8
    vxsvd_.vxst[i]  = 1.0E-08;                        // VXST(II)=1.D-8
    vxsvd_.vxsap[i] = 1.0E-08;                        // VXSAP(II)=1.D-8
    vxsvd_.vxsat[i] = 1.0E-08;                        // VXSAT(II)=1.D-8
    vxsvd_.vxvp[i]  = 1.0E-08;                        // VXVP(II)=1.D-8
    vxsvd_.vxvt[i]  = 1.0E-08;                        // VXVT(II)=1.D-8
    vxsvd_.vxdp[i]  = 1.0E-08;                        // VXDP(II)=1.D-8
    vxsvd_.vxst[i]  = 1.0E-08;                        // VXST(II)=1.D-8
  }

  if (debug)
  {
#ifdef G4VERBOSE
    if (GetVerboseLevel()>0) {
      G4cout <<"AT G4DPMJET2_5Model::Initialise:"  <<G4endl;
    }
#endif
    dprin_.ipri      = debug_level;              // IPRI  = LEVLDB
    dprin_.ipev      = debug_level;              // IPEV  = LEVLDB
    dprin_.ippa      = debug_level;              // IPPA  = LEVLDB
    dprin_.ipco      = debug_level;              // IPCO  = LEVLDB
    dprin_.init      = debug_level;              // INIT  = LEVLDB
    dprin_.iphkk     = debug_level;              // IPHKK = LEVLDB
    pydat1_.mstu[25] = 10;                       // MSTU(26) = 10
  }
  else
  {
    dprin_.ipri      = 0;                        // IPRI  = 0
    dprin_.ipev      = 0;                        // IPEV  = 0
    dprin_.ippa      = 0;                        // IPPA  = 0
    dprin_.ipco      =-2;                        // IPCO  = -2
    dprin_.init      = 0;                        // INIT  = 0
    dprin_.iphkk     = 0;                        // IPHKK = 0
    pydat1_.mstu[25] = 0;                        // MSTU(26) = 10
  }

  for (G4int i=0; i<7; i++)
  {
    diqrej_.idiqre[i] = 0;
    diqrej_.idiqrz[i] = 0;
  }

  for (G4int i=0; i<3; i++)
  {
    diqrej_.idvre[i] = 0;
    diqrej_.ivdre[i] = 0;
    diqrej_.idsre[i] = 0;
    diqrej_.isdre[i] = 0;
    diqrej_.idzre[i] = 0;
    diqrej_.izdre[i] = 0;
  }
  
  diqsum_.ndvuu  = 0;                            // NDVUU     = 0
  diqsum_.ndvus  = 0;                            // NDVUS     = 0
  diqsum_.ndvss  = 0;                            // NDVSS     = 0
  diqsum_.nvduu  = 0;                            // NVDUU     = 0
  diqsum_.nvdus  = 0;                            // NVDUS     = 0
  diqsum_.nvdss  = 0;                            // NVDSS     = 0
  diqsum_.ndsuu  = 0;                            // NDSUU     = 0
  diqsum_.ndsus  = 0;                            // NDSUS     = 0
  diqsum_.ndsss  = 0;                            // NDSSS     = 0
  diqsum_.nsduu  = 0;                            // NSDUU     = 0
  diqsum_.nsdus  = 0;                            // NSDUS     = 0
  diqsum_.nsdss  = 0;                            // NSDSS     = 0
  diqsum_.ndzuu  = 0;                            // NDZUU     = 0
  diqsum_.ndzus  = 0;                            // NDZUS     = 0
  diqsum_.ndzss  = 0;                            // NDZSS     = 0
  diqsum_.nzduu  = 0;                            // NZDUU     = 0
  diqsum_.nzdus  = 0;                            // NZDUS     = 0
  diqsum_.nzdss  = 0;                            // NZDSS     = 0
  diqsum_.nadvuu = 0;                            // NADVUU    = 0
  diqsum_.nadvus = 0;                            // NADVUS    = 0
  diqsum_.nadvss = 0;                            // NADVSS    = 0
  diqsum_.navduu = 0;                            // NAVDUU    = 0
  diqsum_.navdus = 0;                            // NAVDUS    = 0
  diqsum_.navdss = 0;                            // NAVDSS    = 0
  diqsum_.nadsuu = 0;                            // NADSUU    = 0
  diqsum_.nadsus = 0;                            // NADSUS    = 0
  diqsum_.nadsss = 0;                            // NADSSS    = 0
  diqsum_.nasduu = 0;                            // NASDUU    = 0
  diqsum_.nasdus = 0;                            // NASDUS    = 0
  diqsum_.nasdss = 0;                            // NASDSS    = 0
  diqsum_.nadzuu = 0;                            // NADZUU    = 0
  diqsum_.nadzus = 0;                            // NADZUS    = 0
  diqsum_.nadzss = 0;                            // NADZSS    = 0
  diqsum_.nazduu = 0;                            // NAZDUU    = 0
  diqsum_.nazdus = 0;                            // NAZDUS    = 0
  diqsum_.nazdss = 0;                            // NAZDSS    = 0
  hdjase_.nhse1  = 0;                            // NHSE1     = 0
  hdjase_.nhse2  = 0;                            // NHSE2     = 0
  hdjase_.nhse3  = 0;                            // NHSE3     = 0
  hdjase_.nhase1 = 0;                            // NHASE1    = 0
  hdjase_.nhase2 = 0;                            // NHASE2    = 0
  hdjase_.nhase3 = 0;                            // NHASE3    = 0
//
//
// Parton pt distribution.
//
  G4int i      = 1;
  G4double pt1 = 0.0;
  G4double pt2 = 0.0;
  G4int ipt    = 0;
  G4int nevt   = 0;
  parpt_ (&i,&pt1,&pt2,&ipt,&nevt);              
  // CALL PARPT(1,PT1,PT2,IPT,NEVT)
//
//
// Initialise BAMJET, DECAY and HADRIN.
//
  ddatar_ ();                                    // CALL DDATAR
  dhadde_ ();                                    // CALL DHADDE
  dchant_ ();                                    // CALL DCHANT
  dchanh_ ();                                    // CALL DCHANH

  G4double epn = 0.0;
  G4double ppn = 0.0;
  defaul_ (&epn,&ppn);                           // CALL DEFAUL(EPN,PPN)
  defaux_ (&epn,&ppn);                           // CALL DEFAUX(EPN,PPN)

  coulo_.icoul   = 1;                            // ICOUL  = 1
  nuclea_.icoull = 1;                            // ICOULL = 1
  edens_.ieden   = 0;                            // IEDEN = 0
  dprin_.itopd   = 0;                            // ITOPD = 0

  if ( theInitType == DEFAULT ) {
    TAUFOR  = 5.0E+00;
    KTAUGE  = 25;
  } else if ( theInitType == CORSIKA ) {
    TAUFOR  = 5.0E+00;                           // TAUFOR = 5.D0
    KTAUGE  = 25;                                // KTAUGE = 25
  } else if ( theInitType == DPM2_5 ) {
    TAUFOR  = 105.0E+00;                         // TAUFOR = 105.D0
    KTAUGE  = 10;                                // KTAUGE = 10
  } else if ( theInitType == DPM3 ) {
    TAUFOR  = 3.5E+00;
    KTAUGE  = 10;
  }
  taufo_.taufor = TAUFOR;
  taufo_.ktauge = KTAUGE;

  ITAUVE        = 1;                             // ITAUVE = 1
  taufo_.itauve = ITAUVE;
  taufo_.incmod = 1;                             // INCMOD = 1
//
//
// Definition of soft quark distributions, Fermi, Pauli
//
  G4double xseaco = 1.0;                         // XSEACO = 1.00D0
  xseadi_.xseacu  = 1.05 - xseaco;               // XSEACU = 1.05D0-XSEACO

  if ( theInitType == DEFAULT ) {
    UNON           = 3.50;
    UNOM           = 1.11;
    UNOSEA         = 5.0;
    droppt_.fermp  = LTRUE;
    nucimp_.fermod = 0.6;
  } else if ( theInitType == CORSIKA ) {
    UNON           = 3.50;                       // UNON   = 3.50D0
    UNOM           = 1.11;                       // UNOM   = 1.11D0
    UNOSEA         = 5.0;                        // UNOSEA = 5.0D0
    droppt_.fermp  = LTRUE;                      // FERMP  = .TRUE.
    nucimp_.fermod = 0.6;                        // FERMOD = 0.6D0
  } else if ( theInitType == DPM2_5 ) { 
    UNON           = 3.50;                       // UNON   = 3.50D0
    UNOM           = 1.11;                       // UNOM   = 1.11D0
    UNOSEA         = 5.0;                        // UNOSEA = 5.0D0
    droppt_.fermp  = LTRUE;                      // FERMP  = .TRUE.
    nucimp_.fermod = 0.6;                        // FERMOD = 0.6D0
  } else if ( theInitType == DPM3 ) { 
    UNON           = 2.00;
    UNOM           = 1.5;
    UNOSEA         = 5.0;
    droppt_.fermp  = LTRUE;
    nucimp_.fermod = 0.55;
  }
  xseadi_.unon   = UNON;
  xseadi_.unom   = UNOM;
  xseadi_.unosea = UNOSEA;
  
  nuclea_.fermdd = 0.6;                          // FERMDD = 0.6D0
  ferfor_.iferfo = 1;                            // IFERFO = 1
  dprin_.ipaupr  = 0;                            // IPAUPR = 0
  droppt_.lpauli = LTRUE;                        // LPAULI = .TRUE.
//
//
// Definition of cuts for x-sampling.
//
  if ( theInitType == DEFAULT ) {
    CVQ    = 1.8;
    CDQ    = 2.0;
    CSEA   = 0.5;
    SSMIMA = 0.9;
  } else if ( theInitType == CORSIKA ) {
    CVQ    = 1.8;                                // CVQ  = 1.8D0
    CDQ    = 2.0;                                // CDQ  = 2.0D0
    CSEA   = 0.5;                                // CSEA = 0.5D0
    SSMIMA = 0.901;                              // SSMIMA = 0.901D0
  } else if ( theInitType == DPM2_5 ) {
    CVQ    = 1.8;                                // CVQ  = 1.8D0
    CDQ    = 2.0;                                // CDQ  = 2.0D0
    CSEA   = 0.5;                                // CSEA = 0.5D0
    SSMIMA = 1.201;                              // SSMIMA = 1.201D0
  } else if ( theInitType == DPM3 ) {
    CVQ    = 1.0;
    CDQ    = 2.0;
    CSEA   = 0.1;
    SSMIMA = 0.14;
  }
  xseadi_.cvq    = CVQ;
  xseadi_.cdq    = CDQ;
  xseadi_.csea   = CSEA;
  xseadi_.ssmima = SSMIMA;
  
  xseadi_.ssmimq = xseadi_.ssmima * xseadi_.ssmima;
                                                 // SSMIMQ = SSMIMA**2
  if ( theInitType == DEFAULT ) {
    VVMTHR = 0.0;  
  } else if ( theInitType == CORSIKA ) {
    VVMTHR = 0.0;                                // VVMTHR = 0.D0
  } else if ( theInitType == DPM2_5 ) {
    VVMTHR = 0.0;                                // VVMTHR = 0.D0
  } else if ( theInitType == DPM3 ) {
    VVMTHR = 2.0;
  }
  xseadi_.vvmthr = VVMTHR;
//
//
// There is a final call.  Set recombin, seasu3, coninpt, allpart and interdpm
//
  final_.ifinal  = 0;                            // IFINAL = 0
  recom_.irecom  = 0;                            // IRECOM = 0
  seadiq_.lseadi = LTRUE;                        // LSEADI = .TRUE.

  if ( theInitType == DEFAULT ) {
    SEASQ  = 0.5;
    MKCRON = 1;
    CRONCO = 0.64;
  } else if ( theInitType == CORSIKA ) {
    SEASQ  = 0.5;                                // SEASQ  = 0.50D0
    MKCRON = 0;                                  // MKCRON = 0
    CRONCO = 0.0;                                // CRONCO = 0.00D0
  } else if ( theInitType == DPM2_5 ) {
    SEASQ  = 0.5;                                // SEASQ  = 0.50D0
    MKCRON = 1;                                  // MKCRON = 1
    CRONCO = 0.64;                               // CRONCO = 0.64D0
  } else if ( theInitType == DPM3 ) {
    SEASQ  = 1.0;
    MKCRON = 1;
    CRONCO = 0.64;
  }
  seasu3_.seasq  = SEASQ;
  cronin_.mkcron = MKCRON;
  cronin_.cronco = CRONCO;
  
  droppt_.ihada  = LTRUE;                        // IHADA  = .TRUE.
  
//  inxdpm_.intdpm = 0;                          // INTDPM = 0
// Note FORTRAN initialises the variable IROEH  = 0, but this isn't used
// nor is it contained within a common block.
//
//
// Definition for popcork, casadiqu, popcorse
//
  popcck_.pdbck  = 0.0;                         // PDBCK  = 0.D0
  popcck_.ijpock = 0;                           // IJPOCK = 0
  casadi_.icasad = 1;                           // ICASAD = 1
  casadi_.casaxx = 0.05;                        
  // CASAXX = 0.05D0             ! corrected Nov. 2001
  popcck_.pdbse  = 0.45;                        
  // PDBSE  = 0.45D0             ! with baryon stopping
  popcck_.pdbseu = 0.45;                        
  // PDBSEU = 0.45D0             ! with baryon stopping
  // PDBSE  = 0.D0               ! without baryon stopping
  // PDBSEU = 0.D0               ! without baryon stopping
  popcck_.irejck = 0;                           // IREJCK = 0
  popcck_.irejse = 0;                           // IREJSE = 0
  popcck_.irejs3 = 0;                           // IREJS3 = 0
  popcck_.irejs0 = 0;                           // IREJS0 = 0
  popcck_.ick4   = 0;                           // ICK4   = 0
  popcck_.ise4   = 0;                           // ISE4   = 0
  popcck_.ise43  = 0;                           // ISE43  = 0
  popcck_.ihad4  = 0;                           // IHAD4  = 0
  popcck_.ick6   = 0;                           // ICK6   = 0
  popcck_.ise6   = 0;                           // ISE6   = 0
  popcck_.ise63  = 0;                           // ISE63  = 0
  popcck_.ihad6  = 0;                           // IHAD6  = 0
  popcck_.irejsa = 0;                           // IREJSA = 0
  popcck_.ireja3 = 0;                           // IREJA3 = 0
  popcck_.ireja0 = 0;                           // IREJA0 = 0
  popcck_.isea4  = 0;                           // ISEA4  = 0
  popcck_.isea43 = 0;                           // ISEA43 = 0
  popcck_.ihada4 = 0;                           // IHADA4 = 0
  popcck_.isea6  = 0;                           // ISEA6  = 0
  popcck_.isea63 = 0;                           // ISEA63 = 0
  popcck_.ihada6 = 0;                           // IHADA6 = 0

  if ( theInitType == DEFAULT || theInitType == CORSIKA ) {
    popcor_.pdb    = 0.1;                       // PDB    = 0.10D0
  } else if ( theInitType == DPM2_5 ) {
    popcor_.pdb    = 0.1;                       // PDB    = 0.10D0
  } else if ( theInitType == DPM3 ) {
    popcor_.pdb    = 0.15;
  }
  popcor_.ajsdef = 0.0;                         // AJSDEF = 0.D0
//
//
// Definition of fluctuat, intpt, hadroniz, diquarks, singlech, evapor
// (Charmed particles set to decay : IHADRINZ>=2
//
  fluctu_.ifluct = 0;                           // IFLUCT = 0
  droppt_.intpt  = LTRUE;                       // INTPT  = .TRUE.
  colle_.ihadrz  = 2;                           // IHADRZ = 2
  ifragm_.ifrag  = 1;                           // IFRAG  = 1
  promu_.ipromu  = 1;                           // IPROMU = 1
  if (colle_.ihadrz >= 2)
  {
    ifragm_.ifrag  = colle_.ihadrz - 1;
                                                // IFRAG = IHADRZ-1
    lundin_ ();                                 // CALL LUNDIN
  }
  diquax_.idiqua  = 1;                          // IDIQUA = 1
  diquax_.idiquu  = 1;                          // IDIQUU = 1
  diquax_.amedd   = 0.9;                        // AMEDD  = 0.9D0
  sincha_.isicha  = 0;                          // ISICHA = 0
  evappp_.ievap   = 0;                          // IEVAP = 0
  seaqxx_.seaqx   = 0.5;                        // SEAQX = 0.5D0
  seaqxx_.seaqxn  = 0.5;                        // SEAQXN = 0.5D0
  kglaub_.jglaub  = 2;                          // JGLAUB = 2
  hadthr_.ehadth  = 5.0;                        // EHADTH = 5.D0
//
//
// Definitions for hbook, pomtable, cmhisto, central, strucfun
//
  hboo_.ihbook    = 1;                          // IHBOOK = 1
  pomtab_.ipomta  = 1;                          // IPOMTA = 1
  cmhico_. cmhis  = 0.0;                        
  // CMHIS = 0.0D+00           !   Lab System
  zentra_.icentr  = 0;                          // ICENTR = 0
  user2_.istruf   = 222;                        // ISTRUF = 222
  strufu_.istrum  = 0;                          // ISTRUM = 0
  strufu_.istrut  = user2_.istruf / 100;
                                                // ISTRUT = ISTRUF/100
  user2_.istruf   = user2_.istruf - strufu_.istrut*100;
                                                // ISTRUF = ISTRUF-ISTRUT*100
  strufu_.istrum  = user2_.istruf;              // ISTRUM = ISTRUF

  if ( theInitType == DEFAULT ) {
    ISINGD = 1;
    ISINGX = 1;
    IDUBLD = 0;
    SDFRAC = 1.0;
  } else if ( theInitType == CORSIKA ) {
    ISINGD  = 0;                                // ISINGD = 0
    ISINGX  = 0;                                // ISINGX = 0
    IDUBLD  = 0;                                // IDUBLD = 0
    SDFRAC  = 0.0;                              // SDFRAC = 0.
  } else if ( theInitType == DPM2_5 ) {
    ISINGD  = 1;                                // ISINGD = 1
    ISINGX  = 1;                                // ISINGX = 1
    IDUBLD  = 0;                                // IDUBLD = 0
    SDFRAC  = 1.0;                              // SDFRAC = 1.
  } else if ( theInitType == DPM3 ) {
    ISINGD  = 0;
    ISINGX  = 1;
    IDUBLD  = 0;
    SDFRAC  = 1.0;
  }
  diffra_.isingd = ISINGD;
  user2_.isingx  = ISINGX;
  user2_.idubld  = IDUBLD;
  user2_.sdfrac  = SDFRAC;
//
//
// Definitions for start, inforeje, gluxplit, partev, sampt
//  NEVNTS passed to DTMAI as argument, NEVHAD in COMMON
//
  colle_.nfile    = 0;                          // NFILE  = 0
  nstari_.nstart  = 1;                          // NSTART = 1
  stars_.istar2   = 0;                          // ISTAR2 = 0
  stars_.istar3   = 0;                          // ISTAR3 = 0
  user2_.ptlar    = 2.0;                        // PTLAR  = 2.D0

//  G4int iglaub    = 0;                        
// IGLAUB = 0 !! Note that this is just a local variable

//  infore_.ifrej   = 0;                        
// IFREJ  = 0  Note rejection diagnostics not required
  gluspl_.nugluu  = 1;                          // NUGLUU = 1
  gluspl_.nsgluu  = 0;                          // NSGLUU = 0
  colle_.nvers    = 1;                          // NVERS  = 1
  ptsamp_.isampt  = 4;                          // ISAMPT = 4
//
//
// Definitions for selhard, sigmapom, pshow, secinter.
//
  dropjj_.dropjt  = 0.0;                        // DROPJT = 0.D0
  collis_.iophrd  = 2;                          // IOPHRD = 2
  collis_.ptthr   = 3.0;                        // PTTHR  = 3.D0
//  collis_.ptthr2  = collis_.ptthr;            // PTTHR2 = PTTHR
  user2_.cmener   = 100.0;                      // CMENER = 100.D0
  if (strufu_.istrut == 1)
  {
    collis_.ptthr = 2.1 + 0.15*std::pow(std::log10(user2_.cmener/50.),3.0);
    // PTTHR  = 2.1D0+0.15D0*(LOG10(CMENER/50.))**3
  }
  else if (strufu_.istrut == 2)
  {
    collis_.ptthr = 2.5 + 0.12*std::pow(std::log10(user2_.cmener/50.),3.0);
    // PTTHR  = 2.5D0+0.12D0*(LOG10(CMENER/50.))**3
  }
  collis_.ptthr2  = collis_.ptthr;              // PTTHR2 = PTTHR
  pomtyp_.ipim    = 2;                          // IPIM   = 2
  pomtyp_.icon    = 48;                         // ICON   = 48
  pomtyp_.isig    = 10;                         // ISIG   = 10
  pomtyp_.lmax    = 30;                         // LMAX   = 30
  pomtyp_.mmax    = 100;                        // MMAX   = 100
  pomtyp_.nmax    = 2;                          // NMAX   = 2
  pomtyp_.difel   = 0.0;                        // DIFEL  = 0.D0
  pomtyp_.difnu   = 1.0;                        // DIFNU  = 1.D0
  pshow_.ipshow   = 1;                          // IPSHOW = 1
  secint_.isecin  = 0;                          // ISECIN = 0
//
//
// This next bit is associated with evaporation.  I'm not sure if it's needed
// as IEVAP = 0, but will initialise in any case.
//
  if ( theInitType == DEFAULT ) {
    parevt_.levprt  = LTRUE;
    parevt_.ilvmod  = 1;
    parevt_.ldeexg  = LFALSE;
    parevt_.lheavy  = LFALSE;
    frbkcm_.lfrmbk  = LFALSE;
    inpflg_.ifiss   = 0;
  } else if ( theInitType == CORSIKA ) {
    parevt_.levprt  = LTRUE;                    // LEVPRT = .TRUE.
    parevt_.ilvmod  = 1;                        // ILVMOD = 1
    parevt_.ldeexg  = LTRUE;                    // LDEEXG = .TRUE.
    parevt_.lheavy  = LTRUE;                    // LHEAVY = .TRUE.
    frbkcm_.lfrmbk  = LTRUE;                    // LFRMBK = .TRUE.
    inpflg_.ifiss   = 0;                        // IFISS  = 0
  } else if ( theInitType == DPM2_5 ) {
    parevt_.levprt  = LFALSE;                   // LEVPRT = .FALSE.
    parevt_.ilvmod  = 1;                        // ILVMOD = 1
    parevt_.ldeexg  = LFALSE;                   // LDEEXG = .FALSE.
    parevt_.lheavy  = LFALSE;                   // LHEAVY = .FALSE.
    frbkcm_.lfrmbk  = LFALSE;                   // LFRMBK = .FALSE.
    inpflg_.ifiss   = 0;                        // IFISS  = 0
  } else if ( theInitType == DPM3 ) {
    parevt_.levprt  = LFALSE;
    parevt_.ilvmod  = 1;
    parevt_.ldeexg  = LFALSE;
    parevt_.lheavy  = LFALSE;
    frbkcm_.lfrmbk  = LFALSE;
    inpflg_.ifiss   = 0;
  }

  hettp_.nbertp   = lunber;                     // NBERTP = LUNBER

  verboseFortranFile = "fort.6";
  G4int namelen      = verboseFortranFile.length();
  char *ptr1         = new char[namelen+1];
  verboseFortranFile.copy(ptr1,namelen,0);
  ptr1[namelen]      = '\0';
//  ptr1               = const_cast<char*> (verboseFortranFile.c_str());
  ftnlogical opened  = LFALSE; 
  g4dpmjet_open_fort6_ (&namelen, &opened, ptr1);
  delete [] ptr1;
  if (opened == LFALSE)
  {
    G4cout <<"ATTEMPTED TO OPEN fort.6 TO OUTPUT VERBOSE FORTRAN TEXT" 
           <<G4endl;
    G4cout <<"HOWEVER THIS WAS NOT POSSIBLE" <<G4endl;
  }

#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"AT G4DPMJET2_5Model::Initialise: before NUCLEAR.BIN"  <<G4endl;
    G4cout <<"OPENING NUCLEAR.BIN ON FILE UNIT " <<lunber <<G4endl;
  }
#endif
  if ( !getenv("G4DPMJET2_5DATA") )
  {
    G4cout <<"ENVIRONMENT VARIABLE G4DPMJET2_5DATA NOT SET " <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__, 
      "Please setenv G4DPMJET2_5DATA to point to the dpmjet2.5 data files.");
  }
  defaultDirName = G4String(getenv("G4DPMJET2_5DATA")) + "/NUCLEAR.BIN";
  namelen        = defaultDirName.length();
  ptr1           = new char[namelen+1];
  defaultDirName.copy(ptr1,namelen,0);
  ptr1[namelen]  = '\0';
//  ptr1           = const_cast<char*> (defaultDirName.c_str());
  opened         = LFALSE;
  g4dpmjet_open_nuclear_bin_ (&namelen, &lunber, &opened, ptr1);
  delete [] ptr1;
  if (opened == LFALSE)
  {
//
//
// Problems with locating NUCLEAR.BIN file.
//
    G4cout <<"NUCLEAR.BIN FILE NOT FOUND IN DIRECTORY " <<defaultDirName
           <<G4endl;
    throw G4HadronicException(__FILE__, __LINE__,
    "NUCLEAR.BIN file not present.");
  }
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0)
    G4cout <<"AT G4DPMJET2_5Model::Initialise: after NUCLEAR.BIN"  <<G4endl;
#endif
  G4cout << "CALL BERTTP" << G4endl;
  berttp_ ();                                        // CALL BERTTP
  G4cout << "CALL BERTTP done" << G4endl;
  if (evappp_.ievap == 1)
  {
#ifdef G4VERBOSE
    if (GetVerboseLevel()>0)
      G4cout <<"AT G4DPMJET2_5Model::Initialise: before INCINI"  <<G4endl;
#endif
    incini_ ();                                      // CALL INCINI
#ifdef G4VERBOSE
    if (GetVerboseLevel()>0)
      G4cout <<"AT G4DPMJET2_5Model::Initialise: after INCINI"  <<G4endl;
#endif
  }
  g4dpmjet_close_nuclear_bin_ (&lunber);
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0)
    G4cout <<"AT G4DPMJET2_5Model::Initialise: NUCLEAR.BIN closed"  <<G4endl;
#endif

  ptlarg_.xsmax = 0.8;                              // XSMAX  = 0.8D0
  dprin_.itopd  = 0;                                // ITOPD  = 0

  G4int iseed1 = 0;
  G4int iseed2 = 0;
  rd2out_ (&iseed1,&iseed2);
//
//
// This next bit outputs important variables if debug is switched on.
//
#ifdef G4VERBOSE
  if (GetVerboseLevel()>0) {
    G4cout <<"AT G4DPMJET2_5Model::Initialise:" <<G4endl;
    G4cout <<"Printout of important Parameters before DPMJET2.5" <<G4endl;
    G4cout <<"Please note for DPMJET input all numbers are floating point!" 
           <<G4endl;
    G4cout <<"PROJPAR  " <<nucc_.ip       <<" "
                         <<nucc_.ipz      <<G4endl;
    G4cout <<"TARPAR   " <<nucc_.it       <<" "
                         <<nucc_.itz      <<G4endl;
    G4cout <<"MOMENTUM " <<ppn            <<G4endl;
    G4cout <<"ENERGY   " <<epn            <<G4endl;
    G4cout <<"CMENERGY " <<nncms_.umo     <<G4endl;
    G4cout <<"NOFINALE " <<final_.ifinal  <<G4endl;
    G4cout <<"EVAPORAT " <<evappp_.ievap  <<G4endl;
    G4cout <<"OUTLEVEL " <<dprin_.ipri    <<" "
                         <<dprin_.ipev    <<" "
                         <<dprin_.ippa    <<" "
                         <<dprin_.ipco    <<" "
                         <<dprin_.init    <<" "
                         <<dprin_.iphkk   <<G4endl;
    G4cout <<"RANDOMIZ " <<iseed1         <<" "
                         <<iseed2         <<G4endl;
    G4cout <<"STRUCFUN " <<user2_.istruf+100*strufu_.istrut <<G4endl;
    G4cout <<"SAMPT    " <<ptsamp_.isampt <<G4endl;
    G4cout <<"SELHARD  " <<0              <<" "
                         <<collis_.iophrd <<" "
                         <<0              <<" "
                         <<dropjj_.dropjt <<" "
                         <<collis_.ptthr  <<" "
                         <<collis_.ptthr2 << G4endl;
    G4cout <<"SIGMAPOM " <<0              <<" "
                         <<pomtyp_.isig   <<" "
                         <<pomtyp_.ipim + 10*pomtyp_.icon <<" "
                         <<pomtyp_.lmax   <<" "
                         <<pomtyp_.mmax   <<" "
                         <<pomtyp_.nmax   <<G4endl;
    G4cout <<"PSHOWER  " <<pshow_.ipshow  <<G4endl;
    G4cout <<"CENTRAL  " <<zentra_.icentr <<G4endl;
    G4cout <<"CMHISTO  " <<cmhico_.cmhis  <<G4endl;
    G4cout <<"SEASU3   " <<seasu3_.seasq  <<G4endl;
    G4cout <<"RECOMBIN " <<recom_.irecom  <<G4endl;
    G4cout <<"SINGDIFF " <<diffra_.isingd <<G4endl;
    G4cout <<"TAUFOR   " <<taufo_.taufor  <<" "
                         <<taufo_.ktauge  <<" "
                         <<taufo_.itauve  <<G4endl;
    G4cout <<"POPCORN  " <<popcor_.pdb    <<G4endl;
    G4cout <<"POPCORCK " <<popcck_.ijpock <<" "
                         <<popcck_.pdbck  <<G4endl;
    G4cout <<"POPCORSE " <<popcck_.pdbse  <<" "
                         <<popcck_.pdbseu <<G4endl;
    G4cout <<"CASADIQU " <<casadi_.icasad <<" "
                         <<casadi_.casaxx <<G4endl;
    G4cout <<"DIQUARKS " <<diquax_.idiqua <<" "
                         <<diquax_.idiquu <<" "
                         <<diquax_.amedd  <<G4endl;
    G4cout <<"HADRONIZ " <<colle_.ihadrz  <<G4endl;
    G4cout <<"INTPT    " <<droppt_.intpt  <<G4endl;
    G4cout <<"PAULI    " <<droppt_.lpauli <<G4endl;
    G4cout <<"FERMI    " <<droppt_.fermp  <<" "
                         <<nucimp_.fermod <<G4endl;
    G4cout <<"CRONINPT " <<cronin_.mkcron <<" "
                         <<cronin_.cronco <<G4endl;
    G4cout <<"SEADISTR " <<xseadi_.xseacu+0.95 <<" "
                         <<xseadi_.unon   <<" "
                         <<xseadi_.unom   <<" "
                         <<xseadi_.unosea <<G4endl;
    G4cout <<"SEAQUARK " <<seaqxx_.seaqx  <<" "
                         <<seaqxx_.seaqxn <<G4endl;
    G4cout <<"SECINTER " <<secint_.isecin <<G4endl;
    G4cout <<"XCUTS    " <<xseadi_.cvq    <<" "
                         <<xseadi_.cdq    <<" "
                         <<xseadi_.csea   <<" "
                         <<xseadi_.ssmima <<G4endl;
  }
#endif

  bufues_.bnnvv   = 0.001;                      // BNNVV=0.001
  bufues_.bnnss   = 0.001;                      // BNNSS=0.001
  bufues_.bnnsv   = 0.001;                      // BNNSV=0.001
  bufues_.bnnvs   = 0.001;                      // BNNVS=0.001
  bufues_.bnncc   = 0.001;                      // BNNCC=0.001
  bufues_.bnndv   = 0.001;                      // BNNDV=0.001
  bufues_.bnnvd   = 0.001;                      // BNNVD=0.001
  bufues_.bnnds   = 0.001;                      // BNNDS=0.001
  bufues_.bnnsd   = 0.001;                      // BNNSD=0.001
  bufues_.bnnhh   = 0.001;                      // BNNHH=0.001
  bufues_.bnnzz   = 0.001;                      // BNNZZ=0.001
  bufues_.bnndi   = 0.001;                      // BNNDI=0.001
  bufues_.bnnzd   = 0.001;                      // BNNZD=0.001
  bufues_.bnndz   = 0.001;                      // BNNDZ=0.001
  bufues_.bptvv   = 0.0;                        // BPTVV=0.
  bufues_.bptss   = 0.0;                        // BPTSS=0.
  bufues_.bptsv   = 0.0;                        // BPTSV=0.
  bufues_.bptvs   = 0.0;                        // BPTVS=0.
  bufues_.bptcc   = 0.0;                        // BPTCC=0.
  bufues_.bptdv   = 0.0;                        // BPTDV=0.
  bufues_.bptvd   = 0.0;                        // BPTVD=0.
  bufues_.bptds   = 0.0;                        // BPTDS=0.
  bufues_.bptsd   = 0.0;                        // BPTSD=0.
  bufues_.bpthh   = 0.0;                        // BPTHH=0.
  bufues_.bptzz   = 0.0;                        // BPTZZ=0.
  bufues_.bptdi   = 0.0;                        // BPTDI=0.
  bufues_.bptzd   = 0.0;                        // BPTZD=0.
  bufues_.bptdz   = 0.0;                        // BPTDZ=0.
  bufues_.beevv   = 0.0;                        // BEEVV=0.
  bufues_.beess   = 0.0;                        // BEESS=0.
  bufues_.beesv   = 0.0;                        // BEESV=0.
  bufues_.beevs   = 0.0;                        // BEEVS=0.
  bufues_.beecc   = 0.0;                        // BEECC=0.
  bufues_.beedv   = 0.0;                        // BEEDV=0.
  bufues_.beevd   = 0.0;                        // BEEVD=0.
  bufues_.beeds   = 0.0;                        // BEEDS=0.
  bufues_.beesd   = 0.0;                        // BEESD=0.
  bufues_.beehh   = 0.0;                        // BEEHH=0.
  bufues_.beezz   = 0.0;                        // BEEZZ=0.
  bufues_.beedi   = 0.0;                        // BEEDI=0.
  bufues_.beezd   = 0.0;                        // BEEZD=0.
  bufues_.beedz   = 0.0;                        // BEEDZ=0.
  ncoucs_.bcouvv  = 0.0;                        // BCOUVV=0.
  ncoucs_.bcouss  = 0.0;                        // BCOUSS=0.
  ncoucs_.bcousv  = 0.0;                        // BCOUSV=0.
  ncoucs_.bcouvs  = 0.0;                        // BCOUVS=0.
  ncoucs_.bcouzz  = 0.0;                        // BCOUZZ=0.
  ncoucs_.bcouhh  = 0.0;                        // BCOUHH=0.
  ncoucs_.bcouds  = 0.0;                        // BCOUDS=0.
  ncoucs_.bcousd  = 0.0;                        // BCOUSD=0.
  ncoucs_.bcoudz  = 0.0;                        // BCOUDZ=0.
  ncoucs_.bcouzd  = 0.0;                        // BCOUZD=0.
  ncoucs_.bcoudi  = 0.0;                        // BCOUDI=0.
  ncoucs_.bcoudv  = 0.0;                        // BCOUDV=0.
  ncoucs_.bcouvd  = 0.0;                        // BCOUVD=0.
  ncoucs_.bcoucc  = 0.0;                        // BCOUCC=0.
//
//
// Initialisation of
//   ANNVV, ANNSS ... ANNDZ
//   PTVV, PTSS ... PTDZ
//   EEVV, EESS ... EEDZ
//   ACOUVV, ACOUSS, ... ACOUCC
// now all moved to ApplyYourself member function.
//
//  droppt_.ipadis = LFALSE;                        // IPADIS = .FALSE.
//  droppt_.ihadvv = LFALSE;                        // IHADVV = .FALSE.
//  droppt_.ihadsv = LFALSE;                        // IHADSV = .FALSE.
//  droppt_.ihadvs = LFALSE;                        // IHADVS = .FALSE.

  nucc_.ijtarg   = 1;                               // IJTARG=1

//
//
// The following commented out since it seems to have more to do with histogram
// generation.
//
//  i = 1;
//  G4int idummy;
//  distr_ (&i,&nucc_.ijproj,&ppn,&idummy);        
// CALL DISTR( 1,IJPROJ,PPN,IDUMMY )
  if (pomtyp_.ipim == 2) {prblm2_ (&user2_.cmener);}
//
//
// Initialise hard scattering & transverse momentum for soft scattering
//
  i = 0;
  jtdtu_ (&i);                                     // CALL JTDTU( 0 )
  i = 0;
  G4double pt;
  samppt_ (&i,&pt);                                // CALL SAMPPT(0,PT)
}
////////////////////////////////////////////////////////////////////////////////
//
#endif
