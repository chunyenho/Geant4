
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$
//
/// \file hbook/src/ExG4HbookNtupleManager.cc
/// \brief Implementation of the ExG4HbookNtupleManager class

// Author: Ivana Hrivnacova, 15/06/2011  (ivana@ipno.in2p3.fr)

#ifdef G4_USE_HBOOK

#include "ExG4HbookNtupleManager.hh"
#include "ExG4HbookFileManager.hh"
#include "G4AnalysisManagerState.hh"
#include "G4AnalysisUtilities.hh"
#include "G4UnitsTable.hh"

#include <iostream>

using namespace G4Analysis;

//_____________________________________________________________________________
ExG4HbookNtupleManager::ExG4HbookNtupleManager(const G4AnalysisManagerState& state)
 : G4VNtupleManager(state),
   fNtupleHbookIdOffset(-1),
   fNtupleDescriptionVector(),
   fNtupleVector()
{
}

//_____________________________________________________________________________
ExG4HbookNtupleManager::~ExG4HbookNtupleManager()
{  
  // Reset();

  std::vector<ExG4HbookNtupleDescription*>::iterator it;  
  for (it = fNtupleDescriptionVector.begin(); it != fNtupleDescriptionVector.end(); it++ ) {
    delete *it;
  }   
}

// 
// private methods
//

//_____________________________________________________________________________
void ExG4HbookNtupleManager::SetNtupleHbookIdOffset()
{
// Set  fH1HbookIdOffset if needed

  if ( fNtupleHbookIdOffset == -1 ) {
    if ( fFirstId > 0 ) 
      fNtupleHbookIdOffset = 0;
    else
      fNtupleHbookIdOffset = 1;
        
    if ( fNtupleHbookIdOffset > 0 ) {
      G4ExceptionDescription description;
      description << "Ntuple will be defined in HBOOK with ID = G4_firstNtupleId + 1";
      G4Exception("ExG4HbookNtupleManager::SetNtupleHbookIdOffset()",
                  "Analysis_W013", JustWarning, description);
    }              
  }
}  

//_____________________________________________________________________________
void ExG4HbookNtupleManager::CreateNtuplesFromBooking()
{
// Create ntuple from ntuple_booking.

  if ( ! fNtupleDescriptionVector.size() ) return;     
  
  // Set fNtupleHbookIdOffset if needed
  SetNtupleHbookIdOffset();
  
  G4int index = 0;
  std::vector<ExG4HbookNtupleDescription*>::iterator itn;  
  for (itn = fNtupleDescriptionVector.begin(); itn != fNtupleDescriptionVector.end(); itn++ ) {

    tools::ntuple_booking* ntupleBooking = (*itn)->fNtupleBooking;
    if ( ! ntupleBooking ) continue;

#ifdef G4VERBOSE
    if ( fState.GetVerboseL4() ) 
      fState.GetVerboseL4()
        ->Message("create from booking", "ntuple", ntupleBooking->name());
#endif

    // Create an "ntuple" directory both in memory and in the file
    fFileManager->CreateNtupleDirectory();
    G4int hbookIndex = fNtupleHbookIdOffset + index + fFirstId;
    ++index;

    // We should be under //PAWC/LUN1/ntuple
    (*itn)->fNtuple
      = new tools::hbook::wntuple(hbookIndex, G4cout, *ntupleBooking);
    fNtupleVector.push_back((*itn)->fNtuple);  

    if ( ntupleBooking->columns().size() ) {
      // store ntuple columns in local maps
      const std::vector<tools::column_booking>& columns 
        = ntupleBooking->columns();
      std::vector<tools::column_booking>::const_iterator it;
      G4int counter = 0;
      for ( it = columns.begin(); it!=columns.end(); ++it) {
        if ( it->cls_id() == tools::_cid(int(0) ) ) {
          (*itn)->fNtupleIColumnMap[counter++] 
            = (*itn)->fNtuple->find_column<int>(it->name());
        }
        else if( it->cls_id() == tools::_cid(float(0) ) ) {
          (*itn)->fNtupleFColumnMap[counter++] 
            = (*itn)->fNtuple->find_column<float>(it->name());
        } 
        else if(it->cls_id()== tools::_cid(double(0))) {
          (*itn)->fNtupleDColumnMap[counter++] 
            = (*itn)->fNtuple->find_column<double>(it->name());
        }
        else {
          G4ExceptionDescription description;
          description << "      " 
                      << "Unsupported column type " << it->name();
          G4Exception("G4HbookAnalysisManager::CreateNtupleFromBooking()",
                      "Analysis_W002", JustWarning, description);
        }
      }
    }
    FinishNtuple();
#ifdef G4VERBOSE
  if ( fState.GetVerboseL3() ) 
    fState.GetVerboseL3()
      ->Message("create from booking", "ntuple", ntupleBooking->name());
#endif
  }  
}   

//_____________________________________________________________________________
tools::hbook::wntuple::column<int>*    
ExG4HbookNtupleManager::GetNtupleIColumn(G4int ntupleId, G4int columnId) const
{
  ExG4HbookNtupleDescription* ntupleDecription
    = GetNtupleInFunction(ntupleId, "GetNtupleIColumn");
  if ( ! ntupleDecription ) return 0;

  std::map<G4int, tools::hbook::wntuple::column<int>* >& ntupleIColumnMap
    = ntupleDecription->fNtupleIColumnMap;
  std::map<G4int, tools::hbook::wntuple::column<int>* >::const_iterator it
    = ntupleIColumnMap.find(columnId);
  if ( it == ntupleIColumnMap.end() ) {
    G4ExceptionDescription description;
    description << "      "  << "ntupleId " << ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::GetNtupleIColumn()",
                "Analysis_W011", JustWarning, description);
    return 0;
  }
  
  return it->second;
}  
    
//_____________________________________________________________________________
tools::hbook::wntuple::column<float>*  
ExG4HbookNtupleManager::GetNtupleFColumn(G4int ntupleId, G4int columnId) const
{
  ExG4HbookNtupleDescription* ntupleDecription
    = GetNtupleInFunction(ntupleId, "GetNtupleFColumn");
  if ( ! ntupleDecription ) return 0;

  std::map<G4int, tools::hbook::wntuple::column<float>* >& ntupleFColumnMap
    = ntupleDecription->fNtupleFColumnMap;
  std::map<G4int, tools::hbook::wntuple::column<float>* >::const_iterator it
    = ntupleFColumnMap.find(columnId);
  if ( it == ntupleFColumnMap.end() ) {
    G4ExceptionDescription description;
    description << "      "  << "ntupleId " << ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::GetNtupleFColumn()",
                "Analysis_W011", JustWarning, description);
    return 0;
  }
  
  return it->second;
}  

//_____________________________________________________________________________
tools::hbook::wntuple::column<double>* 
ExG4HbookNtupleManager::GetNtupleDColumn(G4int ntupleId, G4int columnId) const
{
  ExG4HbookNtupleDescription* ntupleDecription
    = GetNtupleInFunction(ntupleId, "GetNtupleDColumn");
  if ( ! ntupleDecription ) return 0;

  std::map<G4int, tools::hbook::wntuple::column<double>* >& ntupleDColumnMap
    = ntupleDecription->fNtupleDColumnMap;
  std::map<G4int, tools::hbook::wntuple::column<double>* >::const_iterator it
    = ntupleDColumnMap.find(columnId);
  if ( it == ntupleDColumnMap.end() ) {
    G4ExceptionDescription description;
    description << "      "  << "ntupleId " << ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::GetNtupleDColumn()",
                "Analysis_W011", JustWarning, description);
    return 0;
  }
  
  return it->second;
}  
 
//_____________________________________________________________________________
void ExG4HbookNtupleManager::Reset()
{
// Reset ntuple  

  std::vector<ExG4HbookNtupleDescription*>::iterator it3;  
  for (it3 = fNtupleDescriptionVector.begin(); it3 != fNtupleDescriptionVector.end(); it3++ ) {
    delete (*it3)->fNtuple;
    (*it3)->fNtuple = 0;
  }  
  fNtupleVector.clear(); 
}  
 
//
// protected methods
//

//_____________________________________________________________________________
ExG4HbookNtupleDescription* ExG4HbookNtupleManager::GetNtupleInFunction(
                                      G4int id, 
                                      G4String functionName, G4bool warn,
                                      G4bool /*onlyIfActive*/) const
{                                      
  G4int index = id - fFirstId;
  if ( index < 0 || index >= G4int(fNtupleDescriptionVector.size()) ) {
    if ( warn) {
      G4String inFunction = "G4HbookAnalysisManager::";
      inFunction += functionName;
      G4ExceptionDescription description;
      description << "      " << "ntuple " << id << " does not exist.";
      G4Exception(inFunction, "Analysis_W011", JustWarning, description);
    }
    return 0;         
  }
  
  return fNtupleDescriptionVector[index];
}
  
// 
// public methods
//

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtuple(const G4String& name, 
                                           const G4String& title)
{
  // Create an "ntuple" directory both in memory and in the file
  if ( fFileManager->IsFile() ) 
    fFileManager->CreateNtupleDirectory();

#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("create", "ntuple", name);
#endif

  // Create ntuple description
  G4int index = fNtupleDescriptionVector.size();
  ExG4HbookNtupleDescription* ntupleDescription
    = new ExG4HbookNtupleDescription();
  fNtupleDescriptionVector.push_back(ntupleDescription);  

  // Create ntuple booking
  ntupleDescription->fNtupleBooking 
    = new tools::ntuple_booking(name, title);
           // ntuple booking object is deleted in destructor

  // Set fNtupleHbookIdOffset if needed
  SetNtupleHbookIdOffset();
  
  // Create ntuple if the file is open
  // We should be under //PAWC/LUN1/ntuple
  if ( fFileManager->IsFile() ) {
    G4int hbookIndex = fNtupleHbookIdOffset + index + fFirstId;
    ntupleDescription->fNtuple 
      = new tools::hbook::wntuple(hbookIndex, name);
           // ntuple object is deleted when closing a file
    fNtupleVector.push_back(ntupleDescription->fNtuple);       
  }  

#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << index + fFirstId; 
    fState.GetVerboseL2()->Message("create", "ntuple", description);
  }  
#endif

  return index + fFirstId;
}                                         

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleIColumn(const G4String& name,
                                                  std::vector<int>* vector)
{
  G4int ntupleId = fNtupleDescriptionVector.size() + fFirstId - 1;
  return CreateNtupleIColumn(ntupleId, name, vector);
}  

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleFColumn(const G4String& name,
                                                  std::vector<float>* vector)
{
  G4int ntupleId = fNtupleDescriptionVector.size() + fFirstId - 1;
  return CreateNtupleFColumn(ntupleId, name, vector);
}  

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleDColumn(const G4String& name,
                                                  std::vector<double>* vector)
{
  G4int ntupleId = fNtupleDescriptionVector.size() + fFirstId - 1;
  return CreateNtupleDColumn(ntupleId, name, vector);
}  

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleSColumn(const G4String& name)
{
  G4int ntupleId = fNtupleDescriptionVector.size() + fFirstId - 1;
  return CreateNtupleSColumn(ntupleId, name);
}  

//_____________________________________________________________________________
void ExG4HbookNtupleManager::FinishNtuple()
{ 
  G4int ntupleId = fNtupleDescriptionVector.size() + fFirstId - 1;
  FinishNtuple(ntupleId);
}
  
//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleIColumn(G4int ntupleId, 
                                                    const G4String& name,
                                                    std::vector<int>* vector)
{
  // Vector columns are not supported in HBOOK
  if ( vector ) {
    G4ExceptionDescription description;
    description << "      " 
      << "Vector columns are not supported in HBOOK."; 
    G4Exception("(ExG4HbookNtupleManager::CreateNtupleIColumn)",
                "Analysis_W002", FatalException, description);
    return kInvalidId;            
  }
                
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL4()->Message("create", "ntuple I column", description);
  }  
#endif

  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "CreateNtupleIColumn");
  tools::ntuple_booking* ntupleBooking
    = ntupleDescription->fNtupleBooking;  

  if ( ! ntupleBooking ) {
    G4ExceptionDescription description;
    description << "      " 
                << "Ntuple " << ntupleId << " has to be created first. ";
    G4Exception("G4HbookAnalysisManager::CreateNtupleIColumn()",
                "Analysis_W002", JustWarning, description);
    return kInvalidId;       
  }

  // Save column info in booking
  G4int index = ntupleBooking->columns().size();
  ntupleBooking->add_column<int>(name);  
 
  // Create column if ntuple already exists
  if ( ntupleDescription->fNtuple ) {
    tools::hbook::wntuple::column<int>* column 
      = ntupleDescription->fNtuple->create_column<int>(name);  
    ntupleDescription->fNtupleIColumnMap[index] = column;
  }  
    
  fLockFirstNtupleColumnId = true;

#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL2()->Message("create", "ntuple I column", description);
  }  
#endif

  return index + fFirstNtupleColumnId;       
}                                         

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleFColumn(G4int ntupleId, 
                                                  const G4String& name,
                                                  std::vector<float>* vector)
{ 
  // Vector columns are not supported in HBOOK
  if ( vector ) {
    G4ExceptionDescription description;
    description << "      " 
      << "Vector columns are not supported in HBOOK."; 
    G4Exception("(ExG4HbookNtupleManager::CreateNtupleFColumn)",
                "Analysis_W002", FatalException, description);
    return kInvalidId;            
  }
                
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL4()->Message("create", "ntuple F column", description);
  } 
#endif

  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "CreateNtupleFColumn");
  tools::ntuple_booking* ntupleBooking
    = ntupleDescription->fNtupleBooking;  

  if ( ! ntupleBooking ) {
    G4ExceptionDescription description;
    description << "      " 
                << "Ntuple " << ntupleId << " has to be created first. ";
    G4Exception("G4HbookAnalysisManager::CreateNtupleFColumn()",
                "Analysis_W002", JustWarning, description);
    return kInvalidId;       
  }

  // Save column info in booking
  G4int index = ntupleBooking->columns().size();
  ntupleBooking->add_column<float>(name);  
 
  // Create column if ntuple already exists
  if ( ntupleDescription->fNtuple ) {
    tools::hbook::wntuple::column<float>* column 
      = ntupleDescription->fNtuple->create_column<float>(name);  
    ntupleDescription->fNtupleFColumnMap[index] = column;
  }
    
  fLockFirstNtupleColumnId = true;

#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL2()->Message("create", "ntuple F column", description);
  }  
#endif

  return index + fFirstNtupleColumnId;       
}                                         


//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleDColumn(G4int ntupleId, 
                                                  const G4String& name,
                                                  std::vector<double>* vector)
{
  // Vector columns are not supported in HBOOK
  if ( vector ) {
    G4ExceptionDescription description;
    description << "      " 
      << "Vector columns are not supported in HBOOK."; 
    G4Exception("(ExG4HbookNtupleManager::CreateNtupleDColumn)",
                "Analysis_W002", FatalException, description);
    return kInvalidId;            
  }
                
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL4()->Message("create", "ntuple D column", description);
  }  
#endif

  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "CreateNtupleDColumn");
  tools::ntuple_booking* ntupleBooking
    = ntupleDescription->fNtupleBooking;  

  if ( ! ntupleBooking ) {
    G4ExceptionDescription description;
    description << "      " 
                << "Ntuple " << ntupleId << " has to be created first. ";
    G4Exception("G4HbookAnalysisManager::CreateNtupleDColumn()",
                "Analysis_W002", JustWarning, description);
    return kInvalidId;       
  }

  // Save column info in booking
  G4int index = ntupleBooking->columns().size();
  ntupleBooking->add_column<double>(name);  
 
  // Create column if ntuple already exists
  if ( ntupleDescription->fNtuple ) {
    tools::hbook::wntuple::column<double>* column 
      = ntupleDescription->fNtuple->create_column<double>(name);  
    ntupleDescription->fNtupleDColumnMap[index] = column;
  }
    
  fLockFirstNtupleColumnId = true;

#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) {
    G4ExceptionDescription description;
    description << name << " ntupleId " << ntupleId; 
    fState.GetVerboseL2()->Message("create", "ntuple D column", description);
  }  
#endif

  return index + fFirstNtupleColumnId;       
}                                         

//_____________________________________________________________________________
G4int ExG4HbookNtupleManager::CreateNtupleSColumn(G4int /*ntupleId*/, 
                                                  const G4String& /*name*/)
{
  G4ExceptionDescription description;
  description << "      " 
    << "Columns of string type are not supported in HBOOK."; 
  G4Exception("(ExG4HbookNtupleManager::CreateNtupleDColumn)",
              "Analysis_W002", FatalException, description);
  return kInvalidId;
}

//_____________________________________________________________________________
void ExG4HbookNtupleManager::FinishNtuple(G4int ntupleId)
{ 
  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "CreateNtupleDColumn");
  tools::hbook::wntuple* ntuple = ntupleDescription->fNtuple;  

  if ( ! ntuple ) return;

#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()
      ->Message("finish", "ntuple", ntupleDescription->fNtupleBooking->name());
#endif

  // Return to //PAWC/LUN1 :
  tools::hbook::CHCDIR("//PAWC/LUN1"," ");

  //fNtuple->add_row_beg();
#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) 
    fState.GetVerboseL2()
      ->Message("finish", "ntuple", ntupleDescription->fNtupleBooking->name());
#endif
}
  
//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleIColumn(G4int columnId, G4int value)
{
  return FillNtupleIColumn(fFirstId, columnId, value);
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleFColumn(G4int columnId, G4float value)
{
  return FillNtupleFColumn(fFirstId, columnId, value);
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleDColumn(G4int columnId, G4double value)
{
  return FillNtupleDColumn(fFirstId, columnId, value);
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleSColumn(G4int columnId, 
                                                 const G4String& value)
{
  return FillNtupleSColumn(fFirstId, columnId, value);
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::AddNtupleRow()
{ 
  return AddNtupleRow(fFirstId);
}  

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleIColumn(
                                            G4int ntupleId, G4int columnId, 
                                            G4int value)
{
  tools::hbook::wntuple::column<int>* column 
    = GetNtupleIColumn(ntupleId, columnId);
  if ( ! column ) {
    G4ExceptionDescription description;
    description << "      " << "ntupleId " <<  ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::FillNtupleIColumn()",
                "Analysis_W011", JustWarning, description);
    return false;
  }  
  
  column->fill(value);
 #ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " ntupleId " << ntupleId  
                << " columnId " << columnId << " value " << value;
    fState.GetVerboseL4()->Message("fill", "ntuple I column", description);
  }  
#endif
 return true;       
}                                         
//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleFColumn(
                                            G4int ntupleId, G4int columnId, 
                                            G4float value)
{
  tools::hbook::wntuple::column<float>* column 
    = GetNtupleFColumn(ntupleId, columnId);
  if ( ! column ) {
    G4ExceptionDescription description;
    description << "      " << "ntupleId " <<  ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::FillNtupleFColumn()",
                "Analysis_W011", JustWarning, description);
    return false;
  }  
  
  column->fill(value);
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " ntupleId " << ntupleId  
                << " columnId " << columnId << " value " << value;
    fState.GetVerboseL4()->Message("fill", "ntuple F column", description);
  }  
#endif
  return true;       
}                                         
//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleDColumn(
                                            G4int ntupleId, G4int columnId, 
                                            G4double value)
{
  tools::hbook::wntuple::column<double>* column 
    = GetNtupleDColumn(ntupleId, columnId);
  if ( ! column ) {
    G4ExceptionDescription description;
    description << "      " << "ntupleId " <<  ntupleId
                << "column " << columnId << " does not exist.";
    G4Exception("G4HbookAnalysisManager::FillNtupleDColumn()",
                "Analysis_W011", JustWarning, description);
    return false;
  }  
  
  column->fill(value);
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " ntupleId " << ntupleId  
                << " columnId " << columnId << " value " << value;
    fState.GetVerboseL4()->Message("fill", "ntuple D column", description);
  }  
#endif
  return true;       
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::FillNtupleSColumn(
                                   G4int /*ntupleId*/, G4int /*columnId*/, 
                                   const G4String& /*value*/)
{
  G4ExceptionDescription description;
  description << "      " 
    << "Columns of string type are not supported in HBOOK."; 
  G4Exception("(ExG4HbookNtupleManager::FillNtupleSColumn)",
              "Analysis_W011", JustWarning, description);
  return kInvalidId;
}  

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::AddNtupleRow(G4int ntupleId)
{ 
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " ntupleId " << ntupleId;  
    fState.GetVerboseL4()->Message("add", "ntuple row", description);
  }  
#endif

  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "AddNtupleRow");

  if ( ! ntupleDescription || ! ntupleDescription->fNtuple ) {
    G4ExceptionDescription description;
    description << "      " << " ntupleId " << ntupleId 
                << " does not exist. ";
    G4Exception("G4HbookAnalysisManager::AddNtupleRow()",
                "Analysis_W011", JustWarning, description);
    return false;
  }  
  
  ntupleDescription->fNtuple->add_row();
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " ntupleId " << ntupleId;  
    fState.GetVerboseL4()->Message("add", "ntuple row", description, true);
  }  
#endif
  return true;
}
 
//_____________________________________________________________________________
tools::hbook::wntuple* ExG4HbookNtupleManager::GetNtuple() const
{
  return GetNtuple(fFirstId);
}

//_____________________________________________________________________________
tools::hbook::wntuple* ExG4HbookNtupleManager::GetNtuple(G4int ntupleId) const
{
  ExG4HbookNtupleDescription* ntupleDescription
    = GetNtupleInFunction(ntupleId, "GetNtuple");
    
  return ntupleDescription->fNtuple;  
}

//_____________________________________________________________________________
G4bool ExG4HbookNtupleManager::SetNtupleHbookIdOffset(G4int offset) 
{
  if ( fNtupleDescriptionVector.size() ) {
    G4ExceptionDescription description;
    description 
      << "Cannot set NtupleHbookIdOffset as some ntuples already exist.";
    G4Exception("G4HbookAnalysisManager::SetNtupleHbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }
  
  if ( fFirstId + offset < 1 ) {
    G4ExceptionDescription description;
    description << "The first ntuple HBOOK id must be >= 1.";
    G4Exception("G4HbookAnalysisManager::SetNtupleHbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }
  
  fNtupleHbookIdOffset = offset;
  return true;
}  

#endif
