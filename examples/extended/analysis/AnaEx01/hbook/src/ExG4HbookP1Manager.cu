
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$
//
/// \file hbook/src/ExG4HbookP1Manager.cc
/// \brief Implementation of the ExG4HbookP1Manager class

// Author: Ivana Hrivnacova, 03/11/2014  (ivana@ipno.in2p3.fr)

#ifdef G4_USE_HBOOK

#include "ExG4HbookP1Manager.hh"
#include "ExG4HbookFileManager.hh"
#include "G4HnManager.hh"
#include "G4AnalysisManagerState.hh"
#include "G4AnalysisUtilities.hh"
#include "G4BinScheme.hh"

#include <fstream>

using namespace G4Analysis;

//_____________________________________________________________________________
ExG4HbookP1Manager::ExG4HbookP1Manager(const G4AnalysisManagerState& state)
 : G4VP1Manager(state),
   fBaseToolsManager("P1"),
   fFileManager(0),
   fP1HbookIdOffset(-1),
   fP1Vector(),
   fP1BookingVector(),
   fP1NameIdMap()
{
}

//_____________________________________________________________________________
ExG4HbookP1Manager::~ExG4HbookP1Manager()
{  
  // Delete p1
  Reset();

  // Delete p1 booking 
  std::vector<p1_booking*>::iterator it;
  for ( it = fP1BookingVector.begin(); it != fP1BookingVector.end(); it++ ) {
    delete *it;
  }  
}

//
// utility functions
//

namespace {

//_____________________________________________________________________________
void ConvertToFloat(const std::vector<G4double>& doubleVector,
                    std::vector<float>& floatVector)
{
  for (G4int i=0; i<G4int(doubleVector.size()); ++i) 
    floatVector.push_back((float)doubleVector[i]);
}                        

//_____________________________________________________________________________
void UpdateP1Information(G4HnInformation* hnInformation,
                          const G4String& xunitName, 
                          const G4String& yunitName, 
                          const G4String& xfcnName,
                          const G4String& yfcnName,
                          G4BinScheme xbinScheme)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);
  
  G4HnDimensionInformation* xInformation 
    = hnInformation->GetHnDimensionInformation(G4HnInformation::kX);
  xInformation->fUnitName = xunitName;
  xInformation->fFcnName = xfcnName;
  xInformation->fUnit = xunit;
  xInformation->fFcn = xfcn;
  xInformation->fBinScheme = xbinScheme;

  G4HnDimensionInformation* yInformation 
    = hnInformation->GetHnDimensionInformation(G4HnInformation::kY);
  yInformation->fUnitName = yunitName;
  yInformation->fFcnName = yfcnName;
  yInformation->fUnit = yunit;
  yInformation->fFcn = yfcn;
  yInformation->fBinScheme = kLinearBinScheme;
}  

//_____________________________________________________________________________
p1_booking* CreateP1Booking(const G4String& title,
                   G4int nbins, G4double xmin, G4double xmax,
                   G4double ymin, G4double ymax,
                   const G4String& xunitName,
                   const G4String& yunitName,
                   const G4String& xfcnName,
                   const G4String& yfcnName,
                   G4BinScheme xbinScheme)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);

  p1_booking* p1Booking = 0; 
  if ( xbinScheme != kLogBinScheme ) {
    if ( xbinScheme == kUserBinScheme ) {
      // This should never happen, but let's make sure about it
      // by issuing a warning
      G4ExceptionDescription description;
      description 
        << "    User binning scheme setting was ignored." << G4endl
        << "    Linear binning will be applied with given (nbins, xmin, xmax) values";
      G4Exception("ExG4HbookP1Manager::CreateP1",
                "Analysis_W013", JustWarning, description);
    }              
    p1Booking = new p1_booking(nbins, xfcn(xmin/xunit), xfcn(xmax/xunit), 
                               yfcn(ymin/yunit), yfcn(ymax/yunit)); 
                    // p1_booking object is deleted in destructor
  }
  else {
    // Compute edges
    std::vector<G4double> edges;
    ComputeEdges(nbins, xmin, xmax, xunit, xfcn, xbinScheme, edges);
    p1Booking = new p1_booking(edges, yfcn(ymin/yunit), yfcn(ymax/yunit)); 
                    // p1_booking object is deleted in destructor
  }

  p1Booking->fTitle = title;
  UpdateTitle(p1Booking->fTitle, xunitName, xfcnName);  

  return p1Booking;
}

//_____________________________________________________________________________
p1_booking* CreateP1Booking(const G4String& title,
                   const std::vector<G4double>& edges,
                   G4double ymin, G4double ymax,
                   const G4String& xunitName,
                   const G4String& yunitName,
                   const G4String& xfcnName,
                   const G4String& yfcnName)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);

  // Apply function
  std::vector<G4double> newEdges;
  ComputeEdges(edges, xunit, xfcn, newEdges);
  
  p1_booking* p1Booking = new p1_booking(newEdges, yfcn(ymin/yunit), yfcn(ymax/yunit)); 
                    // p1_booking object is deleted in destructor

  p1Booking->fTitle = title;
  UpdateTitle(p1Booking->fTitle, xunitName, xfcnName);  
  
  return p1Booking;
}

//_____________________________________________________________________________
void UpdateP1Booking(p1_booking* p1Booking,
                     G4int nbins, G4double xmin, G4double xmax,  
                     G4double ymin, G4double ymax,  
                     const G4String& xunitName,
                     const G4String& yunitName,
                     const G4String& xfcnName,
                     const G4String& yfcnName,
                     const G4String& xbinSchemeName)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);
  G4BinScheme xbinScheme = GetBinScheme(xbinSchemeName);

  if ( xbinScheme != kLogBinScheme ) {
    if ( xbinScheme == kUserBinScheme ) {
      // This should never happen, but let's make sure about it
      // by issuing a warning
      G4ExceptionDescription description;
      description 
        << "    User binning scheme setting was ignored." << G4endl
        << "    Linear binning will be applied with given (nbins, xmin, xmax) values";
      G4Exception("ExG4HbookP1Manager::SetP1",
                "Analysis_W013", JustWarning, description);
    }              
    p1Booking->fNbins = nbins;
    p1Booking->fXmin = xfcn(xmin/xunit);
    p1Booking->fXmax = xfcn(xmax/xunit);
    p1Booking->fYmin = yfcn(ymin/yunit);
    p1Booking->fYmax = yfcn(ymax/yunit);
  }
  else {
    // Compute edges
    ComputeEdges(nbins, xmin, xmax, xunit, xfcn, xbinScheme, p1Booking->fEdges);
    p1Booking->fYmin = yfcn(ymin/yunit);
    p1Booking->fYmax = yfcn(ymax/yunit);
  }

  UpdateTitle(p1Booking->fTitle, xunitName, xfcnName);  
}     

//_____________________________________________________________________________
void UpdateP1Booking(p1_booking* p1Booking,
                     const std::vector<G4double>& edges,
                     G4double ymin, G4double ymax,  
                     const G4String& xunitName,
                     const G4String& yunitName,
                     const G4String& xfcnName,
                     const G4String& yfcnName)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);

  // Apply function
  ComputeEdges(edges, xunit, xfcn, p1Booking->fEdges);
  p1Booking->fYmin = yfcn(ymin/yunit);
  p1Booking->fYmax = yfcn(ymax/yunit);

  UpdateTitle(p1Booking->fTitle, xunitName, xfcnName);  
}     

//_____________________________________________________________________________
void ConfigureHbookP1(tools::hbook::p1* p1,
                      G4int nbins, G4double xmin, G4double xmax,  
                      G4double ymin, G4double ymax,  
                      const G4String& xunitName,
                      const G4String& yunitName,
                      const G4String& xfcnName,
                      const G4String& yfcnName,
                      G4BinScheme xbinScheme)
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);

  if ( xbinScheme != kLogBinScheme ) {
    if ( xbinScheme == kUserBinScheme ) {
      // This should never happen, but let's make sure about it
      // by issuing a warning
      G4ExceptionDescription description;
      description 
        << "    User binning scheme setting was ignored." << G4endl
        << "    Linear binning will be applied with given (nbins, xmin, xmax) values";
      G4Exception("ExG4HbookP1Manager::SetP1",
                "Analysis_W013", JustWarning, description);
    }              
    // not available !!              
    p1->configure(nbins, xfcn(xmin/xunit), xfcn(xmax/xunit),
                  yfcn(ymin/yunit), yfcn(ymax/yunit));
  }
  else {
    // Compute bins
    std::vector<G4double> edges;
    ComputeEdges(nbins, xmin, xmax, xunit, xfcn, xbinScheme, edges);
    // Convert to float
    std::vector<float> fedges;
    ConvertToFloat(edges, fedges); 

    // not available !!              
    //p1->configure(edges,  yfcn(ymin/yunit), yfcn(ymax/yunit));
  }
}     

//_____________________________________________________________________________
void ConfigureHbookP1(tools::hbook::p1* /*p1*/,
                      const std::vector<G4double>& edges,
                       G4double /*ymin*/, G4double /*ymax*/,  
                      const G4String& xunitName,
                      const G4String& /*yunitName*/,
                      const G4String& xfcnName,
                      const G4String& /*yfcnName*/)
{
  // Apply function to edges
  G4double xunit = GetUnitValue(xunitName);
  //G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  //G4Fcn yfcn = GetFunction(yfcnName);
  std::vector<G4double> newEdges;
  ComputeEdges(edges, xunit, xfcn, newEdges);
  
  // Convert to float
  std::vector<float> newFEdges;
  ConvertToFloat(newEdges, newFEdges); 

  // not available !!              
  //p1->configure(newFEdges);
}

}

// 
// private methods
//

//_____________________________________________________________________________
void ExG4HbookP1Manager::SetP1HbookIdOffset()
{
// Set  fP1HbookIdOffset if needed

  if ( fP1HbookIdOffset == -1 ) {
    if ( fFirstId > 0 ) 
      fP1HbookIdOffset = 0;
    else
      fP1HbookIdOffset = 1;
        
    if ( fP1HbookIdOffset > 0 ) {
      G4ExceptionDescription description;
      description << "P1 will be defined in HBOOK with ID = G4_firstProfileId + 1";
      G4Exception("ExG4HbookP1Manager::SetP1HbookIdOffset()",
                  "Analysis_W013", JustWarning, description);
    }              
  }
}  

//_____________________________________________________________________________
void ExG4HbookP1Manager::AddP1Information(const G4String& name,  
                            const G4String& xunitName, 
                            const G4String& yunitName, 
                            const G4String& xfcnName,
                            const G4String& yfcnName,
                            G4BinScheme xbinScheme) const
{
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);
  fHnManager
    ->AddH2Information(name, xunitName, yunitName, xfcnName, yfcnName, 
                       xunit, yunit, xfcn, yfcn, 
                       xbinScheme, xbinScheme);
}  

//_____________________________________________________________________________
G4int ExG4HbookP1Manager::CreateP1FromBooking(p1_booking* p1Booking, 
                                              G4bool chDir)
{
// Create p1 from p1_booking.

  if ( chDir ) {
    // Go to profiles directory if defined
    if ( fFileManager->GetProfileDirectoryName() != "" ) {
      G4String profilePath = "//PAWC/LUN1/";
      profilePath.append(fFileManager->GetProfileDirectoryName().data());
      tools::hbook::CHCDIR(profilePath.data()," ");
    }
  }    

  G4int index = fP1Vector.size();
  G4int id = index + fFirstId;    
  G4HnInformation* 
    info = fHnManager->GetHnInformation(id, "CreateP1FromBooking");
  // Hbook index
  G4int hbookIndex = fP1HbookIdOffset + index + fFirstId;
  
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("create from booking", "p1", info->GetName());
#endif

  // Create p1
  tools::hbook::p1* p1 = 0; 
  if ( ! p1Booking->fEdges.size() ) {
    p1 = new tools::hbook::p1(
               hbookIndex, p1Booking->fTitle, 
               p1Booking->fNbins, p1Booking->fXmin, p1Booking->fXmax, 
               p1Booking->fYmin, p1Booking->fYmax);
  }
  else {               
    // Convert to float
    std::vector<float> newEdges;
    ConvertToFloat(p1Booking->fEdges, newEdges); 

    // not supported
    //p1 = new tools::hbook::p1(hbookIndex, p1Booking->fTitle, newEdges,
    //                          p1Booking->fYmin, p1Booking->fYmax);
  }
                           
  fP1Vector.push_back(p1);
  
  if ( chDir ) {
    if ( fFileManager->GetProfileDirectoryName() != "" ) {
      // Return to //PAWC/LUN1 :
      tools::hbook::CHCDIR("//PAWC/LUN1"," ");
    }  
  }
  
#ifdef G4VERBOSE
  if ( fState.GetVerboseL3() ) { 
    G4ExceptionDescription description;
    description << " name : " << info->GetName() << " hbook index : " << hbookIndex; 
    fState.GetVerboseL3()->Message("create from booking", "p1", description);
  }  
#endif
  
  return id;
}  

//_____________________________________________________________________________
G4int ExG4HbookP1Manager::RegisterP1Booking(const G4String& name, 
                                            p1_booking* p1Booking)
{
  // Register p1
  G4int index = fP1BookingVector.size();  
  fP1BookingVector.push_back(p1Booking);
  fP1NameIdMap[name] = index + fFirstId;

  // Lock id
  fLockFirstId = true;

  return index + fFirstId;
}  

//_____________________________________________________________________________
void ExG4HbookP1Manager::BeginCreateP1(const G4String& name)
{
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("create", "P1", name);
#endif

  // Set  fP1HbookIdOffset if needed
  SetP1HbookIdOffset();
}

//_____________________________________________________________________________
G4int ExG4HbookP1Manager::FinishCreateP1(
                               const G4String& name, p1_booking* p1Booking,
                               const G4String& xunitName, 
                               const G4String& yunitName, 
                               const G4String& xfcnName,
                               const G4String& yfcnName,
                               G4BinScheme xbinScheme)
{
  // Register p1 booking
  G4int id = RegisterP1Booking(name, p1Booking);
  
  // Save P1 information
  AddP1Information(name, xunitName, yunitName, xfcnName, yfcnName, xbinScheme);

  // Create p1 if the file is open
  if ( fFileManager->IsFile() ) {
    CreateP1FromBooking(p1Booking);
  }

#ifdef G4VERBOSE
  if ( fState.GetVerboseL2() ) { 
    G4int hbookIndex = fP1HbookIdOffset + id;
    G4ExceptionDescription description;
    description << " name : " << name << " hbook index : " << hbookIndex; 
    fState.GetVerboseL2()->Message("create", "P1", description);
  }  
#endif

  return id;
}                                         

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::BeginSetP1(
                               G4int id,
                               p1_booking* p1Booking,
                               G4HnInformation* info)
{                                
  p1Booking = GetP1Booking(id, false);
  if ( ! p1Booking ) {
    G4ExceptionDescription description;
    description << "      " << "profile " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::SetP1()",
                "Analysis_W011", JustWarning, description);
    return false;
  }

  info = fHnManager->GetHnInformation(id,"SetP1");
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("configure", "P1", info->GetName());
#endif

  return true;
}
  
//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::FinishSetP1(
                               G4int id,
                               G4HnInformation* info,
                               const G4String& xunitName, 
                               const G4String& yunitName, 
                               const G4String& xfcnName,
                               const G4String& yfcnName,
                               G4BinScheme xbinScheme)
{                                
  // Update information
  UpdateP1Information(info, xunitName, yunitName, xfcnName, yfcnName, xbinScheme);

  // Set activation
  fHnManager->SetActivation(id, true); 
  
  return true;
}
  
                                        
//_____________________________________________________________________________
void ExG4HbookP1Manager::CreateP1sFromBooking()
{
// Create all p1 from p1_booking.

  // Do nothing if any p1 profile already exists
  // or no p1 profiles are booked
  if ( fP1Vector.size() || ( fP1BookingVector.size() == 0 ) ) return;       

  // Go to profiles directory if defined
  if ( fFileManager->GetProfileDirectoryName() != "" ) {
    G4String profilePath = "//PAWC/LUN1/";
    profilePath.append(fFileManager->GetProfileDirectoryName().data());
    tools::hbook::CHCDIR(profilePath.data()," ");
  }  

  // Create profiles
  std::vector<p1_booking*>::const_iterator it;
  for ( it = fP1BookingVector.begin(); it != fP1BookingVector.end(); ++it) {
    CreateP1FromBooking(*it, false);
  }  
  
  // Return backi from profiles directory if defined
  if ( fFileManager->GetProfileDirectoryName() != "" ) {
    // Return to //PAWC/LUN1 :
    tools::hbook::CHCDIR("//PAWC/LUN1"," ");
  }  
}   

//_____________________________________________________________________________
void ExG4HbookP1Manager::Reset()
{
// Reset profiles and ntuple  

  // Delete profiles
  std::vector<tools::hbook::p1*>::iterator it;
  for (it = fP1Vector.begin(); it != fP1Vector.end(); it++ ) {
    delete *it;
  }  

  // Clear vectors
  fP1Vector.clear();
}  
 
//_____________________________________________________________________________
p1_booking*  ExG4HbookP1Manager::GetP1Booking(G4int id, G4bool warn) const 
{
  G4int index = id - fFirstId;
  if ( index < 0 || index >= G4int(fP1BookingVector.size()) ) {
    if ( warn) {
      G4ExceptionDescription description;
      description << "      " << "profile " << id << " does not exist.";
      G4Exception("G4HbookAnalysisManager::GetP1Booking()",
                  "Analysis_W011", JustWarning, description);
    }
    return 0;         
  }

  return fP1BookingVector[index];
}

//
// protected methods
//
/*
//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::WriteOnAscii(std::ofstream& output)
{
// Write selected objects on ASCII file
// (Only P1 implemented by now)
// According to the implementation by Michel Maire, originally in
// extended examples.

  // p1 profiles
  for ( G4int i=0; i<G4int(fP1Vector.size()); ++i ) {
    G4int id = i + fFirstId;
    G4HnInformation* info 
      = fHnManager->GetHnInformation(id, "WriteOnAscii"); 
    // skip writing if activation is enabled and P1 is inactivated
    if ( ! info->fAscii ) continue; 
    tools::hbook::p1* p1 = fP1Vector[i];

#ifdef G4VERBOSE
    if ( fState.GetVerboseL3() ) 
      fState.GetVerboseL3()->Message("write on ascii", "p1", info->GetName());
#endif
  
    output << "\n  1D profile " << id << ": " << p1->title() 
           << "\n \n \t     X \t\t     Y" << G4endl;
    
    for (G4int j=0; j< G4int(p1->axis().bins()); ++j) {
       output << "  " << j << "\t" 
              << p1->axis().bin_center(j) << "\t"
              << p1->bin_height(j) << G4endl;
    } 
  }
  
  return true;
}  
*/
//_____________________________________________________________________________
tools::hbook::p1*  ExG4HbookP1Manager::GetP1InFunction(G4int id, 
                                      G4String functionName, G4bool warn,
                                      G4bool onlyIfActive) const
{
  G4int index = id - fFirstId;
  if ( index < 0 || index >= G4int(fP1Vector.size()) ) {
    if ( warn) {
      G4String inFunction = "ExG4HbookP1Manager::";
      inFunction += functionName;
      G4ExceptionDescription description;
      description << "      " << "profile " << id << " does not exist.";
      G4Exception(inFunction, "Analysis_W011", JustWarning, description);
    }
    return 0;         
  }
  
  // Do not return profile if inactive 
  if ( fState.GetIsActivation() && onlyIfActive && ( ! fHnManager->GetActivation(id) ) ) {
    return 0; 
  }  
  
  return fP1Vector[index];
}  
                                      
// 
// public methods
//

//_____________________________________________________________________________
G4int ExG4HbookP1Manager::CreateP1(
                               const G4String& name, const G4String& title,
                               G4int nbins, G4double xmin, G4double xmax,
                               G4double ymin, G4double ymax,
                               const G4String& xunitName, 
                               const G4String& yunitName, 
                               const G4String& xfcnName,
                               const G4String& yfcnName,
                               const G4String& xbinSchemeName)
{
  BeginCreateP1(name);

  G4BinScheme xbinScheme = GetBinScheme(xbinSchemeName);

  // Create p1 booking
  p1_booking* p1Booking 
    = CreateP1Booking(title, nbins, xmin, xmax, ymin, ymax, 
                      xunitName, yunitName, xfcnName, yfcnName, xbinScheme);
    
  return FinishCreateP1(name, p1Booking, 
                        xunitName, yunitName, xfcnName, yfcnName, xbinScheme); 
}                                         

//_____________________________________________________________________________
G4int ExG4HbookP1Manager::CreateP1(
                               const G4String& name, const G4String& title,
                               const std::vector<G4double>& edges,
                               G4double ymin, G4double ymax,
                               const G4String& xunitName, 
                               const G4String& yunitName, 
                               const G4String& xfcnName,
                               const G4String& yfcnName)
{                       
  BeginCreateP1(name);

  // Create p1 booking
  p1_booking* p1Booking 
    = CreateP1Booking(title, edges, ymin, ymax,
                      xunitName, yunitName, xfcnName, yfcnName);
    
  return FinishCreateP1(name, p1Booking,
                        xunitName, yunitName, xfcnName, yfcnName, kUserBinScheme); 
}                                         


//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1(G4int id,
                               G4int nbins, G4double xmin, G4double xmax,
                               G4double ymin, G4double ymax,
                               const G4String& xunitName, const G4String& yunitName,
                               const G4String& xfcnName, const G4String& yfcnName,
                               const G4String& xbinSchemeName)
{                                
  p1_booking* p1Booking = 0;
  G4HnInformation* info = 0;

  if ( ! BeginSetP1(id, p1Booking, info) ) return false; 

  G4BinScheme xbinScheme = GetBinScheme(xbinSchemeName);

  // Update P1 booking
  UpdateP1Booking(p1Booking, nbins, xmin, xmax, ymin, ymax, 
                  xunitName, yunitName, xfcnName, yfcnName, xbinScheme);

  // Re-configure profile if it was already defined
  if ( fP1Vector.size() ) {
    tools::hbook::p1* p1 = GetP1(id);
    ConfigureHbookP1(p1, nbins, xmin, xmax, ymin, ymax, 
                     xunitName, yunitName, xfcnName, yfcnName, xbinScheme);
  }  
  
  return FinishSetP1(id, info, 
                     xunitName, yunitName, xfcnName, yfcnName, xbinScheme);
}
  
//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1(G4int id,
                               const std::vector<G4double>& edges,
                               G4double ymin, G4double ymax,
                               const G4String& xunitName, const G4String& yunitName,
                               const G4String& xfcnName, const G4String& yfcnName)
{                                
  p1_booking* p1Booking = 0;
  G4HnInformation* info = 0;

  if ( ! BeginSetP1(id, p1Booking, info) ) return false; 

  // Update P1 booking
  UpdateP1Booking(p1Booking, edges, ymin, ymax, 
                  xunitName, yunitName, xfcnName, yfcnName);

  // Re-configure profile if it was already defined
  if ( fP1Vector.size() ) {
    tools::hbook::p1* p1 = GetP1(id);
    ConfigureHbookP1(p1, edges, ymin, ymax,
                     xunitName, yunitName, xfcnName, yfcnName);
  }  
  
  return 
    FinishSetP1(id, info, 
                xunitName, yunitName, xfcnName, yfcnName, kUserBinScheme);
}
  
//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::ScaleP1(G4int id, G4double factor)
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "ScaleP1", false, false);
  if ( ! p1 ) return false;

  return p1->scale(factor);
}  

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::FillP1(G4int id, G4double xvalue, G4double yvalue,
                                  G4double weight)
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "FillP1", true, false);
  if ( ! p1 ) return false;

  if ( fState.GetIsActivation() && ( ! fHnManager->GetActivation(id) ) ) {
    //G4cout << "Skipping FillP1 for " << id << G4endl; 
    return false; 
  }  

  G4HnDimensionInformation* xInfo 
    = fHnManager->GetHnDimensionInformation(id, G4HnInformation::kX, "FillP1");
  G4HnDimensionInformation* yInfo 
    = fHnManager->GetHnDimensionInformation(id, G4HnInformation::kY, "FillP1");

  p1->fill(xInfo->fFcn(xvalue/xInfo->fUnit), 
           yInfo->fFcn(yvalue/yInfo->fUnit), weight);
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " id " << id 
                << " xvalue " << xvalue 
                << " xfcn(xvalue/xunit) " <<  xInfo->fFcn(xvalue/xInfo->fUnit) 
                << " yvalue " << yvalue
                << " yfcn(yvalue/yunit) " <<  yInfo->fFcn(yvalue/yInfo->fUnit) 
                << " weight " << weight;
    fState.GetVerboseL4()->Message("fill", "P1", description);
  }  
#endif
  return true;
}

//_____________________________________________________________________________
tools::hbook::p1*  ExG4HbookP1Manager::GetP1(G4int id, G4bool warn,
                                             G4bool onlyIfActive) const 
{
  return GetP1InFunction(id, "GetP1", warn, onlyIfActive);
}

//_____________________________________________________________________________
G4int  ExG4HbookP1Manager::GetP1Id(const G4String& name, G4bool warn) const
{
  std::map<G4String, G4int>::const_iterator it = fP1NameIdMap.find(name);
  if ( it ==  fP1NameIdMap.end() ) {  
    if ( warn) {
      G4String inFunction = "ExG4HbookP1Manager::GetP1Id";
      G4ExceptionDescription description;
      description << "      " << "profile " << name << " does not exist.";
      G4Exception(inFunction, "Analysis_W011", JustWarning, description);
    }
    return -1;         
  }
  return it->second;
}  
                                      
//_____________________________________________________________________________
G4int ExG4HbookP1Manager::GetP1Nbins(G4int id) const
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1Nbins");
  if ( ! p1 ) return 0;
  
  return fBaseToolsManager.GetNbins(p1->axis());
}  

//_____________________________________________________________________________
G4double ExG4HbookP1Manager::GetP1Xmin(G4int id) const
{
// Returns xmin value with applied unit and profile function

  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1Xmin");
  if ( ! p1 ) return 0;
  
  return fBaseToolsManager.GetMin(p1->axis());
}  

//_____________________________________________________________________________
G4double ExG4HbookP1Manager::GetP1Xmax(G4int id) const
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1Xmax");
  if ( ! p1 ) return 0;
  
  return fBaseToolsManager.GetMax(p1->axis());
}  

//_____________________________________________________________________________
G4double ExG4HbookP1Manager::GetP1XWidth(G4int id) const
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1XWidth", true, false);
  if ( ! p1 ) return 0;
  
  G4int nbins = p1->axis().bins();
  if ( ! nbins ) {
    G4ExceptionDescription description;
    description << "    nbins = 0 (for p1 id = " << id << ").";
    G4Exception("ExG4HbookP1Manager::GetP1Width",
                "Analysis_W014", JustWarning, description);
    return 0;
  }              
  
  return fBaseToolsManager.GetWidth(p1->axis());
}

//_____________________________________________________________________________
G4double ExG4HbookP1Manager::GetP1Ymin(G4int id) const
{
// Returns xmin value with applied unit and profile function

  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1Ymin", true, false);
  if ( ! p1 ) return 0;
  
  // not available
  //return p1->min_v();

  G4String inFunction = "ExG4HbookP1Manager::GetP1Ymin";
  G4ExceptionDescription description;
  description << "Get function not available.";
  G4Exception(inFunction, "Analysis_W011", JustWarning, description);
  return 0;
}

//_____________________________________________________________________________
G4double ExG4HbookP1Manager::GetP1Ymax(G4int id) const
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1Ymax", true, false);
  if ( ! p1 ) return 0;
  
  // not available
  //return p1->max_v();

  G4String inFunction = "ExG4HbookP1Manager::GetP1Ymax";
  G4ExceptionDescription description;
  description << "Get function not available.";
  G4Exception(inFunction, "Analysis_W011", JustWarning, description);
  return 0;
}

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1Title(G4int id, const G4String& title)
{
  p1_booking* p1Booking = GetP1Booking(id, false);
  if ( ! p1Booking ) {
    G4ExceptionDescription description;
    description << "      " << "profile " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::SetP1Title()",
                "Analysis_W011", JustWarning, description);
    return false;
  }

  p1Booking->fTitle = title;
  return true;
}  

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1XAxisTitle(G4int id, const G4String& title)
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "SetP1XAxisTitle");
  if ( ! p1 ) return false;
  
  p1->add_annotation(tools::hbook::key_axis_x_title(), title);
  return true;
}  

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1YAxisTitle(G4int id, const G4String& title)
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "SetP1YAxisTitle");
  if ( ! p1 ) return false;
  
  p1->add_annotation(tools::hbook::key_axis_y_title(), title);
  return true;
}  

//_____________________________________________________________________________
G4String ExG4HbookP1Manager::GetP1Title(G4int id) const
{
  p1_booking* p1Booking = GetP1Booking(id, false);
  if ( ! p1Booking ) {
    G4ExceptionDescription description;
    description << "      " << "profile " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::GetP1Title()",
                "Analysis_W011", JustWarning, description);
    return "";
  }
  
  return p1Booking->fTitle;
}  


//_____________________________________________________________________________
G4String ExG4HbookP1Manager::GetP1XAxisTitle(G4int id) const 
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1XAxisTitle");
  if ( ! p1 ) return "";
  
  G4String title;
  G4bool result = p1->annotation(tools::hbook::key_axis_x_title(), title);
  if ( ! result ) {
    G4ExceptionDescription description;
    description << "    Failed to get x_axis title for p1 id = " << id << ").";
    G4Exception("ExG4HbookP1Manager::GetP1XAxisTitle",
                "Analysis_W014", JustWarning, description);
    return "";
  }
  
  return title;              
}  

//_____________________________________________________________________________
G4String ExG4HbookP1Manager::GetP1YAxisTitle(G4int id) const 
{
  tools::hbook::p1* p1 = GetP1InFunction(id, "GetP1YAxisTitle");
  if ( ! p1 ) return "";
  
  G4String title;
  G4bool result = p1->annotation(tools::hbook::key_axis_y_title(), title);
  if ( ! result ) {
    G4ExceptionDescription description;
    description << "    Failed to get y_axis title for p1 id = " << id << ").";
    G4Exception("ExG4HbookP1Manager::GetP1YAxisTitle",
                "Analysis_W014", JustWarning, description);
    return "";
  }
  
  return title;              
}  

//_____________________________________________________________________________
G4bool ExG4HbookP1Manager::SetP1HbookIdOffset(G4int offset) 
{
  if ( fP1Vector.size() ) {
    G4ExceptionDescription description;
    description 
      << "Cannot set P1HbookIdOffset as some P1 profiles already exist.";
    G4Exception("G4HbookAnalysisManager::SetP1HbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }
  
  if ( fFirstId + offset < 1 ) {
    G4ExceptionDescription description;
    description << "The first profile HBOOK id must be >= 1.";
    G4Exception("G4HbookAnalysisManager::SetP1HbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }
  
  fP1HbookIdOffset = offset;
  return true;
}  

#endif
