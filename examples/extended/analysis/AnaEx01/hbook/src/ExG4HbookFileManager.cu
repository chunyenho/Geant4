
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$
//
/// \file hbook/src/ExG4HbookFileManager.cc
/// \brief Implementation of the ExG4HbookFileManager class

// Author: Ivana Hrivnacova, 15/06/2011  (ivana@ipno.in2p3.fr)

#ifdef G4_USE_HBOOK

#include "ExG4HbookFileManager.hh"
#include "G4AnalysisManagerState.hh"

#include <iostream>

const G4String ExG4HbookFileManager::fgkDefaultNtupleDirectoryName = "ntuple";

//_____________________________________________________________________________
ExG4HbookFileManager::ExG4HbookFileManager(const G4AnalysisManagerState& state)
 : G4VFileManager(state),
   fFile(0)
{
}

//_____________________________________________________________________________
ExG4HbookFileManager::~ExG4HbookFileManager()
{  
  delete fFile;  
}

// 
// public methods
//

//_____________________________________________________________________________
G4bool ExG4HbookFileManager::OpenFile(const G4String& fileName)
{
  // Keep file name
  fFileName =  fileName;
  G4String name = GetFullFileName();

#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("open", "analysis file", name);
#endif
  
  // delete a previous file if it exists
  if ( fFile ) delete fFile; 
  
  tools::hbook::CHCDIR("//PAWC"," ");
  
  unsigned int unit = 1;
  fFile = new tools::hbook::wfile(std::cout, name, unit);
  if ( ! fFile->is_valid() ) {
    G4ExceptionDescription description;
    description << "      " << "Cannot open file " << fileName;
    G4Exception("G4HbookAnalysisManager::OpenFile()",
                "Analysis_W001", JustWarning, description);
    return false;       
  }

  // At this point, in HBOOK, we should have :
  //   - created a //LUN1 directory attached to the file
  //   - created a //PAWC/LUN1 in memory
  //   - be in the directory //PAWC/LUN1.

  // create an "histo" HBOOK directory both in memory and in the file :
  if ( fHistoDirectoryName != "" ) {
    tools::hbook::CHCDIR("//PAWC/LUN1"," ");
    tools::hbook::CHMDIR(fHistoDirectoryName.data()," ");
    tools::hbook::CHCDIR("//LUN1"," ");
    tools::hbook::CHMDIR(fHistoDirectoryName.data()," ");
  }
  // the five upper lines could have been done with :
  //fFile->cd_home();
  //fFile->mkcd("histo");

  fLockFileName = true;
  fLockHistoDirectoryName = true;
  fLockNtupleDirectoryName = true;

#ifdef G4VERBOSE
  if ( fState.GetVerboseL1() ) 
    fState.GetVerboseL1()->Message("open", "analysis file", name);
#endif
  
  return true;
}  
  
//_____________________________________________________________________________
G4bool ExG4HbookFileManager::WriteFile() 
{
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("write", "file", GetFullFileName());
#endif

  // Return to //PAWC/LUN1 :
  //tools::hbook::CHCDIR("//PAWC/LUN1"," ");
  G4bool result = fFile->write();  

#ifdef G4VERBOSE
  if ( fState.GetVerboseL1() ) 
    fState.GetVerboseL1()->Message("write", "file", GetFullFileName(), result);
#endif

  return result;  
}

//_____________________________________________________________________________
G4bool ExG4HbookFileManager::CloseFile()
{
  // close file
  G4bool result = fFile->close();  
  fLockFileName = false;

  return result;
} 

//_____________________________________________________________________________
void ExG4HbookFileManager::CreateNtupleDirectory()
{
// Create an "ntuple" directory both in memory and in the file

  static G4bool isDone = false;
  
  // Do not create directory more than once
  if (isDone) return;
  
  fFile->cd_home();      //go under //PAWC/LUN1
  if ( fNtupleDirectoryName == "" )
    fFile->mkcd(fgkDefaultNtupleDirectoryName.data());
  else  
    fFile->mkcd(fNtupleDirectoryName.data());
  fLockNtupleDirectoryName = true;
  isDone = false;
}
                                     
#endif
