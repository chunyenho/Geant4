
#include <hip/hip_runtime.h>
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id$
//
/// \file hbook/src/ExG4HbookH2Manager.cc
/// \brief Implementation of the ExG4HbookH2Manager class

// Author: Ivana Hrivnacova, 15/06/2011  (ivana@ipno.in2p3.fr)

#ifdef G4_USE_HBOOK

#include "ExG4HbookH2Manager.hh"
#include "ExG4HbookFileManager.hh"
#include "G4HnManager.hh"
#include "G4AnalysisManagerState.hh"
#include "G4AnalysisUtilities.hh"

#include <iostream>

using namespace G4Analysis;

const G4int ExG4HbookH2Manager::fgkDefaultH2HbookIdOffset = 100;

//_____________________________________________________________________________
ExG4HbookH2Manager::ExG4HbookH2Manager(const G4AnalysisManagerState& state)
 : G4VH2Manager(state),
   fBaseToolsManager("H2"),
   fFileManager(0),
   fH2HbookIdOffset(-1),
   fH2Vector(),
   fH2BookingVector(),
   fH2NameIdMap()
{
}

//_____________________________________________________________________________
ExG4HbookH2Manager::~ExG4HbookH2Manager()
{  
  // Delete h2
  Reset();

  // Delete h2 booking 
  std::vector<h2_booking*>::iterator it2;
  for ( it2 = fH2BookingVector.begin(); it2 != fH2BookingVector.end(); it2++ ) {
    delete *it2;
  }  
}

// 
// private methods
//

//_____________________________________________________________________________
void ExG4HbookH2Manager::SetH2HbookIdOffset()
{
// Set  fH2HbookIdOffset if needed

  if ( fH2HbookIdOffset == -1 ) {
    if ( fFirstId > 0 ) 
      fH2HbookIdOffset = 0;
    else
      fH2HbookIdOffset = 1;
        
    if ( fH2HbookIdOffset > 0 ) {
      G4ExceptionDescription description;
      description << "H2 will be defined in HBOOK with ID = G4_firstHistoId + 1";
      G4Exception("ExG4HbookH2Manager::SetH1HbookIdOffset",
                  "Analysis_W013", JustWarning, description);
    }              
  }
}  

//_____________________________________________________________________________
void ExG4HbookH2Manager::CreateH2sFromBooking()
{
// Create h2 from h2_booking.

  // Do nothing if any h2 histogram already exists
  // or no h2 histograms are booked
  if ( fH2Vector.size() || ( fH2BookingVector.size() == 0 ) ) return;       

  // Go to histograms directory
  if ( fFileManager->GetHistoDirectoryName() != "" ) {
    G4String histoPath = "//PAWC/LUN1/";
    histoPath.append(fFileManager->GetHistoDirectoryName().data());
    tools::hbook::CHCDIR(histoPath.data()," ");
  }  
  
  // Create histograms
  G4int index = 0;
  std::vector<h2_booking*>::const_iterator it;
  for ( it = fH2BookingVector.begin(); it != fH2BookingVector.end(); ++it) {
    // Get information
    G4int id = index + fFirstId;    
    G4HnInformation* info = fHnManager->GetHnInformation(id, "CreateH2FromBooking");
    // Hbook index
    G4int hbookIndex = fH2HbookIdOffset + index + fFirstId;
    ++index;

#ifdef G4VERBOSE
    if ( fState.GetVerboseL3() ) 
      fState.GetVerboseL3()->Message("create from booking", "h2", info->GetName());
#endif

    // Create h2
    tools::hbook::h2* h2 
      = new tools::hbook::h2(hbookIndex, (*it)->fTitle, 
                             (*it)->fNxbins, (*it)->fXmin, (*it)->fXmax,
                             (*it)->fNybins, (*it)->fYmin, (*it)->fYmax);
    fH2Vector.push_back(h2);

#ifdef G4VERBOSE
    if ( fState.GetVerboseL3() ) { 
      G4ExceptionDescription description;
      description << " name : " << info->GetName() << " hbook index : " << hbookIndex; 
      fState.GetVerboseL3()->Message("create from booking", "h2", description);
    }  
#endif
  } 
  
  if ( fFileManager->GetHistoDirectoryName() != "" ) {
    // Return to //PAWC/LUN1 :
    tools::hbook::CHCDIR("//PAWC/LUN1"," ");
  }  
}   

//_____________________________________________________________________________
void ExG4HbookH2Manager::Reset()
{
// Reset histograms and ntuple  

  // Delete histograms
  std::vector<tools::hbook::h2*>::iterator it2;
  for (it2 = fH2Vector.begin(); it2 != fH2Vector.end(); it2++ ) {
    delete *it2;
  }  

  // Clear vectors
  fH2Vector.clear();
}  
 
//_____________________________________________________________________________
h2_booking*  ExG4HbookH2Manager::GetH2Booking(G4int id, G4bool warn) const 
{
  G4int index = id - fFirstId;
  if ( index < 0 || index >= G4int(fH2BookingVector.size()) ) {
    if ( warn) {
      G4ExceptionDescription description;
      description << "      " << "histo " << id << " does not exist.";
      G4Exception("G4HbookAnalysisManager::GetH2Booking()",
                  "Analysis_W011", JustWarning, description);
    }
    return 0;         
  }
  
  return fH2BookingVector[index];
}

//
// protected methods
//

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::WriteOnAscii(std::ofstream& /*output*/)
{
// Write selected objects on ASCII file
// According to the implementation by Michel Maire, originally in
// extended examples.
// Not yet available for H2

  return false;
}  

//_____________________________________________________________________________
tools::hbook::h2*  ExG4HbookH2Manager::GetH2InFunction(G4int id, 
                                      G4String functionName, G4bool warn,
                                      G4bool onlyIfActive) const
{                                      
  G4int index = id - fFirstId;
  if ( index < 0 || index >= G4int(fH2Vector.size()) ) {
    if ( warn) {
      G4String inFunction = "ExG4HbookH2Manager::";
      inFunction += functionName;
      G4ExceptionDescription description;
      description << "      " << "histogram " << id << " does not exist.";
      G4Exception(inFunction, "Analysis_W011", JustWarning, description);
    }
    return 0;         
  }

  // Do not return histogram if inactive 
  if ( fState.GetIsActivation()  && onlyIfActive && ( ! fHnManager->GetActivation(id) ) ) {
    return 0; 
  }  
  
  return fH2Vector[index];
}
  
// 
// public methods
//

//_____________________________________________________________________________
G4int ExG4HbookH2Manager::CreateH2(const G4String& name, const G4String& title,
                               G4int nxbins, G4double xmin, G4double xmax,
                               G4int nybins, G4double ymin, G4double ymax,
                               const G4String& xunitName, const G4String& yunitName,
                               const G4String& xfcnName, const G4String& yfcnName,
                               const G4String& xbinSchemeName,
                               const G4String& ybinSchemeName)
                               
{
  // HBook does not support user defined binning for H2
  if ( xbinSchemeName != "linear" || ybinSchemeName != "linear" ) {
    G4ExceptionDescription description;
    description 
      << "      " 
      << "Logarithimc binning is not supported for H2.";
    G4Exception("ExG4HbookH2Manager::CreateH2",
                "Analysis_F015", FatalException, description);
  }              

#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("create", "H2", name);
#endif

  // Create h2 booking & information
  G4int index = fH2BookingVector.size();
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);
  G4BinScheme xbinScheme = GetBinScheme(xbinSchemeName);
  G4BinScheme ybinScheme = GetBinScheme(ybinSchemeName);
  G4String newTitle(title);
  UpdateTitle(newTitle, xunitName, xfcnName);  
  UpdateTitle(newTitle, yunitName, yfcnName);  

  h2_booking* h2Booking = new h2_booking(nxbins, xfcn(xmin), xfcn(xmax), 
                                         nybins, yfcn(ymin), yfcn(ymax)); 
           // h2_booking object is deleted in destructor
  h2Booking->fTitle = newTitle;
  fH2BookingVector.push_back(h2Booking);
  fHnManager
    ->AddH2Information(name, xunitName, yunitName, xfcnName, yfcnName, 
                       xunit, yunit, xfcn, yfcn, xbinScheme, ybinScheme);
  
  // Set fH1HbookIdOffset if needed
  SetH2HbookIdOffset();
  
  // Hbook index
  G4int hbookIndex = fH2HbookIdOffset + index + fFirstId;

  // Create h2 if the file is open
  if ( fFileManager->IsFile() ) {
    // Go to histograms directory
    G4String histoPath = "//PAWC/LUN1/";
    if ( fFileManager->GetHistoDirectoryName() != "" ) {
      histoPath.append(fFileManager->GetHistoDirectoryName().data());
    }  
    tools::hbook::CHCDIR(histoPath.data()," ");

    // Create histogram    
    tools::hbook::h2* h2 
      = new tools::hbook::h2(hbookIndex, title, 
                             nxbins, xfcn(xmin), xfcn(xmax), 
                             nybins, yfcn(ymin), yfcn(ymax));
            // h2 objects are deleted when closing a file.
    fH2Vector.push_back(h2);

    // Return to //PAWC/LUN1 
    if ( fFileManager->GetHistoDirectoryName() != "" ) {
      tools::hbook::CHCDIR("//PAWC/LUN1"," ");
    }
  }    

  fLockFirstId = true;

#ifdef G4VERBOSE
    if ( fState.GetVerboseL2() ) {
      G4ExceptionDescription description;
      description << " name : " << name << " hbook index : " << hbookIndex; 
      fState.GetVerboseL2()->Message("create", "H2", description);
    }  
#endif

  fH2NameIdMap[name] = index + fFirstId;
  return index + fFirstId;
}                                         

//_____________________________________________________________________________
G4int ExG4HbookH2Manager::CreateH2(const G4String& /*name*/,  const G4String& /*title*/,
                          const std::vector<G4double>& /*xedges*/,
                          const std::vector<G4double>& /*yedges*/,
                          const G4String& /*xunitName*/, const G4String& /*yunitName*/,
                          const G4String& /*xfcnName*/, const G4String& /*yfcnName*/)
{                          
  // HBook does not support user defined binning for H2
  G4ExceptionDescription description;
  description 
    << "      " 
    << "User defined binning is not supported for H2.";
  G4Exception("ExG4HbookH2Manager::CreateH2",
              "Analysis_F015", FatalException, description);
  return 0;              
}              
                               
//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2(G4int id,
                                G4int nxbins, G4double xmin, G4double xmax, 
                                G4int nybins, G4double ymin, G4double ymax,
                                const G4String& xunitName, const G4String& yunitName,
                                const G4String& xfcnName, const G4String& yfcnName,
                                const G4String& xbinScheme, const G4String& ybinScheme)
{                                
  // HBook does not support user defined binning for H2
  if ( xbinScheme != "linear" || ybinScheme != "linear" ) {
    G4ExceptionDescription description;
    description 
      << "      " 
      << "Logarithimc binning is not supported for H2.";
    G4Exception("ExG4HbookH2Manager::CreateH2",
                "Analysis_F015", FatalException, description);
  }              

  h2_booking* h2Booking = GetH2Booking(id, false);
  if ( ! h2Booking ) {
    G4ExceptionDescription description;
    description << "      " << "histogram " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::SetH2()",
                "Analysis_W011", JustWarning, description);
    return false;
  }

  G4HnInformation* hnInfo = fHnManager->GetHnInformation(id, "SetH2");
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) 
    fState.GetVerboseL4()->Message("configure", "H2", hnInfo->GetName());
#endif

  // Keep new parameters in booking
  G4double xunit = GetUnitValue(xunitName);
  G4double yunit = GetUnitValue(yunitName);
  G4Fcn xfcn = GetFunction(xfcnName);
  G4Fcn yfcn = GetFunction(yfcnName);

  h2Booking->fNxbins = nxbins;
  h2Booking->fXmin = xfcn(xmin/xunit);
  h2Booking->fXmax = xfcn(xmax/xunit);
  h2Booking->fNybins = nybins;
  h2Booking->fYmin = yfcn(ymin/yunit);
  h2Booking->fYmax = yfcn(ymax/yunit);
  
  // Keep new parameters in information
  G4HnDimensionInformation* xInfo 
    = hnInfo->GetHnDimensionInformation(G4HnInformation::kX);
  xInfo->fUnitName = xunitName;
  xInfo->fFcnName = xfcnName;
  xInfo->fUnit = xunit;
  xInfo->fFcn = xfcn;
    
  G4HnDimensionInformation* yInfo 
    = hnInfo->GetHnDimensionInformation(G4HnInformation::kY);
  yInfo->fUnitName = yunitName;
  yInfo->fFcnName = yfcnName;
  yInfo->fUnit = yunit;
  yInfo->fFcn = yfcn;
  fHnManager->SetActivation(id, true); 

  G4String newTitle(h2Booking->fTitle);
  UpdateTitle(newTitle, xunitName, xfcnName);  
  UpdateTitle(newTitle, yunitName, yfcnName);  
  h2Booking->fTitle = newTitle;  
  
  // Re-configure histogram if it was already defined
  if ( fH2Vector.size() ) {
    tools::hbook::h2* h2 = GetH2(id);
    h2->configure(nxbins, xfcn(xmin/xunit), xfcn(xmax/xunit), 
                  nybins, yfcn(ymin/yunit), yfcn(ymax/yunit));
  }  
  
  return true;
}
                                  
//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2(G4int /*id*/,
                            const std::vector<G4double>& /*xedges*/,
                            const std::vector<G4double>& /*yedges*/,
                            const G4String& /*xunitName*/, const G4String& /*yunitName*/,
                            const G4String& /*xfcnName*/, const G4String& /*yfcnName*/)
{                          
  // HBook does not support user defined binning for H2
  G4ExceptionDescription description;
  description 
    << "      " 
    << "User defined binning is not supported for H2.";
  G4Exception("ExG4HbookH2Manager::CreateH2",
              "Analysis_F015", FatalException, description);

  return false;              
}              
                            
//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::ScaleH2(G4int id, G4double factor)
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "ScaleH2", false, false);
  if ( ! h2 ) return false;

  return h2->scale(factor);
}  
                           
//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::FillH2(G4int id, 
                                       G4double xvalue, G4double yvalue,
                                       G4double weight)
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "FillH2", true, false);
  if ( ! h2 ) return false;

  if ( fState.GetIsActivation() && ( ! fHnManager->GetActivation(id) ) ) return false; 

  G4HnDimensionInformation* xInfo 
    = fHnManager->GetHnDimensionInformation(id, G4HnInformation::kX, "FillH2");
  G4HnDimensionInformation* yInfo 
    = fHnManager->GetHnDimensionInformation(id, G4HnInformation::kY, "FillH2");
  h2->fill(xInfo->fFcn(xvalue/xInfo->fUnit), 
           yInfo->fFcn(yvalue/yInfo->fUnit), weight);
#ifdef G4VERBOSE
  if ( fState.GetVerboseL4() ) {
    G4ExceptionDescription description;
    description << " id " << id 
                << " xvalue " << xvalue 
                << " xfcn(xvalue/xunit) " <<  xInfo->fFcn(xvalue/xInfo->fUnit) 
                << " yvalue " << yvalue
                << " yfcn(yvalue/yunit) " <<  yInfo->fFcn(yvalue/yInfo->fUnit) 
                << " weight " << weight;
    fState.GetVerboseL4()->Message("fill", "H2", description);
  }  
#endif
  return true;
}

//_____________________________________________________________________________
tools::hbook::h2*  ExG4HbookH2Manager::GetH2(G4int id, G4bool warn,
                                                   G4bool onlyIfActive) const 
{
  return GetH2InFunction(id, "GetH2", warn, onlyIfActive);
}

//_____________________________________________________________________________
G4int  ExG4HbookH2Manager::GetH2Id(const G4String& name, G4bool warn) const
{
  std::map<G4String, G4int>::const_iterator it = fH2NameIdMap.find(name);
  if ( it ==  fH2NameIdMap.end() ) {  
    if ( warn) {
      G4String inFunction = "ExG4HbookH2Manager::GetH2Id";
      G4ExceptionDescription description;
      description << "      " << "histogram " << name << " does not exist.";
      G4Exception(inFunction, "Analysis_W011", JustWarning, description);
    }
    return -1;         
  }
  return it->second;
}  
                                      
//_____________________________________________________________________________
G4int ExG4HbookH2Manager::GetH2Nxbins(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2NXbins");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetNbins(h2->axis_x());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2Xmin(G4int id) const
{
// Returns xmin value with applied unit and histogram function

  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2Xmin");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetMin(h2->axis_x());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2Xmax(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2Xmax");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetMin(h2->axis_x());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2XWidth(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2XWidth", true, false);
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetWidth(h2->axis_x());
}  

//_____________________________________________________________________________
G4int ExG4HbookH2Manager::GetH2Nybins(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2NYbins");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetNbins(h2->axis_y());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2Ymin(G4int id) const
{
// Returns xmin value with applied unit and histogram function

  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2Ymin");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetMin(h2->axis_y());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2Ymax(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2Ymax");
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetMax(h2->axis_y());
}  

//_____________________________________________________________________________
G4double ExG4HbookH2Manager::GetH2YWidth(G4int id) const
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2YWidth", true, false);
  if ( ! h2 ) return 0;
  
  return fBaseToolsManager.GetWidth(h2->axis_y());
}  

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2Title(G4int id, const G4String& title)
{
  h2_booking* h2Booking = GetH2Booking(id, false);
  if ( ! h2Booking ) {
    G4ExceptionDescription description;
    description << "      " << "histogram " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::SetH2Title()",
                "Analysis_W011", JustWarning, description);
    return false;
  }

  h2Booking->fTitle = title;
  return true;
}  

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2XAxisTitle(G4int id, const G4String& title)
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "SetH2XAxisTitle");
  if ( ! h2 ) return false;
  
  return fBaseToolsManager.SetAxisTitle(*h2, ExG4HbookBaseHnManager::kX, title);
}  

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2YAxisTitle(G4int id, const G4String& title)
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "SetH2YAxisTitle");
  if ( ! h2 ) return false;
  
  return fBaseToolsManager.SetAxisTitle(*h2, ExG4HbookBaseHnManager::kY, title);
}  

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2ZAxisTitle(G4int id, const G4String& title)
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "SetH2ZAxisTitle");
  if ( ! h2 ) return false;
  
  return fBaseToolsManager.SetAxisTitle(*h2, ExG4HbookBaseHnManager::kZ, title);
}  

//_____________________________________________________________________________
G4String ExG4HbookH2Manager::GetH2Title(G4int id) const
{
  h2_booking* h2Booking = GetH2Booking(id, false);
  if ( ! h2Booking ) {
    G4ExceptionDescription description;
    description << "      " << "histogram " << id << " does not exist.";
    G4Exception("G4HbookAnalysisManager::GetH2Title()",
                "Analysis_W011", JustWarning, description);
    return "";
  }
  
  return h2Booking->fTitle;
}  


//_____________________________________________________________________________
G4String ExG4HbookH2Manager::GetH2XAxisTitle(G4int id) const 
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2XAxisTitle");
  if ( ! h2 ) return "";
  
  return fBaseToolsManager.GetAxisTitle(*h2, ExG4HbookBaseHnManager::kX);
} 

//_____________________________________________________________________________
G4String ExG4HbookH2Manager::GetH2YAxisTitle(G4int id) const 
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2YAxisTitle");
  if ( ! h2 ) return "";
  
  return fBaseToolsManager.GetAxisTitle(*h2, ExG4HbookBaseHnManager::kY);
}  

//_____________________________________________________________________________
G4String ExG4HbookH2Manager::GetH2ZAxisTitle(G4int id) const 
{
  tools::hbook::h2* h2 = GetH2InFunction(id, "GetH2ZAxisTitle");
  if ( ! h2 ) return "";
  
  return fBaseToolsManager.GetAxisTitle(*h2, ExG4HbookBaseHnManager::kZ);
}  

//_____________________________________________________________________________
G4bool ExG4HbookH2Manager::SetH2HbookIdOffset(G4int offset) 
{
  if ( fH2Vector.size() ) {
    G4ExceptionDescription description;
    description 
      << "Cannot set H2HbookIdOffset as some H2 histogramms already exist.";
    G4Exception("G4HbookAnalysisManager::SetH2HbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }

  if ( fFirstId + offset < 1 ) {
    G4ExceptionDescription description;
    description << "The first histogram HBOOK id must be >= 1.";
    G4Exception("G4HbookAnalysisManager::SetH1HbookIdOffset()",
                 "Analysis_W013", JustWarning, description);
    return false;             
  }
  
  fH2HbookIdOffset = offset;
  return true;
}  

#endif
