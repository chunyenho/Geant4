//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file medical/fanoCavity/src/PhysListEmStandard_option3.cc
/// \brief Implementation of the PhysListEmStandard_option3 class
//
// $Id: PhysListEmStandard_option3.cc 86064 2014-11-07 08:49:32Z gcosmo $
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "PhysListEmStandard_option3.hh"
#include "DetectorConstruction.hh"

#include "G4ParticleDefinition.hh"
#include "G4ProcessManager.hh"

#include "G4ComptonScattering.hh"
#include "MyKleinNishinaCompton.hh"
#include "G4GammaConversion.hh"
#include "G4PhotoElectricEffect.hh"

#include "G4eMultipleScattering.hh"

#include "G4eIonisation.hh"
#include "MyMollerBhabhaModel.hh"
#include "G4eBremsstrahlung.hh"
#include "G4eplusAnnihilation.hh"

#include "G4hIonisation.hh"
#include "G4hMultipleScattering.hh"

#include "G4EmProcessOptions.hh"
#include "G4MscStepLimitType.hh"

#include "G4SystemOfUnits.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

PhysListEmStandard_option3::PhysListEmStandard_option3(const G4String& name,
                               DetectorConstruction* det)
: G4VPhysicsConstructor(name), fDetector(det)
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

PhysListEmStandard_option3::~PhysListEmStandard_option3()
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void PhysListEmStandard_option3::ConstructProcess()
{
  // Add standard EM Processes
  //

  aParticleIterator->reset();
  while( (*aParticleIterator)() ){
    G4ParticleDefinition* particle = aParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    G4String particleName = particle->GetParticleName();
     
    if (particleName == "gamma") {
      // gamma
    
      G4ComptonScattering* compton = new G4ComptonScattering();
      MyKleinNishinaCompton* comptonModel = 
        new MyKleinNishinaCompton(fDetector);
      comptonModel->SetCSFactor(1000.);      
      compton->SetEmModel(comptonModel );
            
      pmanager->AddDiscreteProcess(new G4PhotoElectricEffect);
      pmanager->AddDiscreteProcess(compton);
      pmanager->AddDiscreteProcess(new G4GammaConversion);
      
    } else if (particleName == "e-") {
      //electron
      G4eMultipleScattering* msc = new G4eMultipleScattering();
            
      G4eIonisation* eIoni = new G4eIonisation();
      eIoni->SetEmModel(new MyMollerBhabhaModel);
                         
      pmanager->AddProcess(msc,                       -1, 1, 1);
      pmanager->AddProcess(eIoni,                     -1, 2, 2);
///      pmanager->AddProcess(new G4eBremsstrahlung,     -1, 3, 3);
            
    } else if (particleName == "e+") {
      //positron
      G4eMultipleScattering* msc = new G4eMultipleScattering();
            
      G4eIonisation* pIoni = new G4eIonisation();
      pIoni->SetEmModel(new MyMollerBhabhaModel);
                               
      pmanager->AddProcess(msc,                       -1, 1, 1);
      pmanager->AddProcess(pIoni,                     -1, 2, 2);
///      pmanager->AddProcess(new G4eBremsstrahlung,     -1, 3, 3);
      pmanager->AddProcess(new G4eplusAnnihilation,    0,-1, 3);
             
    } else if( particleName == "proton" ) {
      //proton  
      pmanager->AddProcess(new G4hMultipleScattering, -1, 1, 1);
      pmanager->AddProcess(new G4hIonisation,         -1, 2, 2);
    }
  }

  // Em options
  //
  // Main options and setting parameters are shown here.
  // Several of them have default values.
  //
  G4EmProcessOptions emOptions;
  
  //physics tables
  //
  emOptions.SetMinEnergy(100*eV);        //default    
  emOptions.SetMaxEnergy(10*GeV);        //default  
  emOptions.SetDEDXBinning(8*20);        //default=8*7
  emOptions.SetLambdaBinning(8*20);      //default=8*7
      
  //multiple coulomb scattering
  //
  emOptions.SetMscStepLimitation(fUseDistanceToBoundary);  
      
  //energy loss
  //
  emOptions.SetStepFunction(0.2, 10*um);        //default=(0.2, 1*mm)   
           
  //build CSDA range
  //
  emOptions.SetBuildCSDARange(true);            //default=false
  emOptions.SetMaxEnergyForCSDARange(10*GeV);  
  emOptions.SetDEDXBinningForCSDARange(8*20);   //default=8*7
          
  //ionization
  //
  emOptions.SetSubCutoff(false);         //default
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

