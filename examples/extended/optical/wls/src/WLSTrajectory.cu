#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: WLSTrajectory.cc 72065 2013-07-05 09:54:59Z gcosmo $
//
/// \file optical/wls/src/WLSTrajectory.cc
/// \brief Implementation of the WLSTrajectory class
//
//
#include "G4AttDef.hh"
#include "G4AttValue.hh"
#include "G4AttDefStore.hh"

#include "G4UIcommand.hh"
#include "G4UnitsTable.hh"

#include "WLSTrajectory.hh"
#include "WLSTrajectoryPoint.hh"
#include "G4ParticleTable.hh"
#include "G4ParticleTypes.hh"

#include "G4Polyline.hh"
#include "G4Colour.hh"
#include "G4VisAttributes.hh"
#include "G4VVisManager.hh"
#include "G4Polymarker.hh"

//#define G4ATTDEBUG
#ifdef G4ATTDEBUG
#include "G4AttCheck.hh"
#endif

G4ThreadLocal G4Allocator<WLSTrajectory>* WLSTrajectoryAllocator=0;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

WLSTrajectory::WLSTrajectory()
    : fpPointsContainer(0), fTrackID(0), fParentID(0),
      fPDGCharge(0.0), fPDGEncoding(0), fParticleName(""),
      fInitialMomentum(G4ThreeVector())
{
    fWLS         = false;
    fDrawIt      = false;
    fForceNoDraw = false;
    fForceDraw   = false;

    fParticleDefinition = NULL;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

WLSTrajectory::WLSTrajectory(const G4Track* aTrack)
{
    fParticleDefinition = aTrack->GetDefinition();
    fParticleName = fParticleDefinition->GetParticleName();
    fPDGCharge = fParticleDefinition->GetPDGCharge();
    fPDGEncoding = fParticleDefinition->GetPDGEncoding();
    fTrackID = aTrack->GetTrackID();
    fParentID = aTrack->GetParentID();
    fInitialMomentum = aTrack->GetMomentum();
    fpPointsContainer = new WLSTrajectoryPointContainer();
    // Following is for the first trajectory point
    fpPointsContainer->push_back(new WLSTrajectoryPoint(aTrack));

    fWLS         = false;
    fDrawIt      = false;
    fForceNoDraw = false;
    fForceDraw   = false;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

WLSTrajectory::WLSTrajectory(WLSTrajectory & right) : G4VTrajectory()
{
    fParticleDefinition=right.fParticleDefinition;
    fParticleName = right.fParticleName;
    fPDGCharge = right.fPDGCharge;
    fPDGEncoding = right.fPDGEncoding;
    fTrackID = right.fTrackID;
    fParentID = right.fParentID;
    fInitialMomentum = right.fInitialMomentum;
    fpPointsContainer = new WLSTrajectoryPointContainer();

    for(size_t i=0;i<right.fpPointsContainer->size();++i) {
        WLSTrajectoryPoint* rightPoint
            = (WLSTrajectoryPoint*)((*(right.fpPointsContainer))[i]);
        fpPointsContainer->push_back(new WLSTrajectoryPoint(*rightPoint));
    }

    fWLS         = right.fWLS;
    fDrawIt      = right.fDrawIt;
    fForceNoDraw = right.fForceNoDraw;
    fForceDraw   = right.fForceDraw;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

WLSTrajectory::~WLSTrajectory()
{
    for(size_t i=0;i<fpPointsContainer->size();++i){
        delete  (*fpPointsContainer)[i];
    }
    fpPointsContainer->clear();

    delete fpPointsContainer;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void WLSTrajectory::ShowTrajectory(std::ostream& os) const
{
    // Invoke the default implementation in G4VTrajectory...
    G4VTrajectory::ShowTrajectory(os);
    // ... or override with your own code here.
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void WLSTrajectory::DrawTrajectory() const
{
    // i_mode is no longer available as an argument of G4VTrajectory.
    // In this exampple it was always called with an argument of 50.
    const G4int i_mode = 50;
    // Consider using commands /vis/modeling/trajectories.

    // Invoke the default implementation in G4VTrajectory...
    // G4VTrajectory::DrawTrajectory(i_mode);
    // ... or override with your own code here.

    //Taken from G4VTrajectory and modified to select colours based on particle
    //type and to selectively eliminate drawing of certain trajectories.

    if (!fForceDraw && (!fDrawIt || fForceNoDraw)) return;

    // If i_mode>=0, draws a trajectory as a polyline and, if i_mode!=0,
    // adds markers - yellow circles for step points and magenta squares
    // for auxiliary points, if any - whose screen size in pixels is
    // given by std::abs(i_mode)/1000.  E.g: i_mode = 5000 gives easily
    // visible markers.

    G4VVisManager* pVVisManager = G4VVisManager::GetConcreteInstance();
    if (!pVVisManager) return;

    const G4double markerSize = std::abs(i_mode)/1000;
    G4bool lineRequired (i_mode >= 0);
    G4bool markersRequired (markerSize > 0.);

    G4Polyline trajectoryLine;
    G4Polymarker stepPoints;
    G4Polymarker auxiliaryPoints;

    for (G4int i = 0; i < GetPointEntries() ; i++) {
      G4VTrajectoryPoint* aTrajectoryPoint = GetPoint(i);
      const std::vector<G4ThreeVector>* auxiliaries
        = aTrajectoryPoint->GetAuxiliaryPoints();
      if (auxiliaries) {
        for (size_t iAux = 0; iAux < auxiliaries->size(); ++iAux) {
          const G4ThreeVector pos((*auxiliaries)[iAux]);
          if (lineRequired) {
            trajectoryLine.push_back(pos);
          }
          if (markersRequired) {
            auxiliaryPoints.push_back(pos);
          }
        }
      }
      const G4ThreeVector pos(aTrajectoryPoint->GetPosition());
      if (lineRequired) {
        trajectoryLine.push_back(pos);
      }
      if (markersRequired) {
        stepPoints.push_back(pos);
      }
    }

    if (lineRequired) {
      G4Colour colour;

      if(fParticleDefinition==G4OpticalPhoton::OpticalPhotonDefinition()){
        if(fWLS) //WLS photons are red
          colour = G4Colour(1.,0.,0.);
        else{ //Scintillation and Cerenkov photons are green
          colour = G4Colour(0.,1.,0.);
        }
      }
      else //All other particles are blue
        colour = G4Colour(0.,0.,1.);

      G4VisAttributes trajectoryLineAttribs(colour);
      trajectoryLine.SetVisAttributes(&trajectoryLineAttribs);
      pVVisManager->Draw(trajectoryLine);
    }
    if (markersRequired) {
      auxiliaryPoints.SetMarkerType(G4Polymarker::squares);
      auxiliaryPoints.SetScreenSize(markerSize);
      auxiliaryPoints.SetFillStyle(G4VMarker::filled);
      G4VisAttributes auxiliaryPointsAttribs(G4Colour(0.,1.,1.));  // Magenta
      auxiliaryPoints.SetVisAttributes(&auxiliaryPointsAttribs);
      pVVisManager->Draw(auxiliaryPoints);

      stepPoints.SetMarkerType(G4Polymarker::circles);
      stepPoints.SetScreenSize(markerSize);
      stepPoints.SetFillStyle(G4VMarker::filled);
      G4VisAttributes stepPointsAttribs(G4Colour(1.,1.,0.));  // Yellow.
      stepPoints.SetVisAttributes(&stepPointsAttribs);
      pVVisManager->Draw(stepPoints);
    }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void WLSTrajectory::AppendStep(const G4Step* aStep)
{
    fpPointsContainer->push_back(new WLSTrajectoryPoint(aStep));
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4ParticleDefinition* WLSTrajectory::GetParticleDefinition()
{
    return (G4ParticleTable::GetParticleTable()->FindParticle(fParticleName));
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void WLSTrajectory::MergeTrajectory(G4VTrajectory* secondTrajectory)
{
    if(!secondTrajectory) return;

    WLSTrajectory* second = (WLSTrajectory*)secondTrajectory;
    G4int ent = second->GetPointEntries();
    // initial point of the second trajectory should not be merged
    for(G4int i=1; i<ent; ++i) {
        fpPointsContainer->push_back((*(second->fpPointsContainer))[i]);
    }
    delete (*second->fpPointsContainer)[0];
    second->fpPointsContainer->clear();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

const std::map<G4String,G4AttDef>* WLSTrajectory::GetAttDefs() const
{
    G4bool isNew;
    std::map<G4String,G4AttDef>* store
        = G4AttDefStore::GetInstance("Trajectory",isNew);

    if (isNew) {

      G4String ID("ID");
      (*store)[ID] = G4AttDef(ID,"Track ID","Bookkeeping","","G4int");

      G4String PID("PID");
      (*store)[PID] = G4AttDef(PID,"Parent ID","Bookkeeping","","G4int");

      G4String PN("PN");
      (*store)[PN] = G4AttDef(PN,"Particle Name","Physics","","G4String");

      G4String Ch("Ch");
      (*store)[Ch] = G4AttDef(Ch,"Charge","Physics","e+","G4double");

      G4String PDG("PDG");
      (*store)[PDG] = G4AttDef(PDG,"PDG Encoding","Physics","","G4int");

      G4String IMom("IMom");
      (*store)[IMom] = G4AttDef(IMom,
                       "Momentum of track at start of trajectory",
                       "Physics","G4BestUnit","G4ThreeVector");

      G4String IMag("IMag");
      (*store)[IMag] = G4AttDef(IMag,
                       "Magnitude of momentum of track at start of trajectory",
                       "Physics","G4BestUnit","G4double");

        G4String NTP("NTP");
        (*store)[NTP] = G4AttDef(NTP,"No. of points","Bookkeeping","","G4int");

    }
    return store;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

std::vector<G4AttValue>* WLSTrajectory::CreateAttValues() const
{
  std::vector<G4AttValue>* values = new std::vector<G4AttValue>;

  values->push_back
    (G4AttValue("ID",G4UIcommand::ConvertToString(fTrackID),""));

  values->push_back
    (G4AttValue("PID",G4UIcommand::ConvertToString(fParentID),""));

  values->push_back(G4AttValue("PN",fParticleName,""));

  values->push_back
    (G4AttValue("Ch",G4UIcommand::ConvertToString(fPDGCharge),""));

  values->push_back
    (G4AttValue("PDG",G4UIcommand::ConvertToString(fPDGEncoding),""));

  values->push_back
    (G4AttValue("IMom",G4BestUnit(fInitialMomentum,"Energy"),""));

  values->push_back
    (G4AttValue("IMag",G4BestUnit(fInitialMomentum.mag(),"Energy"),""));

  values->push_back
    (G4AttValue("NTP",G4UIcommand::ConvertToString(GetPointEntries()),""));

#ifdef G4ATTDEBUG
  G4cout << G4AttCheck(values,GetAttDefs());
#endif
    return values;
}
