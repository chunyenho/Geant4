#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10PrimaryGeneratorAction.cc
/// \brief Implementation of the Em10PrimaryGeneratorAction class
//
//
// $Id: Em10PrimaryGeneratorAction.cc 73033 2013-08-15 09:24:45Z gcosmo $
//
//

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "Em10PrimaryGeneratorAction.hh"

#include "Em10DetectorConstruction.hh"
#include "Em10PrimaryGeneratorMessenger.hh"

#include "G4Event.hh"
#include "G4ParticleGun.hh"
#include "G4ParticleTable.hh"
#include "G4ParticleDefinition.hh"
#include "Randomize.hh"
#include "G4ios.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

 G4String Em10PrimaryGeneratorAction::thePrimaryParticleName="proton";

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10PrimaryGeneratorAction::Em10PrimaryGeneratorAction(
                                            Em10DetectorConstruction*)
//                                            Em10DetectorConstruction* Em10DC)
:G4VUserPrimaryGeneratorAction(),
// Em10Detector(Em10DC),
 rndmFlag("off"),xvertex(0.),yvertex(0.),zvertex(0.),
 vertexdefined(false)
{
  G4int n_particle = 1;
  particleGun  = new G4ParticleGun(n_particle);

  //create a messenger for this class
  gunMessenger = new Em10PrimaryGeneratorMessenger(this);

  // default particle kinematic

  G4ParticleTable* particleTable = G4ParticleTable::GetParticleTable();
  G4String particleName;
  G4ParticleDefinition* particle
                    = particleTable->FindParticle(particleName="proton");
  particleGun->SetParticleDefinition(particle);

  thePrimaryParticleName = particle->GetParticleName();

  particleGun->SetParticleMomentumDirection(G4ThreeVector(0.,0.,1.));
  particleGun->SetParticleEnergy(100.*GeV);

  zvertex = 0.0 ; //  -0.5*(Em10Detector->GetAbsorberThickness());
  particleGun->SetParticlePosition(G4ThreeVector(xvertex,yvertex,zvertex));

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10PrimaryGeneratorAction::~Em10PrimaryGeneratorAction()
{
  delete particleGun;
  delete gunMessenger;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10PrimaryGeneratorAction::GeneratePrimaries(G4Event* anEvent)
{
  //this function is called at the begining of event
  //
  thePrimaryParticleName = particleGun->GetParticleDefinition()->
                                                GetParticleName();
  /* ****************************************************
  G4double x0,y0,z0;
  if(vertexdefined)
  {
    x0 = xvertex;
    y0 = yvertex;
    z0 = zvertex;
  }
  else
  {
    x0 = 0.;
    y0 = 0.;
    z0 = 0.; // -0.5*(Em10Detector->GetWorldSizeZ()) ;
  }
  G4double r0,phi0;
  if (rndmFlag == "on")
  {
      r0 = Em10Detector->GetAbsorberRadius())*std::sqrt(G4UniformRand();
      phi0 = twopi*G4UniformRand();
      x0 = r0*std::cos(phi0);
      y0 = r0*std::sin(phi0);
  }
  ********************************************* */
  //  particleGun->SetParticlePosition(G4ThreeVector(x0,y0,z0));
  particleGun->SetParticlePosition(G4ThreeVector(0.,0.,0.));
  particleGun->GeneratePrimaryVertex(anEvent);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4String Em10PrimaryGeneratorAction::GetPrimaryName()
{
   return thePrimaryParticleName;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10PrimaryGeneratorAction::Setzvertex(G4double z)
{
  vertexdefined = true;
  zvertex = z;
  G4cout << " Z coordinate of the primary vertex = " << zvertex/mm <<
            " mm." << G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10PrimaryGeneratorAction::Setxvertex(G4double x)
{
  vertexdefined = true;
  xvertex = x;
  G4cout << " X coordinate of the primary vertex = " << xvertex/mm <<
            " mm." << G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10PrimaryGeneratorAction::Setyvertex(G4double y)
{
  vertexdefined = true;
  yvertex = y;
  G4cout << " Y coordinate of the primary vertex = " << yvertex/mm <<
            " mm." << G4endl;
}
