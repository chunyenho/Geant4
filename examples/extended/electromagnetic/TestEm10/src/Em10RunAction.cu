#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10RunAction.cc
/// \brief Implementation of the Em10RunAction class
//
//
// $Id: Em10RunAction.cc 67268 2013-02-13 11:38:40Z ihrivnac $
//
// 


#include "Em10RunAction.hh"
#include "Em10RunMessenger.hh"

#include "G4Run.hh"
#include "G4UImanager.hh"
#include "G4VVisManager.hh"
#include "G4ios.hh"
#include <iomanip>

#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"
#include "Randomize.hh"

//////////////////////////////////////////////////////////////////////////////

Em10RunAction::Em10RunAction()
  :G4UserRunAction(),
   histName("histfile"),nbinStep(0),nbinEn(0),nbinTt(0),nbinTb(0),
   nbinTsec(0),nbinTh(0),nbinThback(0),nbinR(0),nbinGamma(0),
   nbinvertexz(0)
{
  runMessenger = new Em10RunMessenger(this);
  saveRndm = 1;  

  EnergySumAbs = EnergySquareSumAbs = tlSumAbs = tlsquareSumAbs = 
    nStepSumCharged = nStepSum2Charged= nStepSumNeutral = nStepSum2Neutral=
    TotNbofEvents = SumCharged= SumNeutral=Sum2Charged=Sum2Neutral=Selectron=
    Spositron=Transmitted=Reflected  =dStep=entryStep=underStep=overStep=
    dEn = entryEn= underEn=overEn=dTt = entryTt=underTt=overTt=Ttmean=
    Tt2mean=dTb = entryTb=underTb=overTb=Tbmean=Tb2mean=dTsec =entryTsec=
    underTsec=overTsec=dTh = entryTh=underTh=overTh=dThback =entryThback=
    underThback=overThback=dR  =entryR =underR =overR =Rmean=R2mean=dEGamma= 
    entryGamma = underGamma=overGamma=dz=entryvertexz=undervertexz=
    oververtexz=0.;
}

////////////////////////////////////////////////////////////////////////////

Em10RunAction::~Em10RunAction()
{
  delete runMessenger;
}

////////////////////////////////////////////////////////////////////////////////

void Em10RunAction::bookHisto()
{
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::BeginOfRunAction(const G4Run* aRun)
{  
  G4cout << "### Run " << aRun->GetRunID() << " start." << G4endl;
  
  // save Rndm status
  if (saveRndm > 0)
  { 
      CLHEP::HepRandom::showEngineStatus();
      CLHEP::HepRandom::saveEngineStatus("beginOfRun.rndm");
  }  
  G4UImanager* UI = G4UImanager::GetUIpointer();
   
  G4VVisManager* pVVisManager = G4VVisManager::GetConcreteInstance();

  if(pVVisManager)    UI->ApplyCommand("/vis/scene/notifyHandlers");

      
  EnergySumAbs = 0. ;
  EnergySquareSumAbs = 0.;
  tlSumAbs = 0. ;
  tlsquareSumAbs = 0. ;
  nStepSumCharged = 0. ;
  nStepSum2Charged= 0. ;
  nStepSumNeutral = 0. ;
  nStepSum2Neutral= 0. ;
  TotNbofEvents = 0. ;
  SumCharged=0.;
  SumNeutral=0.;
  Sum2Charged=0.;
  Sum2Neutral=0.;
  Selectron=0.;
  Spositron=0.;

  Transmitted=0.;
  Reflected  =0.;

  //  plot definitions 
   
  if(nbinStep>0)
  {
    dStep=(Stephigh-Steplow)/nbinStep;
    entryStep=0.;
    underStep=0.;
    overStep=0.;
    for(G4int ist=0; ist<200; ist++)
    {
      distStep[ist]=0.;
    }
  }      
  if(nbinEn>0)
  {
    dEn = (Enhigh-Enlow)/nbinEn ;
    entryEn=0.;
    underEn=0.;
    overEn=0.;

    for (G4int ien=0; ien<200; ien++)   distEn[ien]=0.;
  }
  if(nbinTt>0)
  {
    dTt = (Tthigh-Ttlow)/nbinTt ;
    entryTt=0.;
    underTt=0.;
    overTt=0.;

    for (G4int itt=0; itt<200; itt++)  distTt[itt]=0.;

    Ttmean=0.;
    Tt2mean=0.;
  }
  if(nbinTb>0)
  {
    dTb = (Tbhigh-Tblow)/nbinTb ;
    entryTb=0.;
    underTb=0.;
    overTb=0.;
    for (G4int itt=0; itt<200; itt++)
    {
      distTb[itt]=0.;
    }
    Tbmean=0.;
    Tb2mean=0.;
  }
  if(nbinTsec>0)
  {
    dTsec = (Tsechigh-Tseclow)/nbinTsec ;
    entryTsec=0.;
    underTsec=0.;
    overTsec=0.;
    for (G4int its=0; its<200; its++)
    {
      distTsec[its]=0.;
    }
  }
  if(nbinTh>0)
  {
    dTh = (Thhigh-Thlow)/nbinTh ;
    entryTh=0.;
    underTh=0.;
    overTh=0.;
    for (G4int ith=0; ith<200; ith++)
    {
      distTh[ith]=0.;
    }
  }

  if(nbinThback>0)
  {
    dThback = (Thhighback-Thlowback)/nbinThback ;
    entryThback=0.;
    underThback=0.;
    overThback=0.;
    for (G4int ithback=0; ithback<200; ithback++)
    {
      distThback[ithback]=0.;
    }
  }


  if(nbinR >0)
  {
    dR  = (Rhigh-Rlow)/nbinR  ;
    entryR =0.;
    underR =0.;
    overR =0.;
    for (G4int ir =0; ir<200; ir++)
    {
      distR[ir]=0.;
    }
    Rmean=0.;
    R2mean=0.;
  }

  if(nbinGamma>0)
  {
    dEGamma = std::log(EhighGamma/ElowGamma)/nbinGamma ;
    entryGamma = 0.;
    underGamma=0.;
    overGamma=0.;
    for (G4int ig=0; ig<200; ig++)
    {
      distGamma[ig]=0.;
    }
  } 
  if(nbinvertexz>0)
  {
    dz=(zhigh-zlow)/nbinvertexz;
    entryvertexz=0.;
    undervertexz=0.;
    oververtexz=0.;
    for(G4int iz=0; iz<200; iz++)
    {
      distvertexz[iz]=0.;
    }
  }

  bookHisto();
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::EndOfRunAction(const G4Run*)
{
  G4double sAbs,sigAbs,sigstep,sigcharged,signeutral;
  if(0.0 >= TotNbofEvents) { return; }

  tlSumAbs /= TotNbofEvents ;
  sAbs = tlsquareSumAbs/TotNbofEvents-tlSumAbs*tlSumAbs ;
  if(sAbs>0.)
    sAbs = std::sqrt(sAbs/TotNbofEvents) ;
  else
    sAbs = 0. ;
  
  EnergySumAbs /= TotNbofEvents ;
  sigAbs = EnergySquareSumAbs/TotNbofEvents-EnergySumAbs*EnergySumAbs;
  if(sigAbs>0.)
    sigAbs = std::sqrt(sigAbs/TotNbofEvents);
  else
    sigAbs = 0.;

  nStepSumCharged /= TotNbofEvents ;
  sigstep = nStepSum2Charged/TotNbofEvents-nStepSumCharged*nStepSumCharged;
  if(sigstep>0.)
    sigstep = std::sqrt(sigstep/TotNbofEvents);
  else
    sigstep = 0.;
  G4double sigch=sigstep ;
  
  nStepSumNeutral /= TotNbofEvents ;
  sigstep = nStepSum2Neutral/TotNbofEvents-nStepSumNeutral*nStepSumNeutral;
  if(sigstep>0.)
    sigstep = std::sqrt(sigstep/TotNbofEvents);
  else
    sigstep = 0.;
  G4double signe=sigstep ;
  
  SumCharged /= TotNbofEvents;
  sigcharged = Sum2Charged/TotNbofEvents-SumCharged*SumCharged; 
  if(sigcharged>0.)
    sigcharged = std::sqrt(sigcharged/TotNbofEvents);
  else
    sigcharged = 0. ;
 
  SumNeutral /= TotNbofEvents;
  signeutral = Sum2Neutral/TotNbofEvents-SumNeutral*SumNeutral; 
  if(signeutral>0.)
    signeutral = std::sqrt(signeutral/TotNbofEvents);
  else
    signeutral = 0. ;
 
  Selectron /= TotNbofEvents ;
  Spositron /= TotNbofEvents ;

  Transmitted /=TotNbofEvents ;
  Reflected   /=TotNbofEvents ;
  G4cout << " ================== run summary =====================" << G4endl;
  G4int prec = G4cout.precision(6);
  G4cout << " end of Run TotNbofEvents = " <<  
           TotNbofEvents << G4endl ;
  G4cout << "    mean charged track length   in absorber=" <<
           tlSumAbs/mm      << " +- " << sAbs/mm    <<
          "  mm  " << G4endl; 
  G4cout << G4endl;
  G4cout << "            mean energy deposit in absorber=" <<
           EnergySumAbs/MeV << " +- " << sigAbs/MeV <<
          "  MeV " << G4endl ;
  G4cout << G4endl ;
  G4cout << " mean number of steps in absorber (charged) =" <<
           nStepSumCharged         << " +- " << sigch     <<
          "      " << G4endl ;
  G4cout << " mean number of steps in absorber (neutral) =" <<
           nStepSumNeutral         << " +- " << signe     <<
          "      " << G4endl ;
  G4cout << G4endl ;
  G4cout << "   mean number of charged secondaries = " <<
           SumCharged << " +- " << sigcharged << G4endl;  
  G4cout << G4endl ;
  G4cout << "   mean number of neutral secondaries = " <<
           SumNeutral << " +- " << signeutral << G4endl;  
  G4cout << G4endl ;
  
  G4cout << "   mean number of e-s =" << Selectron << 
            "  and e+s =" << Spositron << G4endl;
  G4cout << G4endl; 
  
  G4cout << "(number) transmission coeff=" << Transmitted <<
            "  reflection coeff=" << Reflected << G4endl;
  G4cout << G4endl; 

  if(nbinStep>0)
  {G4double E , dnorm, norm ;
   G4cout << "   step number/event distribution " << G4endl ;
   G4cout << "#entries=" << entryStep << "    #underflows=" << underStep <<
             "    #overflows=" << overStep << G4endl ;
   if( entryStep>0.)
   {
     E = Steplow - dStep ;
     norm = TotNbofEvents ;
     G4cout << " bin nb   nsteplow     entries     normalized " << G4endl ;
     for(G4int iss=0; iss<nbinStep; iss++)
     {
      E += dStep ;
      dnorm = distStep[iss]/norm;
      G4cout << std::setw(5) << iss << std::setw(10) << E << 
                std::setw(12) << distStep[iss] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }     
  }
  if(nbinEn > 0)
  {
    std::ofstream fileOut("distribution.out", std::ios::out ) ;
    fileOut.setf( std::ios::scientific, std::ios::floatfield );

    std::ofstream normOut("normDist.out", std::ios::out ) ;
    normOut.setf( std::ios::scientific, std::ios::floatfield );

    G4double E , dnorm, norm,fmax,Emp ;
    Emp=-999.999 ;
    G4cout << " energy deposit distribution " << G4endl ;
    G4cout << "#entries=" << entryEn << "    #underflows=" << underEn <<
             "    #overflows=" << overEn << G4endl ;
    if( entryEn>0.)
    {
      E = Enlow - dEn ;
      norm = TotNbofEvents*1.0 ;   // *dEn ;
      G4cout << " bin nb      Elow      entries     normalized " << G4endl ;
      fmax = 0. ;

      for(G4int ien=0; ien<nbinEn; ien++)
      {
        E += dEn ;

        if(distEn[ien]>fmax)
        {
          fmax = distEn[ien] ;
          Emp = E ;                // most probable roughly
        }
        dnorm = distEn[ien]/norm;

        G4cout << std::setw(5) << ien << std::setw(10) << E/keV << 
                  std::setw(12) << distEn[ien] <<
                  std::setw(12) << dnorm << G4endl ;

        fileOut << E/keV << "\t"<< distEn[ien] << G4endl ;
        normOut << E/keV << "\t"<< dnorm << G4endl ;
      }
      G4cout << G4endl;
      G4int ii ;
      G4double E1,E2 ;
      E1=-1.e6 ;
      E2=+1.e6 ;
      E = Enlow -dEn ;
      ii = -1;

      for(G4int i1=0; i1<nbinEn; i1++)
      {
        E += dEn ;
        if(ii<0)
        {
          if(distEn[i1] >= 0.5*fmax)
          {
            E1=E ;
            ii=i1 ;
          }
        }
      }
      E = Enlow -dEn ;

      for(G4int i2=0; i2<nbinEn; i2++)
      {
        E += dEn ;

        if(distEn[i2] >= 0.5*fmax)   E2=E ;
      }
      G4cout << " Emp = " << std::setw(15) << Emp/MeV << "   width="
            << std::setw(15) << (E2-E1)/MeV <<   "  MeV " << G4endl;
      G4cout << G4endl ;
    }     
  }
  if(nbinTt>0)
  {
     G4double E , dnorm, norm ,sig;
   G4cout << " transmitted energy distribution " << G4endl ;
   G4cout << "#entries=" << entryTt << "    #underflows=" << underTt <<
             "    #overflows=" << overTt << G4endl ;
   if( entryTt>0.)
   {
     Ttmean /= entryTt;
     sig=Tt2mean/entryTt-Ttmean*Ttmean ;
     if(sig<=0.)
       sig=0.;
     else
       sig=std::sqrt(sig/entryTt) ;
     G4cout << " mean energy of transmitted particles=" << Ttmean/keV << 
               " +- " << sig/keV << "  keV." << G4endl;
     E = Ttlow - dTt ;
     norm = TotNbofEvents*dTt ;
     G4cout << " bin nb      Elow      entries     normalized " << G4endl ;
     for(G4int itt=0; itt<nbinTt; itt++)
     {
      E += dTt ;
      dnorm = distTt[itt]/norm;
      G4cout << std::setw(5) << itt << std::setw(10) << E << 
                std::setw(12) << distTt[itt] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }     
  }
  if(nbinTb>0)
  {
     G4double E , dnorm, norm ,sig;
   G4cout << " backscattered energy distribution " << G4endl ;
   G4cout << "#entries=" << entryTb << "    #underflows=" << underTb <<
             "    #overflows=" << overTb << G4endl ;
   if( entryTb>0.)
   {
     Tbmean /= entryTb;
     sig=Tb2mean/entryTb-Tbmean*Tbmean ;
     if(sig<=0.)
       sig=0.;
     else
       sig=std::sqrt(sig/entryTb) ;
     G4cout << " mean energy of backscattered particles=" << Tbmean/keV << 
               " +- " << sig/keV << "  keV." << G4endl;
     E = Tblow - dTb ;
     norm = TotNbofEvents*dTb ;
     G4cout << " bin nb      Elow      entries     normalized " << G4endl ;
     for(G4int itt=0; itt<nbinTb; itt++)
     {
      E += dTb ;
      dnorm = distTb[itt]/norm;
      G4cout << std::setw(5) << itt << std::setw(10) << E << 
                std::setw(12) << distTb[itt] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }     
  }
  if(nbinTsec>0)
  {G4double E , dnorm, norm ;
   G4cout << " energy distribution of charged secondaries " << G4endl ;
   G4cout << "#entries=" << entryTsec << "    #underflows=" << underTsec <<
             "    #overflows=" << overTsec << G4endl ;
   if( entryTsec>0.)
   {
     E = Tseclow - dTsec ;
     norm = TotNbofEvents*dTsec ;
     G4cout << " bin nb      Elow      entries     normalized " << G4endl ;
     for(G4int itt=0; itt<nbinTsec; itt++)
     {
      E += dTsec ;
      dnorm = distTsec[itt]/norm;
      G4cout << std::setw(5) << itt << std::setw(10) << E << 
                std::setw(12) << distTsec[itt] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }     
  }

  if(nbinR >0)
  {G4double R , dnorm, norm,sig  ;
   G4cout << "  R  distribution " << G4endl ;
   G4cout << "#entries=" << entryR  << "    #underflows=" << underR  <<
             "    #overflows=" << overR  << G4endl ;
   if( entryR >0.)
   {
     Rmean /= entryR;
     sig = R2mean/entryR - Rmean*Rmean;
     if(sig<=0.) sig=0. ;
     else        sig = std::sqrt(sig/entryR) ;
     G4cout << " mean lateral displacement at exit=" << Rmean/mm << " +- "
            << sig/mm << "  mm." << G4endl ; 
     R = Rlow - dR  ;
     norm = TotNbofEvents*dR  ;
     G4cout << " bin nb      Rlow      entries     normalized " << G4endl ;
     for(G4int ier=0; ier<nbinR ; ier++)
     {
      R+= dR  ;
      dnorm = distR[ier]/norm;
      G4cout << std::setw(5) << ier << std::setw(10) << R  <<
                std::setw(12) << distR[ier] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }
  }

  if(nbinTh>0)
  {G4double Th,Thdeg, dnorm, norm,fac0,fnorm,pere,Thpere,Thmean,sum;
   G4cout << "      angle   distribution " << G4endl ;
   G4cout << "#entries=" << entryTh << "    #underflows=" << underTh <<
             "    #overflows=" << overTh << G4endl ;
   if( entryTh>0.)
   {
     Th= Thlow - dTh ;
     norm = TotNbofEvents ;
     if(distTh[0] == 0.)
       fac0 = 1. ;
     else
       fac0 = 1./distTh[0] ;
     pere = 1./std::exp(1.) ;

     G4cout << " bin nb  Thlowdeg      Thlowrad      " <<
               " entries         normalized " << G4endl ;
     Thpere = 0. ;
     sum = 0. ;
     Thmean = 0. ;
     for(G4int ien=0; ien<nbinTh; ien++)
     {
      Th+= dTh ;
      Thdeg = Th*180./pi ;
      sum += distTh[ien] ;
      Thmean += distTh[ien]*(Th+0.5*dTh) ;
      dnorm = distTh[ien]/norm;
      fnorm = fac0*distTh[ien] ;
      if( fnorm > pere)
        Thpere = Th ; 
      G4cout << std::setw(5) << ien << std::setw(10) << Thdeg << "   " <<
                std::setw(10) << Th << "  " <<   
                std::setw(12) << distTh[ien] << "  " <<
                std::setw(12) << dnorm << "  " << std::setw(12) << fnorm <<G4endl ;
     }
     Thmean /= sum ;
     G4cout << G4endl;
     G4cout << " mean = " << Thmean << "  rad  or " << 180.*Thmean/pi <<
               " deg." << G4endl;
     G4cout << " theta(1/e)=" << Thpere << " - " << Thpere+dTh << " rad   "
            << " or " << 180.*Thpere/pi << " - " << 180.*(Thpere+dTh)/pi 
            << " deg." << G4endl;
     G4cout << G4endl;
   }
  }

  if(nbinThback>0)
  {G4double Thb,Thdegb, dnormb, normb,fac0b,fnormb,pereb,Thpereb,Thmeanb,sumb;
   G4cout << " backscattering angle   distribution " << G4endl ;
   G4cout << "#entries=" << entryThback << "    #underflows=" << underThback <<
             "    #overflows=" << overThback << G4endl ;
   if( entryThback>0.)
   {
     Thb= Thlowback - dThback ;
     normb = TotNbofEvents ;
     if(distThback[0] == 0.)
       fac0b = 1. ;
     else
       fac0b = 1./distThback[0] ;
     pereb = 1./std::exp(1.) ;

     G4cout << " bin nb  Thlowdeg      Thlowrad      " <<
               " entries         normalized " << G4endl ;
     Thpereb = 0. ;
     sumb = 0. ;
     Thmeanb = 0. ;
     for(G4int ien=0; ien<nbinThback; ien++)
     {
      Thb+= dThback ;
      Thdegb = Thb*180./pi ;
      sumb += distThback[ien] ;
      Thmeanb += distThback[ien]*(Thb+0.5*dThback) ;
      dnormb = distThback[ien]/normb;
      fnormb = fac0b*distThback[ien] ;
      if( fnormb > pereb)
        Thpereb = Thb ;
      G4cout << std::setw(5) << ien << std::setw(10) << Thdegb << "   " <<
                std::setw(10) << Thb << "  " <<
                std::setw(12) << distThback[ien] << "  " <<
                std::setw(12) << dnormb << "  " << std::setw(12) << fnormb <<G4endl ;
     }
     Thmeanb /= sumb ;
     G4cout << G4endl;
     G4cout << " mean = " << Thmeanb << "  rad  or " << 180.*Thmeanb/pi <<
               " deg." << G4endl;
     G4cout << " theta(1/e)=" << Thpereb << " - " << Thpereb+dThback << " rad   "
            << " or " << 180.*Thpereb/pi << " - " << 180.*(Thpereb+dThback)/pi
            << " deg." << G4endl;
     G4cout << G4endl;
   }
  }

  if(nbinGamma>0)
  {G4double E , fact,dnorm, norm  ;
   G4cout << " gamma energy distribution " << G4endl ;
   G4cout << "#entries=" << entryGamma << "    #underflows=" << underGamma <<
             "    #overflows=" << overGamma << G4endl ;
   if( entryGamma>0.)
   {
     fact=std::exp(dEGamma) ;
     E = ElowGamma/fact  ;
     norm = TotNbofEvents*dEGamma;
     G4cout << " bin nb         Elow      entries       normalized " << G4endl ;
     for(G4int itt=0; itt<nbinGamma; itt++)
     {
      E *= fact ;
      dnorm = distGamma[itt]/norm;
      G4cout << std::setw(5) << itt << std::setw(13) << E << 
                std::setw(12) << distGamma[itt] <<
                std::setw(15) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }     
  }

  if(nbinvertexz >0)
  {G4double z , dnorm, norm  ;
   G4cout << " vertex Z  distribution " << G4endl ;
   G4cout << "#entries=" << entryvertexz  << "    #underflows=" << undervertexz  <<
             "    #overflows=" << oververtexz  << G4endl ;
   if( entryvertexz >0.)
   {
     z =zlow - dz  ;
     norm = TotNbofEvents*dz  ;
     G4cout << " bin nb      zlow      entries     normalized " << G4endl ;
     for(G4int iez=0; iez<nbinvertexz ; iez++)
     {
      z+= dz  ;
      if(std::fabs(z)<1.e-12) z=0.;
      dnorm = distvertexz[iez]/norm;
      G4cout << std::setw(5) << iez << std::setw(10) << z  <<
                std::setw(12) << distvertexz[iez] <<
                std::setw(12) << dnorm << G4endl ;
     }
     G4cout << G4endl;
   }
  }
  
  G4cout.precision(prec);
  
  if (G4VVisManager::GetConcreteInstance())
  {
    G4UImanager::GetUIpointer()->ApplyCommand("/vis/viewer/update");
  }

  // save Rndm status

  if (saveRndm == 1)
  { 
    CLHEP::HepRandom::showEngineStatus();
    CLHEP::HepRandom::saveEngineStatus("endOfRun.rndm");
  }     
}

///////////////////////////////////////////////////////////////////////////

void Em10RunAction::CountEvent()
{
  TotNbofEvents += 1. ;
}

/////////////////////////////////////////////////////////////////////////

void Em10RunAction::AddnStepsCharged(G4double nstp)
{
  nStepSumCharged += nstp;
  nStepSum2Charged += nstp*nstp;
}

////////////////////////////////////////////////////////////////////////

void Em10RunAction::AddnStepsNeutral(G4double nstp)
{
  nStepSumNeutral += nstp;
  nStepSum2Neutral += nstp*nstp;
}

////////////////////////////////////////////////////////////////////////////

void Em10RunAction::AddEdeps(G4double Eabs)
{
  EnergySumAbs += Eabs;
  EnergySquareSumAbs += Eabs*Eabs;
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::AddTrackLength(G4double tlabs)
{
  tlSumAbs += tlabs;
  tlsquareSumAbs += tlabs*tlabs ;
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::AddTrRef(G4double tr,G4double ref)
{
  Transmitted += tr ;
  Reflected   += ref;
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillNbOfSteps(G4double)// ns)
{
}

//////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillEn(G4double En)
{
  G4double bin ;
  G4int ibin;

  entryEn += 1. ;
  if(Enlow < Enhigh) {

    if(En < Enlow)          underEn += 1. ;
    else if( En >= Enhigh)  overEn  += 1. ;
    else
    {
      bin = (En-Enlow)/dEn;
      ibin= (G4int)bin;
      if(ibin < 0) { ibin = 0; }
      if(ibin > 199) { ibin = 199; }
      distEn[ibin] += 1.;
    }
  }
}

////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillTt(G4double) // En)
{
}

//////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillTb(G4double) // En)
{
}

///////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillTsec(G4double) // En)
{
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillGammaSpectrum(G4double) // En)
{
}

////////////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillTh(G4double) // Th)
{
}

//////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillThBack(G4double) // Th)
{
}

//////////////////////////////////////////////////////////////////////////

void Em10RunAction::FillR(G4double) // R )
{
}

/////////////////////////////////////////////////////////////////////////////

void Em10RunAction::Fillvertexz(G4double) // z )
{
}

//////////////////////////////////////////////////////////////////////////////

void Em10RunAction::SethistName(G4String name)
{
  histName = name ;
  G4cout << " hist file = " << histName << G4endl;
}

void Em10RunAction::SetnbinStep(G4int nbin)
{
  if(nbin> 0 && nbin<= 200) {
    nbinStep = nbin;
    G4cout << " Nb of bins in #step plot = " << nbinStep << G4endl ;
  }
}

void Em10RunAction::SetSteplow(G4double low)
{
  if(low >= 0.0){ 
    Steplow = low ;
    G4cout << " low  in the #step plot = " << Steplow << G4endl ;
  }
}
void Em10RunAction::SetStephigh(G4double high)
{
  if(high > 0.0) {
    Stephigh = high;
    G4cout << " high in the #step plot = " << Stephigh << G4endl ;
  }
}

////////////////////////////////////////////////////////////////////////

void Em10RunAction::SetnbinEn(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinEn = nbin;
    G4cout << " Nb of bins in Edep plot = " << nbinEn << G4endl ;
  }
}

void Em10RunAction::SetEnlow(G4double Elow)
{
  if(Elow >= 0.0) {
    Enlow = Elow ;
    G4cout << " Elow  in the  Edep plot = " << Enlow << G4endl ;
  }
}

void Em10RunAction::SetEnhigh(G4double Ehigh)
{
  if(Ehigh > 0.0) {
    Enhigh = Ehigh ;
    G4cout << " Ehigh in the  Edep plot = " << Enhigh << G4endl ;
  }
}

/////////////////////////////////////////////////////////////////////////

void Em10RunAction::SetnbinGamma(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinGamma = nbin;
    G4cout << " Nb of bins in gamma spectrum plot = " << nbinGamma << G4endl ;
  }
}

void Em10RunAction::SetElowGamma(G4double Elow)
{
  if(Elow >= 0.0) {
    ElowGamma = Elow;
    G4cout << " Elow  in the gamma spectrum plot = " << ElowGamma << G4endl ;
  }
}

void Em10RunAction::SetEhighGamma(G4double Ehigh)
{
  if(Ehigh > 0.0) {
    EhighGamma = Ehigh;
    G4cout << " Ehigh in the gamma spectrum plot = " << EhighGamma << G4endl ;
  }
}

void Em10RunAction::SetnbinTt(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinTt = nbin;
    G4cout << " Nb of bins in Etransmisssion plot = " << nbinTt << G4endl ;
  }
}

void Em10RunAction::SetTtlow(G4double Elow)
{
  if(Elow >= 0.0) {
    Ttlow = Elow;
    G4cout << " Elow  in the  Etransmission plot = " << Ttlow << G4endl ;
  }
}

void Em10RunAction::SetTthigh(G4double Ehigh)
{
  if(Ehigh > 0.0) {
    Tthigh = Ehigh;
    G4cout << " Ehigh in the  Etransmission plot = " << Tthigh << G4endl ;
  }
}

void Em10RunAction::SetnbinTb(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinTb = nbin;
    G4cout << " Nb of bins in Ebackscattered plot = " << nbinTb << G4endl ;
  }
}

void Em10RunAction::SetTblow(G4double Elow)
{

  Tblow = Elow ;
  G4cout << " Elow  in the  Ebackscattered plot = " << Tblow << G4endl ;
}

void Em10RunAction::SetTbhigh(G4double Ehigh)
{
  Tbhigh = Ehigh ;
  G4cout << " Ehigh in the  Ebackscattered plot = " << Tbhigh << G4endl ;
}

void Em10RunAction::SetnbinTsec(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinTsec = nbin;
    G4cout << " Nb of bins in Tsecondary  plot = " << nbinTsec << G4endl ;
  }
}

void Em10RunAction::SetTseclow(G4double Elow)
{
  Tseclow = Elow ;
  G4cout << " Elow  in the  Tsecondary plot = " << Tseclow << G4endl ;
}

void Em10RunAction::SetTsechigh(G4double Ehigh)
{
  Tsechigh = Ehigh ;
  G4cout << " Ehigh in the  Tsecondary plot = " << Tsechigh << G4endl ;
}
 
void Em10RunAction::SetnbinR(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinR  = nbin;
    G4cout << " Nb of bins in R plot = " << nbinR << G4endl ;
  }
}

void Em10RunAction::SetRlow(G4double rlow)
{
  Rlow = rlow ;
  G4cout << " Rlow  in the  R plot = " << Rlow << G4endl ;
}

void Em10RunAction::SetRhigh(G4double rhigh)
{
  Rhigh = rhigh ;
  G4cout << " Rhigh in the R plot = " << Rhigh << G4endl ;
}

void Em10RunAction::Setnbinzvertex(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinvertexz  = nbin;
    G4cout << " Nb of bins in Z plot = " << nbinvertexz << G4endl ;
  }
}

void Em10RunAction::Setzlow(G4double z)
{
  zlow = z ;
  G4cout << " zlow  in the  Z plot = " << zlow << G4endl ;
}

void Em10RunAction::Setzhigh(G4double z)
{
  zhigh = z ;
  G4cout << " zhigh in the Z plot = " << zhigh << G4endl ;
}

void Em10RunAction::SetnbinTh(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinTh = nbin;
    G4cout << " Nb of bins in Theta plot = " << nbinTh << G4endl ;
  }
}

void Em10RunAction::SetThlow(G4double Tlow)
{
  Thlow = Tlow ;
  G4cout << " Tlow  in the  Theta plot = " << Thlow << G4endl ;
}

void Em10RunAction::SetThhigh(G4double Thigh)
{
  Thhigh = Thigh ;
  G4cout << " Thigh in the Theta plot = " << Thhigh << G4endl ;
}

void Em10RunAction::SetnbinThBack(G4int nbin)
{
  if(nbin > 0 && nbin <= 200) {
    nbinThback = nbin;
    G4cout << " Nb of bins in Theta plot = " << nbinThback << G4endl ;
  }
}

void Em10RunAction::SetThlowBack(G4double Tlow)
{
  Thlowback = Tlow ;
  G4cout << " Tlow  in the  Theta plot = " << Thlowback << G4endl ;
}

void Em10RunAction::SetThhighBack(G4double Thigh)
{
  Thhighback = Thigh ;
  G4cout << " Thigh in the Theta plot = " << Thhighback << G4endl ;
}

void Em10RunAction::CountParticles(G4double nch,G4double nne)
{
  SumCharged += nch ;
  SumNeutral += nne ;
  Sum2Charged += nch*nch ;
  Sum2Neutral += nne*nne ;
}

void Em10RunAction::AddEP(G4double nele,G4double npos) 
{
  Selectron += nele;
  Spositron += npos;
}

//
//
////////////////////////////////////////////////////////////////////////
