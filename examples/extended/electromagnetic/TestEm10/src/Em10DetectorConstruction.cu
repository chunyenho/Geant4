#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10DetectorConstruction.cc
/// \brief Implementation of the Em10DetectorConstruction class
//
//
// $Id: Em10DetectorConstruction.cc 73033 2013-08-15 09:24:45Z gcosmo $
//
//

#include "Em10DetectorConstruction.hh"
#include "Em10DetectorMessenger.hh"
#include "Em10CalorimeterSD.hh"
#include "Em10Materials.hh"

#include "G4Material.hh"
#include "G4Box.hh"
#include "G4LogicalVolume.hh"
#include "G4PVPlacement.hh"
#include "G4UniformMagField.hh"
#include "G4FieldManager.hh"
#include "G4TransportationManager.hh"
#include "G4SDManager.hh"
#include "G4GeometryManager.hh"
#include "G4RunManager.hh"

#include "G4Region.hh"
#include "G4RegionStore.hh"
#include "G4PhysicalVolumeStore.hh"
#include "G4LogicalVolumeStore.hh"
#include "G4SolidStore.hh"
#include "G4ProductionCuts.hh"

#include "G4VisAttributes.hh"
#include "G4Colour.hh"

#include "G4UnitsTable.hh"
#include "G4SystemOfUnits.hh"
#include "G4ios.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10DetectorConstruction::Em10DetectorConstruction()
  :G4VUserDetectorConstruction(),
  fWorldChanged(false), fAbsorberMaterial(0), fGapMat(0), fSetUp("simpleALICE"),
  fWorldMaterial(0), fSolidWorld(0), fLogicWorld(0), fPhysicsWorld(0),
//   fSolidRadSlice(0), fLogicRadSlice(0),  fPhysicRadSlice(0),
   fSolidRadiator(0),  fLogicRadiator(0),   fPhysicsRadiator(0),
   fRadiatorMat(0), fPipe(false), fPipeField(false),
   fSolidAbsorber(0),  fLogicAbsorber(0),   fPhysicsAbsorber(0),
   fMagField(0),       fCalorimeterSD(0),   fRegGasDet(0),
   fRadRegion(0), fMat(0)
{
  fDetectorMessenger = new Em10DetectorMessenger(this);
  fMat               = new Em10Materials();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10DetectorConstruction::~Em10DetectorConstruction()
{
  delete fDetectorMessenger;
  delete fMat;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4VPhysicalVolume* Em10DetectorConstruction::Construct()
{
  return ConstructDetectorXTR();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

G4VPhysicalVolume* Em10DetectorConstruction::ConstructDetectorXTR()
{
 // Cleanup old geometry

  G4GeometryManager::GetInstance()->OpenGeometry();
  G4PhysicalVolumeStore::GetInstance()->Clean();
  G4LogicalVolumeStore::GetInstance()->Clean();
  G4SolidStore::GetInstance()->Clean();

  if( fSetUp == "simpleALICE" )
  {
    return SimpleSetUpALICE();
  }
  else if( fSetUp == "alice06" )
  {
    return SetUpALICE06();
  }
  else if( fSetUp == "bari05" )
  {
    return SetUpBari05();
  }
  else if( fSetUp == "harris73" )
  {
    return SetUpHarris73();
  }
  else if( fSetUp == "watase86" )
  {
    return SetUpWatase86();
  }
  else if( fSetUp == "barr90" )
  {
    return SetUpBarr90();
  }
  else
  {
    G4cout <<
    "Experimental setup is unsupported. Check /XTRdetector/setup " <<G4endl;
    G4cout<<"Run default: barr90 "<<G4endl;
    return SetUpBarr90();

    //  return 0;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Simplified setup for ALICE XTR test beam (~2004). 
// Runs by : TestEm10 salice.mac

G4VPhysicalVolume* Em10DetectorConstruction::SimpleSetUpALICE()
{
  fWorldSizeZ = 400.*cm;
  fWorldSizeR = 20.*cm;

  // Radiator and detector parameters

  fRadThickness = 0.020*mm;
  fGasGap       = 0.250*mm;  
  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fFoilNumber   = 220;

  fAbsorberThickness = 38.3*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 10.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  40.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* Mylar = fMat->GetMaterial("Mylar");
  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* Al   = fMat->GetMaterial("Al");

  G4double foilDensity =  1.39*g/cm3;
  // Mylar // 0.91*g/cm3;  // CH2 0.534*g/cm3; //Li
  G4double gasDensity  =  1.2928*mg/cm3;
  // Air // 1.977*mg/cm3; // CO2 0.178*mg/cm3; //He
  G4double totDensity  = foilDensity*foilGasRatio + 
                                             gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;
 
  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( Mylar, fractionFoil );
  radiatorMat->AddMaterial( Air, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat =  radiatorMat;
  fFoilMat     = Mylar; // CH2; // Kapton; // Mylar ; // Li ; // CH2 ;
  fGasMat      = Air; // CO2; // He; //
  
  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Xe15CO2");
 
  fGapMat          = fAbsorberMaterial;

  fWorldMaterial    = Air; // CO2;

  fSolidWorld = new G4Box("World", fWorldSizeR,fWorldSizeR,fWorldSizeZ/2.);
 
  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");
 
  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for window inside windowR for

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);

  fWindowZ = fStartZ + fRadThick + fWindowThick/2. + 15.0*mm;

  //  G4Box* solidWindowR = new G4Box("WindowR",fAbsorberRadius+0.001,
  //                                        fAbsorberRadius+0.001,
  //                                        fWindowThick/2.+0.001  );

  //  G4LogicalVolume* logicWindowR = new G4LogicalVolume(solidWindowR,
  //                                   fWorldMaterial, "WindowR");
 
  //  G4VPhysicalVolume*    physiWindowR = new G4PVPlacement(0,
  //                      G4ThreeVector(0.,0.,fWindowZ),
  //                      "WindowR",logicWindowR,fPhysicsWorld,false,0);
  // window

  //  G4Box* solidWindow = new G4Box("Window",fAbsorberRadius,
  //                                 fAbsorberRadius, fWindowThick/2.);
 
  //  G4LogicalVolume* logicWindow = new G4LogicalVolume(solidWindow,
  //                                   fWindowMat, "Window");

  //  G4VPhysicalVolume* physiWindow = 
  //                         new G4PVPlacement(0, G4ThreeVector(0.,0.,0.),
  //                         "Window", logicWindow, physiWindowR, false, 0); 

  fGapZ = fWindowZ + fWindowThick/2. + fGapThick/2. + 0.01*mm;

  fElectrodeZ = fGapZ + fGapThick/2. + fElectrodeThick/2. + 0.01*mm;

  // Absorber

  fAbsorberZ = fElectrodeZ + fElectrodeThick/2. + 
                  fAbsorberThickness/2. + 0.01*mm;

  fSolidAbsorber = new G4Box("Absorber", fAbsorberRadius,
                                 fAbsorberRadius, fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");

  fPhysicsAbsorber = new G4PVPlacement(0, G4ThreeVector(0.,0.,fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Setup for ALICE XTR test beam (~2004). With He beam-pipe
// Runs by : TestEm10 alice06.mac

G4VPhysicalVolume* Em10DetectorConstruction::SetUpALICE06()
{
  fWorldSizeZ = 600.*cm;
  fWorldSizeR = 22.*cm;

  // Radiator and detector parameters

  //fRadThickness = 0.01*mm;    // Gamma XTR (malz: 0.01)
  //fGasGap       = 0.19*mm;    // Gamma XTR (malz: 0.09)
  //fFoilNumber   = 240;        // Gamma XTR (malz: 480)

  fRadThickness = 0.020*mm;  // Reg1
  fGasGap       = 0.500*mm;  // Reg1
  fFoilNumber   = 120;       // Reg1

  //fRadThickness = 0.013*mm;  // Anton
  //fGasGap       = 0.060*mm;  // Anton
  //fFoilNumber   = 550;       // Anton

  // fRadThickness = 0.020*mm; // Reg2
  // fGasGap       = 0.250*mm; // Reg2 
  // fFoilNumber   = 220;      // Reg2

  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fAbsorberThickness = 37.*mm; // 38.3*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fPipeLength     = 160.0*cm;
  fMylarThick     = 20.0*micrometer;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 100.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  40.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* Mylar = fMat->GetMaterial("Mylar");
  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* Al   = fMat->GetMaterial("Al");
  G4Material* CH2   = fMat->GetMaterial("CH2");
  G4Material* He   = fMat->GetMaterial("He");

  G4double foilDensity = CH2->GetDensity();
  G4double gasDensity  = Air->GetDensity();  
  G4double totDensity  = foilDensity*foilGasRatio + 
                                              gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  1.0 - fractionFoil;
// gasDensity*(1.0-foilGasRatio)/totDensity ;  
    
  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( CH2, fractionFoil );
  radiatorMat->AddMaterial( Air, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat = radiatorMat;
  fFoilMat     = CH2; // Kapton; // Mylar ; // Li ; // CH2 ;
  fGasMat      = Air; // CO2; // He; //

  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Xe15CO2");

  // pipe material is assumed to be He + small admixture of air
  /* 
  foilGasRatio = 0.000001;
  foilDensity  = Air->GetDensity();
  gasDensity   = He->GetDensity();
  totDensity   = foilDensity*foilGasRatio + gasDensity*( 1.0 - foilGasRatio );

  fractionFoil =  foilDensity*foilGasRatio/totDensity;
  fractionGas  =  1.0 - fractionFoil;
  // gasDensity*(1.0 - foilGasRatio)/totDensity;  

  fPipeMat = new G4Material("pipeMat"  , totDensity,  2);
  fPipeMat->AddMaterial( Air, fractionFoil );
  fPipeMat->AddMaterial( He,  fractionGas  );
  */
  fPipeMat = He;

  fGapMat           = fAbsorberMaterial;

  fWorldMaterial    = Air;

  fSolidWorld = new G4Box("World", fWorldSizeR, fWorldSizeR, fWorldSizeZ/2.);

  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;

  // fRadZ = -fRadThick/2. - fElectrodeThick;
  // if ( fabs(pipe) > 1.e-15 ) fRadZ -= ( fPipeLength/2. + pipeDist );

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");

  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for radiator

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);

  // Drift Electrode on both sides of Radiator:

  G4double zElectrode1 = fRadZ - fRadThick/2. - fElectrodeThick/2.;
  G4double zElectrode2 = fRadZ + fRadThick/2. + fElectrodeThick/2.;
  /*
  G4Box* solidElectrode = new G4Box("Electrode",fAbsorberRadius*1.1,
                                                fAbsorberRadius*1.1,
                                                fElectrodeThick/2.);

  G4LogicalVolume* logicElectrode = new G4LogicalVolume(solidElectrode,
                                                        fElectrodeMat,
                                                        "Electrode");

  G4VPhysicalVolume*    physiElectrode1 = new G4PVPlacement(0,
                                       G4ThreeVector(0.,0.,zElectrode1),
                                      "Electrode1",logicElectrode,
                                       fPhysicsWorld,false,0);

  G4VPhysicalVolume*    physiElectrode2 = new G4PVPlacement(0,
                                       G4ThreeVector(0.,0.,zElectrode2),
                                      "Electrode1",logicElectrode,
                                       fPhysicsWorld,false,0);
  */
  G4cout<<"zElectrode1 = "<<zElectrode1/mm<<" mm"<<G4endl;
  G4cout<<"zElectrode2 = "<<zElectrode2/mm<<" mm"<<G4endl;
  G4cout<<"fElectrodeThick = "<<fElectrodeThick/mm<<" mm"<<G4endl<<G4endl;

  // Helium Pipe:

  //Distance between pipe and radiator / absorber
  G4double pipeDist      = 1.*cm;
  G4double fieldStrength = 1.0*tesla;  // 0.01*tesla; // field strength in pipe
  G4double alphaB        = 90.*degree;
  fPipe     =  true;   // 0.;  //  use helium pipe is setup

  fPipeField     =  true;   // field in helium pipe used?

  G4double zPipe = zElectrode2 + fElectrodeThick/2. +
                   pipeDist/2. + fPipeLength/2.;

  if ( fPipe )
  {

    G4Box* solidPipe = new G4Box("Pipe",fAbsorberRadius*0.5,
                                 fAbsorberRadius*0.5,
                                 fPipeLength/2. );

    G4LogicalVolume* logicPipe = new G4LogicalVolume(solidPipe,
                                                     fPipeMat, //fWorldMaterial
                                                     "Pipe");

    //    G4VPhysicalVolume*    physiPipe = new G4PVPlacement(0,
    //                                 G4ThreeVector(0., 0., zPipe),
    //                                "Pipe1",logicPipe,
    //                                  fPhysicsWorld,false,0);

    G4cout<<"zPipe = "<<zPipe/mm<<" mm"<<G4endl;
    G4cout<<"fPipeLength = "<<fPipeLength/mm<<" mm"<<G4endl<<G4endl;

    // magnetic field in Pipe:

    if ( fPipeField )
    {
      if( fMagField ) delete fMagField; //delete the existing mag field

       fMagField =
           new G4UniformMagField(G4ThreeVector(fieldStrength*std::sin(alphaB), 
                                 0., fieldStrength*std::cos(alphaB)));
      // fMagField = new G4UniformMagField(G4ThreeVector(fieldStrength,0.,0.));
      // fMagField = new G4UniformMagField(G4ThreeVector(0.,0.,fieldStrength));
      G4FieldManager* fieldMgr = new G4FieldManager(fMagField);
      fieldMgr->SetDetectorField(fMagField);
      fieldMgr->CreateChordFinder(fMagField);
      logicPipe->SetFieldManager(fieldMgr, true);
    }

  }
  else   G4cout<<"No Helium pipe is used"<<G4endl<<G4endl;

  // Mylar Foil on both sides of helium pipe:

  G4double zMylar1 = zPipe - fPipeLength/2. - fMylarThick/2. - 0.001*mm;
  G4double zMylar2 = zPipe + fPipeLength/2. + fMylarThick/2. + 0.001*mm;

  //  G4Box* solidMylar = new G4Box("MylarB",fAbsorberRadius*0.6,
  //                              fAbsorberRadius*0.6,
  //                               fMylarThick/2.);

  //  G4LogicalVolume* logicMylar = new G4LogicalVolume(solidMylar,
  //                                                  fWindowMat,
  //                                                  "MylarL");

  if ( fPipe )
  {

    //    G4VPhysicalVolume* physiMylar1 = new G4PVPlacement(0,
    //                         G4ThreeVector( 0., 0., zMylar1),
    //                          "Mylar1", logicMylar, fPhysicsWorld,
    //                                      false, 0);

    //  G4VPhysicalVolume* physiMylar2 = new G4PVPlacement(0,
    //                             G4ThreeVector(0., 0., zMylar2),
    //                            "Mylar2", logicMylar, fPhysicsWorld,
    //                               false, 0);

      G4cout<<"zMylar1 = "<<zMylar1/mm<<" mm"<<G4endl;
      G4cout<<"zMylar2 = "<<zMylar2/mm<<" mm"<<G4endl;
      G4cout<<"fMylarThick = "<<fMylarThick/mm<<" mm"<<G4endl<<G4endl;
  }

  // Mylar Foil on Chamber:

  G4double zMylar = zElectrode2 + fElectrodeThick/2. + fMylarThick/2. + 1.0*mm;

  // if ( fPipe )
  {
    zMylar += ( fPipeLength + pipeDist );
  }
  //  G4VPhysicalVolume*    physiMylar = new G4PVPlacement(0,
  //                       G4ThreeVector(0., 0., zMylar),
  //                      "Mylar",logicMylar,fPhysicsWorld,false,0);

  G4cout<<"zMylar = "<<zMylar/mm<<" mm"<<G4endl;
  G4cout<<"fMylarThick = "<<fMylarThick/mm<<" mm"<<G4endl<<G4endl;

  // Absorber

  fAbsorberZ = zMylar + fMylarThick + fAbsorberThickness/2.;

  fSolidAbsorber = new G4Box("Absorber",
                             fAbsorberRadius,
                             // fAbsorberRadius,
                             // 10.*mm,
                             10.*mm,
                             fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");

  fPhysicsAbsorber = new G4PVPlacement(0,
                         G4ThreeVector(0., 0., fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Setup for Bari INFN XTR test beam (~2004) at CERN. With He beam-pipe
// M. Brigida et al, NIM A550 (2005) 157-168
// Runs by : TestEm10 bari05.mac

G4VPhysicalVolume* Em10DetectorConstruction::SetUpBari05()
{
  fWorldSizeZ = 600.*cm;
  fWorldSizeR = 22.*cm;

  // Radiator and detector parameters

  //fRadThickness = 0.01*mm;    // Gamma XTR (malz: 0.01)
  //fGasGap       = 0.19*mm;    // Gamma XTR (malz: 0.09)
  //fFoilNumber   = 240;        // Gamma XTR (malz: 480)

  //fRadThickness = 0.020*mm;  // Reg1
  //fGasGap       = 0.500*mm;  // Reg1
  //fFoilNumber   = 120;       // Reg1

  //fRadThickness = 0.013*mm;  // Anton
  //fGasGap       = 0.230*mm;  // Anton
  //fFoilNumber   = 550;       // Anton

  fRadThickness = 0.0055*mm; // Reg2
  fGasGap       = 0.23*mm; // Reg2
  fFoilNumber   = 191;      // Reg2

  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fAbsorberThickness = 0.4*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fPipeLength = 50.0*cm;
  fMylarThick     = 20.0*micrometer;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 100.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  40.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* Mylar = fMat->GetMaterial("Mylar");
  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* Al   = fMat->GetMaterial("Al");
  G4Material* CH2   = fMat->GetMaterial("CH2");
  G4Material* He   = fMat->GetMaterial("He");

  G4double foilDensity =  0.91*g/cm3;  
  // CH2 1.39*g/cm3; // Mylar //  0.534*g/cm3; //Li
  G4double gasDensity  =  1.2928*mg/cm3;
  // Air // 1.977*mg/cm3; // CO2 0.178*mg/cm3; // He
  G4double totDensity  = foilDensity*foilGasRatio + 
                                           gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;

  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( CH2, fractionFoil );
  radiatorMat->AddMaterial( Air, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat = radiatorMat;
  fFoilMat     = CH2; // Kapton; // Mylar ; // Li ; // CH2 ;
  fGasMat      = Air; // CO2; // He; //

  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Si");

  // pipe material is assumed to be He + small admixture of air

  foilGasRatio = 0.99999;
  foilDensity  = 1.2928*mg/cm3; // Air
  gasDensity   = 0.178*mg/cm3; // He
  totDensity   = foilDensity*foilGasRatio + gasDensity*(1.0-foilGasRatio);

  fractionFoil =  foilDensity*foilGasRatio/totDensity;
  fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;

  fPipeMat = new G4Material("pipeMat"  , totDensity,  2);
  fPipeMat->AddMaterial( Air, fractionFoil );
  fPipeMat->AddMaterial( He,  fractionGas  );

  // fPipeMat = He;

  fGapMat           = fAbsorberMaterial;

  fWorldMaterial    = Air;

  fSolidWorld = new G4Box("World", fWorldSizeR,fWorldSizeR,fWorldSizeZ/2.);

  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;
  // fRadZ = -fRadThick/2. - fElectrodeThick;
  // if ( fabs(pipe) > 1.e-15 ) fRadZ -= ( fPipeLength/2. + pipeDist );

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");

  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for radiator

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);

  // Drift Electrode on both sides of Radiator:

  //  G4Box* solidElectrode = new G4Box("Electrode",fAbsorberRadius*1.1,
  //                                            fAbsorberRadius*1.1,
  //                                             fElectrodeThick/2.);

  //  G4LogicalVolume* logicElectrode = new G4LogicalVolume(solidElectrode,
  //                                                       fElectrodeMat,
  //                                                        "Electrode");

  G4double zElectrode1 = fRadZ - fRadThick/2. - fElectrodeThick/2.;
  G4double zElectrode2 = fRadZ + fRadThick/2. + fElectrodeThick/2.;

  //  G4VPhysicalVolume*    physiElectrode1 = new G4PVPlacement(0,
  //                                       G4ThreeVector(0.,0.,zElectrode1),
  //                                     "Electrode1",logicElectrode,
  //                                      fPhysicsWorld,false,0);

  // G4VPhysicalVolume*    physiElectrode2 = new G4PVPlacement(0,
  //                                      G4ThreeVector(0.,0.,zElectrode2),
  //                                    "Electrode1",logicElectrode,
  //                                     fPhysicsWorld,false,0);

  G4cout<<"zElectrode1 = "<<zElectrode1/mm<<" mm"<<G4endl;
  G4cout<<"zElectrode2 = "<<zElectrode2/mm<<" mm"<<G4endl;
  G4cout<<"fElectrodeThick = "<<fElectrodeThick/mm<<" mm"<<G4endl<<G4endl;

  // Helium Pipe:

  G4double pipe     = 1.0;   // use helium pipe is setup

  G4double pipeDist = 1.*cm;  //Distance between pipe and radiator / absorber

  G4double zPipe = zElectrode2 + fElectrodeThick/2. + 
                                                fPipeLength/2. + pipeDist/2.;

  // G4double field         = 1.0;   // field in helium pipe used?
  // G4double fieldStrength = 1.0*tesla;  // field strength in pipe

  if ( std::fabs(pipe) > 1.e-15 )
  {

    //    G4Box* solidPipe = new G4Box("Pipe",fAbsorberRadius*0.5,
    //                              fAbsorberRadius*0.5,
    //                              fPipeLength/2. );

    //    G4LogicalVolume* logicPipe = new G4LogicalVolume(solidPipe,
    //                                                  fPipeMat,
    //                                                  "Pipe");

    // magnetic field in Pipe:
    // if( fMagField ) delete fMagField; //delete the existing mag field
    // fMagField = new G4UniformMagField(G4ThreeVector(fieldStrength,0.,0.));
    // G4FieldManager* fieldMgr= new G4FieldManager(fMagField);
    // fieldMgr->SetDetectorField(fMagField);
    // fieldMgr->CreateChordFinder(fMagField);
    // if ( fabs(field) > 1.e-15 ) logicPipe->SetFieldManager(fieldMgr, true);

    //    G4VPhysicalVolume*    physiPipe = new G4PVPlacement(0,
    //                                   G4ThreeVector(0.,0.,zPipe),
    //                                  "Pipe1",logicPipe,
    //                                   fPhysicsWorld,false,0);

    G4cout<<"zPipe = "<<zPipe/mm<<" mm"<<G4endl;
    G4cout<<"fPipeLength = "<<fPipeLength/mm<<" mm"<<G4endl<<G4endl;

  }
  else   G4cout<<"No Helium pipe is used"<<G4endl<<G4endl;

  // Mylar Foil on both sides of helium pipe:

  G4double zMylar1 = zPipe - fPipeLength/2. - fMylarThick/2 - 0.01*mm;
  G4double zMylar2 = zPipe + fPipeLength/2. + fMylarThick/2 + 0.01*mm;

  //  G4Box* solidMylar = new G4Box("Mylar",fAbsorberRadius*0.6,
  //                              fAbsorberRadius*0.6,
  //                              fMylarThick/2.);

  //  G4LogicalVolume* logicMylar = new G4LogicalVolume(solidMylar,
  //                                                  fWindowMat,
  //                                                  "Mylar");

  if ( std::fabs(pipe) > 1.e-15 )
  {

    //    G4VPhysicalVolume* physiMylar1 = new G4PVPlacement(0,
    //                           G4ThreeVector( 0., 0., zMylar1),
    //                            "Mylar1", logicMylar, fPhysicsWorld,
    //                                        false, 0);

    //  G4VPhysicalVolume* physiMylar2 = new G4PVPlacement(0,
    //                             G4ThreeVector(0.,0.,zMylar2),
    //                             "Mylar2", logicMylar, fPhysicsWorld,
    //                                false, 0);

      G4cout<<"zMylar1 = "<<zMylar1/mm<<" mm"<<G4endl;
      G4cout<<"zMylar2 = "<<zMylar2/mm<<" mm"<<G4endl;
      G4cout<<"fMylarThick = "<<fMylarThick/mm<<" mm"<<G4endl<<G4endl;

  }

  // Mylar Foil on Chamber:

  G4double zMylar = zElectrode2 + fElectrodeThick/2. + fMylarThick/2. + 1.0*mm;

  if ( std::fabs(pipe) > 1.e-15 ) zMylar += ( fPipeLength + pipeDist );

  //  G4VPhysicalVolume*    physiMylar = new G4PVPlacement(0,
  //                       G4ThreeVector(0.,0.,zMylar),
  //                      "Mylar",logicMylar,fPhysicsWorld,false,0);

  G4cout<<"zMylar = "<<zMylar/mm<<" mm"<<G4endl;
  G4cout<<"fMylarThick = "<<fMylarThick/mm<<" mm"<<G4endl<<G4endl;

  // Absorber

  fAbsorberZ = zMylar + fMylarThick/2. + fAbsorberThickness/2.;

  fSolidAbsorber = new G4Box("Absorber",
                             // fAbsorberRadius, fAbsorberRadius,
                             10.*mm,10.*mm,
                              fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");

  fPhysicsAbsorber = new G4PVPlacement(0, G4ThreeVector(0.,0.,fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");  
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Setuo from F. Harris et al NIM 107 (1973) 413-422 (fig.5b)

G4VPhysicalVolume* Em10DetectorConstruction::SetUpHarris73()
{
  fWorldSizeZ = 400.*cm;
  fWorldSizeR = 20.*cm;

  // Radiator and detector parameters

  fRadThickness = 0.0127*mm;
  fGasGap       = 0.762*mm;
  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fFoilNumber   = 100;

  fAbsorberThickness = 15.0*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 10.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  40.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* Mylar = fMat->GetMaterial("Mylar");
  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* Al   = fMat->GetMaterial("Al");

  G4double foilDensity =  1.39*g/cm3;
  // Mylar // 0.91*g/cm3;  // CH2 0.534*g/cm3; //Li
  G4double gasDensity  =  1.2928*mg/cm3;
  // Air // 1.977*mg/cm3; // CO2 0.178*mg/cm3; // He
 
  G4double totDensity  = foilDensity*foilGasRatio +
                                            gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;

  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( Mylar, fractionFoil );
  radiatorMat->AddMaterial( Air, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat =  radiatorMat;
  fFoilMat     = Mylar;
  fGasMat      = Air;

  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Kr7CH4");
 
  fGapMat          = fAbsorberMaterial;

  fWorldMaterial    = Air; // CO2;

  fSolidWorld = new G4Box("World", fWorldSizeR,fWorldSizeR,fWorldSizeZ/2.);
 
  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");

  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for window inside windowR for

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);
 
  fWindowZ = fStartZ + fRadThick + fWindowThick/2. + 15.0*mm;

  // G4Box* solidWindowR = new G4Box("WindowR",fAbsorberRadius+0.001,
  //                                        fAbsorberRadius+0.001,
  //                                        fWindowThick/2.+0.001  ); 

  //  G4LogicalVolume* logicWindowR = new G4LogicalVolume(solidWindowR,
  //                                    fWorldMaterial, "WindowR");
 
  //  G4VPhysicalVolume*    physiWindowR = new G4PVPlacement(0,
  //                       G4ThreeVector(0.,0.,fWindowZ),
  //                             "WindowR",logicWindowR,fPhysicsWorld,false,0);
  // window
 
  // G4Box* solidWindow = new G4Box("Window",fAbsorberRadius,
  //                                  fAbsorberRadius, fWindowThick/2.);

  //  G4LogicalVolume* logicWindow = new G4LogicalVolume(solidWindow,
  //                                   fWindowMat, "Window");

  //  G4VPhysicalVolume*    physiWindow = 
  //                        new G4PVPlacement(0, G4ThreeVector(0.,0.,0.),
  //                        "Window", logicWindow, physiWindowR, false, 0);

  fGapZ = fWindowZ + fWindowThick/2. + fGapThick/2. + 0.01*mm;

  fElectrodeZ = fGapZ + fGapThick/2. + fElectrodeThick/2. + 0.01*mm;

  // Absorber

  fAbsorberZ = fElectrodeZ + fElectrodeThick/2. +
                                             fAbsorberThickness/2. + 0.01*mm;

  fSolidAbsorber = new G4Box("Absorber", fAbsorberRadius,
                                 fAbsorberRadius, fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");

  fPhysicsAbsorber = new G4PVPlacement(0, G4ThreeVector(0.,0.,fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");  
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Setuo from Y. Watase et al, NIM A248  (1986) 379-388 (fig.7; Li, e-, 2 Gev/c)

G4VPhysicalVolume* Em10DetectorConstruction::SetUpWatase86()
{
  fWorldSizeZ = 400.*cm;
  fWorldSizeR = 20.*cm;

  // Radiator and detector parameters

  fRadThickness = 0.04*mm;
  fGasGap       = 0.126*mm;
  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fFoilNumber   = 300;

  fAbsorberThickness = 30.0*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 10.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  30.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* Li = fMat->GetMaterial("Li");
  //  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* He   = fMat->GetMaterial("He");
  G4Material* Al   = fMat->GetMaterial("Al");
  G4Material* Mylar = fMat->GetMaterial("Mylar");

  G4double foilDensity = 0.534*g/cm3;
  //Li  1.39*g/cm3; // Mylar 0.91*g/cm3;  // CH2
  G4double gasDensity  = 0.178*mg/cm3;
  // He 1.2928*mg/cm3; // Air // 1.977*mg/cm3; // CO2
 
  G4double totDensity  = foilDensity*foilGasRatio + 
                                            gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;

  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( Li, fractionFoil );
  radiatorMat->AddMaterial( He, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat =  radiatorMat;
  fFoilMat     = Li;
  fGasMat      = He;  

  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Xe10CH4");
 
  fGapMat          = fAbsorberMaterial;

  fWorldMaterial    = He; // Air; // CO2 ;

  fSolidWorld = new G4Box("World", fWorldSizeR,fWorldSizeR,fWorldSizeZ/2.);

  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");

  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for window inside windowR for

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);
 
  fWindowZ = fStartZ + fRadThick + fWindowThick/2. + 15.0*mm;

  // G4Box* solidWindowR = new G4Box("WindowR",fAbsorberRadius+0.001,
  //                                         fAbsorberRadius+0.001,
  //                                         fWindowThick/2.+0.001  );

  // G4LogicalVolume* logicWindowR = new G4LogicalVolume(solidWindowR,
  //                                    fWorldMaterial, "WindowR");
 
  //  G4VPhysicalVolume*    physiWindowR = new G4PVPlacement(0,
  //                    G4ThreeVector(0.,0.,fWindowZ),
  //                          "WindowR",logicWindowR,fPhysicsWorld,false,0);
  // window
 
  // G4Box* solidWindow = new G4Box("Window",fAbsorberRadius,
  //                                 fAbsorberRadius, fWindowThick/2.);

  //  G4LogicalVolume* logicWindow = new G4LogicalVolume(solidWindow,
  //                                    fWindowMat, "Window");

  //  G4VPhysicalVolume*    physiWindow =
  //                        new G4PVPlacement(0, G4ThreeVector(0.,0.,0.),
  //                        "Window", logicWindow, physiWindowR, false, 0);

  fGapZ = fWindowZ + fWindowThick/2. + fGapThick/2. + 0.01*mm;

  fElectrodeZ = fGapZ + fGapThick/2. + fElectrodeThick/2. + 0.01*mm;

  // Absorber

  fAbsorberZ = fElectrodeZ + fElectrodeThick/2. +
                                              fAbsorberThickness/2. + 0.01*mm;

  fSolidAbsorber = new G4Box("Absorber", fAbsorberRadius,
                                 fAbsorberRadius, fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");

  fPhysicsAbsorber = new G4PVPlacement(0, G4ThreeVector(0.,0.,fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");  
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

// Setuo from G.D. Barr et al NIM A294 (1990) 465-472 (fig.11)

G4VPhysicalVolume* Em10DetectorConstruction::SetUpBarr90()
{
  fWorldSizeZ = 400.*cm;
  fWorldSizeR = 20.*cm;

  // Radiator and detector parameters

  fRadThickness = 0.019*mm;
  fGasGap       = 0.6*mm;
  foilGasRatio  = fRadThickness/(fRadThickness+fGasGap);

  fFoilNumber   = 350;

  fAbsorberThickness = 50.0*mm;

  fAbsorberRadius   = 100.*mm;
  fAbsorberZ        = 136.*cm;

  fWindowThick    = 51.0*micrometer;
  fElectrodeThick = 10.0*micrometer;
  fGapThick       =  10.0*cm;

  fDetThickness =  50.0*mm;
  fDetLength    = 200.0*cm;
  fDetGap       =   0.01*mm;

  fStartR       = 40*cm;
  fStartZ       = 100.0*mm;

  fModuleNumber = 1;

  // Preparation of mixed radiator material

  G4Material* CH2 = fMat->GetMaterial("CH2");
  G4Material* CO2 = fMat->GetMaterial("CO2");
  G4Material* Air   = fMat->GetMaterial("Air");
  G4Material* Al   = fMat->GetMaterial("Al");
  G4Material* Mylar = fMat->GetMaterial("Mylar");

  G4double foilDensity = 0.91*g/cm3;
  // CH21.39*g/cm3; // Mylar //  0.534*g/cm3; //Li
  G4double gasDensity  = 1.977*mg/cm3;
  // CO2 1.2928*mg/cm3; // Air //  0.178*mg/cm3; // He
 
  G4double totDensity  = foilDensity*foilGasRatio +
                                             gasDensity*(1.0-foilGasRatio);

  G4double fractionFoil =  foilDensity*foilGasRatio/totDensity;
  G4double fractionGas  =  gasDensity*(1.0-foilGasRatio)/totDensity;
 
  G4Material* radiatorMat = new G4Material("radiatorMat"  , totDensity,
                                                  2);
  radiatorMat->AddMaterial( CH2, fractionFoil );
  radiatorMat->AddMaterial( CO2, fractionGas  );

  // default materials of the detector and TR radiator

  fRadiatorMat =  radiatorMat;
  fFoilMat     = CH2;
  fGasMat      = CO2;  

  fWindowMat    = Mylar;
  fElectrodeMat = Al;

  fAbsorberMaterial = fMat->GetMaterial("Xe55He15CH4");
 
  fGapMat          = fAbsorberMaterial;

  fWorldMaterial    =  Air; // CO2; //

  fSolidWorld = new G4Box("World", fWorldSizeR,fWorldSizeR,fWorldSizeZ/2.);

  fLogicWorld = new G4LogicalVolume(fSolidWorld,  fWorldMaterial,  "World");

  fPhysicsWorld = new G4PVPlacement(0, G4ThreeVector(), "World",
                                 fLogicWorld, 0,  false, 0);

  // TR radiator envelope

  fRadThick = fFoilNumber*(fRadThickness + fGasGap) - fGasGap + fDetGap;

  fRadZ = fStartZ + 0.5*fRadThick;

  fSolidRadiator = new G4Box("Radiator",1.1*fAbsorberRadius ,
                              1.1*fAbsorberRadius,  0.5*fRadThick );

  fLogicRadiator = new G4LogicalVolume(fSolidRadiator, fRadiatorMat,
                                       "Radiator");

  fPhysicsRadiator = new G4PVPlacement(0,
                                     G4ThreeVector(0,0,fRadZ),
                                     "Radiator", fLogicRadiator,
                                     fPhysicsWorld, false,        0 );

  // create region for window inside windowR for

  if( fRadRegion != 0 ) delete fRadRegion;
  if( fRadRegion == 0 ) fRadRegion = new G4Region("XTRradiator");
  fRadRegion->AddRootLogicalVolume(fLogicRadiator);

  fWindowZ = fStartZ + fRadThick + fWindowThick/2. + 15.0*mm;

  // G4Box* solidWindowR = new G4Box("WindowR",fAbsorberRadius+0.001,
  //                                         fAbsorberRadius+0.001,
  //                                         fWindowThick/2.+0.001  );

  // G4LogicalVolume* logicWindowR = new G4LogicalVolume(solidWindowR,
  //                                    fWorldMaterial, "WindowR");
  //
  //  G4VPhysicalVolume*    physiWindowR = new G4PVPlacement(0,
  //                       G4ThreeVector(0.,0.,fWindowZ),
  //                             "WindowR",logicWindowR,fPhysicsWorld,false,0);
  // window

  // G4Box* solidWindow = new G4Box("Window",fAbsorberRadius,
  //                                 fAbsorberRadius, fWindowThick/2.);

  // G4LogicalVolume* logicWindow = new G4LogicalVolume(solidWindow,
  //                                   fWindowMat, "Window");

  //  G4VPhysicalVolume*    physiWindow =
  //                        new G4PVPlacement(0, G4ThreeVector(0.,0.,0.),
  //                        "Window", logicWindow, physiWindowR, false, 0);

  fGapZ = fWindowZ + fWindowThick/2. + fGapThick/2. + 0.01*mm;

  fElectrodeZ = fGapZ + fGapThick/2. + fElectrodeThick/2. + 0.01*mm;

  // Absorber

  fAbsorberZ = fElectrodeZ + fElectrodeThick/2. +
                                             fAbsorberThickness/2. + 0.01*mm;

  fSolidAbsorber = new G4Box("Absorber", fAbsorberRadius,
                                 fAbsorberRadius, fAbsorberThickness/2.);

  fLogicAbsorber = new G4LogicalVolume(fSolidAbsorber, fAbsorberMaterial,
                                                "Absorber");
 
  fPhysicsAbsorber = new G4PVPlacement(0, G4ThreeVector(0.,0.,fAbsorberZ),
                                       "Absorber", fLogicAbsorber,
                                        fPhysicsWorld,  false,  0);

  if( fRegGasDet != 0 ) delete fRegGasDet;
  if( fRegGasDet == 0 ) fRegGasDet = new G4Region("XTRdEdxDetector");
  fRegGasDet->AddRootLogicalVolume(fLogicAbsorber);

  // Sensitive Detectors: Absorber

  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  if(!fCalorimeterSD)
  {
    fCalorimeterSD = new Em10CalorimeterSD("CalorSD",this);
    SDman->AddNewDetector( fCalorimeterSD );
  }
  if (fLogicAbsorber)  fLogicAbsorber->SetSensitiveDetector(fCalorimeterSD);

  PrintGeometryParameters();

  return fPhysicsWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::TestOld()
{
  //  G4double inch = 2.54*cm;
  // G4double  mil = inch/1000.0;
  //   G4double GetzstartAbs()           {return zstartAbs;};
  //  G4double GetzendAbs()             {return zendAbs;};
  // void ComputeCalorParameters();

  //  void SetGammaCut(G4double    cut){fGammaCut    = cut;};
  // void SetElectronCut(G4double cut){fElectronCut = cut;};
  //  void SetPositronCut(G4double cut){fPositronCut = cut;};
  // G4int fModelNumber ; // selection of parametrisation model1-10
  //   void SetAlphaPlate (G4double val){fAlphaPlate = val;};
  //   void SetAlphaGas   (G4double val){fAlphaGas   = val;};

  // G4double           fAlphaPlate ;
  // G4double           fAlphaGas ;

  // fAlphaPlate   = 160.0;
  // fAlphaGas     = 160.0;
  // fModelNumber  = 0;

  // create commands for interactive definition of the calorimeter

  // fGammaCut    = 23*mm;
  // fElectronCut = 23*mm; 
  // fPositronCut = 23*mm; 

  // G4cout << *(G4Material::GetMaterialTable()) << G4endl;

  //  G4int i, j ;
  // G4int j ;
  //  G4double zModule, zRadiator, rModule, rRadiator ;

  // complete the Calor parameters definition and Print

  //ComputeCalorParameters();

  // zRadiator ;
 
  // World

  // if(solidWorld) delete solidWorld ;
  // if(logicWorld) delete logicWorld ;
  // if(physiWorld) delete physiWorld ;
 
  //  if(solidRadiator) delete solidRadiator;
  //  if(logicRadiator) delete logicRadiator;
  //  if(physiRadiator) delete physiRadiator;

  //  radThick *= 1.02 ;

  //  if(fSolidRadSlice) delete fSolidRadSlice;
  //  if(fLogicRadSlice) delete fLogicRadSlice; 
  //  if(fPhysicRadSlice) delete fPhysicRadSlice;
  // fSolidRadSlice = new G4Box("RadSlice",fAbsorberRadius,
  //   fAbsorberRadius,0.5*fRadThickness );

  // fLogicRadSlice = new G4LogicalVolume(fSolidRadSlice,fRadiatorMat,
  //                                          "RadSlice",0,0,0);

//    for(j=0;j<fFoilNumber;j++)
//    {
//
//      zRadiator = zModule + j*(fRadThickness + fGasGap) ;
//      G4cout<<zRadiator/mm<<" mm"<<"\t" ;
//      //   G4cout<<"j = "<<j<<"\t" ;
// 
//      fPhysicRadSlice =
//          new G4PVPlacement(0,G4ThreeVector(0.,0.,zRadiator-zRad),
//                                         "RadSlice",fLogicRadSlice,
//                                          physiRadiator,false,j);
//     }
//  G4cout<<G4endl ;
 
    // fRadRegion->RemoveRootLogicalVolume(logicWindowR);
  // G4ProductionCuts* cutsR = 0;
    // cutsR = new G4ProductionCuts();
    // fRadRegion->SetProductionCuts(cutsR);

  // else  // Second time - get a cut object from region
  {
    // cutsR = fRadRegion->GetProductionCuts();
  }

  // cutsR->SetProductionCut(fGammaCut,"gamma");
  // cutsR->SetProductionCut(fElectronCut,"e-");
  // cutsR->SetProductionCut(fPositronCut,"e+");
  // G4Box* solidGap = new G4Box("Gap",fAbsorberRadius, fAbsorberRadius,
  //                                fGapThick/2.     ) ;
 
  // G4LogicalVolume* logicGap = new G4LogicalVolume(solidGap,fGapMat, "Gap");

  // G4VPhysicalVolume*    physiGap = new G4PVPlacement(0,
  //                                        G4ThreeVector(0.,0.,zGap),
  //                                    "Gap",logicGap,physiWorld,false,0);

  // G4Box* solidElectrode = new G4Box("Electrode",fAbsorberRadius,
  //                                  fAbsorberRadius, fElectrodeThick/2. );

  // G4LogicalVolume* logicElectrode = new G4LogicalVolume(solidElectrode,
  //                                     fElectrodeMat, "Electrode");

  //  G4VPhysicalVolume*    physiElectrode = new G4PVPlacement(0,
  //                                         G4ThreeVector(0.,0.,zElectrode),
  //                                    "Electrode",logicElectrode,
  //                                     physiWorld,false,0);
    //  if(solidAbsorber) delete solidAbsorber;
    //  if(logicAbsorber) delete logicAbsorber;
    //  if(physiAbsorber) delete physiAbsorber;
//   if (fAbsorberThickness > 0.)
//  {
//  }

    // fRegGasDet->RemoveRootLogicalVolume(logicAbsorber);
  // G4ProductionCuts* cuts = 0;
    // cuts = new G4ProductionCuts();
    //  fRegGasDet->SetProductionCuts(cuts);
  // else  // Second time - get a cut object from region
  {
    //  cuts = fRegGasDet->GetProductionCuts();
  }

  // cuts->SetProductionCut(fGammaCut,"gamma");
  // cuts->SetProductionCut(fElectronCut,"e-");
  // cuts->SetProductionCut(fPositronCut,"e+");

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::PrintGeometryParameters()
{
  G4cout << "\n The  WORLD   is made of "
       << fWorldSizeZ/mm << "mm of " << fWorldMaterial->GetName();
  G4cout << ", the transverse size (R) of the world is " << 
                                         fWorldSizeR/mm << " mm. " << G4endl;
  G4cout << " The ABSORBER is made of "
       << fAbsorberThickness/mm << "mm of " << fAbsorberMaterial->GetName();
  G4cout << ", the transverse size (R) is " << fAbsorberRadius/mm << 
            " mm. " << G4endl;
  G4cout << " Z position of the (middle of the) absorber " 
         << fAbsorberZ/mm << "  mm." << G4endl;

  G4cout<<"fRadZ = "<<fRadZ/mm<<" mm"<<G4endl;
 
  G4cout<<"fStartZ = "<<fStartZ/mm<<" mm"<<G4endl;

  G4cout<<"fRadThick = "<<fRadThick/mm<<" mm"<<G4endl;
  G4cout<<"fFoilNumber = "<<fFoilNumber<<G4endl;
  G4cout<<"fRadiatorMat = "<<fRadiatorMat->GetName()<<G4endl;
  G4cout<<"WorldMaterial = "<<fWorldMaterial->GetName()<<G4endl;
  //  G4cout<<"fAbsorberZ = "<<fAbsorberZ/mm<<" mm"<<G4endl;
  G4cout << G4endl;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetAbsorberMaterial(G4String materialChoice)
{
  // get the pointer to the material table
  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();

  // search the material by its name
  G4Material* pttoMaterial;

  for (size_t J=0 ; J<theMaterialTable->size() ; J++)
  {
    pttoMaterial = (*theMaterialTable)[J];
 
    if(pttoMaterial->GetName() == materialChoice)
    {
      fAbsorberMaterial = pttoMaterial;
      fLogicAbsorber->SetMaterial(pttoMaterial);
        // PrintCalorParameters();
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetRadiatorMaterial(G4String materialChoice)
{
  // get the pointer to the material table

  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();

  // search the material by its name

  G4Material* pttoMaterial;
  for (size_t J=0 ; J<theMaterialTable->size() ; J++)
  {
    pttoMaterial = (*theMaterialTable)[J];

    if(pttoMaterial->GetName() == materialChoice)
    {
      fRadiatorMat = pttoMaterial;
//      fLogicRadSlice->SetMaterial(pttoMaterial);
      // PrintCalorParameters();
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetWorldMaterial(G4String materialChoice)
{
  // get the pointer to the material table
  const G4MaterialTable* theMaterialTable = G4Material::GetMaterialTable();

  // search the material by its name
  G4Material* pttoMaterial;

  for (size_t J=0 ; J<theMaterialTable->size() ; J++)
  {
    pttoMaterial = (*theMaterialTable)[J];
 
    if(pttoMaterial->GetName() == materialChoice)
    {
      fWorldMaterial = pttoMaterial;
      fLogicWorld->SetMaterial(pttoMaterial);
       //  PrintCalorParameters();
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetAbsorberThickness(G4double val)
{
  // change Absorber thickness and recompute the calorimeter parameters
  fAbsorberThickness = val;
  //  ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetRadiatorThickness(G4double val)
{
  // change XTR radiator thickness and recompute the calorimeter parameters
  fRadThickness = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetGasGapThickness(G4double val)
{
  // change XTR gas gap thickness and recompute the calorimeter parameters
  fGasGap = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetAbsorberRadius(G4double val)
{
  // change the transverse size and recompute the calorimeter parameters
  fAbsorberRadius = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetWorldSizeZ(G4double val)
{
  fWorldChanged=true;
  fWorldSizeZ = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetWorldSizeR(G4double val)
{
  fWorldChanged=true;
  fWorldSizeR = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetAbsorberZpos(G4double val)
{
  fAbsorberZ  = val;
  // ComputeCalorParameters();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::SetMagField(G4double)
{
  //apply a global uniform magnetic field along X axis

  /* *********************************************************

  G4FieldManager* fieldMgr
   = G4TransportationManager::GetTransportationManager()->GetFieldManager();

  if(magField) delete magField;             //delete the existing magn field

  if(fieldValue!=0.)                        // create a new one if non null
  {
    magField = new G4UniformMagField(G4ThreeVector(fieldValue,0.,0.));
    fieldMgr->SetDetectorField(magField);
    fieldMgr->CreateChordFinder(magField);
  }
  else
  {
    magField = 0;
    fieldMgr->SetDetectorField(magField);
  }

  *************************************************************** */

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10DetectorConstruction::UpdateGeometry()
{
  G4RunManager::GetRunManager()->DefineWorldVolume(ConstructDetectorXTR());
}
