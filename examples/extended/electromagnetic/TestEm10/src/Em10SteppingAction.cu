#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10SteppingAction.cc
/// \brief Implementation of the Em10SteppingAction class
//
//
// $Id: Em10SteppingAction.cc 73033 2013-08-15 09:24:45Z gcosmo $
//
//

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "Em10DetectorConstruction.hh"
#include "G4Electron.hh"
#include "G4Positron.hh"
#include "G4Gamma.hh"
#include "Em10SteppingAction.hh"
#include "Em10EventAction.hh"
#include "Em10RunAction.hh"
#include "G4Event.hh"
#include "G4VPhysicalVolume.hh"
#include "G4PhysicalConstants.hh"
#include "G4ios.hh"
#include <iomanip>

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10SteppingAction::Em10SteppingAction(Em10EventAction* EA,
                                       Em10RunAction* RA)
  :G4UserSteppingAction(),eventaction (EA),runaction (RA),
   IDold(-1)
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

Em10SteppingAction::~Em10SteppingAction()
{}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void Em10SteppingAction::UserSteppingAction(const G4Step* aStep)
{

  G4double Theta,Thetaback,Ttrans,Tback,Tsec,Egamma,yend,zend,rend;

  G4int evno = eventaction->GetEventno();

  const G4Track* track = aStep->GetTrack();
  const G4StepPoint* prePoint = aStep->GetPreStepPoint();
  G4int trackID = track->GetTrackID();
  G4int parentID = track->GetParentID();

  const G4DynamicParticle* dynParticle = track->GetDynamicParticle();
  const G4ParticleDefinition* particle = dynParticle->GetDefinition();
  G4VPhysicalVolume* preVol = prePoint->GetPhysicalVolume();
  G4VPhysicalVolume* postVol = aStep->GetPostStepPoint()->GetPhysicalVolume();

  IDnow = evno+10000*trackID+100000000*parentID;
  if(IDnow != IDold) {
    IDold=IDnow;
    if(trackID > 1 && (particle == G4Electron::Electron() ||
                       particle == G4Positron::Positron() ||
                       particle == G4Gamma::Gamma())) {
      runaction->Fillvertexz(track->GetVertexPosition().z());

      if(preVol->GetName()=="Absorber") {
        if(particle == G4Gamma::Gamma()) {
          eventaction->AddNeutral();
        } else {
          eventaction->AddCharged();
          Tsec = track->GetKineticEnergy();
          Tsec += aStep->GetTotalEnergyDeposit();
          runaction->FillTsec(Tsec);
          if(particle == G4Electron::Electron()) {
            eventaction->AddE();
          } else {
            eventaction->AddP();
          }
        }
      }
    }
  }

  if(preVol->GetName()=="Absorber") {
    if(particle == G4Electron::Electron() ||
       particle == G4Positron::Positron()) {
      eventaction->CountStepsCharged();

    } else if(particle == G4Gamma::Gamma()) {
      eventaction->CountStepsNeutral();
    }

    if(prePoint->GetStepStatus() == fGeomBoundary &&
       preVol != postVol) {

      if(trackID == 1) {
        if(track->GetMomentumDirection().z()>0.) {

          eventaction->SetTr();
          Theta = std::acos(track->GetMomentumDirection().z());
          runaction->FillTh(Theta);
          Ttrans = track->GetKineticEnergy();
          runaction->FillTt(Ttrans);
          yend= aStep->GetTrack()->GetPosition().y();
          zend= aStep->GetTrack()->GetPosition().x();
          rend = std::sqrt(yend*yend+zend*zend);
          runaction->FillR(rend);

        } else {
          eventaction->SetRef();
          Thetaback = std::acos(aStep->GetTrack()->GetMomentumDirection().z());
          Thetaback -= 0.5*pi;
          runaction->FillThBack(Thetaback);
          Tback  = aStep->GetTrack()->GetKineticEnergy();
          runaction->FillTb(Tback);
        }
      }
      if(track->GetMomentumDirection().z()>0. &&
         particle == G4Gamma::Gamma()) {
 
        Egamma = aStep->GetTrack()->GetKineticEnergy();
        runaction->FillGammaSpectrum(Egamma);
      }
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
