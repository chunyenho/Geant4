//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10PhysicsList.cc
/// \brief Implementation of the Em10PhysicsList class
//
//
// $Id: Em10PhysicsList.cc 68585 2013-04-01 23:35:07Z adotti $
//

#include "Em10PhysicsList.hh"
#include "Em10DetectorConstruction.hh"
#include "Em10PhysicsListMessenger.hh"

#include "G4ParticleDefinition.hh"
#include "G4ProcessManager.hh"
#include "G4ParticleTable.hh"
#include "G4ParticleTypes.hh"
#include "G4Material.hh"

#include "G4UnitsTable.hh"
#include "G4SystemOfUnits.hh"
#include "G4ios.hh"
#include <iomanip>

#include "G4Region.hh"
#include "G4RegionStore.hh"

#include "G4ProductionCuts.hh"
#include "G4EmProcessOptions.hh"

#include "G4ComptonScattering.hh"
#include "G4GammaConversion.hh"
#include "G4PhotoElectricEffect.hh"

#include "G4eMultipleScattering.hh"
#include "G4MuMultipleScattering.hh"
#include "G4hMultipleScattering.hh"

#include "G4eIonisation.hh"
#include "G4eBremsstrahlung.hh"
#include "G4eplusAnnihilation.hh"
#include "G4PAIModel.hh"
#include "G4PAIPhotonModel.hh"

#include "G4SynchrotronRadiation.hh"

#include "G4MuIonisation.hh"
#include "G4MuBremsstrahlung.hh"
#include "G4MuPairProduction.hh"

#include "G4hIonisation.hh"

#include "G4Decay.hh"

#include "G4VXTRenergyLoss.hh"
#include "G4RegularXTRadiator.hh"
#include "G4TransparentRegXTRadiator.hh"
#include "G4GammaXTRadiator.hh"
#include "G4StrawTubeXTRadiator.hh"

#include "G4XTRGammaRadModel.hh"
#include "G4XTRRegularRadModel.hh"
#include "G4XTRTransparentRegRadModel.hh"
#include "Em10XTRTransparentRegRadModel.hh"

#include "Em10StepCut.hh"

/////////////////////////////////////////////////////////////
//
//

Em10PhysicsList::Em10PhysicsList(Em10DetectorConstruction* p)
  :  G4VUserPhysicsList(),
     MaxChargedStep(DBL_MAX),
     theeminusStepCut(0),            theeplusStepCut(0),
     fRadiatorCuts(0),fDetectorCuts(0),fXTRModel("transpM")
{
  pDet = p;

  // world cuts

  defaultCutValue = 1.*mm;
  cutForGamma     = defaultCutValue;
  cutForElectron  = defaultCutValue;
  cutForPositron  = defaultCutValue;

  // Region cuts

  fGammaCut    = defaultCutValue;
  fElectronCut = defaultCutValue;
  fPositronCut = defaultCutValue;

  SetVerboseLevel(1);
  physicsListMessenger = new Em10PhysicsListMessenger(this);
}

Em10PhysicsList::~Em10PhysicsList()
{
  delete physicsListMessenger; 
}

///////////////////////////////////////////////////////////////////////////
//
//

void Em10PhysicsList::ConstructParticle()
{
  // In this method, static member functions should be called
  // for all particles which you want to use.
  // This ensures that objects of these particle types will be
  // created in the program. 

  ConstructBosons();
  ConstructLeptons();
  ConstructMesons();
  ConstructBarions();
}

////////////////////////////////////////////////////////////////////////////
//
//

void Em10PhysicsList::ConstructBosons()
{
  // gamma
  G4Gamma::GammaDefinition();
}

void Em10PhysicsList::ConstructLeptons()
{
  // leptons

  G4Electron::ElectronDefinition();
  G4Positron::PositronDefinition();
  G4MuonPlus::MuonPlusDefinition();
  G4MuonMinus::MuonMinusDefinition();

  G4NeutrinoE::NeutrinoEDefinition();
  G4AntiNeutrinoE::AntiNeutrinoEDefinition();
  G4NeutrinoMu::NeutrinoMuDefinition();
  G4AntiNeutrinoMu::AntiNeutrinoMuDefinition();
}

void Em10PhysicsList::ConstructMesons()
{
 //  mesons

  G4PionPlus::PionPlusDefinition();
  G4PionMinus::PionMinusDefinition();
  G4PionZero::PionZeroDefinition();
  G4KaonPlus::KaonPlusDefinition();
  G4KaonMinus::KaonMinusDefinition();
}


void Em10PhysicsList::ConstructBarions()
{
//  barions

  G4Proton::ProtonDefinition();
  G4AntiProton::AntiProtonDefinition();
}


///////////////////////////////////////////////////////////////////////
//
//

void Em10PhysicsList::ConstructProcess()
{
  AddTransportation();
  ConstructEM();
  ConstructGeneral();
}

/////////////////////////////////////////////////////////////////////////////
//
//

void Em10PhysicsList::ConstructEM()
{
  
  // G4cout<<"fMinElectronEnergy = "<<fMinElectronEnergy/keV<<" keV"<<G4endl;
  // G4cout<<"fMinGammaEnergy = "<<fMinGammaEnergy/keV<<" keV"<<G4endl;
  G4cout<<"XTR model = "<<fXTRModel<<G4endl;

  const G4RegionStore* theRegionStore = G4RegionStore::GetInstance();
  G4Region* gas = theRegionStore->GetRegion("XTRdEdxDetector");

  G4VXTRenergyLoss* processXTR = 0;

  if(fXTRModel == "gammaR" )          
  {      
    // G4GammaXTRadiator* 
    processXTR = new G4GammaXTRadiator(pDet->GetLogicalRadiator(),
                                       100.,
                                       100.,
                                       pDet->GetFoilMaterial(),
                                       pDet->GetGasMaterial(),
                                       pDet->GetFoilThick(),
                                       pDet->GetGasThick(),
                                       pDet->GetFoilNumber(),
                                       "GammaXTRadiator");
  }
  else if(fXTRModel == "gammaM" ) 
  {
    // G4XTRGammaRadModel* 
    processXTR = new G4XTRGammaRadModel(pDet->GetLogicalRadiator(),
                                       100.,
                                       100.,
                                       pDet->GetFoilMaterial(),
                                       pDet->GetGasMaterial(),
                                       pDet->GetFoilThick(),
                                       pDet->GetGasThick(),
                                       pDet->GetFoilNumber(),
                                       "GammaXTRadiator");
  }
  else if(fXTRModel == "strawR" ) 
  {

    // G4StrawTubeXTRadiator* 
    processXTR = new G4StrawTubeXTRadiator(pDet->GetLogicalRadiator(),
                                         pDet->GetFoilMaterial(),
                                         pDet->GetGasMaterial(),
                                0.53,           // pDet->GetFoilThick(),
                                3.14159,           // pDet->GetGasThick(),
                                         pDet->GetAbsorberMaterial(),
                                         true,
                                         "strawXTRadiator");
  }
  else if(fXTRModel == "regR" ) 
  {      
    // G4RegularXTRadiator* 
    processXTR = new G4RegularXTRadiator(pDet->GetLogicalRadiator(),
                                         pDet->GetFoilMaterial(),
                                         pDet->GetGasMaterial(),
                                         pDet->GetFoilThick(),
                                         pDet->GetGasThick(),
                                         pDet->GetFoilNumber(),
                                         "RegularXTRadiator");            
  }
  else if(fXTRModel == "transpR" ) 
  {
    // G4TransparentRegXTRadiator* 
    processXTR = new G4TransparentRegXTRadiator(pDet->GetLogicalRadiator(),
                                         pDet->GetFoilMaterial(),
                                         pDet->GetGasMaterial(),
                                         pDet->GetFoilThick(),
                                         pDet->GetGasThick(),
                                         pDet->GetFoilNumber(),
                                         "RegularXTRadiator");
  }
  else if(fXTRModel == "regM" ) 
  {
    // G4XTRRegularRadModel* 
    processXTR = new G4XTRRegularRadModel(pDet->GetLogicalRadiator(),
                                         pDet->GetFoilMaterial(),
                                         pDet->GetGasMaterial(),
                                         pDet->GetFoilThick(),
                                         pDet->GetGasThick(),
                                         pDet->GetFoilNumber(),
                                         "RegularXTRadiator");
       
  }
  else if(fXTRModel == "transpM" ) 
  { 
    // G4XTRTransparentRegRadModel* 
    // processXTR = new G4XTRTransparentRegRadModel(pDet->GetLogicalRadiator(),
    processXTR = new Em10XTRTransparentRegRadModel(pDet->GetLogicalRadiator(),
                                         pDet->GetFoilMaterial(),
                                         pDet->GetGasMaterial(),
                                         pDet->GetFoilThick(),
                                         pDet->GetGasThick(),
                                         pDet->GetFoilNumber(),
                                         "RegularXTRadiator");
  }     
  else
  {
    G4Exception("Invalid XTR model name", "InvalidSetup",
                 FatalException, "XTR model name is out of the name list");
  }     
  //  processXTR->SetCompton(true);
  processXTR->SetVerboseLevel(1);

  theParticleIterator->reset();

  while( (*theParticleIterator)() )
  {
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    G4String particleName = particle->GetParticleName();

    if (particleName == "gamma")
    {
      // Construct processes for gamma

      pmanager->AddDiscreteProcess(new G4PhotoElectricEffect);
      pmanager->AddDiscreteProcess(new G4ComptonScattering);
      pmanager->AddDiscreteProcess(new G4GammaConversion);

    }
    else if (particleName == "e-")
    {
      // Construct processes for electron
      theeminusStepCut = new Em10StepCut();
      theeminusStepCut->SetMaxStep(MaxChargedStep) ;
      G4eIonisation* eioni = new G4eIonisation();
      G4PAIModel*     pai = new G4PAIModel(particle,"PAIModel");
      eioni->AddEmModel(0,pai,pai,gas);

      pmanager->AddProcess(new G4eMultipleScattering,-1,1,1);
      pmanager->AddProcess(eioni,-1,2,2);
      pmanager->AddProcess(new G4eBremsstrahlung,-1,3,3);
      pmanager->AddDiscreteProcess(processXTR);
      pmanager->AddDiscreteProcess(new G4SynchrotronRadiation);
      pmanager->AddDiscreteProcess(theeminusStepCut);

    }
    else if (particleName == "e+")
    {
      // Construct processes for positron

      theeplusStepCut = new Em10StepCut();
      theeplusStepCut->SetMaxStep(MaxChargedStep) ;
      G4eIonisation* eioni = new G4eIonisation();
      G4PAIModel*     pai = new G4PAIModel(particle,"PAIModel");
      eioni->AddEmModel(0,pai,pai,gas);

      pmanager->AddProcess(new G4eMultipleScattering,-1,1,1);
      pmanager->AddProcess(eioni,-1,2,2);
      pmanager->AddProcess(new G4eBremsstrahlung,-1,3,3);
      pmanager->AddProcess(new G4eplusAnnihilation,0,-1,4);
      pmanager->AddDiscreteProcess(processXTR);
      pmanager->AddDiscreteProcess(new G4SynchrotronRadiation);
      pmanager->AddDiscreteProcess(theeplusStepCut);

    }
    else if( particleName == "mu+" ||
             particleName == "mu-"    )
    {
     // Construct processes for muon+

      Em10StepCut* muonStepCut = new Em10StepCut();
      muonStepCut->SetMaxStep(MaxChargedStep) ;

      G4MuIonisation* muioni = new G4MuIonisation() ;

      G4PAIModel*     pai = new G4PAIModel(particle,"PAIModel");
      muioni->AddEmModel(0,pai,pai,gas);

      pmanager->AddProcess(new G4MuMultipleScattering(),-1,1,1);
      pmanager->AddProcess(muioni,-1,2,2);
      pmanager->AddProcess(new G4MuBremsstrahlung(),-1,3,3);
      pmanager->AddProcess(new G4MuPairProduction(),-1,4,4);
      pmanager->AddProcess( muonStepCut,-1,-1,5);

    }
    else if (
                particleName == "proton"
               || particleName == "antiproton"
               || particleName == "pi+"
               || particleName == "pi-"
               || particleName == "kaon+"
               || particleName == "kaon-"
              )
    {
      Em10StepCut* thehadronStepCut = new Em10StepCut();
      thehadronStepCut->SetMaxStep(MaxChargedStep) ;

      G4hIonisation* thehIonisation = new G4hIonisation();
      G4PAIModel*     pai = new G4PAIModel(particle,"PAIModel");
      thehIonisation->AddEmModel(0,pai,pai,gas);

      pmanager->AddProcess(new G4hMultipleScattering,-1,1,1);
      pmanager->AddProcess(thehIonisation,-1,2,2);
      pmanager->AddProcess( thehadronStepCut,-1,-1,3);

    }
  }
  G4EmProcessOptions opt;
  opt.SetApplyCuts(true);
}

void Em10PhysicsList::ConstructGeneral()
{
  // Add Decay Process

  G4Decay* theDecayProcess = new G4Decay();
  theParticleIterator->reset();

  while( (*theParticleIterator)() )
  {
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();

    if (theDecayProcess->IsApplicable(*particle)) 
    { 
      pmanager ->AddProcess(theDecayProcess);

      // set ordering for PostStepDoIt and AtRestDoIt

      pmanager ->SetProcessOrdering(theDecayProcess, idxPostStep);
      pmanager ->SetProcessOrdering(theDecayProcess, idxAtRest);
    }
  }
}

/////////////////////////////////////////////////////////////////////////////

void Em10PhysicsList::SetCuts()
{
  // set cut values for gamma at first and for e- second and next for e+,
  // because some processes for e+/e- need cut values for gamma
 
  SetCutValue(cutForGamma, "gamma", "DefaultRegionForTheWorld");
  SetCutValue(cutForElectron, "e-", "DefaultRegionForTheWorld");
  SetCutValue(cutForPositron, "e+", "DefaultRegionForTheWorld");

  if (verboseLevel > 0)
  {
    G4cout << "Em10PhysicsList::SetCuts:";
    G4cout << "CutLength for e-, e+ and gamma is: " 
           << G4BestUnit(defaultCutValue,"Length") << G4endl;
  }
  
  if( !fRadiatorCuts ) SetRadiatorCuts();

  G4Region* region = (G4RegionStore::GetInstance())->GetRegion("XTRradiator");
  region->SetProductionCuts(fRadiatorCuts);
  G4cout << "Radiator cuts are set" << G4endl;

  if( !fDetectorCuts ) SetDetectorCuts();
  region = (G4RegionStore::GetInstance())->GetRegion("XTRdEdxDetector");
  region->SetProductionCuts(fDetectorCuts);
  G4cout << "Detector cuts are set" << G4endl;

  if (verboseLevel > 1)     DumpCutValuesTable();
}

///////////////////////////////////////////////////////////////////////////

void Em10PhysicsList::SetGammaCut(G4double val)
{
  cutForGamma = val;
}

///////////////////////////////////////////////////////////////////////////

void Em10PhysicsList::SetElectronCut(G4double val)
{
  cutForElectron = val;
}

////////////////////////////////////////////////////////////////////////////

void Em10PhysicsList::SetMaxStep(G4double step)
{
  MaxChargedStep = step ;
  G4cout << " MaxChargedStep=" << MaxChargedStep << G4endl;
  G4cout << G4endl;
}

/////////////////////////////////////////////////////

void Em10PhysicsList::SetRadiatorCuts()
{
  if( !fRadiatorCuts ) fRadiatorCuts = new G4ProductionCuts();

  fRadiatorCuts->SetProductionCut(fGammaCut, idxG4GammaCut);
  fRadiatorCuts->SetProductionCut(fElectronCut, idxG4ElectronCut);
  fRadiatorCuts->SetProductionCut(fPositronCut, idxG4PositronCut);

  G4cout<<"Radiator gamma cut    = "<<fGammaCut/mm<<" mm"<<G4endl;
  G4cout<<"Radiator electron cut = "<<fElectronCut/mm<<" mm"<<G4endl;
  G4cout<<"Radiator positron cut = "<<fPositronCut/mm<<" mm"<<G4endl;
}

/////////////////////////////////////////////////////////////

void Em10PhysicsList::SetDetectorCuts()
{
  if( !fDetectorCuts ) fDetectorCuts = new G4ProductionCuts();

  fDetectorCuts->SetProductionCut(fGammaCut, idxG4GammaCut);
  fDetectorCuts->SetProductionCut(fElectronCut, idxG4ElectronCut);
  fDetectorCuts->SetProductionCut(fPositronCut, idxG4PositronCut);

  G4cout<<"Detector gamma cut    = "<<fGammaCut/mm<<" mm"<<G4endl;
  G4cout<<"Detector electron cut = "<<fElectronCut/mm<<" mm"<<G4endl;
  G4cout<<"Detector positron cut = "<<fPositronCut/mm<<" mm"<<G4endl;

}
