//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10RunMessenger.cc
/// \brief Implementation of the Em10RunMessenger class
//
//
// $Id: Em10RunMessenger.cc 67268 2013-02-13 11:38:40Z ihrivnac $
//
// 

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "Em10RunMessenger.hh"

#include "Em10RunAction.hh"
#include "G4UIdirectory.hh"
#include "G4UIcmdWithAnInteger.hh"
#include "G4UIcmdWithADoubleAndUnit.hh"
#include "G4UIcmdWithADouble.hh"
#include "G4UIcmdWithAString.hh"
#include "G4ios.hh"
#include "globals.hh"
#include "Randomize.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

Em10RunMessenger::Em10RunMessenger(Em10RunAction* RA)
:G4UImessenger(),runAction (RA)
{
  plotDir = new G4UIdirectory("/plots/");
  plotDir->SetGuidance("plot control");

  sethistNameCmd = new G4UIcmdWithAString("/plots/sethistName",this);
  sethistNameCmd->SetGuidance("set name for the histogram file"); 

  setnbinStepCmd = new G4UIcmdWithAnInteger("/plots/setnbinStep",this);
  setnbinStepCmd->SetGuidance("set nb of bins in #step plot");
  setnbinStepCmd->SetParameterName("nbinStep",false);

  setSteplowCmd = new G4UIcmdWithADouble("/plots/setSteplow",this);
  setSteplowCmd->SetGuidance("set lower limit for #step plot ");
  setSteplowCmd->SetParameterName("Steplow",false);

  setStephighCmd = new G4UIcmdWithADouble("/plots/setStephigh",this);
  setStephighCmd->SetGuidance("set upper limit for #step plot ");
  setStephighCmd->SetParameterName("Stephigh",false);

  setnbinEnCmd = new G4UIcmdWithAnInteger("/plots/setnbinEn",this);
  setnbinEnCmd->SetGuidance("set nb of bins in Edep plot");
  setnbinEnCmd->SetParameterName("nbinE",false);

  setEnlowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setEnlow",this);
  setEnlowCmd->SetGuidance("set lower limit for Edep plot ");
  setEnlowCmd->SetParameterName("Elow",false);
  setEnlowCmd->SetUnitCategory("Energy");
  
  setEnhighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setEnhigh",this);
  setEnhighCmd->SetGuidance("set upper limit for Edep plot ");
  setEnhighCmd->SetParameterName("Ehigh",false);
  setEnhighCmd->SetUnitCategory("Energy");

  setnbinGammaCmd = new G4UIcmdWithAnInteger("/plots/setnbinGamma",this);
  setnbinGammaCmd->SetGuidance("set nb of bins in gamma spectrum plot");
  setnbinGammaCmd->SetParameterName("nbinGamma",false);

  setElowGammaCmd = new G4UIcmdWithADoubleAndUnit("/plots/setElowGamma",this);
  setElowGammaCmd->SetGuidance("set lower limit for gamma spectrum plot ");
  setElowGammaCmd->SetParameterName("ElowGamma",false);
  setElowGammaCmd->SetUnitCategory("Energy");

  setEhighGammaCmd = new G4UIcmdWithADoubleAndUnit("/plots/setEhighGamma",this);
  setEhighGammaCmd->SetGuidance("set upper limit for gamma spectrum plot ");
  setEhighGammaCmd->SetParameterName("EhighGamma",false);
  setEhighGammaCmd->SetUnitCategory("Energy");

  setnbinTtCmd = new G4UIcmdWithAnInteger("/plots/setnbinTt",this);
  setnbinTtCmd->SetGuidance("set nb of bins in Etransmitted plot");
  setnbinTtCmd->SetParameterName("nbinTt",false);

  setTtlowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTtlow",this);
  setTtlowCmd->SetGuidance("set lower limit for Etransmitted plot ");
  setTtlowCmd->SetParameterName("Ttlow",false);

  setTthighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTthigh",this);
  setTthighCmd->SetGuidance("set upper limit for Etransmitted plot ");
  setTthighCmd->SetParameterName("Tthigh",false);

  setnbinTbCmd = new G4UIcmdWithAnInteger("/plots/setnbinTb",this);
  setnbinTbCmd->SetGuidance("set nb of bins in Ebackscattering plot");
  setnbinTbCmd->SetParameterName("nbinTb",false);

  setTblowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTblow",this);
  setTblowCmd->SetGuidance("set lower limit for Ebackscattered plot ");
  setTblowCmd->SetParameterName("Tblow",false);

  setTbhighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTbhigh",this);
  setTbhighCmd->SetGuidance("set upper limit for Ebackscattered plot ");
  setTbhighCmd->SetParameterName("Tbhigh",false);

  setnbinTsecCmd = new G4UIcmdWithAnInteger("/plots/setnbinTsec",this);
  setnbinTsecCmd->SetGuidance("set nb of bins in charged Tsecondary plot");
  setnbinTsecCmd->SetParameterName("nbinTsec",false);

  setTseclowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTseclow",this);
  setTseclowCmd->SetGuidance("set lower limit for charged Tsecondary plot ");
  setTseclowCmd->SetParameterName("Tseclow",false);

  setTsechighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setTsechigh",this);
  setTsechighCmd->SetGuidance("set upper limit for charged Tsecondary plot ");
  setTsechighCmd->SetParameterName("Tsechigh",false);

  setnbinRCmd = new G4UIcmdWithAnInteger("/plots/setnbinR",this);
  setnbinRCmd->SetGuidance("set nb of bins in R plot");
  setnbinRCmd->SetParameterName("nbinR",false);

  setRlowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setRlow",this);
  setRlowCmd->SetGuidance("set lower limit for R plot ");
  setRlowCmd->SetParameterName("Rlow",false);

  setRhighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setRhigh",this);
  setRhighCmd->SetGuidance("set upper limit for R plot ");
  setRhighCmd->SetParameterName("Rhigh",false);

  setnbinzvertexCmd = new G4UIcmdWithAnInteger("/plots/setnbinzvertex",this);
  setnbinzvertexCmd->SetGuidance("set nb of bins in Z vertex plot");
  setnbinzvertexCmd->SetParameterName("nbinZ",false);

  setzlowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setzlow",this);
  setzlowCmd->SetGuidance("set lower limit for Z vertex plot ");
  setzlowCmd->SetParameterName("zlow",false);

  setzhighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setzhigh",this);
  setzhighCmd->SetGuidance("set upper limit for Z vertex plot ");
  setzhighCmd->SetParameterName("zhigh",false);

  setnbinThCmd = new G4UIcmdWithAnInteger("/plots/setnbinTh",this);
  setnbinThCmd->SetGuidance("set nb of bins in Theta transmitted plot");
  setnbinThCmd->SetParameterName("nbinTh",false);

  setThlowCmd = new G4UIcmdWithADoubleAndUnit("/plots/setThlow",this);
  setThlowCmd->SetGuidance("set lower limit for Theta transmitted plot ");
  setThlowCmd->SetParameterName("Thlow",false);

  setThhighCmd = new G4UIcmdWithADoubleAndUnit("/plots/setThhigh",this);
  setThhighCmd->SetGuidance("set upper limit for Theta transmitted plot ");
  setThhighCmd->SetParameterName("Thhigh",false);

  setnbinThbackCmd = new G4UIcmdWithAnInteger("/plots/setnbinThback",this);
  setnbinThbackCmd->SetGuidance("set nb of bins in backscattering Theta plot");
  setnbinThbackCmd->SetParameterName("nbinThback",false);

  setThlowbackCmd = new G4UIcmdWithADoubleAndUnit("/plots/setThlowback",this);
  setThlowbackCmd->SetGuidance("set lower limit for backscattering Theta plot ");
  setThlowbackCmd->SetParameterName("Thlowback",false);

  setThhighbackCmd = new G4UIcmdWithADoubleAndUnit("/plots/setThhighback",this);
  setThhighbackCmd->SetGuidance("set upper limit for backscattering Theta plot ");
  setThhighbackCmd->SetParameterName("Thhighback",false);
    
  RndmDir = new G4UIdirectory("/rndm/");
  RndmDir->SetGuidance("Rndm status control.");
  
  RndmSaveCmd = new G4UIcmdWithAnInteger("/rndm/save",this);
  RndmSaveCmd->SetGuidance("set frequency to save rndm status on external files.");
  RndmSaveCmd->SetGuidance("freq = 0 not saved");
  RndmSaveCmd->SetGuidance("freq > 0 saved on: beginOfRun.rndm");
  RndmSaveCmd->SetGuidance("freq = 1 saved on:   endOfRun.rndm");
  RndmSaveCmd->SetGuidance("freq = 2 saved on: endOfEvent.rndm");    
  RndmSaveCmd->SetParameterName("frequency",false);
  RndmSaveCmd->SetRange("frequency>=0 && frequency<=2");
  RndmSaveCmd->AvailableForStates(G4State_PreInit,G4State_Idle); 
         
  RndmReadCmd = new G4UIcmdWithAString("/rndm/read",this);
  RndmReadCmd->SetGuidance("get rndm status from an external file.");
  RndmReadCmd->SetParameterName("fileName",true);
  RndmReadCmd->SetDefaultValue ("beginOfRun.rndm");
  RndmReadCmd->AvailableForStates(G4State_PreInit,G4State_Idle);  
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

Em10RunMessenger::~Em10RunMessenger()
{
  delete sethistNameCmd;

  delete setnbinStepCmd;
  delete setSteplowCmd;
  delete setStephighCmd;

  delete setnbinEnCmd;
  delete setEnlowCmd;
  delete setEnhighCmd;

  delete setnbinGammaCmd;
  delete setElowGammaCmd;
  delete setEhighGammaCmd;

  delete setnbinTtCmd;
  delete setTtlowCmd;
  delete setTthighCmd;

  delete setnbinTbCmd;
  delete setTblowCmd;
  delete setTbhighCmd;

  delete setnbinTsecCmd;
  delete setTseclowCmd;
  delete setTsechighCmd;

  delete setnbinRCmd;
  delete setRlowCmd;
  delete setRhighCmd;

  delete setnbinzvertexCmd;
  delete setzlowCmd;
  delete setzhighCmd;

  delete setnbinThCmd;
  delete setThlowCmd;
  delete setThhighCmd;

  delete setnbinThbackCmd;
  delete setThlowbackCmd;
  delete setThhighbackCmd;

  delete plotDir;
  
  delete RndmSaveCmd; delete RndmReadCmd; delete RndmDir;  
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void Em10RunMessenger::SetNewValue(G4UIcommand* command,G4String newValues)
{
  if( command == sethistNameCmd)
    runAction
    ->SethistName(newValues) ;
    
  if( command == setnbinStepCmd)
    runAction
    ->SetnbinStep(setnbinStepCmd->GetNewIntValue(newValues));

  if( command == setSteplowCmd)
    runAction
    ->SetSteplow( setSteplowCmd->GetNewDoubleValue(newValues));

  if( command == setStephighCmd)
    runAction
    ->SetStephigh( setStephighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinEnCmd)
    runAction
    ->SetnbinEn(setnbinEnCmd->GetNewIntValue(newValues));

  if( command == setEnlowCmd)
    runAction
    ->SetEnlow( setEnlowCmd->GetNewDoubleValue(newValues));

  if( command == setEnhighCmd)
    runAction
    ->SetEnhigh( setEnhighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinGammaCmd)
    runAction
    ->SetnbinGamma(setnbinGammaCmd->GetNewIntValue(newValues));

  if( command == setElowGammaCmd)
    runAction
    ->SetElowGamma( setElowGammaCmd->GetNewDoubleValue(newValues));

  if( command == setEhighGammaCmd)
    runAction
    ->SetEhighGamma( setEhighGammaCmd->GetNewDoubleValue(newValues));

  if( command == setnbinTtCmd)
    runAction
    ->SetnbinTt(setnbinTtCmd->GetNewIntValue(newValues));

  if( command == setTtlowCmd)
    runAction
    ->SetTtlow( setTtlowCmd->GetNewDoubleValue(newValues));

  if( command == setTthighCmd)
    runAction
    ->SetTthigh( setTthighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinTbCmd)
    runAction
    ->SetnbinTb(setnbinTbCmd->GetNewIntValue(newValues));

  if( command == setTblowCmd)
    runAction
    ->SetTblow( setTblowCmd->GetNewDoubleValue(newValues));

  if( command == setTbhighCmd)
    runAction
    ->SetTbhigh( setTbhighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinTsecCmd)
    runAction
    ->SetnbinTsec(setnbinTsecCmd->GetNewIntValue(newValues));

  if( command == setTseclowCmd)
    runAction
    ->SetTseclow( setTseclowCmd->GetNewDoubleValue(newValues));

  if( command == setTsechighCmd)
    runAction
    ->SetTsechigh( setTsechighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinRCmd)
    runAction
    ->SetnbinR(setnbinRCmd->GetNewIntValue(newValues));

  if( command == setRlowCmd)
    runAction
    ->SetRlow( setRlowCmd->GetNewDoubleValue(newValues));

  if( command == setRhighCmd)
    runAction
    ->SetRhigh( setRhighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinzvertexCmd)
    runAction
    ->Setnbinzvertex(setnbinzvertexCmd->GetNewIntValue(newValues));

  if( command == setzlowCmd)
    runAction
    ->Setzlow( setzlowCmd->GetNewDoubleValue(newValues));

  if( command == setzhighCmd)
    runAction
    ->Setzhigh( setzhighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinThCmd)
    runAction
    ->SetnbinTh(setnbinThCmd->GetNewIntValue(newValues));

  if( command == setThlowCmd)
    runAction
    ->SetThlow( setThlowCmd->GetNewDoubleValue(newValues));

  if( command == setThhighCmd)
    runAction
    ->SetThhigh( setThhighCmd->GetNewDoubleValue(newValues));

  if( command == setnbinThbackCmd)
    runAction
    ->SetnbinThBack(setnbinThbackCmd->GetNewIntValue(newValues));

  if( command == setThlowbackCmd)
    runAction
    ->SetThlowBack( setThlowbackCmd->GetNewDoubleValue(newValues));

  if( command == setThhighbackCmd)
    runAction
    ->SetThhighBack( setThhighbackCmd->GetNewDoubleValue(newValues));
 
  if (command == RndmSaveCmd)
      runAction->SetRndmFreq(RndmSaveCmd->GetNewIntValue(newValues));
                 
  if (command == RndmReadCmd)
    { G4cout << "\n---> rndm status restored from file: " << newValues << G4endl;
      CLHEP::HepRandom::restoreEngineStatus(newValues);
      CLHEP::HepRandom::showEngineStatus();
    }   
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

   
