//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10CalorHit.cc
/// \brief Implementation of the Em10CalorHit class
//
//
// $Id: Em10CalorHit.cc 66241 2012-12-13 18:34:42Z gunter $
//
// 

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "Em10CalorHit.hh"

G4Allocator<Em10CalorHit> Em10CalorHitAllocator;

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

Em10CalorHit::Em10CalorHit()
{
   EdepAbs = 0.; TrackLengthAbs = 0.;
   EdepGap = 0.; TrackLengthGap = 0.;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

Em10CalorHit::~Em10CalorHit()
{;}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
/*
Em10CalorHit::Em10CalorHit(const Em10CalorHit& right)
{
  EdepAbs = right.EdepAbs; TrackLengthAbs = right.TrackLengthAbs;
  EdepGap = right.EdepGap; TrackLengthGap = right.TrackLengthGap;
}
*/
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

const Em10CalorHit& Em10CalorHit::operator=(const Em10CalorHit& right)
{
  EdepAbs = right.EdepAbs; TrackLengthAbs = right.TrackLengthAbs;
  EdepGap = right.EdepGap; TrackLengthGap = right.TrackLengthGap;
  return *this;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void Em10CalorHit::Print()
{;}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

