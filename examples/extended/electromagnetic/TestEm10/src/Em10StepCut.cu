//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
/// \file electromagnetic/TestEm10/src/Em10StepCut.cc
/// \brief Implementation of the Em10StepCut class
//
//
// $Id: Em10StepCut.cc 66241 2012-12-13 18:34:42Z gunter $
//

#include "Em10StepCut.hh"

#include "G4Step.hh"
#include "G4UserLimits.hh"
#include "G4VParticleChange.hh"
#include "G4EnergyLossTables.hh"

Em10StepCut::Em10StepCut(const G4String& aName)
  : G4VDiscreteProcess(aName),MaxChargedStep(DBL_MAX)
{
   if (verboseLevel>0) {
     G4cout << GetProcessName() << " is created "<< G4endl;
   }
}

Em10StepCut::~Em10StepCut()
{
}

Em10StepCut::Em10StepCut(Em10StepCut& right)
    :G4VDiscreteProcess(right)
{}

void Em10StepCut::SetMaxStep(G4double step)
{
  MaxChargedStep = step ;
}


