//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: exrdmDetectorMessenger.cc 68030 2013-03-13 13:51:27Z gcosmo $
//
/// \file radioactivedecay/rdecay02/src/exrdmDetectorMessenger.cc
/// \brief Implementation of the exrdmDetectorMessenger class
//
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

#include "exrdmDetectorMessenger.hh"

#include "exrdmDetectorConstruction.hh"
#include "G4UIdirectory.hh"
#include "G4UIcmdWithAString.hh"
#include "G4UIcmdWithADoubleAndUnit.hh"
#include "globals.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

exrdmDetectorMessenger::exrdmDetectorMessenger(exrdmDetectorConstruction* myDet)
:G4UImessenger(),
 fMyDetector(myDet),
 fExrdmDir(0),
 fDetDir(0),
 fTargMatCmd(0),
 fDetectMatCmd(0),
 fTargRadiusCmd(0),
 fDetectThicknessCmd(0),
 fTargLengthCmd(0),
 fDetectLengthCmd(0)
{ 
  fExrdmDir = new G4UIdirectory("/exrdm/");
  fExrdmDir->SetGuidance("UI commands specific to this example.");
  
  fDetDir = new G4UIdirectory("/exrdm/det/");
  fDetDir->SetGuidance("detector control.");
  
  fTargMatCmd = new G4UIcmdWithAString("/exrdm/det/setTargetMate",this);
  fTargMatCmd->SetGuidance("Select Material of the Target.");
  fTargMatCmd->SetParameterName("choice",false);
  fTargMatCmd->AvailableForStates(G4State_PreInit,G4State_Idle);
  
  fTargRadiusCmd = new G4UIcmdWithADoubleAndUnit("/exrdm/det/setTargetRadius", this);
  fTargRadiusCmd->SetGuidance("Set the Target Radius.");
  fTargRadiusCmd->SetUnitCategory("Length");
  fTargRadiusCmd->SetParameterName("choice",false);
  fTargRadiusCmd->AvailableForStates(G4State_PreInit);
  
  fTargLengthCmd = new G4UIcmdWithADoubleAndUnit("/exrdm/det/setTargetLength", this);
  fTargLengthCmd->SetGuidance("Set the Target Length.");
  fTargLengthCmd->SetUnitCategory("Length");
  fTargLengthCmd->SetParameterName("choice",false);
  fTargLengthCmd->AvailableForStates(G4State_PreInit);

  fDetectMatCmd = new G4UIcmdWithAString("/exrdm/det/setDetectorMate",this);
  fDetectMatCmd->SetGuidance("Select Material of the Detector.");
  fDetectMatCmd->SetParameterName("choice",false);
  fDetectMatCmd->AvailableForStates(G4State_PreInit,G4State_Idle);  

  fDetectThicknessCmd =
                 new G4UIcmdWithADoubleAndUnit("/exrdm/det/setDetectorThickness",this);
  fDetectThicknessCmd->SetGuidance("Set the Detector Thickness.");
  fDetectThicknessCmd->SetUnitCategory("Length");
  fDetectThicknessCmd->SetParameterName("choice",false);
  fDetectThicknessCmd->AvailableForStates(G4State_PreInit);

  fDetectLengthCmd =
                 new G4UIcmdWithADoubleAndUnit("/exrdm/det/setDetectorLength",this);
  fDetectLengthCmd->SetGuidance("Set the Detector Length.");
  fDetectLengthCmd->SetUnitCategory("Length");
  fDetectLengthCmd->SetParameterName("choice",false);
  fDetectLengthCmd->AvailableForStates(G4State_PreInit);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

exrdmDetectorMessenger::~exrdmDetectorMessenger()
{
  delete fTargMatCmd;
  delete fDetectMatCmd;
  delete fTargRadiusCmd;
  delete fDetectThicknessCmd;
  delete fTargLengthCmd;
  delete fDetectLengthCmd;
  delete fDetDir;
  delete fExrdmDir;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void exrdmDetectorMessenger::SetNewValue(G4UIcommand* command, G4String newValue)
{ 
  if( command == fTargMatCmd )fMyDetector->SetTargetMaterial(newValue);
  else if ( command == fTargLengthCmd ) 
    fMyDetector->SetTargetLength(fTargLengthCmd->GetNewDoubleValue(newValue));
  else if ( command == fTargRadiusCmd ) 
    fMyDetector->SetTargetRadius(fTargLengthCmd->GetNewDoubleValue(newValue));
  else if( command == fDetectMatCmd )
    fMyDetector->SetDetectorMaterial(newValue);
  else if (command == fDetectLengthCmd ) 
    fMyDetector->SetDetectorLength(
                     fDetectLengthCmd->GetNewDoubleValue(newValue));
  else if (command == fDetectThicknessCmd ) 
    fMyDetector->SetDetectorThickness(
                              fDetectThicknessCmd->GetNewDoubleValue(newValue));
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
