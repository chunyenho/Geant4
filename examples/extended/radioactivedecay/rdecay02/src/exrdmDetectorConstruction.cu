#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: exrdmDetectorConstruction.cc 84601 2014-10-17 07:44:52Z gcosmo $
//
/// \file radioactivedecay/rdecay02/src/exrdmDetectorConstruction.cc
/// \brief Implementation of the exrdmDetectorConstruction class
//
#include "exrdmDetectorConstruction.hh"
#include "exrdmDetectorMessenger.hh"
//#include "exrdmDetectorSD.hh"
#include "G4UImanager.hh"
#include "G4Tubs.hh"
#include "G4LogicalVolume.hh"
#include "G4PVPlacement.hh"
//#include "G4SDManager.hh"
#include "G4Region.hh"
#include "G4RegionStore.hh"

#include "exrdmMaterial.hh"

#include "G4VisAttributes.hh"
#include "G4Colour.hh"
#include "G4PhysicalConstants.hh"
#include "G4SystemOfUnits.hh"

#include "G4ios.hh"
#include <sstream>

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
 
exrdmDetectorConstruction::exrdmDetectorConstruction()
:G4VUserDetectorConstruction(),
 fSolidWorld(0),  fLogicWorld(0),  fPhysiWorld(0),
 fSolidTarget(0), fLogicTarget(0), fPhysiTarget(0), 
 fSolidDetector(0),fLogicDetector(0),fPhysiDetector(0),
 fDetectorMessenger(0),
 fMaterialsManager(0),
 fDefaultMater(0),fTargetMater(0),fDetectorMater(0),
 fTargetLength (1.*cm), fTargetRadius(0.5*cm),
 fDetectorLength(5.0 * cm), fDetectorThickness(2.0 * cm),
 fWorldLength (std::max(fTargetLength,fDetectorLength)),
 fWorldRadius (fTargetRadius + fDetectorThickness),
 fTargetRegion(0), fDetectorRegion(0)
{
  fDetectorMessenger = new exrdmDetectorMessenger(this);
  DefineMaterials();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
 
exrdmDetectorConstruction::~exrdmDetectorConstruction()
{
  delete fDetectorMessenger;             
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
 
void exrdmDetectorConstruction::DefineMaterials()
{
//--------- Material definition ---------

  fMaterialsManager = new exrdmMaterial();
  // Lead
  fMaterialsManager->AddMaterial("Lead","Pb",11.3*g/cm3,"");
  //Germanium detector
  fMaterialsManager->AddMaterial("Germanium","Ge",5.323*g/cm3,""); 
  //CsI
  fMaterialsManager->AddMaterial("CsI","Cs-I",4.51*g/cm3,"");

  // G4cout << G4endl << "The materials defined are : " << G4endl << G4endl;
  // G4cout << *(G4Material::GetMaterialTable()) << G4endl;
    
  fDefaultMater = fMaterialsManager->GetMaterial("Air");
  fTargetMater  = fMaterialsManager->GetMaterial("CsI");
  fDetectorMater = fMaterialsManager->GetMaterial("Germanium");
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
 
G4VPhysicalVolume* exrdmDetectorConstruction::Construct()
{
//--------- Definitions of Solids, Logical Volumes, Physical Volumes ---------
  //--------- Sizes of the principal geometrical components (solids)  ---------

  fWorldLength = std::max(fTargetLength,fDetectorLength);
  fWorldRadius = fTargetRadius + fDetectorThickness;
   
  //------------------------------ 
  // World
  //------------------------------ 

 fSolidWorld= new G4Tubs("world",0.,fWorldRadius,fWorldLength/2.,0.,twopi);
 fLogicWorld= new G4LogicalVolume( fSolidWorld, fDefaultMater, "World", 0, 0, 0);
  
  //  Must place the World Physical volume unrotated at (0,0,0).
  // 
  fPhysiWorld = new G4PVPlacement(0,              // no rotation
                                 G4ThreeVector(), // at (0,0,0)
                                 fLogicWorld,     // its logical volume
                                 "World",         // its name
                                 0,               // its mother  volume
                                 false,           // no boolean operations
                                 0);              // no field specific to volume
                                 
  //------------------------------ 
  // Target
  //------------------------------
  
  G4ThreeVector positionTarget = G4ThreeVector(0,0,0);
   
  fSolidTarget = new G4Tubs("target",0.,fTargetRadius,fTargetLength/2.,0.,twopi);
  fLogicTarget = new G4LogicalVolume(fSolidTarget,fTargetMater,"Target",0,0,0);
  fPhysiTarget = new G4PVPlacement(0,              // no rotation
                                  positionTarget,  // at (x,y,z)
                                  fLogicTarget,    // its logical volume
                                  "Target",        // its name
                                  fLogicWorld,     // its mother  volume
                                  false,           // no boolean operations
                                  0);              // no particular field 

  //  G4cout << "Target is a cylinder with rdius of " << targetradius/cm << " cm of " 
  //       << fTargetMater->GetName() << G4endl;

  //------------------------------ 
  // Detector
  //------------------------------
  
  G4ThreeVector positionDetector = G4ThreeVector(0,0,0);
  
  fSolidDetector = new G4Tubs("detector",fTargetRadius,fWorldRadius,
                                                          fDetectorLength/2.,0.,twopi);
  fLogicDetector = new G4LogicalVolume(fSolidDetector ,fDetectorMater,
                                                                      "Detector",0,0,0);
  fPhysiDetector = new G4PVPlacement(0,             // no rotation
                                  positionDetector, // at (x,y,z)
                                  fLogicDetector,   // its logical volume
                                  "Detector",       // its name
                                  fLogicWorld,      // its mother  volume
                                  false,            // no boolean operations
                                  0);               // no particular field 

  //------------------------------------------------ 
  // Sensitive detectors
  //------------------------------------------------ 

  //  G4SDManager* SDman = G4SDManager::GetSDMpointer();

  // G4String detectortargetSDname = "exrdm/DetectorTargetSD";
  // exrdmDetectorSD* aDetectorSD = new exrdmDetectorSD( detectorTargetSDname );
  // SDman->AddNewDetector( aDetectorSD );
  //fLogicTarget->SetSensitiveDetector( aDetectorSD );
  // fLogicDetector->SetSensitiveDetector( aDetectorSD );
  //
  //-------------------------------------------------
  // regions
  //
  //  if(fTargetRegion) delete fTargetRegion;
  // if(fDetectorRegion) delete fDetectorRegion;
  fTargetRegion = new G4Region("Target");
  fDetectorRegion   = new G4Region("Detector");
  fTargetRegion->AddRootLogicalVolume(fLogicTarget);
  fDetectorRegion->AddRootLogicalVolume(fLogicDetector);

  //--------- Visualization attributes -------------------------------
  fLogicWorld->SetVisAttributes(G4VisAttributes::Invisible);
  G4VisAttributes* TargetVisAtt= new G4VisAttributes(G4Colour(1.0,1.0,1.0));
  fLogicTarget ->SetVisAttributes(TargetVisAtt);
  G4VisAttributes* DetectorVisAtt= new G4VisAttributes(G4Colour(1.0,1.0,.0));
  fLogicDetector->SetVisAttributes(DetectorVisAtt);


  //------------ set the incident position ------

 // get the pointer to the User Interface manager 
    
  G4UImanager* UI = G4UImanager::GetUIpointer();  
  //      UI->ApplyCommand("/run/verbose 1");
  //      UI->ApplyCommand("/event/verbose 2");
  //      UI->ApplyCommand("/tracking/verbose 1");  

  G4double zpos = -fWorldLength/2.;
  G4String command = "/gps/pos/centre ";
  std::ostringstream os;
  os << zpos ; 
  G4String xs = os.str();
  UI->ApplyCommand(command+"0. 0. "+xs+" mm");
  UI->ApplyCommand("/gps/pos/type Point");
  command = "/gps/position ";
  //  UI->ApplyCommand(command+"0. 0. "+xs+" mm");
  UI->ApplyCommand("/gps/particle proton");
  UI->ApplyCommand("/gps/direction 0 0 1");
  UI->ApplyCommand("/gps/energy 100 MeV");
  //       
  
  return fPhysiWorld;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......
 
void exrdmDetectorConstruction::SetTargetMaterial(G4String materialName)
{
  // search the material by its name 
  G4Material* pttoMaterial = G4Material::GetMaterial(materialName);  
  if (pttoMaterial)
     {fTargetMater = pttoMaterial;
      if (fLogicTarget) fLogicTarget->SetMaterial(pttoMaterial); 
      G4cout << "\n----> The target has been changed to " << fTargetLength/cm
                     << " cm of "
             << materialName << G4endl;
     }             
}
 
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo......

void exrdmDetectorConstruction::SetDetectorMaterial(G4String materialName)
{
  // search the material by its name 
  G4Material* pttoMaterial = G4Material::GetMaterial(materialName);  
  if (pttoMaterial)
     {fDetectorMater = pttoMaterial;
      if (fLogicDetector) fLogicDetector->SetMaterial(pttoMaterial); 
      G4cout << "\n----> The Deetctor has been changed to" << fDetectorLength/cm
                     << " cm of "
             << materialName << G4endl;
     }             
}
//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo.....
