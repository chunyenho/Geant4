//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
// $Id: exrdmHisto.cc 78592 2014-01-08 10:30:37Z gcosmo $
//
/// \file radioactivedecay/rdecay02/src/exrdmHisto.cc
/// \brief Implementation of the exrdmHisto class
//
#ifdef G4ANALYSIS_USE
#include <AIDA/AIDA.h>
#endif
//
#ifdef G4ANALYSIS_USE_ROOT
#include "TROOT.h"
#include "TApplication.h"
#include "TGClient.h"
#include "TCanvas.h"
#include "TSystem.h"
#include "TTree.h"
#include "TBranch.h"
#include "TFile.h"
#include "TH1D.h"
#include "TNtuple.h"
#endif

#include "exrdmHisto.hh"
#include "exrdmHistoMessenger.hh"
#include "G4ParticleTable.hh"

#include "G4Tokenizer.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
exrdmHisto::exrdmHisto()
:fHistName("exrdm"), fHistType("root"),
 fNHisto(0), fNTuple(0), fVerbose(0),
 fDefaultAct(1)
{
#ifdef G4ANALYSIS_USE
  fAida = 0;
  fTree = 0;
#endif

#ifdef G4ANALYSIS_USE_ROOT
  fROOThisto.clear();
  fROOTntup.clear();
  fRarray.clear();
  fRcol.clear();
#endif

  fActive.clear();
  fBins.clear();
  fXmin.clear();
  fXmax.clear();
  fUnit.clear();
  fIds.clear();
  fTitles.clear();
  fTupleName.clear();
  fTupleId.clear();
  fTupleList.clear();
  fTupleListROOT.clear();

  fMessenger = new exrdmHistoMessenger(this);
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

exrdmHisto::~exrdmHisto()
{
#ifdef G4ANALYSIS_USE
  fHisto.clear();
  fNtup.clear();
#endif
#ifdef G4ANALYSIS_USE_ROOT
  //FIXME : G.Barrand : the below is crashy.
  //        In principle the TH are deleted
  //        when doing the TFile::Close !
  //         In fact the fHfileROOT should 
  //        be deleted in Save(). And I am pretty
  //        sure that the TApplication is not needed.
  //
  // removed by F.Lei
  //  for(G4int i=0; i<fNHisto; i++) {
  //   if(fROOThisto[i]) delete fROOThisto[i];
  // }
#endif
  delete fMessenger;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::Book()
{
#ifdef G4ANALYSIS_USE
  G4cout << "### exrdmHisto books " << fNHisto << " histograms " << G4endl; 
  // Creating the analysis factory
  fAida = AIDA_createAnalysisFactory();
  if(!fAida) {
    G4cout << "ERROR: can't get AIDA." << G4endl; 
    return;
  }
  // Creating the fTree factory
 {AIDA::ITreeFactory* tf = fAida->createTreeFactory(); 
  // Creating a fTree mapped to a new fAida file.
  G4String fileName = fHistName + "." + fHistType;
  if (fHistType == "root") fileName = fHistName + "_aida." + fHistType;
  fTree = tf->create(fileName,fHistType,false,true,"");
  delete tf;
  if(!fTree) { 
    G4cout << "ERROR: Tree store " << fHistName  << " is not created!" << G4endl; 
    return;
  }
  G4cout << "Tree store  : " << fTree->storeName() << G4endl;}
  // Creating a histogram factory, whose histograms will be handled by the fTree
 {AIDA::IHistogramFactory* hf = fAida->createHistogramFactory(*fTree);
  // Creating an 1-dimensional histograms in the root directory of the fTree
  for(G4int i=0; i<fNHisto; i++) {
    if(fActive[i]) {
      if(fVerbose>1)
        G4cout<<"Book: histogram "<< i << " id= " << fIds[i] <<G4endl;
      G4String tit = fIds[i];
      if(fHistType == "root") tit = "h" + fIds[i];
      fHisto[i] = hf->createHistogram1D(tit, fTitles[i], fBins[i], fXmin[i],
                                        fXmax[i]);
    }
  }
  delete hf;
  G4cout << "AIDA histograms are booked" << G4endl;}

  // Creating a tuple factory, whose tuples will be handled by the fTree  
 {AIDA::ITupleFactory* tpf =  fAida->createTupleFactory( *fTree );
  G4cout << "AIDA will Book " << fNTuple << " ntuples" << G4endl;
  for(G4int i=0; i<fNTuple; i++) {
    if(fTupleList[i] != "") {
      G4cout << "Creating Ntuple: " << fTupleName[i] <<":" <<fTupleList[i]
             << G4endl;
      fNtup[i] = tpf->create(fTupleId[i], fTupleName[i], fTupleList[i],"");
    }
  }
  delete tpf;
  G4cout << "AIDA ntuples are booked" << G4endl;}
#endif

#ifdef G4ANALYSIS_USE_ROOT
//  new TApplication("App", ((int *)0), ((char **)0));
  G4String fileNameROOT = fHistName + G4String(".root");
  fHfileROOT = new TFile(fileNameROOT.c_str() ,"RECREATE","ROOT file for exRDM");
  G4cout << "Root file: " << fileNameROOT << G4endl;
  // Creating an 1-dimensional histograms in the root directory of the fTree
  for(G4int i=0; i<fNHisto; i++) {
    if(fActive[i]) {
      G4String id = G4String("h")+fIds[i];
      fROOThisto[i] = new TH1D(id, fTitles[i], fBins[i], fXmin[i], fXmax[i]);
      G4cout << "ROOT Histo " << fIds[i] << " " << fTitles[i] << " booked "
             << G4endl;
    }
  }
  // Now the ntuples  
  for(G4int i=0; i<fNTuple; i++) {
    if(fTupleListROOT[i] != "") {
      G4String id = G4String("t")+fTupleId[i];
      G4cout << "Creating Ntuple "<<fTupleId[i] << " in ROOT file: " 
             << fTupleName[i] << G4endl;
      fROOTntup[i] = new TNtuple(id, fTupleName[i], fTupleListROOT[i]);
      G4cout << "ROOT Ntuple " << id << " " << fTupleName[i] <<" "
             << fTupleListROOT[i]<< " booked " << G4endl;
    }
  }
#endif

} 

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::Save()
{
#ifdef G4ANALYSIS_USE
  // Write histogram file
  fTree->commit();
  G4cout << "Closing the AIDA fTree..." << G4endl;
  fTree->close();
  G4cout << "Histograms and Ntuples are saved" << G4endl;
  delete fTree;
  fTree = 0;
  delete fAida;
  fAida = 0;
  {for(G4int i=0; i<fNHisto; i++) fHisto[i] = 0;}
  {for(G4int i=0; i<fNTuple; i++) fNtup[i] = 0;}
#endif
#ifdef G4ANALYSIS_USE_ROOT
  G4cout << "ROOT: files writing..." << G4endl;
  fHfileROOT->Write();
  G4cout << "ROOT: files closing..." << G4endl;
  fHfileROOT->Close();
  //
  // F.Lei added following Guy's suggestion!
  delete fHfileROOT;

#endif
}


//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::Add1D(const G4String& id, const G4String& name, G4int nb, 
                       G4double x1, G4double x2, G4double u)
{
  if(fVerbose > 0) {
    G4cout << "New histogram will be booked: #" << id << "  <" << name 
           << "  " << nb << "  " << x1 << "  " << x2 << "  " << u 
           << G4endl;
  }
  fNHisto++;
  x1 /= u;
  x2 /= u;
  fActive.push_back(fDefaultAct);
  fBins.push_back(nb);
  fXmin.push_back(x1);
  fXmax.push_back(x2);
  fUnit.push_back(u);
  fIds.push_back(id);
  fTitles.push_back(name);
#ifdef G4ANALYSIS_USE
  fHisto.push_back(0);
#endif
#ifdef G4ANALYSIS_USE_ROOT
  fROOThisto.push_back(0);
#endif
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::SetHisto1D(G4int i, G4int nb, G4double x1, G4double x2, G4double u)
{
  if(i>=0 && i<fNHisto) {
    if(fVerbose > 0) {
      G4cout << "Update histogram: #" << i  
             << "  " << nb << "  " << x1 << "  " << x2 << "  " << u 
             << G4endl;
    }
    fBins[i] = nb;
    fXmin[i] = x1;
    fXmax[i] = x2;
    fUnit[i] = u;
  } else {
    G4cout << "exrdmHisto::setexrdmHisto1D: WARNING! wrong histogram index "
           << i << G4endl;
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::FillHisto(G4int i, G4double x, G4double w)
{
  if(fVerbose > 1) {
    G4cout << "fill histogram: #" << i << " at x= " << x 
           << "  weight= " << w
           << G4endl;   
  }
#ifdef G4ANALYSIS_USE
  if(i>=0 && i<fNHisto) {
    fHisto[i]->fill(x/fUnit[i], w);
  } else {
    G4cout << "exrdmHisto::fill: WARNING! wrong AIDA histogram index "
           << i << G4endl;
  }
#endif
#ifdef G4ANALYSIS_USE_ROOT  
  if(i>=0 && i<fNHisto) {
    fROOThisto[i]->Fill(x/fUnit[i],w);
  } else {
    G4cout << "exrdmHisto::fill: WARNING! wrong ROOT histogram index "
           << i << G4endl;
  }
#endif
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::ScaleHisto(G4int i, G4double x)
{
  if(fVerbose > 0) {
    G4cout << "Scale histogram: #" << i << " by factor " << x << G4endl;   
  }
#ifdef G4ANALYSIS_USE
  if(i>=0 && i<fNHisto) {
    fHisto[i]->scale(x);
    G4cout << "exrdmHisto::scale: WARNING! wrong AIDA histogram index "
           << i << G4endl;
  }
#endif
#ifdef G4ANALYSIS_USE_ROOT  
  if(i>=0 && i<fNHisto) {
    fROOThisto[i]->Scale(x);
  } else {
    G4cout << "exrdmHisto::scale: WARNING! wrong ROOT histogram index "
           << i << G4endl;
  }
#endif
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#ifdef G4ANALYSIS_USE
void exrdmHisto::AddTuple(const G4String& w1, const G4String& w2,
                          const G4String& w3 )
#else
#ifdef G4ANALYSIS_USE_ROOT
void exrdmHisto::AddTuple(const G4String& w1, const G4String& w2,
                          const G4String& w3 )
#else
void exrdmHisto::AddTuple(const G4String& w1, const G4String& w2,
                          const G4String& )
#endif
#endif

{
  //G4cout << w1 << " " << w2 << " " << w3 << G4endl;
  fNTuple++;
  fTupleId.push_back(w1);
  fTupleName.push_back(w2) ;
#ifdef G4ANALYSIS_USE
  fTupleList.push_back(w3);
  fNtup.push_back(0);
#endif

#ifdef G4ANALYSIS_USE_ROOT
  std::vector<float> ar;
  ar.clear();
  for (size_t i = 0; i < 20; i++) ar.push_back(0.);
  fRarray.push_back(ar);
  // convert AIDA header to ROOT header for ntuple
  G4Tokenizer next(w3);
  G4String token = next();
  G4String ROOTList1 = "" ;
  G4int col = 0;
  while ( token != "") {
   token = next();
   if (token == ",") token = next();
   if (token.contains(",")) token.remove(token.size()-1);
   ROOTList1 = ROOTList1 + token + G4String(":");
   col++;
  }
  G4String ROOTList = ROOTList1.substr(0,ROOTList1.length()-2);
//  G4cout << ROOTList << G4endl;
  fTupleListROOT.push_back(ROOTList);
  fROOTntup.push_back(0);
  fRcol.push_back(col-1);
#endif
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::FillTuple(G4int i, const G4String& parname, G4double x)
{
  if(fVerbose > 1) 
    G4cout << "fill tuple # " << i 
           <<" with  parameter <" << parname << "> = " << x << G4endl; 
#ifdef G4ANALYSIS_USE
  if(fNtup[i]) fNtup[i]->fill(fNtup[i]->findColumn(parname), x);
#endif
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::FillTuple(G4int i, G4int col, G4double x)
{
  if(fVerbose > 1) {
    G4cout << "fill tuple # " << i 
           <<" in column < " << col << "> = " << x << G4endl; 
  }
#ifdef G4ANALYSIS_USE
  if(fNtup[i]) fNtup[i]->fill(col,double(x));
#endif

#ifdef G4ANALYSIS_USE_ROOT  
  if(fROOTntup[i]) (fRarray[i])[col] = float(x);
#endif

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::FillTuple(G4int i, const G4String& parname, G4String& x)
{
  if(fVerbose > 1) {
    G4cout << "fill tuple # " << i 
           <<" with  parameter <" << parname << "> = " << x << G4endl; 
  }
#ifdef G4ANALYSIS_USE
  if(fNtup[i]) fNtup[i]->fill(fNtup[i]->findColumn(parname), x);
#endif

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::AddRow(G4int i)
{
  if(fVerbose > 1) G4cout << "Added a raw #" << i << " to tuple" << G4endl; 
#ifdef G4ANALYSIS_USE
  if(fNtup[i]) fNtup[i]->addRow();
#endif

#ifdef G4ANALYSIS_USE_ROOT
  float *ar=new float[fRcol[i]];
  for (G4int j=0; j < fRcol[i]; j++) {
//      G4cout << i << " " << fRarray[i][j] << G4endl;
      ar[j] = fRarray[i][j];       
  }  
  if(fROOTntup[i]) fROOTntup[i]->Fill(ar);
  delete ar;
#endif

} 

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::SetFileName(const G4String& nam) 
{
  fHistName = nam;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

const G4String& exrdmHisto::GetFileName() const
{
  return fHistName;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void exrdmHisto::SetFileType(const G4String& nam) 
{
  fHistType = nam;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

const G4String& exrdmHisto::FileType() const
{
  return fHistType;
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

