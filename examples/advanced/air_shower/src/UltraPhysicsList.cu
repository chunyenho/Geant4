//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// --------------------------------------------------------------
//                 GEANT 4 - ULTRA experiment example
// --------------------------------------------------------------
//
// Code developed by:
// B. Tome, M.C. Espirito-Santo, A. Trindade, P. Rodrigues 
//
//    ****************************************************
//    *      UltraPhysicsList.cc
//    ****************************************************
//
//    Ultra Physics List class; Standard and Low Energy EM processes are defined for
//    the relevant particles. Optical processes are declared.
//
#include "G4ios.hh"
//#include "iomanip.h"
#include "globals.hh"

#include "UltraPhysicsList.hh"

#include "G4ParticleDefinition.hh"
#include "G4ParticleTypes.hh"
#include "G4ParticleWithCuts.hh"
#include "G4ParticleTable.hh"
#include "G4Material.hh"
#include "G4MaterialTable.hh"
#include "G4ProcessManager.hh"
#include "G4ProcessVector.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

UltraPhysicsList::UltraPhysicsList() :  G4VUserPhysicsList() {;}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

UltraPhysicsList::~UltraPhysicsList() {;}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructParticle()
{
  // In this method, static member functions should be called
  // for all particles which you want to use.
  // This ensures that objects of these particle types will be
  // created in the program.

  ConstructBosons();
  ConstructLeptons();
  ConstructMesons();
  ConstructBaryons();

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructBosons()
{
  // pseudo-particles
  G4Geantino::GeantinoDefinition();
  G4ChargedGeantino::ChargedGeantinoDefinition();

  // gamma
  G4Gamma::GammaDefinition();

  // optical photon
  G4OpticalPhoton::OpticalPhotonDefinition();

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructLeptons()
{
  // leptons
  G4Electron::ElectronDefinition();
  G4Positron::PositronDefinition();
  G4NeutrinoE::NeutrinoEDefinition();
  G4AntiNeutrinoE::AntiNeutrinoEDefinition();
  G4MuonPlus::MuonPlusDefinition();
  G4MuonMinus::MuonMinusDefinition();
  G4NeutrinoMu::NeutrinoMuDefinition();
  G4AntiNeutrinoMu::AntiNeutrinoMuDefinition();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructMesons()
{
 //  mesons
  G4PionPlus::PionPlusDefinition();
  G4PionMinus::PionMinusDefinition();
  G4PionZero::PionZeroDefinition();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructBaryons()
{
//  barions
  G4Proton::ProtonDefinition();
  G4AntiProton::AntiProtonDefinition();
  G4Neutron::NeutronDefinition();
  G4AntiNeutron::AntiNeutronDefinition();
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructProcess()
{
  AddTransportation();
  ConstructGeneral();
  ConstructEM();
  ConstructOp();

}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4Decay.hh"

void UltraPhysicsList::ConstructGeneral()
{
  G4Decay* theDecayProcess = new G4Decay();
  theParticleIterator->reset();
  while( (*theParticleIterator)() ){
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    if (theDecayProcess->IsApplicable(*particle)) {
      pmanager->AddDiscreteProcess(theDecayProcess);
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4ComptonScattering.hh"
#include "G4GammaConversion.hh"
#include "G4PhotoElectricEffect.hh"

#include "G4eMultipleScattering.hh"
#include "G4MuMultipleScattering.hh"
#include "G4hMultipleScattering.hh"

#include "G4eIonisation.hh"
#include "G4eBremsstrahlung.hh"
#include "G4eplusAnnihilation.hh"

#include "G4MuIonisation.hh"
#include "G4MuBremsstrahlung.hh"
#include "G4MuPairProduction.hh"

#include "G4hIonisation.hh"

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::ConstructEM()
{
  theParticleIterator->reset();
  while( (*theParticleIterator)() ){
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    G4String particleName = particle->GetParticleName();

    if (particleName == "gamma") {
    // gamma
      // Construct processes for gamma
    pmanager->AddDiscreteProcess(new G4GammaConversion());
    pmanager->AddDiscreteProcess(new G4ComptonScattering());
    pmanager->AddDiscreteProcess(new G4PhotoElectricEffect());

    } else if (particleName == "e-") {
    //electron
      // Construct processes for electron
      pmanager->AddProcess(new G4eMultipleScattering(),-1,1,1);
      pmanager->AddProcess(new G4eIonisation(),-1,2,2);
      pmanager->AddProcess(new G4eBremsstrahlung(),-1,-1,3);

    } else if (particleName == "e+") {
    //positron
      // Construct processes for positron
      pmanager->AddProcess(new G4eMultipleScattering(),-1,1,1);
      pmanager->AddProcess(new G4eIonisation(),-1,2,2);
      pmanager->AddProcess(new G4eBremsstrahlung(),-1,-1,3);
      pmanager->AddProcess(new G4eplusAnnihilation(),0,-1,4);

    } else if( particleName == "mu+" ||
               particleName == "mu-"    ) {
    //muon
     // Construct processes for muon
     pmanager->AddProcess(new G4MuMultipleScattering(),-1,1,1);
     pmanager->AddProcess(new G4MuIonisation(),-1,2,2);
     pmanager->AddProcess(new G4MuBremsstrahlung(),-1,-1,3);
     pmanager->AddProcess(new G4MuPairProduction(),-1,-1,4);

    } else {
      if ((particle->GetPDGCharge() != 0.0) &&
          (particle->GetParticleName() != "chargedgeantino")) {
     // all others charged particles except geantino
       pmanager->AddProcess(new G4hMultipleScattering(),-1,1,1);
       pmanager->AddProcess(new G4hIonisation(),-1,2,2);
     }
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

#include "G4Cerenkov.hh"
#include "G4Scintillation.hh"
#include "G4OpAbsorption.hh"
#include "G4OpRayleigh.hh"
#include "G4OpBoundaryProcess.hh"

void UltraPhysicsList::ConstructOp()
{
  // this Cerenkov Process
  G4Cerenkov*   theCerenkovProcess = new G4Cerenkov("Cerenkov");
  // this absorption process inside optical media
  G4OpAbsorption* theAbsorptionProcess = new G4OpAbsorption();
  // Rayleigh scattering for optical photons (aerogel radiators)
  G4OpRayleigh*   theRayleighScatteringProcess = new G4OpRayleigh();
  // Boundary process definition Class
  G4OpBoundaryProcess* theBoundaryProcess = new G4OpBoundaryProcess();

  // Chose level 0 (no verbose)
  theCerenkovProcess           -> SetVerboseLevel(0);
  theAbsorptionProcess         -> SetVerboseLevel(0);
  theRayleighScatteringProcess -> SetVerboseLevel(0);
  theBoundaryProcess           -> SetVerboseLevel(0);

 
// Chose MaxNumPhotons that can be generated. Lets ignore this for now
//   G4int MaxNumPhotons = 300;
//   theCerenkovProcess->SetMaxNumPhotonsPerStep(MaxNumPhotons);
  theCerenkovProcess->SetTrackSecondariesFirst(true);

  theParticleIterator->reset();
  while( (*theParticleIterator)() ){
    G4ParticleDefinition* particle = theParticleIterator->value();
    G4ProcessManager* pmanager = particle->GetProcessManager();
    G4String particleName = particle->GetParticleName();

    if (theCerenkovProcess->IsApplicable(*particle)) {
      pmanager->AddProcess(theCerenkovProcess);
      pmanager->SetProcessOrdering(theCerenkovProcess,idxPostStep);
    }


    if (particleName == "opticalphoton") {
      G4cout << ">>>>>>>>>>>>>> AddDiscreteProcess to OpticalPhoton " << G4endl;
      pmanager->AddDiscreteProcess(theAbsorptionProcess);
      pmanager->AddDiscreteProcess(theRayleighScatteringProcess);
      pmanager->AddDiscreteProcess(theBoundaryProcess);
    }
  }
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....

void UltraPhysicsList::SetCuts()
{
  if (verboseLevel >1){
    G4cout << "UltraPhysicsList::SetCuts:";
  }  
  //  " G4VUserPhysicsList::SetCutsWithDefault" method sets 
  //   the default cut value for all particle types 
  SetCutsWithDefault();   
}

//....oooOO0OOooo........oooOO0OOooo........oooOO0OOooo........oooOO0OOooo....
